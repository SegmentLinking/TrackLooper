#include "hip/hip_runtime.h"
# include "PixelTriplet.cuh"
#include "allocate.h"
#include "Kernels.cuh"

SDL::pixelTriplets::pixelTriplets()
{
    pixelSegmentIndices = nullptr;
    tripletIndices = nullptr;
    nPixelTriplets = nullptr;
    totOccupancyPixelTriplets = nullptr;
    pixelRadius = nullptr;
    tripletRadius = nullptr;
    pt = nullptr;
    isDup = nullptr;
    partOfPT5 = nullptr;
    centerX = nullptr;
    centerY = nullptr;
    hitIndices = nullptr;
    lowerModuleIndices = nullptr;
    logicalLayers = nullptr;
    rzChiSquared = nullptr;
    rPhiChiSquared = nullptr;
    rPhiChiSquaredInwards = nullptr;
}

void SDL::pixelTriplets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,pixelSegmentIndices);
    cms::cuda::free_device(dev,tripletIndices);
    cms::cuda::free_device(dev,nPixelTriplets);
    cms::cuda::free_device(dev,totOccupancyPixelTriplets);
    cms::cuda::free_device(dev,pixelRadius);
    cms::cuda::free_device(dev,tripletRadius);
    cms::cuda::free_device(dev,pt);
    cms::cuda::free_device(dev,isDup);
    cms::cuda::free_device(dev,partOfPT5);
    cms::cuda::free_device(dev, centerX);
    cms::cuda::free_device(dev, centerY);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, rPhiChiSquared);
    cms::cuda::free_device(dev, rPhiChiSquaredInwards);
    cms::cuda::free_device(dev, rzChiSquared);
}
void SDL::pixelTriplets::freeMemory(hipStream_t stream)
{
    hipFree(pixelSegmentIndices);
    hipFree(tripletIndices);
    hipFree(nPixelTriplets);
    hipFree(totOccupancyPixelTriplets);
    hipFree(pixelRadius);
    hipFree(tripletRadius);
    hipFree(pt);
    hipFree(isDup);
    hipFree(partOfPT5);
    hipFree(centerX);
    hipFree(centerY);
    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(lowerModuleIndices);
    hipFree(rPhiChiSquared);
    hipFree(rPhiChiSquaredInwards);
    hipFree(rzChiSquared);
}

SDL::pixelTriplets::~pixelTriplets()
{
}

void SDL::pixelTriplets::resetMemory(unsigned int maxPixelTriplets,hipStream_t stream)
{
    hipMemsetAsync(pixelSegmentIndices,0, maxPixelTriplets * sizeof(unsigned int),stream);
    hipMemsetAsync(tripletIndices, 0,maxPixelTriplets * sizeof(unsigned int),stream);
    hipMemsetAsync(nPixelTriplets, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(totOccupancyPixelTriplets, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(pixelRadius, 0,maxPixelTriplets * sizeof(FPX),stream);
    hipMemsetAsync(tripletRadius, 0,maxPixelTriplets * sizeof(FPX),stream);
    hipMemsetAsync(pt, 0,maxPixelTriplets * 6*sizeof(FPX),stream);
    hipMemsetAsync(isDup, 0,maxPixelTriplets * sizeof(bool),stream);
    hipMemsetAsync(partOfPT5, 0,maxPixelTriplets * sizeof(bool),stream);
}

void SDL::createPixelTripletsInExplicitMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets, hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    pixelTripletsInGPU.pixelSegmentIndices       =(unsigned int*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.tripletIndices            =(unsigned int*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.nPixelTriplets            =(unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    pixelTripletsInGPU.totOccupancyPixelTriplets =(unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    pixelTripletsInGPU.pixelRadius               =(FPX*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(FPX),stream);
    pixelTripletsInGPU.tripletRadius             =(FPX*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(FPX),stream);
    pixelTripletsInGPU.pt                        =(FPX*)cms::cuda::allocate_device(dev,maxPixelTriplets * 6*sizeof(FPX),stream);
    pixelTripletsInGPU.isDup                     =(bool*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(bool),stream);
    pixelTripletsInGPU.partOfPT5                 =(bool*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(bool),stream);
    pixelTripletsInGPU.centerX                   = (FPX*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(FPX), stream);
    pixelTripletsInGPU.centerY                   = (FPX*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(FPX), stream);
    pixelTripletsInGPU.lowerModuleIndices        = (uint16_t*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(uint16_t) * 5, stream);
    pixelTripletsInGPU.hitIndices                = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(unsigned int) * 10, stream);
    pixelTripletsInGPU.logicalLayers             = (uint8_t*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(uint8_t) * 5, stream);

    pixelTripletsInGPU.rPhiChiSquared = (float*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(float), stream);
    pixelTripletsInGPU.rPhiChiSquaredInwards = (float*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(float), stream);
    pixelTripletsInGPU.rzChiSquared = (float*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(float), stream);
#else
    hipMalloc(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.totOccupancyPixelTriplets, sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool));
    hipMalloc(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool));
    hipMalloc(&pixelTripletsInGPU.centerX, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.centerY, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.logicalLayers, maxPixelTriplets * sizeof(uint8_t) * 5);
    hipMalloc(&pixelTripletsInGPU.hitIndices, maxPixelTriplets * sizeof(unsigned int) * 10);
    hipMalloc(&pixelTripletsInGPU.lowerModuleIndices, maxPixelTriplets * sizeof(uint16_t) * 5);
    hipMalloc(&pixelTripletsInGPU.rPhiChiSquared, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.rPhiChiSquaredInwards, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.rzChiSquared, maxPixelTriplets * sizeof(float));
#endif
    hipMemsetAsync(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int),stream);
    hipMemsetAsync(pixelTripletsInGPU.totOccupancyPixelTriplets, 0, sizeof(unsigned int),stream);
    hipMemsetAsync(pixelTripletsInGPU.partOfPT5, 0, maxPixelTriplets*sizeof(bool),stream);
    hipStreamSynchronize(stream);

    pixelTripletsInGPU.eta = pixelTripletsInGPU.pt + maxPixelTriplets;
    pixelTripletsInGPU.phi = pixelTripletsInGPU.pt + maxPixelTriplets * 2;
    pixelTripletsInGPU.eta_pix = pixelTripletsInGPU.pt + maxPixelTriplets *3;
    pixelTripletsInGPU.phi_pix = pixelTripletsInGPU.pt + maxPixelTriplets * 4;
    pixelTripletsInGPU.score = pixelTripletsInGPU.pt + maxPixelTriplets * 5;

}

ALPAKA_FN_ACC void SDL::addPixelTripletToMemory(struct modules& modulesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, struct pixelTriplets& pixelTripletsInGPU, unsigned int pixelSegmentIndex, unsigned int tripletIndex, float pixelRadius, float tripletRadius, float centerX, float centerY, float rPhiChiSquared, float rPhiChiSquaredInwards, float rzChiSquared, unsigned int pixelTripletIndex, float pt, float eta, float phi, float eta_pix, float phi_pix,float score)
{
    pixelTripletsInGPU.pixelSegmentIndices[pixelTripletIndex] = pixelSegmentIndex;
    pixelTripletsInGPU.tripletIndices[pixelTripletIndex] = tripletIndex;
    pixelTripletsInGPU.pixelRadius[pixelTripletIndex] = __F2H(pixelRadius);
    pixelTripletsInGPU.tripletRadius[pixelTripletIndex] = __F2H(tripletRadius);
    pixelTripletsInGPU.pt[pixelTripletIndex] = __F2H(pt);
    pixelTripletsInGPU.eta[pixelTripletIndex] = __F2H(eta);
    pixelTripletsInGPU.phi[pixelTripletIndex] = __F2H(phi);
    pixelTripletsInGPU.eta_pix[pixelTripletIndex] = __F2H(eta_pix);
    pixelTripletsInGPU.phi_pix[pixelTripletIndex] = __F2H(phi_pix);
    pixelTripletsInGPU.isDup[pixelTripletIndex] = 0;
    pixelTripletsInGPU.score[pixelTripletIndex] = __F2H(score);

    pixelTripletsInGPU.centerX[pixelTripletIndex] = __F2H(centerX);
    pixelTripletsInGPU.centerY[pixelTripletIndex] = __F2H(centerY);
    pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex] = 0;
    pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex + 1] = 0;
    pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex + 2] = tripletsInGPU.logicalLayers[tripletIndex * 3];
    pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex + 3] = tripletsInGPU.logicalLayers[tripletIndex * 3 + 1];
    pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex + 4] = tripletsInGPU.logicalLayers[tripletIndex * 3 + 2];

    pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex] = segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];
    pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex + 1] = segmentsInGPU.outerLowerModuleIndices[pixelSegmentIndex];
    pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex + 2] = tripletsInGPU.lowerModuleIndices[3 * tripletIndex];
    pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex + 3] = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 1];
    pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex + 4] = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 2];
 
    unsigned int pixelInnerMD = segmentsInGPU.mdIndices[2 * pixelSegmentIndex];
    unsigned int pixelOuterMD = segmentsInGPU.mdIndices[2 * pixelSegmentIndex + 1];

    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex] = mdsInGPU.anchorHitIndices[pixelInnerMD];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 1] = mdsInGPU.outerHitIndices[pixelInnerMD];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 2] = mdsInGPU.anchorHitIndices[pixelOuterMD];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 3] = mdsInGPU.outerHitIndices[pixelOuterMD];

    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 4] = tripletsInGPU.hitIndices[6 * tripletIndex];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 5] = tripletsInGPU.hitIndices[6 * tripletIndex + 1];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 6] = tripletsInGPU.hitIndices[6 * tripletIndex + 2];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 7] = tripletsInGPU.hitIndices[6 * tripletIndex + 3];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 8] = tripletsInGPU.hitIndices[6 * tripletIndex + 4];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 9] = tripletsInGPU.hitIndices[6 * tripletIndex + 5];
    pixelTripletsInGPU.rPhiChiSquared[pixelTripletIndex] = rPhiChiSquared;
    pixelTripletsInGPU.rPhiChiSquaredInwards[pixelTripletIndex] = rPhiChiSquaredInwards;
    pixelTripletsInGPU.rzChiSquared[pixelTripletIndex] = rzChiSquared;
}

ALPAKA_FN_ACC float SDL::computeRadiusFromThreeAnchorHitspT3(float* xs, float* ys, float& g, float& f)
{
    float radius = 0;

    //writing manual code for computing radius, which obviously sucks
    //TODO:Use fancy inbuilt libraries like cuBLAS or cuSOLVE for this!
    //(g,f) -> center
    //first anchor hit - (x1,y1), second anchor hit - (x2,y2), third anchor hit - (x3, y3)

    /*
    if((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0)
    {
        return -1; //WTF man three collinear points!
    }
    */

    float denomInv = 1.f/((ys[0] - ys[2]) * (xs[1] - xs[2]) - (xs[0] - xs[2]) * (ys[1] - ys[2]));

    float xy1sqr = xs[0] * xs[0] + ys[0] * ys[0];

    float xy2sqr = xs[1] * xs[1] + ys[1] * ys[1];

    float xy3sqr = xs[2] * xs[2] + ys[2] * ys[2];

    g = 0.5f * ((ys[2] - ys[1]) * xy1sqr + (ys[0] - ys[2]) * xy2sqr + (ys[1] - ys[0]) * xy3sqr) * denomInv;

    f = 0.5f * ((xs[1] - xs[2]) * xy1sqr + (xs[2] - xs[0]) * xy2sqr + (xs[0] - xs[1]) * xy3sqr) * denomInv;

    float c = ((xs[1] * ys[2] - xs[2] * ys[1]) * xy1sqr + (xs[2] * ys[0] - xs[0] * ys[2]) * xy2sqr + (xs[0] * ys[1] - xs[1] * ys[0]) * xy3sqr) * denomInv;

    if(((ys[0] - ys[2]) * (xs[1] - xs[2]) - (xs[0] - xs[2]) * (ys[1] - ys[2]) == 0) || (g * g + f * f - c < 0))
    {
#ifdef Warnings
        printf("three collinear points or FATAL! r^2 < 0!\n");
#endif
        radius = -1;
    }
    else
      radius = sqrtf(g * g  + f * f - c);

    return radius;
}


ALPAKA_FN_ACC bool SDL::runPixelTripletDefaultAlgo(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, unsigned int& pixelSegmentIndex, unsigned int tripletIndex, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, float& centerX, float& centerY, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards, bool
        runChiSquaredCuts)
{
    bool pass = true;

    //run pT4 compatibility between the pixel segment and inner segment, and between the pixel and outer segment of the triplet

    uint16_t pixelModuleIndex = segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];

    uint16_t lowerModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex];
    uint16_t middleModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 1];
    uint16_t upperModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 2];

    {
    //placeholder
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    // pixel segment vs inner segment of the triplet
    pass = pass and runPixelTrackletDefaultAlgopT3(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, lowerModuleIndex, middleModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    if(not pass) return pass;

    //pixel segment vs outer segment of triplet
    pass = pass and runPixelTrackletDefaultAlgopT3(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, middleModuleIndex, upperModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex + 1], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    if(not pass) return pass;
    }

    //pt matching between the pixel ptin and the triplet circle pt
    unsigned int pixelSegmentArrayIndex = pixelSegmentIndex - rangesInGPU.segmentModuleIndices[pixelModuleIndex];
    float pixelSegmentPt = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
    float pixelSegmentPtError = segmentsInGPU.ptErr[pixelSegmentArrayIndex];
    float pixelSegmentPx = segmentsInGPU.px[pixelSegmentArrayIndex];
    float pixelSegmentPy = segmentsInGPU.py[pixelSegmentArrayIndex];
    float pixelSegmentPz = segmentsInGPU.pz[pixelSegmentArrayIndex];
    int pixelSegmentCharge = segmentsInGPU.charge[pixelSegmentArrayIndex];

    float pixelG = segmentsInGPU.circleCenterX[pixelSegmentArrayIndex];
    float pixelF = segmentsInGPU.circleCenterY[pixelSegmentArrayIndex];
    float pixelRadiusPCA = segmentsInGPU.circleRadius[pixelSegmentArrayIndex];

    unsigned int pixelInnerMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex];
    unsigned int pixelOuterMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex + 1];

    //unsigned int pixelAnchorHitIndex1 = mdsInGPU.anchorHitIndices[pixelInnerMDIndex];
    //unsigned int pixelAnchorHitIndex2 = mdsInGPU.anchorHitIndices[pixelOuterMDIndex];

    pixelRadius = pixelSegmentPt * kR1GeVf;
    pixelRadiusError = pixelSegmentPtError * kR1GeVf;
    unsigned int tripletInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex];
    unsigned int tripletOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex + 1];

    unsigned int firstMDIndex = segmentsInGPU.mdIndices[2 * tripletInnerSegmentIndex];
    unsigned int secondMDIndex = segmentsInGPU.mdIndices[2 * tripletInnerSegmentIndex + 1];
    unsigned int thirdMDIndex = segmentsInGPU.mdIndices[2 * tripletOuterSegmentIndex + 1];

    float xs[3] = {mdsInGPU.anchorX[firstMDIndex], mdsInGPU.anchorX[secondMDIndex], mdsInGPU.anchorX[thirdMDIndex]};
    float ys[3] = {mdsInGPU.anchorY[firstMDIndex], mdsInGPU.anchorY[secondMDIndex], mdsInGPU.anchorY[thirdMDIndex]};

    float g,f;
    tripletRadius = computeRadiusFromThreeAnchorHitspT3(xs, ys, g,f);    

    pass = pass and passRadiusCriterion(modulesInGPU, pixelRadius, pixelRadiusError, tripletRadius, lowerModuleIndex, middleModuleIndex, upperModuleIndex);
    if(not pass) return pass;


    uint16_t lowerModuleIndices[3] = {lowerModuleIndex, middleModuleIndex, upperModuleIndex};

    if(runChiSquaredCuts and pixelSegmentPt < 5.0f)
    {
        float rts[3] = {mdsInGPU.anchorRt[firstMDIndex], mdsInGPU.anchorRt[secondMDIndex], mdsInGPU.anchorRt[thirdMDIndex]};
        float xs[3] = {mdsInGPU.anchorX[firstMDIndex], mdsInGPU.anchorX[secondMDIndex], mdsInGPU.anchorX[thirdMDIndex]};
        float ys[3] = {mdsInGPU.anchorY[firstMDIndex], mdsInGPU.anchorY[secondMDIndex], mdsInGPU.anchorY[thirdMDIndex]};
        float zs[3] = {mdsInGPU.anchorZ[firstMDIndex], mdsInGPU.anchorZ[secondMDIndex], mdsInGPU.anchorZ[thirdMDIndex]};
        float rtPix[2] = {mdsInGPU.anchorRt[pixelInnerMDIndex], mdsInGPU.anchorRt[pixelOuterMDIndex]};
        float xPix[2] = {mdsInGPU.anchorX[pixelInnerMDIndex], mdsInGPU.anchorX[pixelOuterMDIndex]};
        float yPix[2] = {mdsInGPU.anchorY[pixelInnerMDIndex], mdsInGPU.anchorY[pixelOuterMDIndex]};
        float zPix[2] = {mdsInGPU.anchorZ[pixelInnerMDIndex], mdsInGPU.anchorZ[pixelOuterMDIndex]};

        rzChiSquared = computePT3RZChiSquared(modulesInGPU, lowerModuleIndices, rtPix, xPix, yPix, zPix, rts, xs, ys, zs, pixelSegmentPt, pixelSegmentPx, pixelSegmentPy, pixelSegmentPz, pixelSegmentCharge);
        pass = pass and passPT3RZChiSquaredCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rzChiSquared);
        if(not pass) return pass;
    }

    rPhiChiSquared = computePT3RPhiChiSquared(modulesInGPU, lowerModuleIndices, pixelG, pixelF, pixelRadiusPCA, xs, ys);

    if(runChiSquaredCuts and pixelSegmentPt < 5.0f)
    {
        pass = pass and passPT3RPhiChiSquaredCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rPhiChiSquared);
        if(not pass) return pass;
    }

    float xPix[2] = {mdsInGPU.anchorX[pixelInnerMDIndex], mdsInGPU.anchorX[pixelOuterMDIndex]};
    float yPix[2] = {mdsInGPU.anchorY[pixelInnerMDIndex], mdsInGPU.anchorY[pixelOuterMDIndex]};
    rPhiChiSquaredInwards = computePT3RPhiChiSquaredInwards(modulesInGPU, g, f, tripletRadius, xPix, yPix);

    if(runChiSquaredCuts and pixelSegmentPt < 5.0f)
    {
        pass = pass and passPT3RPhiChiSquaredInwardsCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rPhiChiSquaredInwards);
        if(not pass) return pass;
    }
    return pass;
}

ALPAKA_FN_ACC bool SDL::passPT3RPhiChiSquaredInwardsCuts(struct modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, float& chiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
 
    if(layer1 == 7 and layer2 == 8 and layer3 == 9) // endcap layer 1,2,3, ps
    {
        return chiSquared < 22016.8055f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14) // endcap layer 1,2,3 layer3->2s
    {
        return chiSquared < 935179.56807f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 10) // endcap layer 2,3,4
    {
        return chiSquared < 29064.12959f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15) // endcap layer 2,3,4, layer3->2s      
    {
        return chiSquared < 935179.5681f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3) // barrel 1,2,3
    {
        return chiSquared < 1370.0113195101474f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7) // barrel 1,2 endcap 1
    {
        return chiSquared < 5492.110048314815f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4) // barrel 2,3,4
    {
        return chiSquared < 4160.410806470067f;
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8) // barrel 1, endcap 1,2
    {
        return chiSquared < 29064.129591225726f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7) // barrel 2,3 endcap 1
    {
        return chiSquared < 12634.215376250893f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12) // barrel 2,3, endcap 1->2s
    {
        return chiSquared < 353821.69361145404f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8) // barrel2, endcap 1,2
    {
        return chiSquared < 33393.26076341235f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13) //barrel 2, endcap 1, endcap2->2s
    {
        return chiSquared < 935179.5680742573f;
    }

    return true;
}

ALPAKA_FN_ACC float SDL::computePT3RPhiChiSquaredInwards(struct modules& modulesInGPU, float& g, float& f, float& r, float* xPix, float* yPix)
{
    float residual = (xPix[0] - g) * (xPix[0] -g) + (yPix[0] - f) * (yPix[0] - f) - r * r;
    float chiSquared = residual * residual;
    residual = (xPix[1] - g) * (xPix[1] -g) + (yPix[1] - f) * (yPix[1] - f) - r * r;
    chiSquared += residual * residual;

    //chiSquared /= 2;
    chiSquared *= 0.5f;
    return chiSquared;
}

ALPAKA_FN_ACC bool SDL::passPT3RZChiSquaredCuts(struct modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, float& rzChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);

    if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return rzChiSquared < 13.6067f; //CMSSW 12 geo, add charge, new pixel map
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return rzChiSquared < 5.5953f; //CMSSW 12 geo, add charge, new pixel map
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return rzChiSquared < 3.9263f; //CMSSW 12 geo, add charge, new pixel map
    }
/*    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {   
        return rzChiSquared < 16.836f; //CMSSW 12 geo, add charge, new pixel map
    }
*/
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return rzChiSquared < 9.4377f; //CMSSW 12 geo, add charge, new pixel map
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return rzChiSquared < 9.9975f; //CMSSW 12 geo, add charge, new pixel map
    }

    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return rzChiSquared < 8.6369f; //CMSSW 12 geo, add charge, new pixel map
    }

    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return rzChiSquared < 37.945f; //CMSSW 12 geo, add charge, new pixel map
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return rzChiSquared < 43.0167f; //CMSSW 12 geo, add charge, new pixel map
    }
    
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return rzChiSquared < 8.6923f; //CMSSW 12 geo, add charge, new pixel map
    }

    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return rzChiSquared < 11.9672f; //CMSSW 12 geo, add charge, new pixel map
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return rzChiSquared < 16.2133f; //CMSSW 12 geo, add charge, new pixel map
    }

    //default - category not found!
    return true;
}

ALPAKA_FN_ACC float SDL::computePT3RZChiSquared(struct modules& modulesInGPU, uint16_t* lowerModuleIndices, float* rtPix, float* xPix, float* yPix, float* zPix, float* rts, float* xs, float* ys, float* zs, float pixelSegmentPt, float pixelSegmentPx, float pixelSegmentPy, float pixelSegmentPz, int pixelSegmentCharge)
{ 
    float slope = (zPix[1] - zPix[0])/(rtPix[1] - rtPix[0]);
    float residual = 0;
    float error = 0;
    //hardcoded array indices!!!
    float RMSE = 0;

    float Px=pixelSegmentPx, Py=pixelSegmentPy, Pz=pixelSegmentPz;
    int charge=pixelSegmentCharge;
    float x1 = xPix[1]/100;
    float y1 = yPix[1]/100;
    float z1 = zPix[1]/100;
    float r1 = rtPix[1]/100;

    float B = 3.8112;
    float a = -0.299792*B*charge;

    for(size_t i = 0; i < 3; i++)
    {
        float xsi = xs[i]/100;
        float ysi = ys[i]/100;
        float zsi = zs[i]/100;
        float rtsi = rts[i]/100;
        uint16_t lowerModuleIndex = lowerModuleIndices[i];
        const int moduleType = modulesInGPU.moduleType[lowerModuleIndex];
        const int moduleSide = modulesInGPU.sides[lowerModuleIndex];
        const int moduleSubdet = modulesInGPU.subdets[lowerModuleIndex];

        // calculation is detailed documented here https://indico.cern.ch/event/1185895/contributions/4982756/attachments/2526561/4345805/helix%20pT3%20summarize.pdf
        float diffr,diffz;
        float p = sqrt(Px*Px+Py*Py+Pz*Pz);

        float rou = a/p;
        if (moduleSubdet == SDL::Endcap){
            float s = (zsi-z1)*p/Pz;
            float x = x1 + Px/a*sin(rou*s)-Py/a*(1-cos(rou*s));
            float y = y1 + Py/a*sin(rou*s)+Px/a*(1-cos(rou*s));
//          float z = z1+Pz/p*s;
            diffr = fabs(rtsi-sqrt(x*x+y*y))*100;
        }

        if (moduleSubdet == SDL::Barrel){
            float paraA = r1*r1 + 2*(Px*Px+Py*Py)/(a*a) + 2*(y1*Px-x1*Py)/a - rtsi*rtsi;
            float paraB = 2*(x1*Px+y1*Py)/a;
            float paraC = 2*(y1*Px-x1*Py)/a+2*(Px*Px+Py*Py)/(a*a);
            //termA+paraB*sin(\rho s)  = paraC*sqrt(1-sin(\rho s)*sin(\rho s))
            float A=paraB*paraB+paraC*paraC;
            float B=2*paraA*paraB;
            float C=paraA*paraA-paraC*paraC;
            //A*sin(\rho s)*sin(\rho s)+B*sin(\rho s)+C=0;
            float sol1 = (-B+sqrt(B*B-4*A*C))/(2*A);
            float sol2 = (-B-sqrt(B*B-4*A*C))/(2*A);
            float solz1 = asin(sol1)/rou*Pz/p+z1;
            float solz2 = asin(sol2)/rou*Pz/p+z1;
            float diffz1 = fabs(solz1-zsi)*100;
            float diffz2 = fabs(solz2-zsi)*100;
            diffz = min(diffz1,diffz2);
        }

        residual = moduleSubdet == SDL::Barrel ? diffz : diffr ;

        //PS Modules
        if(moduleType == 0)
        {
            error = 0.15f;
        }
        else //2S modules
        {
            error = 5.0f;
        }

        //special dispensation to tilted PS modules!
        if(moduleType == 0 and moduleSubdet == SDL::Barrel and moduleSide != Center)
        {
            float drdz = modulesInGPU.drdzs[lowerModuleIndex];
            error /= sqrtf(1 + drdz * drdz);
        }
        RMSE += (residual * residual)/(error * error);
    }

    RMSE = sqrtf(0.2f * RMSE); //the constant doesn't really matter....

    return RMSE;
}

//TODO: merge this one and the pT5 function later into a single function
ALPAKA_FN_ACC float SDL::computePT3RPhiChiSquared(struct modules& modulesInGPU, uint16_t* lowerModuleIndices, float& g, float& f, float& radius, float* xs, float* ys)
{
    float delta1[3], delta2[3], slopes[3];
    bool isFlat[3];
    float chiSquared = 0;
    float inv1 = 0.01f/0.009f;
    float inv2 = 0.15f/0.009f;
    //float inv3 = 2.4f/0.009f;
    for(size_t i = 0; i < 3; i++)
    {
        ModuleType moduleType = modulesInGPU.moduleType[lowerModuleIndices[i]];
        short moduleSubdet = modulesInGPU.subdets[lowerModuleIndices[i]];
        short moduleSide = modulesInGPU.sides[lowerModuleIndices[i]];
        float drdz = modulesInGPU.drdzs[lowerModuleIndices[i]];
        slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
        //category 1 - barrel PS flat
        if(moduleSubdet == Barrel and moduleType == PS and moduleSide == Center)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            delta2[i] = inv1;//1.1111f;//0.01;
            slopes[i] = -999;
            isFlat[i] = true;
        }

        //category 2 - barrel 2S
        else if(moduleSubdet == Barrel and moduleType == TwoS)
        {
            delta1[i] = 1;//0.009;
            delta2[i] = 1;//0.009;
            slopes[i] = -999;
            isFlat[i] = true;
        }

        //category 3 - barrel PS tilted
        else if(moduleSubdet == Barrel and moduleType == PS and moduleSide != Center)
        {

            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;
            delta2[i] = (inv2 * drdz/sqrtf(1 + drdz * drdz));
        }

        //category 4 - endcap PS
        else if(moduleSubdet == Endcap and moduleType == PS)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            /*despite the type of the module layer of the lower module index,
            all anchor hits are on the pixel side and all non-anchor hits are
            on the strip side!*/        
            delta2[i] = inv2;//16.6666f;//0.15f;
        }

        //category 5 - endcap 2S
        else if(moduleSubdet == Endcap and moduleType == TwoS)
        {
            delta1[i] = 1;//0.009;
            delta2[i] = 500*inv1;//555.5555f;//5.f;
            isFlat[i] = false;
        }
#ifdef Warnings
        else
        {
            printf("ERROR!!!!! I SHOULDN'T BE HERE!!!! subdet = %d, type = %d, side = %d\n", moduleSubdet, moduleType, moduleSide);
        }
#endif
    }
    // this for loop is kept to keep the physics results the same but I think this is a bug in the original code. This was kept at 5 and not nPoints
    //for(size_t i = 3; i < 5; i++)
    //{
    //    delta1[i] /= 0.009f;
    //    delta2[i] /= 0.009f;
    //}
    //chiSquared = computeChiSquared(3, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius); //unclear why but the pT3 version doesn't work despite being copy and pasted
    chiSquared = computeChiSquaredpT3(3, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius); //unclear why but the pT3 version doesn't work despite being copy and pasted
    
    return chiSquared;
}


//90pc threshold
ALPAKA_FN_ACC bool SDL::passPT3RPhiChiSquaredCuts(struct modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, float& chiSquared)
{

    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);

    if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return chiSquared < 7.003f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return chiSquared < 0.5f;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return chiSquared < 8.046f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return chiSquared < 0.575f;
    }

    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return chiSquared < 5.304f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return chiSquared < 10.6211f;
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 4.617f;
    }

    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 8.046f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return chiSquared < 0.435f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return chiSquared < 9.244f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return chiSquared < 0.287f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return chiSquared < 18.509f;
    }

    return true;
}

ALPAKA_FN_ACC bool SDL::passRadiusCriterion(struct modules& modulesInGPU, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, uint16_t& lowerModuleIndex, uint16_t& middleModuleIndex, uint16_t& upperModuleIndex)
{
    if(modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionEEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[middleModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[upperModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBBE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else
    {
        return passRadiusCriterionBBB(pixelRadius, pixelRadiusError, tripletRadius);
    }

}

/*bounds for high Pt taken from : http://uaf-10.t2.ucsd.edu/~bsathian/SDL/T5_efficiency/efficiencies/new_efficiencies/efficiencies_20210513_T5_recovering_high_Pt_efficiencies/highE_radius_matching/highE_bounds.txt */
ALPAKA_FN_ACC bool SDL::passRadiusCriterionBBB(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0.15624f;
    float pixelInvRadiusErrorBound = 0.17235f;

    if(pixelRadius > 2.0f * kR1GeVf)
    {
        pixelInvRadiusErrorBound = 0.6375f;
        tripletInvRadiusErrorBound = 0.6588f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));

    return checkIntervalOverlappT3(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);
}

ALPAKA_FN_ACC bool SDL::passRadiusCriterionBBE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0.45972f;
    float pixelInvRadiusErrorBound = 0.19644f;

    if(pixelRadius > 2.0f * kR1GeVf)
    {
        pixelInvRadiusErrorBound = 0.6805f;
        tripletInvRadiusErrorBound = 0.8557f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));

    return checkIntervalOverlappT3(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

ALPAKA_FN_ACC bool SDL::passRadiusCriterionBEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 1.59294f;
    float pixelInvRadiusErrorBound = 0.255181f;

    if(pixelRadius > 2.0f * kR1GeVf) //as good as not having selections
    {
        pixelInvRadiusErrorBound = 2.2091f;
        tripletInvRadiusErrorBound = 2.3548f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));
    pixelRadiusInvMin = fmaxf(pixelRadiusInvMin, 0);

    return checkIntervalOverlappT3(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

ALPAKA_FN_ACC bool SDL::passRadiusCriterionEEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 1.7006f;
    float pixelInvRadiusErrorBound = 0.26367f;

    if(pixelRadius > 2.0f * kR1GeVf) //as good as not having selections
    {
        pixelInvRadiusErrorBound = 2.286f;
        tripletInvRadiusErrorBound = 2.436f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));
    pixelRadiusInvMin = fmaxf(0, pixelRadiusInvMin);

    return checkIntervalOverlappT3(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);
}

__global__ void SDL::createPixelTripletsInGPUFromMapv2(struct SDL::modules& modulesInGPU, struct SDL::objectRanges& rangesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU, unsigned int* connectedPixelSize, unsigned int* connectedPixelIndex, unsigned int nPixelSegments)
{
    //newgrid with map
    int blockxSize = blockDim.x*gridDim.x;
    int blockySize = blockDim.y*gridDim.y;
    //unsigned int offsetIndex = blockIdx.x * blockDim.x + threadIdx.x;
    // loop over pLS  and inside loop over outer T3s in modules.connectedPixels[[0, connectedPixelSize[i_pLS]) + connectedPixelIndex[i_pLS]]
    for(int i_pLS = blockIdx.y * blockDim.y + threadIdx.y; i_pLS < nPixelSegments; i_pLS += blockySize)
    {

      auto iLSModule_max = connectedPixelIndex[i_pLS] + connectedPixelSize[i_pLS];
      for (int iLSModule = connectedPixelIndex[i_pLS] + blockIdx.z; iLSModule < iLSModule_max; iLSModule += gridDim.z)
      {
        uint16_t tripletLowerModuleIndex = modulesInGPU.connectedPixels[iLSModule]; //connected pixels will have the appopriate lower module index by default!
#ifdef Warnings
        if(tripletLowerModuleIndex >= *modulesInGPU.nLowerModules) {
          printf("tripletLowerModuleIndex %d >= modulesInGPU.nLowerModules %d \n", tripletLowerModuleIndex, modulesInGPU.nLowerModules);
          continue;//sanity check
        }
#endif
        if(modulesInGPU.moduleType[tripletLowerModuleIndex] == SDL::TwoS) continue;//return; //Removes 2S-2S :FIXME: filter these out in the pixel map

        uint16_t pixelModuleIndex = *modulesInGPU.nLowerModules;
        unsigned int nOuterTriplets = tripletsInGPU.nTriplets[tripletLowerModuleIndex];
        if(nOuterTriplets == 0) continue;//return;

        unsigned int pixelSegmentIndex = rangesInGPU.segmentModuleIndices[pixelModuleIndex] + i_pLS;

        if(segmentsInGPU.isDup[i_pLS]) continue;
        if(segmentsInGPU.partOfPT5[i_pLS]) continue;//don't make pT3s for those pixels that are part of pT5

        short layer2_adjustment;// = 2 - modulesInGPU.layers[tripletLowerModuleIndex];
        if(modulesInGPU.layers[tripletLowerModuleIndex] == 1)
        {
            layer2_adjustment = 1;
        } //get upper segment to be in second layer
        else if( modulesInGPU.layers[tripletLowerModuleIndex] == 2)
        {
            layer2_adjustment = 0;
        } // get lower segment to be in second layer        
        else
        {
            continue;
        }

        //fetch the triplet
        for(unsigned int outerTripletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x; outerTripletArrayIndex< nOuterTriplets; outerTripletArrayIndex +=blockxSize)
        {
            unsigned int outerTripletIndex = rangesInGPU.tripletModuleIndices[tripletLowerModuleIndex] + outerTripletArrayIndex;
            if(modulesInGPU.moduleType[tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 1]] == SDL::TwoS) continue;//REMOVES PS-2S

            if(tripletsInGPU.partOfPT5[outerTripletIndex]) continue;//don't create pT3s for T3s accounted in pT5s

            float pixelRadius, pixelRadiusError, tripletRadius, rPhiChiSquared, rzChiSquared, rPhiChiSquaredInwards, centerX, centerY;
            bool success = runPixelTripletDefaultAlgo(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius, pixelRadiusError, tripletRadius, centerX, centerY, rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards);

            if(success)
            {
                float phi = mdsInGPU.anchorPhi[segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex]+layer2_adjustment]];
                float eta = mdsInGPU.anchorEta[segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex]+layer2_adjustment]];
                float eta_pix = segmentsInGPU.eta[i_pLS];
                float phi_pix = segmentsInGPU.phi[i_pLS];
                float pt = segmentsInGPU.ptIn[i_pLS];
                float score = rPhiChiSquared+rPhiChiSquaredInwards;
                unsigned int totOccupancyPixelTriplets = atomicAdd(pixelTripletsInGPU.totOccupancyPixelTriplets, 1);
                if(totOccupancyPixelTriplets >= N_MAX_PIXEL_TRIPLETS)
                {
#ifdef Warnings
                    printf("Pixel Triplet excess alert!\n");
#endif
                }
                else
                {
                    unsigned int pixelTripletIndex = atomicAdd(pixelTripletsInGPU.nPixelTriplets, 1);
                    addPixelTripletToMemory(modulesInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, pixelTripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius,tripletRadius, centerX, centerY, rPhiChiSquared, rPhiChiSquaredInwards, rzChiSquared, pixelTripletIndex, pt,eta,phi,eta_pix,phi_pix,score);
                    tripletsInGPU.partOfPT3[outerTripletIndex] = true;
                }
            }
        } // for outerTripletArrayIndex
      } // for iLSModule < iLSModule_max
    } // for i_pLS
}

ALPAKA_FN_ACC void SDL::runDeltaBetaIterationspT3(float& betaIn, float& betaOut, float& betaAv, float & pt_beta, float sdIn_dr, float sdOut_dr, float dr, float lIn)
{
    if (lIn == 0)
    {
        betaOut += copysign(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaOut);
        return;
    }

    if (betaIn * betaOut > 0.f and (fabsf(pt_beta) < 4.f * SDL::pt_betaMax or (lIn >= 11 and fabsf(pt_beta) < 8.f * SDL::pt_betaMax)))   //and the pt_beta is well-defined; less strict for endcap-endcap
    {

        const float betaInUpd  = betaIn + copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        const float betaOutUpd = betaOut + copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaOut); //FIXME: need a faster version
        betaAv = 0.5f * (betaInUpd + betaOutUpd);

        //1st update
        //pt_beta = dr * k2Rinv1GeVf / sinf(betaAv); //get a better pt estimate
        const float pt_beta_inv = 1.f/fabsf(dr * k2Rinv1GeVf / sinf(betaAv)); //get a better pt estimate

        betaIn  += copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf *pt_beta_inv, SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaOut += copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf *pt_beta_inv, SDL::sinAlphaMax)), betaOut); //FIXME: need a faster version
        //update the av and pt
        betaAv = 0.5f * (betaIn + betaOut);
        //2nd update
        pt_beta = dr * SDL::k2Rinv1GeVf / sinf(betaAv); //get a better pt estimate
    }
    else if (lIn < 11 && fabsf(betaOut) < 0.2f * fabsf(betaIn) && fabsf(pt_beta) < 12.f * SDL::pt_betaMax)   //use betaIn sign as ref
    {

        const float pt_betaIn = dr * k2Rinv1GeVf / sinf(betaIn);

        const float betaInUpd  = betaIn + copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_betaIn), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        const float betaOutUpd = betaOut + copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_betaIn), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaAv = (fabsf(betaOut) > 0.2f * fabsf(betaIn)) ? (0.5f * (betaInUpd + betaOutUpd)) : betaInUpd;

        //1st update
        pt_beta = dr * SDL::k2Rinv1GeVf / sin(betaAv); //get a better pt estimate
        betaIn  += copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaOut += copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        //update the av and pt
        betaAv = 0.5f * (betaIn + betaOut);
        //2nd update
        pt_beta = dr * SDL::k2Rinv1GeVf / sin(betaAv); //get a better pt estimate

    }
}

ALPAKA_FN_ACC float SDL::computeChiSquaredpT3(int nPoints, float* xs, float* ys, float* delta1, float* delta2, float* slopes, bool* isFlat, float g, float f, float radius)
{
    // given values of (g, f, radius) and a set of points (and its uncertainties)
    //compute chi squared
    float c = g*g + f*f - radius*radius;
    float chiSquared = 0.f;
    float absArctanSlope, angleM, xPrime, yPrime, sigma;
    for(size_t i = 0; i < nPoints; i++)
    {
        absArctanSlope = ((slopes[i] != SDL::SDL_INF) ? fabs(atanf(slopes[i])) : 0.5f*float(M_PI)); // Since C++ can't represent infinity, SDL_INF = 123456789 was used to represent infinity in the data table
        if(xs[i] > 0 and ys[i] > 0)
        {
            angleM = 0.5f*float(M_PI) - absArctanSlope;
        }
        else if(xs[i] < 0 and ys[i] > 0)
        {
            angleM = absArctanSlope + 0.5f*float(M_PI);
        }
        else if(xs[i] < 0 and ys[i] < 0)
        {
            angleM = -(absArctanSlope + 0.5f*float(M_PI));
        }
        else if(xs[i] > 0 and ys[i] < 0)
        {
            angleM = -(0.5f*float(M_PI) - absArctanSlope);
        }

        if(not isFlat[i])
        {
            xPrime = xs[i] * cosf(angleM) + ys[i] * sinf(angleM);
            yPrime = ys[i] * cosf(angleM) - xs[i] * sinf(angleM);
        }
        else
        {
            xPrime = xs[i];
            yPrime = ys[i];
        }
        sigma = 2 * sqrtf((xPrime * delta1[i]) * (xPrime * delta1[i]) + (yPrime * delta2[i]) * (yPrime * delta2[i]));
        chiSquared +=  (xs[i] * xs[i] + ys[i] * ys[i] - 2 * g * xs[i] - 2 * f * ys[i] + c) * (xs[i] * xs[i] + ys[i] * ys[i] - 2 * g * xs[i] - 2 * f * ys[i] + c) / (sigma * sigma);
    }
    return chiSquared;
}

ALPAKA_FN_ACC ALPAKA_FN_INLINE bool SDL::runTripletDefaultAlgoPPBB(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU ,struct segments& segmentsInGPU, uint16_t& pixelModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int thirdMDIndex, unsigned int& fourthMDIndex, float& /*z_OutLo*/, float& /*rt_OutLo*/, float& dPhiPos, float& dPhi, float& betaIn,
        float& betaOut, float& pt_beta, float& zLo, float& zHi, float& zLoPointed, float& zHiPointed, float& sdlCut, float& betaOutCut, float& deltaBetaCut) // pixel to BB and BE segments
{
    bool pass = true;

    bool isPS_OutLo = (modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS);

    float rt_InLo = mdsInGPU.anchorRt[firstMDIndex];
    float rt_InUp = mdsInGPU.anchorRt[secondMDIndex];
    float rt_OutLo = mdsInGPU.anchorRt[thirdMDIndex];
    float rt_OutUp = mdsInGPU.anchorRt[fourthMDIndex];

    float z_InLo = mdsInGPU.anchorZ[firstMDIndex];
    float z_InUp = mdsInGPU.anchorZ[secondMDIndex];
    float z_OutLo = mdsInGPU.anchorZ[thirdMDIndex];
    float z_OutUp = mdsInGPU.anchorZ[fourthMDIndex];

    float x_InLo = mdsInGPU.anchorX[firstMDIndex];
    float x_InUp = mdsInGPU.anchorX[secondMDIndex];
    float x_OutLo = mdsInGPU.anchorX[thirdMDIndex];
    float x_OutUp = mdsInGPU.anchorX[fourthMDIndex];

    float y_InLo = mdsInGPU.anchorY[firstMDIndex];
    float y_InUp = mdsInGPU.anchorY[secondMDIndex];
    float y_OutLo = mdsInGPU.anchorY[thirdMDIndex];
    float y_OutUp = mdsInGPU.anchorY[fourthMDIndex];

    float rt_InOut = rt_InUp;
    //float& z_InOut = z_InUp;

    pass = pass and (fabsf(deltaPhi(x_InUp, y_InUp, x_OutLo, y_OutLo)) <= 0.5f * float(M_PI));
    if(not pass) return pass;

    unsigned int pixelSegmentArrayIndex = innerSegmentIndex - rangesInGPU.segmentModuleIndices[pixelModuleIndex];
    float ptIn = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
    float ptSLo = ptIn;
    float px = segmentsInGPU.px[pixelSegmentArrayIndex];
    float py = segmentsInGPU.py[pixelSegmentArrayIndex];
    float pz = segmentsInGPU.pz[pixelSegmentArrayIndex];
    float ptErr = segmentsInGPU.ptErr[pixelSegmentArrayIndex];
    float etaErr = segmentsInGPU.etaErr[pixelSegmentArrayIndex];
    ptSLo = fmaxf(ptCut, ptSLo - 10.0f*fmaxf(ptErr, 0.005f*ptSLo));
    ptSLo = fminf(10.0f, ptSLo);


    float alpha1GeV_OutLo = asinf(fminf(rt_OutLo * k2Rinv1GeVf / ptCut, sinAlphaMax));
    //float rtRatio_OutLoInLo = rt_OutLo / rt_InLo; // Outer segment beginning rt divided by inner segment beginning rt;
    const float rtRatio_OutLoInOut = rt_OutLo / rt_InOut; // Outer segment beginning rt divided by inner segment beginning rt;

    float dzDrtScale = tanf(alpha1GeV_OutLo) / alpha1GeV_OutLo; // The track can bend in r-z plane slightly
    const float zpitch_InLo = 0.05f;
    const float zpitch_InOut = 0.05f;
    float zpitch_OutLo = (isPS_OutLo ? pixelPSZpitch : strip2SZpitch);
    float zGeom = zpitch_InLo + zpitch_OutLo;
    zHi = z_InUp + (z_InUp + deltaZLum) * (rtRatio_OutLoInOut - 1.f) * (z_InUp < 0.f ? 1.f : dzDrtScale) + (zpitch_InOut + zpitch_OutLo);
    zLo = z_InUp + (z_InUp - deltaZLum) * (rtRatio_OutLoInOut - 1.f) * (z_InUp > 0.f ? 1.f : dzDrtScale) - (zpitch_InOut + zpitch_OutLo); //slope-correction only on outer end

    pass = pass and ((z_OutLo >= zLo) & (z_OutLo <= zHi));
    if(not pass) return pass;

    const float coshEta = sqrtf(ptIn * ptIn + pz * pz) / ptIn;
    // const float drt_OutLo_InLo = (rt_OutLo - rt_InLo);
    const float drt_OutLo_InUp = (rt_OutLo - rt_InUp);
    //const float invRt_InLo = 1.f / rt_InLo;
    //const float r3_InLo = sqrtf(z_InLo * z_InLo + rt_InLo * rt_InLo);
    const float r3_InUp = sqrtf(z_InUp * z_InUp + rt_InUp * rt_InUp);

    float drt_InSeg = rt_InOut - rt_InLo;
    //float dz_InSeg = z_InOut - z_InLo;
    //float dr3_InSeg = sqrtf(rt_InOut * rt_InOut + z_InOut * z_InOut) - sqrtf(rt_InLo * rt_InLo + z_InLo * z_InLo);

    const float sdlThetaMulsF = 0.015f * sqrtf(0.1f + 0.2f * (rt_OutLo - rt_InUp) / 50.f) * sqrtf(r3_InUp / rt_InUp);
    const float sdlMuls = sdlThetaMulsF * 3.f / ptCut * 4.f; // will need a better guess than x4?

    float dzErr = drt_OutLo_InUp*etaErr*coshEta; //FIXME: check with the calc in the endcap
    dzErr *= dzErr;
    dzErr += 0.03f*0.03f; // pixel size x2. ... random for now
    dzErr *= 9.f; //3 sigma
    dzErr += sdlMuls*sdlMuls*drt_OutLo_InUp*drt_OutLo_InUp/3.f*coshEta*coshEta;//sloppy
    dzErr += zGeom*zGeom;
    dzErr = sqrtf(dzErr);

    const float dzDrIn = pz / ptIn;
    const float zWindow = dzErr / drt_InSeg * drt_OutLo_InUp + zGeom;
    const float dzMean = dzDrIn * drt_OutLo_InUp *
        (1.f + drt_OutLo_InUp * drt_OutLo_InUp * 4 * k2Rinv1GeVf * k2Rinv1GeVf / ptIn /
         ptIn / 24.f); // with curved path correction
    // Constructing upper and lower bound
    zLoPointed = z_InUp + dzMean - zWindow;
    zHiPointed = z_InUp + dzMean + zWindow;

    pass =  pass and ((z_OutLo >= zLoPointed) & (z_OutLo <= zHiPointed));
    if(not pass) return pass;

    const float sdlPVoff = 0.1f / rt_OutLo;
    sdlCut = alpha1GeV_OutLo + sqrtf(sdlMuls * sdlMuls + sdlPVoff * sdlPVoff);

#ifdef CUT_VALUE_DEBUG
    dPhiPos = deltaPhi(x_InUp, y_InUp, x_OutUp, y_OutUp);
    //no dphipos cut
#endif

    float midPointX = 0.5f * (x_InLo + x_OutLo);
    float midPointY = 0.5f * (y_InLo + y_OutLo);
    float midPointZ = 0.5f * (z_InLo + z_OutLo);

    float diffX = x_OutLo - x_InLo;
    float diffY = y_OutLo - y_InLo;
    float diffZ = z_OutLo - z_InLo;


    dPhi = deltaPhi(midPointX, midPointY, diffX, diffY);

    pass = pass and (fabsf(dPhi) <= sdlCut);
    if(not pass) return pass;

    //lots of array accesses below this...

    float alpha_InLo  = __H2F(segmentsInGPU.dPhiChanges[innerSegmentIndex]);
    float alpha_OutLo = __H2F(segmentsInGPU.dPhiChanges[outerSegmentIndex]);

    bool isEC_lastLayer = modulesInGPU.subdets[outerOuterLowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS;

    float alpha_OutUp,alpha_OutUp_highEdge,alpha_OutUp_lowEdge;
    alpha_OutUp = deltaPhi(x_OutUp, y_OutUp, x_OutUp - x_OutLo, y_OutUp - y_OutLo);

    alpha_OutUp_highEdge = alpha_OutUp;
    alpha_OutUp_lowEdge = alpha_OutUp;

    float tl_axis_x = x_OutUp - x_InUp;
    float tl_axis_y = y_OutUp - y_InUp;
    float tl_axis_z = z_OutUp - z_InUp;

    float tl_axis_highEdge_x = tl_axis_x;
    float tl_axis_highEdge_y = tl_axis_y;

    float tl_axis_lowEdge_x = tl_axis_x;
    float tl_axis_lowEdge_y = tl_axis_y;

    betaIn = -deltaPhi(px, py, tl_axis_x, tl_axis_y);
    float betaInRHmin = betaIn;
    float betaInRHmax = betaIn;

    betaOut = -alpha_OutUp + deltaPhi(x_OutUp, y_OutUp, tl_axis_x, tl_axis_y);

    float betaOutRHmin = betaOut;
    float betaOutRHmax = betaOut;

    if(isEC_lastLayer)
    {
        alpha_OutUp_highEdge = deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], mdsInGPU.anchorHighEdgeX[fourthMDIndex] - x_OutLo, mdsInGPU.anchorHighEdgeY[fourthMDIndex] - y_OutLo);
        alpha_OutUp_lowEdge = deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], mdsInGPU.anchorLowEdgeX[fourthMDIndex] - x_OutLo, mdsInGPU.anchorLowEdgeY[fourthMDIndex] - y_OutLo);

        tl_axis_highEdge_x = mdsInGPU.anchorHighEdgeX[fourthMDIndex] - x_InUp;
        tl_axis_highEdge_y = mdsInGPU.anchorHighEdgeY[fourthMDIndex] - y_InUp;
        tl_axis_lowEdge_x = mdsInGPU.anchorLowEdgeX[fourthMDIndex] - x_InUp;
        tl_axis_lowEdge_y = mdsInGPU.anchorLowEdgeY[fourthMDIndex] - y_InUp;

        betaOutRHmin = -alpha_OutUp_highEdge + deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], tl_axis_highEdge_x, tl_axis_highEdge_y);
        betaOutRHmax = -alpha_OutUp_lowEdge + deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], tl_axis_lowEdge_x, tl_axis_lowEdge_y);
    }

    //beta computation
    float drt_tl_axis = sqrtf(tl_axis_x * tl_axis_x + tl_axis_y * tl_axis_y);
    //float drt_tl_lowEdge = sqrtf(tl_axis_lowEdge_x * tl_axis_lowEdge_x + tl_axis_lowEdge_y * tl_axis_lowEdge_y);
    //float drt_tl_highEdge = sqrtf(tl_axis_highEdge_x * tl_axis_highEdge_x + tl_axis_highEdge_y * tl_axis_highEdge_y);

    //innerOuterAnchor - innerInnerAnchor
    const float rt_InSeg = sqrtf((x_InUp - x_InLo) * (x_InUp - x_InLo) + (y_InUp - y_InLo) * (y_InUp - y_InLo));

    //no betaIn cut for the pixels
    float betaAv = 0.5f * (betaIn + betaOut);
    pt_beta = ptIn;

    int lIn = 0;
    int lOut = isEC_lastLayer ? 11 : 5;
    float sdOut_dr = sqrtf((x_OutUp - x_OutLo) * (x_OutUp - x_OutLo) + (y_OutUp - y_OutLo) * (y_OutUp - y_OutLo));
    float sdOut_d = rt_OutUp - rt_OutLo;

    //const float diffDr = fabsf(rt_InSeg - sdOut_dr) / fabsf(rt_InSeg + sdOut_dr);

    runDeltaBetaIterationspT3(betaIn, betaOut, betaAv, pt_beta, rt_InSeg, sdOut_dr, drt_tl_axis, lIn);

    const float betaInMMSF = (fabsf(betaInRHmin + betaInRHmax) > 0) ? (2.f * betaIn / fabsf(betaInRHmin + betaInRHmax)) : 0.; //mean value of min,max is the old betaIn
    const float betaOutMMSF = (fabsf(betaOutRHmin + betaOutRHmax) > 0) ? (2.f * betaOut / fabsf(betaOutRHmin + betaOutRHmax)) : 0.;
    betaInRHmin *= betaInMMSF;
    betaInRHmax *= betaInMMSF;
    betaOutRHmin *= betaOutMMSF;
    betaOutRHmax *= betaOutMMSF;

    const float dBetaMuls = sdlThetaMulsF * 4.f / fminf(fabsf(pt_beta), SDL::pt_betaMax); //need to confirm the range-out value of 7 GeV
    const float alphaInAbsReg =  fmaxf(fabsf(alpha_InLo), asinf(fminf(rt_InUp * k2Rinv1GeVf / 3.0f, sinAlphaMax)));
    const float alphaOutAbsReg = fmaxf(fabsf(alpha_OutLo), asinf(fminf(rt_OutLo * k2Rinv1GeVf / 3.0f, sinAlphaMax)));
    const float dBetaInLum = lIn < 11 ? 0.0f : fabsf(alphaInAbsReg*deltaZLum / z_InUp);
    const float dBetaOutLum = lOut < 11 ? 0.0f : fabsf(alphaOutAbsReg*deltaZLum / z_OutLo);
    const float dBetaLum2 = (dBetaInLum + dBetaOutLum) * (dBetaInLum + dBetaOutLum);

    const float sinDPhi = sinf(dPhi);
    const float dBetaRIn2 = 0; // TODO-RH

    float dBetaROut = 0;
    if(isEC_lastLayer)
    {
        dBetaROut = (sqrtf(mdsInGPU.anchorHighEdgeX[fourthMDIndex] * mdsInGPU.anchorHighEdgeX[fourthMDIndex] + mdsInGPU.anchorHighEdgeY[fourthMDIndex] * mdsInGPU.anchorHighEdgeY[fourthMDIndex]) - sqrtf(mdsInGPU.anchorLowEdgeX[fourthMDIndex] * mdsInGPU.anchorLowEdgeX[fourthMDIndex] + mdsInGPU.anchorLowEdgeY[fourthMDIndex] * mdsInGPU.anchorLowEdgeY[fourthMDIndex])) * sinDPhi / drt_tl_axis;
    }

    const float dBetaROut2 =  dBetaROut * dBetaROut;

    betaOutCut = asinf(fminf(drt_tl_axis*k2Rinv1GeVf / ptCut, sinAlphaMax)) //FIXME: need faster version
        + (0.02f / sdOut_d) + sqrtf(dBetaLum2 + dBetaMuls*dBetaMuls);

    //Cut #6: The real beta cut
    pass = pass and (fabsf(betaOut) < betaOutCut);
    if(not pass) return pass;

    //const float pt_betaOut = drt_tl_axis * k2Rinv1GeVf / sin(betaOut);
    const float dBetaRes = 0.02f / fminf(sdOut_d, drt_InSeg);
    const float dBetaCut2 = (dBetaRes*dBetaRes * 2.0f + dBetaMuls * dBetaMuls + dBetaLum2 + dBetaRIn2 + dBetaROut2
            + 0.25f * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)) * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)));
    float dBeta = betaIn - betaOut;
#ifdef CUT_VALUE_DEBUG
    deltaBetaCut = sqrtf(dBetaCut2);
#endif

    pass = pass and (dBeta * dBeta <= dBetaCut2);

    return pass;
}

ALPAKA_FN_ACC ALPAKA_FN_INLINE bool SDL::runTripletDefaultAlgoPPEE(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU ,struct segments& segmentsInGPU, uint16_t& pixelModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int& thirdMDIndex, unsigned int& fourthMDIndex, float& /*z_OutLo*/, float& /*rt_OutLo*/, float& deltaPhiPos, float& dPhi, float& betaIn,
        float& betaOut, float& pt_beta, float& zLo, float& rtLo, float& rtHi, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ) // pixel to EE segments
{
    bool pass = true;
    bool isPS_OutLo = (modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS);


    float z_InLo = mdsInGPU.anchorZ[firstMDIndex];
    float z_InUp = mdsInGPU.anchorZ[secondMDIndex];
    float z_OutLo = mdsInGPU.anchorZ[thirdMDIndex];
    float z_OutUp = mdsInGPU.anchorZ[fourthMDIndex];

    pass =  pass and (z_InUp * z_OutLo > 0);
    if(not pass) return pass;

    float rt_InLo = mdsInGPU.anchorRt[firstMDIndex];
    float rt_InUp = mdsInGPU.anchorRt[secondMDIndex];
    float rt_OutLo = mdsInGPU.anchorRt[thirdMDIndex];
    float rt_OutUp = mdsInGPU.anchorRt[fourthMDIndex];

    float x_InLo = mdsInGPU.anchorX[firstMDIndex];
    float x_InUp = mdsInGPU.anchorX[secondMDIndex];
    float x_OutLo = mdsInGPU.anchorX[thirdMDIndex];
    float x_OutUp = mdsInGPU.anchorX[fourthMDIndex];

    float y_InLo = mdsInGPU.anchorY[firstMDIndex];
    float y_InUp = mdsInGPU.anchorY[secondMDIndex];
    float y_OutLo = mdsInGPU.anchorY[thirdMDIndex];
    float y_OutUp = mdsInGPU.anchorY[fourthMDIndex];

    unsigned int pixelSegmentArrayIndex = innerSegmentIndex - rangesInGPU.segmentModuleIndices[pixelModuleIndex];

    float ptIn = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
    float ptSLo = ptIn;
    float px = segmentsInGPU.px[pixelSegmentArrayIndex];
    float py = segmentsInGPU.py[pixelSegmentArrayIndex];
    float pz = segmentsInGPU.pz[pixelSegmentArrayIndex];
    float ptErr = segmentsInGPU.ptErr[pixelSegmentArrayIndex];
    float etaErr = segmentsInGPU.etaErr[pixelSegmentArrayIndex];

    ptSLo = fmaxf(ptCut, ptSLo - 10.0f*fmaxf(ptErr, 0.005f*ptSLo));
    ptSLo = fminf(10.0f, ptSLo);

    float rtOut_o_rtIn = rt_OutLo/rt_InUp;
    const float zpitch_InLo = 0.05f;
    float zpitch_OutLo = (isPS_OutLo ? pixelPSZpitch : strip2SZpitch);
    float zGeom = zpitch_InLo + zpitch_OutLo;

    const float sdlSlope = asinf(fminf(rt_OutLo * k2Rinv1GeVf / ptCut, sinAlphaMax));
    const float dzDrtScale = tanf(sdlSlope) / sdlSlope;//FIXME: need approximate value
    zLo = z_InUp + (z_InUp - deltaZLum) * (rtOut_o_rtIn - 1.f) * (z_InUp > 0.f ? 1.f : dzDrtScale) - zGeom; //slope-correction only on outer end


    const float dLum = copysignf(deltaZLum, z_InUp);
    bool isOutSgInnerMDPS = modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS;

    const float rtGeom1 = isOutSgInnerMDPS ? pixelPSZpitch : strip2SZpitch;//FIXME: make this chosen by configuration for lay11,12 full PS
    const float zGeom1 = copysignf(zGeom, z_InUp); //used in B-E region
    rtLo = rt_InUp * (1.f + (z_OutLo- z_InUp - zGeom1) / (z_InUp + zGeom1 + dLum) / dzDrtScale) - rtGeom1; //slope correction only on the lower end


    float zInForHi = z_InUp - zGeom1 - dLum;
    if (zInForHi * z_InUp < 0)
        zInForHi = copysignf(0.1f, z_InUp);
    rtHi = rt_InUp * (1.f + (z_OutLo - z_InUp + zGeom1) / zInForHi) + rtGeom1;

    // Cut #2: rt condition
    pass =  pass and ((rt_OutLo >= rtLo) & (rt_OutLo <= rtHi));
    if(not pass) return pass;

    const float dzOutInAbs = fabsf(z_OutLo - z_InUp);
    const float coshEta = hypotf(ptIn, pz) / ptIn;
    const float multDzDr = dzOutInAbs*coshEta/(coshEta*coshEta - 1.f);
    const float r3_InUp = sqrtf(z_InUp * z_InUp + rt_InUp * rt_InUp);
    const float sdlThetaMulsF = 0.015f * sqrtf(0.1f + 0.2f * (rt_OutLo - rt_InUp) / 50.f) * sqrtf(r3_InUp / rt_InUp);
    const float sdlMuls = sdlThetaMulsF * 3.f / ptCut * 4.f; // will need a better guess than x4?

    float drtErr = etaErr*multDzDr;
    drtErr *= drtErr;
    drtErr += 0.03f*0.03f; // pixel size x2. ... random for now
    drtErr *= 9.f; //3 sigma
    drtErr += sdlMuls*sdlMuls*multDzDr*multDzDr/3.f*coshEta*coshEta;//sloppy: relative muls is 1/3 of total muls
    drtErr = sqrtf(drtErr);
    const float drtDzIn = fabsf(ptIn / pz);//all tracks are out-going in endcaps?

    const float drt_OutLo_InUp = (rt_OutLo - rt_InUp); // drOutIn

    const float rtWindow = drtErr + rtGeom1;
    const float drtMean = drtDzIn * dzOutInAbs *
        (1.f - drt_OutLo_InUp * drt_OutLo_InUp * 4 * k2Rinv1GeVf * k2Rinv1GeVf / ptIn /
         ptIn / 24.f); // with curved path correction
    const float rtLo_point = rt_InUp + drtMean - rtWindow;
    const float rtHi_point = rt_InUp + drtMean + rtWindow;

    // Cut #3: rt-z pointed
    pass =  pass and ((rt_OutLo >= rtLo_point) & (rt_OutLo <= rtHi_point));
    if(not pass) return pass;

    const float alpha1GeV_OutLo = asinf(fminf(rt_OutLo * k2Rinv1GeVf / ptCut, sinAlphaMax));
    const float sdlPVoff = 0.1f / rt_OutLo;
    sdlCut = alpha1GeV_OutLo + sqrtf(sdlMuls * sdlMuls + sdlPVoff * sdlPVoff);

    deltaPhiPos = deltaPhi(x_InUp, y_InUp, x_OutUp, y_OutUp);

    float midPointX = 0.5f * (x_InLo + x_OutLo);
    float midPointY = 0.5f * (y_InLo + y_OutLo);
    float midPointZ = 0.5f * (z_InLo + z_OutLo);

    float diffX = x_OutLo - x_InLo;
    float diffY = y_OutLo - y_InLo;
    float diffZ = z_OutLo - z_InLo;

    dPhi = deltaPhi(midPointX, midPointY, diffX, diffY);

    // Cut #5: deltaPhiChange
    pass =  pass and (fabsf(dPhi) <= sdlCut);
    if(not pass) return pass;

    float alpha_InLo  = __H2F(segmentsInGPU.dPhiChanges[innerSegmentIndex]);
    float alpha_OutLo = __H2F(segmentsInGPU.dPhiChanges[outerSegmentIndex]);

    bool isEC_lastLayer = modulesInGPU.subdets[outerOuterLowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS;

    float alpha_OutUp,alpha_OutUp_highEdge,alpha_OutUp_lowEdge;

    alpha_OutUp = deltaPhi(x_OutUp, y_OutUp, x_OutUp - x_OutLo, y_OutUp - y_OutLo);
    alpha_OutUp_highEdge = alpha_OutUp;
    alpha_OutUp_lowEdge = alpha_OutUp;

    float tl_axis_x = x_OutUp - x_InUp;
    float tl_axis_y = y_OutUp - y_InUp;
    float tl_axis_z = z_OutUp - z_InUp;

    float tl_axis_highEdge_x = tl_axis_x;
    float tl_axis_highEdge_y = tl_axis_y;

    float tl_axis_lowEdge_x = tl_axis_x;
    float tl_axis_lowEdge_y = tl_axis_y;

    betaIn = -deltaPhi(px, py, tl_axis_x, tl_axis_y);
    float betaInRHmin = betaIn;
    float betaInRHmax = betaIn;

    betaOut = -alpha_OutUp + deltaPhi(x_OutUp, y_OutUp, tl_axis_x, tl_axis_y);
    float betaOutRHmin = betaOut;
    float betaOutRHmax = betaOut;

    if(isEC_lastLayer)
    {

        alpha_OutUp_highEdge = deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], mdsInGPU.anchorHighEdgeX[fourthMDIndex] - x_OutLo, mdsInGPU.anchorHighEdgeY[fourthMDIndex] - y_OutLo);
        alpha_OutUp_lowEdge = deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], mdsInGPU.anchorLowEdgeX[fourthMDIndex] - x_OutLo, mdsInGPU.anchorLowEdgeY[fourthMDIndex] - y_OutLo);

        tl_axis_highEdge_x = mdsInGPU.anchorHighEdgeX[fourthMDIndex] - x_InUp;
        tl_axis_highEdge_y = mdsInGPU.anchorHighEdgeY[fourthMDIndex] - y_InUp;
        tl_axis_lowEdge_x = mdsInGPU.anchorLowEdgeX[fourthMDIndex] - x_InUp;
        tl_axis_lowEdge_y = mdsInGPU.anchorLowEdgeY[fourthMDIndex] - y_InUp;

        betaOutRHmin = -alpha_OutUp_highEdge + deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], tl_axis_highEdge_x, tl_axis_highEdge_y);
        betaOutRHmax = -alpha_OutUp_lowEdge + deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], tl_axis_lowEdge_x, tl_axis_lowEdge_y);
    }

    //beta computation
    float drt_tl_axis = sqrtf(tl_axis_x * tl_axis_x + tl_axis_y * tl_axis_y);
    //float drt_tl_lowEdge = sqrtf(tl_axis_lowEdge_x * tl_axis_lowEdge_x + tl_axis_lowEdge_y * tl_axis_lowEdge_y);
    //float drt_tl_highEdge = sqrtf(tl_axis_highEdge_x * tl_axis_highEdge_x + tl_axis_highEdge_y * tl_axis_highEdge_y);
//no betaIn cut for the pixels
    const float rt_InSeg = sqrtf((x_InUp - x_InLo) * (x_InUp - x_InLo) + (y_InUp - y_InLo) * (y_InUp - y_InLo));

    float betaAv = 0.5f * (betaIn + betaOut);
    pt_beta = ptIn;

    int lIn = 0;
    int lOut = isEC_lastLayer ? 11 : 5;
    float sdOut_dr = sqrtf((x_OutUp - x_OutLo) * (x_OutUp - x_OutLo) + (y_OutUp - y_OutLo) * (y_OutUp - y_OutLo));
    float sdOut_d = rt_OutUp - rt_OutLo;

    //const float diffDr = fabsf(rt_InSeg - sdOut_dr) / fabsf(rt_InSeg + sdOut_dr);

    runDeltaBetaIterationspT3(betaIn, betaOut, betaAv, pt_beta, rt_InSeg, sdOut_dr, drt_tl_axis, lIn);

    const float betaInMMSF = (fabsf(betaInRHmin + betaInRHmax) > 0) ? (2.f * betaIn / fabsf(betaInRHmin + betaInRHmax)) : 0.; //mean value of min,max is the old betaIn
    const float betaOutMMSF = (fabsf(betaOutRHmin + betaOutRHmax) > 0) ? (2.f * betaOut / fabsf(betaOutRHmin + betaOutRHmax)) : 0.;
    betaInRHmin *= betaInMMSF;
    betaInRHmax *= betaInMMSF;
    betaOutRHmin *= betaOutMMSF;
    betaOutRHmax *= betaOutMMSF;

    const float dBetaMuls = sdlThetaMulsF * 4.f / fminf(fabsf(pt_beta), SDL::pt_betaMax); //need to confirm the range-out value of 7 GeV

    const float alphaInAbsReg =  fmaxf(fabsf(alpha_InLo), asinf(fminf(rt_InUp * k2Rinv1GeVf / 3.0f, sinAlphaMax)));
    const float alphaOutAbsReg = fmaxf(fabsf(alpha_OutLo), asinf(fminf(rt_OutLo * k2Rinv1GeVf / 3.0f, sinAlphaMax)));
    const float dBetaInLum = lIn < 11 ? 0.0f : fabsf(alphaInAbsReg*deltaZLum / z_InUp);
    const float dBetaOutLum = lOut < 11 ? 0.0f : fabsf(alphaOutAbsReg*deltaZLum / z_OutLo);
    const float dBetaLum2 = (dBetaInLum + dBetaOutLum) * (dBetaInLum + dBetaOutLum);

    const float sinDPhi = sinf(dPhi);
    const float dBetaRIn2 = 0; // TODO-RH

    float dBetaROut = 0;
    if(isEC_lastLayer)
    {
        dBetaROut = (sqrtf(mdsInGPU.anchorHighEdgeX[fourthMDIndex] * mdsInGPU.anchorHighEdgeX[fourthMDIndex] + mdsInGPU.anchorHighEdgeY[fourthMDIndex] * mdsInGPU.anchorHighEdgeY[fourthMDIndex]) - sqrtf(mdsInGPU.anchorLowEdgeX[fourthMDIndex] * mdsInGPU.anchorLowEdgeX[fourthMDIndex] + mdsInGPU.anchorLowEdgeY[fourthMDIndex] * mdsInGPU.anchorLowEdgeY[fourthMDIndex])) * sinDPhi / drt_tl_axis;
    }

    const float dBetaROut2 =  dBetaROut * dBetaROut;

    betaOutCut = asinf(fminf(drt_tl_axis*k2Rinv1GeVf / ptCut, sinAlphaMax)) //FIXME: need faster version
        + (0.02f / sdOut_d) + sqrtf(dBetaLum2 + dBetaMuls*dBetaMuls);

    //Cut #6: The real beta cut
    pass =  pass and (fabsf(betaOut) < betaOutCut);
    if(not pass) return pass;

   // const float pt_betaOut = drt_tl_axis * k2Rinv1GeVf / sin(betaOut);
    float drt_InSeg = rt_InUp - rt_InLo;

    const float dBetaRes = 0.02f / fminf(sdOut_d, drt_InSeg);
    const float dBetaCut2 = (dBetaRes*dBetaRes * 2.0f + dBetaMuls * dBetaMuls + dBetaLum2 + dBetaRIn2 + dBetaROut2
            + 0.25f * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)) * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)));
    float dBeta = betaIn - betaOut;
#ifdef CUT_VALUE_DEBUG
    deltaBetaCut = sqrtf(dBetaCut2);
#endif

    pass =  pass and (dBeta * dBeta <= dBetaCut2);
    return pass;
}

ALPAKA_FN_ACC ALPAKA_FN_INLINE bool SDL::runPixelTrackletDefaultAlgopT3(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, uint16_t& pixelLowerModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& deltaPhi, float& betaIn, float& betaOut, float& pt_beta, float& zLo, float& zHi, float& rtLo, float& rtHi, float& zLoPointed, float& zHiPointed, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ)
{
    zLo = -999;
    zHi = -999;
    rtLo = -999;
    rtHi = -999;
    zLoPointed = -999;
    zHiPointed = -999;
    kZ = -999;
    betaInCut = -999;

    short outerInnerLowerModuleSubdet = modulesInGPU.subdets[outerInnerLowerModuleIndex];
    short outerOuterLowerModuleSubdet = modulesInGPU.subdets[outerOuterLowerModuleIndex];

    unsigned int firstMDIndex = segmentsInGPU.mdIndices[2 * innerSegmentIndex];
    unsigned int secondMDIndex = segmentsInGPU.mdIndices[2 * innerSegmentIndex + 1];

    unsigned int thirdMDIndex = segmentsInGPU.mdIndices[2 * outerSegmentIndex];
    unsigned int fourthMDIndex = segmentsInGPU.mdIndices[2 * outerSegmentIndex + 1];

    if(outerInnerLowerModuleSubdet == SDL::Barrel and (outerOuterLowerModuleSubdet == SDL::Barrel or outerOuterLowerModuleSubdet == SDL::Endcap))
    {
      return runTripletDefaultAlgoPPBB(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, pixelLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, zHi, zLoPointed, zHiPointed, sdlCut, betaOutCut, deltaBetaCut);
    }
    else if(outerInnerLowerModuleSubdet == SDL::Endcap and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
      return runTripletDefaultAlgoPPEE(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, pixelLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, rtLo, rtHi, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    }
    return false;

}
ALPAKA_FN_ACC bool SDL::checkIntervalOverlappT3(const float& firstMin, const float& firstMax, const float& secondMin, const float& secondMax)
{
    return ((firstMin <= secondMin) & (secondMin < firstMax)) |  ((secondMin < firstMin) & (firstMin < secondMax));
}


SDL::pixelQuintuplets::pixelQuintuplets()
{
    pixelIndices = nullptr;
    T5Indices = nullptr;
    nPixelQuintuplets = nullptr;
    totOccupancyPixelQuintuplets = nullptr;
    isDup = nullptr;
    score = nullptr;
    pixelRadius = nullptr;
    quintupletRadius = nullptr;
    centerX = nullptr;
    centerY = nullptr;
    logicalLayers = nullptr;
    hitIndices = nullptr;
    lowerModuleIndices = nullptr;
}

SDL::pixelQuintuplets::~pixelQuintuplets()
{
}

void SDL::pixelQuintuplets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,pixelIndices);
    cms::cuda::free_device(dev,T5Indices);
    cms::cuda::free_device(dev,nPixelQuintuplets);
    cms::cuda::free_device(dev,totOccupancyPixelQuintuplets);
    cms::cuda::free_device(dev,isDup);
    cms::cuda::free_device(dev,score);
    cms::cuda::free_device(dev,eta);
    cms::cuda::free_device(dev,phi);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, centerX);
    cms::cuda::free_device(dev, centerY);
    cms::cuda::free_device(dev, pixelRadius);
    cms::cuda::free_device(dev, quintupletRadius);
    cms::cuda::free_device(dev, rzChiSquared);
    cms::cuda::free_device(dev, rPhiChiSquared);
    cms::cuda::free_device(dev, rPhiChiSquaredInwards);
}
void SDL::pixelQuintuplets::freeMemory(hipStream_t stream)
{
    hipFree(pixelIndices);
    hipFree(T5Indices);
    hipFree(nPixelQuintuplets);
    hipFree(totOccupancyPixelQuintuplets);
    hipFree(isDup);
    hipFree(score);
    hipFree(eta);
    hipFree(phi);

    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(lowerModuleIndices);
    hipFree(pixelRadius);
    hipFree(quintupletRadius);
    hipFree(centerX);
    hipFree(centerY);
    hipFree(rzChiSquared);
    hipFree(rPhiChiSquared);
    hipFree(rPhiChiSquaredInwards);
    hipStreamSynchronize(stream);
}

void SDL::pixelQuintuplets::resetMemory(unsigned int maxPixelQuintuplets,hipStream_t stream)
{
    hipMemsetAsync(pixelIndices,0, maxPixelQuintuplets * sizeof(unsigned int),stream);
    hipMemsetAsync(T5Indices,0, maxPixelQuintuplets * sizeof(unsigned int),stream);
    hipMemsetAsync(nPixelQuintuplets,0, sizeof(unsigned int),stream);
    hipMemsetAsync(totOccupancyPixelQuintuplets,0, sizeof(unsigned int),stream);
    hipMemsetAsync(isDup,0, maxPixelQuintuplets * sizeof(bool),stream);
    hipMemsetAsync(score,0, maxPixelQuintuplets * sizeof(FPX),stream);
    hipMemsetAsync(eta , 0, maxPixelQuintuplets * sizeof(FPX),stream);
    hipMemsetAsync(phi , 0, maxPixelQuintuplets * sizeof(FPX),stream);
}

void SDL::createPixelQuintupletsInExplicitMemory(struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int maxPixelQuintuplets,hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    pixelQuintupletsInGPU.pixelIndices        = (unsigned int*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.T5Indices           = (unsigned int*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.nPixelQuintuplets   = (unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.totOccupancyPixelQuintuplets   = (unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.isDup               = (bool*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(bool),stream);
    pixelQuintupletsInGPU.score               = (FPX*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(FPX),stream);
    pixelQuintupletsInGPU.eta                 = (FPX*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(FPX),stream);
    pixelQuintupletsInGPU.phi                 = (FPX*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(FPX),stream);
    pixelQuintupletsInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * 14 * sizeof(unsigned int), stream);
    pixelQuintupletsInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * 7 * sizeof(uint8_t), stream);
    pixelQuintupletsInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * 7 * sizeof(uint16_t), stream);
    pixelQuintupletsInGPU.centerX          = (FPX*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(FPX), stream);
    pixelQuintupletsInGPU.centerY          = (FPX*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(FPX), stream);
    pixelQuintupletsInGPU.pixelRadius      = (FPX*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(FPX), stream);
    pixelQuintupletsInGPU.quintupletRadius = (FPX*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(FPX), stream);
     pixelQuintupletsInGPU.rzChiSquared          = (float*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(float), stream);
    pixelQuintupletsInGPU.rPhiChiSquared      = (float*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(float), stream);
    pixelQuintupletsInGPU.rPhiChiSquaredInwards = (float*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(float), stream);
   
#else
    hipMalloc(&pixelQuintupletsInGPU.pixelIndices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.T5Indices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.nPixelQuintuplets, sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.totOccupancyPixelQuintuplets, sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.isDup, maxPixelQuintuplets * sizeof(bool));
    hipMalloc(&pixelQuintupletsInGPU.score, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.eta  , maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.phi  , maxPixelQuintuplets * sizeof(FPX));

    hipMalloc(&pixelQuintupletsInGPU.logicalLayers, maxPixelQuintuplets * 7 *sizeof(uint8_t));
    hipMalloc(&pixelQuintupletsInGPU.hitIndices, maxPixelQuintuplets * 14 * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.lowerModuleIndices, maxPixelQuintuplets * 7 * sizeof(uint16_t));
    hipMalloc(&pixelQuintupletsInGPU.pixelRadius, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.quintupletRadius, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.centerX, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.centerY, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.rzChiSquared, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.rPhiChiSquared, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.rPhiChiSquaredInwards, maxPixelQuintuplets * sizeof(unsigned int));
#endif
    hipMemsetAsync(pixelQuintupletsInGPU.nPixelQuintuplets, 0, sizeof(unsigned int),stream);
    hipMemsetAsync(pixelQuintupletsInGPU.totOccupancyPixelQuintuplets, 0, sizeof(unsigned int),stream);
  hipStreamSynchronize(stream);
}

ALPAKA_FN_ACC void SDL::addPixelQuintupletToMemory(struct modules& modulesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct quintuplets& quintupletsInGPU, struct pixelQuintuplets& pixelQuintupletsInGPU, unsigned int pixelIndex, unsigned int T5Index, unsigned int pixelQuintupletIndex, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards, float score, float eta, float phi, float& pixelRadius, float& quintupletRadius,
        float& centerX, float& centerY)
{
    pixelQuintupletsInGPU.pixelIndices[pixelQuintupletIndex] = pixelIndex;
    pixelQuintupletsInGPU.T5Indices[pixelQuintupletIndex] = T5Index;
    pixelQuintupletsInGPU.isDup[pixelQuintupletIndex] = 0;
    pixelQuintupletsInGPU.score[pixelQuintupletIndex] = __F2H(score);
    pixelQuintupletsInGPU.eta[pixelQuintupletIndex]   = __F2H(eta);
    pixelQuintupletsInGPU.phi[pixelQuintupletIndex]   = __F2H(phi);

    pixelQuintupletsInGPU.pixelRadius[pixelQuintupletIndex] = __F2H(pixelRadius);
    pixelQuintupletsInGPU.quintupletRadius[pixelQuintupletIndex] = __F2H(quintupletRadius);
    pixelQuintupletsInGPU.centerX[pixelQuintupletIndex] = __F2H(centerX);
    pixelQuintupletsInGPU.centerY[pixelQuintupletIndex] = __F2H(centerY);

    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex] = 0;
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 1] = 0;
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 2] = quintupletsInGPU.logicalLayers[T5Index * 5];
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 3] = quintupletsInGPU.logicalLayers[T5Index * 5 + 1];
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 4] = quintupletsInGPU.logicalLayers[T5Index * 5 + 2];
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 5] = quintupletsInGPU.logicalLayers[T5Index * 5 + 3];
    pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex + 6] = quintupletsInGPU.logicalLayers[T5Index * 5 + 4];

    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex] = segmentsInGPU.innerLowerModuleIndices[pixelIndex];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 1] = segmentsInGPU.outerLowerModuleIndices[pixelIndex];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 2] = quintupletsInGPU.lowerModuleIndices[T5Index * 5];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 3] = quintupletsInGPU.lowerModuleIndices[T5Index * 5 + 1];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 4] = quintupletsInGPU.lowerModuleIndices[T5Index * 5 + 2];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 5] = quintupletsInGPU.lowerModuleIndices[T5Index * 5 + 3];
    pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex + 6] = quintupletsInGPU.lowerModuleIndices[T5Index * 5 + 4];

    unsigned int pixelInnerMD = segmentsInGPU.mdIndices[2 * pixelIndex];
    unsigned int pixelOuterMD = segmentsInGPU.mdIndices[2 * pixelIndex + 1];

    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex] = mdsInGPU.anchorHitIndices[pixelInnerMD];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 1] = mdsInGPU.outerHitIndices[pixelInnerMD];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 2] = mdsInGPU.anchorHitIndices[pixelOuterMD];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 3] = mdsInGPU.outerHitIndices[pixelOuterMD];

    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 4] = quintupletsInGPU.hitIndices[10 * T5Index];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 5] = quintupletsInGPU.hitIndices[10 * T5Index + 1];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 6] = quintupletsInGPU.hitIndices[10 * T5Index + 2];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 7] = quintupletsInGPU.hitIndices[10 * T5Index + 3];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 8] = quintupletsInGPU.hitIndices[10 * T5Index + 4];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 9] = quintupletsInGPU.hitIndices[10 * T5Index + 5];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 10] = quintupletsInGPU.hitIndices[10 * T5Index + 6];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 11] = quintupletsInGPU.hitIndices[10 * T5Index + 7];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 12] = quintupletsInGPU.hitIndices[10 * T5Index + 8];
    pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex + 13] = quintupletsInGPU.hitIndices[10 * T5Index + 9];
        
    pixelQuintupletsInGPU.rzChiSquared[pixelQuintupletIndex] = rzChiSquared;
    pixelQuintupletsInGPU.rPhiChiSquared[pixelQuintupletIndex] = rPhiChiSquared;
    pixelQuintupletsInGPU.rPhiChiSquaredInwards[pixelQuintupletIndex] = rPhiChiSquaredInwards;
}

ALPAKA_FN_ACC bool SDL::runPixelQuintupletDefaultAlgo(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, struct quintuplets& quintupletsInGPU, unsigned int& pixelSegmentIndex, unsigned int& quintupletIndex, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards, float& pixelRadius, float& quintupletRadius, float& centerX, float& centerY, unsigned int pixelSegmentArrayIndex)
{
    bool pass = true;
    
//    unsigned int pixelModuleIndex = *modulesInGPU.nLowerModules;//segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];

    unsigned int T5InnerT3Index = quintupletsInGPU.tripletIndices[2 * quintupletIndex];
    unsigned int T5OuterT3Index = quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1];

    float pixelRadiusTemp, pixelRadiusError, tripletRadius, rPhiChiSquaredTemp, rzChiSquaredTemp, rPhiChiSquaredInwardsTemp, centerXTemp, centerYTemp;

    pass = pass and runPixelTripletDefaultAlgo(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, pixelSegmentIndex, T5InnerT3Index, pixelRadiusTemp, pixelRadiusError, tripletRadius, centerXTemp, centerYTemp, rzChiSquaredTemp, rPhiChiSquaredTemp, rPhiChiSquaredInwardsTemp, false);
    if(not pass) return false;

    unsigned int firstSegmentIndex = tripletsInGPU.segmentIndices[2 * T5InnerT3Index];
    unsigned int secondSegmentIndex = tripletsInGPU.segmentIndices[2 * T5InnerT3Index + 1];
    unsigned int thirdSegmentIndex = tripletsInGPU.segmentIndices[2 * T5OuterT3Index];
    unsigned int fourthSegmentIndex = tripletsInGPU.segmentIndices[2 * T5OuterT3Index + 1];

    unsigned int pixelInnerMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex];
    unsigned int pixelOuterMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex + 1];
    unsigned int firstMDIndex = segmentsInGPU.mdIndices[2 * firstSegmentIndex];
    unsigned int secondMDIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex];
    unsigned int thirdMDIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex + 1];
    unsigned int fourthMDIndex = segmentsInGPU.mdIndices[2 * thirdSegmentIndex + 1];
    unsigned int fifthMDIndex = segmentsInGPU.mdIndices[2 * fourthSegmentIndex + 1];

    uint16_t lowerModuleIndex1 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex];
    uint16_t lowerModuleIndex2 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 1];
    uint16_t lowerModuleIndex3 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 2];
    uint16_t lowerModuleIndex4 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 3];
    uint16_t lowerModuleIndex5 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 4];

    uint16_t lowerModuleIndices[5] = {lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5};
    
    float zPix[2] = {mdsInGPU.anchorZ[pixelInnerMDIndex], mdsInGPU.anchorZ[pixelOuterMDIndex]};
    float rtPix[2] = {mdsInGPU.anchorRt[pixelInnerMDIndex], mdsInGPU.anchorRt[pixelOuterMDIndex]};
    float zs[5] = {mdsInGPU.anchorZ[firstMDIndex], mdsInGPU.anchorZ[secondMDIndex], mdsInGPU.anchorZ[thirdMDIndex], mdsInGPU.anchorZ[fourthMDIndex], mdsInGPU.anchorZ[fifthMDIndex]};
    float rts[5] = {mdsInGPU.anchorRt[firstMDIndex], mdsInGPU.anchorRt[secondMDIndex], mdsInGPU.anchorRt[thirdMDIndex], mdsInGPU.anchorRt[fourthMDIndex], mdsInGPU.anchorRt[fifthMDIndex]};

    rzChiSquared = computePT5RZChiSquared(modulesInGPU, lowerModuleIndices, rtPix, zPix, rts, zs);

    if(pixelRadius < 5.0f * kR1GeVf)
    {
        pass = pass and passPT5RZChiSquaredCuts(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rzChiSquared);
        if(not pass) return pass;
    }

    //outer T5
    float xs[5] = {mdsInGPU.anchorX[firstMDIndex], mdsInGPU.anchorX[secondMDIndex], mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorX[fifthMDIndex]};
    float ys[5] = {mdsInGPU.anchorY[firstMDIndex], mdsInGPU.anchorY[secondMDIndex], mdsInGPU.anchorY[thirdMDIndex], mdsInGPU.anchorY[fourthMDIndex], mdsInGPU.anchorY[fifthMDIndex]};

    //get the appropriate radii and centers
    centerX  = segmentsInGPU.circleCenterX[1];
    centerX  = segmentsInGPU.circleCenterX[pixelSegmentArrayIndex];
    centerY = segmentsInGPU.circleCenterY[pixelSegmentArrayIndex];
    pixelRadius = segmentsInGPU.circleRadius[pixelSegmentArrayIndex];

    float T5CenterX = quintupletsInGPU.regressionG[quintupletIndex];
    float T5CenterY = quintupletsInGPU.regressionF[quintupletIndex];
    quintupletRadius = quintupletsInGPU.regressionRadius[quintupletIndex];

    rPhiChiSquared = computePT5RPhiChiSquared(modulesInGPU, lowerModuleIndices, centerX, centerY, pixelRadius, xs, ys);

    if(pixelRadius < 5.0f * kR1GeVf)
    {
        pass = pass and passPT5RPhiChiSquaredCuts(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rPhiChiSquared);
        if(not pass) return pass;
    }

    float xPix[] = {mdsInGPU.anchorX[pixelInnerMDIndex], mdsInGPU.anchorX[pixelOuterMDIndex]};
    float yPix[] = {mdsInGPU.anchorY[pixelInnerMDIndex], mdsInGPU.anchorY[pixelOuterMDIndex]};
    rPhiChiSquaredInwards = computePT5RPhiChiSquaredInwards(modulesInGPU, T5CenterX, T5CenterY, quintupletRadius, xPix, yPix);

    if(quintupletsInGPU.regressionRadius[quintupletIndex] < 5.0f * kR1GeVf)
    {
        pass = pass and passPT5RPhiChiSquaredInwardsCuts(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rPhiChiSquaredInwards); 
        if(not pass) return pass;
    }
    //trusting the T5 regression center to also be a good estimate..
    centerX = (centerX + T5CenterX)/2;
    centerY = (centerY + T5CenterY)/2;

    //other cuts will be filled here!
    return pass;
}

ALPAKA_FN_ACC bool SDL::passPT5RPhiChiSquaredCuts(struct modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float rPhiChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 48.921f;
        }
        else if(layer4 == 4 and layer5 == 12)
        {
            return rPhiChiSquared < 97.948f;
        }
        else if(layer4 == 4 and layer5 == 5)
        {
            return rPhiChiSquared < 129.3f;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return rPhiChiSquared < 56.21f;
        }
        else if(layer4 == 7 and layer5 == 8)
        {
            return rPhiChiSquared < 74.198f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rPhiChiSquared < 21.265f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 37.058f;
        }
        else if(layer4 == 8 and layer5 == 9)
        {
            return rPhiChiSquared < 42.578f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 32.253f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 37.058f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 97.947f;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return rPhiChiSquared < 129.3f;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return rPhiChiSquared < 170.68f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {   
            return rPhiChiSquared < 48.92f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 74.2f;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 14 and layer5 == 15)
        {
            return rPhiChiSquared < 42.58f;
        }
        else if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 37.06f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 48.92f;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rPhiChiSquared < 85.25f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return rPhiChiSquared < 42.58f;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return rPhiChiSquared < 37.06f;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return rPhiChiSquared < 37.06f;
        }
    }
    return true;
}

ALPAKA_FN_ACC bool SDL::passPT5RPhiChiSquaredInwardsCuts(struct modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float rPhiChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 451.141f;
        }
        else if(layer4 == 4 and layer5 == 12)
        {
            return rPhiChiSquared < 786.173f;
        }
        else if(layer4 == 4 and layer5 == 5)
        {
            return rPhiChiSquared < 595.545f;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return rPhiChiSquared < 581.339f;
        }
        else if(layer4 == 7 and layer5 == 8)
        {
            return rPhiChiSquared < 112.537f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rPhiChiSquared < 225.322f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 1192.402f;
        }
        else if(layer4 == 8 and layer5 == 9)
        {
            return rPhiChiSquared < 786.173f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 1037.817f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 1808.536f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 684.253f;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return rPhiChiSquared < 684.253f;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return rPhiChiSquared < 684.253f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {   
            return rPhiChiSquared < 451.141f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 518.34f;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 14 and layer5 == 15)
        {
            return rPhiChiSquared < 2077.92f;
        }
        else if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 74.20f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 1808.536f;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rPhiChiSquared < 786.173f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return rPhiChiSquared < 1574.076f;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return rPhiChiSquared < 5492.11f;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return rPhiChiSquared < 2743.037f;
        }
    }
    return true;
}

ALPAKA_FN_ACC float SDL::computePT5RPhiChiSquared(struct modules& modulesInGPU, uint16_t* lowerModuleIndices, float& g, float& f, float& radius, float* xs, float* ys)
{
    /*
       Compute circle parameters from 3 pixel hits, and then use them to compute the chi squared for the outer hits
    */

    float delta1[5], delta2[5], slopes[5];
    bool isFlat[5];
    float chiSquared = 0;

    computeSigmasForRegression_pT5(modulesInGPU, lowerModuleIndices, delta1, delta2, slopes, isFlat);
    chiSquared = computeChiSquaredpT5(5, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius);

    return chiSquared;
}

ALPAKA_FN_ACC bool SDL::passPT5RZChiSquaredCuts(struct modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& rzChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    
    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rzChiSquared < 451.141f;
        }
        else if(layer4 == 4 and layer5 == 12)
        {
            return rzChiSquared < 392.654f;
        }
        else if(layer4 == 4 and layer5 == 5)
        {
            return rzChiSquared < 225.322f;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return rzChiSquared < 595.546f;
        }
        else if(layer4 == 7 and layer5 == 8)
        {
            return rzChiSquared < 196.111f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rzChiSquared < 297.446f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {   
            return rzChiSquared < 451.141f;
        }
        else if(layer4 == 8 and layer5 == 9)
        {
            return rzChiSquared < 518.339f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return rzChiSquared < 341.75f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rzChiSquared < 341.75f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rzChiSquared < 392.655f;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return rzChiSquared < 341.75f;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return rzChiSquared < 112.537f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer4 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rzChiSquared < 595.545f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rzChiSquared < 74.198f;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 14 and layer5 == 15)
        {
            return rzChiSquared < 518.339f;
        }
        else if(layer4 == 9 and layer5 == 10)
        {
            return rzChiSquared < 8.046f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rzChiSquared < 451.141f;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rzChiSquared < 56.207f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return rzChiSquared < 64.578f;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return rzChiSquared < 85.250f;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return rzChiSquared < 85.250f;
        }
    }
    return true;
}

ALPAKA_FN_ACC float SDL::computePT5RPhiChiSquaredInwards(struct modules& modulesInGPU, float& g, float& f, float& r, float* xPix, float* yPix)
{
    /*Using the computed regression center and radius, compute the chi squared for the pixels*/
    float chiSquared = 0;   
    for(size_t i = 0; i < 2; i++)
    {
        float residual = (xPix[i] - g) * (xPix[i] -g) + (yPix[i] - f) * (yPix[i] - f) - r * r;
        chiSquared += residual * residual;
    }
    chiSquared *= 0.5f;
    return chiSquared;
}


ALPAKA_FN_ACC float SDL::computePT5RZChiSquared(struct modules& modulesInGPU, uint16_t* lowerModuleIndices, float* rtPix, float* zPix, float* rts, float* zs)
{
    //use the two anchor hits of the pixel segment to compute the slope
    //then compute the pseudo chi squared of the five outer hits

    float slope = (zPix[1] - zPix[0]) / (rtPix[1] - rtPix[0]);
    float residual = 0;
    float error = 0;
    //hardcoded array indices!!!
    float RMSE = 0;
    for(size_t i = 0; i < 5; i++)
    {
        uint16_t& lowerModuleIndex = lowerModuleIndices[i];
        const int moduleType = modulesInGPU.moduleType[lowerModuleIndex];
        const int moduleSide = modulesInGPU.sides[lowerModuleIndex];
        const int moduleSubdet = modulesInGPU.subdets[lowerModuleIndex];
 
        residual = (moduleSubdet == SDL::Barrel) ? (zs[i] - zPix[0]) - slope * (rts[i] - rtPix[0]) : (rts[i] - rtPix[0]) - (zs[i] - zPix[0])/slope;
        float& drdz = modulesInGPU.drdzs[lowerModuleIndex]; 
        //PS Modules
        if(moduleType == 0)
        {
            error = 0.15f;
        }
        else //2S modules
        {
            error = 5.0f;
        }

        //special dispensation to tilted PS modules!
        if(moduleType == 0 and moduleSubdet == SDL::Barrel and moduleSide != Center)
        {
            //error *= 1.f/sqrtf(1.f + drdz * drdz);
            error /= sqrtf(1.f + drdz * drdz);
        }
        RMSE += (residual * residual)/(error * error);
    }

    RMSE = sqrtf(0.2f * RMSE);
    return RMSE;
}
ALPAKA_FN_ACC void SDL::computeSigmasForRegression_pT5(SDL::modules& modulesInGPU, const uint16_t* lowerModuleIndices, float* delta1, float* delta2, float* slopes, bool* isFlat, int nPoints, bool anchorHits)
{
   /*bool anchorHits required to deal with a weird edge case wherein
     the hits ultimately used in the regression are anchor hits, but the
     lower modules need not all be Pixel Modules (in case of PS). Similarly,
     when we compute the chi squared for the non-anchor hits, the "partner module"
     need not always be a PS strip module, but all non-anchor hits sit on strip
     modules.
    */
    ModuleType moduleType;
    short moduleSubdet, moduleSide;
    float inv1 = 0.01f/0.009f;
    float inv2 = 0.15f/0.009f;
    float inv3 = 2.4f/0.009f;
    for(size_t i=0; i<nPoints; i++)
    {
        moduleType = modulesInGPU.moduleType[lowerModuleIndices[i]];
        moduleSubdet = modulesInGPU.subdets[lowerModuleIndices[i]];
        moduleSide = modulesInGPU.sides[lowerModuleIndices[i]];
        float& drdz = modulesInGPU.drdzs[lowerModuleIndices[i]];
        slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
        //category 1 - barrel PS flat
        if(moduleSubdet == Barrel and moduleType == PS and moduleSide == Center)
        {
            //delta1[i] = 0.01;
            //delta2[i] = 0.01;
            delta1[i] = inv1;//1.1111f;//0.01;
            delta2[i] = inv1;//1.1111f;//0.01;
            slopes[i] = -999.f;
            isFlat[i] = true;
        }

        //category 2 - barrel 2S
        else if(moduleSubdet == Barrel and moduleType == TwoS)
        {
            //delta1[i] = 0.009;
            //delta2[i] = 0.009;
            delta1[i] = 1.f;//0.009;
            delta2[i] = 1.f;//0.009;
            slopes[i] = -999.f;
            isFlat[i] = true;
        }

        //category 3 - barrel PS tilted
        else if(moduleSubdet == Barrel and moduleType == PS and moduleSide != Center)
        {

            //delta1[i] = 0.01;
            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            if(anchorHits)
            {
                //delta2[i] = (0.15f * drdz/sqrtf(1 + drdz * drdz));
                delta2[i] = (inv2 * drdz/sqrtf(1 + drdz * drdz));
                //delta2[i] = (inv2 * drdz*rsqrt(1 + drdz * drdz));
            }
            else
            {
                //delta2[i] = (2.4f * drdz/sqrtf(1 + drdz * drdz));
                delta2[i] = (inv3 * drdz/sqrtf(1 + drdz * drdz));
                //delta2[i] = (inv3 * drdz*rsqrt(1 + drdz * drdz));
            }
        }
        //category 4 - endcap PS
        else if(moduleSubdet == Endcap and moduleType == PS)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            //delta1[i] = 0.01;
            isFlat[i] = false;

            /*despite the type of the module layer of the lower module index,
            all anchor hits are on the pixel side and all non-anchor hits are
            on the strip side!*/
            if(anchorHits)
            {
                delta2[i] = inv2;//16.6666f;//0.15f;
                //delta2[i] = 0.15f;
            }
            else
            {
                //delta2[i] = 2.4f;
                delta2[i] = inv3;//266.666f;//2.4f;
            }
        }

        //category 5 - endcap 2S
        else if(moduleSubdet == Endcap and moduleType == TwoS)
        {
            //delta1[i] = 0.009;
            //delta2[i] = 5.f;
            delta1[i] = 1.f;//0.009;
            delta2[i] = 500.f*inv1;//555.5555f;//5.f;
            isFlat[i] = false;
        }
#ifdef Warnings
        else
        {
            printf("ERROR!!!!! I SHOULDN'T BE HERE!!!! subdet = %d, type = %d, side = %d\n", moduleSubdet, moduleType, moduleSide);
        }
#endif
    }
    //divide everyone by the smallest possible values of delta1 and delta2
//    for(size_t i = 0; i < 5; i++)
//    {
//        delta1[i] /= 0.009;
//        delta2[i] /= 0.009;
//    }
}

__global__ void SDL::createPixelQuintupletsInGPUFromMapv2(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int* connectedPixelSize, unsigned int* connectedPixelIndex, unsigned int nPixelSegments, struct SDL::objectRanges& rangesInGPU)
{
    int blockxSize = blockDim.x*gridDim.x;
    int blockySize = blockDim.y*gridDim.y;
    for(int i_pLS = blockIdx.y * blockDim.y + threadIdx.y; i_pLS< nPixelSegments; i_pLS += blockySize)
    {
          auto iLSModule_max = connectedPixelIndex[i_pLS] + connectedPixelSize[i_pLS];
      for( int iLSModule = connectedPixelIndex[i_pLS] + blockIdx.z; iLSModule<iLSModule_max; iLSModule += gridDim.z)
      {

        //these are actual module indices
        uint16_t quintupletLowerModuleIndex = modulesInGPU.connectedPixels[iLSModule];
        if(quintupletLowerModuleIndex >= *modulesInGPU.nLowerModules) continue;
        if( modulesInGPU.moduleType[quintupletLowerModuleIndex] == SDL::TwoS) continue;
        uint16_t pixelModuleIndex = *modulesInGPU.nLowerModules;
        if(segmentsInGPU.isDup[i_pLS]) continue;
        unsigned int nOuterQuintuplets = quintupletsInGPU.nQuintuplets[quintupletLowerModuleIndex];

        if(nOuterQuintuplets == 0) continue;

        unsigned int pixelSegmentIndex = rangesInGPU.segmentModuleIndices[pixelModuleIndex] + i_pLS;

        //fetch the quintuplet
        for(unsigned int outerQuintupletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x; outerQuintupletArrayIndex< nOuterQuintuplets; outerQuintupletArrayIndex +=blockxSize)
        {

            unsigned int quintupletIndex = rangesInGPU.quintupletModuleIndices[quintupletLowerModuleIndex] + outerQuintupletArrayIndex;

            if(quintupletsInGPU.isDup[quintupletIndex]) continue;

            float rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards, pixelRadius, quintupletRadius, centerX, centerY;

            bool success = runPixelQuintupletDefaultAlgo(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, quintupletsInGPU, pixelSegmentIndex, quintupletIndex, rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards, pixelRadius, quintupletRadius, centerX, centerY,static_cast<unsigned int>(i_pLS));
            if(success)
            {
                unsigned int totOccupancyPixelQuintuplets = atomicAdd(pixelQuintupletsInGPU.totOccupancyPixelQuintuplets, 1);
                if(totOccupancyPixelQuintuplets >= N_MAX_PIXEL_QUINTUPLETS)
                {
#ifdef Warnings
                    printf("Pixel Quintuplet excess alert!\n");
#endif
                }
                else
                {
                    unsigned int pixelQuintupletIndex = atomicAdd(pixelQuintupletsInGPU.nPixelQuintuplets, 1);
                    float eta = __H2F(quintupletsInGPU.eta[quintupletIndex]);
                    float phi = __H2F(quintupletsInGPU.phi[quintupletIndex]);

                    addPixelQuintupletToMemory(modulesInGPU, mdsInGPU, segmentsInGPU, quintupletsInGPU, pixelQuintupletsInGPU, pixelSegmentIndex, quintupletIndex, pixelQuintupletIndex,rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards, rPhiChiSquared, eta, phi, pixelRadius, quintupletRadius, centerX, centerY);

                    tripletsInGPU.partOfPT5[quintupletsInGPU.tripletIndices[2 * quintupletIndex]] = true;
                    tripletsInGPU.partOfPT5[quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1]] = true;
                    segmentsInGPU.partOfPT5[i_pLS] = true;
                    quintupletsInGPU.partOfPT5[quintupletIndex] = true;
                }// tot occupancy
            }//end success
        } //end T5
      }//end iLS
    }// end i_pLS
}// 
ALPAKA_FN_ACC void SDL::runDeltaBetaIterationspT5(float& betaIn, float& betaOut, float& betaAv, float & pt_beta, float sdIn_dr, float sdOut_dr, float dr, float lIn)
{
    if (lIn == 0)
    {
        betaOut += copysign(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaOut);
        return;
    }

    if (betaIn * betaOut > 0.f and (fabsf(pt_beta) < 4.f * SDL::pt_betaMax or (lIn >= 11 and fabsf(pt_beta) < 8.f * SDL::pt_betaMax)))   //and the pt_beta is well-defined; less strict for endcap-endcap
    {

        const float betaInUpd  = betaIn + copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        const float betaOutUpd = betaOut + copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaOut); //FIXME: need a faster version
        betaAv = 0.5f * (betaInUpd + betaOutUpd);

        //1st update
        //pt_beta = dr * k2Rinv1GeVf / sinf(betaAv); //get a better pt estimate
        const float pt_beta_inv = 1.f/fabsf(dr * k2Rinv1GeVf / sinf(betaAv)); //get a better pt estimate

        betaIn  += copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf *pt_beta_inv, SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaOut += copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf *pt_beta_inv, SDL::sinAlphaMax)), betaOut); //FIXME: need a faster version
        //update the av and pt
        betaAv = 0.5f * (betaIn + betaOut);
        //2nd update
        pt_beta = dr * SDL::k2Rinv1GeVf / sinf(betaAv); //get a better pt estimate
    }
    else if (lIn < 11 && fabsf(betaOut) < 0.2f * fabsf(betaIn) && fabsf(pt_beta) < 12.f * SDL::pt_betaMax)   //use betaIn sign as ref
    {

        const float pt_betaIn = dr * k2Rinv1GeVf / sinf(betaIn);

        const float betaInUpd  = betaIn + copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_betaIn), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        const float betaOutUpd = betaOut + copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_betaIn), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaAv = (fabsf(betaOut) > 0.2f * fabsf(betaIn)) ? (0.5f * (betaInUpd + betaOutUpd)) : betaInUpd;

        //1st update
        pt_beta = dr * SDL::k2Rinv1GeVf / sin(betaAv); //get a better pt estimate
        betaIn  += copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaOut += copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        //update the av and pt
        betaAv = 0.5f * (betaIn + betaOut);
        //2nd update
        pt_beta = dr * SDL::k2Rinv1GeVf / sin(betaAv); //get a better pt estimate

    }
}
ALPAKA_FN_ACC bool SDL::checkIntervalOverlappT5(const float& firstMin, const float& firstMax, const float& secondMin, const float& secondMax)
{
    return ((firstMin <= secondMin) & (secondMin < firstMax)) |  ((secondMin < firstMin) & (firstMin < secondMax));
}

ALPAKA_FN_ACC float SDL::computeChiSquaredpT5(int nPoints, float* xs, float* ys, float* delta1, float* delta2, float* slopes, bool* isFlat, float g, float f, float radius)
{
    // given values of (g, f, radius) and a set of points (and its uncertainties)
    //compute chi squared
    float c = g*g + f*f - radius*radius;
    float chiSquared = 0.f;
    float absArctanSlope, angleM, xPrime, yPrime, sigma;
    for(size_t i = 0; i < nPoints; i++)
    {
        absArctanSlope = ((slopes[i] != SDL::SDL_INF) ? fabs(atanf(slopes[i])) : 0.5f*float(M_PI)); // Since C++ can't represent infinity, SDL_INF = 123456789 was used to represent infinity in the data table
        if(xs[i] > 0 and ys[i] > 0)
        {
            angleM = 0.5f*float(M_PI) - absArctanSlope;
        }
        else if(xs[i] < 0 and ys[i] > 0)
        {
            angleM = absArctanSlope + 0.5f*float(M_PI);
        }
        else if(xs[i] < 0 and ys[i] < 0)
        {
            angleM = -(absArctanSlope + 0.5f*float(M_PI));
        }
        else if(xs[i] > 0 and ys[i] < 0)
        {
            angleM = -(0.5f*float(M_PI) - absArctanSlope);
        }

        if(not isFlat[i])
        {
            xPrime = xs[i] * cosf(angleM) + ys[i] * sinf(angleM);
            yPrime = ys[i] * cosf(angleM) - xs[i] * sinf(angleM);
        }
        else
        {
            xPrime = xs[i];
            yPrime = ys[i];
        }
        sigma = 2 * sqrtf((xPrime * delta1[i]) * (xPrime * delta1[i]) + (yPrime * delta2[i]) * (yPrime * delta2[i]));
        chiSquared +=  (xs[i] * xs[i] + ys[i] * ys[i] - 2 * g * xs[i] - 2 * f * ys[i] + c) * (xs[i] * xs[i] + ys[i] * ys[i] - 2 * g * xs[i] - 2 * f * ys[i] + c) / (sigma * sigma);
    }
    return chiSquared;
}

ALPAKA_FN_ACC ALPAKA_FN_INLINE bool SDL::runpT5DefaultAlgoPPBB(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU ,struct segments& segmentsInGPU, uint16_t& pixelModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int thirdMDIndex, unsigned int& fourthMDIndex, float& z_OutLo, float& rt_OutLo, float& dPhiPos, float& dPhi, float& betaIn,
        float& betaOut, float& pt_beta, float& zLo, float& zHi, float& zLoPointed, float& zHiPointed, float& sdlCut, float& betaOutCut, float& deltaBetaCut) // pixel to BB and BE segments
{
    bool pass = true;

    bool isPS_OutLo = (modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS);

    float rt_InLo = mdsInGPU.anchorRt[firstMDIndex];
    float rt_InUp = mdsInGPU.anchorRt[secondMDIndex];
    rt_OutLo = mdsInGPU.anchorRt[thirdMDIndex];
    float rt_OutUp = mdsInGPU.anchorRt[fourthMDIndex];

    float z_InLo = mdsInGPU.anchorZ[firstMDIndex];
    float z_InUp = mdsInGPU.anchorZ[secondMDIndex];
    z_OutLo = mdsInGPU.anchorZ[thirdMDIndex];
    float z_OutUp = mdsInGPU.anchorZ[fourthMDIndex];

    float x_InLo = mdsInGPU.anchorX[firstMDIndex];
    float x_InUp = mdsInGPU.anchorX[secondMDIndex];
    float x_OutLo = mdsInGPU.anchorX[thirdMDIndex];
    float x_OutUp = mdsInGPU.anchorX[fourthMDIndex];

    float y_InLo = mdsInGPU.anchorY[firstMDIndex];
    float y_InUp = mdsInGPU.anchorY[secondMDIndex];
    float y_OutLo = mdsInGPU.anchorY[thirdMDIndex];
    float y_OutUp = mdsInGPU.anchorY[fourthMDIndex];

    float& rt_InOut = rt_InUp;
    //float& z_InOut = z_InUp;

    pass = pass and (fabsf(deltaPhi(x_InUp, y_InUp, x_OutLo, y_OutLo)) <= 0.5f * float(M_PI));
    if(not pass) return pass;

    unsigned int pixelSegmentArrayIndex = innerSegmentIndex - rangesInGPU.segmentModuleIndices[pixelModuleIndex];
    float ptIn = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
    float ptSLo = ptIn;
    float px = segmentsInGPU.px[pixelSegmentArrayIndex];
    float py = segmentsInGPU.py[pixelSegmentArrayIndex];
    float pz = segmentsInGPU.pz[pixelSegmentArrayIndex];
    float ptErr = segmentsInGPU.ptErr[pixelSegmentArrayIndex];
    float etaErr = segmentsInGPU.etaErr[pixelSegmentArrayIndex];
    ptSLo = fmaxf(ptCut, ptSLo - 10.0f*fmaxf(ptErr, 0.005f*ptSLo));
    ptSLo = fminf(10.0f, ptSLo);


    float alpha1GeV_OutLo = asinf(fminf(rt_OutLo * k2Rinv1GeVf / ptCut, sinAlphaMax));
    //float rtRatio_OutLoInLo = rt_OutLo / rt_InLo; // Outer segment beginning rt divided by inner segment beginning rt;
    const float rtRatio_OutLoInOut = rt_OutLo / rt_InOut; // Outer segment beginning rt divided by inner segment beginning rt;

    float dzDrtScale = tanf(alpha1GeV_OutLo) / alpha1GeV_OutLo; // The track can bend in r-z plane slightly
    const float zpitch_InLo = 0.05f;
    const float zpitch_InOut = 0.05f;
    float zpitch_OutLo = (isPS_OutLo ? pixelPSZpitch : strip2SZpitch);
    float zGeom = zpitch_InLo + zpitch_OutLo;
    zHi = z_InUp + (z_InUp + deltaZLum) * (rtRatio_OutLoInOut - 1.f) * (z_InUp < 0.f ? 1.f : dzDrtScale) + (zpitch_InOut + zpitch_OutLo);
    zLo = z_InUp + (z_InUp - deltaZLum) * (rtRatio_OutLoInOut - 1.f) * (z_InUp > 0.f ? 1.f : dzDrtScale) - (zpitch_InOut + zpitch_OutLo); //slope-correction only on outer end

    pass = pass and ((z_OutLo >= zLo) & (z_OutLo <= zHi));
    if(not pass) return pass;

    const float coshEta = sqrtf(ptIn * ptIn + pz * pz) / ptIn;
    // const float drt_OutLo_InLo = (rt_OutLo - rt_InLo);
    const float drt_OutLo_InUp = (rt_OutLo - rt_InUp);
    //const float invRt_InLo = 1.f / rt_InLo;
    //const float r3_InLo = sqrtf(z_InLo * z_InLo + rt_InLo * rt_InLo);
    const float r3_InUp = sqrtf(z_InUp * z_InUp + rt_InUp * rt_InUp);

    float drt_InSeg = rt_InOut - rt_InLo;
    //float dz_InSeg = z_InOut - z_InLo;
    //float dr3_InSeg = sqrtf(rt_InOut * rt_InOut + z_InOut * z_InOut) - sqrtf(rt_InLo * rt_InLo + z_InLo * z_InLo);

    const float sdlThetaMulsF = 0.015f * sqrtf(0.1f + 0.2f * (rt_OutLo - rt_InUp) / 50.f) * sqrtf(r3_InUp / rt_InUp);
    const float sdlMuls = sdlThetaMulsF * 3.f / ptCut * 4.f; // will need a better guess than x4?

    float dzErr = drt_OutLo_InUp*etaErr*coshEta; //FIXME: check with the calc in the endcap
    dzErr *= dzErr;
    dzErr += 0.03f*0.03f; // pixel size x2. ... random for now
    dzErr *= 9.f; //3 sigma
    dzErr += sdlMuls*sdlMuls*drt_OutLo_InUp*drt_OutLo_InUp/3.f*coshEta*coshEta;//sloppy
    dzErr += zGeom*zGeom;
    dzErr = sqrtf(dzErr);

    const float dzDrIn = pz / ptIn;
    const float zWindow = dzErr / drt_InSeg * drt_OutLo_InUp + zGeom;
    const float dzMean = dzDrIn * drt_OutLo_InUp *
        (1.f + drt_OutLo_InUp * drt_OutLo_InUp * 4 * k2Rinv1GeVf * k2Rinv1GeVf / ptIn /
         ptIn / 24.f); // with curved path correction
    // Constructing upper and lower bound
    zLoPointed = z_InUp + dzMean - zWindow;
    zHiPointed = z_InUp + dzMean + zWindow;

    pass =  pass and ((z_OutLo >= zLoPointed) & (z_OutLo <= zHiPointed));
    if(not pass) return pass;

    const float sdlPVoff = 0.1f / rt_OutLo;
    sdlCut = alpha1GeV_OutLo + sqrtf(sdlMuls * sdlMuls + sdlPVoff * sdlPVoff);

    dPhiPos = deltaPhi(x_InUp, y_InUp, x_OutUp, y_OutUp);

    //no dphipos cut
    float midPointX = 0.5f * (x_InLo + x_OutLo);
    float midPointY = 0.5f * (y_InLo + y_OutLo);
    float midPointZ = 0.5f * (z_InLo + z_OutLo);

    float diffX = x_OutLo - x_InLo;
    float diffY = y_OutLo - y_InLo;
    float diffZ = z_OutLo - z_InLo;


    dPhi = deltaPhi(midPointX, midPointY, diffX, diffY);

    pass = pass and (fabsf(dPhi) <= sdlCut);
    if(not pass) return pass;

    //lots of array accesses below this...

    float alpha_InLo  = __H2F(segmentsInGPU.dPhiChanges[innerSegmentIndex]);
    float alpha_OutLo = __H2F(segmentsInGPU.dPhiChanges[outerSegmentIndex]);

    bool isEC_lastLayer = modulesInGPU.subdets[outerOuterLowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS;

    float alpha_OutUp,alpha_OutUp_highEdge,alpha_OutUp_lowEdge;
    alpha_OutUp = deltaPhi(x_OutUp, y_OutUp, x_OutUp - x_OutLo, y_OutUp - y_OutLo);

    alpha_OutUp_highEdge = alpha_OutUp;
    alpha_OutUp_lowEdge = alpha_OutUp;

    float tl_axis_x = x_OutUp - x_InUp;
    float tl_axis_y = y_OutUp - y_InUp;
    float tl_axis_z = z_OutUp - z_InUp;

    float tl_axis_highEdge_x = tl_axis_x;
    float tl_axis_highEdge_y = tl_axis_y;

    float tl_axis_lowEdge_x = tl_axis_x;
    float tl_axis_lowEdge_y = tl_axis_y;

    betaIn = -deltaPhi(px, py, tl_axis_x, tl_axis_y);
    float betaInRHmin = betaIn;
    float betaInRHmax = betaIn;

    betaOut = -alpha_OutUp + deltaPhi(x_OutUp, y_OutUp, tl_axis_x, tl_axis_y);

    float betaOutRHmin = betaOut;
    float betaOutRHmax = betaOut;

    if(isEC_lastLayer)
    {
        alpha_OutUp_highEdge = deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], mdsInGPU.anchorHighEdgeX[fourthMDIndex] - x_OutLo, mdsInGPU.anchorHighEdgeY[fourthMDIndex] - y_OutLo);
        alpha_OutUp_lowEdge = deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], mdsInGPU.anchorLowEdgeX[fourthMDIndex] - x_OutLo, mdsInGPU.anchorLowEdgeY[fourthMDIndex] - y_OutLo);

        tl_axis_highEdge_x = mdsInGPU.anchorHighEdgeX[fourthMDIndex] - x_InUp;
        tl_axis_highEdge_y = mdsInGPU.anchorHighEdgeY[fourthMDIndex] - y_InUp;
        tl_axis_lowEdge_x = mdsInGPU.anchorLowEdgeX[fourthMDIndex] - x_InUp;
        tl_axis_lowEdge_y = mdsInGPU.anchorLowEdgeY[fourthMDIndex] - y_InUp;

        betaOutRHmin = -alpha_OutUp_highEdge + deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], tl_axis_highEdge_x, tl_axis_highEdge_y);
        betaOutRHmax = -alpha_OutUp_lowEdge + deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], tl_axis_lowEdge_x, tl_axis_lowEdge_y);
    }

    //beta computation
    float drt_tl_axis = sqrtf(tl_axis_x * tl_axis_x + tl_axis_y * tl_axis_y);
    //float drt_tl_lowEdge = sqrtf(tl_axis_lowEdge_x * tl_axis_lowEdge_x + tl_axis_lowEdge_y * tl_axis_lowEdge_y);
    //float drt_tl_highEdge = sqrtf(tl_axis_highEdge_x * tl_axis_highEdge_x + tl_axis_highEdge_y * tl_axis_highEdge_y);

    //innerOuterAnchor - innerInnerAnchor
    const float rt_InSeg = sqrtf((x_InUp - x_InLo) * (x_InUp - x_InLo) + (y_InUp - y_InLo) * (y_InUp - y_InLo));

    //no betaIn cut for the pixels
    float betaAv = 0.5f * (betaIn + betaOut);
    pt_beta = ptIn;

    int lIn = 0;
    int lOut = isEC_lastLayer ? 11 : 5;
    float sdOut_dr = sqrtf((x_OutUp - x_OutLo) * (x_OutUp - x_OutLo) + (y_OutUp - y_OutLo) * (y_OutUp - y_OutLo));
    float sdOut_d = rt_OutUp - rt_OutLo;

    //const float diffDr = fabsf(rt_InSeg - sdOut_dr) / fabsf(rt_InSeg + sdOut_dr);

    runDeltaBetaIterationspT5(betaIn, betaOut, betaAv, pt_beta, rt_InSeg, sdOut_dr, drt_tl_axis, lIn);

    const float betaInMMSF = (fabsf(betaInRHmin + betaInRHmax) > 0) ? (2.f * betaIn / fabsf(betaInRHmin + betaInRHmax)) : 0.; //mean value of min,max is the old betaIn
    const float betaOutMMSF = (fabsf(betaOutRHmin + betaOutRHmax) > 0) ? (2.f * betaOut / fabsf(betaOutRHmin + betaOutRHmax)) : 0.;
    betaInRHmin *= betaInMMSF;
    betaInRHmax *= betaInMMSF;
    betaOutRHmin *= betaOutMMSF;
    betaOutRHmax *= betaOutMMSF;

    const float dBetaMuls = sdlThetaMulsF * 4.f / fminf(fabsf(pt_beta), SDL::pt_betaMax); //need to confirm the range-out value of 7 GeV
    const float alphaInAbsReg =  fmaxf(fabsf(alpha_InLo), asinf(fminf(rt_InUp * k2Rinv1GeVf / 3.0f, sinAlphaMax)));
    const float alphaOutAbsReg = fmaxf(fabsf(alpha_OutLo), asinf(fminf(rt_OutLo * k2Rinv1GeVf / 3.0f, sinAlphaMax)));
    const float dBetaInLum = lIn < 11 ? 0.0f : fabsf(alphaInAbsReg*deltaZLum / z_InUp);
    const float dBetaOutLum = lOut < 11 ? 0.0f : fabsf(alphaOutAbsReg*deltaZLum / z_OutLo);
    const float dBetaLum2 = (dBetaInLum + dBetaOutLum) * (dBetaInLum + dBetaOutLum);

    const float sinDPhi = sinf(dPhi);
    const float dBetaRIn2 = 0; // TODO-RH

    float dBetaROut = 0;
    if(isEC_lastLayer)
    {
        dBetaROut = (sqrtf(mdsInGPU.anchorHighEdgeX[fourthMDIndex] * mdsInGPU.anchorHighEdgeX[fourthMDIndex] + mdsInGPU.anchorHighEdgeY[fourthMDIndex] * mdsInGPU.anchorHighEdgeY[fourthMDIndex]) - sqrtf(mdsInGPU.anchorLowEdgeX[fourthMDIndex] * mdsInGPU.anchorLowEdgeX[fourthMDIndex] + mdsInGPU.anchorLowEdgeY[fourthMDIndex] * mdsInGPU.anchorLowEdgeY[fourthMDIndex])) * sinDPhi / drt_tl_axis;
    }

    const float dBetaROut2 =  dBetaROut * dBetaROut;

    betaOutCut = asinf(fminf(drt_tl_axis*k2Rinv1GeVf / ptCut, sinAlphaMax)) //FIXME: need faster version
        + (0.02f / sdOut_d) + sqrtf(dBetaLum2 + dBetaMuls*dBetaMuls);

    //Cut #6: The real beta cut
    pass = pass and (fabsf(betaOut) < betaOutCut);
    if(not pass) return pass;

    //const float pt_betaOut = drt_tl_axis * k2Rinv1GeVf / sin(betaOut);
    const float dBetaRes = 0.02f / fminf(sdOut_d, drt_InSeg);
    const float dBetaCut2 = (dBetaRes*dBetaRes * 2.0f + dBetaMuls * dBetaMuls + dBetaLum2 + dBetaRIn2 + dBetaROut2
            + 0.25f * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)) * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)));
    float dBeta = betaIn - betaOut;
    deltaBetaCut = sqrtf(dBetaCut2);

    pass = pass and (dBeta * dBeta <= dBetaCut2);

    return pass;
}

ALPAKA_FN_ACC ALPAKA_FN_INLINE bool SDL::runpT5DefaultAlgoPPEE(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU ,struct segments& segmentsInGPU, uint16_t& pixelModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int& thirdMDIndex, unsigned int& fourthMDIndex, float& z_OutLo, float& rt_OutLo, float& deltaPhiPos, float& dPhi, float& betaIn,
        float& betaOut, float& pt_beta, float& zLo, float& rtLo, float& rtHi, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ) // pixel to EE segments
{
    bool pass = true;
    bool isPS_OutLo = (modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS);


    float z_InLo = mdsInGPU.anchorZ[firstMDIndex];
    float z_InUp = mdsInGPU.anchorZ[secondMDIndex];
    z_OutLo = mdsInGPU.anchorZ[thirdMDIndex];
    float z_OutUp = mdsInGPU.anchorZ[fourthMDIndex];

    pass =  pass and (z_InUp * z_OutLo > 0);
    if(not pass) return pass;

    float rt_InLo = mdsInGPU.anchorRt[firstMDIndex];
    float rt_InUp = mdsInGPU.anchorRt[secondMDIndex];
    rt_OutLo = mdsInGPU.anchorRt[thirdMDIndex];
    float rt_OutUp = mdsInGPU.anchorRt[fourthMDIndex];

    float x_InLo = mdsInGPU.anchorX[firstMDIndex];
    float x_InUp = mdsInGPU.anchorX[secondMDIndex];
    float x_OutLo = mdsInGPU.anchorX[thirdMDIndex];
    float x_OutUp = mdsInGPU.anchorX[fourthMDIndex];

    float y_InLo = mdsInGPU.anchorY[firstMDIndex];
    float y_InUp = mdsInGPU.anchorY[secondMDIndex];
    float y_OutLo = mdsInGPU.anchorY[thirdMDIndex];
    float y_OutUp = mdsInGPU.anchorY[fourthMDIndex];

    unsigned int pixelSegmentArrayIndex = innerSegmentIndex - rangesInGPU.segmentModuleIndices[pixelModuleIndex];

    float ptIn = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
    float ptSLo = ptIn;
    float px = segmentsInGPU.px[pixelSegmentArrayIndex];
    float py = segmentsInGPU.py[pixelSegmentArrayIndex];
    float pz = segmentsInGPU.pz[pixelSegmentArrayIndex];
    float ptErr = segmentsInGPU.ptErr[pixelSegmentArrayIndex];
    float etaErr = segmentsInGPU.etaErr[pixelSegmentArrayIndex];

    ptSLo = fmaxf(ptCut, ptSLo - 10.0f*fmaxf(ptErr, 0.005f*ptSLo));
    ptSLo = fminf(10.0f, ptSLo);

    float rtOut_o_rtIn = rt_OutLo/rt_InUp;
    const float zpitch_InLo = 0.05f;
    float zpitch_OutLo = (isPS_OutLo ? pixelPSZpitch : strip2SZpitch);
    float zGeom = zpitch_InLo + zpitch_OutLo;

    const float sdlSlope = asinf(fminf(rt_OutLo * k2Rinv1GeVf / ptCut, sinAlphaMax));
    const float dzDrtScale = tanf(sdlSlope) / sdlSlope;//FIXME: need approximate value
    zLo = z_InUp + (z_InUp - deltaZLum) * (rtOut_o_rtIn - 1.f) * (z_InUp > 0.f ? 1.f : dzDrtScale) - zGeom; //slope-correction only on outer end


    const float dLum = copysignf(deltaZLum, z_InUp);
    bool isOutSgInnerMDPS = modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS;

    const float rtGeom1 = isOutSgInnerMDPS ? pixelPSZpitch : strip2SZpitch;//FIXME: make this chosen by configuration for lay11,12 full PS
    const float zGeom1 = copysignf(zGeom, z_InUp); //used in B-E region
    rtLo = rt_InUp * (1.f + (z_OutLo- z_InUp - zGeom1) / (z_InUp + zGeom1 + dLum) / dzDrtScale) - rtGeom1; //slope correction only on the lower end


    float zInForHi = z_InUp - zGeom1 - dLum;
    if (zInForHi * z_InUp < 0)
        zInForHi = copysignf(0.1f, z_InUp);
    rtHi = rt_InUp * (1.f + (z_OutLo - z_InUp + zGeom1) / zInForHi) + rtGeom1;

    // Cut #2: rt condition
    pass =  pass and ((rt_OutLo >= rtLo) & (rt_OutLo <= rtHi));
    if(not pass) return pass;

    const float dzOutInAbs = fabsf(z_OutLo - z_InUp);
    const float coshEta = hypotf(ptIn, pz) / ptIn;
    const float multDzDr = dzOutInAbs*coshEta/(coshEta*coshEta - 1.f);
    const float r3_InUp = sqrtf(z_InUp * z_InUp + rt_InUp * rt_InUp);
    const float sdlThetaMulsF = 0.015f * sqrtf(0.1f + 0.2f * (rt_OutLo - rt_InUp) / 50.f) * sqrtf(r3_InUp / rt_InUp);
    const float sdlMuls = sdlThetaMulsF * 3.f / ptCut * 4.f; // will need a better guess than x4?

    float drtErr = etaErr*multDzDr;
    drtErr *= drtErr;
    drtErr += 0.03f*0.03f; // pixel size x2. ... random for now
    drtErr *= 9.f; //3 sigma
    drtErr += sdlMuls*sdlMuls*multDzDr*multDzDr/3.f*coshEta*coshEta;//sloppy: relative muls is 1/3 of total muls
    drtErr = sqrtf(drtErr);
    const float drtDzIn = fabsf(ptIn / pz);//all tracks are out-going in endcaps?

    const float drt_OutLo_InUp = (rt_OutLo - rt_InUp); // drOutIn

    const float rtWindow = drtErr + rtGeom1;
    const float drtMean = drtDzIn * dzOutInAbs *
        (1.f - drt_OutLo_InUp * drt_OutLo_InUp * 4 * k2Rinv1GeVf * k2Rinv1GeVf / ptIn /
         ptIn / 24.f); // with curved path correction
    const float rtLo_point = rt_InUp + drtMean - rtWindow;
    const float rtHi_point = rt_InUp + drtMean + rtWindow;

    // Cut #3: rt-z pointed
    pass =  pass and ((rt_OutLo >= rtLo_point) & (rt_OutLo <= rtHi_point));
    if(not pass) return pass;

    const float alpha1GeV_OutLo = asinf(fminf(rt_OutLo * k2Rinv1GeVf / ptCut, sinAlphaMax));
    const float sdlPVoff = 0.1f / rt_OutLo;
    sdlCut = alpha1GeV_OutLo + sqrtf(sdlMuls * sdlMuls + sdlPVoff * sdlPVoff);

    deltaPhiPos = deltaPhi(x_InUp, y_InUp, x_OutUp, y_OutUp);

    float midPointX = 0.5f * (x_InLo + x_OutLo);
    float midPointY = 0.5f * (y_InLo + y_OutLo);
    float midPointZ = 0.5f * (z_InLo + z_OutLo);

    float diffX = x_OutLo - x_InLo;
    float diffY = y_OutLo - y_InLo;
    float diffZ = z_OutLo - z_InLo;

    dPhi = deltaPhi(midPointX, midPointY, diffX, diffY);

    // Cut #5: deltaPhiChange
    pass =  pass and (fabsf(dPhi) <= sdlCut);
    if(not pass) return pass;

    float alpha_InLo  = __H2F(segmentsInGPU.dPhiChanges[innerSegmentIndex]);
    float alpha_OutLo = __H2F(segmentsInGPU.dPhiChanges[outerSegmentIndex]);

    bool isEC_lastLayer = modulesInGPU.subdets[outerOuterLowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS;

    float alpha_OutUp,alpha_OutUp_highEdge,alpha_OutUp_lowEdge;

    alpha_OutUp = deltaPhi(x_OutUp, y_OutUp, x_OutUp - x_OutLo, y_OutUp - y_OutLo);
    alpha_OutUp_highEdge = alpha_OutUp;
    alpha_OutUp_lowEdge = alpha_OutUp;

    float tl_axis_x = x_OutUp - x_InUp;
    float tl_axis_y = y_OutUp - y_InUp;
    float tl_axis_z = z_OutUp - z_InUp;

    float tl_axis_highEdge_x = tl_axis_x;
    float tl_axis_highEdge_y = tl_axis_y;

    float tl_axis_lowEdge_x = tl_axis_x;
    float tl_axis_lowEdge_y = tl_axis_y;

    betaIn = -deltaPhi(px, py, tl_axis_x, tl_axis_y);
    float betaInRHmin = betaIn;
    float betaInRHmax = betaIn;

    betaOut = -alpha_OutUp + deltaPhi(x_OutUp, y_OutUp, tl_axis_x, tl_axis_y);
    float betaOutRHmin = betaOut;
    float betaOutRHmax = betaOut;

    if(isEC_lastLayer)
    {

        alpha_OutUp_highEdge = deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], mdsInGPU.anchorHighEdgeX[fourthMDIndex] - x_OutLo, mdsInGPU.anchorHighEdgeY[fourthMDIndex] - y_OutLo);
        alpha_OutUp_lowEdge = deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], mdsInGPU.anchorLowEdgeX[fourthMDIndex] - x_OutLo, mdsInGPU.anchorLowEdgeY[fourthMDIndex] - y_OutLo);

        tl_axis_highEdge_x = mdsInGPU.anchorHighEdgeX[fourthMDIndex] - x_InUp;
        tl_axis_highEdge_y = mdsInGPU.anchorHighEdgeY[fourthMDIndex] - y_InUp;
        tl_axis_lowEdge_x = mdsInGPU.anchorLowEdgeX[fourthMDIndex] - x_InUp;
        tl_axis_lowEdge_y = mdsInGPU.anchorLowEdgeY[fourthMDIndex] - y_InUp;

        betaOutRHmin = -alpha_OutUp_highEdge + deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], tl_axis_highEdge_x, tl_axis_highEdge_y);
        betaOutRHmax = -alpha_OutUp_lowEdge + deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], tl_axis_lowEdge_x, tl_axis_lowEdge_y);
    }

    //beta computation
    float drt_tl_axis = sqrtf(tl_axis_x * tl_axis_x + tl_axis_y * tl_axis_y);
    //float drt_tl_lowEdge = sqrtf(tl_axis_lowEdge_x * tl_axis_lowEdge_x + tl_axis_lowEdge_y * tl_axis_lowEdge_y);
    //float drt_tl_highEdge = sqrtf(tl_axis_highEdge_x * tl_axis_highEdge_x + tl_axis_highEdge_y * tl_axis_highEdge_y);
//no betaIn cut for the pixels
    const float rt_InSeg = sqrtf((x_InUp - x_InLo) * (x_InUp - x_InLo) + (y_InUp - y_InLo) * (y_InUp - y_InLo));

    float betaAv = 0.5f * (betaIn + betaOut);
    pt_beta = ptIn;

    int lIn = 0;
    int lOut = isEC_lastLayer ? 11 : 5;
    float sdOut_dr = sqrtf((x_OutUp - x_OutLo) * (x_OutUp - x_OutLo) + (y_OutUp - y_OutLo) * (y_OutUp - y_OutLo));
    float sdOut_d = rt_OutUp - rt_OutLo;

    //const float diffDr = fabsf(rt_InSeg - sdOut_dr) / fabsf(rt_InSeg + sdOut_dr);

    runDeltaBetaIterationspT5(betaIn, betaOut, betaAv, pt_beta, rt_InSeg, sdOut_dr, drt_tl_axis, lIn);

    const float betaInMMSF = (fabsf(betaInRHmin + betaInRHmax) > 0) ? (2.f * betaIn / fabsf(betaInRHmin + betaInRHmax)) : 0.; //mean value of min,max is the old betaIn
    const float betaOutMMSF = (fabsf(betaOutRHmin + betaOutRHmax) > 0) ? (2.f * betaOut / fabsf(betaOutRHmin + betaOutRHmax)) : 0.;
    betaInRHmin *= betaInMMSF;
    betaInRHmax *= betaInMMSF;
    betaOutRHmin *= betaOutMMSF;
    betaOutRHmax *= betaOutMMSF;

    const float dBetaMuls = sdlThetaMulsF * 4.f / fminf(fabsf(pt_beta), SDL::pt_betaMax); //need to confirm the range-out value of 7 GeV

    const float alphaInAbsReg =  fmaxf(fabsf(alpha_InLo), asinf(fminf(rt_InUp * k2Rinv1GeVf / 3.0f, sinAlphaMax)));
    const float alphaOutAbsReg = fmaxf(fabsf(alpha_OutLo), asinf(fminf(rt_OutLo * k2Rinv1GeVf / 3.0f, sinAlphaMax)));
    const float dBetaInLum = lIn < 11 ? 0.0f : fabsf(alphaInAbsReg*deltaZLum / z_InUp);
    const float dBetaOutLum = lOut < 11 ? 0.0f : fabsf(alphaOutAbsReg*deltaZLum / z_OutLo);
    const float dBetaLum2 = (dBetaInLum + dBetaOutLum) * (dBetaInLum + dBetaOutLum);

    const float sinDPhi = sinf(dPhi);
    const float dBetaRIn2 = 0; // TODO-RH

    float dBetaROut = 0;
    if(isEC_lastLayer)
    {
        dBetaROut = (sqrtf(mdsInGPU.anchorHighEdgeX[fourthMDIndex] * mdsInGPU.anchorHighEdgeX[fourthMDIndex] + mdsInGPU.anchorHighEdgeY[fourthMDIndex] * mdsInGPU.anchorHighEdgeY[fourthMDIndex]) - sqrtf(mdsInGPU.anchorLowEdgeX[fourthMDIndex] * mdsInGPU.anchorLowEdgeX[fourthMDIndex] + mdsInGPU.anchorLowEdgeY[fourthMDIndex] * mdsInGPU.anchorLowEdgeY[fourthMDIndex])) * sinDPhi / drt_tl_axis;
    }

    const float dBetaROut2 =  dBetaROut * dBetaROut;

    betaOutCut = asinf(fminf(drt_tl_axis*k2Rinv1GeVf / ptCut, sinAlphaMax)) //FIXME: need faster version
        + (0.02f / sdOut_d) + sqrtf(dBetaLum2 + dBetaMuls*dBetaMuls);

    //Cut #6: The real beta cut
    pass =  pass and (fabsf(betaOut) < betaOutCut);
    if(not pass) return pass;

   // const float pt_betaOut = drt_tl_axis * k2Rinv1GeVf / sin(betaOut);
    float drt_InSeg = rt_InUp - rt_InLo;

    const float dBetaRes = 0.02f / fminf(sdOut_d, drt_InSeg);
    const float dBetaCut2 = (dBetaRes*dBetaRes * 2.0f + dBetaMuls * dBetaMuls + dBetaLum2 + dBetaRIn2 + dBetaROut2
            + 0.25f * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)) * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)));
    float dBeta = betaIn - betaOut;
    deltaBetaCut = sqrtf(dBetaCut2);

    pass =  pass and (dBeta * dBeta <= dBetaCut2);
    return pass;
}

ALPAKA_FN_ACC ALPAKA_FN_INLINE bool SDL::runpT5DefaultAlgo(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, uint16_t& pixelLowerModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& deltaPhi, float& betaIn, float& betaOut, float& pt_beta, float& zLo, float& zHi, float& rtLo, float& rtHi, float& zLoPointed, float& zHiPointed, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ)
{
    zLo = -999;
    zHi = -999;
    rtLo = -999;
    rtHi = -999;
    zLoPointed = -999;
    zHiPointed = -999;
    kZ = -999;
    betaInCut = -999;

    short outerInnerLowerModuleSubdet = modulesInGPU.subdets[outerInnerLowerModuleIndex];
    short outerOuterLowerModuleSubdet = modulesInGPU.subdets[outerOuterLowerModuleIndex];

    unsigned int firstMDIndex = segmentsInGPU.mdIndices[2 * innerSegmentIndex];

    unsigned int secondMDIndex = segmentsInGPU.mdIndices[2 * innerSegmentIndex + 1];
    unsigned int thirdMDIndex = segmentsInGPU.mdIndices[2 * outerSegmentIndex];
    unsigned int fourthMDIndex = segmentsInGPU.mdIndices[2 * outerSegmentIndex + 1];

    if(outerInnerLowerModuleSubdet == SDL::Barrel and outerOuterLowerModuleSubdet == SDL::Barrel)
    {
        return runpT5DefaultAlgoPPBB(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, pixelLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, zHi, zLoPointed, zHiPointed, sdlCut, betaOutCut, deltaBetaCut);
    }
    else if(outerInnerLowerModuleSubdet == SDL::Barrel and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
        return runpT5DefaultAlgoPPBB(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, pixelLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, zHi, zLoPointed, zHiPointed, sdlCut, betaOutCut, deltaBetaCut);
    }
    else if(outerInnerLowerModuleSubdet == SDL::Endcap and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
        return runpT5DefaultAlgoPPEE(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, pixelLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, rtLo, rtHi, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    }
    return false;

}
