#include "hip/hip_runtime.h"
# include "PixelTriplet.cuh"
# include "PixelTracklet.cuh"
#include "allocate.h"

SDL::pixelTriplets::pixelTriplets()
{
    pixelSegmentIndices = nullptr;
    tripletIndices = nullptr;
    nPixelTriplets = nullptr;
    pixelRadius = nullptr;
    tripletRadius = nullptr;
    pt = nullptr;
    isDup = nullptr;
    partOfPT5 = nullptr;
#ifdef CUT_VALUE_DEBUG
    pixelRadiusError = nullptr;
    rzChiSquared = nullptr;
    rPhiChiSquared = nullptr;
    rPhiChiSquaredInwards = nullptr;
#endif
}

void SDL::pixelTriplets::freeMemoryCache()
{
#ifdef Explicit_PT3
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,pixelSegmentIndices);
    cms::cuda::free_device(dev,tripletIndices);
    cms::cuda::free_device(dev,nPixelTriplets);
    cms::cuda::free_device(dev,pixelRadius);
    cms::cuda::free_device(dev,tripletRadius);
    cms::cuda::free_device(dev,pt);
    cms::cuda::free_device(dev,isDup);
    cms::cuda::free_device(dev,partOfPT5);
#else
    cms::cuda::free_managed(pixelSegmentIndices);
    cms::cuda::free_managed(tripletIndices);
    cms::cuda::free_managed(nPixelTriplets);
    cms::cuda::free_managed(pixelRadius);
    cms::cuda::free_managed(tripletRadius);
    cms::cuda::free_managed(pt);
    cms::cuda::free_managed(isDup);
    cms::cuda::free_managed(partOfPT5);
#endif
}
void SDL::pixelTriplets::freeMemory(hipStream_t stream)
{
    //hipFreeAsync(pixelSegmentIndices,stream);
    //hipFreeAsync(tripletIndices,stream);
    //hipFreeAsync(nPixelTriplets,stream);
    //hipFreeAsync(pixelRadius,stream);
    //hipFreeAsync(tripletRadius,stream);
    //hipFreeAsync(pt,stream);
    //hipFreeAsync(isDup,stream);
    //hipFreeAsync(partOfPT5,stream);
    hipFree(pixelSegmentIndices);
    hipFree(tripletIndices);
    hipFree(nPixelTriplets);
    hipFree(pixelRadius);
    hipFree(tripletRadius);
    hipFree(pt);
    hipFree(isDup);
    hipFree(partOfPT5);
#ifdef CUT_VALUE_DEBUG
    hipFree(pixelRadiusError);
    hipFree(rPhiChiSquared);
    hipFree(rPhiChiSquaredInwards);
    hipFree(rzChiSquared);
#endif
}

SDL::pixelTriplets::~pixelTriplets()
{
}

void SDL::pixelTriplets::resetMemory(unsigned int maxPixelTriplets,hipStream_t stream)
{
    hipMemsetAsync(pixelSegmentIndices,0, maxPixelTriplets * sizeof(unsigned int),stream);
    hipMemsetAsync(tripletIndices, 0,maxPixelTriplets * sizeof(unsigned int),stream);
    hipMemsetAsync(nPixelTriplets, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(pixelRadius, 0,maxPixelTriplets * sizeof(__half),stream);
    hipMemsetAsync(tripletRadius, 0,maxPixelTriplets * sizeof(__half),stream);
    hipMemsetAsync(pt, 0,maxPixelTriplets * 6*sizeof(__half),stream);
    hipMemsetAsync(isDup, 0,maxPixelTriplets * sizeof(bool),stream);
    hipMemsetAsync(partOfPT5, 0,maxPixelTriplets * sizeof(bool),stream);
}
void SDL::createPixelTripletsInUnifiedMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets,hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    pixelTripletsInGPU.pixelSegmentIndices =(unsigned int*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.tripletIndices      =(unsigned int*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.nPixelTriplets      =(unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    pixelTripletsInGPU.pixelRadius         =(__half*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(__half),stream);
    pixelTripletsInGPU.tripletRadius       =(__half*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(__half),stream);
    pixelTripletsInGPU.pt                  =(__half*)cms::cuda::allocate_managed(maxPixelTriplets * 6*sizeof(__half),stream);
    pixelTripletsInGPU.isDup               =(bool*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(bool),stream);
    pixelTripletsInGPU.partOfPT5           =(bool*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(bool),stream);

#else
    hipMallocManaged(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMallocManaged(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMallocManaged(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMallocManaged(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(__half));
    hipMallocManaged(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(__half));
    hipMallocManaged(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(__half));
    hipMallocManaged(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool));
    hipMallocManaged(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool));
#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&pixelTripletsInGPU.pixelRadiusError, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.rPhiChiSquared, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.rPhiChiSquaredInwards, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.rzChiSquared, maxPixelTriplets * sizeof(float));
#endif
#endif
    pixelTripletsInGPU.eta = pixelTripletsInGPU.pt + maxPixelTriplets;
    pixelTripletsInGPU.phi = pixelTripletsInGPU.pt + maxPixelTriplets * 2;
    pixelTripletsInGPU.eta_pix = pixelTripletsInGPU.pt + maxPixelTriplets *3;
    pixelTripletsInGPU.phi_pix = pixelTripletsInGPU.pt + maxPixelTriplets * 4;
    pixelTripletsInGPU.score = pixelTripletsInGPU.pt + maxPixelTriplets * 5;
    hipMemsetAsync(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int),stream);
}

void SDL::createPixelTripletsInExplicitMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets, hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    pixelTripletsInGPU.pixelSegmentIndices =(unsigned int*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.tripletIndices      =(unsigned int*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.nPixelTriplets      =(unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    pixelTripletsInGPU.pixelRadius         =(__half*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(__half),stream);
    pixelTripletsInGPU.tripletRadius       =(__half*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(__half),stream);
    pixelTripletsInGPU.pt                  =(__half*)cms::cuda::allocate_device(dev,maxPixelTriplets * 6*sizeof(__half),stream);
    pixelTripletsInGPU.isDup               =(bool*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(bool),stream);
    pixelTripletsInGPU.partOfPT5           =(bool*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(bool),stream);

#else
    //hipMallocAsync(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(unsigned int),stream);
    //hipMallocAsync(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(unsigned int),stream);
    //hipMallocAsync(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int),stream);
    //hipMallocAsync(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(float),stream);
    //hipMallocAsync(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(float),stream);
    //hipMallocAsync(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(float),stream);
    //hipMallocAsync(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool),stream);
    //hipMallocAsync(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool),stream);
    hipMalloc(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(__half));
    hipMalloc(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(__half));
    hipMalloc(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(__half));
    hipMalloc(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool));
    hipMalloc(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool));
#endif

    hipMemsetAsync(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int),stream);
hipStreamSynchronize(stream);
    pixelTripletsInGPU.eta = pixelTripletsInGPU.pt + maxPixelTriplets;
    pixelTripletsInGPU.phi = pixelTripletsInGPU.pt + maxPixelTriplets * 2;
    pixelTripletsInGPU.eta_pix = pixelTripletsInGPU.pt + maxPixelTriplets *3;
    pixelTripletsInGPU.phi_pix = pixelTripletsInGPU.pt + maxPixelTriplets * 4;
    pixelTripletsInGPU.score = pixelTripletsInGPU.pt + maxPixelTriplets * 5;

}

#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int pixelSegmentIndex, unsigned int tripletIndex, float pixelRadius, float pixelRadiusError, float tripletRadius, float rPhiChiSquared, float rPhiChiSquaredInwards, float rzChiSquared, unsigned int pixelTripletIndex, float pt, float eta, float phi, float eta_pix, float phi_pix, float score)
#else
__device__ void SDL::addPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int pixelSegmentIndex, unsigned int tripletIndex, float pixelRadius, float tripletRadius, unsigned int pixelTripletIndex, float pt, float eta, float phi, float eta_pix, float phi_pix,float score)
#endif
{
    pixelTripletsInGPU.pixelSegmentIndices[pixelTripletIndex] = pixelSegmentIndex;
    pixelTripletsInGPU.tripletIndices[pixelTripletIndex] = tripletIndex;
    pixelTripletsInGPU.pixelRadius[pixelTripletIndex] = __float2half(pixelRadius);
    pixelTripletsInGPU.tripletRadius[pixelTripletIndex] = __float2half(tripletRadius);
    pixelTripletsInGPU.pt[pixelTripletIndex] = __float2half(pt);
    pixelTripletsInGPU.eta[pixelTripletIndex] = __float2half(eta);
    pixelTripletsInGPU.phi[pixelTripletIndex] = __float2half(phi);
    pixelTripletsInGPU.eta_pix[pixelTripletIndex] = __float2half(eta_pix);
    pixelTripletsInGPU.phi_pix[pixelTripletIndex] = __float2half(phi_pix);
    pixelTripletsInGPU.isDup[pixelTripletIndex] = 0;
    pixelTripletsInGPU.score[pixelTripletIndex] = __float2half(score);

#ifdef CUT_VALUE_DEBUG
    pixelTripletsInGPU.pixelRadiusError[pixelTripletIndex] = pixelRadiusError;
    pixelTripletsInGPU.rPhiChiSquared[pixelTripletIndex] = rPhiChiSquared;
    pixelTripletsInGPU.rPhiChiSquaredInwards[pixelTripletIndex] = rPhiChiSquaredInwards;
    pixelTripletsInGPU.rzChiSquared[pixelTripletIndex] = rzChiSquared;
#endif
}
__device__ void SDL::rmPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU,unsigned int pixelTripletIndex)
{
    pixelTripletsInGPU.isDup[pixelTripletIndex] = 1;
}

__device__ float SDL::computeRadiusFromThreeAnchorHitspT3(float x1, float y1, float x2, float y2, float x3, float y3, float& g, float& f)
{
    float radius = 0;

    //writing manual code for computing radius, which obviously sucks
    //TODO:Use fancy inbuilt libraries like cuBLAS or cuSOLVE for this!
    //(g,f) -> center
    //first anchor hit - (x1,y1), second anchor hit - (x2,y2), third anchor hit - (x3, y3)

    /*
    if((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0)
    {
        return -1; //WTF man three collinear points!
    }
    */

    float denomInv = 1.f/((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3));

    float xy1sqr = x1 * x1 + y1 * y1;

    float xy2sqr = x2 * x2 + y2 * y2;

    float xy3sqr = x3 * x3 + y3 * y3;

    g = 0.5f * ((y3 - y2) * xy1sqr + (y1 - y3) * xy2sqr + (y2 - y1) * xy3sqr) * denomInv;

    f = 0.5f * ((x2 - x3) * xy1sqr + (x3 - x1) * xy2sqr + (x1 - x2) * xy3sqr) * denomInv;

    float c = ((x2 * y3 - x3 * y2) * xy1sqr + (x3 * y1 - x1 * y3) * xy2sqr + (x1 * y2 - x2 * y1) * xy3sqr) * denomInv;

    if(((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0) || (g * g + f * f - c < 0))
    {
        printf("three collinear points or FATAL! r^2 < 0!\n");
  radius = -1;
    }
    else
      radius = sqrtf(g * g  + f * f - c);

    return radius;
}

__device__ bool SDL::runPixelTripletDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, unsigned int& pixelSegmentIndex, unsigned int tripletIndex, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards, bool runChiSquaredCuts)
{
    bool pass = true;

    //run pT4 compatibility between the pixel segment and inner segment, and between the pixel and outer segment of the triplet


    //placeholder
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    unsigned int pixelModuleIndex = segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];

    unsigned int lowerModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex];
    unsigned int middleModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 1];
    unsigned int upperModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 2];


    // pixel segment vs inner segment of the triplet
    pass = pass & runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, lowerModuleIndex, middleModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, 600/*N_MAX_SEGMENTS_PER_MODULE*/);

    //pixel segment vs outer segment of triplet
    pass = pass & runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, middleModuleIndex, upperModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex + 1], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, 600/*N_MAX_SEGMENTS_PER_MODULE*/);

    //pt matching between the pixel ptin and the triplet circle pt
    unsigned int pixelSegmentArrayIndex = pixelSegmentIndex - (pixelModuleIndex * 600);
    float pixelSegmentPt = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
    float pixelSegmentPtError = segmentsInGPU.ptErr[pixelSegmentArrayIndex];

    unsigned int pixelInnerMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex];
    unsigned int pixelOuterMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex + 1];

    unsigned int pixelAnchorHitIndex1 = mdsInGPU.hitIndices[2 * pixelInnerMDIndex];
    unsigned int pixelNonAnchorHitIndex1 = mdsInGPU.hitIndices[2 * pixelInnerMDIndex + 1];
    unsigned int pixelAnchorHitIndex2 = mdsInGPU.hitIndices[2 * pixelOuterMDIndex];
    unsigned int pixelNonAnchorHitIndex2 = mdsInGPU.hitIndices[2 * pixelOuterMDIndex + 1];

    pixelRadius = pixelSegmentPt/(2.f * k2Rinv1GeVf);
    pixelRadiusError = pixelSegmentPtError/(2.f * k2Rinv1GeVf);
    unsigned int tripletInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex];
    unsigned int tripletOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex + 1];

    unsigned int innerMDAnchorHitIndex = segmentsInGPU.innerMiniDoubletAnchorHitIndices[tripletInnerSegmentIndex];
    unsigned int middleMDAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[tripletInnerSegmentIndex];
    unsigned int outerMDAnchorHitIndex = segmentsInGPU.outerMiniDoubletAnchorHitIndices[tripletOuterSegmentIndex];

    float x1 = hitsInGPU.xs[innerMDAnchorHitIndex];
    float x2 = hitsInGPU.xs[middleMDAnchorHitIndex];
    float x3 = hitsInGPU.xs[outerMDAnchorHitIndex];

    float y1 = hitsInGPU.ys[innerMDAnchorHitIndex];
    float y2 = hitsInGPU.ys[middleMDAnchorHitIndex];
    float y3 = hitsInGPU.ys[outerMDAnchorHitIndex];
    float g,f;
    
    tripletRadius = computeRadiusFromThreeAnchorHitspT3(x1, y1, x2, y2, x3, y3,g,f);
    
    pass = pass & passRadiusCriterion(modulesInGPU, pixelRadius, pixelRadiusError, tripletRadius, lowerModuleIndex, middleModuleIndex, upperModuleIndex);

    unsigned int anchorHits[] = {innerMDAnchorHitIndex, middleMDAnchorHitIndex, outerMDAnchorHitIndex};
    unsigned int pixelAnchorHits[] = {pixelAnchorHitIndex1, pixelAnchorHitIndex2};
    unsigned int lowerModuleIndices[] = {lowerModuleIndex, middleModuleIndex, upperModuleIndex};

    rzChiSquared = computePT3RZChiSquared(modulesInGPU, hitsInGPU, segmentsInGPU, pixelAnchorHitIndex1, pixelAnchorHitIndex2, anchorHits, lowerModuleIndices);

    rPhiChiSquared = computePT3RPhiChiSquared(modulesInGPU, hitsInGPU, segmentsInGPU, pixelSegmentArrayIndex, anchorHits, lowerModuleIndices);

    rPhiChiSquaredInwards = computePT3RPhiChiSquaredInwards(modulesInGPU, hitsInGPU, tripletRadius, g, f, pixelAnchorHits);

    if(runChiSquaredCuts and pixelSegmentPt < 5.0f)
    {
        pass = pass & passPT3RZChiSquaredCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rzChiSquared);
        pass = pass & passPT3RPhiChiSquaredCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rPhiChiSquared);

        pass = pass & passPT3RPhiChiSquaredInwardsCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rPhiChiSquaredInwards);
    }


    return pass;

}

__device__ bool SDL::passPT3RPhiChiSquaredInwardsCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, float& chiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    
    if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return chiSquared < 22016.8055f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return chiSquared < 935179.56807f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return chiSquared < 29064.12959f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return chiSquared < 935179.5681f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return chiSquared < 1370.0113195101474f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return chiSquared < 5492.110048314815f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return chiSquared < 4160.410806470067f;
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 29064.129591225726f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return chiSquared < 12634.215376250893f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return chiSquared < 353821.69361145404f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 33393.26076341235f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return chiSquared < 935179.5680742573f;
    }

    return true;
}

__device__ float SDL::computePT3RPhiChiSquaredInwards(struct modules& modulesInGPU, struct hits& hitsInGPU, float& r, float& g, float& f, unsigned int* pixelAnchorHits)
{
    float x,y;
    float chiSquared = 0;
    for(size_t i = 0; i < 2; i++)
    {
        x = hitsInGPU.xs[pixelAnchorHits[i]];
        y = hitsInGPU.ys[pixelAnchorHits[i]];
        float residual = (x - g) * (x -g) + (y - f) * (y - f) - r * r;
        chiSquared += residual * residual;
    }
    //chiSquared /= 2;
    chiSquared *= 0.5f;
    return chiSquared;
}

__device__ bool SDL::passPT3RZChiSquaredCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, float& rzChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);

    if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return rzChiSquared < 85.2499f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return rzChiSquared < 85.2499f;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return rzChiSquared < 74.19805f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return rzChiSquared < 97.9479f;
    }

    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return rzChiSquared < 451.1407f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return rzChiSquared < 595.546f;
    }

    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return rzChiSquared < 518.339f;
    }

    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return rzChiSquared < 684.253f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return rzChiSquared < 684.253f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return rzChiSquared  < 392.654f;
    }

    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return rzChiSquared < 518.339f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return rzChiSquared < 518.339f;
    }

    //default - category not found!
    return true;
}

__device__ float SDL::computePT3RZChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, struct segments& segmentsInGPU, unsigned int& pixelAnchorHitIndex1, unsigned int& pixelAnchorHitIndex2, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    float& rtPix1 = hitsInGPU.rts[pixelAnchorHitIndex1];
    float& rtPix2 = hitsInGPU.rts[pixelAnchorHitIndex2];
    float& zPix1 = hitsInGPU.zs[pixelAnchorHitIndex1];
    float& zPix2 = hitsInGPU.zs[pixelAnchorHitIndex2];
    float slope = (zPix2 - zPix1)/(rtPix2 - rtPix1);
    float rtAnchor, zAnchor;
    float residual = 0;
    float error = 0;
    //hardcoded array indices!!!
    float RMSE = 0;
    float drdz;
    for(size_t i = 0; i < 3; i++)
    {
        unsigned int& anchorHitIndex = anchorHits[i];
        unsigned int& lowerModuleIndex = lowerModuleIndices[i];
        rtAnchor = hitsInGPU.rts[anchorHitIndex];
        zAnchor = hitsInGPU.zs[anchorHitIndex];

        const int moduleType = modulesInGPU.moduleType[lowerModuleIndex];
        const int moduleSide = modulesInGPU.sides[lowerModuleIndex];
        const int moduleLayerType = modulesInGPU.moduleLayerType[lowerModuleIndex];
        const int layer = modulesInGPU.layers[lowerModuleIndex] + 6 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex] == SDL::TwoS);
        
        residual = (layer <= 6) ?  (zAnchor - zPix1) - slope * (rtAnchor - rtPix1) : (rtAnchor - rtPix1) - (zAnchor - zPix1)/slope;
        
        //PS Modules
        if(moduleType == 0)
        {
            error = 0.15f;
        }
        else //2S modules
        {
            error = 5.0f;
        }

        //special dispensation to tilted PS modules!
        if(moduleType == 0 and layer <= 6 and moduleSide != Center)
        {
            if(moduleLayerType == Strip)
            {
                drdz = modulesInGPU.drdzs[lowerModuleIndex];
            }
            else
            {
                drdz = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(lowerModuleIndex)];
            }

            error /= sqrtf(1 + drdz * drdz);
        }
        RMSE += (residual * residual)/(error * error);
    }

    RMSE = sqrtf(0.2f * RMSE); //the constant doesn't really matter....
    return RMSE;
}

//TODO: merge this one and the pT5 function later into a single function
__device__ float SDL::computePT3RPhiChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, struct segments& segmentsInGPU, unsigned int pixelSegmentArrayIndex, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    float g = segmentsInGPU.circleCenterX[pixelSegmentArrayIndex];
    float f = segmentsInGPU.circleCenterY[pixelSegmentArrayIndex];
    float radius = segmentsInGPU.circleRadius[pixelSegmentArrayIndex];
    float delta1[3], delta2[3], slopes[3];
    bool isFlat[3];
    float xs[3];
    float ys[3];
    float chiSquared = 0;
    ModuleType moduleType;
    short moduleSubdet, moduleSide;
    ModuleLayerType moduleLayerType;
    float drdz;
    float inv1 = 0.01f/0.009f;
    float inv2 = 0.15f/0.009f;
    float inv3 = 2.4f/0.009f;
    for(size_t i = 0; i < 3; i++)
    {
        xs[i] = hitsInGPU.xs[anchorHits[i]];
        ys[i] = hitsInGPU.ys[anchorHits[i]];
    //}

    ////computeSigmasForRegression(modulesInGPU, lowerModuleIndices, delta1, delta2, slopes, isFlat, 3);
    //for(size_t i=0; i<3; i++)
    //{
        moduleType = modulesInGPU.moduleType[lowerModuleIndices[i]];
        moduleSubdet = modulesInGPU.subdets[lowerModuleIndices[i]];
        moduleSide = modulesInGPU.sides[lowerModuleIndices[i]];
        moduleLayerType = modulesInGPU.moduleLayerType[lowerModuleIndices[i]];
        //category 1 - barrel PS flat
        if(moduleSubdet == Barrel and moduleType == PS and moduleSide == Center)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            delta2[i] = inv1;//1.1111f;//0.01;
            slopes[i] = -999;
            isFlat[i] = true;
        }

        //category 2 - barrel 2S
        else if(moduleSubdet == Barrel and moduleType == TwoS)
        {
            delta1[i] = 1;//0.009;
            delta2[i] = 1;//0.009;
            slopes[i] = -999;
            isFlat[i] = true;
        }

        //category 3 - barrel PS tilted
        else if(moduleSubdet == Barrel and moduleType == PS and moduleSide != Center)
        {

            //get drdz
            if(moduleLayerType == Strip)
            {
                drdz = modulesInGPU.drdzs[lowerModuleIndices[i]];
                slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
            }
            else
            {
                drdz = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(lowerModuleIndices[i])];
                slopes[i] = modulesInGPU.slopes[modulesInGPU.partnerModuleIndex(lowerModuleIndices[i])];
            }

            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            if(anchorHits)
            {
                //delta2[i] = (0.15f * drdz/sqrtf(1 + drdz * drdz));
                delta2[i] = (inv2 * drdz/sqrtf(1 + drdz * drdz));
            }
            else
            {
                //delta2[i] = (2.4f * drdz/sqrtf(1 + drdz * drdz));
                delta2[i] = (inv3 * drdz/sqrtf(1 + drdz * drdz));
            }
        }

        //category 4 - endcap PS
        else if(moduleSubdet == Endcap and moduleType == PS)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            if(moduleLayerType == Strip)
            {
                slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
            }
            else
            {
                slopes[i] = modulesInGPU.slopes[modulesInGPU.partnerModuleIndex(lowerModuleIndices[i])];

            }
            isFlat[i] = false;

            /*despite the type of the module layer of the lower module index,
            all anchor hits are on the pixel side and all non-anchor hits are
            on the strip side!*/
            if(anchorHits)
            {
                delta2[i] = inv2;//16.6666f;//0.15f;
            }
            else
            {
                delta2[i] = inv3;//266.666f;//2.4f;
            }
        }

        //category 5 - endcap 2S
        else if(moduleSubdet == Endcap and moduleType == TwoS)
        {
            delta1[i] = 1;//0.009;
            delta2[i] = 500*inv1;//555.5555f;//5.f;
            slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
            isFlat[i] = false;
        }
        else
        {
            printf("ERROR!!!!! I SHOULDN'T BE HERE!!!! subdet = %d, type = %d, side = %d\n", moduleSubdet, moduleType, moduleSide);
        }
    }
    // this for loop is kept to keep the physics results the same but I think this is a bug in the original code. This was kept at 5 and not nPoints
    for(size_t i = 3; i < 5; i++)
    {
        delta1[i] /= 0.009f;
        delta2[i] /= 0.009f;
    }
    chiSquared = computeChiSquared(3, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius);
    
    return chiSquared;
}


//90pc threshold
__device__ bool SDL::passPT3RPhiChiSquaredCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, float& chiSquared)
{

    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);

    if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return chiSquared < 7.003f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return chiSquared < 0.5f;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return chiSquared < 8.046f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return chiSquared < 0.575f;
    }

    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return chiSquared < 5.304f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return chiSquared < 10.6211f;
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 4.617f;
    }

    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 8.046f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return chiSquared < 0.435f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return chiSquared < 9.244f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return chiSquared < 0.287f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return chiSquared < 18.509f;
    }

    return true;
}

__device__ bool SDL::passRadiusCriterion(struct modules& modulesInGPU, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, unsigned int lowerModuleIndex, unsigned int middleModuleIndex, unsigned int upperModuleIndex)
{
    if(modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionEEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[middleModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[upperModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBBE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else
    {
        return passRadiusCriterionBBB(pixelRadius, pixelRadiusError, tripletRadius);
    }

    //return ((modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap) & (passRadiusCriterionEEE(pixelRadius, pixelRadiusError, tripletRadius))) | ((modulesInGPU.subdets[middleModuleIndex] == SDL::Endcap) & (passRadiusCriterionBEE(pixelRadius, pixelRadiusError, tripletRadius))) | ((modulesInGPU.subdets[upperModuleIndex] == SDL::Endcap) & (passRadiusCriterionBBE(pixelRadius, pixelRadiusError, tripletRadius))) |  (passRadiusCriterionBBB(pixelRadius, pixelRadiusError, tripletRadius));

}

/*bounds for high Pt taken from : http://uaf-10.t2.ucsd.edu/~bsathian/SDL/T5_efficiency/efficiencies/new_efficiencies/efficiencies_20210513_T5_recovering_high_Pt_efficiencies/highE_radius_matching/highE_bounds.txt */
__device__ bool SDL::passRadiusCriterionBBB(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0.15624f;
    float pixelInvRadiusErrorBound = 0.17235f;

    if(pixelRadius > 2.0f/(2.f * k2Rinv1GeVf))
    {
        pixelInvRadiusErrorBound = 0.6375f;
        tripletInvRadiusErrorBound = 0.6588f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);
}

__device__ bool SDL::passRadiusCriterionBBE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0.45972f;
    float pixelInvRadiusErrorBound = 0.19644f;

    if(pixelRadius > 2.0f/(2 * k2Rinv1GeVf))
    {
        pixelInvRadiusErrorBound = 0.6805f;
        tripletInvRadiusErrorBound = 0.8557f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

__device__ bool SDL::passRadiusCriterionBEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 1.59294f;
    float pixelInvRadiusErrorBound = 0.255181f;

    if(pixelRadius > 2.0f/(2 * k2Rinv1GeVf)) //as good as not having selections
    {
        pixelInvRadiusErrorBound = 2.2091f;
        tripletInvRadiusErrorBound = 2.3548f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));
    pixelRadiusInvMin = fmaxf(pixelRadiusInvMin, 0);

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

__device__ bool SDL::passRadiusCriterionEEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 1.7006f;
    float pixelInvRadiusErrorBound = 0.26367f;

    if(pixelRadius > 2.0f/(2 * k2Rinv1GeVf)) //as good as not having selections
    {
        pixelInvRadiusErrorBound = 2.286f;
        tripletInvRadiusErrorBound = 2.436f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));
    pixelRadiusInvMin = fmaxf(0, pixelRadiusInvMin);

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

