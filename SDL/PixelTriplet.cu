#include "PixelTriplet.cuh"

SDL::pixelTriplets::pixelTriplets()
{
    pixelSegmentIndices = nullptr;
    tripletIndices = nullptr;
    nPixelTriplets = nullptr;
    totOccupancyPixelTriplets = nullptr;
    pixelRadius = nullptr;
    tripletRadius = nullptr;
    pt = nullptr;
    isDup = nullptr;
    partOfPT5 = nullptr;
    centerX = nullptr;
    centerY = nullptr;
    hitIndices = nullptr;
    lowerModuleIndices = nullptr;
    logicalLayers = nullptr;
    rzChiSquared = nullptr;
    rPhiChiSquared = nullptr;
    rPhiChiSquaredInwards = nullptr;
}

void SDL::pixelTriplets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,pixelSegmentIndices);
    cms::cuda::free_device(dev,tripletIndices);
    cms::cuda::free_device(dev,nPixelTriplets);
    cms::cuda::free_device(dev,totOccupancyPixelTriplets);
    cms::cuda::free_device(dev,pixelRadius);
    cms::cuda::free_device(dev,tripletRadius);
    cms::cuda::free_device(dev,pt);
    cms::cuda::free_device(dev,isDup);
    cms::cuda::free_device(dev,partOfPT5);
    cms::cuda::free_device(dev, centerX);
    cms::cuda::free_device(dev, centerY);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, rPhiChiSquared);
    cms::cuda::free_device(dev, rPhiChiSquaredInwards);
    cms::cuda::free_device(dev, rzChiSquared);
}

void SDL::pixelTriplets::freeMemory(hipStream_t stream)
{
    hipFree(pixelSegmentIndices);
    hipFree(tripletIndices);
    hipFree(nPixelTriplets);
    hipFree(totOccupancyPixelTriplets);
    hipFree(pixelRadius);
    hipFree(tripletRadius);
    hipFree(pt);
    hipFree(isDup);
    hipFree(partOfPT5);
    hipFree(centerX);
    hipFree(centerY);
    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(lowerModuleIndices);
    hipFree(rPhiChiSquared);
    hipFree(rPhiChiSquaredInwards);
    hipFree(rzChiSquared);
}

SDL::pixelTriplets::~pixelTriplets()
{
}

void SDL::createPixelTripletsInExplicitMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets, hipStream_t stream)
{
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    pixelTripletsInGPU.pixelSegmentIndices       =(unsigned int*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.tripletIndices            =(unsigned int*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.nPixelTriplets            =(int*)cms::cuda::allocate_device(dev,sizeof(int),stream);
    pixelTripletsInGPU.totOccupancyPixelTriplets =(int*)cms::cuda::allocate_device(dev,sizeof(int),stream);
    pixelTripletsInGPU.pixelRadius               =(FPX*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(FPX),stream);
    pixelTripletsInGPU.tripletRadius             =(FPX*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(FPX),stream);
    pixelTripletsInGPU.pt                        =(FPX*)cms::cuda::allocate_device(dev,maxPixelTriplets * 6*sizeof(FPX),stream);
    pixelTripletsInGPU.isDup                     =(bool*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(bool),stream);
    pixelTripletsInGPU.partOfPT5                 =(bool*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(bool),stream);
    pixelTripletsInGPU.centerX                   = (FPX*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(FPX), stream);
    pixelTripletsInGPU.centerY                   = (FPX*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(FPX), stream);
    pixelTripletsInGPU.lowerModuleIndices        = (uint16_t*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(uint16_t) * 5, stream);
    pixelTripletsInGPU.hitIndices                = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(unsigned int) * 10, stream);
    pixelTripletsInGPU.logicalLayers             = (uint8_t*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(uint8_t) * 5, stream);

    pixelTripletsInGPU.rPhiChiSquared = (float*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(float), stream);
    pixelTripletsInGPU.rPhiChiSquaredInwards = (float*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(float), stream);
    pixelTripletsInGPU.rzChiSquared = (float*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(float), stream);
#else
    hipMalloc(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.nPixelTriplets, sizeof(int));
    hipMalloc(&pixelTripletsInGPU.totOccupancyPixelTriplets, sizeof(int));
    hipMalloc(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool));
    hipMalloc(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool));
    hipMalloc(&pixelTripletsInGPU.centerX, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.centerY, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.logicalLayers, maxPixelTriplets * sizeof(uint8_t) * 5);
    hipMalloc(&pixelTripletsInGPU.hitIndices, maxPixelTriplets * sizeof(unsigned int) * 10);
    hipMalloc(&pixelTripletsInGPU.lowerModuleIndices, maxPixelTriplets * sizeof(uint16_t) * 5);
    hipMalloc(&pixelTripletsInGPU.rPhiChiSquared, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.rPhiChiSquaredInwards, maxPixelTriplets * sizeof(float));
    hipMalloc(&pixelTripletsInGPU.rzChiSquared, maxPixelTriplets * sizeof(float));
#endif
    hipMemsetAsync(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(int),stream);
    hipMemsetAsync(pixelTripletsInGPU.totOccupancyPixelTriplets, 0, sizeof(int),stream);
    hipMemsetAsync(pixelTripletsInGPU.partOfPT5, 0, maxPixelTriplets*sizeof(bool),stream);
    hipStreamSynchronize(stream);

    pixelTripletsInGPU.eta = pixelTripletsInGPU.pt + maxPixelTriplets;
    pixelTripletsInGPU.phi = pixelTripletsInGPU.pt + maxPixelTriplets * 2;
    pixelTripletsInGPU.eta_pix = pixelTripletsInGPU.pt + maxPixelTriplets *3;
    pixelTripletsInGPU.phi_pix = pixelTripletsInGPU.pt + maxPixelTriplets * 4;
    pixelTripletsInGPU.score = pixelTripletsInGPU.pt + maxPixelTriplets * 5;
}

SDL::pixelQuintuplets::pixelQuintuplets()
{
    pixelIndices = nullptr;
    T5Indices = nullptr;
    nPixelQuintuplets = nullptr;
    totOccupancyPixelQuintuplets = nullptr;
    isDup = nullptr;
    score = nullptr;
    pixelRadius = nullptr;
    quintupletRadius = nullptr;
    centerX = nullptr;
    centerY = nullptr;
    logicalLayers = nullptr;
    hitIndices = nullptr;
    lowerModuleIndices = nullptr;
}

SDL::pixelQuintuplets::~pixelQuintuplets()
{
}

void SDL::pixelQuintuplets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,pixelIndices);
    cms::cuda::free_device(dev,T5Indices);
    cms::cuda::free_device(dev,nPixelQuintuplets);
    cms::cuda::free_device(dev,totOccupancyPixelQuintuplets);
    cms::cuda::free_device(dev,isDup);
    cms::cuda::free_device(dev,score);
    cms::cuda::free_device(dev,eta);
    cms::cuda::free_device(dev,phi);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, centerX);
    cms::cuda::free_device(dev, centerY);
    cms::cuda::free_device(dev, pixelRadius);
    cms::cuda::free_device(dev, quintupletRadius);
    cms::cuda::free_device(dev, rzChiSquared);
    cms::cuda::free_device(dev, rPhiChiSquared);
    cms::cuda::free_device(dev, rPhiChiSquaredInwards);
}

void SDL::pixelQuintuplets::freeMemory(hipStream_t stream)
{
    hipFree(pixelIndices);
    hipFree(T5Indices);
    hipFree(nPixelQuintuplets);
    hipFree(totOccupancyPixelQuintuplets);
    hipFree(isDup);
    hipFree(score);
    hipFree(eta);
    hipFree(phi);

    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(lowerModuleIndices);
    hipFree(pixelRadius);
    hipFree(quintupletRadius);
    hipFree(centerX);
    hipFree(centerY);
    hipFree(rzChiSquared);
    hipFree(rPhiChiSquared);
    hipFree(rPhiChiSquaredInwards);
    hipStreamSynchronize(stream);
}

void SDL::createPixelQuintupletsInExplicitMemory(struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int maxPixelQuintuplets,hipStream_t stream)
{
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    pixelQuintupletsInGPU.pixelIndices        = (unsigned int*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.T5Indices           = (unsigned int*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.nPixelQuintuplets   = (int*)cms::cuda::allocate_device(dev,sizeof(int),stream);
    pixelQuintupletsInGPU.totOccupancyPixelQuintuplets   = (int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.isDup               = (bool*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(bool),stream);
    pixelQuintupletsInGPU.score               = (FPX*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(FPX),stream);
    pixelQuintupletsInGPU.eta                 = (FPX*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(FPX),stream);
    pixelQuintupletsInGPU.phi                 = (FPX*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(FPX),stream);
    pixelQuintupletsInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * 14 * sizeof(unsigned int), stream);
    pixelQuintupletsInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * 7 * sizeof(uint8_t), stream);
    pixelQuintupletsInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * 7 * sizeof(uint16_t), stream);
    pixelQuintupletsInGPU.centerX          = (FPX*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(FPX), stream);
    pixelQuintupletsInGPU.centerY          = (FPX*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(FPX), stream);
    pixelQuintupletsInGPU.pixelRadius      = (FPX*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(FPX), stream);
    pixelQuintupletsInGPU.quintupletRadius = (FPX*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(FPX), stream);
    pixelQuintupletsInGPU.rzChiSquared          = (float*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(float), stream);
    pixelQuintupletsInGPU.rPhiChiSquared      = (float*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(float), stream);
    pixelQuintupletsInGPU.rPhiChiSquaredInwards = (float*)cms::cuda::allocate_device(dev, maxPixelQuintuplets * sizeof(float), stream);
#else
    hipMalloc(&pixelQuintupletsInGPU.pixelIndices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.T5Indices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.nPixelQuintuplets, sizeof(int));
    hipMalloc(&pixelQuintupletsInGPU.totOccupancyPixelQuintuplets, sizeof(int));
    hipMalloc(&pixelQuintupletsInGPU.isDup, maxPixelQuintuplets * sizeof(bool));
    hipMalloc(&pixelQuintupletsInGPU.score, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.eta  , maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.phi  , maxPixelQuintuplets * sizeof(FPX));

    hipMalloc(&pixelQuintupletsInGPU.logicalLayers, maxPixelQuintuplets * 7 *sizeof(uint8_t));
    hipMalloc(&pixelQuintupletsInGPU.hitIndices, maxPixelQuintuplets * 14 * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.lowerModuleIndices, maxPixelQuintuplets * 7 * sizeof(uint16_t));
    hipMalloc(&pixelQuintupletsInGPU.pixelRadius, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.quintupletRadius, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.centerX, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.centerY, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.rzChiSquared, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.rPhiChiSquared, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.rPhiChiSquaredInwards, maxPixelQuintuplets * sizeof(unsigned int));
#endif
    hipMemsetAsync(pixelQuintupletsInGPU.nPixelQuintuplets, 0, sizeof(int),stream);
    hipMemsetAsync(pixelQuintupletsInGPU.totOccupancyPixelQuintuplets, 0, sizeof(int),stream);
    hipStreamSynchronize(stream);
}
