#include "hip/hip_runtime.h"
# include "PixelTriplet.cuh"
# include "PixelTracklet.cuh"
#include "allocate.h"
#include "Kernels.cuh"

SDL::pixelTriplets::pixelTriplets()
{
    pixelSegmentIndices = nullptr;
    tripletIndices = nullptr;
    nPixelTriplets = nullptr;
    totOccupancyPixelTriplets = nullptr;
    pixelRadius = nullptr;
    tripletRadius = nullptr;
    pt = nullptr;
    isDup = nullptr;
    partOfPT5 = nullptr;
    centerX = nullptr;
    centerY = nullptr;
    hitIndices = nullptr;
    lowerModuleIndices = nullptr;
    logicalLayers = nullptr;
#ifdef CUT_VALUE_DEBUG
    pixelRadiusError = nullptr;
    rzChiSquared = nullptr;
    rPhiChiSquared = nullptr;
    rPhiChiSquaredInwards = nullptr;
#endif
}

void SDL::pixelTriplets::freeMemoryCache()
{
#ifdef Explicit_PT3
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,pixelSegmentIndices);
    cms::cuda::free_device(dev,tripletIndices);
    cms::cuda::free_device(dev,nPixelTriplets);
    cms::cuda::free_device(dev,totOccupancyPixelTriplets);
    cms::cuda::free_device(dev,pixelRadius);
    cms::cuda::free_device(dev,tripletRadius);
    cms::cuda::free_device(dev,pt);
    cms::cuda::free_device(dev,isDup);
    cms::cuda::free_device(dev,partOfPT5);
    cms::cuda::free_device(dev, centerX);
    cms::cuda::free_device(dev, centerY);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, lowerModuleIndices);
#else
    cms::cuda::free_managed(pixelSegmentIndices);
    cms::cuda::free_managed(tripletIndices);
    cms::cuda::free_managed(nPixelTriplets);
    cms::cuda::free_managed(totOccupancyPixelTriplets);
    cms::cuda::free_managed(pixelRadius);
    cms::cuda::free_managed(tripletRadius);
    cms::cuda::free_managed(pt);
    cms::cuda::free_managed(isDup);
    cms::cuda::free_managed(partOfPT5);
    cms::cuda::free_managed(centerX);
    cms::cuda::free_managed(centerY);
    cms::cuda::free_managed(hitIndices);
    cms::cuda::free_managed(logicalLayers);
    cms::cuda::free_managed(lowerModuleIndices);

#endif
}
void SDL::pixelTriplets::freeMemory(hipStream_t stream)
{
    hipFree(pixelSegmentIndices);
    hipFree(tripletIndices);
    hipFree(nPixelTriplets);
    hipFree(totOccupancyPixelTriplets);
    hipFree(pixelRadius);
    hipFree(tripletRadius);
    hipFree(pt);
    hipFree(isDup);
    hipFree(partOfPT5);
    hipFree(centerX);
    hipFree(centerY);
    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(lowerModuleIndices);
#ifdef CUT_VALUE_DEBUG
    hipFree(pixelRadiusError);
    hipFree(rPhiChiSquared);
    hipFree(rPhiChiSquaredInwards);
    hipFree(rzChiSquared);
#endif
}

SDL::pixelTriplets::~pixelTriplets()
{
}

void SDL::pixelTriplets::resetMemory(unsigned int maxPixelTriplets,hipStream_t stream)
{
    hipMemsetAsync(pixelSegmentIndices,0, maxPixelTriplets * sizeof(unsigned int),stream);
    hipMemsetAsync(tripletIndices, 0,maxPixelTriplets * sizeof(unsigned int),stream);
    hipMemsetAsync(nPixelTriplets, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(totOccupancyPixelTriplets, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(pixelRadius, 0,maxPixelTriplets * sizeof(FPX),stream);
    hipMemsetAsync(tripletRadius, 0,maxPixelTriplets * sizeof(FPX),stream);
    hipMemsetAsync(pt, 0,maxPixelTriplets * 6*sizeof(FPX),stream);
    hipMemsetAsync(isDup, 0,maxPixelTriplets * sizeof(bool),stream);
    hipMemsetAsync(partOfPT5, 0,maxPixelTriplets * sizeof(bool),stream);
}
void SDL::createPixelTripletsInUnifiedMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets,hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    pixelTripletsInGPU.pixelSegmentIndices       =(unsigned int*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.tripletIndices            =(unsigned int*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.nPixelTriplets            =(unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    pixelTripletsInGPU.totOccupancyPixelTriplets =(unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    pixelTripletsInGPU.pixelRadius               =(FPX*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(FPX),stream);
    pixelTripletsInGPU.tripletRadius             =(FPX*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(FPX),stream);
    pixelTripletsInGPU.pt                        =(FPX*)cms::cuda::allocate_managed(maxPixelTriplets * 6*sizeof(FPX),stream);
    pixelTripletsInGPU.isDup                     =(bool*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(bool),stream);
    pixelTripletsInGPU.partOfPT5                 =(bool*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(bool),stream);

    pixelTripletsInGPU.centerX = (FPX*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(FPX),stream);
    pixelTripletsInGPU.centerY = (FPX*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(FPX),stream);
    pixelTripletsInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(uint16_t) * 5, stream);
    pixelTripletsInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(unsigned int) * 10, stream);
    pixelTripletsInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_managed(maxPixelTriplets * sizeof(uint8_t) * 5, stream);

#else
    hipMallocManaged(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMallocManaged(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMallocManaged(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMallocManaged(&pixelTripletsInGPU.totOccupancyPixelTriplets, sizeof(unsigned int));
    hipMallocManaged(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(FPX));
    hipMallocManaged(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(FPX));
    hipMallocManaged(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(FPX));
    hipMallocManaged(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool));
    hipMallocManaged(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool));

    hipMallocManaged(&pixelTripletsInGPU.centerX, maxPixelTriplets * sizeof(FPX));
    hipMallocManaged(&pixelTripletsInGPU.centerY, maxPixelTriplets * sizeof(FPX));
    hipMallocManaged(&pixelTripletsInGPU.logicalLayers, maxPixelTriplets * sizeof(uint8_t) * 5);
    hipMallocManaged(&pixelTripletsInGPU.hitIndices, maxPixelTriplets * sizeof(unsigned int) * 10);
    hipMallocManaged(&pixelTripletsInGPU.lowerModuleIndices, maxPixelTriplets * sizeof(uint16_t) * 5);
#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&pixelTripletsInGPU.pixelRadiusError, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.rPhiChiSquared, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.rPhiChiSquaredInwards, maxPixelTriplets * sizeof(float));
    hipMallocManaged(&pixelTripletsInGPU.rzChiSquared, maxPixelTriplets * sizeof(float));
#endif
#endif
    pixelTripletsInGPU.eta = pixelTripletsInGPU.pt + maxPixelTriplets;
    pixelTripletsInGPU.phi = pixelTripletsInGPU.pt + maxPixelTriplets * 2;
    pixelTripletsInGPU.eta_pix = pixelTripletsInGPU.pt + maxPixelTriplets *3;
    pixelTripletsInGPU.phi_pix = pixelTripletsInGPU.pt + maxPixelTriplets * 4;
    pixelTripletsInGPU.score = pixelTripletsInGPU.pt + maxPixelTriplets * 5;
    hipMemsetAsync(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int),stream);
    hipMemsetAsync(pixelTripletsInGPU.totOccupancyPixelTriplets, 0, sizeof(unsigned int),stream);
    hipMemsetAsync(pixelTripletsInGPU.partOfPT5, 0, maxPixelTriplets*sizeof(bool),stream);
    hipStreamSynchronize(stream);
}

void SDL::createPixelTripletsInExplicitMemory(struct pixelTriplets& pixelTripletsInGPU, unsigned int maxPixelTriplets, hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    pixelTripletsInGPU.pixelSegmentIndices       =(unsigned int*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.tripletIndices            =(unsigned int*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(unsigned int),stream);
    pixelTripletsInGPU.nPixelTriplets            =(unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    pixelTripletsInGPU.totOccupancyPixelTriplets =(unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    pixelTripletsInGPU.pixelRadius               =(FPX*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(FPX),stream);
    pixelTripletsInGPU.tripletRadius             =(FPX*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(FPX),stream);
    pixelTripletsInGPU.pt                        =(FPX*)cms::cuda::allocate_device(dev,maxPixelTriplets * 6*sizeof(FPX),stream);
    pixelTripletsInGPU.isDup                     =(bool*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(bool),stream);
    pixelTripletsInGPU.partOfPT5                 =(bool*)cms::cuda::allocate_device(dev,maxPixelTriplets * sizeof(bool),stream);
    pixelTripletsInGPU.centerX                   = (FPX*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(FPX), stream);
    pixelTripletsInGPU.centerY                   = (FPX*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(FPX), stream);
    pixelTripletsInGPU.lowerModuleIndices        = (uint16_t*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(uint16_t) * 5, stream);
    pixelTripletsInGPU.hitIndices                = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(unsigned int) * 10, stream);
    pixelTripletsInGPU.logicalLayers             = (uint8_t*)cms::cuda::allocate_device(dev, maxPixelTriplets * sizeof(uint8_t) * 5, stream);
#else
    hipMalloc(&pixelTripletsInGPU.pixelSegmentIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.tripletIndices, maxPixelTriplets * sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.nPixelTriplets, sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.totOccupancyPixelTriplets, sizeof(unsigned int));
    hipMalloc(&pixelTripletsInGPU.pixelRadius, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.tripletRadius, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.pt, maxPixelTriplets * 6*sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.isDup, maxPixelTriplets * sizeof(bool));
    hipMalloc(&pixelTripletsInGPU.partOfPT5, maxPixelTriplets * sizeof(bool));
    hipMalloc(&pixelTripletsInGPU.centerX, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.centerY, maxPixelTriplets * sizeof(FPX));
    hipMalloc(&pixelTripletsInGPU.logicalLayers, maxPixelTriplets * sizeof(uint8_t) * 5);
    hipMalloc(&pixelTripletsInGPU.hitIndices, maxPixelTriplets * sizeof(unsigned int) * 10);
    hipMalloc(&pixelTripletsInGPU.lowerModuleIndices, maxPixelTriplets * sizeof(uint16_t) * 5);
#endif
    hipMemsetAsync(pixelTripletsInGPU.nPixelTriplets, 0, sizeof(unsigned int),stream);
    hipMemsetAsync(pixelTripletsInGPU.totOccupancyPixelTriplets, 0, sizeof(unsigned int),stream);
    hipMemsetAsync(pixelTripletsInGPU.partOfPT5, 0, maxPixelTriplets*sizeof(bool),stream);
    hipStreamSynchronize(stream);

    pixelTripletsInGPU.eta = pixelTripletsInGPU.pt + maxPixelTriplets;
    pixelTripletsInGPU.phi = pixelTripletsInGPU.pt + maxPixelTriplets * 2;
    pixelTripletsInGPU.eta_pix = pixelTripletsInGPU.pt + maxPixelTriplets *3;
    pixelTripletsInGPU.phi_pix = pixelTripletsInGPU.pt + maxPixelTriplets * 4;
    pixelTripletsInGPU.score = pixelTripletsInGPU.pt + maxPixelTriplets * 5;

}

#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addPixelTripletToMemory(struct modules& modulesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, struct pixelTriplets& pixelTripletsInGPU, unsigned int pixelSegmentIndex, unsigned int tripletIndex, float pixelRadius, float pixelRadiusError, float tripletRadius, float centerX, float centerY, float rPhiChiSquared, float rPhiChiSquaredInwards, float rzChiSquared, unsigned int pixelTripletIndex, float pt, float eta, float phi, float eta_pix, float phi_pix, float score)
#else
__device__ void SDL::addPixelTripletToMemory(struct modules& modulesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, struct pixelTriplets& pixelTripletsInGPU, unsigned int pixelSegmentIndex, unsigned int tripletIndex, float pixelRadius, float tripletRadius, float centerX, float centerY, unsigned int pixelTripletIndex, float pt, float eta, float phi, float eta_pix, float phi_pix,float score)
#endif
{
    pixelTripletsInGPU.pixelSegmentIndices[pixelTripletIndex] = pixelSegmentIndex;
    pixelTripletsInGPU.tripletIndices[pixelTripletIndex] = tripletIndex;
    pixelTripletsInGPU.pixelRadius[pixelTripletIndex] = __F2H(pixelRadius);
    pixelTripletsInGPU.tripletRadius[pixelTripletIndex] = __F2H(tripletRadius);
    pixelTripletsInGPU.pt[pixelTripletIndex] = __F2H(pt);
    pixelTripletsInGPU.eta[pixelTripletIndex] = __F2H(eta);
    pixelTripletsInGPU.phi[pixelTripletIndex] = __F2H(phi);
    pixelTripletsInGPU.eta_pix[pixelTripletIndex] = __F2H(eta_pix);
    pixelTripletsInGPU.phi_pix[pixelTripletIndex] = __F2H(phi_pix);
    pixelTripletsInGPU.isDup[pixelTripletIndex] = 0;
    pixelTripletsInGPU.score[pixelTripletIndex] = __F2H(score);

    pixelTripletsInGPU.centerX[pixelTripletIndex] = __F2H(centerX);
    pixelTripletsInGPU.centerY[pixelTripletIndex] = __F2H(centerY);
    pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex] = 0;
    pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex + 1] = 0;
    pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex + 2] = tripletsInGPU.logicalLayers[tripletIndex * 3];
    pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex + 3] = tripletsInGPU.logicalLayers[tripletIndex * 3 + 1];
    pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex + 4] = tripletsInGPU.logicalLayers[tripletIndex * 3 + 2];

    pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex] = segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];
    pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex + 1] = segmentsInGPU.outerLowerModuleIndices[pixelSegmentIndex];
    pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex + 2] = tripletsInGPU.lowerModuleIndices[3 * tripletIndex];
     pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex + 3] = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 1];
      pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex + 4] = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 2];
 
    unsigned int pixelInnerMD = segmentsInGPU.mdIndices[2 * pixelSegmentIndex];
    unsigned int pixelOuterMD = segmentsInGPU.mdIndices[2 * pixelSegmentIndex + 1];

    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex] = mdsInGPU.anchorHitIndices[pixelInnerMD];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 1] = mdsInGPU.outerHitIndices[pixelInnerMD];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 2] = mdsInGPU.anchorHitIndices[pixelOuterMD];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 3] = mdsInGPU.outerHitIndices[pixelOuterMD];

    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 4] = tripletsInGPU.hitIndices[6 * tripletIndex];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 5] = tripletsInGPU.hitIndices[6 * tripletIndex + 1];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 6] = tripletsInGPU.hitIndices[6 * tripletIndex + 2];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 7] = tripletsInGPU.hitIndices[6 * tripletIndex + 3];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 8] = tripletsInGPU.hitIndices[6 * tripletIndex + 4];
    pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex + 9] = tripletsInGPU.hitIndices[6 * tripletIndex + 5];
#ifdef CUT_VALUE_DEBUG
    pixelTripletsInGPU.pixelRadiusError[pixelTripletIndex] = pixelRadiusError;
    pixelTripletsInGPU.rPhiChiSquared[pixelTripletIndex] = rPhiChiSquared;
    pixelTripletsInGPU.rPhiChiSquaredInwards[pixelTripletIndex] = rPhiChiSquaredInwards;
    pixelTripletsInGPU.rzChiSquared[pixelTripletIndex] = rzChiSquared;
#endif

}
__device__ void SDL::rmPixelTripletToMemory(struct pixelTriplets& pixelTripletsInGPU,unsigned int pixelTripletIndex)
{
    pixelTripletsInGPU.isDup[pixelTripletIndex] = 1;
}

__device__ float SDL::computeRadiusFromThreeAnchorHitspT3(float* xs, float* ys, float& g, float& f)
{
    float radius = 0;

    //writing manual code for computing radius, which obviously sucks
    //TODO:Use fancy inbuilt libraries like cuBLAS or cuSOLVE for this!
    //(g,f) -> center
    //first anchor hit - (x1,y1), second anchor hit - (x2,y2), third anchor hit - (x3, y3)

    /*
    if((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0)
    {
        return -1; //WTF man three collinear points!
    }
    */

    float denomInv = 1.f/((ys[0] - ys[2]) * (xs[1] - xs[2]) - (xs[0] - xs[2]) * (ys[1] - ys[2]));

    float xy1sqr = xs[0] * xs[0] + ys[0] * ys[0];

    float xy2sqr = xs[1] * xs[1] + ys[1] * ys[1];

    float xy3sqr = xs[2] * xs[2] + ys[2] * ys[2];

    g = 0.5f * ((ys[2] - ys[1]) * xy1sqr + (ys[0] - ys[2]) * xy2sqr + (ys[1] - ys[0]) * xy3sqr) * denomInv;

    f = 0.5f * ((xs[1] - xs[2]) * xy1sqr + (xs[2] - xs[0]) * xy2sqr + (xs[0] - xs[1]) * xy3sqr) * denomInv;

    float c = ((xs[1] * ys[2] - xs[2] * ys[1]) * xy1sqr + (xs[2] * ys[0] - xs[0] * ys[2]) * xy2sqr + (xs[0] * ys[1] - xs[1] * ys[0]) * xy3sqr) * denomInv;

    if(((ys[0] - ys[2]) * (xs[1] - xs[2]) - (xs[0] - xs[2]) * (ys[1] - ys[2]) == 0) || (g * g + f * f - c < 0))
    {
        printf("three collinear points or FATAL! r^2 < 0!\n");
  radius = -1;
    }
    else
      radius = sqrtf(g * g  + f * f - c);

    return radius;
}


__device__ bool SDL::runPixelTripletDefaultAlgo(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, unsigned int& pixelSegmentIndex, unsigned int tripletIndex, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, float& centerX, float& centerY, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards, bool runChiSquaredCuts)
{
    bool pass = true;

    //run pT4 compatibility between the pixel segment and inner segment, and between the pixel and outer segment of the triplet

    //placeholder
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    uint16_t pixelModuleIndex = segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];

    uint16_t lowerModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex];
    uint16_t middleModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 1];
    uint16_t upperModuleIndex = tripletsInGPU.lowerModuleIndices[3 * tripletIndex + 2];

    // pixel segment vs inner segment of the triplet
    pass = pass and runPixelTrackletDefaultAlgo(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, lowerModuleIndex, middleModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    if(not pass) return pass;

    //pixel segment vs outer segment of triplet
    pass = pass and runPixelTrackletDefaultAlgo(modulesInGPU, rangesInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, middleModuleIndex, upperModuleIndex, pixelSegmentIndex, tripletsInGPU.segmentIndices[2 * tripletIndex + 1], zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    if(not pass) return pass;

    //pt matching between the pixel ptin and the triplet circle pt
    unsigned int pixelSegmentArrayIndex = pixelSegmentIndex - rangesInGPU.segmentModuleIndices[pixelModuleIndex];
    float pixelSegmentPt = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
    float pixelSegmentPtError = segmentsInGPU.ptErr[pixelSegmentArrayIndex];

    unsigned int pixelInnerMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex];
    unsigned int pixelOuterMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex + 1];

    unsigned int pixelAnchorHitIndex1 = mdsInGPU.anchorHitIndices[pixelInnerMDIndex];
    unsigned int pixelAnchorHitIndex2 = mdsInGPU.anchorHitIndices[pixelOuterMDIndex];

    pixelRadius = pixelSegmentPt/(2.f * k2Rinv1GeVf);
    pixelRadiusError = pixelSegmentPtError/(2.f * k2Rinv1GeVf);
    unsigned int tripletInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex];
    unsigned int tripletOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * tripletIndex + 1];

    unsigned int firstMDIndex = segmentsInGPU.mdIndices[2 * tripletInnerSegmentIndex];
    unsigned int secondMDIndex = segmentsInGPU.mdIndices[2 * tripletInnerSegmentIndex + 1];
    unsigned int thirdMDIndex = segmentsInGPU.mdIndices[2 * tripletOuterSegmentIndex + 1];

    float g,f;
    float xs[] = {mdsInGPU.anchorX[firstMDIndex], mdsInGPU.anchorX[secondMDIndex], mdsInGPU.anchorX[thirdMDIndex]};
    float ys[] = {mdsInGPU.anchorY[firstMDIndex], mdsInGPU.anchorY[secondMDIndex], mdsInGPU.anchorY[thirdMDIndex]};

    tripletRadius = computeRadiusFromThreeAnchorHitspT3(xs, ys, g,f);    
    pass = pass and passRadiusCriterion(modulesInGPU, pixelRadius, pixelRadiusError, tripletRadius, lowerModuleIndex, middleModuleIndex, upperModuleIndex);
    if(not pass) return pass;

    float rts[] = {mdsInGPU.anchorRt[firstMDIndex], mdsInGPU.anchorRt[secondMDIndex], mdsInGPU.anchorRt[thirdMDIndex]};
    float zs[] = {mdsInGPU.anchorZ[firstMDIndex], mdsInGPU.anchorZ[secondMDIndex], mdsInGPU.anchorZ[thirdMDIndex]};
    float rtPix[] = {mdsInGPU.anchorRt[pixelInnerMDIndex], mdsInGPU.anchorRt[pixelOuterMDIndex]};
    float zPix[] = {mdsInGPU.anchorZ[pixelInnerMDIndex], mdsInGPU.anchorZ[pixelOuterMDIndex]};

    uint16_t lowerModuleIndices[] = {lowerModuleIndex, middleModuleIndex, upperModuleIndex};
    rzChiSquared = computePT3RZChiSquared(modulesInGPU, lowerModuleIndices, rtPix, zPix, rts, zs);

    if(runChiSquaredCuts and pixelSegmentPt < 5.0f)
    {
        pass = pass and passPT3RZChiSquaredCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rzChiSquared);
        if(not pass) return pass;
    }

    float pixelG = segmentsInGPU.circleCenterX[pixelSegmentArrayIndex];
    float pixelF = segmentsInGPU.circleCenterY[pixelSegmentArrayIndex];
    float pixelRadiusPCA = segmentsInGPU.circleRadius[pixelSegmentArrayIndex];

    rPhiChiSquared = computePT3RPhiChiSquared(modulesInGPU, lowerModuleIndices, pixelG, pixelF, pixelRadiusPCA, xs, ys);

    if(runChiSquaredCuts and pixelSegmentPt < 5.0f)
    {
        pass = pass and passPT3RPhiChiSquaredCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rPhiChiSquared);
        if(not pass) return pass;
    }

    float xPix[] = {mdsInGPU.anchorX[pixelInnerMDIndex], mdsInGPU.anchorX[pixelOuterMDIndex]};
    float yPix[] = {mdsInGPU.anchorY[pixelInnerMDIndex], mdsInGPU.anchorY[pixelOuterMDIndex]};
    rPhiChiSquaredInwards = computePT3RPhiChiSquaredInwards(modulesInGPU, g, f, tripletRadius, xPix, yPix);

    if(runChiSquaredCuts and pixelSegmentPt < 5.0f)
    {
        pass = pass and passPT3RPhiChiSquaredInwardsCuts(modulesInGPU, lowerModuleIndex, middleModuleIndex, upperModuleIndex, rPhiChiSquaredInwards);
        if(not pass) return pass;
    }

    return pass;
}

__device__ bool SDL::passPT3RPhiChiSquaredInwardsCuts(struct modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, float& chiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    
    if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return chiSquared < 22016.8055f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return chiSquared < 935179.56807f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return chiSquared < 29064.12959f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return chiSquared < 935179.5681f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return chiSquared < 1370.0113195101474f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return chiSquared < 5492.110048314815f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return chiSquared < 4160.410806470067f;
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 29064.129591225726f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return chiSquared < 12634.215376250893f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return chiSquared < 353821.69361145404f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 33393.26076341235f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return chiSquared < 935179.5680742573f;
    }

    return true;
}

__device__ float SDL::computePT3RPhiChiSquaredInwards(struct modules& modulesInGPU, float& g, float& f, float& r, float* xPix, float* yPix)
{
    float residual = (xPix[0] - g) * (xPix[0] -g) + (yPix[0] - f) * (yPix[0] - f) - r * r;
    float chiSquared = residual * residual;
    residual = (xPix[1] - g) * (xPix[1] -g) + (yPix[1] - f) * (yPix[1] - f) - r * r;
    chiSquared += residual * residual;

    //chiSquared /= 2;
    chiSquared *= 0.5f;
    return chiSquared;
}

__device__ bool SDL::passPT3RZChiSquaredCuts(struct modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, float& rzChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);

    if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return rzChiSquared < 85.2499f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return rzChiSquared < 85.2499f;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return rzChiSquared < 74.19805f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return rzChiSquared < 97.9479f;
    }

    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return rzChiSquared < 451.1407f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return rzChiSquared < 595.546f;
    }

    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return rzChiSquared < 518.339f;
    }

    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return rzChiSquared < 684.253f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return rzChiSquared < 684.253f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return rzChiSquared  < 392.654f;
    }

    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return rzChiSquared < 518.339f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return rzChiSquared < 518.339f;
    }

    //default - category not found!
    return true;
}

__device__ float SDL::computePT3RZChiSquared(struct modules& modulesInGPU, uint16_t* lowerModuleIndices, float* rtPix, float* zPix, float* rts, float* zs)
{ 
    float slope = (zPix[1] - zPix[0])/(rtPix[1] - rtPix[0]);
    float residual = 0;
    float error = 0;
    //hardcoded array indices!!!
    float RMSE = 0;
    for(size_t i = 0; i < 3; i++)
    {
        uint16_t& lowerModuleIndex = lowerModuleIndices[i];
        const int moduleType = modulesInGPU.moduleType[lowerModuleIndex];
        const int moduleSide = modulesInGPU.sides[lowerModuleIndex];
        const int moduleSubdet = modulesInGPU.subdets[lowerModuleIndex];

        residual = moduleSubdet == SDL::Barrel ? (zs[i] - zPix[0]) - slope * (rts[i] - rtPix[0]) : (rts[i] - rtPix[0]) - (zs[i] - zPix[0])/slope;
 
        //PS Modules
        if(moduleType == 0)
        {
            error = 0.15f;
        }
        else //2S modules
        {
            error = 5.0f;
        }

        //special dispensation to tilted PS modules!
        if(moduleType == 0 and moduleSubdet == SDL::Barrel and moduleSide != Center)
        {
            float& drdz = modulesInGPU.drdzs[lowerModuleIndex];
            error /= sqrtf(1 + drdz * drdz);
        }
        RMSE += (residual * residual)/(error * error);
    }

    RMSE = sqrtf(0.2f * RMSE); //the constant doesn't really matter....
    return RMSE;
}

//TODO: merge this one and the pT5 function later into a single function
__device__ float SDL::computePT3RPhiChiSquared(struct modules& modulesInGPU, uint16_t* lowerModuleIndices, float& g, float& f, float& radius, float* xs, float* ys)
{
    float delta1[3], delta2[3], slopes[3];
    bool isFlat[3];
    float chiSquared = 0;
    ModuleType moduleType;
    short moduleSubdet, moduleSide;
    float drdz;
    float inv1 = 0.01f/0.009f;
    float inv2 = 0.15f/0.009f;
    float inv3 = 2.4f/0.009f;
    for(size_t i = 0; i < 3; i++)
    {
        moduleType = modulesInGPU.moduleType[lowerModuleIndices[i]];
        moduleSubdet = modulesInGPU.subdets[lowerModuleIndices[i]];
        moduleSide = modulesInGPU.sides[lowerModuleIndices[i]];
        float& drdz = modulesInGPU.drdzs[lowerModuleIndices[i]];
        slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
        //category 1 - barrel PS flat
        if(moduleSubdet == Barrel and moduleType == PS and moduleSide == Center)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            delta2[i] = inv1;//1.1111f;//0.01;
            slopes[i] = -999;
            isFlat[i] = true;
        }

        //category 2 - barrel 2S
        else if(moduleSubdet == Barrel and moduleType == TwoS)
        {
            delta1[i] = 1;//0.009;
            delta2[i] = 1;//0.009;
            slopes[i] = -999;
            isFlat[i] = true;
        }

        //category 3 - barrel PS tilted
        else if(moduleSubdet == Barrel and moduleType == PS and moduleSide != Center)
        {

            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;
            delta2[i] = (inv2 * drdz/sqrtf(1 + drdz * drdz));
        }

        //category 4 - endcap PS
        else if(moduleSubdet == Endcap and moduleType == PS)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            /*despite the type of the module layer of the lower module index,
            all anchor hits are on the pixel side and all non-anchor hits are
            on the strip side!*/        
            delta2[i] = inv2;//16.6666f;//0.15f;
        }

        //category 5 - endcap 2S
        else if(moduleSubdet == Endcap and moduleType == TwoS)
        {
            delta1[i] = 1;//0.009;
            delta2[i] = 500*inv1;//555.5555f;//5.f;
            isFlat[i] = false;
        }
        else
        {
            printf("ERROR!!!!! I SHOULDN'T BE HERE!!!! subdet = %d, type = %d, side = %d\n", moduleSubdet, moduleType, moduleSide);
        }
    }
    // this for loop is kept to keep the physics results the same but I think this is a bug in the original code. This was kept at 5 and not nPoints
    for(size_t i = 3; i < 5; i++)
    {
        delta1[i] /= 0.009f;
        delta2[i] /= 0.009f;
    }
    chiSquared = computeChiSquared(3, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius);
    
    return chiSquared;
}


//90pc threshold
__device__ bool SDL::passPT3RPhiChiSquaredCuts(struct modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, float& chiSquared)
{

    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);

    if(layer1 == 8 and layer2 == 9 and layer3 == 10)
    {
        return chiSquared < 7.003f;
    }
    else if(layer1 == 8 and layer2 == 9 and layer3 == 15)
    {
        return chiSquared < 0.5f;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        return chiSquared < 8.046f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14)
    {
        return chiSquared < 0.575f;
    }

    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        return chiSquared < 5.304f;
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        return chiSquared < 10.6211f;
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 4.617f;
    }

    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        return chiSquared < 8.046f;
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13)
    {
        return chiSquared < 0.435f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        return chiSquared < 9.244f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12)
    {
        return chiSquared < 0.287f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        return chiSquared < 18.509f;
    }

    return true;
}

__device__ bool SDL::passRadiusCriterion(struct modules& modulesInGPU, float& pixelRadius, float& pixelRadiusError, float& tripletRadius, uint16_t& lowerModuleIndex, uint16_t& middleModuleIndex, uint16_t& upperModuleIndex)
{
    if(modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionEEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[middleModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBEE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else if(modulesInGPU.subdets[upperModuleIndex] == SDL::Endcap)
    {
        return passRadiusCriterionBBE(pixelRadius, pixelRadiusError, tripletRadius);
    }
    else
    {
        return passRadiusCriterionBBB(pixelRadius, pixelRadiusError, tripletRadius);
    }

}

/*bounds for high Pt taken from : http://uaf-10.t2.ucsd.edu/~bsathian/SDL/T5_efficiency/efficiencies/new_efficiencies/efficiencies_20210513_T5_recovering_high_Pt_efficiencies/highE_radius_matching/highE_bounds.txt */
__device__ bool SDL::passRadiusCriterionBBB(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0.15624f;
    float pixelInvRadiusErrorBound = 0.17235f;

    if(pixelRadius > 2.0f/(2.f * k2Rinv1GeVf))
    {
        pixelInvRadiusErrorBound = 0.6375f;
        tripletInvRadiusErrorBound = 0.6588f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);
}

__device__ bool SDL::passRadiusCriterionBBE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 0.45972f;
    float pixelInvRadiusErrorBound = 0.19644f;

    if(pixelRadius > 2.0f/(2 * k2Rinv1GeVf))
    {
        pixelInvRadiusErrorBound = 0.6805f;
        tripletInvRadiusErrorBound = 0.8557f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

__device__ bool SDL::passRadiusCriterionBEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 1.59294f;
    float pixelInvRadiusErrorBound = 0.255181f;

    if(pixelRadius > 2.0f/(2 * k2Rinv1GeVf)) //as good as not having selections
    {
        pixelInvRadiusErrorBound = 2.2091f;
        tripletInvRadiusErrorBound = 2.3548f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));
    pixelRadiusInvMin = fmaxf(pixelRadiusInvMin, 0);

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);

}

__device__ bool SDL::passRadiusCriterionEEE(float& pixelRadius, float& pixelRadiusError, float& tripletRadius)
{
    float tripletInvRadiusErrorBound = 1.7006f;
    float pixelInvRadiusErrorBound = 0.26367f;

    if(pixelRadius > 2.0f/(2 * k2Rinv1GeVf)) //as good as not having selections
    {
        pixelInvRadiusErrorBound = 2.286f;
        tripletInvRadiusErrorBound = 2.436f;
    }

    float tripletRadiusInvMax = (1 + tripletInvRadiusErrorBound)/tripletRadius;
    float tripletRadiusInvMin = fmaxf((1 - tripletInvRadiusErrorBound)/tripletRadius, 0);

    float pixelRadiusInvMax = fmaxf((1 + pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius - pixelRadiusError));
    float pixelRadiusInvMin = fminf((1 - pixelInvRadiusErrorBound)/pixelRadius, 1.f/(pixelRadius + pixelRadiusError));
    pixelRadiusInvMin = fmaxf(0, pixelRadiusInvMin);

    return checkIntervalOverlap(tripletRadiusInvMin, tripletRadiusInvMax, pixelRadiusInvMin, pixelRadiusInvMax);
}

