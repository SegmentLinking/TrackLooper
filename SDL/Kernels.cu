#include "hip/hip_runtime.h"
#include "Kernels.cuh"
#include "allocate.h"

#ifndef NESTED_PARA
__global__ void createMiniDoubletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU)
{
    int lowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    //int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;
    if(lowerModuleArrayIndex >= (*modulesInGPU.nLowerModules)) return; //extra precaution

    int lowerModuleIndex = modulesInGPU.lowerModuleIndices[lowerModuleArrayIndex];
    int upperModuleIndex = modulesInGPU.partnerModuleIndex(lowerModuleIndex);

    if(modulesInGPU.hitRanges[lowerModuleIndex * 2] == -1) return;
    if(modulesInGPU.hitRanges[upperModuleIndex * 2] == -1) return;
    unsigned int nLowerHits = modulesInGPU.hitRanges[lowerModuleIndex * 2 + 1] - modulesInGPU.hitRanges[lowerModuleIndex * 2] + 1;
    unsigned int nUpperHits = modulesInGPU.hitRanges[upperModuleIndex * 2 + 1] - modulesInGPU.hitRanges[upperModuleIndex * 2] + 1;

#ifdef NEWGRID_MD
    int lowerHitIndex =  (blockIdx.y * blockDim.y + threadIdx.y) / nUpperHits;
    int upperHitIndex =  (blockIdx.y * blockDim.y + threadIdx.y) % nUpperHits;
#else
    int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;
#endif

    //consider assigining a dummy computation function for these
    if(lowerHitIndex >= nLowerHits) return;
    if(upperHitIndex >= nUpperHits) return;

    unsigned int lowerHitArrayIndex = modulesInGPU.hitRanges[lowerModuleIndex * 2] + lowerHitIndex;
    unsigned int upperHitArrayIndex = modulesInGPU.hitRanges[upperModuleIndex * 2] + upperHitIndex;

    float dz, drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;

#ifdef CUT_VALUE_DEBUG
    float dzCut, drtCut, miniCut;
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz,  drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut);
#else
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange);
#endif

    if(success)
    {
        unsigned int mdModuleIndex = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);
        if(mdModuleIndex >= N_MAX_MD_PER_MODULES)
        {
            #ifdef Warnings
            if(mdModuleIndex == N_MAX_MD_PER_MODULES)
                printf("Mini-doublet excess alert! Module index =  %d\n",lowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int mdIndex = lowerModuleIndex * N_MAX_MD_PER_MODULES + mdModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz,drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut, mdIndex);
#else
        addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIndex);
#endif

        }

    }
}
#else
__global__ void createMiniDoubletsFromLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, unsigned int lowerModuleIndex, unsigned int upperModuleIndex, unsigned int nLowerHits, unsigned int nUpperHits)
{
    unsigned int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;

    //consider assigining a dummy computation function for these
    if(lowerHitIndex >= nLowerHits) return;
    if(upperHitIndex >= nUpperHits) return;

    unsigned int lowerHitArrayIndex = modulesInGPU.hitRanges[lowerModuleIndex * 2] + lowerHitIndex;
    unsigned int upperHitArrayIndex = modulesInGPU.hitRanges[upperModuleIndex * 2] + upperHitIndex;

    float dz, drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;

#ifdef CUT_VALUE_DEBUG
    float dzCut, drtCut, miniCut;
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz,  drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut);
#else
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange);
#endif

    if(success)
    {
        unsigned int mdModuleIndex = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);

        if(mdModuleIndex >= N_MAX_MD_PER_MODULES)
        {
            #ifdef Warnings
            if(mdModuleIndex == N_MAX_MD_PER_MODULES)
                printf("Mini-doublet excess alert! Module index = %d\n",lowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int mdIndex = lowerModuleIndex * N_MAX_MD_PER_MODULES + mdModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz,drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut, mdIndex);
#else
            addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIndex);
#endif
        }

    }
}


__global__ void createMiniDoubletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU)
{
    int lowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(lowerModuleArrayIndex >= (*modulesInGPU.nLowerModules)) return; //extra precaution

    int lowerModuleIndex = modulesInGPU.lowerModuleIndices[lowerModuleArrayIndex];
    int upperModuleIndex = modulesInGPU.partnerModuleIndex(lowerModuleIndex);

    if(modulesInGPU.hitRanges[lowerModuleIndex * 2] == -1) return;
    if(modulesInGPU.hitRanges[upperModuleIndex * 2] == -1) return;

    unsigned int nLowerHits = modulesInGPU.hitRanges[lowerModuleIndex * 2 + 1] - modulesInGPU.hitRanges[lowerModuleIndex * 2] + 1;
    unsigned int nUpperHits = modulesInGPU.hitRanges[upperModuleIndex * 2 + 1] - modulesInGPU.hitRanges[upperModuleIndex * 2] + 1;

    dim3 nThreads(1,16,16);
    dim3 nBlocks(1,nLowerHits % nThreads.y == 0 ? nLowerHits/nThreads.y : nLowerHits/nThreads.y + 1, nUpperHits % nThreads.z == 0 ? nUpperHits/nThreads.z : nUpperHits/nThreads.z + 1);

    createMiniDoubletsFromLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, lowerModuleIndex, upperModuleIndex, nLowerHits, nUpperHits);


}
#endif

#ifndef NESTED_PARA
__global__ void createSegmentsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{
#ifdef NEWGRID_Seg
    int innerLowerModuleArrayIdx = blockIdx.z * blockDim.z + threadIdx.z;
    int outerLowerModuleArrayIdx = blockIdx.y * blockDim.y + threadIdx.y;
#else
    int xAxisIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int innerMDArrayIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int outerMDArrayIdx = blockIdx.z * blockDim.z + threadIdx.z;

    int innerLowerModuleArrayIdx = xAxisIdx/MAX_CONNECTED_MODULES;
    int outerLowerModuleArrayIdx = xAxisIdx % MAX_CONNECTED_MODULES; //need this index from the connected module array
#endif
    if(innerLowerModuleArrayIdx >= *modulesInGPU.nLowerModules) return;

    unsigned int innerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerLowerModuleArrayIdx];

    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerLowerModuleIndex];

    if(outerLowerModuleArrayIdx >= nConnectedModules) return;

    unsigned int outerLowerModuleIndex = modulesInGPU.moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + outerLowerModuleArrayIdx];

    unsigned int nInnerMDs = mdsInGPU.nMDs[innerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[innerLowerModuleIndex];
    unsigned int nOuterMDs = mdsInGPU.nMDs[outerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[outerLowerModuleIndex];

#ifdef NEWGRID_Seg
    if (nInnerMDs*nOuterMDs == 0) return;
    int innerMDArrayIdx = (blockIdx.x * blockDim.x + threadIdx.x) / nOuterMDs;
    int outerMDArrayIdx = (blockIdx.x * blockDim.x + threadIdx.x) % nOuterMDs;
#endif

    if(innerMDArrayIdx >= nInnerMDs) return;
    if(outerMDArrayIdx >= nOuterMDs) return;

    unsigned int innerMDIndex = modulesInGPU.mdRanges[innerLowerModuleIndex * 2] + innerMDArrayIdx;
    unsigned int outerMDIndex = modulesInGPU.mdRanges[outerLowerModuleIndex * 2] + outerMDArrayIdx;

    float zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD;

    unsigned int innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex;

    dPhiMin = 0;
    dPhiMax = 0;
    dPhiChangeMin = 0;
    dPhiChangeMax = 0;
    float zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold, dAlphaInnerMDOuterMDThreshold;

    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
            dAlphaInnerMDOuterMDThreshold, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);

    if(success)
    {
        unsigned int segmentModuleIdx = atomicAdd(&segmentsInGPU.nSegments[innerLowerModuleIndex],1);
        if(segmentModuleIdx >= N_MAX_SEGMENTS_PER_MODULE)
        {
            #ifdef Warnings
            if(segmentModuleIdx == N_MAX_SEGMENTS_PER_MODULE)
                printf("Segment excess alert! Module index = %d\n",innerLowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int segmentIdx = innerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentModuleIdx;
#ifdef CUT_VALUE_DEBUG
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
                dAlphaInnerMDOuterMDThreshold, segmentIdx);
#else
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, segmentIdx);
#endif

        }
    }
}
#else

__global__ void createSegmentsFromInnerLowerModule(struct SDL::modules&modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerLowerModuleIndex, unsigned int nInnerMDs)
{
    unsigned int outerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int innerMDArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int outerMDArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    unsigned int outerLowerModuleIndex = modulesInGPU.moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + outerLowerModuleArrayIndex];

    unsigned int nOuterMDs = mdsInGPU.nMDs[outerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[outerLowerModuleIndex];
    if(innerMDArrayIndex >= nInnerMDs) return;
    if(outerMDArrayIndex >= nOuterMDs) return;

    unsigned int innerMDIndex = innerLowerModuleIndex * N_MAX_MD_PER_MODULES + innerMDArrayIndex;
    unsigned int outerMDIndex = outerLowerModuleIndex * N_MAX_MD_PER_MODULES + outerMDArrayIndex;

    float zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD;

    unsigned int innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex;

    dPhiMin = 0;
    dPhiMax = 0;
    dPhiChangeMin = 0;
    dPhiChangeMax = 0;
    float zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold, dAlphaInnerMDOuterMDThreshold;

    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
            dAlphaInnerMDOuterMDThreshold, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);


    if(success)
    {
        unsigned int segmentModuleIdx = atomicAdd(&segmentsInGPU.nSegments[innerLowerModuleIndex],1);
        if(segmentModuleIdx >= N_MAX_SEGMENTS_PER_MODULE)
        {
            #ifdef Warnings
            if(segmentModuleIdx == N_MAX_SEGMENTS_PER_MODULE)
                printf("Segment excess alert! Module index = %d\n",innerLowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int segmentIdx = innerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentModuleIdx;
#ifdef CUT_VALUE_DEBUG
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
                dAlphaInnerMDOuterMDThreshold, segmentIdx);
#else
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, segmentIdx);
#endif

        }

    }

}

__global__ void createSegmentsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{
    int innerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerLowerModuleArrayIndex];
    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerLowerModuleIndex];
    unsigned int nInnerMDs = mdsInGPU.nMDs[innerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[innerLowerModuleIndex];

    if(nConnectedModules == 0) return;

    if(nInnerMDs == 0) return;
    dim3 nThreads(1,16,16);
    dim3 nBlocks((nConnectedModules % nThreads.x == 0 ? nConnectedModules/nThreads.x : nConnectedModules/nThreads.x + 1), (nInnerMDs % nThreads.y == 0 ? nInnerMDs/nThreads.y : nInnerMDs/nThreads.y + 1), (N_MAX_MD_PER_MODULES % nThreads.z == 0 ? N_MAX_MD_PER_MODULES/nThreads.z : N_MAX_MD_PER_MODULES/nThreads.z + 1));

    createSegmentsFromInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerLowerModuleIndex,nInnerMDs);

}
#endif

#ifndef NESTED_PARA
#ifdef NEWGRID_Tracklet
__global__ void createTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int *index_gpu)
{
  //int innerInnerLowerModuleArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;
  int innerInnerLowerModuleArrayIndex = index_gpu[blockIdx.z * blockDim.z + threadIdx.z];
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
  unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];

  if(nInnerSegments == 0) return;

  int outerInnerLowerModuleArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  int innerSegmentArrayIndex = (blockIdx.x * blockDim.x + threadIdx.x) % nInnerSegments;
  int outerSegmentArrayIndex = (blockIdx.x * blockDim.x + threadIdx.x) / nInnerSegments;

  if(innerSegmentArrayIndex >= nInnerSegments) return;

  //outer inner lower module array indices should be obtained from the partner module of the inner segment's outer lower module
  unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

  unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

  //number of possible outer segment inner MD lower modules
  unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
  if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

  unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

  unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
  if(outerSegmentArrayIndex >= nOuterSegments) return;

  unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

  //for completeness - outerOuterLowerModuleIndex
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

  //with both segment indices obtained, run the tracklet algorithm
  float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta;

    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses


  if(success)
    {
      unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
      if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
      {
          #ifdef Warnings
          if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
              printf("Tracklet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
          #endif
      }
      else
      {
          unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
          addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
          addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);

#endif

      }
    }
}
#endif
#else
__global__ void createTrackletsFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    if(innerSegmentArrayIndex >= nInnerSegments) return;
        //outer inner lower module array indices should be obtained from the partner module of the inner segment's outer lower module
    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;


    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    //number of possible outer segment inner MD lower modules
    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm

    float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta;

    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses

   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
        {
            #ifdef Warnings
            if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
                printf("Tracklet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);

#endif
        }
   }



}

__global__ void createTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
  int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
  unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
  if(nInnerSegments == 0) return;

  dim3 nThreads(1,16,16);
  dim3 nBlocks(MAX_CONNECTED_MODULES % nThreads.x  == 0 ? MAX_CONNECTED_MODULES / nThreads.x : MAX_CONNECTED_MODULES / nThreads.x + 1 ,nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,N_MAX_SEGMENTS_PER_MODULE % nThreads.z == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.z : N_MAX_SEGMENTS_PER_MODULE/nThreads.z + 1);

  createTrackletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex,nInnerSegments,innerInnerLowerModuleArrayIndex);

}
#endif


#ifdef NEWGRID_Tracklet
__global__ void createTrackletsFromTriplets(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::tracklets& trackletsInGPU,unsigned int *threadIdx_gpu, unsigned int *threadIdx_gpu_offset)
{

  int innerInnerLowerModuleArrayIndex = threadIdx_gpu[blockIdx.y * blockDim.y + threadIdx.y];
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int nTriplets = tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex] > N_MAX_TRIPLETS_PER_MODULE ? N_MAX_TRIPLETS_PER_MODULE : tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex];

  if(nTriplets == 0) return;
  int innerTripletArrayIndex = threadIdx_gpu_offset[blockIdx.y * blockDim.y + threadIdx.y];
  int outerTripletArrayIndex = (blockIdx.x * blockDim.x + threadIdx.x);

//////////////////////////////////////////////////////////
  if(innerTripletArrayIndex >= nTriplets) return;

  //outer inner lower module array indices should be obtained from the partner module of the inner Triplet's outer lower module
  unsigned int innerTripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + innerTripletArrayIndex;
  unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 1]];//same as innerOuterInnerLowerModuleIndex
        if(outerTripletArrayIndex < fminf(tripletsInGPU.nTriplets[outerInnerInnerLowerModuleIndex],N_MAX_TRIPLETS_PER_MODULE))
        {
            unsigned int outerTripletIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;
            unsigned int innerOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
            unsigned int outerInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];

            if(innerOuterSegmentIndex == outerInnerSegmentIndex)
            {
              unsigned int innerSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex];
              unsigned int outerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex + 1];
              unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];
              unsigned int outerInnerLowerModuleIndex = segmentsInGPU.innerLowerModuleIndices[outerSegmentIndex];
              unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];
              float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta;
              unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];

              float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
              bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses

              if(success)
              {
                   unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
                   if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
                   {
                       #ifdef Warnings
                       if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
                           printf("Tracklet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
                       #endif
                   }
                   else
                   {
                       unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
                        #ifdef CUT_VALUE_DEBUG
                       addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);
                        #else
                       addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);
                        #endif
                   }
              }
            }
        }
}
#else
__global__ void createTrackletsFromTriplets(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::tracklets& trackletsInGPU/*,unsigned int *index_gpu*/)
{
  int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int nTriplets = tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex] > N_MAX_TRIPLETS_PER_MODULE ? N_MAX_TRIPLETS_PER_MODULE : tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex];

  if(nTriplets == 0) return;
    dim3 nThreads(16,16,1);
    dim3 nBlocks(nTriplets % nThreads.x == 0 ? nTriplets / nThreads.x : nTriplets / nThreads.x + 1, N_MAX_TRIPLETS_PER_MODULE % nThreads.y == 0 ? N_MAX_TRIPLETS_PER_MODULE / nThreads.y : N_MAX_TRIPLETS_PER_MODULE / nThreads.y + 1, 1);
    createTrackletsFromTripletsP2<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,tripletsInGPU,trackletsInGPU,innerInnerLowerModuleArrayIndex,nTriplets);

}
__global__ void createTrackletsFromTripletsP2(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::tracklets& trackletsInGPU/*,unsigned int *index_gpu*/,unsigned int innerInnerLowerModuleArrayIndex, unsigned int nTriplets)
{
  int innerTripletArrayIndex = (blockIdx.x * blockDim.x + threadIdx.x);// % nTriplets;
  int outerTripletArrayIndex = (blockIdx.y * blockDim.y + threadIdx.y);// / nTriplets;
  if(innerTripletArrayIndex >= nTriplets) return;

  //outer inner lower module array indices should be obtained from the partner module of the inner Triplet's outer lower module
  unsigned int innerTripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + innerTripletArrayIndex;
  unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 1]];//same as innerOuterInnerLowerModuleIndex
        if(outerTripletArrayIndex < fminf(tripletsInGPU.nTriplets[outerInnerInnerLowerModuleIndex],N_MAX_TRIPLETS_PER_MODULE))
        {
            unsigned int outerTripletIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;
            unsigned int innerOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
            unsigned int outerInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];

            if(innerOuterSegmentIndex == outerInnerSegmentIndex)
            {
              unsigned int innerSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex];
              unsigned int outerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex + 1];
              unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];
              unsigned int outerInnerLowerModuleIndex = segmentsInGPU.innerLowerModuleIndices[outerSegmentIndex];
              unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];
              float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta;
              unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];

              float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
              bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses

              if(success)
              {
                   unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
                   if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
                   {
                       #ifdef Warnings
                       if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
                           printf("Tracklet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
                       #endif
                   }
                   else
                   {
                       unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
                        #ifdef CUT_VALUE_DEBUG
                       addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);
                        #else
                       addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);
                        #endif
                   }
              }
            }
        }
}
#endif
#ifndef NESTED_PARA
__global__ void createPixelTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, unsigned int* threadIdx_gpu, unsigned int *threadIdx_gpu_offset)
{
  int outerInnerLowerModuleArrayIndex = threadIdx_gpu[blockIdx.y * blockDim.y + threadIdx.y];
  if(outerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

  unsigned int outerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerInnerLowerModuleArrayIndex];
  unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1; //last dude
  unsigned int pixelLowerModuleArrayIndex = modulesInGPU.reverseLookupLowerModuleIndices[pixelModuleIndex]; //should be the same as nLowerModules
  unsigned int nInnerSegments = segmentsInGPU.nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[pixelModuleIndex];
  unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
  if(nOuterSegments == 0) return;
  if(nInnerSegments == 0) return;
  if(modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::TwoS) return; //REMOVES 2S-2S

  int innerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int outerSegmentArrayIndex = threadIdx_gpu_offset[blockIdx.y * blockDim.y + threadIdx.y];
  if(innerSegmentArrayIndex >= nInnerSegments) return;
  if(outerSegmentArrayIndex >= nOuterSegments) return;
  unsigned int innerSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;
  unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];
  if(modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS) return; //REMOVES PS-2S
  float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta;

  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
  bool success = runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
  if(success)
    {
      unsigned int trackletModuleIndex = atomicAdd(pixelTrackletsInGPU.nPixelTracklets, 1);
      if(trackletModuleIndex >= N_MAX_PIXEL_TRACKLETS_PER_MODULE)
        {
            #ifdef Warnings
	  if(trackletModuleIndex == N_MAX_PIXEL_TRACKLETS_PER_MODULE)
	    printf("Pixel Tracklet excess alert! Module index = %d\n",pixelModuleIndex);
            #endif
        }
      else
        {
	  unsigned int trackletIndex = trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
	  addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);
#else
	  addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);
#endif
        }
    }
}
__global__ void createPixelTrackletsInGPUFromMap(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, unsigned int* connectedPixelSize, unsigned int* connectedPixelIndex,unsigned int nInnerSegs,unsigned int* seg_pix_gpu, unsigned int* seg_pix_gpu_offset, unsigned int totalSegs)
{
  //newgrid with map
  unsigned int offsetIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if(offsetIndex >= totalSegs) return;

  int segmentArrayIndex = seg_pix_gpu_offset[offsetIndex];  
  int pixelArrayIndex = seg_pix_gpu[offsetIndex];  
  if(pixelArrayIndex >= nInnerSegs) return;// don't exceed # of pLS
  if( segmentArrayIndex >= connectedPixelSize[pixelArrayIndex]) return; // don't exceed # connected segment modules for this pixel

  unsigned int outerInnerLowerModuleArrayIndex;// This will be the index of the module that connects to this pixel.
    unsigned int temp = connectedPixelIndex[pixelArrayIndex]+segmentArrayIndex; //gets module index for segment
    outerInnerLowerModuleArrayIndex = modulesInGPU.connectedPixels[temp]; //gets module index for segment
  if(outerInnerLowerModuleArrayIndex >= *modulesInGPU.nModules - 1) return;
  unsigned int outerInnerLowerModuleIndex = /*modulesInGPU.lowerModuleIndices[*/outerInnerLowerModuleArrayIndex;//];

  unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1; //last dude
  unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
  if(nOuterSegments == 0) return;
  if(modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::TwoS) return; //REMOVES 2S-2S

//  int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;
  int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if(outerSegmentArrayIndex >= nOuterSegments) return;
  unsigned int innerSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + pixelArrayIndex;
  unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];
  if(modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS) return; //REMOVES PS-2S
  float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta;

  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
  bool success = runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
  if(success)
    {
      unsigned int trackletModuleIndex = atomicAdd(pixelTrackletsInGPU.nPixelTracklets, 1);
      if(trackletModuleIndex >= N_MAX_PIXEL_TRACKLETS_PER_MODULE)
        {
            #ifdef Warnings
	  if(trackletModuleIndex == N_MAX_PIXEL_TRACKLETS_PER_MODULE)
	    printf("Pixel Tracklet excess alert! Module index = %d\n",pixelModuleIndex);
            #endif
        }
      else
        {
	  unsigned int trackletIndex = trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
	  addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);
#else
	  addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);
#endif
        }
    }
}

#else
__global__ void createPixelTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU)
{
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x; // loop for modules for segments lower hit
    if(outerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return; // don't exceed number of modules

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerInnerLowerModuleArrayIndex]; // correspond to module number index
    unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1; // pixel module index
    unsigned int pixelLowerModuleArrayIndex = modulesInGPU.reverseLookupLowerModuleIndices[pixelModuleIndex]; //should be the same as nLowerModules
    unsigned int nInnerSegments = segmentsInGPU.nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[pixelModuleIndex]; // number of pLS
    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex]; // number of segments from module corresponding to each module.
    if(nOuterSegments == 0) return;
    if(nInnerSegments == 0) return;
    if(modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::TwoS) return; //REMOVES 2S-2S

    dim3 nThreads(16,16,1);
    dim3 nBlocks(nInnerSegments % nThreads.x == 0 ? nInnerSegments / nThreads.x : nInnerSegments / nThreads.x + 1, nOuterSegments % nThreads.y == 0 ? nOuterSegments / nThreads.y : nOuterSegments / nThreads.y + 1, 1);

    createPixelTrackletsFromOuterInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelTrackletsInGPU, outerInnerLowerModuleIndex, nInnerSegments, nOuterSegments, pixelModuleIndex, pixelLowerModuleArrayIndex);

}
__global__ void createPixelTrackletsFromOuterInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, unsigned int outerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int nOuterSegments, unsigned int pixelModuleIndex, unsigned int pixelLowerModuleArrayIndex)
{
    int innerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;// looping over pixels
    int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;//looping over segments
    if(innerSegmentArrayIndex >= nInnerSegments) return; // not over # of pLS
    if(outerSegmentArrayIndex >= nOuterSegments) return; // not over # of segments for this module
    unsigned int innerSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex; // get this pixel index Just innerSegmentArrayIndex'th value (1-pLS)
    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex; // get this segment Index for this this module
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex]; // get corresponding outer module index for this segment
    if(modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS) return; //REMOVES PS-2S
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta;
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runPixelTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses

    if(success)
    {
        unsigned int trackletModuleIndex = atomicAdd(pixelTrackletsInGPU.nPixelTracklets, 1);
        if(trackletModuleIndex >= N_MAX_PIXEL_TRACKLETS_PER_MODULE)
        {
            #ifdef Warnings
            if(trackletModuleIndex == N_MAX_PIXEL_TRACKLETS_PER_MODULE)
                printf("Pixel Tracklet excess alert! Module index = %d\n",pixelModuleIndex);
            #endif
        }
        else
        {
            unsigned int trackletIndex = trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
                addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
            addPixelTrackletToMemory(pixelTrackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);
#endif
        }
    }
}
#endif

__global__ void createTrackletsWithAGapFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    //Proposal 1 : Inner kernel takes care of both loops
    int xAxisIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex =  blockIdx.y * blockDim.y + threadIdx.y;
    int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    if(innerSegmentArrayIndex >= nInnerSegments) return;

    int middleLowerModuleArrayIndex = xAxisIndex / MAX_CONNECTED_MODULES;
    int outerInnerLowerModuleArrayIndex = xAxisIndex % MAX_CONNECTED_MODULES;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;
    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    //first check for middle modules
    unsigned int nMiddleLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
    if(middleLowerModuleArrayIndex >= nMiddleLowerModules) return;

    unsigned int middleLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + middleLowerModuleArrayIndex];

    //second check for outerInnerLowerMoules
    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[middleLowerModuleIndex];
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[middleLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm

   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut, pt_beta;
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
        {
            #ifdef Warnings
            if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
                 printf("T4x excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
            #endif
        }
        else
        {

            unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,trackletIndex);
#endif

        }
   }
}

__global__ void createTrackletsWithAGapInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    //outer kernel for proposal 1
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    dim3 nThreads(1,16,16);
    dim3 nBlocks((MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) % nThreads.x  == 0 ? (MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) / nThreads.x : (MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) / nThreads.x + 1 ,nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,N_MAX_SEGMENTS_PER_MODULE % nThreads.z == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.z : N_MAX_SEGMENTS_PER_MODULE/nThreads.z + 1);

    createTrackletsWithAGapFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex,nInnerSegments,innerInnerLowerModuleArrayIndex);

}

#ifndef NESTED_PARA
#ifdef NEWGRID_Trips
__global__ void createTripletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, unsigned int *index_gpu)
{
  int innerInnerLowerModuleArrayIndex = index_gpu[blockIdx.z * blockDim.z + threadIdx.z];
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

  unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
  unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerInnerLowerModuleIndex];
  if(nConnectedModules == 0) return;

  unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];

  int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  int outerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if(innerSegmentArrayIndex >= nInnerSegments) return;

  unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

  //middle lower module - outer lower module of inner segment
  unsigned int middleLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

  unsigned int nOuterSegments = segmentsInGPU.nSegments[middleLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[middleLowerModuleIndex];
  if(outerSegmentArrayIndex >= nOuterSegments) return;

  unsigned int outerSegmentIndex = middleLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

  float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut, pt_beta;
  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    bool success = runTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

  if(success)
    {
      unsigned int tripletModuleIndex = atomicAdd(&tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex], 1);
      if(tripletModuleIndex >= N_MAX_TRIPLETS_PER_MODULE)
      {
          #ifdef Warnings
          if(tripletModuleIndex == N_MAX_TRIPLETS_PER_MODULE)
              printf("Triplet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
          #endif
      }
      unsigned int tripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + tripletModuleIndex;
#ifdef CUT_VALUE_DEBUG

        addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut,pt_beta, zLo,zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, tripletIndex);

#else
      addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, betaIn, betaOut, pt_beta, tripletIndex);
#endif
    }
}
#endif
#else
__global__ void createTripletsFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int nConnectedModules, unsigned int innerInnerLowerModuleArrayIndex)
{
    int innerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if(innerSegmentArrayIndex >= nInnerSegments) return;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

    //middle lower module - outer lower module of inner segment
    unsigned int middleLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[middleLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[middleLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;
    unsigned int outerSegmentIndex = middleLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta;
  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    bool success = runTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    if(success)
    {
        unsigned int tripletModuleIndex = atomicAdd(&tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex], 1);
        if(tripletModuleIndex >= N_MAX_TRIPLETS_PER_MODULE)
        {
            #ifdef Warnings
            if(tripletModuleIndex == N_MAX_TRIPLETS_PER_MODULE)
                printf("Triplet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
            #endif
        }
        else
        {
            unsigned int tripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + tripletModuleIndex;
#ifdef CUT_VALUE_DEBUG

            addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo,zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, tripletIndex);

#else
        addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, betaIn, betaOut, pt_beta, tripletIndex);
#endif

        }
    }
}

__global__ void createTripletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU)
{
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex] ;
    if(nInnerSegments == 0) return;

    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerInnerLowerModuleIndex];
    if(nConnectedModules == 0) return;

    dim3 nThreads(16,16,1);
    dim3 nBlocks(nInnerSegments % nThreads.x == 0 ? nInnerSegments / nThreads.x : nInnerSegments / nThreads.x + 1, N_MAX_SEGMENTS_PER_MODULE % nThreads.y == 0 ? N_MAX_SEGMENTS_PER_MODULE / nThreads.y : N_MAX_SEGMENTS_PER_MODULE / nThreads.y + 1);

    createTripletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, innerInnerLowerModuleIndex, nInnerSegments, nConnectedModules, innerInnerLowerModuleArrayIndex);
}
#endif

__global__ void addT5asTrackCandidateInGPU(struct SDL::modules& modulesInGPU,struct SDL::quintuplets& quintupletsInGPU,struct SDL::trackCandidates& trackCandidatesInGPU)
{

  int innerInnerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules or modulesInGPU.quintupletModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1) return;
  unsigned int nQuints = quintupletsInGPU.nQuintuplets[innerInnerInnerLowerModuleArrayIndex];
  if (nQuints > N_MAX_QUINTUPLETS_PER_MODULE) {nQuints = N_MAX_QUINTUPLETS_PER_MODULE;}
  int innerObjectArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if(innerObjectArrayIndex >= nQuints) return;
  int quintupletIndex = modulesInGPU.quintupletModuleIndices[innerInnerInnerLowerModuleArrayIndex] + innerObjectArrayIndex;

  //don't add duplicate T5s or T5s that are accounted in pT5s
  if(quintupletsInGPU.isDup[quintupletIndex] or quintupletsInGPU.partOfPT5[quintupletIndex])
  {
      return;
  }

  unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
  atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT5[innerInnerInnerLowerModuleArrayIndex],1);
  unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
  addTrackCandidateToMemory(trackCandidatesInGPU, 4/*track candidate type T5=4*/, quintupletIndex, quintupletIndex, trackCandidateIdx);
}

__global__ void addpT2asTrackCandidateInGPU(struct SDL::modules& modulesInGPU,struct SDL::pixelTracklets& pixelTrackletsInGPU,struct SDL::trackCandidates& trackCandidatesInGPU)
{
  int pixelTrackletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int pixelLowerModuleArrayIndex = *modulesInGPU.nLowerModules;
  unsigned int nPixelTracklets = *pixelTrackletsInGPU.nPixelTracklets;
  if(pixelTrackletArrayIndex >= nPixelTracklets) return;
  int pixelTrackletIndex = pixelTrackletArrayIndex;
  unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
  atomicAdd(trackCandidatesInGPU.nTrackCandidatespT2,1);
  unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
  addTrackCandidateToMemory(trackCandidatesInGPU, 3/*track candidate type pT2=3*/, pixelTrackletIndex, pixelTrackletIndex, trackCandidateIdx);
}

__global__ void addpT3asTrackCandidateInGPU(struct SDL::modules& modulesInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU,struct SDL::segments& segmentsInGPU, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU)
{
  int pixelTripletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int pixelLowerModuleArrayIndex = *modulesInGPU.nLowerModules;
  unsigned int nPixelTriplets = *pixelTripletsInGPU.nPixelTriplets;
  if(pixelTripletArrayIndex >= nPixelTriplets) return;
  int pixelTripletIndex = pixelTripletArrayIndex;

  if(pixelTripletsInGPU.isDup[pixelTripletIndex])  
  {
      return;
  }


  //cross cleaning step
  float eta1 = pixelTripletsInGPU.eta_pix[pixelTripletIndex]; 
  float phi1 = pixelTripletsInGPU.phi_pix[pixelTripletIndex]; 
  int pixelModuleIndex = *modulesInGPU.nModules - 1;
  unsigned int prefix = pixelModuleIndex*N_MAX_SEGMENTS_PER_MODULE;
  for (unsigned int jx=0; jx<*pixelQuintupletsInGPU.nPixelQuintuplets; jx++){
       unsigned int pLS_jx = pixelQuintupletsInGPU.pixelIndices[jx];
       float eta2 = segmentsInGPU.eta[pLS_jx - prefix];
       float phi2 = segmentsInGPU.phi[pLS_jx - prefix];
       float dEta = abs(eta1-eta2);
       float dPhi = abs(phi1-phi2);
       if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
       float dR2 = dEta*dEta + dPhi*dPhi;
       if(dR2 < 0.0000001) return;


  }


  unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
  atomicAdd(trackCandidatesInGPU.nTrackCandidatespT3,1);
  unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
  addTrackCandidateToMemory(trackCandidatesInGPU, 5/*track candidate type pT3=5*/, pixelTripletIndex, pixelTripletIndex, trackCandidateIdx);

}


__global__ void addpT5asTrackCandidateInGPU(struct SDL::modules& modulesInGPU, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU)
{
  int pixelQuintupletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int pixelLowerModuleArrayIndex = *modulesInGPU.nLowerModules;
  unsigned int nPixelQuintuplets = *pixelQuintupletsInGPU.nPixelQuintuplets;
  if(pixelQuintupletArrayIndex >= nPixelQuintuplets) return;
  int pixelQuintupletIndex = pixelQuintupletArrayIndex;
  if(pixelQuintupletsInGPU.isDup[pixelQuintupletIndex])  
  {
      return;
  }
  unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
  atomicAdd(trackCandidatesInGPU.nTrackCandidatespT5,1);
  unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
  

  addTrackCandidateToMemory(trackCandidatesInGPU, 7/*track candidate type pT5=7*/, pixelQuintupletsInGPU.pixelIndices[pixelQuintupletIndex], pixelQuintupletsInGPU.T5Indices[pixelQuintupletIndex], trackCandidateIdx);

}

#ifndef NESTED_PARA
__global__ void createPixelTrackCandidatesInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, unsigned int* threadIdx_gpu, unsigned int* threadIdx_gpu_offset)
{
  unsigned int outerInnerInnerLowerModuleArrayIndex = threadIdx_gpu[blockIdx.y * blockDim.y + threadIdx.y];
  if(outerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  //FIXME:Cheapo module map - We care about pT4s and pTCs Only if the outerInnerInnerLowerModule is "connected" to the pixel module

  int outerInnerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerInnerInnerLowerModuleArrayIndex];
  if(modulesInGPU.moduleType[outerInnerInnerLowerModuleIndex] == SDL::TwoS) return;

  unsigned int pixelLowerModuleArrayIndex = *modulesInGPU.nLowerModules;

  unsigned int nPixelTracklets = *(pixelTrackletsInGPU.nPixelTracklets);
  //capping
  if(nPixelTracklets > N_MAX_PIXEL_TRACKLETS_PER_MODULE)
    nPixelTracklets = N_MAX_PIXEL_TRACKLETS_PER_MODULE;

  unsigned int nOuterLayerTracklets = trackletsInGPU.nTracklets[outerInnerInnerLowerModuleArrayIndex];
  if(nOuterLayerTracklets > N_MAX_TRACKLETS_PER_MODULE)
    {
      nOuterLayerTracklets = N_MAX_TRACKLETS_PER_MODULE;
    }
  unsigned int nOuterLayerTriplets = tripletsInGPU.nTriplets[outerInnerInnerLowerModuleArrayIndex];
  if(nOuterLayerTriplets > N_MAX_TRIPLETS_PER_MODULE)
    {
      nOuterLayerTriplets = N_MAX_TRIPLETS_PER_MODULE;
    }

  unsigned int nThreadsForNestedKernel = max(nOuterLayerTracklets,nOuterLayerTriplets);
  if(nThreadsForNestedKernel == 0) return;

  int pixelTrackletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

  int outerObjectArrayIndex = threadIdx_gpu_offset[blockIdx.y * blockDim.y+ threadIdx.y];
  if(pixelTrackletArrayIndex >= nPixelTracklets) return;

  int pixelTrackletIndex = pixelTrackletArrayIndex;
  int outerObjectIndex = 0;
  short trackCandidateType;
  bool success;

  //pT4-T4
  if(outerObjectArrayIndex < nOuterLayerTracklets)
    {
      outerObjectIndex = outerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;

        //part 2 of cheapo module map : only considering tracklets with PS-PS inner segment
        if(modulesInGPU.moduleType[trackletsInGPU.lowerModuleIndices[4 * outerObjectIndex + 1]] == SDL::PS)
        {
	        success = runTrackCandidateDefaultAlgoTwoTracklets(pixelTrackletsInGPU, trackletsInGPU, tripletsInGPU, pixelTrackletIndex, outerObjectIndex, trackCandidateType);
	    if(success)
        {
	        unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
	        atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[pixelLowerModuleArrayIndex],1);
	        if(trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
            {
                #ifdef Warnings
    		  if(innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                {

		            printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                }
                    #endif
            }
	        else
            {
		    if(modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] == -1)
            {
                #ifdef Warnings
		        printf("Track candidates: no memory for pixel lower module index at %d\n",innerInnerInnerLowerModuleArrayIndex);
                #endif

            }
		  else
		    {
		      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
		      addTrackCandidateToMemory(trackCandidatesInGPU, 5/*trackCandidateType*/, pixelTrackletIndex, outerObjectIndex, trackCandidateIdx);
                    }

                }
            }
        }
    }

  //pT4-T3
  if(outerObjectArrayIndex < nOuterLayerTriplets)
    {
      outerObjectIndex = outerInnerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + outerObjectArrayIndex;

      //part 2 of cheapo module map : only considering tracklets with PS-PS inner segment
      if(modulesInGPU.moduleType[tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 1]] == SDL::PS)
        {
	  success = runTrackCandidateDefaultAlgoTrackletToTriplet(pixelTrackletsInGPU, trackletsInGPU, tripletsInGPU, pixelTrackletIndex, outerObjectIndex, trackCandidateType);
	  if(success)
            {
	      unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
	      atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[pixelLowerModuleArrayIndex],1);
	      if(trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
		  if(innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                    {

		      printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
	      else
                {
		  if(modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] == -1)
                    {
                       #ifdef Warnings
		      printf("Track candidates: no memory for pixel lower module index at %d\n",innerInnerInnerLowerModuleArrayIndex);
                       #endif

                    }
		  else
		    {
		      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
		      addTrackCandidateToMemory(trackCandidatesInGPU, 6/*trackCandidateType*/, pixelTrackletIndex, outerObjectIndex, trackCandidateIdx);
                    }

                }
            }
        }

    }
}

#else
__global__ void createPixelTrackCandidatesInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU)
{
    unsigned int outerInnerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(outerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

    int outerInnerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerInnerInnerLowerModuleArrayIndex];
    if(modulesInGPU.moduleType[outerInnerInnerLowerModuleIndex] == SDL::TwoS) return;

    unsigned int pixelLowerModuleArrayIndex = *modulesInGPU.nLowerModules;

    unsigned int nPixelTracklets = *(pixelTrackletsInGPU.nPixelTracklets);
    //capping
    if(nPixelTracklets > N_MAX_PIXEL_TRACKLETS_PER_MODULE)
        nPixelTracklets = N_MAX_PIXEL_TRACKLETS_PER_MODULE;

    unsigned int nOuterLayerTracklets = trackletsInGPU.nTracklets[outerInnerInnerLowerModuleArrayIndex];
    if(nOuterLayerTracklets > N_MAX_TRACKLETS_PER_MODULE)
    {
        nOuterLayerTracklets = N_MAX_TRACKLETS_PER_MODULE;
    }
    unsigned int nOuterLayerTriplets = tripletsInGPU.nTriplets[outerInnerInnerLowerModuleArrayIndex];
    if(nOuterLayerTriplets > N_MAX_TRIPLETS_PER_MODULE)
    {
        nOuterLayerTriplets = N_MAX_TRIPLETS_PER_MODULE;
    }

    unsigned int nThreadsForNestedKernel = max(nOuterLayerTracklets,nOuterLayerTriplets);
    if(nThreadsForNestedKernel == 0) return;

    dim3 nThreads(16,16,1);
    dim3 nBlocks( nPixelTracklets % nThreads.x == 0 ? nPixelTracklets/nThreads.x : nPixelTracklets/nThreads.x + 1, nThreadsForNestedKernel % nThreads.y == 0 ? nThreadsForNestedKernel/nThreads.y : nThreadsForNestedKernel/nThreads.y + 1, 1);

    createPixelTrackCandidatesFromOuterInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, trackletsInGPU, pixelTrackletsInGPU, tripletsInGPU, trackCandidatesInGPU, pixelLowerModuleArrayIndex, outerInnerInnerLowerModuleArrayIndex, nPixelTracklets, nOuterLayerTracklets, nOuterLayerTriplets);
}


__global__ void createPixelTrackCandidatesFromOuterInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::pixelTracklets& pixelTrackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, unsigned int pixelLowerModuleArrayIndex, unsigned int outerInnerInnerLowerModuleArrayIndex, unsigned int nPixelTracklets, unsigned int nOuterLayerTracklets, unsigned int nOuterLayerTriplets)
{
    int pixelTrackletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerObjectArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if(pixelTrackletArrayIndex >= nPixelTracklets) return;

    int pixelTrackletIndex = pixelTrackletArrayIndex;
    int outerObjectIndex = 0;
    short trackCandidateType;
    bool success;

    //pT4-T4
    if(outerObjectArrayIndex < nOuterLayerTracklets)
    {
        outerObjectIndex = outerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;

        //part 2 of cheapo module map : only considering tracklets with PS-PS inner segment
       if(modulesInGPU.moduleType[trackletsInGPU.lowerModuleIndices[4 * outerObjectIndex + 1]] == SDL::PS)
        {
            success = runTrackCandidateDefaultAlgoTwoTracklets(pixelTrackletsInGPU, trackletsInGPU, tripletsInGPU, pixelTrackletIndex, outerObjectIndex, trackCandidateType);
            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[pixelLowerModuleArrayIndex],1);
                if(trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
                    if(innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                    {

                        printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
                else
                {
                    if(modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] == -1)
                    {
                       #ifdef Warnings
                       printf("Track candidates: no memory for pixel lower module index at %d\n",innerInnerInnerLowerModuleArrayIndex);
                       #endif

                    }
                    else
                   {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
                        addTrackCandidateToMemory(trackCandidatesInGPU, 5/*pT2-T4 trackCandidateType*/, pixelTrackletIndex, outerObjectIndex, trackCandidateIdx);
                    }

                }
            }
        }
    }

    //pT4-T3
    if(outerObjectArrayIndex < nOuterLayerTriplets)
    {
        outerObjectIndex = outerInnerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + outerObjectArrayIndex;

        //part 2 of cheapo module map : only considering tracklets with PS-PS inner segment
        if(modulesInGPU.moduleType[tripletsInGPU.lowerModuleIndices[3 * outerObjectIndex + 1]] == SDL::PS)
        {
            success = runTrackCandidateDefaultAlgoTrackletToTriplet(pixelTrackletsInGPU, trackletsInGPU, tripletsInGPU, pixelTrackletIndex, outerObjectIndex, trackCandidateType);
            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[pixelLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[pixelLowerModuleArrayIndex],1);
                if(trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
                    if(innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE)
                    {

                        printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
                else
                {
                    if(modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] == -1)
                    {
                       #ifdef Warnings
                       printf("Track candidates: no memory for pixel lower module index at %d\n",innerInnerInnerLowerModuleArrayIndex);
                       #endif

                    }
                    else
                   {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[pixelLowerModuleArrayIndex] + trackCandidateModuleIdx;
                        addTrackCandidateToMemory(trackCandidatesInGPU, 6/* pT2-T3 trackCandidateType*/, pixelTrackletIndex, outerObjectIndex, trackCandidateIdx);
                    }

                }
            }
        }

    }
}
#endif

#ifndef NESTED_PARA
__global__ void createTrackCandidatesInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, unsigned int* threadIdx_gpu, unsigned int *threadIdx_gpu_offset)
{
  //inner tracklet/triplet inner segment inner MD lower module
  int innerInnerInnerLowerModuleArrayIndex = threadIdx_gpu[blockIdx.y * blockDim.y + threadIdx.y];
  //hack to include pixel detector
  if(innerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

  unsigned int nTracklets = trackletsInGPU.nTracklets[innerInnerInnerLowerModuleArrayIndex];
  if(nTracklets > N_MAX_TRACKLETS_PER_MODULE)
    {
      nTracklets = N_MAX_TRACKLETS_PER_MODULE;
    }

  unsigned int nTriplets = tripletsInGPU.nTriplets[innerInnerInnerLowerModuleArrayIndex]; // should be zero for the pixels
  if(nTriplets > N_MAX_TRIPLETS_PER_MODULE)
    {
      nTriplets = N_MAX_TRIPLETS_PER_MODULE;
    }

  unsigned int temp = max(nTracklets,nTriplets);
  unsigned int MAX_OBJECTS = max(N_MAX_TRACKLETS_PER_MODULE, N_MAX_TRIPLETS_PER_MODULE);

  if(temp == 0) return;

  int innerObjectArrayIndex = threadIdx_gpu_offset[blockIdx.y * blockDim.y + threadIdx.y];
  int outerObjectArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

  int innerObjectIndex = 0;
  int outerObjectIndex = 0;
  short trackCandidateType;
  bool success;

  //step 1 tracklet-tracklet
  if(innerObjectArrayIndex < nTracklets)
    {
      innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
      unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];/*same as innerOuterInnerLowerModuleIndex*/

      if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {

	  outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;

	  success = runTrackCandidateDefaultAlgoTwoTracklets(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);

	  if(success)
            {
	      unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
	      atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[innerInnerInnerLowerModuleArrayIndex],1);
	      if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
		  if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                    {
		      printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
	      else
                {
		  if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                       #ifdef Warnings
		      printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                       #endif

                    }
		  else
		    {
		      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
		      addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);
                    }

                }
            }

        }
    }

  //step 2 tracklet-triplet
  if(innerObjectArrayIndex < nTracklets)
    {
      innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
      unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];//same as innerOuterInnerLowerModuleIndex
      if(outerObjectArrayIndex < fminf(tripletsInGPU.nTriplets[outerInnerInnerLowerModuleIndex],N_MAX_TRIPLETS_PER_MODULE))
        {
	  outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRIPLETS_PER_MODULE + outerObjectArrayIndex;
	  success = runTrackCandidateDefaultAlgoTrackletToTriplet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
	  if(success)
            {
	      unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
	      atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T3[innerInnerInnerLowerModuleArrayIndex],1);
	      if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
		  if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                    {
		      printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
	      else
                {

		  if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                        #ifdef Warnings
		      printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                        #endif
                    }
		  else
                    {
		      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;

		      addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);
                    }
                }
            }

        }
    }
  //step 3 triplet-tracklet
  if(innerObjectArrayIndex < nTriplets)
    {
      innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + innerObjectArrayIndex;
      unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletsInGPU.lowerModuleIndices[3 * innerObjectIndex + 1]];//same as innerOuterInnerLowerModuleIndex

      if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {
	  outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;
	  success = runTrackCandidateDefaultAlgoTripletToTracklet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
	  if(success)
            {
	      unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
	      atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT3T4[innerInnerInnerLowerModuleArrayIndex],1);
	      if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                   #ifdef Warnings
		  if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
		    printf("Track Candidate excess alert! Module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                   #endif
                }
	      else
                {
		  if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                        #ifdef Warnings
		      printf("Track candidates: no memory for module at module index = %d, outer T4 module index = %d\n",innerInnerInnerLowerModuleArrayIndex, outerInnerInnerLowerModuleIndex);
                        #endif
                    }
		  else
                    {
		      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
		      addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);

                    }
                }
            }

        }
    }
}

#else
__global__ void createTrackCandidatesInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU)
{
    //inner tracklet/triplet inner segment inner MD lower module
    int innerInnerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //hack to include pixel detector
    if(innerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

    unsigned int nTracklets = trackletsInGPU.nTracklets[innerInnerInnerLowerModuleArrayIndex];
    if(nTracklets > N_MAX_TRACKLETS_PER_MODULE)
    {
        nTracklets = N_MAX_TRACKLETS_PER_MODULE;
    }

    unsigned int nTriplets = tripletsInGPU.nTriplets[innerInnerInnerLowerModuleArrayIndex]; // should be zero for the pixels
    if(nTriplets > N_MAX_TRIPLETS_PER_MODULE)
    {
        nTriplets = N_MAX_TRIPLETS_PER_MODULE;
    }

    unsigned int temp = max(nTracklets,nTriplets);
    unsigned int MAX_OBJECTS = max(N_MAX_TRACKLETS_PER_MODULE, N_MAX_TRIPLETS_PER_MODULE);

    if(temp == 0) return;

    //triplets and tracklets are stored directly using lower module array index
    dim3 nThreads(16,16,1);
    dim3 nBlocks(temp % nThreads.x == 0 ? temp / nThreads.x : temp / nThreads.x + 1, MAX_OBJECTS % nThreads.y == 0 ? MAX_OBJECTS / nThreads.y : MAX_OBJECTS / nThreads.y + 1, 1);

    createTrackCandidatesFromInnerInnerInnerLowerModule<<<nBlocks, nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, trackletsInGPU, tripletsInGPU, trackCandidatesInGPU,innerInnerInnerLowerModuleArrayIndex,nTracklets,nTriplets);
}

__global__ void createTrackCandidatesFromInnerInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, unsigned int innerInnerInnerLowerModuleArrayIndex, unsigned int nInnerTracklets, unsigned int nInnerTriplets)
{
    int innerObjectArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerObjectArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

    int innerObjectIndex = 0;
    int outerObjectIndex = 0;
    short trackCandidateType;
    bool success;
    //step 1 tracklet-tracklet
    if(innerObjectArrayIndex < nInnerTracklets)
    {
        innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
        unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];/*same as innerOuterInnerLowerModuleIndex*/

        if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {

            outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;

            success = runTrackCandidateDefaultAlgoTwoTracklets(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);

            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[innerInnerInnerLowerModuleArrayIndex],1);
                if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
                    if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                    {
                        printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
                else
                {
                    if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                       #ifdef Warnings
                       printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                       #endif

                    }
                    else
                   {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
                        addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);
                    }

                }
            }

        }
    }
    //step 2 tracklet-triplet
    if(innerObjectArrayIndex < nInnerTracklets)
    {
        innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
        unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];//same as innerOuterInnerLowerModuleIndex
        if(outerObjectArrayIndex < fminf(tripletsInGPU.nTriplets[outerInnerInnerLowerModuleIndex],N_MAX_TRIPLETS_PER_MODULE))
        {
            outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRIPLETS_PER_MODULE + outerObjectArrayIndex;
            success = runTrackCandidateDefaultAlgoTrackletToTriplet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T3[innerInnerInnerLowerModuleArrayIndex],1);
                if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                    #ifdef Warnings
                    if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                    {
                        printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    #endif
                }
                else
                {

                    if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                        #ifdef Warnings
                        printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                        #endif
                    }
                    else
                    {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;

                        addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);
                    }
                }
            }

        }
    }

    //step 3 triplet-tracklet
    if(innerObjectArrayIndex < nInnerTriplets)
    {
        innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + innerObjectArrayIndex;
        unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletsInGPU.lowerModuleIndices[3 * innerObjectIndex + 1]];//same as innerOuterInnerLowerModuleIndex

        if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {
            outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;
            success = runTrackCandidateDefaultAlgoTripletToTracklet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT3T4[innerInnerInnerLowerModuleArrayIndex],1);
	        if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                   #ifdef Warnings
                   if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                       printf("Track Candidate excess alert! Module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                   #endif
                }
                else
                {
                    if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                        #ifdef Warnings
                        printf("Track candidates: no memory for module at module index = %d, outer T4 module index = %d\n",innerInnerInnerLowerModuleArrayIndex, outerInnerInnerLowerModuleIndex);
                        #endif
                    }
                    else
                    {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
                        addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);

                    }
                }
            }

        }
    }
}
#endif


#ifndef NESTED_PARA
__global__ void createPixelTripletsInGPUFromMap(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU, unsigned int* connectedPixelSize, unsigned int* connectedPixelIndex, unsigned int nPixelSegments, unsigned int* seg_pix_gpu, unsigned int* seg_pix_gpu_offset, unsigned int totalSegs)
{
    //newgrid with map
    unsigned int offsetIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(offsetIndex >= totalSegs)  return;

    int segmentModuleIndex = seg_pix_gpu_offset[offsetIndex];
    int pixelSegmentArrayIndex = seg_pix_gpu[offsetIndex];
    if(pixelSegmentArrayIndex >= nPixelSegments) return;
    if(segmentModuleIndex >= connectedPixelSize[pixelSegmentArrayIndex]) return;

    unsigned int tripletLowerModuleIndex; //index of the module that connects to this pixel
    unsigned int tempIndex = connectedPixelIndex[pixelSegmentArrayIndex] + segmentModuleIndex; //gets module array index for segment

    //these are actual module indices
    tripletLowerModuleIndex = modulesInGPU.connectedPixels[tempIndex];
    unsigned int tripletLowerModuleArrayIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletLowerModuleIndex];
    if(tripletLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

    unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1;
    unsigned int nOuterTriplets = min(tripletsInGPU.nTriplets[tripletLowerModuleArrayIndex], N_MAX_TRIPLETS_PER_MODULE);

    if(nOuterTriplets == 0) return;
    if(modulesInGPU.moduleType[tripletLowerModuleIndex] == SDL::TwoS) return; //Removes 2S-2S

    //fetch the triplet
    unsigned int outerTripletArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if(outerTripletArrayIndex >= nOuterTriplets) return;
    unsigned int pixelSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + pixelSegmentArrayIndex;
    unsigned int outerTripletIndex = tripletLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;
    if(modulesInGPU.moduleType[tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 1]] == SDL::TwoS) return; //REMOVES PS-2S

    if(segmentsInGPU.isDup[pixelSegmentArrayIndex]) return;
    if(segmentsInGPU.partOfPT5[pixelSegmentArrayIndex]) return; //don't make pT3s for those pixels that are part of pT5
    if(tripletsInGPU.partOfPT5[outerTripletIndex]) return; //don't create pT3s for T3s accounted in pT5s

    float pixelRadius, pixelRadiusError, tripletRadius, rPhiChiSquared, rzChiSquared, rPhiChiSquaredInwards;
    bool success = runPixelTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius, pixelRadiusError, tripletRadius, rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards);

    if(success)
    {

        short layer2_adjustment;
        if(modulesInGPU.layers[tripletLowerModuleIndex] == 1){layer2_adjustment = 1;} //get upper segment to be in second layer
        else if( modulesInGPU.layers[tripletLowerModuleIndex] == 2){layer2_adjustment = 0;} // get lower segment to be in second layer
        else{return;} // ignore anything else
        float phi = hitsInGPU.phis[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex]+layer2_adjustment]]];
        float eta = hitsInGPU.etas[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex]+layer2_adjustment]]];
        float eta_pix = segmentsInGPU.eta[pixelSegmentArrayIndex];
        float phi_pix = segmentsInGPU.phi[pixelSegmentArrayIndex];
        float pt = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
        float score = rPhiChiSquared;//scorepT3(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,tripletsInGPU,pixelSegmentIndex,outerTripletIndex,pt,segmentsInGPU.pz[pixelSegmentArrayIndex]);
        //printf("%f\n",score);
        //if(score >100){return;}
        unsigned int pixelTripletIndex = atomicAdd(pixelTripletsInGPU.nPixelTriplets, 1);
        if(pixelTripletIndex >= N_MAX_PIXEL_TRIPLETS)
        {
            #ifdef Warnings
            if(pixelTripletIndex == N_MAX_PIXEL_TRIPLETS)
            {
               printf("Pixel Triplet excess alert!\n");
            }
            #endif
        }
        else
        {
#ifdef CUT_VALUE_DEBUG
            addPixelTripletToMemory(pixelTripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius, pixelRadiusError, tripletRadius, rPhiChiSquared, rPhiChiSquaredInwards, rzChiSquared, pixelTripletIndex, pt, eta, phi, eta_pix, phi_pix, score);
#else
            addPixelTripletToMemory(pixelTripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius,tripletRadius, pixelTripletIndex, pt,eta,phi,eta_pix,phi_pix,score);
#endif
        }
    }

}

#else

__global__ void createPixelTripletsFromOuterInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU, unsigned int outerTripletInnerLowerModuleArrayIndex, unsigned int nPixelSegments, unsigned int nOuterTriplets, unsigned int pixelModuleIndex)
{
   int pixelSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
   int outerTripletArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

   if(pixelSegmentArrayIndex >= nPixelSegments) return;
   if(outerTripletArrayIndex >= nOuterTriplets) return;

   unsigned int pixelSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + pixelSegmentArrayIndex;
   unsigned int outerTripletIndex = outerTripletInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;


   if(segmentsInGPU.isDup[pixelSegmentArrayIndex]) return;
   if(segmentsInGPU.partOfPT5[pixelSegmentArrayIndex]) return;
   if(tripletsInGPU.partOfPT5[outerTripletIndex]) return; //don't create pT3s for T3s accounted in pT5s


   if(modulesInGPU.moduleType[tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 1]] == SDL::TwoS) return; //REMOVES PS-2S

   float pixelRadius, pixelRadiusError, tripletRadius, rPhiChiSquared, rzChiSquared, rPhiChiSquaredInwards;
   bool success = runPixelTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius, pixelRadiusError, tripletRadius, rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards);

   if(success)
   {
        unsigned int tripletLowerModuleIndex = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex];
        short layer2_adjustment;
        if(modulesInGPU.layers[tripletLowerModuleIndex] == 1){layer2_adjustment = 1;} //get upper segment to be in second layer
        else if( modulesInGPU.layers[tripletLowerModuleIndex] == 2){layer2_adjustment = 0;} // get lower segment to be in second layer
        else{return;} // ignore anything else
        float phi = hitsInGPU.phis[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex]+layer2_adjustment]]];
        float eta = hitsInGPU.etas[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTripletIndex]+layer2_adjustment]]];
        float eta_pix = segmentsInGPU.eta[pixelSegmentArrayIndex];
        float phi_pix = segmentsInGPU.phi[pixelSegmentArrayIndex];
        float pt = segmentsInGPU.ptIn[pixelSegmentArrayIndex];
        float score = rPhiChiSquared;//scorepT3(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,tripletsInGPU,pixelSegmentIndex,outerTripletIndex,pt,segmentsInGPU.pz[pixelSegmentArrayIndex]);

       unsigned int pixelTripletIndex = atomicAdd(pixelTripletsInGPU.nPixelTriplets, 1);
       if(pixelTripletIndex >= N_MAX_PIXEL_TRIPLETS)
       {
            #ifdef Warnings
            if(pixelTripletIndex == N_MAX_PIXEL_TRIPLETS)
            {
               printf("Pixel Triplet excess alert!\n");
            }
            #endif
       }
       else
       {
#ifdef CUT_VALUE_DEBUG
           addPixelTripletToMemory(pixelTripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius, pixelRadiusError, tripletRadius, rPhiChiSquared, rPhiChiSquaredInwards, rzChiSquared, pixelTripletIndex, pt, eta, phi, eta_pix, phi_pix, score);
#else
           addPixelTripletToMemory(pixelTripletsInGPU, pixelSegmentIndex, outerTripletIndex, pixelRadius,tripletRadius, pixelTripletIndex,pt,eta,phi,eta_pix,phi_pix,score);
#endif
       }
   }
}

__global__ void createPixelTripletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU)
{
    int outerTripletInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

    //lower modules 2 and 3 are taken from the triplet!
    if(outerTripletInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

    unsigned int nOuterTriplets = tripletsInGPU.nTriplets[outerTripletInnerLowerModuleArrayIndex] > N_MAX_TRIPLETS_PER_MODULE ? N_MAX_TRIPLETS_PER_MODULE : tripletsInGPU.nTriplets[outerTripletInnerLowerModuleArrayIndex];

    unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1;
    unsigned int nPixelSegments = segmentsInGPU.nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[pixelModuleIndex];

    //El-cheapo map applied on the inner segment
    unsigned int outerTripletInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerTripletInnerLowerModuleArrayIndex];
    if(modulesInGPU.moduleType[outerTripletInnerLowerModuleIndex]== SDL::TwoS) return; //REMOVES 2S-2S

    if(nOuterTriplets == 0) return;
    dim3 nThreads(16,16,1);
    dim3 nBlocks(nPixelSegments % nThreads.x == 0 ? nPixelSegments / nThreads.x : nPixelSegments / nThreads.x + 1, nOuterTriplets % nThreads.y == 0 ? nOuterTriplets / nThreads.y : nOuterTriplets / nThreads.y + 1, 1);

    createPixelTripletsFromOuterInnerLowerModule<<<nBlocks, nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, pixelTripletsInGPU, outerTripletInnerLowerModuleArrayIndex, nPixelSegments, nOuterTriplets, pixelModuleIndex);
}

#endif

#ifndef NESTED_PARA
__global__ void createQuintupletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, unsigned int* threadIdx_gpu, unsigned int* threadIdx_gpu_offset, int nTotalTriplets)
{
    int gidy = blockIdx.y * blockDim.y + threadIdx.y;
    int np = gridDim.y * blockDim.y;
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int iter=gidy; iter < nTotalTriplets; iter+=np) {
      //int lowerModuleArray1 = threadIdx_gpu[blockIdx.y * blockDim.y + threadIdx.y];
    int lowerModuleArray1 = threadIdx_gpu[iter];

    //this if statement never gets executed!
    if(lowerModuleArray1  >= *modulesInGPU.nLowerModules) continue;

    unsigned int nInnerTriplets = tripletsInGPU.nTriplets[lowerModuleArray1];

    //unsigned int innerTripletArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    //unsigned int innerTripletArrayIndex = threadIdx_gpu_offset[blockIdx.y * blockDim.y + threadIdx.y];
    //unsigned int outerTripletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int innerTripletArrayIndex = threadIdx_gpu_offset[iter];
    unsigned int outerTripletArrayIndex = gidx;

    if(innerTripletArrayIndex >= nInnerTriplets) continue;

    unsigned int innerTripletIndex = lowerModuleArray1 * N_MAX_TRIPLETS_PER_MODULE + innerTripletArrayIndex;
    unsigned int lowerModule1 = modulesInGPU.lowerModuleIndices[lowerModuleArray1];
    //these are actual module indices!! not lower module indices!
    unsigned int lowerModule2 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 1];
    unsigned int lowerModule3 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 2];
    unsigned int lowerModuleArray3 = modulesInGPU.reverseLookupLowerModuleIndices[lowerModule3];
    unsigned int nOuterTriplets = min(tripletsInGPU.nTriplets[lowerModuleArray3], N_MAX_TRIPLETS_PER_MODULE);

    if(outerTripletArrayIndex >= nOuterTriplets) continue;
    unsigned int outerTripletIndex = lowerModuleArray3 * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;
    //these are actual module indices!!
    unsigned int lowerModule4 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 1];
    unsigned int lowerModule5 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 2];

    float innerRadius, innerRadiusMin, innerRadiusMin2S, innerRadiusMax, innerRadiusMax2S, outerRadius, outerRadiusMin, outerRadiusMin2S, outerRadiusMax, outerRadiusMax2S, bridgeRadius, bridgeRadiusMin, bridgeRadiusMin2S, bridgeRadiusMax, bridgeRadiusMax2S, regressionG, regressionF, regressionRadius, chiSquared, nonAnchorChiSquared; //required for making distributions

    bool success = runQuintupletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerTripletIndex, outerTripletIndex, innerRadius, innerRadiusMin, innerRadiusMax, outerRadius, outerRadiusMin, outerRadiusMax, bridgeRadius, bridgeRadiusMin, bridgeRadiusMax, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S,
            outerRadiusMax2S, regressionG, regressionF, regressionRadius, chiSquared, nonAnchorChiSquared);

   if(success)
   {
       short layer2_adjustment;
       int layer = modulesInGPU.layers[lowerModule1];
       if(layer == 1){layer2_adjustment = 1;} //get upper segment to be in second layer
       else if( layer == 2){layer2_adjustment = 0;} // get lower segment to be in second layer
       else{return;} // ignore anything else TODO: move this to start, before object is made (faster)
       unsigned int quintupletModuleIndex = atomicAdd(&quintupletsInGPU.nQuintuplets[lowerModuleArray1], 1);
       if(quintupletModuleIndex >= N_MAX_QUINTUPLETS_PER_MODULE)
       {
#ifdef Warnings
           if(quintupletModuleIndex ==  N_MAX_QUINTUPLETS_PER_MODULE)
               printf("Quintuplet excess alert! Module index = %d\n", lowerModuleArray1);
#endif
       }
       else
       {
           //this if statement should never get executed!
           if(modulesInGPU.quintupletModuleIndices[lowerModuleArray1] == -1)
           {
                printf("Quintuplets : no memory for module at module index = %d\n", lowerModuleArray1);
           }
           else
           {
                unsigned int quintupletIndex = modulesInGPU.quintupletModuleIndices[lowerModuleArray1] +  quintupletModuleIndex;
                float phi = hitsInGPU.phis[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+layer2_adjustment]]]];
                float eta = hitsInGPU.etas[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+layer2_adjustment]]]];
                float pt = (innerRadius+outerRadius)*3.8*1.602/(2*100*5.39);
                //float scores[3];
                //scores[1] = scoreT5(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,tripletsInGPU,innerTripletIndex,outerTripletIndex,layer);
                float scores[4];// still fills all values,but only rphi sum is actually used for the cuts. Others may be removed later.
                scoreT5(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,tripletsInGPU,innerTripletIndex,outerTripletIndex,layer,scores);
                scores[0] = chiSquared;
                scores[2] = chiSquared + nonAnchorChiSquared;
                //printf("%f %f %f %f\n",scores[0],scores[2],scores[1],scores[3]);
#ifdef CUT_VALUE_DEBUG
                addQuintupletToMemory(quintupletsInGPU, innerTripletIndex, outerTripletIndex, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerRadius, innerRadiusMin, innerRadiusMax, outerRadius, outerRadiusMin, outerRadiusMax, bridgeRadius, bridgeRadiusMin, bridgeRadiusMax, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, regressionG, regressionF, regressionRadius, chiSquared, nonAnchorChiSquared,
                        pt, eta, phi, scores, layer, quintupletIndex);
#else
                addQuintupletToMemory(quintupletsInGPU, innerTripletIndex, outerTripletIndex, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerRadius, outerRadius, regressionG, regressionF, regressionRadius, pt,eta,phi,scores,layer,quintupletIndex);
#endif
            }
        }
    }
    }
}

#else
__global__ void createQuintupletsFromInnerInnerLowerModule(SDL::modules& modulesInGPU, SDL::hits& hitsInGPU, SDL::miniDoublets& mdsInGPU, SDL::segments& segmentsInGPU, SDL::triplets& tripletsInGPU, SDL::quintuplets& quintupletsInGPU, unsigned int lowerModuleArray1, unsigned int nInnerTriplets)
{
   int innerTripletArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
   int outerTripletArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

   if(innerTripletArrayIndex >= nInnerTriplets) return;

   unsigned int innerTripletIndex = lowerModuleArray1 * N_MAX_TRIPLETS_PER_MODULE + innerTripletArrayIndex;
   unsigned int lowerModule1 = modulesInGPU.lowerModuleIndices[lowerModuleArray1];
   //these are actual module indices!!! not lower module indices
   unsigned int lowerModule2 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 1];
   unsigned int lowerModule3 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 2];
   unsigned int lowerModuleArray3 = modulesInGPU.reverseLookupLowerModuleIndices[lowerModule3];

   unsigned int nOuterTriplets = min(tripletsInGPU.nTriplets[lowerModuleArray3], N_MAX_TRIPLETS_PER_MODULE);

   if(outerTripletArrayIndex >= nOuterTriplets) return;

   unsigned int outerTripletIndex = lowerModuleArray3 * N_MAX_TRIPLETS_PER_MODULE + outerTripletArrayIndex;
    //these are actual module indices!!
    unsigned int lowerModule4 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 1];
    unsigned int lowerModule5 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 2];

    float innerRadius, innerRadiusMin, innerRadiusMin2S, innerRadiusMax, innerRadiusMax2S, outerRadius, outerRadiusMin, outerRadiusMin2S, outerRadiusMax, outerRadiusMax2S, bridgeRadius, bridgeRadiusMin, bridgeRadiusMin2S, bridgeRadiusMax, bridgeRadiusMax2S, regressionG, regressionF, regressionRadius, chiSquared, nonAnchorChiSquared; //required for making distributions
    bool success = runQuintupletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerTripletIndex, outerTripletIndex, innerRadius, innerRadiusMin, innerRadiusMax, outerRadius, outerRadiusMin, outerRadiusMax, bridgeRadius, bridgeRadiusMin, bridgeRadiusMax, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S,
            outerRadiusMax2S, regressionG, regressionF, regressionRadius, chiSquared, nonAnchorChiSquared);

   if(success)
   {
       short layer2_adjustment;
       int layer = modulesInGPU.layers[lowerModule1];
       if(layer == 1){layer2_adjustment = 1;} //get upper segment to be in second layer
       else if( layer == 2){layer2_adjustment = 0;} // get lower segment to be in second layer
       else{return;} // ignore anything else TODO: move this to start, before object is made (faster)

       unsigned int quintupletModuleIndex = atomicAdd(&quintupletsInGPU.nQuintuplets[lowerModuleArray1], 1);
       if(quintupletModuleIndex >= N_MAX_QUINTUPLETS_PER_MODULE)
       {
#ifdef Warnings
           if(quintupletModuleIndex ==  N_MAX_QUINTUPLETS_PER_MODULE)
               printf("Quintuplet excess alert! Module index = %d\n", lowerModuleArray1);
#endif
       }
       else
       {
           if(modulesInGPU.quintupletModuleIndices[lowerModuleArray1] == -1)
           {
                printf("Quintuplets : no memory for module at module index = %d\n", lowerModuleArray1);
           }
           else
           {
                unsigned int quintupletIndex = modulesInGPU.quintupletModuleIndices[lowerModuleArray1] +  quintupletModuleIndex;

                float phi = hitsInGPU.phis[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+layer2_adjustment]]]];
                float eta = hitsInGPU.etas[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+layer2_adjustment]]]];
                float pt = (innerRadius+outerRadius)*3.8*1.602/(2*100*5.39);
                //float scores[3];
                //scores[1] = scoreT5(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,tripletsInGPU,innerTripletIndex,outerTripletIndex,layer);
                //scores[0] = chiSquared;
                //scores[2] = chiSquared+ nonAnchorChiSquared//+scores[1];
                float scores[4];
                scoreT5(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,tripletsInGPU,innerTripletIndex,outerTripletIndex,layer,scores);
                scores[0] = chiSquared;
                scores[2] = chiSquared + nonAnchorChiSquared;
                //printf("scores: %f %f %f\n",scores[0],scores[1],scores[2]);
                
#ifdef CUT_VALUE_DEBUG
                addQuintupletToMemory(quintupletsInGPU, innerTripletIndex, outerTripletIndex, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerRadius, innerRadiusMin, innerRadiusMax, outerRadius, outerRadiusMin, outerRadiusMax, bridgeRadius, bridgeRadiusMin, bridgeRadiusMax, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, regressionG, regressionF, regressionRadius, chiSquared, nonAnchorChiSquared, pt,eta,phi,scores,layer, quintupletIndex);
#else
                addQuintupletToMemory(quintupletsInGPU, innerTripletIndex, outerTripletIndex, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerRadius, outerRadius, regressionG, regressionF, regressionRadius, pt,eta,phi,scores,layer, quintupletIndex);
#endif

            }
        }
    }
}

__global__ void createQuintupletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU)
{
    int innerInnerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x; //inner triplet inner segment inner MD

    //no quintuplets can be formed for these folks - no need to run inner kernels for them!

    if(innerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules or modulesInGPU.quintupletModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1) return;

    unsigned int nInnerTriplets = min(tripletsInGPU.nTriplets[innerInnerInnerLowerModuleArrayIndex], N_MAX_TRIPLETS_PER_MODULE);
    if(nInnerTriplets == 0) return;

    dim3 nThreads(16,16,1);
    dim3 nBlocks(nInnerTriplets % nThreads.x == 0 ? nInnerTriplets / nThreads.x : nInnerTriplets / nThreads.x + 1, N_MAX_TRIPLETS_PER_MODULE % nThreads.y == 0 ? N_MAX_TRIPLETS_PER_MODULE / nThreads.y : N_MAX_TRIPLETS_PER_MODULE / nThreads.y + 1);

    createQuintupletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, quintupletsInGPU, innerInnerInnerLowerModuleArrayIndex, nInnerTriplets);

}

#endif

#ifndef NESTED_PARA
__global__ void createPixelQuintupletsInGPUFromMap(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int* connectedPixelSize, unsigned int* connectedPixelIndex, unsigned int nPixelSegments, unsigned int* seg_pix_gpu, unsigned int* seg_pix_gpu_offset, unsigned int totalSegs)
{
    unsigned int offsetIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(offsetIndex >= totalSegs) return;

    int segmentModuleIndex = seg_pix_gpu_offset[offsetIndex];
    int pixelSegmentArrayIndex = seg_pix_gpu[offsetIndex];
    if(pixelSegmentArrayIndex >= nPixelSegments) return;
    if(segmentModuleIndex >= connectedPixelSize[pixelSegmentArrayIndex]) return;

    unsigned int quintupletLowerModuleIndex; //index of the module that connects to this pixel
    unsigned int tempIndex = connectedPixelIndex[pixelSegmentArrayIndex] + segmentModuleIndex; //gets module array index for segment

    //these are actual module indices
    quintupletLowerModuleIndex = modulesInGPU.connectedPixels[tempIndex];
    unsigned int quintupletLowerModuleArrayIndex = modulesInGPU.reverseLookupLowerModuleIndices[quintupletLowerModuleIndex];
    if(quintupletLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

    unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1;
    unsigned int nOuterQuintuplets = min(quintupletsInGPU.nQuintuplets[quintupletLowerModuleArrayIndex], N_MAX_QUINTUPLETS_PER_MODULE);

    if(nOuterQuintuplets == 0) return;

    //fetch the quintuplet
    unsigned int outerQuintupletArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if(outerQuintupletArrayIndex >= nOuterQuintuplets) return;
    unsigned int pixelSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + pixelSegmentArrayIndex;
    
    unsigned int quintupletIndex = modulesInGPU.quintupletModuleIndices[quintupletLowerModuleArrayIndex] + outerQuintupletArrayIndex;

    if(quintupletsInGPU.isDup[quintupletIndex]) return;

    float rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards;

    bool success = runPixelQuintupletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, quintupletsInGPU, pixelSegmentIndex, quintupletIndex, rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards);

    if(success)
    {
       unsigned int pixelQuintupletIndex = atomicAdd(pixelQuintupletsInGPU.nPixelQuintuplets, 1);
       if(pixelQuintupletIndex >= N_MAX_PIXEL_QUINTUPLETS)
       {
            #ifdef Warnings
            if(pixelQuintupletIndex == N_MAX_PIXEL_QUINTUPLETS)
            {
               printf("Pixel Quintuplet excess alert!\n");
            }
            #endif
       }
       else
       {
#ifdef CUT_VALUE_DEBUG
           addPixelQuintupletToMemory(pixelQuintupletsInGPU, pixelSegmentIndex, quintupletIndex, pixelQuintupletIndex,rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards, rPhiChiSquared);

#else
           addPixelQuintupletToMemory(pixelQuintupletsInGPU, pixelSegmentIndex, quintupletIndex, pixelQuintupletIndex,rPhiChiSquared);
#endif
//           //mark the relevant T5 and pT3 here!
//           quintupletsInGPU.partOfPT5[quintupletIndex] = true;
//           unsigned int innerTripletIndex = quintupletsInGPU.tripletIndices[2 * quintupletIndex];
//           unsigned int outerTripletIndex = quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1];
//           tripletsInGPU.partOfPT5[innerTripletIndex] = true;
//           tripletsInGPU.partOfPT5[outerTripletIndex] = true;
//           segmentsInGPU.partOfPT5[pixelSegmentArrayIndex] = true;
       }

    }
}


#else
__global__ void createPixelQuintupletsFromFirstModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int nPixelSegments, unsigned int nOuterQuintuplets, unsigned int firstLowerModuleArrayIndex, unsigned int pixelModuleIndex)
{
    unsigned int pixelSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int outerQuintupletArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if(pixelSegmentArrayIndex >= nPixelSegments) return; 
    if(outerQuintupletArrayIndex >= nOuterQuintuplets) return;

    unsigned int pixelSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + pixelSegmentArrayIndex;
    unsigned int quintupletIndex = modulesInGPU.quintupletModuleIndices[firstLowerModuleArrayIndex] + outerQuintupletArrayIndex;

    if(segmentsInGPU.isDup[pixelSegmentArrayIndex]) return;//skip duplicated pLS
    if(quintupletsInGPU.isDup[quintupletIndex]) return;//skip duplicated T5s

    float rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards;

    bool success = runPixelQuintupletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, quintupletsInGPU, pixelSegmentIndex, quintupletIndex, rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards);

    if(success)
    {
       //printf("pT5 rz chi2: %f\n",rzChiSquared);
       unsigned int pixelQuintupletIndex = atomicAdd(pixelQuintupletsInGPU.nPixelQuintuplets, 1);
       if(pixelQuintupletIndex >= N_MAX_PIXEL_QUINTUPLETS)
       {
            #ifdef Warnings
            if(pixelQuintupletIndex == N_MAX_PIXEL_QUINTUPLETS)
            {
               printf("Pixel Quintuplet excess alert!\n");
            }
            #endif
       }
       else
       {
#ifdef CUT_VALUE_DEBUG
           addPixelQuintupletToMemory(pixelQuintupletsInGPU, pixelSegmentIndex, quintupletIndex, pixelQuintupletIndex,rzChiSquared, rPhiChiSquared, rPhiChiSquaredInwards, rPhiChiSquared);

#else
           addPixelQuintupletToMemory(pixelQuintupletsInGPU, pixelSegmentIndex, quintupletIndex, pixelQuintupletIndex,/*score*/rPhiChiSquared);
#endif
//           //mark the relevant T5 and pT3 here!
//           quintupletsInGPU.partOfPT5[quintupletIndex] = true;
//           unsigned int innerTripletIndex = quintupletsInGPU.tripletIndices[2 * quintupletIndex];
//           unsigned int outerTripletIndex = quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1];
//           tripletsInGPU.partOfPT5[innerTripletIndex] = true;
//           tripletsInGPU.partOfPT5[outerTripletIndex] = true;
//           segmentsInGPU.partOfPT5[pixelSegmentArrayIndex] = true;
       }

    }
}


__global__ void createPixelQuintupletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU)
{
    unsigned int firstLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(firstLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

    unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1;
    unsigned int nPixelSegments = segmentsInGPU.nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[pixelModuleIndex];

    unsigned int nOuterQuintuplets = min(quintupletsInGPU.nQuintuplets[firstLowerModuleArrayIndex], N_MAX_QUINTUPLETS_PER_MODULE);
    if(nOuterQuintuplets == 0) return;

    //no "pre-selections". Straight up pT5 selections
    dim3 nThreads(16, 16, 1);
    dim3 nBlocks(nPixelSegments % nThreads.x == 0 ? nPixelSegments / nThreads.x : nPixelSegments / nThreads.x + 1, nOuterQuintuplets % nThreads.y == 0 ? nOuterQuintuplets / nThreads.y : nOuterQuintuplets / nThreads.y + 1, 1);

    createPixelQuintupletsFromFirstModule<<<nBlocks, nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, quintupletsInGPU, pixelQuintupletsInGPU, nPixelSegments, nOuterQuintuplets, firstLowerModuleArrayIndex, pixelModuleIndex);

}

#endif

__device__ void scoreT5(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU,struct SDL::segments& segmentsInGPU,struct SDL::triplets& tripletsInGPU, unsigned int innerTrip, unsigned int outerTrip, int layer, float* scores)
{
        int hits1[10] = {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1};
        hits1[0] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTrip]]]; // inner triplet inner segment inner md inner hit
        hits1[1] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTrip]]+1]; // inner triplet inner segment inner md outer hit
        hits1[2] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTrip]+1]]; // inner triplet inner segment outer md inner hit
        hits1[3] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTrip]+1]+1]; // inner triplet inner segment outer md outer hit
        hits1[4] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTrip+1]+1]]; // inner triplet outer segment outer md inner hit
        hits1[5] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTrip+1]+1]+1]; // inner triplet outer segment outer md outer hit
        hits1[6] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTrip+1]]]; // outer triplet outersegment inner md inner hit
        hits1[7] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTrip+1]]+1]; // outer triplet outersegment inner md outer hit
        hits1[8] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTrip+1]+1]]; // outer triplet outersegment outer md inner hit
        hits1[9] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTrip+1]+1]+1]; // outer triplet outersegment outer md outer hit

        unsigned int mod1 = hitsInGPU.moduleIndices[hits1[0]];
        SDL::ModuleLayerType type1 = modulesInGPU.moduleLayerType[mod1];
        unsigned int mod2 = hitsInGPU.moduleIndices[hits1[6-2*layer]];//4 for layer=1 (second hit in 3rd layer), 2 for layer=2 (second hit in third layer)
        SDL::ModuleLayerType type2 = modulesInGPU.moduleLayerType[mod1];
        float r1,r2,z1,z2;

        if(type1 == 0){//lower hit is pixel
         r1 = hitsInGPU.rts[hits1[0]];
         z1 = hitsInGPU.zs[hits1[0]];
        }else{//upper hit is pixel
         r1 = hitsInGPU.rts[hits1[1]];
         z1 = hitsInGPU.zs[hits1[1]];
        }
        if(type2==0){//lower hit is pixel
         r2 = hitsInGPU.rts[hits1[6-2*layer]];
         z2 = hitsInGPU.zs[hits1[6-2*layer]];
        }else{
         r2 = hitsInGPU.rts[hits1[7-2*layer]];
         z2 = hitsInGPU.zs[hits1[7-2*layer]];
        }
        float slope_barrel = (z2-z1)/(r2-r1);
        float slope_endcap = (r2-r1)/(z2-z1);

        //least squares
        float rsum=0, zsum=0, r2sum=0,rzsum=0;
        float rsum_e=0, zsum_e=0, r2sum_e=0,rzsum_e=0;
        for(int i =0; i < 10; i++){
          rsum += hitsInGPU.rts[hits1[i]];
          zsum += hitsInGPU.zs[hits1[i]];
          r2sum += hitsInGPU.rts[hits1[i]]*hitsInGPU.rts[hits1[i]];
          rzsum += hitsInGPU.rts[hits1[i]]*hitsInGPU.zs[hits1[i]];

          rsum_e += hitsInGPU.zs[hits1[i]];
          zsum_e += hitsInGPU.rts[hits1[i]];
          r2sum_e += hitsInGPU.zs[hits1[i]]*hitsInGPU.zs[hits1[i]];
          rzsum_e += hitsInGPU.zs[hits1[i]]*hitsInGPU.rts[hits1[i]];
        }
        float slope_lsq = (10*rzsum - rsum*zsum)/(10*r2sum-rsum*rsum);
        float b = (r2sum*zsum-rsum*rzsum)/(r2sum*10-rsum*rsum);
        float slope_lsq_e = (10*rzsum_e - rsum_e*zsum_e)/(10*r2sum_e-rsum_e*rsum_e);
        float b_e = (r2sum_e*zsum_e-rsum_e*rzsum_e)/(r2sum_e*10-rsum_e*rsum_e);


        float score=0;
        float score_lsq=0;
        for( int i=0; i <10; i++){
          float z = hitsInGPU.zs[hits1[i]];
          float r = hitsInGPU.rts[hits1[i]]; // cm
          float subdet = modulesInGPU.subdets[hitsInGPU.moduleIndices[hits1[i]]];
          float drdz = modulesInGPU.drdzs[hitsInGPU.moduleIndices[hits1[i]]];
          float var=0;
          float var_lsq=0;
          if(subdet == 5) {// 5== barrel
            var = slope_barrel*(r-r1) - (z-z1);
            var_lsq = slope_lsq*(r-r1) - (z-z1);
          }else{
            var = slope_endcap*(z-z1) - (r-r1);
            var_lsq = slope_lsq_e*(z-z1) - (r-r1);
          }
          float err;
          if(modulesInGPU.moduleLayerType[hitsInGPU.moduleIndices[hits1[i]]]==0){
            err=0.15*cos(atan(drdz));//(1.5mm)^2
          }else{ err=5.0*cos(atan(drdz));}//(5cm)^2
          score += (var*var) / (err*err);
          score_lsq += (var_lsq*var_lsq) / (err*err);
        }
        //printf("%f %f\n",score,score_lsq);
        scores[1] = score;
        scores[3] = score_lsq;
        //return score;
}
__device__ int inline checkHitsT5(unsigned int ix, unsigned int jx,struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU,struct SDL::quintuplets& quintupletsInGPU)
{
        unsigned int hits1[10];// = {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1};
        unsigned int hits2[10];// = {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1};
        hits1[0] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]]]]; // inner triplet inner segment inner md inner hit
        hits1[1] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]]]+1]; // inner triplet inner segment inner md outer hit
        hits1[2] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]]+1]]; // inner triplet inner segment outer md inner hit
        hits1[3] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]]+1]+1]; // inner triplet inner segment outer md outer hit
        hits1[4] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]+1]+1]]; // inner triplet outer segment outer md inner hit
        hits1[5] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix]+1]+1]+1]; // inner triplet outer segment outer md outer hit
        hits1[6] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix+1]+1]]]; // outer triplet outersegment inner md inner hit
        hits1[7] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix+1]+1]]+1]; // outer triplet outersegment inner md outer hit
        hits1[8] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix+1]+1]+1]]; // outer triplet outersegment outer md inner hit
        hits1[9] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*ix+1]+1]+1]+1]; // outer triplet outersegment outer md outer hit

        hits2[0] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]]]; // inner triplet inner segment inner md inner hit
        hits2[1] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]]+1]; // inner triplet inner segment inner md outer hit
        hits2[2] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]+1]]; // inner triplet inner segment outer md inner hit
        hits2[3] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]]+1]+1]; // inner triplet inner segment outer md outer hit
        hits2[4] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]+1]+1]]; // inner triplet outer segment outer md inner hit
        hits2[5] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx]+1]+1]+1]; // inner triplet outer segment outer md outer hit
        hits2[6] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]]]; // outer triplet outersegment inner md inner hit
        hits2[7] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]]+1]; // outer triplet outersegment inner md outer hit
        hits2[8] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]+1]]; // outer triplet outersegment outer md inner hit
        hits2[9] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*quintupletsInGPU.tripletIndices[2*jx+1]+1]+1]+1]; // outer triplet outersegment outer md outer hit
        int nMatched =0;
        for (int i =0; i<10;i++){
          bool matched = false;
          for (int j =0; j<10; j++){
            if(hits1[i] == hits2[j]){matched = true; break;}
          }
          if(matched){nMatched++;}
        }
        return nMatched;
}
__device__ int duplicateCounter;
__global__ void removeDupQuintupletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU,bool secondPass)
{
      int dup_count=0;
      for(unsigned int lowmod1=blockIdx.x*blockDim.x+threadIdx.x; lowmod1<*modulesInGPU.nLowerModules;lowmod1+=blockDim.x*gridDim.x){
      for(unsigned int ix1=blockIdx.y*blockDim.y+threadIdx.y; ix1<quintupletsInGPU.nQuintuplets[lowmod1]; ix1+=blockDim.y*gridDim.y){
        unsigned int ix = modulesInGPU.quintupletModuleIndices[lowmod1] + ix1;
        if(secondPass && quintupletsInGPU.isDup[ix]==1)
        {
            continue;
        }
        float pt1  = quintupletsInGPU.pt[ix];
        float eta1 = quintupletsInGPU.eta[ix];
        float phi1 = quintupletsInGPU.phi[ix];
        bool isDup = false;
        for(unsigned int lowmod=0; lowmod<*modulesInGPU.nLowerModules;lowmod++){
        for(unsigned int jx1=0; jx1<quintupletsInGPU.nQuintuplets[lowmod]; jx1++){
          unsigned int jx = modulesInGPU.quintupletModuleIndices[lowmod] + jx1;
          if(ix==jx){continue;}
          if(secondPass && quintupletsInGPU.isDup[jx]==1)
          {
              continue;
          }
          float pt2  = quintupletsInGPU.pt[jx];
          float eta2 = quintupletsInGPU.eta[jx];
          float phi2 = quintupletsInGPU.phi[jx];
          float dEta = abs(eta1-eta2);
          float dPhi = abs(phi1-phi2);
          if (dEta > 0.1){continue;}
          if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
          if (abs(dPhi) > 0.1){continue;}
          float dR2 = dEta*dEta + dPhi*dPhi;
          int nMatched = checkHitsT5(ix,jx,mdsInGPU,segmentsInGPU,tripletsInGPU,quintupletsInGPU);
          if(nMatched >=7){
            dup_count++;
            if(secondPass){
              if( quintupletsInGPU.score_rphisum[ix] > quintupletsInGPU.score_rphisum[jx] ){
                rmQuintupletToMemory(quintupletsInGPU,ix);continue; // keept shorted track
              }
            }
              if( quintupletsInGPU.score_rphisum[ix] > quintupletsInGPU.score_rphisum[jx] ){
              rmQuintupletToMemory(quintupletsInGPU,ix);continue; // keept shorted track
            }
              if( (quintupletsInGPU.score_rphisum[ix] == quintupletsInGPU.score_rphisum[jx]) && (ix<jx)){
              rmQuintupletToMemory(quintupletsInGPU,ix);continue; // keept shorted track
            }
          }
        }}
      }}
}

__device__ float scorepT3(struct SDL::modules& modulesInGPU,struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU,struct SDL::segments& segmentsInGPU,struct SDL::triplets& tripletsInGPU, unsigned int innerPix, unsigned int outerTrip, float pt, float pz)
{
        unsigned int hits1[10];// = {-1,-1,-1,-1};
        hits1[0] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*innerPix]];
        hits1[1] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*innerPix]+1];
        hits1[2] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*innerPix+1]];
        hits1[3] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*innerPix+1]+1];
        hits1[4] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTrip]]];// outer trip, inner seg, inner md, inner hit
        hits1[5] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTrip]]+1];// o t, is, im oh
        hits1[6] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTrip]+1]]; //ot is om ih
        hits1[7] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTrip]+1]+1];
        hits1[8] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTrip+1]+1]];// ot os om ih
        hits1[9] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*outerTrip+1]+1]+1];

        float r1 = hitsInGPU.rts[hits1[0]];
        float z1 = hitsInGPU.zs[hits1[0]]; 
        float r2 = hitsInGPU.rts[hits1[3]];
        float z2 = hitsInGPU.zs[hits1[3]]; 

        float slope_barrel = (z2-z1)/(r2-r1);
        float slope_endcap = (r2-r1)/(z2-z1);

        float score = 0;
        for(unsigned int i=4; i <10; i++){
          float z = hitsInGPU.zs[hits1[i]];
          float r = hitsInGPU.rts[hits1[i]]; // cm
          float subdet = modulesInGPU.subdets[hitsInGPU.moduleIndices[hits1[i]]];
          float drdz = modulesInGPU.drdzs[hitsInGPU.moduleIndices[hits1[i]]];
          float var=0;
          if(subdet == 5) {// 5== barrel
            var = slope_barrel*(r-r1) - (z-z1);
          }else{
            var = slope_endcap*(z-z1) - (r-r1);
          }
          float err;
          if(modulesInGPU.moduleLayerType[hitsInGPU.moduleIndices[hits1[i]]]==0){
            err=0.15*cos(atan(drdz));//(1.5mm)^2
          }else{ err=5.0*cos(atan(drdz));}//(5cm)^2
          score += (var*var) / (err*err);
        }
        //printf("pT3 score: %f\n",score);
        return score;
}
__device__ inline int checkHitspT5(unsigned int ix, unsigned int jx,struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::hits& hitsInGPU)
{
        int phits1[4] = {-1,-1,-1,-1};
        int phits2[4] = {-1,-1,-1,-1};
          phits1[0] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*ix]]];
          phits1[1] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*ix+1]]];
          phits1[2] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*ix]+1]];
          phits1[3] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*ix+1]+1]];

          phits2[0] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*jx]]];
          phits2[1] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*jx+1]]];
          phits2[2] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*jx]+1]];
          phits2[3] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*jx+1]+1]];

        int npMatched =0;
        for (int i =0; i<4;i++){
          bool pmatched = false;
          if(phits1[i] == -1){continue;}
          for (int j =0; j<4; j++){
            if(phits2[j] == -1){continue;}
            if(phits1[i] == phits2[j]){pmatched = true; break;}
          }
          if(pmatched){npMatched++;}
        }
        return npMatched;
}
__device__ inline int* checkHitspT3(unsigned int ix, unsigned int jx,struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU,struct SDL::hits& hitsInGPU)
{
        int phits1[4] = {-1,-1,-1,-1};
        int phits2[4] = {-1,-1,-1,-1};
          phits1[0] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTripletsInGPU.pixelSegmentIndices[ix]]]];
          phits1[1] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTripletsInGPU.pixelSegmentIndices[ix]+1]]];
          phits1[2] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTripletsInGPU.pixelSegmentIndices[ix]]+1]];
          phits1[3] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTripletsInGPU.pixelSegmentIndices[ix]+1]+1]];

          phits2[0] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTripletsInGPU.pixelSegmentIndices[jx]]]];
          phits2[1] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTripletsInGPU.pixelSegmentIndices[jx]+1]]];
          phits2[2] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTripletsInGPU.pixelSegmentIndices[jx]]+1]];
          phits2[3] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*pixelTripletsInGPU.pixelSegmentIndices[jx]+1]+1]];

        int npMatched =0;
        for (int i =0; i<4;i++){
          bool pmatched = false;
          for (int j =0; j<4; j++){
            if(phits1[i] == phits2[j]){pmatched = true; break;}
          }
          if(pmatched){npMatched++;}
        }
        int hits1[6] = {-1,-1,-1,-1,-1,-1};
        int hits2[6] = {-1,-1,-1,-1,-1,-1};
          hits1[0] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[ix]]]];// outer trip, inner seg, inner md, inner hit
          hits1[1] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[ix]]]+1];// o t, is, im oh
          hits1[2] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[ix]]+1]]; //ot is om ih
          hits1[3] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[ix]]+1]+1];
          hits1[4] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[ix]+1]+1]];// ot os om ih
          hits1[5] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[ix]+1]+1]+1];

          hits2[0] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[jx]]]];// outer trip, inner seg, inner md, inner hit
          hits2[1] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[jx]]]+1];// o t, is, im oh
          hits2[2] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[jx]]+1]]; //ot is om ih
          hits2[3] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[jx]]+1]+1];
          hits2[4] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[jx]+1]+1]];// ot os om ih
          hits2[5] = mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*pixelTripletsInGPU.tripletIndices[jx]+1]+1]+1];

        int nMatched =0;
        for (int i =0; i<6;i++){
          bool matched = false;
          for (int j =0; j<6; j++){
            if(hits1[i] == hits2[j]){matched = true; break;}
          }
          if(matched){nMatched++;}
        }

        //if((nMatched >= 6) & (npMatched >= 4)){return true;}
        //if((nMatched >= 2) & (npMatched >= 1)){return true;}
        //if((nMatched  + npMatched >= 10)){return true;}
        int matched[2] = {npMatched,nMatched};
        return matched;//nMatched+npMatched;
}
__device__ int duplicateCounter_pT3 =0;
__global__ void removeDupPixelTripletsInGPUFromMap(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU, struct SDL::triplets& tripletsInGPU)
{
    int dup_count=0;
    //for (unsigned int ix=0; ix<*pixelTrackletsInGPU.nPixelTracklets; ix++){
    for (unsigned int ix=blockIdx.x*blockDim.x+threadIdx.x; ix<*pixelTripletsInGPU.nPixelTriplets; ix+=blockDim.x*gridDim.x)
    {
      bool isDup = false;
//      if(pixelTripletsInGPU.isDup[ix]){continue;}
//      float pt1 = pixelTrackletsInGPU.pt[ix];
      float eta1_pix = pixelTripletsInGPU.eta_pix[ix];
      float phi1_pix = pixelTripletsInGPU.phi_pix[ix];
      float eta1     = pixelTripletsInGPU.eta[ix];
      float phi1     = pixelTripletsInGPU.phi[ix];
      //float pt1     = pixelTripletsInGPU.pt[ix];
      //for (unsigned int jx=ix+1; jx<*pixelTripletsInGPU.nPixelTriplets-1; jx++){
      for (unsigned int jx=0; jx<*pixelTripletsInGPU.nPixelTriplets; jx++){
       // if(pixelTripletsInGPU.isDup[jx]){continue;}
        //float pt2 = pixelTripletsInGPU.pt[jx];
        //if(abs(1./pt1 - 1./pt2) > 0.5){continue;}
        float eta2_pix = pixelTripletsInGPU.eta_pix[jx];
        float phi2_pix = pixelTripletsInGPU.phi_pix[jx];
        float dEta_pix = abs(eta1_pix-eta2_pix);
        float dPhi_pix = abs(phi1_pix-phi2_pix);
        if(dPhi_pix > M_PI){dPhi_pix = dPhi_pix - 2*M_PI;}
        //if (dEta_pix > 0.005){continue;}
        //if (abs(dPhi_pix) > 0.005){continue;}
        float dR2_pix = dEta_pix*dEta_pix + dPhi_pix*dPhi_pix;
        //if(dR2_pix < 0.0001){
        //  isDup=true;break;
        //}

        float eta2 = pixelTripletsInGPU.eta[jx];
        float phi2 = pixelTripletsInGPU.phi[jx];
        float dEta = abs(eta1-eta2);
        float dPhi = abs(phi1-phi2);
        if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
        //if (dEta > 0.005){continue;}
        //if (dPhi > 0.005){continue;}
        float dR2 = dEta*dEta + dPhi*dPhi;
        int* nMatched = checkHitspT3(ix,jx,mdsInGPU,segmentsInGPU,tripletsInGPU,pixelTripletsInGPU,hitsInGPU); 
        //if(nMatched ==5){
        //  dup_count++;
        //  if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > 2){
        //    rmPixelTripletToMemory(pixelTripletsInGPU,ix);break; // keept shorted track
        //  }
        //}
        //if(nMatched ==6){
        //  dup_count++;
        //  if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > 2){
        //    rmPixelTripletToMemory(pixelTripletsInGPU,ix);break; // keept shorted track
        //  }
        //}
        //if(nMatched >=6){
        //  dup_count++;
        //  //if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > .2){
        //  if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > 0){
        //    rmPixelTripletToMemory(pixelTripletsInGPU,ix);break; // keept shorted track
        //  }
        //  else if(nMatched ==10 && ix-jx >0){
        //    rmPixelTripletToMemory(pixelTripletsInGPU,ix);break; // keept shorted track
        //  }
        //}
        //else if(dR2 < 0.00001 || dR2_pix < 0.00001){
        //  dup_count++;
        //  //if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > .2){
        //  if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > 0){
        //    rmPixelTripletToMemory(pixelTripletsInGPU,ix);break; // keept shorted track
        //  }
        //}
        unsigned int pLS_ix = pixelTripletsInGPU.pixelSegmentIndices[ix];
        unsigned int pLS_jx = pixelTripletsInGPU.pixelSegmentIndices[jx];
        //if(nMatched[0] >=1){
        //printf("%d\n",nMatched[0]);
        //}
        //if((nMatched[1] >= 4) || (pLS_ix == pLS_jx))
        if((nMatched[0] + nMatched[1]) >= 7)
        {
          dup_count++;
          //if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > .2){
          if( pixelTripletsInGPU.score[ix] > pixelTripletsInGPU.score[jx])
          {
                rmPixelTripletToMemory(pixelTripletsInGPU,ix);
                break;
          }
          if( (pixelTripletsInGPU.score[ix] == pixelTripletsInGPU.score[jx]) && (ix<jx))
          {
                rmPixelTripletToMemory(pixelTripletsInGPU,ix);
                break;
          }
        }
        //if(nMatched ==8){
        //  dup_count++;
        //  //if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > 0.6){
        //  if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > 0.6){
        //    rmPixelTripletToMemory(pixelTripletsInGPU,ix);break; // keept shorted track
        //  }
        //}
        //if(nMatched ==9){
        //  //if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > 0.01){
        //  if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > 0.01){
        //    rmPixelTripletToMemory(pixelTripletsInGPU,ix);break; // keept shorted track
        //  }
        //}
        //if(nMatched ==10){
        //  dup_count++;
        //  if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] >= 0){
        //    rmPixelTripletToMemory(pixelTripletsInGPU,ix);break; // keept shorted track
        //  }
        //}

      }
    }
}

__global__ void markUsedObjects(struct SDL::modules& modulesInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, struct SDL::quintuplets& quintupletsInGPU)
{
    for (unsigned int ix=blockIdx.x*blockDim.x+threadIdx.x; ix<*pixelQuintupletsInGPU.nPixelQuintuplets; ix+=blockDim.x*gridDim.x){
           //mark the relevant T5 and pT3 here!
           if(pixelQuintupletsInGPU.isDup[ix]) {continue;}
           unsigned int quintupletIndex = pixelQuintupletsInGPU.T5Indices[ix];
           unsigned int pixelSegmentArrayIndex = pixelQuintupletsInGPU.pixelIndices[ix]- ((*modulesInGPU.nModules - 1)* N_MAX_SEGMENTS_PER_MODULE);
           quintupletsInGPU.partOfPT5[quintupletIndex] = true;
           unsigned int innerTripletIndex = quintupletsInGPU.tripletIndices[2 * quintupletIndex];
           unsigned int outerTripletIndex = quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1];
           tripletsInGPU.partOfPT5[innerTripletIndex] = true;
           tripletsInGPU.partOfPT5[outerTripletIndex] = true;
           segmentsInGPU.partOfPT5[pixelSegmentArrayIndex] = true;
    
    }
}
__global__ void removeDupPixelQuintupletsInGPUFromMap(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, struct SDL::quintuplets& quintupletsInGPU)
{
    int dup_count=0;
    //for (unsigned int ix=0; ix<*pixelTrackletsInGPU.nPixelTracklets; ix++){
    for (unsigned int ix=blockIdx.x*blockDim.x+threadIdx.x; ix<*pixelQuintupletsInGPU.nPixelQuintuplets; ix+=blockDim.x*gridDim.x){
      bool isDup = false;
//      if(pixelTripletsInGPU.isDup[ix]){continue;}
//      float pt1 = pixelTrackletsInGPU.pt[ix];
//      float eta1_pix = pixelTripletsInGPU.eta_pix[ix];
//      float phi1_pix = pixelTripletsInGPU.phi_pix[ix];
//      float eta1     = pixelTripletsInGPU.eta[ix];
//      float phi1     = pixelTripletsInGPU.phi[ix];
      //float pt1     = pixelTripletsInGPU.pt[ix];
      //for (unsigned int jx=ix+1; jx<*pixelTripletsInGPU.nPixelTriplets-1; jx++){
      for (unsigned int jx=0; jx<*pixelQuintupletsInGPU.nPixelQuintuplets; jx++){
      if(ix==jx){continue;}
       // if(pixelTripletsInGPU.isDup[jx]){continue;}
        //float pt2 = pixelTripletsInGPU.pt[jx];
        //if(abs(1./pt1 - 1./pt2) > 0.5){continue;}
//        float eta2_pix = pixelTripletsInGPU.eta_pix[jx];
//        float phi2_pix = pixelTripletsInGPU.phi_pix[jx];
//        float dEta_pix = abs(eta1_pix-eta2_pix);
//        float dPhi_pix = abs(phi1_pix-phi2_pix);
//        if(dPhi_pix > M_PI){dPhi_pix = dPhi_pix - 2*M_PI;}
//        //if (dEta_pix > 0.005){continue;}
//        //if (abs(dPhi_pix) > 0.005){continue;}
//        float dR2_pix = dEta_pix*dEta_pix + dPhi_pix*dPhi_pix;
//        //if(dR2_pix < 0.0001){
//        //  isDup=true;break;
//        //}
//
//        float eta2 = pixelTripletsInGPU.eta[jx];
//        float phi2 = pixelTripletsInGPU.phi[jx];
//        float dEta = abs(eta1-eta2);
//        float dPhi = abs(phi1-phi2);
//        if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
//        //if (dEta > 0.005){continue;}
//        //if (dPhi > 0.005){continue;}
//        float dR2 = dEta*dEta + dPhi*dPhi;
        unsigned int T5_ix = pixelQuintupletsInGPU.T5Indices[ix];
        unsigned int T5_jx = pixelQuintupletsInGPU.T5Indices[jx];
        unsigned int pLS_ix = pixelQuintupletsInGPU.pixelIndices[ix];
        unsigned int pLS_jx = pixelQuintupletsInGPU.pixelIndices[jx];
        int nMatched = checkHitsT5(T5_ix,T5_jx,mdsInGPU,segmentsInGPU,tripletsInGPU,quintupletsInGPU);
        int npMatched = checkHitspT5(pLS_ix,pLS_jx,mdsInGPU,segmentsInGPU,hitsInGPU);
        //if(T5_ix==T5_jx || (pLS_ix==pLS_jx))
        //{
        //  dup_count++;
        //  //if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > .2){
        //  if( pixelQuintupletsInGPU.score[ix] > pixelQuintupletsInGPU.score[jx] )
        //  {
        //        rmPixelQuintupletToMemory(pixelQuintupletsInGPU,ix);
        //        break; // keept shorted track
        //  }
        //  if( (pixelQuintupletsInGPU.score[ix] == pixelQuintupletsInGPU.score[jx]) && (ix<jx))
        //  {
        //        rmPixelQuintupletToMemory(pixelQuintupletsInGPU,ix);
        //        break; // keept shorted track
        //  }
        //}
        if((nMatched + npMatched) >=10)
        //if(nMatched >= 7 || (npMatched>=3))
        {
          dup_count++;
          //if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > .2){
          if( pixelQuintupletsInGPU.score[ix] > pixelQuintupletsInGPU.score[jx])
          {
                rmPixelQuintupletToMemory(pixelQuintupletsInGPU,ix);
                break; // keept shorted track
          }
          if( (pixelQuintupletsInGPU.score[ix] == pixelQuintupletsInGPU.score[jx]) && (ix>jx))
          {
                rmPixelQuintupletToMemory(pixelQuintupletsInGPU,ix);
                break; // keept shorted track
          }
        }
       int pixelModuleIndex = *modulesInGPU.nModules - 1;
       unsigned int prefix = pixelModuleIndex*N_MAX_SEGMENTS_PER_MODULE; 
       float eta1 = segmentsInGPU.eta[pLS_ix - prefix];
       float eta2 = segmentsInGPU.eta[pLS_jx - prefix];
       float phi1 = segmentsInGPU.phi[pLS_ix - prefix];
       float phi2 = segmentsInGPU.phi[pLS_jx - prefix];
       float dEta = abs(eta1-eta2);
       float dPhi = abs(phi1-phi2);
       if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
       float dR2 = dEta*dEta + dPhi*dPhi;
       // if(dR2<0.001)
       // {
       //   dup_count++;
       //   //if( pixelTripletsInGPU.score[ix] - pixelTripletsInGPU.score[jx] > .2){
       //   if( pixelQuintupletsInGPU.score[ix] - pixelQuintupletsInGPU.score[jx] > 0)
       //   {
       //         rmPixelQuintupletToMemory(pixelQuintupletsInGPU,ix);
       //         break; // keept shorted track
       //   }
       //   if( (pixelQuintupletsInGPU.score[ix] = pixelQuintupletsInGPU.score[jx]) && (ix>jx))
       //   {
       //         rmPixelQuintupletToMemory(pixelQuintupletsInGPU,ix);
       //         break; // keept shorted track
       //   }
       // }

      }
    }
}
__global__ void checkHitspLS(struct SDL::modules& modulesInGPU,struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::hits& hitsInGPU){
     int counter=0;
     int pixelModuleIndex = *modulesInGPU.nModules - 1;
     unsigned int prefix = pixelModuleIndex*N_MAX_SEGMENTS_PER_MODULE;
     unsigned int nPixelSegments = segmentsInGPU.nSegments[pixelModuleIndex];
     if(nPixelSegments >  N_MAX_PIXEL_SEGMENTS_PER_MODULE) { nPixelSegments =  N_MAX_PIXEL_SEGMENTS_PER_MODULE;}
     for(int ix=blockIdx.x*blockDim.x+threadIdx.x;ix<nPixelSegments;ix+=blockDim.x*gridDim.x){
       bool found=false;
       unsigned int phits1[4] ;
       phits1[0] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*(prefix+ix)]]];
       phits1[1] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*(prefix+ix)+1]]];
       phits1[2] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*(prefix+ix)]+1]];
       phits1[3] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*(prefix+ix)+1]+1]];
       float eta_pix1 = segmentsInGPU.eta[ix];
       float phi_pix1 = segmentsInGPU.phi[ix];
       float pt1 = segmentsInGPU.ptIn[ix];
       for(int jx=0;jx<nPixelSegments;jx++){
         if(ix==jx){continue;}
         unsigned int phits2[4] ;
         float ptErr_diff = segmentsInGPU.ptErr[ix] - segmentsInGPU.ptErr[jx];
         if (ptErr_diff>0){continue;}// allows for exact matches to be checked

         phits2[0] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*(prefix+jx)]]];
         phits2[1] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*(prefix+jx)+1]]];
         phits2[2] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*(prefix+jx)]+1]];
         phits2[3] = hitsInGPU.idxs[mdsInGPU.hitIndices[2*segmentsInGPU.mdIndices[2*(prefix+jx)+1]+1]];
         float eta_pix2 = segmentsInGPU.eta[jx];
         float phi_pix2 = segmentsInGPU.phi[jx];
         float pt2 = segmentsInGPU.ptIn[jx];
         if(abs(1/pt1 - 1/pt2)> 0.1){continue;}

         int npMatched =0;
         for (int i =0; i<4;i++){
           bool pmatched = false;
           for (int j =0; j<4; j++){
             if(phits1[i] == phits2[j]){pmatched = true; break;}
           }
           if(pmatched){npMatched++;}
         }
         if((npMatched ==4) && (ix < jx)){ // if exact match, remove only 1
           //printf("pLS npMatched: %d\n",npMatched);
           found=true;break;
         }
         if(npMatched ==3){
           //printf("pLS npMatched: %d\n",npMatched);
           found=true;break;
         }
         float dEta = abs(eta_pix1-eta_pix2);
         float dPhi = abs(phi_pix1-phi_pix2);
         //if (dEta > 0.1){continue;}
         if(dPhi > M_PI){dPhi = dPhi - 2*M_PI;}
         //if (abs(dPhi) > 0.1){continue;}
         float dR2 = dEta*dEta + dPhi*dPhi;
         //if(dR2 < 0.0001){
         //  //printf("dR: %.2f\n",dR2);
         //  found=true;break;
         //}
       }
       if(found){counter++;rmPixelSegmentFromMemory(segmentsInGPU,ix);continue;}
     }
     //printf("%u %d\n",nPixelSegments,counter);
}
