#include "hip/hip_runtime.h"
#include "Triplet.cuh"

void SDL::triplets::resetMemory(unsigned int maxTriplets, unsigned int nLowerModules,hipStream_t stream)
{
    hipMemsetAsync(segmentIndices,0, 5 * maxTriplets * sizeof(unsigned int),stream);
    hipMemsetAsync(nTriplets,0, nLowerModules * sizeof(unsigned int),stream);
    hipMemsetAsync(totOccupancyTriplets,0, nLowerModules * sizeof(unsigned int),stream);
    hipMemsetAsync(betaIn,0, maxTriplets * 3 * sizeof(FPX),stream);
    hipMemsetAsync(partOfPT5,0, maxTriplets * sizeof(bool),stream);
    hipMemsetAsync(partOfT5,0, maxTriplets * sizeof(bool), stream);
    hipMemsetAsync(partOfPT3, 0, maxTriplets * sizeof(bool), stream);
}

__global__ void SDL::createTripletArrayRanges(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct segments& segmentsInGPU)
{

    short module_subdets;
    short module_layers;
    short module_rings;
    float module_eta;
    __shared__ unsigned int nTotalTriplets;
    nTotalTriplets = 0; //start!   
    __syncthreads();

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int np = gridDim.x * blockDim.x;
    for(uint16_t i = gid; i < *modulesInGPU.nLowerModules; i+= np)
    {
        if(segmentsInGPU.nSegments[i] == 0){
          rangesInGPU.tripletModuleIndices[i] = nTotalTriplets;
          rangesInGPU.tripletModuleOccupancy[i] = 0;
          continue;
        }
        module_subdets = modulesInGPU.subdets[i];
        module_layers = modulesInGPU.layers[i];
        module_rings = modulesInGPU.rings[i];
        module_eta = abs(modulesInGPU.eta[i]);
        unsigned int occupancy;
        unsigned int category_number, eta_number;
        if (module_layers<=3 && module_subdets==5) category_number = 0;
        else if (module_layers>=4 && module_subdets==5) category_number = 1;
        else if (module_layers<=2 && module_subdets==4 && module_rings>=11) category_number = 2;
        else if (module_layers>=3 && module_subdets==4 && module_rings>=8) category_number = 2;
        else if (module_layers<=2 && module_subdets==4 && module_rings<=10) category_number = 3;
        else if (module_layers>=3 && module_subdets==4 && module_rings<=7) category_number = 3;
        if (module_eta<0.75) eta_number=0;
        else if (module_eta>0.75 && module_eta<1.5) eta_number=1;
        else if (module_eta>1.5 && module_eta<2.25) eta_number=2;
        else if (module_eta>2.25 && module_eta<3) eta_number=3;

        if (category_number == 0 && eta_number == 0) occupancy = 543;
        else if (category_number == 0 && eta_number == 1) occupancy = 235;
        else if (category_number == 0 && eta_number == 2) occupancy = 88;
        else if (category_number == 0 && eta_number == 3) occupancy = 46;
        else if (category_number == 1 && eta_number == 0) occupancy = 755;
        else if (category_number == 1 && eta_number == 1) occupancy = 347;
        else if (category_number == 2 && eta_number == 1) occupancy = 0;
        else if (category_number == 2 && eta_number == 2) occupancy = 0;
        else if (category_number == 3 && eta_number == 1) occupancy = 38;
        else if (category_number == 3 && eta_number == 2) occupancy = 46;
        else if (category_number == 3 && eta_number == 3) occupancy = 39;

        rangesInGPU.tripletModuleOccupancy[i] = occupancy;
        unsigned int nTotT = atomicAdd(&nTotalTriplets,occupancy);
        rangesInGPU.tripletModuleIndices[i] = nTotT;
    }
    __syncthreads();
    if(threadIdx.x==0){
      *rangesInGPU.device_nTotalTrips = nTotalTriplets;
    }
}

void SDL::createTripletsInExplicitMemory(struct triplets& tripletsInGPU, unsigned int maxTriplets, uint16_t nLowerModules, hipStream_t stream)
{
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    tripletsInGPU.segmentIndices = (unsigned int*)cms::cuda::allocate_device(dev,maxTriplets * sizeof(unsigned int) *2,stream);
    tripletsInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev,maxTriplets * sizeof(uint16_t) *3,stream);
    tripletsInGPU.betaIn = (FPX*)cms::cuda::allocate_device(dev,maxTriplets * sizeof(FPX) *3,stream);
    tripletsInGPU.nTriplets = (int*)cms::cuda::allocate_device(dev,nLowerModules * sizeof(int),stream);
    tripletsInGPU.totOccupancyTriplets = (int*)cms::cuda::allocate_device(dev,nLowerModules * sizeof(int),stream);
    tripletsInGPU.partOfPT5 = (bool*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(bool), stream);
    tripletsInGPU.partOfPT3 = (bool*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(bool), stream);
    tripletsInGPU.partOfT5 = (bool*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(bool), stream);

    tripletsInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_device(dev, maxTriplets * 3 * sizeof(uint8_t), stream);
    tripletsInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, maxTriplets * 6 * sizeof(unsigned int), stream);
    tripletsInGPU.nMemoryLocations = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);

#ifdef CUT_VALUE_DEBUG
    tripletsInGPU.zOut = (float*)cms::cuda::allocate_device(dev, maxTriplets * 4 * sizeof(float), stream);
    tripletsInGPU.zLo = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.zHi = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.zLoPointed = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.zHiPointed = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.sdlCut = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.betaInCut = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.betaOutCut = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.deltaBetaCut = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.rtLo = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.rtHi = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.kZ = (float*)cms::cuda::allocate_device(dev, maxTriplets * sizeof(float), stream);
    tripletsInGPU.rtOut = tripletsInGPU.zOut + maxTriplets;
    tripletsInGPU.deltaPhiPos = tripletsInGPU.zOut + maxTriplets *2;
    tripletsInGPU.deltaPhi = tripletsInGPU.zOut + maxTriplets *3;
#endif

#else
    hipMalloc(&tripletsInGPU.segmentIndices, /*5*/2 * maxTriplets * sizeof(unsigned int));
    hipMalloc(&tripletsInGPU.lowerModuleIndices, 3 * maxTriplets * sizeof(uint16_t));
    hipMalloc(&tripletsInGPU.betaIn, maxTriplets * 3 * sizeof(FPX));
    hipMalloc(&tripletsInGPU.nTriplets, nLowerModules * sizeof(int));
    hipMalloc(&tripletsInGPU.totOccupancyTriplets, nLowerModules * sizeof(int));
    hipMalloc(&tripletsInGPU.partOfPT5, maxTriplets * sizeof(bool));
    hipMalloc(&tripletsInGPU.partOfPT3, maxTriplets * sizeof(bool));
    hipMalloc(&tripletsInGPU.partOfT5, maxTriplets * sizeof(bool));

    hipMalloc(&tripletsInGPU.logicalLayers, maxTriplets * 3 * sizeof(uint8_t));
    hipMalloc(&tripletsInGPU.hitIndices, maxTriplets * 6 * sizeof(unsigned int));
    hipMalloc(&tripletsInGPU.nMemoryLocations, sizeof(unsigned int));

#ifdef CUT_VALUE_DEBUG
    hipMalloc(&tripletsInGPU.zOut, maxTriplets * 4*sizeof(unsigned int));
    hipMalloc(&tripletsInGPU.zLo, maxTriplets * sizeof(float));
    hipMalloc(&tripletsInGPU.zHi, maxTriplets * sizeof(float));
    hipMalloc(&tripletsInGPU.zLoPointed, maxTriplets * sizeof(float));
    hipMalloc(&tripletsInGPU.zHiPointed, maxTriplets * sizeof(float));
    hipMalloc(&tripletsInGPU.sdlCut, maxTriplets * sizeof(float));
    hipMalloc(&tripletsInGPU.betaInCut, maxTriplets * sizeof(float));
    hipMalloc(&tripletsInGPU.betaOutCut, maxTriplets * sizeof(float));
    hipMalloc(&tripletsInGPU.deltaBetaCut, maxTriplets * sizeof(float));
    hipMalloc(&tripletsInGPU.rtLo, maxTriplets * sizeof(float));
    hipMalloc(&tripletsInGPU.rtHi, maxTriplets * sizeof(float));
    hipMalloc(&tripletsInGPU.kZ, maxTriplets * sizeof(float));

    tripletsInGPU.rtOut = tripletsInGPU.zOut + maxTriplets;
    tripletsInGPU.deltaPhiPos = tripletsInGPU.zOut + maxTriplets *2;
    tripletsInGPU.deltaPhi = tripletsInGPU.zOut + maxTriplets *3;
#endif

#endif
    hipMemsetAsync(tripletsInGPU.nTriplets,0,nLowerModules * sizeof(int),stream);
    hipMemsetAsync(tripletsInGPU.totOccupancyTriplets,0,nLowerModules * sizeof(int),stream);
    hipMemsetAsync(tripletsInGPU.partOfPT5,0,maxTriplets * sizeof(bool),stream);
    hipMemsetAsync(tripletsInGPU.partOfPT3,0,maxTriplets * sizeof(bool),stream);
    hipMemsetAsync(tripletsInGPU.partOfT5,0,maxTriplets * sizeof(bool),stream);
    
    hipStreamSynchronize(stream);

    tripletsInGPU.betaOut = tripletsInGPU.betaIn + maxTriplets;
    tripletsInGPU.pt_beta = tripletsInGPU.betaIn + maxTriplets * 2;
}

SDL::triplets::triplets()
{
    segmentIndices = nullptr;
    lowerModuleIndices = nullptr;
    betaIn = nullptr;
    betaOut = nullptr;
    pt_beta = nullptr;
    logicalLayers = nullptr;
    hitIndices = nullptr;
#ifdef CUT_VALUE_DEBUG
    zOut = nullptr;
    rtOut = nullptr;
    deltaPhiPos = nullptr;
    deltaPhi = nullptr;
    zLo = nullptr;
    zHi = nullptr;
    rtLo = nullptr;
    rtHi = nullptr;
    zLoPointed = nullptr;
    zHiPointed = nullptr;
    kZ = nullptr;
    betaInCut = nullptr;
    betaOutCut = nullptr;
    deltaBetaCut = nullptr;
    sdlCut = nullptr;
#endif
}

SDL::triplets::~triplets()
{
}

void SDL::triplets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,segmentIndices);
    cms::cuda::free_device(dev,lowerModuleIndices);
    cms::cuda::free_device(dev,betaIn);
    cms::cuda::free_device(dev,nTriplets);
    cms::cuda::free_device(dev,totOccupancyTriplets);
    cms::cuda::free_device(dev, partOfPT5);
    cms::cuda::free_device(dev, partOfPT3);
    cms::cuda::free_device(dev, partOfT5);
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, nMemoryLocations);
#ifdef CUT_VALUE_DEBUG
    cms::cuda::free_device(dev, zOut);
    cms::cuda::free_device(dev, zLo);
    cms::cuda::free_device(dev, zHi);
    cms::cuda::free_device(dev, zLoPointed);
    cms::cuda::free_device(dev, zHiPointed);
    cms::cuda::free_device(dev, sdlCut);
    cms::cuda::free_device(dev, betaInCut);
    cms::cuda::free_device(dev, betaOutCut);
    cms::cuda::free_device(dev, deltaBetaCut);
    cms::cuda::free_device(dev, rtLo);
    cms::cuda::free_device(dev, rtHi);
    cms::cuda::free_device(dev, kZ);
#endif
}

void SDL::triplets::freeMemory(hipStream_t stream)
{
    hipFree(segmentIndices);
    hipFree(lowerModuleIndices);
    hipFree(nTriplets);
    hipFree(totOccupancyTriplets);
    hipFree(betaIn);
    hipFree(partOfPT5);
    hipFree(partOfPT3);
    hipFree(partOfT5);
    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(nMemoryLocations);
#ifdef CUT_VALUE_DEBUG
    hipFree(zOut);
    hipFree(zLo);
    hipFree(zHi);
    hipFree(rtLo);
    hipFree(rtHi);
    hipFree(zLoPointed);
    hipFree(zHiPointed);
    hipFree(kZ);
    hipFree(betaInCut);
    hipFree(betaOutCut);
    hipFree(deltaBetaCut);
    hipFree(sdlCut);
#endif
    hipStreamSynchronize(stream);
}

__global__ void SDL::addTripletRangesToEventExplicit(struct modules& modulesInGPU, struct triplets& tripletsInGPU, struct objectRanges& rangesInGPU)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int np = gridDim.x * blockDim.x;
    for(uint16_t i = gid; i < *modulesInGPU.nLowerModules; i+= np)
    {
        if(tripletsInGPU.nTriplets[i] == 0)
        {
            rangesInGPU.tripletRanges[i * 2] = -1;
            rangesInGPU.tripletRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU.tripletRanges[i * 2] = rangesInGPU.tripletModuleIndices[i];
            rangesInGPU.tripletRanges[i * 2 + 1] = rangesInGPU.tripletModuleIndices[i] +  tripletsInGPU.nTriplets[i] - 1;
        }
    }
}