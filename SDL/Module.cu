#include "hip/hip_runtime.h"
#include "Module.cuh"

std::map <unsigned int, uint16_t> *SDL::detIdToIndex;
std::map <unsigned int, float> *SDL::module_x;
std::map <unsigned int, float> *SDL::module_y;
std::map <unsigned int, float> *SDL::module_z;
std::map <unsigned int, unsigned int> *SDL::module_type; // 23 : Ph2PSP, 24 : Ph2PSS, 25 : Ph2SS
// https://github.com/cms-sw/cmssw/blob/5e809e8e0a625578aa265dc4b128a93830cb5429/Geometry/TrackerGeometryBuilder/interface/TrackerGeometry.h#L29

void SDL::createRangesInExplicitMemory(struct objectRanges& rangesInGPU,unsigned int nModules,hipStream_t stream, unsigned int nLowerModules)
{
    /* modules stucture object will be created in Event.cu*/
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    rangesInGPU.hitRanges = (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.hitRangesLower = (int*)cms::cuda::allocate_device(dev,nModules * sizeof(int),stream);
    rangesInGPU.hitRangesUpper = (int*)cms::cuda::allocate_device(dev,nModules * sizeof(int),stream);
    rangesInGPU.hitRangesnLower = (int8_t*)cms::cuda::allocate_device(dev,nModules * sizeof(int8_t),stream);
    rangesInGPU.hitRangesnUpper = (int8_t*)cms::cuda::allocate_device(dev,nModules * sizeof(int8_t),stream);
    rangesInGPU.mdRanges = (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.segmentRanges = (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.trackletRanges = (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.tripletRanges = (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.trackCandidateRanges = (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.quintupletRanges = (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.nEligibleT5Modules = (uint16_t*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);

    rangesInGPU.quintupletModuleIndices = (int*)cms::cuda::allocate_device(dev,nLowerModules * sizeof(int),stream);
    rangesInGPU.quintupletModuleOccupancy = (int*)cms::cuda::allocate_device(dev,nLowerModules * sizeof(int),stream);
    rangesInGPU.miniDoubletModuleIndices = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) * sizeof(int), stream);
    rangesInGPU.miniDoubletModuleOccupancy = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) * sizeof(int), stream);
    rangesInGPU.segmentModuleIndices = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) * sizeof(int), stream);
    rangesInGPU.segmentModuleOccupancy = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) * sizeof(int), stream);
    rangesInGPU.tripletModuleIndices = (int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(int), stream);
    rangesInGPU.tripletModuleOccupancy = (int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(int), stream);

    rangesInGPU.device_nTotalMDs = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);
    rangesInGPU.device_nTotalSegs = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);
    rangesInGPU.device_nTotalTrips = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);
    rangesInGPU.device_nTotalQuints = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);

#else
    hipMalloc(&rangesInGPU.hitRanges,nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.hitRangesLower,nModules  * sizeof(int));
    hipMalloc(&rangesInGPU.hitRangesUpper,nModules  * sizeof(int));
    hipMalloc(&rangesInGPU.hitRangesnLower,nModules  * sizeof(int8_t));
    hipMalloc(&rangesInGPU.hitRangesnUpper,nModules  * sizeof(int8_t));
    hipMalloc(&rangesInGPU.mdRanges,nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.segmentRanges,nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.trackletRanges,nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.tripletRanges,nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.trackCandidateRanges, nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.quintupletRanges, nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.nEligibleT5Modules, sizeof(uint16_t));
    hipMalloc(&rangesInGPU.quintupletModuleIndices, nLowerModules * sizeof(int));
    hipMalloc(&rangesInGPU.quintupletModuleOccupancy, nLowerModules * sizeof(int));

    hipMalloc(&rangesInGPU.miniDoubletModuleIndices, (nLowerModules + 1) * sizeof(int));
    hipMalloc(&rangesInGPU.miniDoubletModuleOccupancy, (nLowerModules + 1) * sizeof(int));
    hipMalloc(&rangesInGPU.segmentModuleIndices, (nLowerModules + 1) * sizeof(int));
    hipMalloc(&rangesInGPU.segmentModuleOccupancy, (nLowerModules + 1) * sizeof(int));
    hipMalloc(&rangesInGPU.tripletModuleIndices, nLowerModules * sizeof(int));
    hipMalloc(&rangesInGPU.tripletModuleOccupancy, nLowerModules * sizeof(int));
    
    hipMalloc(&rangesInGPU.device_nTotalMDs, sizeof(unsigned int));
    hipMalloc(&rangesInGPU.device_nTotalSegs, sizeof(unsigned int));
    hipMalloc(&rangesInGPU.device_nTotalTrips, sizeof(unsigned int));
    hipMalloc(&rangesInGPU.device_nTotalQuints, sizeof(unsigned int));

#endif
}

void SDL::createModulesInExplicitMemory(struct modules& modulesInGPU,unsigned int nModules,hipStream_t stream)
{
    /* modules stucture object will be created in Event.cu*/
    hipMalloc(&(modulesInGPU.detIds),nModules * sizeof(unsigned int));
    hipMalloc(&modulesInGPU.moduleMap,nModules * 40 * sizeof(uint16_t));
    hipMalloc(&modulesInGPU.mapIdx, nModules*sizeof(uint16_t));
    hipMalloc(&modulesInGPU.mapdetId, nModules*sizeof(unsigned int));
    hipMalloc(&modulesInGPU.nConnectedModules,nModules * sizeof(uint16_t));
    hipMalloc(&modulesInGPU.drdzs,nModules * sizeof(float));
    hipMalloc(&modulesInGPU.slopes,nModules * sizeof(float));
    hipMalloc(&modulesInGPU.nModules,sizeof(uint16_t));
    hipMalloc(&modulesInGPU.nLowerModules,sizeof(uint16_t));
    hipMalloc(&modulesInGPU.partnerModuleIndices, nModules * sizeof(uint16_t));

    hipMalloc(&modulesInGPU.layers,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.rings,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.modules,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.rods,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.subdets,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.sides,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.eta,nModules * sizeof(float));
    hipMalloc(&modulesInGPU.r,nModules * sizeof(float));
    hipMalloc(&modulesInGPU.isInverted, nModules * sizeof(bool));
    hipMalloc(&modulesInGPU.isLower, nModules * sizeof(bool));
    hipMalloc(&modulesInGPU.isAnchor, nModules * sizeof(bool));
    hipMalloc(&modulesInGPU.moduleType,nModules * sizeof(ModuleType));
    hipMalloc(&modulesInGPU.moduleLayerType,nModules * sizeof(ModuleLayerType));

    hipMemcpyAsync(modulesInGPU.nModules,&nModules,sizeof(uint16_t),hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);
}

void SDL::objectRanges::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,hitRanges);
    cms::cuda::free_device(dev,mdRanges);
    cms::cuda::free_device(dev,segmentRanges);
    cms::cuda::free_device(dev,trackletRanges);
    cms::cuda::free_device(dev,tripletRanges);
    cms::cuda::free_device(dev,trackCandidateRanges);
    cms::cuda::free_device(dev,quintupletRanges);
    cms::cuda::free_device(dev,nEligibleT5Modules);
    cms::cuda::free_device(dev, indicesOfEligibleT5Modules);
    cms::cuda::free_device(dev,quintupletModuleIndices);
    cms::cuda::free_device(dev,quintupletModuleOccupancy);
    cms::cuda::free_device(dev, hitRangesLower);
    cms::cuda::free_device(dev, hitRangesUpper);
    cms::cuda::free_device(dev, hitRangesnLower);
    cms::cuda::free_device(dev, hitRangesnUpper);
    cms::cuda::free_device(dev, miniDoubletModuleIndices);
    cms::cuda::free_device(dev, miniDoubletModuleOccupancy);
    cms::cuda::free_device(dev, segmentModuleIndices);
    cms::cuda::free_device(dev, segmentModuleOccupancy);
    cms::cuda::free_device(dev, tripletModuleIndices);
    cms::cuda::free_device(dev, tripletModuleOccupancy);
    cms::cuda::free_device(dev, device_nTotalMDs);
    cms::cuda::free_device(dev, device_nTotalSegs);
    cms::cuda::free_device(dev, device_nTotalTrips);
    cms::cuda::free_device(dev, device_nTotalQuints);
}

void SDL::objectRanges::freeMemory()
{
    hipFree(hitRanges);
    hipFree(hitRangesLower);
    hipFree(hitRangesUpper);
    hipFree(hitRangesnLower);
    hipFree(hitRangesnUpper);
    hipFree(mdRanges);
    hipFree(segmentRanges);
    hipFree(trackletRanges);
    hipFree(tripletRanges);
    hipFree(trackCandidateRanges);
    hipFree(quintupletRanges);
    hipFree(nEligibleT5Modules);
    hipFree(indicesOfEligibleT5Modules);
    hipFree(quintupletModuleIndices);
    hipFree(quintupletModuleOccupancy);
    hipFree(miniDoubletModuleIndices);
    hipFree(miniDoubletModuleOccupancy);
    hipFree(segmentModuleIndices);
    hipFree(segmentModuleOccupancy);
    hipFree(tripletModuleIndices);
    hipFree(tripletModuleOccupancy);
    hipFree(device_nTotalMDs);
    hipFree(device_nTotalSegs);
    hipFree(device_nTotalTrips);
    hipFree(device_nTotalQuints);
}

void SDL::freeModulesCache(struct modules& modulesInGPU,struct pixelMap& pixelMapping)
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,modulesInGPU.detIds);
    cms::cuda::free_device(dev,modulesInGPU.moduleMap);
    cms::cuda::free_device(dev,modulesInGPU.mapIdx);
    cms::cuda::free_device(dev,modulesInGPU.mapdetId);
    cms::cuda::free_device(dev,modulesInGPU.nConnectedModules);
    cms::cuda::free_device(dev,modulesInGPU.drdzs);
    cms::cuda::free_device(dev,modulesInGPU.slopes);
    cms::cuda::free_device(dev,modulesInGPU.nModules);
    cms::cuda::free_device(dev,modulesInGPU.nLowerModules);
    cms::cuda::free_device(dev,modulesInGPU.layers);
    cms::cuda::free_device(dev,modulesInGPU.rings);
    cms::cuda::free_device(dev,modulesInGPU.modules);
    cms::cuda::free_device(dev,modulesInGPU.rods);
    cms::cuda::free_device(dev,modulesInGPU.subdets);
    cms::cuda::free_device(dev,modulesInGPU.sides);
    cms::cuda::free_device(dev,modulesInGPU.isInverted);
    cms::cuda::free_device(dev,modulesInGPU.isLower);
    cms::cuda::free_device(dev,modulesInGPU.isAnchor);
    cms::cuda::free_device(dev,modulesInGPU.moduleType);
    cms::cuda::free_device(dev,modulesInGPU.moduleLayerType);
    cms::cuda::free_device(dev,modulesInGPU.connectedPixels);
    hipHostFree(pixelMapping.connectedPixelsSizes);
    hipHostFree(pixelMapping.connectedPixelsSizesPos);
    hipHostFree(pixelMapping.connectedPixelsSizesNeg);
    hipHostFree(pixelMapping.connectedPixelsIndex);
    hipHostFree(pixelMapping.connectedPixelsIndexPos);
    hipHostFree(pixelMapping.connectedPixelsIndexNeg);
}

void SDL::freeModules(struct modules& modulesInGPU, struct pixelMap& pixelMapping)
{
    hipFree(modulesInGPU.detIds);
    hipFree(modulesInGPU.moduleMap);
    hipFree(modulesInGPU.mapIdx);
    hipFree(modulesInGPU.mapdetId);
    hipFree(modulesInGPU.nConnectedModules);
    hipFree(modulesInGPU.drdzs);
    hipFree(modulesInGPU.slopes);
    hipFree(modulesInGPU.nModules);
    hipFree(modulesInGPU.nLowerModules);
    hipFree(modulesInGPU.layers);
    hipFree(modulesInGPU.rings);
    hipFree(modulesInGPU.modules);
    hipFree(modulesInGPU.rods);
    hipFree(modulesInGPU.subdets);
    hipFree(modulesInGPU.sides);
    hipFree(modulesInGPU.eta);
    hipFree(modulesInGPU.r);
    hipFree(modulesInGPU.isInverted);
    hipFree(modulesInGPU.isLower);
    hipFree(modulesInGPU.isAnchor);
    hipFree(modulesInGPU.moduleType);
    hipFree(modulesInGPU.moduleLayerType);
    hipFree(modulesInGPU.connectedPixels);
    hipFree(modulesInGPU.partnerModuleIndices);

    hipHostFree(pixelMapping.connectedPixelsSizes);
    hipHostFree(pixelMapping.connectedPixelsSizesPos);
    hipHostFree(pixelMapping.connectedPixelsSizesNeg);
    hipHostFree(pixelMapping.connectedPixelsIndex);
    hipHostFree(pixelMapping.connectedPixelsIndexPos);
    hipHostFree(pixelMapping.connectedPixelsIndexNeg);
}

void SDL::loadModulesFromFile(struct modules& modulesInGPU, uint16_t& nModules, uint16_t& nLowerModules, struct pixelMap& pixelMapping,hipStream_t stream, const char* moduleMetaDataFilePath)
{
    detIdToIndex = new std::map<unsigned int, uint16_t>;
    module_x = new std::map<unsigned int, float>;
    module_y = new std::map<unsigned int, float>;
    module_z = new std::map<unsigned int, float>;
    module_type = new std::map<unsigned int, unsigned int>;

    /*modules structure object will be created in Event.cu*/
    /* Load the whole text file into the map first*/

    std::ifstream ifile;
    ifile.open(moduleMetaDataFilePath);
    if(!ifile.is_open())
    {
        std::cout<<"ERROR! module list file not present!"<<std::endl;
    }
    std::string line;
    uint16_t counter = 0;

    while(std::getline(ifile,line))
    {
        std::stringstream ss(line);
        std::string token;
        int count_number = 0;

        unsigned int temp_detId;
        while(std::getline(ss,token,','))
        {
            if(count_number == 0)
            {
                temp_detId = stoi(token);
                (*detIdToIndex)[temp_detId] = counter;
            }
            if(count_number == 1)
                (*module_x)[temp_detId] = std::stof(token);
            if(count_number == 2)
                (*module_y)[temp_detId] = std::stof(token);
            if(count_number == 3)
                (*module_z)[temp_detId] = std::stof(token);
            if(count_number == 4)
            {
                (*module_type)[temp_detId] = std::stoi(token);
                counter++;
            }
            count_number++;
            if(count_number>4)
                break;
        }

    }
    (*detIdToIndex)[1] = counter; //pixel module is the last module in the module list
    counter++;
    nModules = counter;
    //std::cout<<"Number of modules = "<<nModules<<std::endl;
    createModulesInExplicitMemory(modulesInGPU,nModules,stream);
    unsigned int* host_detIds;
    short* host_layers;
    short* host_rings;
    short* host_rods;
    short* host_modules;
    short* host_subdets;
    short* host_sides;
    float* host_eta;
    float* host_r;
    bool* host_isInverted;
    bool* host_isLower;
    bool* host_isAnchor;
    ModuleType* host_moduleType;
    ModuleLayerType* host_moduleLayerType;
    float* host_slopes;
    float* host_drdzs;
    uint16_t* host_partnerModuleIndices;

    host_detIds = (unsigned int*)cms::cuda::allocate_host(sizeof(unsigned int)*nModules, stream);
    host_layers = (short*)cms::cuda::allocate_host(sizeof(short)*nModules, stream);
    host_rings = (short*)cms::cuda::allocate_host(sizeof(short)*nModules, stream);
    host_rods = (short*)cms::cuda::allocate_host(sizeof(short)*nModules, stream);
    host_modules = (short*)cms::cuda::allocate_host(sizeof(short)*nModules, stream);
    host_subdets = (short*)cms::cuda::allocate_host(sizeof(short)*nModules, stream);
    host_sides = (short*)cms::cuda::allocate_host(sizeof(short)*nModules, stream);
    host_eta = (float*)cms::cuda::allocate_host(sizeof(float)*nModules, stream);
    host_r = (float*)cms::cuda::allocate_host(sizeof(float)*nModules, stream);
    host_isInverted = (bool*)cms::cuda::allocate_host(sizeof(bool)*nModules, stream);
    host_isLower = (bool*)cms::cuda::allocate_host(sizeof(bool)*nModules, stream);
    host_isAnchor = (bool*)cms::cuda::allocate_host(sizeof(bool)*nModules, stream);
    host_moduleType = (ModuleType*)cms::cuda::allocate_host(sizeof(ModuleType)*nModules, stream);
    host_moduleLayerType = (ModuleLayerType*)cms::cuda::allocate_host(sizeof(ModuleLayerType)*nModules, stream);
    host_slopes = (float*)cms::cuda::allocate_host(sizeof(float)*nModules, stream);
    host_drdzs = (float*)cms::cuda::allocate_host(sizeof(float)*nModules, stream);
    host_partnerModuleIndices = (uint16_t*)cms::cuda::allocate_host(sizeof(uint16_t) * nModules, stream);
    
    //reassign detIdToIndex indices here
    nLowerModules = (nModules - 1) / 2;
    uint16_t lowerModuleCounter = 0;
    uint16_t upperModuleCounter = nLowerModules + 1;
    //0 to nLowerModules - 1 => only lower modules, nLowerModules - pixel module, nLowerModules + 1 to nModules => upper modules
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); it++)
    {
        unsigned int detId = it->first;
        float m_x = (*module_x)[detId];
        float m_y = (*module_y)[detId];
        float m_z = (*module_z)[detId];
        unsigned int m_t = (*module_type)[detId];

        float eta,r;

        uint16_t index;
        unsigned short layer,ring,rod,module,subdet,side;
        bool isInverted, isLower;
        if(detId == 1)
        {
            layer = 0;
            ring = 0;
            rod = 0;
            module = 0;
            subdet = 0;
            side = 0;
            isInverted = false;
            isLower = false;
        }
        else
        {
            setDerivedQuantities(detId,layer,ring,rod,module,subdet,side,m_x,m_y,m_z,eta,r);
            isInverted = modulesInGPU.parseIsInverted(subdet, side, module, layer);
            isLower = modulesInGPU.parseIsLower(isInverted, detId);
        }
        if(isLower)
        {
            index = lowerModuleCounter;
            lowerModuleCounter++;
        }
        else if(detId != 1)
        {
            index = upperModuleCounter;
            upperModuleCounter++;
        }
        else
        {
            index = nLowerModules; //pixel
        }
        //reassigning indices!
        (*detIdToIndex)[detId] = index;   
        host_detIds[index] = detId;
        host_layers[index] = layer;
        host_rings[index] = ring;
        host_rods[index] = rod;
        host_modules[index] = module;
        host_subdets[index] = subdet;
        host_sides[index] = side;
        host_eta[index] = eta;
        host_r[index] = r;
        host_isInverted[index] = isInverted;
        host_isLower[index] = isLower;

        //assigning other variables!
        if(detId == 1)
        {
            host_moduleType[index] = PixelModule;
            host_moduleLayerType[index] = SDL::InnerPixelLayer;
            host_slopes[index] = 0;
            host_drdzs[index] = 0;
            host_isAnchor[index] = false;
        }
        else
        {
            host_moduleType[index] = ( m_t == 25 ? SDL::TwoS : SDL::PS );
            host_moduleLayerType[index] = ( m_t == 23 ? SDL::Pixel : SDL::Strip );

            if(host_moduleType[index] == SDL::PS and host_moduleLayerType[index] == SDL::Pixel)
            {
                host_isAnchor[index] = true;
            }
            else if(host_moduleType[index] == SDL::TwoS and host_isLower[index])
            {
                host_isAnchor[index] = true;   
            }
            else
            {
                host_isAnchor[index] = false;
            }

            host_slopes[index] = (subdet == Endcap) ? endcapGeometry.getSlopeLower(detId) : tiltedGeometry.getSlope(detId);
            host_drdzs[index] = (subdet == Barrel) ? tiltedGeometry.getDrDz(detId) : 0;
        }
    }

    //partner module stuff, and slopes and drdz move around
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); it++)
    {
        auto& detId = it->first;
        auto& index = it->second;
        if(detId != 1)
        {
            host_partnerModuleIndices[index] = (*detIdToIndex)[modulesInGPU.parsePartnerModuleId(detId, host_isLower[index], host_isInverted[index])];
            //add drdz and slope importing stuff here!
            if(host_drdzs[index] == 0)
            {
                host_drdzs[index] = host_drdzs[host_partnerModuleIndices[index]];
            }
            if(host_slopes[index] == 0)
            {
                host_slopes[index] = host_slopes[host_partnerModuleIndices[index]];
            }
        }
    }

    hipMemcpyAsync(modulesInGPU.nLowerModules,&nLowerModules,sizeof(uint16_t),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.detIds,host_detIds,nModules*sizeof(unsigned int),hipMemcpyHostToDevice,stream);

    hipMemcpyAsync(modulesInGPU.layers,host_layers,nModules*sizeof(short),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.rings,host_rings,sizeof(short)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.rods,host_rods,sizeof(short)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.modules,host_modules,sizeof(short)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.subdets,host_subdets,sizeof(short)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.sides,host_sides,sizeof(short)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.eta,host_eta,sizeof(float)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.r,host_r,sizeof(float)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.isInverted,host_isInverted,sizeof(bool)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.isLower,host_isLower,sizeof(bool)*nModules,hipMemcpyHostToDevice,stream);

    hipMemcpyAsync(modulesInGPU.moduleType,host_moduleType,sizeof(ModuleType)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.moduleLayerType,host_moduleLayerType,sizeof(ModuleLayerType)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.slopes,host_slopes,sizeof(float)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.isAnchor, host_isAnchor, sizeof(bool) * nModules, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(modulesInGPU.drdzs,host_drdzs,sizeof(float)*nModules,hipMemcpyHostToDevice,stream);

    hipMemcpyAsync(modulesInGPU.partnerModuleIndices, host_partnerModuleIndices, sizeof(uint16_t) * nModules, hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);

    cms::cuda::free_host(host_detIds);
    cms::cuda::free_host(host_layers);
    cms::cuda::free_host(host_rings);
    cms::cuda::free_host(host_rods);
    cms::cuda::free_host(host_modules);
    cms::cuda::free_host(host_subdets);
    cms::cuda::free_host(host_sides);
    cms::cuda::free_host(host_eta);
    cms::cuda::free_host(host_r);
    cms::cuda::free_host(host_isInverted);
    cms::cuda::free_host(host_isLower);
    cms::cuda::free_host(host_isAnchor);
    cms::cuda::free_host(host_moduleType);
    cms::cuda::free_host(host_moduleLayerType);
    cms::cuda::free_host(host_slopes);
    cms::cuda::free_host(host_drdzs);
    cms::cuda::free_host(host_partnerModuleIndices);

    fillConnectedModuleArrayExplicit(modulesInGPU,nModules,stream);
    fillMapArraysExplicit(modulesInGPU, nModules, stream);
    fillPixelMap(modulesInGPU,pixelMapping,stream);
}

void SDL::fillConnectedModuleArray(struct modules& modulesInGPU, unsigned int nModules)
{
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); ++it)
    {
        unsigned int detId = it->first;
        uint16_t index = it->second;
        auto& connectedModules = moduleConnectionMap.getConnectedModuleDetIds(detId);
        modulesInGPU.nConnectedModules[index] = connectedModules.size();
        for(uint16_t i = 0; i< modulesInGPU.nConnectedModules[index];i++)
        {
            modulesInGPU.moduleMap[index * 40 + i] = (*detIdToIndex)[connectedModules[i]];
        }
    }
}

void SDL::fillPixelMap(struct modules& modulesInGPU, struct pixelMap& pixelMapping,hipStream_t stream)
{
    int size_superbins = 45000; //changed to 45000 to reduce memory useage on GPU
    std::vector<unsigned int> connectedModuleDetIds;
    std::vector<unsigned int> connectedModuleDetIds_pos;
    std::vector<unsigned int> connectedModuleDetIds_neg;
    hipHostMalloc(&pixelMapping.connectedPixelsIndex,size_superbins * sizeof(unsigned int));
    hipHostMalloc(&pixelMapping.connectedPixelsSizes,size_superbins * sizeof(unsigned int));
    hipHostMalloc(&pixelMapping.connectedPixelsIndexPos,size_superbins * sizeof(unsigned int));
    hipHostMalloc(&pixelMapping.connectedPixelsSizesPos,size_superbins * sizeof(unsigned int));
    hipHostMalloc(&pixelMapping.connectedPixelsIndexNeg,size_superbins * sizeof(unsigned int));
    hipHostMalloc(&pixelMapping.connectedPixelsSizesNeg,size_superbins * sizeof(unsigned int));

    int totalSizes=0;
    int totalSizes_pos=0;
    int totalSizes_neg=0;
    for(int isuperbin =0; isuperbin<size_superbins; isuperbin++)
    {
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet5 = SDL::moduleConnectionMap_pLStoLayer1Subdet5.getConnectedModuleDetIds(isuperbin+size_superbins);// index adjustment to get high values
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet5 = SDL::moduleConnectionMap_pLStoLayer2Subdet5.getConnectedModuleDetIds(isuperbin+size_superbins);// from the high pt bins
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet5 = SDL::moduleConnectionMap_pLStoLayer3Subdet5.getConnectedModuleDetIds(isuperbin+size_superbins);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet4 = SDL::moduleConnectionMap_pLStoLayer1Subdet4.getConnectedModuleDetIds(isuperbin+size_superbins);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet4 = SDL::moduleConnectionMap_pLStoLayer2Subdet4.getConnectedModuleDetIds(isuperbin+size_superbins);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet4 = SDL::moduleConnectionMap_pLStoLayer3Subdet4.getConnectedModuleDetIds(isuperbin+size_superbins);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer4Subdet4 = SDL::moduleConnectionMap_pLStoLayer4Subdet4.getConnectedModuleDetIds(isuperbin+size_superbins);
        connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer1Subdet5.begin(),connectedModuleDetIds_pLStoLayer1Subdet5.end());
        connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer2Subdet5.begin(),connectedModuleDetIds_pLStoLayer2Subdet5.end());
        connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer3Subdet5.begin(),connectedModuleDetIds_pLStoLayer3Subdet5.end());
        connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer1Subdet4.begin(),connectedModuleDetIds_pLStoLayer1Subdet4.end());
        connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer2Subdet4.begin(),connectedModuleDetIds_pLStoLayer2Subdet4.end());
        connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer3Subdet4.begin(),connectedModuleDetIds_pLStoLayer3Subdet4.end());
        connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer4Subdet4.begin(),connectedModuleDetIds_pLStoLayer4Subdet4.end());

        int sizes =0;
        sizes += connectedModuleDetIds_pLStoLayer1Subdet5.size();
        sizes += connectedModuleDetIds_pLStoLayer2Subdet5.size();
        sizes += connectedModuleDetIds_pLStoLayer3Subdet5.size();
        sizes += connectedModuleDetIds_pLStoLayer1Subdet4.size();
        sizes += connectedModuleDetIds_pLStoLayer2Subdet4.size();
        sizes += connectedModuleDetIds_pLStoLayer3Subdet4.size();
        sizes += connectedModuleDetIds_pLStoLayer4Subdet4.size();
        pixelMapping.connectedPixelsIndex[isuperbin] = totalSizes;
        pixelMapping.connectedPixelsSizes[isuperbin] = sizes;
        totalSizes += sizes;

        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet5_pos = SDL::moduleConnectionMap_pLStoLayer1Subdet5_pos.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet5_pos = SDL::moduleConnectionMap_pLStoLayer2Subdet5_pos.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet5_pos = SDL::moduleConnectionMap_pLStoLayer3Subdet5_pos.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet4_pos = SDL::moduleConnectionMap_pLStoLayer1Subdet4_pos.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet4_pos = SDL::moduleConnectionMap_pLStoLayer2Subdet4_pos.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet4_pos = SDL::moduleConnectionMap_pLStoLayer3Subdet4_pos.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer4Subdet4_pos = SDL::moduleConnectionMap_pLStoLayer4Subdet4_pos.getConnectedModuleDetIds(isuperbin);
        connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer1Subdet5_pos.begin(),connectedModuleDetIds_pLStoLayer1Subdet5_pos.end());
        connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer2Subdet5_pos.begin(),connectedModuleDetIds_pLStoLayer2Subdet5_pos.end());
        connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer3Subdet5_pos.begin(),connectedModuleDetIds_pLStoLayer3Subdet5_pos.end());
        connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer1Subdet4_pos.begin(),connectedModuleDetIds_pLStoLayer1Subdet4_pos.end());
        connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer2Subdet4_pos.begin(),connectedModuleDetIds_pLStoLayer2Subdet4_pos.end());
        connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer3Subdet4_pos.begin(),connectedModuleDetIds_pLStoLayer3Subdet4_pos.end());
        connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer4Subdet4_pos.begin(),connectedModuleDetIds_pLStoLayer4Subdet4_pos.end());

        int sizes_pos =0;
        sizes_pos += connectedModuleDetIds_pLStoLayer1Subdet5_pos.size();
        sizes_pos += connectedModuleDetIds_pLStoLayer2Subdet5_pos.size();
        sizes_pos += connectedModuleDetIds_pLStoLayer3Subdet5_pos.size();
        sizes_pos += connectedModuleDetIds_pLStoLayer1Subdet4_pos.size();
        sizes_pos += connectedModuleDetIds_pLStoLayer2Subdet4_pos.size();
        sizes_pos += connectedModuleDetIds_pLStoLayer3Subdet4_pos.size();
        sizes_pos += connectedModuleDetIds_pLStoLayer4Subdet4_pos.size();
        pixelMapping.connectedPixelsIndexPos[isuperbin] = totalSizes_pos;
        pixelMapping.connectedPixelsSizesPos[isuperbin] = sizes_pos;
        totalSizes_pos += sizes_pos;

        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet5_neg = SDL::moduleConnectionMap_pLStoLayer1Subdet5_neg.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet5_neg = SDL::moduleConnectionMap_pLStoLayer2Subdet5_neg.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet5_neg = SDL::moduleConnectionMap_pLStoLayer3Subdet5_neg.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet4_neg = SDL::moduleConnectionMap_pLStoLayer1Subdet4_neg.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet4_neg = SDL::moduleConnectionMap_pLStoLayer2Subdet4_neg.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet4_neg = SDL::moduleConnectionMap_pLStoLayer3Subdet4_neg.getConnectedModuleDetIds(isuperbin);
        std::vector<unsigned int> connectedModuleDetIds_pLStoLayer4Subdet4_neg = SDL::moduleConnectionMap_pLStoLayer4Subdet4_neg.getConnectedModuleDetIds(isuperbin);
        connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer1Subdet5_neg.begin(),connectedModuleDetIds_pLStoLayer1Subdet5_neg.end());
        connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer2Subdet5_neg.begin(),connectedModuleDetIds_pLStoLayer2Subdet5_neg.end());
        connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer3Subdet5_neg.begin(),connectedModuleDetIds_pLStoLayer3Subdet5_neg.end());
        connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer1Subdet4_neg.begin(),connectedModuleDetIds_pLStoLayer1Subdet4_neg.end());
        connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer2Subdet4_neg.begin(),connectedModuleDetIds_pLStoLayer2Subdet4_neg.end());
        connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer3Subdet4_neg.begin(),connectedModuleDetIds_pLStoLayer3Subdet4_neg.end());
        connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer4Subdet4_neg.begin(),connectedModuleDetIds_pLStoLayer4Subdet4_neg.end());

        int sizes_neg = 0;
        sizes_neg += connectedModuleDetIds_pLStoLayer1Subdet5_neg.size();
        sizes_neg += connectedModuleDetIds_pLStoLayer2Subdet5_neg.size();
        sizes_neg += connectedModuleDetIds_pLStoLayer3Subdet5_neg.size();
        sizes_neg += connectedModuleDetIds_pLStoLayer1Subdet4_neg.size();
        sizes_neg += connectedModuleDetIds_pLStoLayer2Subdet4_neg.size();
        sizes_neg += connectedModuleDetIds_pLStoLayer3Subdet4_neg.size();
        sizes_neg += connectedModuleDetIds_pLStoLayer4Subdet4_neg.size();
        pixelMapping.connectedPixelsIndexNeg[isuperbin] = totalSizes_neg;
        pixelMapping.connectedPixelsSizesNeg[isuperbin] = sizes_neg;
        totalSizes_neg += sizes_neg;
    }

    unsigned int* connectedPixels;
    connectedPixels = (unsigned int*)cms::cuda::allocate_host((totalSizes+totalSizes_pos+totalSizes_neg) * sizeof(unsigned int), stream);
    hipMalloc(&modulesInGPU.connectedPixels,(totalSizes+totalSizes_pos+totalSizes_neg)* sizeof(unsigned int));

    for(int icondet=0; icondet< totalSizes; icondet++)
    {
        connectedPixels[icondet] = (*detIdToIndex)[connectedModuleDetIds[icondet]];
    }
    for(int icondet=0; icondet< totalSizes_pos; icondet++)
    {
        connectedPixels[icondet+totalSizes] = (*detIdToIndex)[connectedModuleDetIds_pos[icondet]];
    }
    for(int icondet=0; icondet< totalSizes_neg; icondet++)
    {
        connectedPixels[icondet+totalSizes+totalSizes_pos] = (*detIdToIndex)[connectedModuleDetIds_neg[icondet]];
    }
    hipMemcpyAsync(modulesInGPU.connectedPixels,connectedPixels,(totalSizes+totalSizes_pos+totalSizes_neg)*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);

    cms::cuda::free_host(connectedPixels);
}

void SDL::fillConnectedModuleArrayExplicit(struct modules& modulesInGPU, unsigned int nModules,hipStream_t stream)
{
    uint16_t* moduleMap;
    uint16_t* nConnectedModules;
    moduleMap = (uint16_t*)cms::cuda::allocate_host(nModules * 40 * sizeof(uint16_t), stream);
    nConnectedModules = (uint16_t*)cms::cuda::allocate_host(nModules * sizeof(uint16_t), stream);
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); ++it)
    {
        unsigned int detId = it->first;
        uint16_t index = it->second;
        auto& connectedModules = moduleConnectionMap.getConnectedModuleDetIds(detId);
        nConnectedModules[index] = connectedModules.size();
        for(uint16_t i = 0; i< nConnectedModules[index];i++)
        {
            moduleMap[index * 40 + i] = (*detIdToIndex)[connectedModules[i]];
        }
    }
    hipMemcpyAsync(modulesInGPU.moduleMap,moduleMap,nModules*40*sizeof(uint16_t),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.nConnectedModules,nConnectedModules,nModules*sizeof(uint16_t),hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);
    cms::cuda::free_host(moduleMap);
    cms::cuda::free_host(nConnectedModules);
}

void SDL::fillMapArraysExplicit(struct modules& modulesInGPU, unsigned int nModules,hipStream_t stream)
{
    uint16_t* mapIdx;
    unsigned int* mapdetId;
    unsigned int counter = 0;
    mapIdx = (uint16_t*)cms::cuda::allocate_host(nModules * sizeof(uint16_t), stream);
    mapdetId = (unsigned int*)cms::cuda::allocate_host(nModules * sizeof(unsigned int), stream);
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); ++it)
    {
        unsigned int detId = it->first;
        unsigned int index = it->second;
        mapIdx[counter] = index;
        mapdetId[counter] = detId;
        counter++;
    }
    hipMemcpyAsync(modulesInGPU.mapIdx,mapIdx,nModules*sizeof(uint16_t),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.mapdetId,mapdetId,nModules*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);
    cms::cuda::free_host(mapIdx);
    cms::cuda::free_host(mapdetId);
}

void SDL::setDerivedQuantities(unsigned int detId, unsigned short& layer, unsigned short& ring, unsigned short& rod, unsigned short& module, unsigned short& subdet, unsigned short& side, float m_x, float m_y, float m_z, float& eta, float& r)
{
    subdet = (detId & (7 << 25)) >> 25;
    side = (subdet == Endcap) ? (detId & (3 << 23)) >> 23 : (detId & (3 << 18)) >> 18;
    layer = (subdet == Endcap) ? (detId & (7 << 18)) >> 18 : (detId & (7 << 20)) >> 20;
    ring = (subdet == Endcap) ? (detId & (15 << 12)) >> 12 : 0;
    module = (detId & (127 << 2)) >> 2;
    rod = (subdet == Endcap) ? 0 : (detId & (127 << 10)) >> 10;

    r = std::sqrt(m_x * m_x + m_y * m_y + m_z * m_z);
    eta = ((m_z > 0) - ( m_z < 0)) * std::acosh(r / std::sqrt(m_x * m_x + m_y * m_y));
}

bool SDL::modules::parseIsInverted(short subdet, short side, short module, short layer)
{
    if (subdet == Endcap)
    {
        if (side == NegZ)
        {
            return module % 2 == 1;
        }
        else if (side == PosZ)
        {
            return module % 2 == 0;
        }
        else
        {
            return 0;
        }
    }
    else if (subdet == Barrel)
    {
        if (side == Center)
        {
            if (layer <= 3)
            {
                return module % 2 == 1;
            }
            else if (layer >= 4)
            {
                return module % 2 == 0;
            }
            else
            {
                return 0;
            }
        }
        else if (side == NegZ or side == PosZ)
        {
            if (layer <= 2)
            {
                return module % 2 == 1;
            }
            else if (layer == 3)
            {
                return module % 2 == 0;
            }
            else
            {
                return 0;
            }
        }
        else
        {
            return 0;
        }
    }
    else
    {
        return 0;
    }
}

bool SDL::modules::parseIsLower(bool isInvertedx, unsigned int detId)
{
    return (isInvertedx) ? !(detId & 1) : (detId & 1);
}

unsigned int SDL::modules::parsePartnerModuleId(unsigned int detId, bool isLowerx, bool isInvertedx)
{
    return isLowerx ? (isInvertedx ? detId - 1 : detId + 1) : (isInvertedx ? detId + 1 : detId - 1);
}

void SDL::resetObjectRanges(struct objectRanges& rangesInGPU, unsigned int nModules,hipStream_t stream)
{
    hipMemsetAsync(rangesInGPU.hitRanges, -1,nModules*2*sizeof(int),stream);
    hipMemsetAsync(rangesInGPU.hitRangesLower, -1,nModules*sizeof(int),stream);
    hipMemsetAsync(rangesInGPU.hitRangesUpper, -1,nModules*sizeof(int),stream);
    hipMemsetAsync(rangesInGPU.hitRangesnLower, -1,nModules*sizeof(int8_t),stream);
    hipMemsetAsync(rangesInGPU.hitRangesnUpper, -1,nModules*sizeof(int8_t),stream);
    hipMemsetAsync(rangesInGPU.mdRanges, -1,nModules*2*sizeof(int),stream);
    hipMemsetAsync(rangesInGPU.segmentRanges, -1,nModules*2*sizeof(int),stream);
    hipMemsetAsync(rangesInGPU.trackletRanges, -1,nModules*2*sizeof(int),stream);
    hipMemsetAsync(rangesInGPU.tripletRanges, -1,nModules*2*sizeof(int),stream);
    hipMemsetAsync(rangesInGPU.trackCandidateRanges, -1,nModules*2*sizeof(int),stream);
    hipMemsetAsync(rangesInGPU.quintupletRanges, -1, nModules*2*sizeof(int),stream);
    hipStreamSynchronize(stream);
}
