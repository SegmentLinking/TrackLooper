# include "Module.cuh"
#include "allocate.h"
std::map <unsigned int, unsigned int> *SDL::detIdToIndex;

void SDL::createModulesInUnifiedMemory(struct modules& modulesInGPU,unsigned int nModules)
{
    /* modules stucture object will be created in Event.cu*/
#ifdef CACHE_ALLOC
    hipStream_t stream=0; 
    modulesInGPU.detIds =            (unsigned int*)cms::cuda::allocate_managed(nModules * sizeof(unsigned int),stream);
    modulesInGPU.moduleMap =         (unsigned int*)cms::cuda::allocate_managed(nModules * 40 * sizeof(unsigned int),stream);
    modulesInGPU.nConnectedModules = (unsigned int*)cms::cuda::allocate_managed(nModules * sizeof(unsigned int),stream);
    modulesInGPU.drdzs =                    (float*)cms::cuda::allocate_managed(nModules * sizeof(float),stream);
    modulesInGPU.slopes =                   (float*)cms::cuda::allocate_managed(nModules * sizeof(float),stream);
    modulesInGPU.nModules =          (unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    modulesInGPU.nLowerModules =     (unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    modulesInGPU.layers =                   (short*)cms::cuda::allocate_managed(nModules * sizeof(short),stream);
    modulesInGPU.rings =                    (short*)cms::cuda::allocate_managed(nModules * sizeof(short),stream);
    modulesInGPU.modules =                  (short*)cms::cuda::allocate_managed(nModules * sizeof(short),stream);
    modulesInGPU.rods =                    (short*)cms::cuda::allocate_managed(nModules * sizeof(short),stream);
    modulesInGPU.subdets =                 (short*)cms::cuda::allocate_managed(nModules * sizeof(short),stream);
    modulesInGPU.sides =                   (short*)cms::cuda::allocate_managed(nModules * sizeof(short),stream);
    modulesInGPU.isInverted =               (bool*)cms::cuda::allocate_managed(nModules * sizeof(bool),stream);
    modulesInGPU.isLower =                  (bool*)cms::cuda::allocate_managed(nModules * sizeof(bool),stream);
    modulesInGPU.nEligibleModules =     (unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);

    modulesInGPU.hitRanges =                 (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    modulesInGPU.mdRanges =                  (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    modulesInGPU.segmentRanges =             (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    modulesInGPU.trackletRanges =            (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    modulesInGPU.tripletRanges =             (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    modulesInGPU.trackCandidateRanges =      (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);

    modulesInGPU.moduleType =         (ModuleType*)cms::cuda::allocate_managed(nModules * sizeof(ModuleType),stream);
    modulesInGPU.moduleLayerType=(ModuleLayerType*)cms::cuda::allocate_managed(nModules * sizeof(ModuleLayerType),stream);
#else
    hipMallocManaged(&modulesInGPU.detIds,nModules * sizeof(unsigned int));
    hipMallocManaged(&modulesInGPU.moduleMap,nModules * 40 * sizeof(unsigned int));
    hipMallocManaged(&modulesInGPU.nConnectedModules,nModules * sizeof(unsigned int));
    hipMallocManaged(&modulesInGPU.drdzs,nModules * sizeof(float));
    hipMallocManaged(&modulesInGPU.slopes,nModules * sizeof(float));
    hipMallocManaged(&modulesInGPU.nModules,sizeof(unsigned int));
    hipMallocManaged(&modulesInGPU.nLowerModules,sizeof(unsigned int));
    hipMallocManaged(&modulesInGPU.layers,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.rings,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.modules,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.rods,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.subdets,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.sides,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.isInverted, nModules * sizeof(bool));
    hipMallocManaged(&modulesInGPU.isLower, nModules * sizeof(bool));
    hipMallocManaged(&modulesInGPU.nEligibleModules,sizeof(unsigned int));

    hipMallocManaged(&modulesInGPU.hitRanges,nModules * 2 * sizeof(int));
    hipMallocManaged(&modulesInGPU.mdRanges,nModules * 2 * sizeof(int));
    hipMallocManaged(&modulesInGPU.segmentRanges,nModules * 2 * sizeof(int));
    hipMallocManaged(&modulesInGPU.trackletRanges,nModules * 2 * sizeof(int));
    hipMallocManaged(&modulesInGPU.tripletRanges,nModules * 2 * sizeof(int));
    hipMallocManaged(&modulesInGPU.trackCandidateRanges, nModules * 2 * sizeof(int));

    hipMallocManaged(&modulesInGPU.moduleType,nModules * sizeof(ModuleType));
    hipMallocManaged(&modulesInGPU.moduleLayerType,nModules * sizeof(ModuleLayerType));
#endif


    *modulesInGPU.nModules = nModules;
}
void SDL::createModulesInExplicitMemory(struct modules& modulesInGPU,unsigned int nModules)
{
    /* modules stucture object will be created in Event.cu*/
#ifdef CACHE_ALLOC
    hipStream_t stream=0; 
    int dev;
    hipGetDevice(&dev);
    modulesInGPU.detIds =            (unsigned int*)cms::cuda::allocate_device(dev,nModules * sizeof(unsigned int),stream);
    modulesInGPU.moduleMap =         (unsigned int*)cms::cuda::allocate_device(dev,nModules * 40 * sizeof(unsigned int),stream);
    modulesInGPU.nConnectedModules = (unsigned int*)cms::cuda::allocate_device(dev,nModules * sizeof(unsigned int),stream);
    modulesInGPU.drdzs =                    (float*)cms::cuda::allocate_device(dev,nModules * sizeof(float),stream);
    modulesInGPU.slopes =                   (float*)cms::cuda::allocate_device(dev,nModules * sizeof(float),stream);
    modulesInGPU.nModules =          (unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    modulesInGPU.nLowerModules =     (unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    modulesInGPU.layers =                   (short*)cms::cuda::allocate_device(dev,nModules * sizeof(short),stream);
    modulesInGPU.rings =                    (short*)cms::cuda::allocate_device(dev,nModules * sizeof(short),stream);
    modulesInGPU.modules =                  (short*)cms::cuda::allocate_device(dev,nModules * sizeof(short),stream);
    modulesInGPU.rods =                     (short*)cms::cuda::allocate_device(dev,nModules * sizeof(short),stream);
    modulesInGPU.subdets =                  (short*)cms::cuda::allocate_device(dev,nModules * sizeof(short),stream);
    modulesInGPU.sides =                    (short*)cms::cuda::allocate_device(dev,nModules * sizeof(short),stream);
    modulesInGPU.isInverted =                (bool*)cms::cuda::allocate_device(dev,nModules * sizeof(bool),stream);
    modulesInGPU.isLower =                   (bool*)cms::cuda::allocate_device(dev,nModules * sizeof(bool),stream);
    modulesInGPU.nEligibleModules =     (unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);

    modulesInGPU.hitRanges =                  (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    modulesInGPU.mdRanges =                   (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    modulesInGPU.segmentRanges =              (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    modulesInGPU.trackletRanges =             (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    modulesInGPU.tripletRanges =              (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    modulesInGPU.trackCandidateRanges =       (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);

    modulesInGPU.moduleType =          (ModuleType*)cms::cuda::allocate_device(dev,nModules * sizeof(ModuleType),stream);
    modulesInGPU.moduleLayerType= (ModuleLayerType*)cms::cuda::allocate_device(dev,nModules * sizeof(ModuleLayerType),stream);
#else
    hipMalloc(&(modulesInGPU.detIds),nModules * sizeof(unsigned int));
    hipMalloc(&modulesInGPU.moduleMap,nModules * 40 * sizeof(unsigned int));
    hipMalloc(&modulesInGPU.nConnectedModules,nModules * sizeof(unsigned int));
    hipMalloc(&modulesInGPU.drdzs,nModules * sizeof(float));
    hipMalloc(&modulesInGPU.slopes,nModules * sizeof(float));
    hipMalloc(&modulesInGPU.nModules,sizeof(unsigned int));
    hipMalloc(&modulesInGPU.nLowerModules,sizeof(unsigned int));
    hipMalloc(&modulesInGPU.layers,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.rings,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.modules,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.rods,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.subdets,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.sides,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.isInverted, nModules * sizeof(bool));
    hipMalloc(&modulesInGPU.isLower, nModules * sizeof(bool));
    hipMalloc(&modulesInGPU.nEligibleModules,sizeof(unsigned int));

    hipMalloc(&modulesInGPU.hitRanges,nModules * 2 * sizeof(int));
    hipMalloc(&modulesInGPU.mdRanges,nModules * 2 * sizeof(int));
    hipMalloc(&modulesInGPU.segmentRanges,nModules * 2 * sizeof(int));
    hipMalloc(&modulesInGPU.trackletRanges,nModules * 2 * sizeof(int));
    hipMalloc(&modulesInGPU.tripletRanges,nModules * 2 * sizeof(int));
    hipMalloc(&modulesInGPU.trackCandidateRanges, nModules * 2 * sizeof(int));

    hipMalloc(&modulesInGPU.moduleType,nModules * sizeof(ModuleType));
    hipMalloc(&modulesInGPU.moduleLayerType,nModules * sizeof(ModuleLayerType));
#endif

    hipMemcpy(modulesInGPU.nModules,&nModules,sizeof(unsigned int),hipMemcpyHostToDevice);
}

void SDL::freeModulesCache(struct modules& modulesInGPU)
{
#ifdef Explicit_Module
  int dev;
  hipGetDevice(&dev);
  cms::cuda::free_device(dev,modulesInGPU.detIds);
  cms::cuda::free_device(dev,modulesInGPU.moduleMap);
  cms::cuda::free_device(dev,modulesInGPU.nConnectedModules);
  cms::cuda::free_device(dev,modulesInGPU.drdzs);
  cms::cuda::free_device(dev,modulesInGPU.slopes);
  cms::cuda::free_device(dev,modulesInGPU.nModules);
  cms::cuda::free_device(dev,modulesInGPU.nLowerModules);
  cms::cuda::free_device(dev,modulesInGPU.layers);
  cms::cuda::free_device(dev,modulesInGPU.rings);
  cms::cuda::free_device(dev,modulesInGPU.modules);
  cms::cuda::free_device(dev,modulesInGPU.rods);
  cms::cuda::free_device(dev,modulesInGPU.subdets);
  cms::cuda::free_device(dev,modulesInGPU.sides);
  cms::cuda::free_device(dev,modulesInGPU.isInverted);
  cms::cuda::free_device(dev,modulesInGPU.isLower);
  cms::cuda::free_device(dev,modulesInGPU.hitRanges);
  cms::cuda::free_device(dev,modulesInGPU.mdRanges);
  cms::cuda::free_device(dev,modulesInGPU.segmentRanges);
  cms::cuda::free_device(dev,modulesInGPU.trackletRanges);
  cms::cuda::free_device(dev,modulesInGPU.tripletRanges);
  cms::cuda::free_device(dev,modulesInGPU.trackCandidateRanges);
  cms::cuda::free_device(dev,modulesInGPU.moduleType);
  cms::cuda::free_device(dev,modulesInGPU.moduleLayerType);
  cms::cuda::free_device(dev,modulesInGPU.lowerModuleIndices);
  cms::cuda::free_device(dev,modulesInGPU.reverseLookupLowerModuleIndices);
  cms::cuda::free_device(dev,modulesInGPU.trackCandidateModuleIndices);
  cms::cuda::free_device(dev,modulesInGPU.nEligibleModules);
#else
  cms::cuda::free_managed(modulesInGPU.detIds);
  cms::cuda::free_managed(modulesInGPU.moduleMap);
  cms::cuda::free_managed(modulesInGPU.nConnectedModules);
  cms::cuda::free_managed(modulesInGPU.drdzs);
  cms::cuda::free_managed(modulesInGPU.slopes);
  cms::cuda::free_managed(modulesInGPU.nModules);
  cms::cuda::free_managed(modulesInGPU.nLowerModules);
  cms::cuda::free_managed(modulesInGPU.layers);
  cms::cuda::free_managed(modulesInGPU.rings);
  cms::cuda::free_managed(modulesInGPU.modules);
  cms::cuda::free_managed(modulesInGPU.rods);
  cms::cuda::free_managed(modulesInGPU.subdets);
  cms::cuda::free_managed(modulesInGPU.sides);
  cms::cuda::free_managed(modulesInGPU.isInverted);
  cms::cuda::free_managed(modulesInGPU.isLower);
  cms::cuda::free_managed(modulesInGPU.hitRanges);
  cms::cuda::free_managed(modulesInGPU.mdRanges);
  cms::cuda::free_managed(modulesInGPU.segmentRanges);
  cms::cuda::free_managed(modulesInGPU.trackletRanges);
  cms::cuda::free_managed(modulesInGPU.tripletRanges);
  cms::cuda::free_managed(modulesInGPU.trackCandidateRanges);
  cms::cuda::free_managed(modulesInGPU.moduleType);
  cms::cuda::free_managed(modulesInGPU.moduleLayerType);
  cms::cuda::free_managed(modulesInGPU.lowerModuleIndices);
  cms::cuda::free_managed(modulesInGPU.reverseLookupLowerModuleIndices);
  cms::cuda::free_managed(modulesInGPU.trackCandidateModuleIndices);
  cms::cuda::free_managed(modulesInGPU.nEligibleModules);
#endif
}
void SDL::freeModules(struct modules& modulesInGPU)
{
  hipFree(modulesInGPU.detIds);
  hipFree(modulesInGPU.moduleMap);
  hipFree(modulesInGPU.nConnectedModules);
  hipFree(modulesInGPU.drdzs);
  hipFree(modulesInGPU.slopes);
  hipFree(modulesInGPU.nModules);
  hipFree(modulesInGPU.nLowerModules);
  hipFree(modulesInGPU.layers);
  hipFree(modulesInGPU.rings);
  hipFree(modulesInGPU.modules);
  hipFree(modulesInGPU.rods);
  hipFree(modulesInGPU.subdets);
  hipFree(modulesInGPU.sides);
  hipFree(modulesInGPU.isInverted);
  hipFree(modulesInGPU.isLower);
  hipFree(modulesInGPU.hitRanges);
  hipFree(modulesInGPU.mdRanges);
  hipFree(modulesInGPU.segmentRanges);
  hipFree(modulesInGPU.trackletRanges);
  hipFree(modulesInGPU.tripletRanges);
  hipFree(modulesInGPU.trackCandidateRanges);
  hipFree(modulesInGPU.moduleType);
  hipFree(modulesInGPU.moduleLayerType);
  hipFree(modulesInGPU.lowerModuleIndices);
  hipFree(modulesInGPU.reverseLookupLowerModuleIndices);
  hipFree(modulesInGPU.trackCandidateModuleIndices);
  hipFree(modulesInGPU.nEligibleModules);
}

void SDL::createLowerModuleIndexMapExplicit(struct modules& modulesInGPU, unsigned int nLowerModules, unsigned int nModules,bool* isLower)
{
    //FIXME:some hacks to get the pixel module in the lower modules index without incrementing nLowerModules counter!
    //Reproduce these hacks in the explicit memory for identical results (or come up with a better method)
    unsigned int* lowerModuleIndices;
    int* reverseLookupLowerModuleIndices;
    hipHostMalloc(&lowerModuleIndices,(nLowerModules + 1) * sizeof(unsigned int));
    hipHostMalloc(&reverseLookupLowerModuleIndices,nModules * sizeof(int));

    unsigned int lowerModuleCounter = 0;
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); it++)
    {
        unsigned int index = it->second;
        unsigned int detId = it->first;
        if(isLower[index])
        {
            lowerModuleIndices[lowerModuleCounter] = index;
            reverseLookupLowerModuleIndices[index] = lowerModuleCounter;
            lowerModuleCounter++;
        }
        else
        {
           reverseLookupLowerModuleIndices[index] = -1;
        }
    }
    //hacky stuff "beyond the index" for the pixel module. nLowerModules will *NOT* cover the pixel module!
    lowerModuleIndices[nLowerModules] = (*detIdToIndex)[1];
    reverseLookupLowerModuleIndices[(*detIdToIndex)[1]] = nLowerModules;
    #ifdef CACHE_ALLOC
    hipStream_t stream =0;
    int dev;
    hipGetDevice(&dev);
    modulesInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_device(dev,(nLowerModules + 1) * sizeof(unsigned int),stream);
    modulesInGPU.reverseLookupLowerModuleIndices = (int*)cms::cuda::allocate_device(dev,nModules * sizeof(int),stream);
    modulesInGPU.trackCandidateModuleIndices = (int*)cms::cuda::allocate_device(dev,(nLowerModules + 1) * sizeof(int),stream);
    #else
    hipMalloc(&modulesInGPU.lowerModuleIndices,(nLowerModules + 1) * sizeof(unsigned int));
    hipMalloc(&modulesInGPU.reverseLookupLowerModuleIndices,nModules * sizeof(int));
    hipMalloc(&modulesInGPU.trackCandidateModuleIndices, (nLowerModules + 1) * sizeof(int));
    #endif
    hipMemcpy(modulesInGPU.lowerModuleIndices,lowerModuleIndices,sizeof(unsigned int)*(nLowerModules+1),hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.reverseLookupLowerModuleIndices,reverseLookupLowerModuleIndices,sizeof(int)*nModules,hipMemcpyHostToDevice);
   
    hipHostFree(lowerModuleIndices);
    hipHostFree(reverseLookupLowerModuleIndices);
}
void SDL::createLowerModuleIndexMap(struct modules& modulesInGPU, unsigned int nLowerModules, unsigned int nModules)
{
    //FIXME:some hacks to get the pixel module in the lower modules index without incrementing nLowerModules counter!
    //Reproduce these hacks in the explicit memory for identical results (or come up with a better method)
    #ifdef CACHE_ALLOC
    hipStream_t stream =0;
    modulesInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_managed((nLowerModules + 1) * sizeof(unsigned int),stream);
    modulesInGPU.reverseLookupLowerModuleIndices = (int*)cms::cuda::allocate_managed(nModules * sizeof(int),stream);
    modulesInGPU.trackCandidateModuleIndices = (int*)cms::cuda::allocate_managed((nLowerModules + 1) * sizeof(int),stream);
    #else
    hipMallocManaged(&modulesInGPU.lowerModuleIndices,(nLowerModules + 1) * sizeof(unsigned int));
    hipMallocManaged(&modulesInGPU.reverseLookupLowerModuleIndices,nModules * sizeof(int));
    hipMallocManaged(&modulesInGPU.trackCandidateModuleIndices, (nLowerModules + 1) * sizeof(int));
    #endif



    unsigned int lowerModuleCounter = 0;
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); it++)
    {
        unsigned int index = it->second;
        unsigned int detId = it->first;
        if(modulesInGPU.isLower[index])
        {
            modulesInGPU.lowerModuleIndices[lowerModuleCounter] = index;
            modulesInGPU.reverseLookupLowerModuleIndices[index] = lowerModuleCounter;
            lowerModuleCounter++;
        }
        else
        {
            modulesInGPU.reverseLookupLowerModuleIndices[index] = -1;
        }
    }
    //hacky stuff "beyond the index" for the pixel module. nLowerModules will *NOT* cover the pixel module!
    modulesInGPU.lowerModuleIndices[nLowerModules] = (*detIdToIndex)[1];
    modulesInGPU.reverseLookupLowerModuleIndices[(*detIdToIndex)[1]] = nLowerModules;

}

void SDL::loadModulesFromFile(struct modules& modulesInGPU, unsigned int& nModules, const char* moduleMetaDataFilePath)
{
    detIdToIndex = new std::map<unsigned int, unsigned int>;

    /*modules structure object will be created in Event.cu*/
    /* Load the whole text file into the unordered_map first*/

    std::ifstream ifile;
    ifile.open(moduleMetaDataFilePath);
    if(!ifile.is_open())
    {
        std::cout<<"ERROR! module list file not present!"<<std::endl;
    }
    std::string line;
    unsigned int counter = 0;
    
    while(std::getline(ifile,line))
    {
        std::stringstream ss(line);
        std::string token;
        bool flag = 0;

        while(std::getline(ss,token,','))
        {
            if(flag == 1) break;
            (*detIdToIndex)[stoi(token)] = counter;
            flag = 1;
            counter++;
        }
    }
    //FIXME:MANUAL INSERTION OF PIXEL MODULE!
    (*detIdToIndex)[1] = counter; //pixel module is the last module in the module list
    counter++;
    nModules = counter;
    std::cout<<"Number of modules = "<<nModules<<std::endl;
#ifdef Explicit_Module
    createModulesInExplicitMemory(modulesInGPU,nModules);
    unsigned int* lowerModuleCounter;// = 0;
    hipHostMalloc(&lowerModuleCounter,sizeof(unsigned int));
    hipMemset(lowerModuleCounter,0,sizeof(unsigned int));
    unsigned int* host_detIds;
    short* host_layers;
    short* host_rings;
    short* host_rods;
    short* host_modules;
    short* host_subdets;
    short* host_sides;
    bool* host_isInverted;
    bool* host_isLower;
    ModuleType* host_moduleType;
    ModuleLayerType* host_moduleLayerType;
    float* host_slopes;
    float* host_drdzs;
    hipHostMalloc(&host_detIds,sizeof(unsigned int)*nModules);
    hipHostMalloc(&host_layers,sizeof(short)*nModules);
    hipHostMalloc(&host_rings,sizeof(short)*nModules);
    hipHostMalloc(&host_rods,sizeof(short)*nModules);
    hipHostMalloc(&host_modules,sizeof(short)*nModules);
    hipHostMalloc(&host_subdets,sizeof(short)*nModules);
    hipHostMalloc(&host_sides,sizeof(short)*nModules);
    hipHostMalloc(&host_isInverted,sizeof(bool)*nModules);
    hipHostMalloc(&host_isLower,sizeof(bool)*nModules);
    hipHostMalloc(&host_moduleType,sizeof(ModuleType)*nModules);
    hipHostMalloc(&host_moduleLayerType,sizeof(ModuleLayerType)*nModules);
    hipHostMalloc(&host_slopes,sizeof(float)*nModules);
    hipHostMalloc(&host_drdzs,sizeof(float)*nModules);
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); it++)
    {
        unsigned int detId = it->first;
        unsigned int index = it->second;
        host_detIds[index] = detId;
        if(detId == 1)
        {
            host_layers[index] = 0;
            host_rings[index] = 0;
            host_rods[index] = 0;
            host_modules[index] = 0;
            host_subdets[index] = SDL::InnerPixel;
            host_sides[index] = 0;
            host_isInverted[index] = 0;
            host_isLower[index] = false;
            host_moduleType[index] = PixelModule;
            host_moduleLayerType[index] = SDL::InnerPixelLayer;
            host_slopes[index] = 0;
            host_drdzs[index] = 0;
        }
        else
        {
            unsigned short layer,ring,rod,module,subdet,side;
            setDerivedQuantities(detId,layer,ring,rod,module,subdet,side);
            host_layers[index] = layer;
            host_rings[index] = ring;
            host_rods[index] = rod;
            host_modules[index] = module;
            host_subdets[index] = subdet;
            host_sides[index] = side;

            host_isInverted[index] = modulesInGPU.parseIsInverted(index,subdet, side,module,layer);
            host_isLower[index] = modulesInGPU.parseIsLower(index, host_isInverted[index], detId);

            host_moduleType[index] = modulesInGPU.parseModuleType(index, subdet, layer, ring);
            host_moduleLayerType[index] = modulesInGPU.parseModuleLayerType(index, host_moduleType[index],host_isInverted[index],host_isLower[index]);

            host_slopes[index] = (subdet == Endcap) ? endcapGeometry.getSlopeLower(detId) : tiltedGeometry.getSlope(detId);
            host_drdzs[index] = (subdet == Barrel) ? tiltedGeometry.getDrDz(detId) : 0;
        }
          lowerModuleCounter[0] += host_isLower[index];
    }

    hipMemcpy(modulesInGPU.nLowerModules,lowerModuleCounter,sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.detIds,host_detIds,nModules*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.layers,host_layers,nModules*sizeof(short),hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.rings,host_rings,sizeof(short)*nModules,hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.rods,host_rods,sizeof(short)*nModules,hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.modules,host_modules,sizeof(short)*nModules,hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.subdets,host_subdets,sizeof(short)*nModules,hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.sides,host_sides,sizeof(short)*nModules,hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.isInverted,host_isInverted,sizeof(bool)*nModules,hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.isLower,host_isLower,sizeof(bool)*nModules,hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.moduleType,host_moduleType,sizeof(ModuleType)*nModules,hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.moduleLayerType,host_moduleLayerType,sizeof(ModuleLayerType)*nModules,hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.slopes,host_slopes,sizeof(float)*nModules,hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.drdzs,host_drdzs,sizeof(float)*nModules,hipMemcpyHostToDevice);
    hipHostFree(host_detIds);
    hipHostFree(host_layers);
    hipHostFree(host_rings);
    hipHostFree(host_rods);
    hipHostFree(host_modules);
    hipHostFree(host_subdets);
    hipHostFree(host_sides);
    hipHostFree(host_isInverted);
    hipHostFree(host_isLower);
    hipHostFree(host_moduleType);
    hipHostFree(host_moduleLayerType);
    hipHostFree(host_slopes);
    hipHostFree(host_drdzs);
    hipHostFree(lowerModuleCounter);
    std::cout<<"number of lower modules (without fake pixel module)= "<<lowerModuleCounter[0]<<std::endl;
    createLowerModuleIndexMapExplicit(modulesInGPU,lowerModuleCounter[0], nModules,host_isLower);
    fillConnectedModuleArrayExplicit(modulesInGPU,nModules);
    resetObjectRanges(modulesInGPU,nModules);

#else
    createModulesInUnifiedMemory(modulesInGPU,nModules);
    unsigned int lowerModuleCounter = 0;
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); it++)
    {
        unsigned int detId = it->first;
        unsigned int index = it->second;
        modulesInGPU.detIds[index] = detId;
        if(detId == 1)
        {
            modulesInGPU.layers[index] = 0;
            modulesInGPU.rings[index] = 0;
            modulesInGPU.rods[index] = 0;
            modulesInGPU.modules[index] = 0;
            modulesInGPU.subdets[index] = SDL::InnerPixel;
            modulesInGPU.sides[index] = 0;
            modulesInGPU.isInverted[index] = 0;
            modulesInGPU.isLower[index] = false;
            modulesInGPU.moduleType[index] = PixelModule;
            modulesInGPU.moduleLayerType[index] = SDL::InnerPixelLayer;
            modulesInGPU.slopes[index] = 0;
            modulesInGPU.drdzs[index] = 0;
        }
        else
        {
            unsigned short layer,ring,rod,module,subdet,side;
            setDerivedQuantities(detId,layer,ring,rod,module,subdet,side);
            modulesInGPU.layers[index] = layer;
            modulesInGPU.rings[index] = ring;
            modulesInGPU.rods[index] = rod;
            modulesInGPU.modules[index] = module;
            modulesInGPU.subdets[index] = subdet;
            modulesInGPU.sides[index] = side;

            modulesInGPU.isInverted[index] = modulesInGPU.parseIsInverted(index);
            modulesInGPU.isLower[index] = modulesInGPU.parseIsLower(index);

            modulesInGPU.moduleType[index] = modulesInGPU.parseModuleType(index);
            modulesInGPU.moduleLayerType[index] = modulesInGPU.parseModuleLayerType(index);

            modulesInGPU.slopes[index] = (subdet == Endcap) ? endcapGeometry.getSlopeLower(detId) : tiltedGeometry.getSlope(detId);
            modulesInGPU.drdzs[index] = (subdet == Barrel) ? tiltedGeometry.getDrDz(detId) : 0;
        }
        if(modulesInGPU.isLower[index]) lowerModuleCounter++;
    }
    *modulesInGPU.nLowerModules = lowerModuleCounter;
    std::cout<<"number of lower modules (without fake pixel module)= "<<*modulesInGPU.nLowerModules<<std::endl;
    createLowerModuleIndexMap(modulesInGPU,lowerModuleCounter, nModules);
    fillConnectedModuleArray(modulesInGPU,nModules);
    resetObjectRanges(modulesInGPU,nModules);
#endif
}

void SDL::fillConnectedModuleArrayExplicit(struct modules& modulesInGPU, unsigned int nModules)
{
    unsigned int* moduleMap;
    unsigned int* nConnectedModules; 
    hipHostMalloc(&moduleMap,nModules * 40 * sizeof(unsigned int));
    hipHostMalloc(&nConnectedModules,nModules * sizeof(unsigned int));
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); ++it)
    {
        unsigned int detId = it->first;
        unsigned int index = it->second;
        auto& connectedModules = moduleConnectionMap.getConnectedModuleDetIds(detId);
        nConnectedModules[index] = connectedModules.size();
        for(unsigned int i = 0; i< nConnectedModules[index];i++)
        {
            moduleMap[index * 40 + i] = (*detIdToIndex)[connectedModules[i]];
        }
    }
    hipMemcpy(modulesInGPU.moduleMap,moduleMap,nModules*40*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(modulesInGPU.nConnectedModules,nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipHostFree(moduleMap);
    hipHostFree(nConnectedModules);
}
void SDL::fillConnectedModuleArray(struct modules& modulesInGPU, unsigned int nModules)
{
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); ++it)
    {
        unsigned int detId = it->first;
        unsigned int index = it->second;
        auto& connectedModules = moduleConnectionMap.getConnectedModuleDetIds(detId);
        modulesInGPU.nConnectedModules[index] = connectedModules.size();
        for(unsigned int i = 0; i< modulesInGPU.nConnectedModules[index];i++)
        {
            modulesInGPU.moduleMap[index * 40 + i] = (*detIdToIndex)[connectedModules[i]];
        }
    }
}

void SDL::setDerivedQuantities(unsigned int detId, unsigned short& layer, unsigned short& ring, unsigned short& rod, unsigned short& module, unsigned short& subdet, unsigned short& side)
{
    subdet = (detId & (7 << 25)) >> 25;
    side = (subdet == Endcap) ? (detId & (3 << 23)) >> 23 : (detId & (3 << 18)) >> 18;
    layer = (subdet == Endcap) ? (detId & (7 << 18)) >> 18 : (detId & (7 << 20)) >> 20;
    ring = (subdet == Endcap) ? (detId & (15 << 12)) >> 12 : 0;
    module = (detId & (127 << 2)) >> 2;
    rod = (subdet == Endcap) ? 0 : (detId & (127 << 10)) >> 10;
}

//auxilliary functions - will be called as needed
bool SDL::modules::parseIsInverted(unsigned int index)
{
    if (subdets[index] == Endcap)
    {
        if (sides[index] == NegZ)
        {
            return modules[index] % 2 == 1;
        }
        else if (sides[index] == PosZ)
        {
            return modules[index] % 2 == 0;
        }
        else
        {
            return 0;
        }
    }
    else if (subdets[index] == Barrel)
    {
        if (sides[index] == Center)
        {
            if (layers[index] <= 3)
            {
                return modules[index] % 2 == 1;
            }
            else if (layers[index] >= 4)
            {
                return modules[index] % 2 == 0;
            }
            else
            {
                return 0;
            }
        }
        else if (sides[index] == NegZ or sides[index] == PosZ)
        {
            if (layers[index] <= 2)
            {
                return modules[index] % 2 == 1;
            }
            else if (layers[index] == 3)
            {
                return modules[index] % 2 == 0;
            }
            else
            {
                return 0;
            }
        }
        else
        {
            return 0;
        }
    }
    else
    {
        return 0;
    }
}
bool SDL::modules::parseIsInverted(unsigned int index, short subdet, short side, short module, short layer)
{
    if (subdet == Endcap)
    {
        if (side == NegZ)
        {
            return module % 2 == 1;
        }
        else if (side == PosZ)
        {
            return module % 2 == 0;
        }
        else
        {
            return 0;
        }
    }
    else if (subdet == Barrel)
    {
        if (side == Center)
        {
            if (layer <= 3)
            {
                return module % 2 == 1;
            }
            else if (layer >= 4)
            {
                return module % 2 == 0;
            }
            else
            {
                return 0;
            }
        }
        else if (side == NegZ or side == PosZ)
        {
            if (layer <= 2)
            {
                return module % 2 == 1;
            }
            else if (layer == 3)
            {
                return module % 2 == 0;
            }
            else
            {
                return 0;
            }
        }
        else
        {
            return 0;
        }
    }
    else
    {
        return 0;
    }
}

bool SDL::modules::parseIsLower(unsigned int index, bool isInvertedx, unsigned int detId)
{
    return (isInvertedx) ? !(detId & 1) : (detId & 1);
}
bool SDL::modules::parseIsLower(unsigned int index)
{
    return (isInverted[index]) ? !(detIds[index] & 1) : (detIds[index] & 1);
}

unsigned int SDL::modules::partnerModuleIndexExplicit(unsigned int index, bool isLowerx, bool isInvertedx)
{
    /*We need to ensure modules with successive det Ids are right next to each other
    or we're dead*/
    if(isLowerx)
    {
        return (isInvertedx ? index - 1: index + 1);
    }
    else
    {
        return (isInvertedx ? index + 1 : index - 1);
    }
}
unsigned int SDL::modules::partnerModuleIndex(unsigned int index)
{
    /*We need to ensure modules with successive det Ids are right next to each other
    or we're dead*/
    if(isLower[index])
    {
        return (isInverted[index] ? index - 1: index + 1);
    }
    else
    {
        return (isInverted[index] ? index + 1 : index - 1);
    }
}

SDL::ModuleType SDL::modules::parseModuleType(unsigned int index, short subdet, short layer, short ring)
{
    if(subdet == Barrel)
    {
        if(layer <= 3)
        {
            return PS;
        }
        else
        {
            return TwoS;
        }
    }
    else
    {
        if(layer <= 2)
        {
            if(ring <= 10)
            {
                return PS;
            }
            else
            {
                return TwoS;
            }
        }
        else
        {
            if(ring <= 7)
            {
                return PS;
            }
            else
            {
                return TwoS;
            }
        }
    }
}
SDL::ModuleType SDL::modules::parseModuleType(unsigned int index)
{
    if(subdets[index] == Barrel)
    {
        if(layers[index] <= 3)
        {
            return PS;
        }
        else
        {
            return TwoS;
        }
    }
    else
    {
        if(layers[index] <= 2)
        {
            if(rings[index] <= 10)
            {
                return PS;
            }
            else
            {
                return TwoS;
            }
        }
        else
        {
            if(rings[index] <= 7)
            {
                return PS;
            }
            else
            {
                return TwoS;
            }
        }
    }
}

SDL::ModuleLayerType SDL::modules::parseModuleLayerType(unsigned int index, ModuleType moduleTypex,bool isInvertedx, bool isLowerx)
{
    if(moduleTypex == TwoS)
    {
        return Strip;
    }
    if(isInvertedx)
    {
        if(isLowerx)
        {
            return Strip;
        }
        else
        {
            return Pixel;
        }
    }
    else
   {
        if(isLowerx)
        {
            return Pixel;
        }
        else
        {
            return Strip;
        }
    }
}
SDL::ModuleLayerType SDL::modules::parseModuleLayerType(unsigned int index)
{
    if(moduleType[index] == TwoS)
    {
        return Strip;
    }
    if(isInverted[index])
    {
        if(isLower[index])
        {
            return Strip;
        }
        else
        {
            return Pixel;
        }
    }
    else
   {
        if(isLower[index])
        {
            return Pixel;
        }
        else
        {
            return Strip;
        }
    }
}

void SDL::resetObjectRanges(struct modules& modulesInGPU, unsigned int nModules)
{
#ifdef Explicit_Module
        hipMemset(modulesInGPU.hitRanges, -1,nModules*2*sizeof(int));
        hipMemset(modulesInGPU.mdRanges, -1,nModules*2*sizeof(int));
        hipMemset(modulesInGPU.segmentRanges, -1,nModules*2*sizeof(int));
        hipMemset(modulesInGPU.trackletRanges, -1,nModules*2*sizeof(int));
        hipMemset(modulesInGPU.tripletRanges, -1,nModules*2*sizeof(int));
        hipMemset(modulesInGPU.trackCandidateRanges, -1,nModules*2*sizeof(int));
#else

#pragma omp parallel for default(shared)
    for(size_t i = 0; i<nModules *2; i++)
    {
        modulesInGPU.hitRanges[i] = -1;
        modulesInGPU.mdRanges[i] = -1;
        modulesInGPU.segmentRanges[i] = -1;
        modulesInGPU.trackletRanges[i] = -1;
        modulesInGPU.tripletRanges[i] = -1;
        modulesInGPU.trackCandidateRanges[i] = -1;
    }
#endif
}
