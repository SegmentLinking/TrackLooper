#include "hip/hip_runtime.h"
# include "Module.cuh"
#include "ModuleConnectionMap.h"
#include "allocate.h"
std::map <unsigned int, uint16_t> *SDL::detIdToIndex;
std::map <unsigned int, float> *SDL::module_x;
std::map <unsigned int, float> *SDL::module_y;
std::map <unsigned int, float> *SDL::module_z;

void SDL::createRangesInUnifiedMemory(struct objectRanges& rangesInGPU,unsigned int nModules,hipStream_t stream, unsigned int nLowerModules)
{
    /* modules stucture object will be created in Event.cu*/
#ifdef CACHE_ALLOC
    rangesInGPU.hitRanges =                 (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    rangesInGPU.hitRangesLower =                 (int*)cms::cuda::allocate_managed(nModules * sizeof(int),stream);
    rangesInGPU.hitRangesUpper =                 (int*)cms::cuda::allocate_managed(nModules * sizeof(int),stream);
    rangesInGPU.hitRangesnLower =                 (int8_t*)cms::cuda::allocate_managed(nModules * sizeof(int8_t),stream);
    rangesInGPU.hitRangesnUpper =                 (int8_t*)cms::cuda::allocate_managed(nModules * sizeof(int8_t),stream);
    rangesInGPU.mdRanges =                  (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    rangesInGPU.segmentRanges =             (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    rangesInGPU.trackletRanges =            (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    rangesInGPU.tripletRanges =             (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    rangesInGPU.trackCandidateRanges =      (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    rangesInGPU.quintupletRanges =          (int*)cms::cuda::allocate_managed(nModules * 2 * sizeof(int),stream);
    rangesInGPU.nEligibleT5Modules =        (uint16_t*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);

    rangesInGPU.quintupletModuleIndices = (int*)cms::cuda::allocate_managed(nLowerModules * sizeof(int),stream);
    rangesInGPU.miniDoubletModuleIndices = (int*)cms::cuda::allocate_managed((nLowerModules + 1) * sizeof(int), stream);
    rangesInGPU.segmentModuleIndices = (int*)cms::cuda::allocate_managed((nLowerModules + 1) * sizeof(int), stream);
    rangesInGPU.tripletModuleIndices = (int*)cms::cuda::allocate_managed(nLowerModules * sizeof(int), stream);

#else
    hipMallocManaged(&rangesInGPU.hitRanges,nModules * 2 * sizeof(int));
    hipMallocManaged(&rangesInGPU.hitRangesLower,nModules  * sizeof(int));
    hipMallocManaged(&rangesInGPU.hitRangesUpper,nModules  * sizeof(int));
    hipMallocManaged(&rangesInGPU.hitRangesnLower,nModules  * sizeof(int8_t));
    hipMallocManaged(&rangesInGPU.hitRangesnUpper,nModules  * sizeof(int8_t));
    hipMallocManaged(&rangesInGPU.mdRanges,nModules * 2 * sizeof(int));
    hipMallocManaged(&rangesInGPU.segmentRanges,nModules * 2 * sizeof(int));
    hipMallocManaged(&rangesInGPU.trackletRanges,nModules * 2 * sizeof(int));
    hipMallocManaged(&rangesInGPU.tripletRanges,nModules * 2 * sizeof(int));
    hipMallocManaged(&rangesInGPU.trackCandidateRanges, nModules * 2 * sizeof(int));
    hipMallocManaged(&rangesInGPU.quintupletRanges, nModules * 2 * sizeof(int));
    hipMallocManaged(&rangesInGPU.nEligibleT5Modules, sizeof(uint16_t));

    hipMallocManaged(&rangesInGPU.quintupletModuleIndices, nLowerModules * sizeof(int));
    hipMallocManaged(&rangesInGPU.miniDoubletModuleIndices, (nLowerModules + 1) * sizeof(int));
    hipMallocManaged(&rangesInGPU.segmentModuleIndices, (nLowerModules + 1) * sizeof(int));
    hipMallocManaged(&rangesInGPU.tripletModuleIndices, nLowerModules * sizeof(int));

#endif
}
void SDL::createRangesInExplicitMemory(struct objectRanges& rangesInGPU,unsigned int nModules,hipStream_t stream, unsigned int nLowerModules)
{
    /* modules stucture object will be created in Event.cu*/
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    rangesInGPU.hitRanges =                  (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.hitRangesLower =                  (int*)cms::cuda::allocate_device(dev,nModules * sizeof(int),stream);
    rangesInGPU.hitRangesUpper =                  (int*)cms::cuda::allocate_device(dev,nModules * sizeof(int),stream);
    rangesInGPU.hitRangesnLower =                  (int8_t*)cms::cuda::allocate_device(dev,nModules * sizeof(int8_t),stream);
    rangesInGPU.hitRangesnUpper =                  (int8_t*)cms::cuda::allocate_device(dev,nModules * sizeof(int8_t),stream);
    rangesInGPU.mdRanges =                   (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.segmentRanges =              (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.trackletRanges =             (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.tripletRanges =              (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.trackCandidateRanges =       (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.quintupletRanges =       (int*)cms::cuda::allocate_device(dev,nModules * 2 * sizeof(int),stream);
    rangesInGPU.nEligibleT5Modules =    (uint16_t*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);

    rangesInGPU.quintupletModuleIndices = (int*)cms::cuda::allocate_device(dev,nLowerModules * sizeof(int),stream);
    rangesInGPU.miniDoubletModuleIndices = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) * sizeof(int), stream);
    rangesInGPU.segmentModuleIndices = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) * sizeof(int), stream);
    rangesInGPU.tripletModuleIndices = (int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(int), stream);

#else
    hipMalloc(&rangesInGPU.hitRanges,nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.hitRangesLower,nModules  * sizeof(int));
    hipMalloc(&rangesInGPU.hitRangesUpper,nModules  * sizeof(int));
    hipMalloc(&rangesInGPU.hitRangesnLower,nModules  * sizeof(int8_t));
    hipMalloc(&rangesInGPU.hitRangesnUpper,nModules  * sizeof(int8_t));
    hipMalloc(&rangesInGPU.mdRanges,nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.segmentRanges,nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.trackletRanges,nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.tripletRanges,nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.trackCandidateRanges, nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.quintupletRanges, nModules * 2 * sizeof(int));
    hipMalloc(&rangesInGPU.nEligibleT5Modules, sizeof(uint16_t));
    hipMalloc(&rangesInGPU.quintupletModuleIndices, nLowerModules * sizeof(int));

    hipMalloc(&rangesInGPU.miniDoubletModuleIndices, (nLowerModules + 1) * sizeof(int));
    hipMalloc(&rangesInGPU.segmentModuleIndices, (nLowerModules + 1) * sizeof(int));
    hipMalloc(&rangesInGPU.tripletModuleIndices, nLowerModules * sizeof(int));

#endif
}
void SDL::createModulesInUnifiedMemory(struct modules& modulesInGPU,unsigned int nModules,hipStream_t stream)
{
    hipMallocManaged(&modulesInGPU.detIds,nModules * sizeof(unsigned int));
    hipMallocManaged(&modulesInGPU.moduleMap,nModules * 40 * sizeof(uint16_t));
    hipMallocManaged(&modulesInGPU.nConnectedModules,nModules * sizeof(uint16_t));
    hipMallocManaged(&modulesInGPU.drdzs,nModules * sizeof(float));
    hipMallocManaged(&modulesInGPU.slopes,nModules * sizeof(float));
    hipMallocManaged(&modulesInGPU.nModules,sizeof(uint16_t));
    hipMallocManaged(&modulesInGPU.nLowerModules,sizeof(uint16_t));
    hipMallocManaged(&modulesInGPU.partnerModuleIndices, nModules * sizeof(uint16_t));

    hipMallocManaged(&modulesInGPU.layers,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.rings,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.modules,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.rods,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.subdets,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.sides,nModules * sizeof(short));
    hipMallocManaged(&modulesInGPU.eta,nModules * sizeof(float));
    hipMallocManaged(&modulesInGPU.r,nModules * sizeof(float));
    hipMallocManaged(&modulesInGPU.isInverted, nModules * sizeof(bool));
    hipMallocManaged(&modulesInGPU.isLower, nModules * sizeof(bool));
    hipMallocManaged(&modulesInGPU.isAnchor, nModules * sizeof(bool));
    hipMallocManaged(&modulesInGPU.moduleType,nModules * sizeof(ModuleType));

    hipMallocManaged(&modulesInGPU.moduleLayerType,nModules * sizeof(ModuleLayerType));

    *modulesInGPU.nModules = nModules;
}
void SDL::createModulesInExplicitMemory(struct modules& modulesInGPU,unsigned int nModules,hipStream_t stream)
{
    /* modules stucture object will be created in Event.cu*/
    hipMalloc(&(modulesInGPU.detIds),nModules * sizeof(unsigned int));
    hipMalloc(&modulesInGPU.moduleMap,nModules * 40 * sizeof(uint16_t));
    hipMalloc(&modulesInGPU.nConnectedModules,nModules * sizeof(uint16_t));
    hipMalloc(&modulesInGPU.drdzs,nModules * sizeof(float));
    hipMalloc(&modulesInGPU.slopes,nModules * sizeof(float));
    hipMalloc(&modulesInGPU.nModules,sizeof(uint16_t));
    hipMalloc(&modulesInGPU.nLowerModules,sizeof(uint16_t));
    hipMalloc(&modulesInGPU.partnerModuleIndices, nModules * sizeof(uint16_t));

    hipMalloc(&modulesInGPU.layers,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.rings,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.modules,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.rods,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.subdets,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.sides,nModules * sizeof(short));
    hipMalloc(&modulesInGPU.eta,nModules * sizeof(float));
    hipMalloc(&modulesInGPU.r,nModules * sizeof(float));
    hipMalloc(&modulesInGPU.isInverted, nModules * sizeof(bool));
    hipMalloc(&modulesInGPU.isLower, nModules * sizeof(bool));
    hipMalloc(&modulesInGPU.isAnchor, nModules * sizeof(bool));
    hipMalloc(&modulesInGPU.moduleType,nModules * sizeof(ModuleType));
    hipMalloc(&modulesInGPU.moduleLayerType,nModules * sizeof(ModuleLayerType));

    hipMemcpyAsync(modulesInGPU.nModules,&nModules,sizeof(uint16_t),hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);
}

void SDL::objectRanges::freeMemoryCache()//struct objectRanges& rangesInGPU)
{
#ifdef Explicit_Module
  int dev;
  hipGetDevice(&dev);
  cms::cuda::free_device(dev,hitRanges);
  cms::cuda::free_device(dev,mdRanges);
  cms::cuda::free_device(dev,segmentRanges);
  cms::cuda::free_device(dev,trackletRanges);
  cms::cuda::free_device(dev,tripletRanges);
  cms::cuda::free_device(dev,trackCandidateRanges);
  cms::cuda::free_device(dev,quintupletRanges);
  cms::cuda::free_device(dev,nEligibleT5Modules);
  cms::cuda::free_device(dev,quintupletModuleIndices);
  cms::cuda::free_device(dev, hitRangesLower);
  cms::cuda::free_device(dev, hitRangesUpper);
  cms::cuda::free_device(dev, hitRangesnLower);
  cms::cuda::free_device(dev, hitRangesnUpper);
  cms::cuda::free_device(dev, miniDoubletModuleIndices);
  cms::cuda::free_device(dev, segmentModuleIndices);
  cms::cuda::free_device(dev, tripletModuleIndices);
#else
  cms::cuda::free_managed(hitRanges);
  cms::cuda::free_managed(mdRanges);
  cms::cuda::free_managed(segmentRanges);
  cms::cuda::free_managed(trackletRanges);
  cms::cuda::free_managed(tripletRanges);
  cms::cuda::free_managed(trackCandidateRanges);
  cms::cuda::free_managed(quintupletRanges);
  cms::cuda::free_managed(nEligibleT5Modules);
  cms::cuda::free_managed(quintupletModuleIndices);
  cms::cuda::free_managed(hitRangesLower);
  cms::cuda::free_managed(hitRangesUpper);
  cms::cuda::free_managed(hitRangesnLower);
  cms::cuda::free_managed(hitRangesnUpper);
  cms::cuda::free_managed(miniDoubletModuleIndices);
  cms::cuda::free_managed(segmentModuleIndices);
  cms::cuda::free_managed(tripletModuleIndices);

#endif
}
void SDL::objectRanges::freeMemory()
{
  hipFree(hitRanges);
  hipFree(hitRangesLower);
  hipFree(hitRangesUpper);
  hipFree(hitRangesnLower);
  hipFree(hitRangesnUpper);
  hipFree(mdRanges);
  hipFree(segmentRanges);
  hipFree(trackletRanges);
  hipFree(tripletRanges);
  hipFree(trackCandidateRanges);
  hipFree(quintupletRanges);
  hipFree(nEligibleT5Modules);
  hipFree(quintupletModuleIndices);
  hipFree(miniDoubletModuleIndices);
  hipFree(segmentModuleIndices);
  hipFree(tripletModuleIndices);
}
void SDL::freeModulesCache(struct modules& modulesInGPU,struct pixelMap& pixelMapping)
{
#ifdef Explicit_Module
  int dev;
  hipGetDevice(&dev);
  cms::cuda::free_device(dev,modulesInGPU.detIds);
  cms::cuda::free_device(dev,modulesInGPU.moduleMap);
  cms::cuda::free_device(dev,modulesInGPU.nConnectedModules);
  cms::cuda::free_device(dev,modulesInGPU.drdzs);
  cms::cuda::free_device(dev,modulesInGPU.slopes);
  cms::cuda::free_device(dev,modulesInGPU.nModules);
  cms::cuda::free_device(dev,modulesInGPU.nLowerModules);
  cms::cuda::free_device(dev,modulesInGPU.layers);
  cms::cuda::free_device(dev,modulesInGPU.rings);
  cms::cuda::free_device(dev,modulesInGPU.modules);
  cms::cuda::free_device(dev,modulesInGPU.rods);
  cms::cuda::free_device(dev,modulesInGPU.subdets);
  cms::cuda::free_device(dev,modulesInGPU.sides);
  cms::cuda::free_device(dev,modulesInGPU.isInverted);
  cms::cuda::free_device(dev,modulesInGPU.isLower);
  cms::cuda::free_device(dev,modulesInGPU.isAnchor);
  cms::cuda::free_device(dev,modulesInGPU.moduleType);
  cms::cuda::free_device(dev,modulesInGPU.moduleLayerType);
  cms::cuda::free_device(dev,modulesInGPU.connectedPixels);
#else
  cms::cuda::free_managed(modulesInGPU.detIds);
  cms::cuda::free_managed(modulesInGPU.moduleMap);
  cms::cuda::free_managed(modulesInGPU.nConnectedModules);
  cms::cuda::free_managed(modulesInGPU.drdzs);
  cms::cuda::free_managed(modulesInGPU.slopes);
  cms::cuda::free_managed(modulesInGPU.nModules);
  cms::cuda::free_managed(modulesInGPU.nLowerModules);
  cms::cuda::free_managed(modulesInGPU.layers);
  cms::cuda::free_managed(modulesInGPU.rings);
  cms::cuda::free_managed(modulesInGPU.modules);
  cms::cuda::free_managed(modulesInGPU.rods);
  cms::cuda::free_managed(modulesInGPU.subdets);
  cms::cuda::free_managed(modulesInGPU.sides);
  cms::cuda::free_managed(modulesInGPU.isInverted);
  cms::cuda::free_managed(modulesInGPU.isLower);
  cms::cuda::free_managed(modulesInGPU.isAnchor);
  cms::cuda::free_managed(modulesInGPU.moduleType);
  cms::cuda::free_managed(modulesInGPU.moduleLayerType);
  cms::cuda::free_managed(modulesInGPU.connectedPixels);
#endif
  hipHostFree(pixelMapping.connectedPixelsSizes);
  hipHostFree(pixelMapping.connectedPixelsSizesPos);
  hipHostFree(pixelMapping.connectedPixelsSizesNeg);
  hipHostFree(pixelMapping.connectedPixelsIndex);
  hipHostFree(pixelMapping.connectedPixelsIndexPos);
  hipHostFree(pixelMapping.connectedPixelsIndexNeg);
}
void SDL::freeModules(struct modules& modulesInGPU, struct pixelMap& pixelMapping,hipStream_t stream)
{

  hipFree(modulesInGPU.detIds);
  hipFree(modulesInGPU.moduleMap);
  hipFree(modulesInGPU.nConnectedModules);
  hipFree(modulesInGPU.drdzs);
  hipFree(modulesInGPU.slopes);
  hipFree(modulesInGPU.nModules);
  hipFree(modulesInGPU.nLowerModules);
  hipFree(modulesInGPU.layers);
  hipFree(modulesInGPU.rings);
  hipFree(modulesInGPU.modules);
  hipFree(modulesInGPU.rods);
  hipFree(modulesInGPU.subdets);
  hipFree(modulesInGPU.sides);
  hipFree(modulesInGPU.eta);
  hipFree(modulesInGPU.r);
  hipFree(modulesInGPU.isInverted);
  hipFree(modulesInGPU.isLower);
  hipFree(modulesInGPU.isAnchor);
  hipFree(modulesInGPU.moduleType);
  hipFree(modulesInGPU.moduleLayerType);
  hipFree(modulesInGPU.connectedPixels);
  hipFree(modulesInGPU.partnerModuleIndices);

  hipHostFree(pixelMapping.connectedPixelsSizes);
  hipHostFree(pixelMapping.connectedPixelsSizesPos);
  hipHostFree(pixelMapping.connectedPixelsSizesNeg);
  hipHostFree(pixelMapping.connectedPixelsIndex);
  hipHostFree(pixelMapping.connectedPixelsIndexPos);
  hipHostFree(pixelMapping.connectedPixelsIndexNeg);
}

void SDL::loadModulesFromFile(struct modules& modulesInGPU, uint16_t& nModules, uint16_t& nLowerModules, struct pixelMap& pixelMapping,hipStream_t stream, const char* moduleMetaDataFilePath)
{
    detIdToIndex = new std::map<unsigned int, uint16_t>;
    module_x = new std::map<unsigned int, float>;
    module_y = new std::map<unsigned int, float>;
    module_z = new std::map<unsigned int, float>;

    /*modules structure object will be created in Event.cu*/
    /* Load the whole text file into the unordered_map first*/

    std::ifstream ifile;
    ifile.open(moduleMetaDataFilePath);
    if(!ifile.is_open())
    {
        std::cout<<"ERROR! module list file not present!"<<std::endl;
    }
    std::string line;
    uint16_t counter = 0;

    while(std::getline(ifile,line))
    {
        std::stringstream ss(line);
        std::string token;
        int count_number = 0;

        unsigned int temp_detId;
        while(std::getline(ss,token,','))
        {
            if(count_number == 0)
            {
                temp_detId = stoi(token);
                (*detIdToIndex)[temp_detId] = counter;
            }
            if(count_number == 1)
                (*module_x)[temp_detId] = std::stof(token);
            if(count_number == 2)
                (*module_y)[temp_detId] = std::stof(token);
            if(count_number == 3)
            {
                (*module_z)[temp_detId] = std::stof(token);
                counter++;
            }
            count_number++;
            if(count_number>3)
                break;
        }

    }
    (*detIdToIndex)[1] = counter; //pixel module is the last module in the module list
    counter++;
    nModules = counter;
    std::cout<<"Number of modules = "<<nModules<<std::endl;
#ifdef Explicit_Module
    createModulesInExplicitMemory(modulesInGPU,nModules,stream);
    unsigned int* host_detIds;
    short* host_layers;
    short* host_rings;
    short* host_rods;
    short* host_modules;
    short* host_subdets;
    short* host_sides;
    float* host_eta;
    float* host_r;
    bool* host_isInverted;
    bool* host_isLower;
    bool* host_isAnchor;
    ModuleType* host_moduleType;
    ModuleLayerType* host_moduleLayerType;
    float* host_slopes;
    float* host_drdzs;
    uint16_t* host_partnerModuleIndices;

    hipHostMalloc(&host_detIds,sizeof(unsigned int)*nModules);
    hipHostMalloc(&host_layers,sizeof(short)*nModules);
    hipHostMalloc(&host_rings,sizeof(short)*nModules);
    hipHostMalloc(&host_rods,sizeof(short)*nModules);
    hipHostMalloc(&host_modules,sizeof(short)*nModules);
    hipHostMalloc(&host_subdets,sizeof(short)*nModules);
    hipHostMalloc(&host_sides,sizeof(short)*nModules);
    hipHostMalloc(&host_eta,sizeof(float)*nModules);
    hipHostMalloc(&host_r,sizeof(float)*nModules);
    hipHostMalloc(&host_isInverted,sizeof(bool)*nModules);
    hipHostMalloc(&host_isLower,sizeof(bool)*nModules);
    hipHostMalloc(&host_isAnchor, sizeof(bool) * nModules);
    hipHostMalloc(&host_moduleType,sizeof(ModuleType)*nModules);
    hipHostMalloc(&host_moduleLayerType,sizeof(ModuleLayerType)*nModules);
    hipHostMalloc(&host_slopes,sizeof(float)*nModules);
    hipHostMalloc(&host_drdzs,sizeof(float)*nModules);
    hipHostMalloc(&host_partnerModuleIndices, sizeof(uint16_t) * nModules);
    
    //reassign detIdToIndex indices here
    nLowerModules = (nModules - 1) / 2;
    uint16_t lowerModuleCounter = 0;
    uint16_t upperModuleCounter = nLowerModules + 1;
    //0 to nLowerModules - 1 => only lower modules, nLowerModules - pixel module, nLowerModules + 1 to nModules => upper modules
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); it++)
    {
        unsigned int detId = it->first;
        float m_x = (*module_x)[detId];
        float m_y = (*module_y)[detId];
        float m_z = (*module_z)[detId];

        float eta,r;

        uint16_t index;
        unsigned short layer,ring,rod,module,subdet,side;
        bool isInverted, isLower;
        if(detId == 1)
        {
            layer = 0;
            ring = 0;
            rod = 0;
            module = 0;
            subdet = 0;
            side = 0;
            isInverted = false;
            isLower = false;
        }
        else
        {
            setDerivedQuantities(detId,layer,ring,rod,module,subdet,side,m_x,m_y,m_z,eta,r);
            isInverted = modulesInGPU.parseIsInverted(subdet, side, module, layer);
            isLower = modulesInGPU.parseIsLower(isInverted, detId);
        }
        if(isLower)
        {
            index = lowerModuleCounter;
            lowerModuleCounter++;
        }
        else if(detId != 1)
        {
            index = upperModuleCounter;
            upperModuleCounter++;
        }
        else
        {
            index = nLowerModules; //pixel
        }
        //reassigning indices!
        (*detIdToIndex)[detId] = index;   
        host_detIds[index] = detId;
        host_layers[index] = layer;
        host_rings[index] = ring;
        host_rods[index] = rod;
        host_modules[index] = module;
        host_subdets[index] = subdet;
        host_sides[index] = side;
        host_eta[index] = eta;
        host_r[index] = r;
        host_isInverted[index] = isInverted;
        host_isLower[index] = isLower;

        //assigning other variables!
        if(detId == 1)
        {
            host_moduleType[index] = PixelModule;
            host_moduleLayerType[index] = SDL::InnerPixelLayer;
            host_slopes[index] = 0;
            host_drdzs[index] = 0;
            host_isAnchor[index] = false;
        }
        else
        {

            host_moduleType[index] = modulesInGPU.parseModuleType(subdet, layer, ring);
            host_moduleLayerType[index] = modulesInGPU.parseModuleLayerType(host_moduleType[index],host_isInverted[index],host_isLower[index]);

            if(host_moduleType[index] == SDL::PS and host_moduleLayerType[index] == SDL::Pixel)
            {
                host_isAnchor[index] = true;
            }
            else if(host_moduleType[index] == SDL::TwoS and host_isLower[index])
            {
                host_isAnchor[index] = true;   
            }
            else
            {
                host_isAnchor[index] = false;
            }

            host_slopes[index] = (subdet == Endcap) ? endcapGeometry.getSlopeLower(detId) : tiltedGeometry.getSlope(detId);
            host_drdzs[index] = (subdet == Barrel) ? tiltedGeometry.getDrDz(detId) : 0;
        }
    }

    //partner module stuff, and slopes and drdz move around
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); it++)
    {
        auto& detId = it->first;
        auto& index = it->second;
        if(detId != 1)
        {
            host_partnerModuleIndices[index] = (*detIdToIndex)[modulesInGPU.parsePartnerModuleId(detId, host_isLower[index], host_isInverted[index])];
            //add drdz and slope importing stuff here!
            if(host_drdzs[index] == 0)
            {
                host_drdzs[index] = host_drdzs[host_partnerModuleIndices[index]];
            }
            if(host_slopes[index] == 0)
            {
                host_slopes[index] = host_slopes[host_partnerModuleIndices[index]];
            }
        }
    }

    hipMemcpyAsync(modulesInGPU.nLowerModules,&nLowerModules,sizeof(uint16_t),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.detIds,host_detIds,nModules*sizeof(unsigned int),hipMemcpyHostToDevice,stream);

    hipMemcpyAsync(modulesInGPU.layers,host_layers,nModules*sizeof(short),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.rings,host_rings,sizeof(short)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.rods,host_rods,sizeof(short)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.modules,host_modules,sizeof(short)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.subdets,host_subdets,sizeof(short)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.sides,host_sides,sizeof(short)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.eta,host_eta,sizeof(float)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.r,host_r,sizeof(float)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.isInverted,host_isInverted,sizeof(bool)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.isLower,host_isLower,sizeof(bool)*nModules,hipMemcpyHostToDevice,stream);

    hipMemcpyAsync(modulesInGPU.moduleType,host_moduleType,sizeof(ModuleType)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.moduleLayerType,host_moduleLayerType,sizeof(ModuleLayerType)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.slopes,host_slopes,sizeof(float)*nModules,hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.isAnchor, host_isAnchor, sizeof(bool) * nModules, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(modulesInGPU.drdzs,host_drdzs,sizeof(float)*nModules,hipMemcpyHostToDevice,stream);

    hipMemcpyAsync(modulesInGPU.partnerModuleIndices, host_partnerModuleIndices, sizeof(uint16_t) * nModules, hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);

    hipHostFree(host_detIds);
    hipHostFree(host_layers);
    hipHostFree(host_rings);
    hipHostFree(host_rods);
    hipHostFree(host_modules);
    hipHostFree(host_subdets);
    hipHostFree(host_sides);
    hipHostFree(host_eta);
    hipHostFree(host_r);
    hipHostFree(host_isInverted);
    hipHostFree(host_isLower);
    hipHostFree(host_isAnchor);
    hipHostFree(host_moduleType);
    hipHostFree(host_moduleLayerType);
    hipHostFree(host_slopes);
    hipHostFree(host_drdzs);
    hipHostFree(host_partnerModuleIndices);
    std::cout<<"number of lower modules (without fake pixel module)= "<<lowerModuleCounter<<std::endl;
    fillConnectedModuleArrayExplicit(modulesInGPU,nModules,stream);
    fillPixelMap(modulesInGPU,pixelMapping,stream);

#else
    createModulesInUnifiedMemory(modulesInGPU,nModules,stream);
    nLowerModules = (nModules - 1) / 2;
    unsigned int lowerModuleCounter = 0;
    unsigned int upperModuleCounter = nLowerModules + 1;
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); it++)
    {
        unsigned int detId = it->first;
        float m_x = (*module_x)[detId];
        float m_y = (*module_y)[detId];
        float m_z = (*module_z)[detId];

        float eta,r;

        uint16_t index;
        unsigned short layer,ring,rod,module,subdet,side;
        bool isInverted, isLower;
        if(detId == 1)
        {
            layer = 0;
            ring = 0;
            rod = 0;
            module = 0;
            subdet = 0;
            side = 0;
            isInverted = false;
            isLower = false;

        }
        else
        {
            setDerivedQuantities(detId,layer,ring,rod,module,subdet,side,m_x,m_y,m_z,eta,r);
            isInverted = modulesInGPU.parseIsInverted(subdet, side, module, layer);
            isLower = modulesInGPU.parseIsLower(isInverted, detId);
        }
        if(isLower)
        {
            index = lowerModuleCounter;
            lowerModuleCounter++;
        }
        else if(detId != 1)
        {
            index = upperModuleCounter;
            upperModuleCounter++;
        }
        else
        {
            index = nLowerModules; //pixel
        }
        //reassigning indices!
        (*detIdToIndex)[detId] = index;   
        modulesInGPU.detIds[index] = detId;
        modulesInGPU.layers[index] = layer;
        modulesInGPU.rings[index] = ring;
        modulesInGPU.rods[index] = rod;
        modulesInGPU.modules[index] = module;
        modulesInGPU.subdets[index] = subdet;
        modulesInGPU.sides[index] = side;
        modulesInGPU.isInverted[index] = isInverted;
        modulesInGPU.isLower[index] = isLower;
        modulesInGPU.eta[index] = eta;
        modulesInGPU.r[index] = r;

        if(detId == 1)
        {
            modulesInGPU.moduleType[index] = PixelModule;
            modulesInGPU.moduleLayerType[index] = SDL::InnerPixelLayer;
            modulesInGPU.slopes[index] = 0;
            modulesInGPU.drdzs[index] = 0;
            modulesInGPU.isAnchor[index] = false;
        }
        else
        {

            modulesInGPU.moduleType[index] = modulesInGPU.parseModuleType(subdet, layer, ring);
            modulesInGPU.moduleLayerType[index] = modulesInGPU.parseModuleLayerType(modulesInGPU.moduleType[index],modulesInGPU.isInverted[index],modulesInGPU.isLower[index]);

            if(modulesInGPU.moduleType[index] == SDL::PS and modulesInGPU.moduleLayerType[index] == SDL::Pixel)
            {
                modulesInGPU.isAnchor[index] = true;
            }
            else if(modulesInGPU.moduleType[index] == SDL::TwoS and modulesInGPU.isLower[index])
            {
                modulesInGPU.isAnchor[index] = true;   
            }
            else
            {
                modulesInGPU.isAnchor[index] = false;
            }

            modulesInGPU.slopes[index] = (subdet == Endcap) ? endcapGeometry.getSlopeLower(detId) : tiltedGeometry.getSlope(detId);
            modulesInGPU.drdzs[index] = (subdet == Barrel) ? tiltedGeometry.getDrDz(detId) : 0;
        }
    }


    //partner module stuff, and slopes and drdz move around
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); it++)
    {
        auto& detId = it->first;
        auto& index = it->second;
        if(detId != 1)
        {
            modulesInGPU.partnerModuleIndices[index] = (*detIdToIndex)[modulesInGPU.parsePartnerModuleId(detId, modulesInGPU.isLower[index], modulesInGPU.isInverted[index])];
            //add drdz and slope importing stuff here!
            if(modulesInGPU.drdzs[index] == 0)
            {
                modulesInGPU.drdzs[index] = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndices[index]];
            }
            if(modulesInGPU.slopes[index] == 0)
            {
                modulesInGPU.slopes[index] = modulesInGPU.slopes[modulesInGPU.partnerModuleIndices[index]];
            }
        }
    }

    *(modulesInGPU.nLowerModules) = nLowerModules;
    std::cout<<"number of lower modules (without fake pixel module)= "<<*modulesInGPU.nLowerModules<<std::endl;
    fillConnectedModuleArray(modulesInGPU,nModules);
    fillPixelMap(modulesInGPU,pixelMapping,stream);
    #endif
}

void SDL::fillConnectedModuleArray(struct modules& modulesInGPU, unsigned int nModules)
{
    uint16_t* moduleMap;
    uint16_t* nConnectedModules;
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); ++it)
    {
        unsigned int detId = it->first;
        uint16_t index = it->second;
        auto& connectedModules = moduleConnectionMap.getConnectedModuleDetIds(detId);
        modulesInGPU.nConnectedModules[index] = connectedModules.size();
        for(uint16_t i = 0; i< modulesInGPU.nConnectedModules[index];i++)
        {
            modulesInGPU.moduleMap[index * 40 + i] = (*detIdToIndex)[connectedModules[i]];
        }
    }
}

void SDL::fillPixelMap(struct modules& modulesInGPU, struct pixelMap& pixelMapping,hipStream_t stream)
{
    int size_superbins = 45000;//SDL::moduleConnectionMap_pLStoLayer1Subdet5.size(); //changed to 45000 to reduce memory useage on GPU
    std::vector<unsigned int> connectedModuleDetIds;
    std::vector<unsigned int> connectedModuleDetIds_pos;
    std::vector<unsigned int> connectedModuleDetIds_neg;
    unsigned int* connectedPixelsIndex;
    unsigned int* connectedPixelsIndexPos;
    unsigned int* connectedPixelsIndexNeg;
    unsigned int* connectedPixelsSizes;
    unsigned int* connectedPixelsSizesPos;
    unsigned int* connectedPixelsSizesNeg;
    hipHostMalloc(&pixelMapping.connectedPixelsIndex,size_superbins * sizeof(unsigned int));
    hipHostMalloc(&pixelMapping.connectedPixelsSizes,size_superbins * sizeof(unsigned int));
    hipHostMalloc(&pixelMapping.connectedPixelsIndexPos,size_superbins * sizeof(unsigned int));
    hipHostMalloc(&pixelMapping.connectedPixelsSizesPos,size_superbins * sizeof(unsigned int));
    hipHostMalloc(&pixelMapping.connectedPixelsIndexNeg,size_superbins * sizeof(unsigned int));
    hipHostMalloc(&pixelMapping.connectedPixelsSizesNeg,size_superbins * sizeof(unsigned int));
    int totalSizes=0;
    int totalSizes_pos=0;
    int totalSizes_neg=0;
    for(int isuperbin =0; isuperbin<size_superbins; isuperbin++)
    {
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet5 = SDL::moduleConnectionMap_pLStoLayer1Subdet5.getConnectedModuleDetIds(isuperbin+size_superbins);// index adjustment to get high values
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet5 = SDL::moduleConnectionMap_pLStoLayer2Subdet5.getConnectedModuleDetIds(isuperbin+size_superbins);// from the high pt bins
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet5 = SDL::moduleConnectionMap_pLStoLayer3Subdet5.getConnectedModuleDetIds(isuperbin+size_superbins);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet4 = SDL::moduleConnectionMap_pLStoLayer1Subdet4.getConnectedModuleDetIds(isuperbin+size_superbins);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet4 = SDL::moduleConnectionMap_pLStoLayer2Subdet4.getConnectedModuleDetIds(isuperbin+size_superbins);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet4 = SDL::moduleConnectionMap_pLStoLayer3Subdet4.getConnectedModuleDetIds(isuperbin+size_superbins);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer4Subdet4 = SDL::moduleConnectionMap_pLStoLayer4Subdet4.getConnectedModuleDetIds(isuperbin+size_superbins);
      connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer1Subdet5.begin(),connectedModuleDetIds_pLStoLayer1Subdet5.end());
      connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer2Subdet5.begin(),connectedModuleDetIds_pLStoLayer2Subdet5.end());
      connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer3Subdet5.begin(),connectedModuleDetIds_pLStoLayer3Subdet5.end());
      connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer1Subdet4.begin(),connectedModuleDetIds_pLStoLayer1Subdet4.end());
      connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer2Subdet4.begin(),connectedModuleDetIds_pLStoLayer2Subdet4.end());
      connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer3Subdet4.begin(),connectedModuleDetIds_pLStoLayer3Subdet4.end());
      connectedModuleDetIds.insert(connectedModuleDetIds.end(),connectedModuleDetIds_pLStoLayer4Subdet4.begin(),connectedModuleDetIds_pLStoLayer4Subdet4.end());

      int sizes =0;
      sizes += connectedModuleDetIds_pLStoLayer1Subdet5.size();
      sizes += connectedModuleDetIds_pLStoLayer2Subdet5.size();
      sizes += connectedModuleDetIds_pLStoLayer3Subdet5.size();
      sizes += connectedModuleDetIds_pLStoLayer1Subdet4.size();
      sizes += connectedModuleDetIds_pLStoLayer2Subdet4.size();
      sizes += connectedModuleDetIds_pLStoLayer3Subdet4.size();
      sizes += connectedModuleDetIds_pLStoLayer4Subdet4.size();
      pixelMapping.connectedPixelsIndex[isuperbin] = totalSizes;
      pixelMapping.connectedPixelsSizes[isuperbin] = sizes;
      totalSizes += sizes;


      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet5_pos = SDL::moduleConnectionMap_pLStoLayer1Subdet5_pos.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet5_pos = SDL::moduleConnectionMap_pLStoLayer2Subdet5_pos.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet5_pos = SDL::moduleConnectionMap_pLStoLayer3Subdet5_pos.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet4_pos = SDL::moduleConnectionMap_pLStoLayer1Subdet4_pos.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet4_pos = SDL::moduleConnectionMap_pLStoLayer2Subdet4_pos.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet4_pos = SDL::moduleConnectionMap_pLStoLayer3Subdet4_pos.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer4Subdet4_pos = SDL::moduleConnectionMap_pLStoLayer4Subdet4_pos.getConnectedModuleDetIds(isuperbin);
      connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer1Subdet5_pos.begin(),connectedModuleDetIds_pLStoLayer1Subdet5_pos.end());
      connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer2Subdet5_pos.begin(),connectedModuleDetIds_pLStoLayer2Subdet5_pos.end());
      connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer3Subdet5_pos.begin(),connectedModuleDetIds_pLStoLayer3Subdet5_pos.end());
      connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer1Subdet4_pos.begin(),connectedModuleDetIds_pLStoLayer1Subdet4_pos.end());
      connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer2Subdet4_pos.begin(),connectedModuleDetIds_pLStoLayer2Subdet4_pos.end());
      connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer3Subdet4_pos.begin(),connectedModuleDetIds_pLStoLayer3Subdet4_pos.end());
      connectedModuleDetIds_pos.insert(connectedModuleDetIds_pos.end(),connectedModuleDetIds_pLStoLayer4Subdet4_pos.begin(),connectedModuleDetIds_pLStoLayer4Subdet4_pos.end());

      int sizes_pos =0;
      sizes_pos += connectedModuleDetIds_pLStoLayer1Subdet5_pos.size();
      sizes_pos += connectedModuleDetIds_pLStoLayer2Subdet5_pos.size();
      sizes_pos += connectedModuleDetIds_pLStoLayer3Subdet5_pos.size();
      sizes_pos += connectedModuleDetIds_pLStoLayer1Subdet4_pos.size();
      sizes_pos += connectedModuleDetIds_pLStoLayer2Subdet4_pos.size();
      sizes_pos += connectedModuleDetIds_pLStoLayer3Subdet4_pos.size();
      sizes_pos += connectedModuleDetIds_pLStoLayer4Subdet4_pos.size();
      pixelMapping.connectedPixelsIndexPos[isuperbin] = totalSizes_pos;
      pixelMapping.connectedPixelsSizesPos[isuperbin] = sizes_pos;
      totalSizes_pos += sizes_pos;


      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet5_neg = SDL::moduleConnectionMap_pLStoLayer1Subdet5_neg.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet5_neg = SDL::moduleConnectionMap_pLStoLayer2Subdet5_neg.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet5_neg = SDL::moduleConnectionMap_pLStoLayer3Subdet5_neg.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer1Subdet4_neg = SDL::moduleConnectionMap_pLStoLayer1Subdet4_neg.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer2Subdet4_neg = SDL::moduleConnectionMap_pLStoLayer2Subdet4_neg.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer3Subdet4_neg = SDL::moduleConnectionMap_pLStoLayer3Subdet4_neg.getConnectedModuleDetIds(isuperbin);
      std::vector<unsigned int> connectedModuleDetIds_pLStoLayer4Subdet4_neg = SDL::moduleConnectionMap_pLStoLayer4Subdet4_neg.getConnectedModuleDetIds(isuperbin);
      connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer1Subdet5_neg.begin(),connectedModuleDetIds_pLStoLayer1Subdet5_neg.end());
      connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer2Subdet5_neg.begin(),connectedModuleDetIds_pLStoLayer2Subdet5_neg.end());
      connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer3Subdet5_neg.begin(),connectedModuleDetIds_pLStoLayer3Subdet5_neg.end());
      connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer1Subdet4_neg.begin(),connectedModuleDetIds_pLStoLayer1Subdet4_neg.end());
      connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer2Subdet4_neg.begin(),connectedModuleDetIds_pLStoLayer2Subdet4_neg.end());
      connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer3Subdet4_neg.begin(),connectedModuleDetIds_pLStoLayer3Subdet4_neg.end());
      connectedModuleDetIds_neg.insert(connectedModuleDetIds_neg.end(),connectedModuleDetIds_pLStoLayer4Subdet4_neg.begin(),connectedModuleDetIds_pLStoLayer4Subdet4_neg.end());

      int sizes_neg =0;
      sizes_neg += connectedModuleDetIds_pLStoLayer1Subdet5_neg.size();
      sizes_neg += connectedModuleDetIds_pLStoLayer2Subdet5_neg.size();
      sizes_neg += connectedModuleDetIds_pLStoLayer3Subdet5_neg.size();
      sizes_neg += connectedModuleDetIds_pLStoLayer1Subdet4_neg.size();
      sizes_neg += connectedModuleDetIds_pLStoLayer2Subdet4_neg.size();
      sizes_neg += connectedModuleDetIds_pLStoLayer3Subdet4_neg.size();
      sizes_neg += connectedModuleDetIds_pLStoLayer4Subdet4_neg.size();
      pixelMapping.connectedPixelsIndexNeg[isuperbin] = totalSizes_neg;
      pixelMapping.connectedPixelsSizesNeg[isuperbin] = sizes_neg;
      totalSizes_neg += sizes_neg;

    }

    unsigned int* connectedPixels;
    hipHostMalloc(&connectedPixels,(totalSizes+totalSizes_pos+totalSizes_neg) * sizeof(unsigned int));
#ifdef Explicit_Module
    hipMalloc(&modulesInGPU.connectedPixels,(totalSizes+totalSizes_pos+totalSizes_neg)* sizeof(unsigned int));
#else
    hipMallocManaged(&modulesInGPU.connectedPixels,(totalSizes+totalSizes_pos+totalSizes_neg)* sizeof(unsigned int));
#endif

    for(int icondet=0; icondet< totalSizes; icondet++){
      connectedPixels[icondet] = (*detIdToIndex)[connectedModuleDetIds[icondet]];
    }
    for(int icondet=0; icondet< totalSizes_pos; icondet++){
      connectedPixels[icondet+totalSizes] = (*detIdToIndex)[connectedModuleDetIds_pos[icondet]];
    }
    for(int icondet=0; icondet< totalSizes_neg; icondet++){
      connectedPixels[icondet+totalSizes+totalSizes_pos] = (*detIdToIndex)[connectedModuleDetIds_neg[icondet]];
    }
    hipMemcpyAsync(modulesInGPU.connectedPixels,connectedPixels,(totalSizes+totalSizes_pos+totalSizes_neg)*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);

    hipHostFree(connectedPixels);
}

void SDL::fillConnectedModuleArrayExplicit(struct modules& modulesInGPU, unsigned int nModules,hipStream_t stream)
{
    uint16_t* moduleMap;
    uint16_t* nConnectedModules;
    hipHostMalloc(&moduleMap,nModules * 40 * sizeof(uint16_t));
    hipHostMalloc(&nConnectedModules,nModules * sizeof(uint16_t));
    for(auto it = (*detIdToIndex).begin(); it != (*detIdToIndex).end(); ++it)
    {
        unsigned int detId = it->first;
        uint16_t index = it->second;
        auto& connectedModules = moduleConnectionMap.getConnectedModuleDetIds(detId);
        nConnectedModules[index] = connectedModules.size();
        for(uint16_t i = 0; i< nConnectedModules[index];i++)
        {
            moduleMap[index * 40 + i] = (*detIdToIndex)[connectedModules[i]];
        }
    }
    hipMemcpyAsync(modulesInGPU.moduleMap,moduleMap,nModules*40*sizeof(uint16_t),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(modulesInGPU.nConnectedModules,nConnectedModules,nModules*sizeof(uint16_t),hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);
    hipHostFree(moduleMap);
    hipHostFree(nConnectedModules);
}

void SDL::setDerivedQuantities(unsigned int detId, unsigned short& layer, unsigned short& ring, unsigned short& rod, unsigned short& module, unsigned short& subdet, unsigned short& side, float m_x, float m_y, float m_z, float& eta, float& r)
{
    subdet = (detId & (7 << 25)) >> 25;
    side = (subdet == Endcap) ? (detId & (3 << 23)) >> 23 : (detId & (3 << 18)) >> 18;
    layer = (subdet == Endcap) ? (detId & (7 << 18)) >> 18 : (detId & (7 << 20)) >> 20;
    ring = (subdet == Endcap) ? (detId & (15 << 12)) >> 12 : 0;
    module = (detId & (127 << 2)) >> 2;
    rod = (subdet == Endcap) ? 0 : (detId & (127 << 10)) >> 10;

    r = std::sqrt(m_x * m_x + m_y * m_y + m_z * m_z);
    eta = ((m_z > 0) - ( m_z < 0)) * std::acosh(r / std::sqrt(m_x * m_x + m_y * m_y));
}

//auxilliary functions - will be called as needed
bool SDL::modules::parseIsInverted(unsigned int index)
{
    if (subdets[index] == Endcap)
    {
        if (sides[index] == NegZ)
        {
            return modules[index] % 2 == 1;
        }
        else if (sides[index] == PosZ)
        {
            return modules[index] % 2 == 0;
        }
        else
        {
            return 0;
        }
    }
    else if (subdets[index] == Barrel)
    {
        if (sides[index] == Center)
        {
            if (layers[index] <= 3)
            {
                return modules[index] % 2 == 1;
            }
            else if (layers[index] >= 4)
            {
                return modules[index] % 2 == 0;
            }
            else
            {
                return 0;
            }
        }
        else if (sides[index] == NegZ or sides[index] == PosZ)
        {
            if (layers[index] <= 2)
            {
                return modules[index] % 2 == 1;
            }
            else if (layers[index] == 3)
            {
                return modules[index] % 2 == 0;
            }
            else
            {
                return 0;
            }
        }
        else
        {
            return 0;
        }
    }
    else
    {
        return 0;
    }
}
bool SDL::modules::parseIsInverted(short subdet, short side, short module, short layer)
{
    if (subdet == Endcap)
    {
        if (side == NegZ)
        {
            return module % 2 == 1;
        }
        else if (side == PosZ)
        {
            return module % 2 == 0;
        }
        else
        {
            return 0;
        }
    }
    else if (subdet == Barrel)
    {
        if (side == Center)
        {
            if (layer <= 3)
            {
                return module % 2 == 1;
            }
            else if (layer >= 4)
            {
                return module % 2 == 0;
            }
            else
            {
                return 0;
            }
        }
        else if (side == NegZ or side == PosZ)
        {
            if (layer <= 2)
            {
                return module % 2 == 1;
            }
            else if (layer == 3)
            {
                return module % 2 == 0;
            }
            else
            {
                return 0;
            }
        }
        else
        {
            return 0;
        }
    }
    else
    {
        return 0;
    }
}

bool SDL::modules::parseIsLower(bool isInvertedx, unsigned int detId)
{
    return (isInvertedx) ? !(detId & 1) : (detId & 1);
}
bool SDL::modules::parseIsLower(unsigned int index)
{
    return (isInverted[index]) ? !(detIds[index] & 1) : (detIds[index] & 1);
}


unsigned int SDL::modules::parsePartnerModuleId(unsigned int detId, bool isLowerx, bool isInvertedx)
{
    return isLowerx ? (isInvertedx ? detId - 1 : detId + 1) : (isInvertedx ? detId + 1 : detId - 1);
}

SDL::ModuleType SDL::modules::parseModuleType(short subdet, short layer, short ring)
{
    if(subdet == Barrel)
    {
        if(layer <= 3)
        {
            return PS;
        }
        else
        {
            return TwoS;
        }
    }
    else
    {
        if(layer <= 2)
        {
            if(ring <= 10)
            {
                return PS;
            }
            else
            {
                return TwoS;
            }
        }
        else
        {
            if(ring <= 7)
            {
                return PS;
            }
            else
            {
                return TwoS;
            }
        }
    }
}
SDL::ModuleType SDL::modules::parseModuleType(unsigned int index)
{
    if(subdets[index] == Barrel)
    {
        if(layers[index] <= 3)
        {
            return PS;
        }
        else
        {
            return TwoS;
        }
    }
    else
    {
        if(layers[index] <= 2)
        {
            if(rings[index] <= 10)
            {
                return PS;
            }
            else
            {
                return TwoS;
            }
        }
        else
        {
            if(rings[index] <= 7)
            {
                return PS;
            }
            else
            {
                return TwoS;
            }
        }
    }
}

SDL::ModuleLayerType SDL::modules::parseModuleLayerType(ModuleType moduleTypex,bool isInvertedx, bool isLowerx)
{
    if(moduleTypex == TwoS)
    {
        return Strip;
    }
    if(isInvertedx)
    {
        if(isLowerx)
        {
            return Strip;
        }
        else
        {
            return Pixel;
        }
    }
    else
   {
        if(isLowerx)
        {
            return Pixel;
        }
        else
        {
            return Strip;
        }
    }
}
SDL::ModuleLayerType SDL::modules::parseModuleLayerType(unsigned int index)
{
    if(moduleType[index] == TwoS)
    {
        return Strip;
    }
    if(isInverted[index])
    {
        if(isLower[index])
        {
            return Strip;
        }
        else
        {
            return Pixel;
        }
    }
    else
   {
        if(isLower[index])
        {
            return Pixel;
        }
        else
        {
            return Strip;
        }
    }
}

void SDL::resetObjectRanges(struct objectRanges& rangesInGPU, unsigned int nModules,hipStream_t stream)
{
//#ifdef Explicit_Module
        hipMemsetAsync(rangesInGPU.hitRanges, -1,nModules*2*sizeof(int),stream);
        hipMemsetAsync(rangesInGPU.hitRangesLower, -1,nModules*sizeof(int),stream);
        hipMemsetAsync(rangesInGPU.hitRangesUpper, -1,nModules*sizeof(int),stream);
        hipMemsetAsync(rangesInGPU.hitRangesnLower, -1,nModules*sizeof(int8_t),stream);
        hipMemsetAsync(rangesInGPU.hitRangesnUpper, -1,nModules*sizeof(int8_t),stream);
        hipMemsetAsync(rangesInGPU.mdRanges, -1,nModules*2*sizeof(int),stream);
        hipMemsetAsync(rangesInGPU.segmentRanges, -1,nModules*2*sizeof(int),stream);
        hipMemsetAsync(rangesInGPU.trackletRanges, -1,nModules*2*sizeof(int),stream);
        hipMemsetAsync(rangesInGPU.tripletRanges, -1,nModules*2*sizeof(int),stream);
        hipMemsetAsync(rangesInGPU.trackCandidateRanges, -1,nModules*2*sizeof(int),stream);
        hipMemsetAsync(rangesInGPU.quintupletRanges, -1, nModules*2*sizeof(int),stream);
        hipStreamSynchronize(stream);
}


/*
   Find the modules that are "staggered neighbours" to a given module
   
   Both barrel central : same module, adjacent rod (modulo number of rods) 
   Both barrel tilted :  same rod, adjacent module (modulo number of modules, which is the same as number of rods for flat)
   One Barrel flat, One Barrel tilted : 
      Left and Center : Left module's rod = 12, center module's module = 1, and left module's module = center module's rod
      Center and Right : Right module's rod = 1 center module's module = max module number, and right module's module = center module's rod

   Endcap : Same ring -> Adjacent modules,
   TODO: Endcap different ring, and tilted->endcap
*/


__device__ const int nEndcapModulesInner[] = {20,24,24,28,32,32,36,40,40,44,52,60,64,72,76};
__device__ const int nEndcapModulesOuter[] = {28,28,32,36,36,40,44,52,56,64,72,76};

__device__ const int nCentralBarrelModules[] = {7,11,15,24,24,24};
__device__ const int nCentralRods[] = {18, 26, 36, 48, 60, 78};

__device__ void findStaggeredNeighbours(struct SDL::modules& modulesInGPU, unsigned int moduleIdx, unsigned int* staggeredNeighbours, unsigned int& counter)
{
    //naive and expensive method
    counter = 0;
    bool flag = false;
    for(size_t i = 0; i < *(modulesInGPU.nLowerModules); i++)
    {
        flag = false;
        unsigned int partnerModuleIdx = i;
        //start
        unsigned int layer1 = modulesInGPU.layers[moduleIdx];
        unsigned int layer2 = modulesInGPU.layers[partnerModuleIdx];
        unsigned int module1 = modulesInGPU.modules[moduleIdx];
        unsigned int module2 = modulesInGPU.modules[partnerModuleIdx];

        if(layer1 != layer2) continue;

        if(modulesInGPU.subdets[moduleIdx] == 4 and modulesInGPU.subdets[partnerModuleIdx] == 4)
        {
            unsigned int ring1 = modulesInGPU.rings[moduleIdx];
            unsigned int ring2 = modulesInGPU.rings[partnerModuleIdx];
            if(ring1 != ring2) continue;

            if((layer1 <=2) and (fabsf(module1 - module2) == 1 or fabsf(module1 % nEndcapModulesInner[ring1 - 1] - module2 % nEndcapModulesInner[ring2 - 1]) == 1))
            {
                flag = true;
            }

            else if((layer1 > 2) and (fabsf(module1 - module2) == 1 or fabsf(module1 % nEndcapModulesOuter[ring1 - 1] - module2 % nEndcapModulesOuter[ring2 - 1]) == 1))
            {
                flag = true;
            }
        }
        else if(modulesInGPU.subdets[moduleIdx] == 5 and modulesInGPU.subdets[partnerModuleIdx] == 5)
        {
            unsigned int rod1 = modulesInGPU.rods[moduleIdx];
            unsigned int rod2 = modulesInGPU.rods[partnerModuleIdx];
            unsigned int side1 = modulesInGPU.sides[moduleIdx];
            unsigned int side2 = modulesInGPU.sides[partnerModuleIdx];
            

            if(side1 == side2)             
            {
                if((fabsf(rod1 - rod2) == 1 and module1 == module2) or (fabsf(module1 - module2) == 1 and rod1 == rod2))
                {
                    flag = true;
                }
                else if(side1 == 3 and side2 == 3 and fabsf(rod1 % nCentralRods[layer1 - 1] - rod2 % nCentralRods[layer2 - 1]) == 1 and module1 == module2)
                {
                    flag = true;
                }
                else if(side1 != 3 and  fabsf(module1 % nCentralRods[layer1 - 1] - module2 % nCentralRods[layer2 - 1]) == 1 and rod1 == rod2)
                {
                    flag = true;
                }
            }
            else
            {
                if(side1 == 1 and side2 == 3 and rod1 == 12 and module2 == 1 and module1 == rod2)
                {
                    flag = true;
                }
                else if(side1 == 3 and side2 == 1 and rod2 == 12 and module1 == 1 and module1 == rod2)
                {
                    flag = true;
                }
                else if(side1 == 2 and side2 == 3 and rod1 == 1 and module2 == nCentralBarrelModules[layer2 - 1] and module1 == rod2)
                {
                    flag = true;
                }
                else if(side1 == 3 and side2 == 2 and module1 == nCentralBarrelModules[layer1 - 1] and rod2 == 1 and rod1 == module2)
                {
                    flag = true;
                }
            }
        }
        if(flag)
        {
            staggeredNeighbours[counter] = i;//deal in lower module indices
            counter++;
        }
    }
}
