#include "hip/hip_runtime.h"
#include "Event.cuh"
#include "allocate.h"

struct SDL::modules* SDL::modulesInGPU = nullptr;
struct SDL::pixelMap* SDL::pixelMapping = nullptr;
uint16_t SDL::nModules;
uint16_t SDL::nLowerModules;

SDL::Event::Event(hipStream_t estream)
{
    int version;
    int driver;
    hipRuntimeGetVersion(&version);
    hipDriverGetVersion(&driver);
    //printf("version: %d Driver %d\n",version, driver);
    stream = estream;
    hitsInGPU = nullptr;
    mdsInGPU = nullptr;
    segmentsInGPU = nullptr;
    tripletsInGPU = nullptr;
    quintupletsInGPU = nullptr;
    trackCandidatesInGPU = nullptr;
    pixelTripletsInGPU = nullptr;
    pixelQuintupletsInGPU = nullptr;
    trackExtensionsInGPU = nullptr;
    rangesInGPU = nullptr;

    hitsInCPU = nullptr;
    rangesInCPU = nullptr;
    mdsInCPU = nullptr;
    segmentsInCPU = nullptr;
    tripletsInCPU = nullptr;
    trackCandidatesInCPU = nullptr;
    modulesInCPU = nullptr;
    modulesInCPUFull = nullptr;
    quintupletsInCPU = nullptr;
    pixelTripletsInCPU = nullptr;
    pixelQuintupletsInCPU = nullptr;
    trackExtensionsInCPU = nullptr;

    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_triplets_by_layer_barrel_[i] = 0;
        n_trackCandidates_by_layer_barrel_[i] = 0;
        n_quintuplets_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
            n_segments_by_layer_endcap_[i] = 0;
            n_triplets_by_layer_endcap_[i] = 0;
            n_trackCandidates_by_layer_endcap_[i] = 0;
            n_quintuplets_by_layer_endcap_[i] = 0;
        }
    }
    //resetObjectsInModule();

}

SDL::Event::~Event()
{
#ifdef CACHE_ALLOC
    if(rangesInGPU){rangesInGPU->freeMemoryCache();}
    if(hitsInGPU){hitsInGPU->freeMemoryCache();}
    if(mdsInGPU){mdsInGPU->freeMemoryCache();}
    if(segmentsInGPU){segmentsInGPU->freeMemoryCache();}
    if(tripletsInGPU){tripletsInGPU->freeMemoryCache();}
    if(quintupletsInGPU){quintupletsInGPU->freeMemoryCache();}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemoryCache();}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemoryCache();}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemoryCache();}
    if(trackExtensionsInGPU){trackExtensionsInGPU->freeMemoryCache();}
#else

    if(rangesInGPU){rangesInGPU->freeMemory();}
    if(hitsInGPU){hitsInGPU->freeMemory();}
    if(mdsInGPU){mdsInGPU->freeMemory(stream);}
    if(segmentsInGPU){segmentsInGPU->freeMemory(stream);}
    if(tripletsInGPU){tripletsInGPU->freeMemory(stream);}
    if(quintupletsInGPU){quintupletsInGPU->freeMemory(stream);}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemory(stream);}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemory(stream);}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemory(stream);}
    if(trackExtensionsInGPU){trackExtensionsInGPU->freeMemory(stream);}
#endif
    if(rangesInGPU != nullptr){cms::cuda::free_host(rangesInGPU);}
    if(mdsInGPU != nullptr){cms::cuda::free_host(mdsInGPU);}
    if(segmentsInGPU!= nullptr){cms::cuda::free_host(segmentsInGPU);}
    if(tripletsInGPU!= nullptr){cms::cuda::free_host(tripletsInGPU);}
    if(trackCandidatesInGPU!= nullptr){cms::cuda::free_host(trackCandidatesInGPU);}
    if(hitsInGPU!= nullptr){cms::cuda::free_host(hitsInGPU);}
    if(pixelTripletsInGPU!= nullptr){cms::cuda::free_host(pixelTripletsInGPU);}
    if(pixelQuintupletsInGPU!= nullptr){cms::cuda::free_host(pixelQuintupletsInGPU);}
    if(quintupletsInGPU!= nullptr){cms::cuda::free_host(quintupletsInGPU);}
    if(trackExtensionsInGPU != nullptr){cms::cuda::free_host(trackExtensionsInGPU);}

    if(hitsInCPU != nullptr)
    {
        delete[] hitsInCPU->idxs;
        delete[] hitsInCPU->xs;
        delete[] hitsInCPU->ys;
        delete[] hitsInCPU->zs;
        delete[] hitsInCPU->moduleIndices;
        delete hitsInCPU->nHits;
        delete hitsInCPU;
    }
    if(rangesInCPU != nullptr)
    {
        delete[] rangesInCPU->quintupletModuleIndices;
        delete rangesInCPU;
    }

    if(mdsInCPU != nullptr)
    {
        delete[] mdsInCPU->anchorHitIndices;
        delete[] mdsInCPU->nMDs;
        delete mdsInCPU->nMemoryLocations;
        delete[] mdsInCPU->totOccupancyMDs;
        delete mdsInCPU;
    }

    if(segmentsInCPU != nullptr)
    {
        delete[] segmentsInCPU->mdIndices;
        delete[] segmentsInCPU->nSegments;
        delete[] segmentsInCPU->totOccupancySegments;
        delete[] segmentsInCPU->innerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->outerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->ptIn;
        delete[] segmentsInCPU->eta;
        delete[] segmentsInCPU->phi;
        delete segmentsInCPU->nMemoryLocations;
        delete segmentsInCPU;
    }

    if(tripletsInCPU != nullptr)
    {
        delete[] tripletsInCPU->segmentIndices;
        delete[] tripletsInCPU->nTriplets;
        delete[] tripletsInCPU->totOccupancyTriplets;
        delete[] tripletsInCPU->betaIn;
        delete[] tripletsInCPU->betaOut;
        delete[] tripletsInCPU->pt_beta;
        delete[] tripletsInCPU->hitIndices;
        delete[] tripletsInCPU->logicalLayers;
        delete[] tripletsInCPU->lowerModuleIndices;
        delete tripletsInCPU->nMemoryLocations;
#ifdef CUT_VALUE_DEBUG
        delete[] tripletsInCPU->zOut;
        delete[] tripletsInCPU->zLo;
        delete[] tripletsInCPU->zHi;
        delete[] tripletsInCPU->zLoPointed;
        delete[] tripletsInCPU->zHiPointed;
        delete[] tripletsInCPU->sdlCut;
        delete[] tripletsInCPU->betaInCut;
        delete[] tripletsInCPU->betaOutCut;
        delete[] tripletsInCPU->deltaBetaCut;
        delete[] tripletsInCPU->rtLo;
        delete[] tripletsInCPU->rtHi;
        delete[] tripletsInCPU->kZ;
#endif
        delete tripletsInCPU;
    }
#ifdef FINAL_T5
    if(quintupletsInCPU != nullptr)
    {
        delete[] quintupletsInCPU->tripletIndices;
        delete[] quintupletsInCPU->nQuintuplets;
        delete[] quintupletsInCPU->totOccupancyQuintuplets;
        delete[] quintupletsInCPU->lowerModuleIndices;
        delete[] quintupletsInCPU->innerRadius;
        delete[] quintupletsInCPU->outerRadius;
        delete[] quintupletsInCPU->regressionRadius;
#ifdef CUT_VALUE_DEBUG
        delete[] quintupletsInCPU->innerRadiusMin;
        delete[] quintupletsInCPU->innerRadiusMin2S;
        delete[] quintupletsInCPU->innerRadiusMax;
        delete[] quintupletsInCPU->innerRadiusMax2S;
        delete[] quintupletsInCPU->bridgeRadius;
        delete[] quintupletsInCPU->bridgeRadiusMin;
        delete[] quintupletsInCPU->bridgeRadiusMin2S;
        delete[] quintupletsInCPU->bridgeRadiusMax;
        delete[] quintupletsInCPU->bridgeRadiusMax2S;
        delete[] quintupletsInCPU->outerRadiusMin;
        delete[] quintupletsInCPU->outerRadiusMin2S;
        delete[] quintupletsInCPU->outerRadiusMax;
        delete[] quintupletsInCPU->outerRadiusMax2S;
        delete[] quintupletsInCPU->chiSquared;
        delete[] quintupletsInCPU->nonAnchorChiSquared;
#endif
        delete quintupletsInCPU;
    }
#endif

    if(pixelTripletsInCPU != nullptr)
    {
        delete[] pixelTripletsInCPU->tripletIndices;
        delete[] pixelTripletsInCPU->pixelSegmentIndices;
        delete[] pixelTripletsInCPU->pixelRadius;
        delete[] pixelTripletsInCPU->tripletRadius;
        delete pixelTripletsInCPU->nPixelTriplets;
        delete pixelTripletsInCPU->totOccupancyPixelTriplets;
#ifdef CUT_VALUE_DEBUG
        delete[] pixelTripletsInCPU->pixelRadiusError;
        delete[] pixelTripletsInCPU->rzChiSquared;
        delete[] pixelTripletsInCPU->rPhiChiSquared;
        delete[] pixelTripletsInCPU->rPhiChiSquaredInwards;
#endif
        delete pixelTripletsInCPU;
    }

    if(pixelQuintupletsInCPU != nullptr)
    {
        delete[] pixelQuintupletsInCPU->pixelIndices;
        delete[] pixelQuintupletsInCPU->T5Indices;
        delete[] pixelQuintupletsInCPU->isDup;
        delete[] pixelQuintupletsInCPU->score;
        delete pixelQuintupletsInCPU->nPixelQuintuplets;
        delete pixelQuintupletsInCPU->totOccupancyPixelQuintuplets;
#ifdef CUT_VALUE_DEBUG
        delete[] pixelQuintupletsInCPU->rzChiSquared;
        delete[] pixelQuintupletsInCPU->rPhiChiSquared;
        delete[] pixelQuintupletsInCPU->rPhiChiSquaredInwards;
#endif
        delete pixelQuintupletsInCPU;
    }

    if(trackCandidatesInCPU != nullptr)
    {
        delete[] trackCandidatesInCPU->objectIndices;
        delete[] trackCandidatesInCPU->trackCandidateType;
        delete[] trackCandidatesInCPU->nTrackCandidates;
        delete[] trackCandidatesInCPU->hitIndices;
        delete[] trackCandidatesInCPU->logicalLayers;
        delete[] trackCandidatesInCPU->partOfExtension;
        delete trackCandidatesInCPU;
    }

    if(trackExtensionsInCPU != nullptr)
    {
        delete[] trackExtensionsInCPU->nTrackExtensions;
        delete[] trackExtensionsInCPU->totOccupancyTrackExtensions;
        delete[] trackExtensionsInCPU->constituentTCTypes;
        delete[] trackExtensionsInCPU->constituentTCIndices;
        delete[] trackExtensionsInCPU->nLayerOverlaps;
        delete[] trackExtensionsInCPU->nHitOverlaps;
        delete[] trackExtensionsInCPU->isDup;
        delete[] trackExtensionsInCPU->regressionRadius;
#ifdef CUT_VALUE_DEBUG
        delete[] trackExtensionsInCPU->rPhiChiSquared;
        delete[] trackExtensionsInCPU->rzChiSquared;
        delete[] trackExtensionsInCPU->innerRadius;
        delete[] trackExtensionsInCPU->outerRadius;
#endif

        delete trackExtensionsInCPU;
    }

    if(modulesInCPU != nullptr)
    {
        delete[] modulesInCPU->nLowerModules;
        delete[] modulesInCPU->nModules;
        delete[] modulesInCPU->detIds;
        delete[] modulesInCPU->isLower;
        delete[] modulesInCPU->layers;
        delete[] modulesInCPU->subdets;
        delete[] modulesInCPU->rings;
        delete[] modulesInCPU->rods;
        delete[] modulesInCPU->modules;
        delete[] modulesInCPU->sides;
        delete[] modulesInCPU->eta;
        delete[] modulesInCPU->r;
        delete[] modulesInCPU;
    }
    if(modulesInCPUFull != nullptr)
    {
        delete[] modulesInCPUFull->detIds;
        delete[] modulesInCPUFull->moduleMap;
        delete[] modulesInCPUFull->nConnectedModules;
        delete[] modulesInCPUFull->drdzs;
        delete[] modulesInCPUFull->slopes;
        delete[] modulesInCPUFull->nModules;
        delete[] modulesInCPUFull->nLowerModules;
        delete[] modulesInCPUFull->layers;
        delete[] modulesInCPUFull->rings;
        delete[] modulesInCPUFull->modules;
        delete[] modulesInCPUFull->rods;
        delete[] modulesInCPUFull->subdets;
        delete[] modulesInCPUFull->sides;
        delete[] modulesInCPUFull->eta;
        delete[] modulesInCPUFull->r;
        delete[] modulesInCPUFull->isInverted;
        delete[] modulesInCPUFull->isLower;


        delete[] modulesInCPUFull->moduleType;
        delete[] modulesInCPUFull->moduleLayerType;
        delete[] modulesInCPUFull;
    }
    SDL::freeEndCapMapMemory();
}

void SDL::Event::resetEvent()
{
#ifdef CACHE_ALLOC
    if(hitsInGPU){hitsInGPU->freeMemoryCache();}
    if(mdsInGPU){mdsInGPU->freeMemoryCache();}
    if(quintupletsInGPU){quintupletsInGPU->freeMemoryCache();}
    if(rangesInGPU){rangesInGPU->freeMemoryCache();}
    if(segmentsInGPU){segmentsInGPU->freeMemoryCache();}
    if(tripletsInGPU){tripletsInGPU->freeMemoryCache();}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemoryCache();}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemoryCache();}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemoryCache();}
    if(trackExtensionsInGPU){trackExtensionsInGPU->freeMemoryCache();}

#else
    if(hitsInGPU){hitsInGPU->freeMemory();}
    if(quintupletsInGPU){quintupletsInGPU->freeMemory(stream);}
    if(rangesInGPU){rangesInGPU->freeMemory();}
    if(mdsInGPU){mdsInGPU->freeMemory(stream);}
    if(segmentsInGPU){segmentsInGPU->freeMemory(stream);}
    if(tripletsInGPU){tripletsInGPU->freeMemory(stream);}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemory(stream);}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemory(stream);}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemory(stream);}
    if(trackExtensionsInGPU){trackExtensionsInGPU->freeMemory(stream);}
#endif
    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_triplets_by_layer_barrel_[i] = 0;
        n_trackCandidates_by_layer_barrel_[i] = 0;
        n_quintuplets_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
            n_segments_by_layer_endcap_[i] = 0;
            n_triplets_by_layer_endcap_[i] = 0;
            n_trackCandidates_by_layer_endcap_[i] = 0;
            n_quintuplets_by_layer_endcap_[i] = 0;
        }
    }
    if(hitsInGPU){cms::cuda::free_host(hitsInGPU);
    hitsInGPU = nullptr;}
    if(mdsInGPU){cms::cuda::free_host(mdsInGPU);
    mdsInGPU = nullptr;}
    if(rangesInGPU){cms::cuda::free_host(rangesInGPU);
    rangesInGPU = nullptr;}
    if(segmentsInGPU){cms::cuda::free_host(segmentsInGPU);
    segmentsInGPU = nullptr;}
    if(tripletsInGPU){cms::cuda::free_host(tripletsInGPU);
    tripletsInGPU = nullptr;}
      if(quintupletsInGPU){cms::cuda::free_host(quintupletsInGPU);
      quintupletsInGPU = nullptr;}
    if(trackCandidatesInGPU){cms::cuda::free_host(trackCandidatesInGPU);
    trackCandidatesInGPU = nullptr;}
    if(pixelTripletsInGPU){cms::cuda::free_host(pixelTripletsInGPU);
    pixelTripletsInGPU = nullptr;}
    if(pixelQuintupletsInGPU){cms::cuda::free_host(pixelQuintupletsInGPU);
    pixelQuintupletsInGPU = nullptr;}
    if(trackExtensionsInGPU){cms::cuda::free_host(trackExtensionsInGPU);
    trackExtensionsInGPU = nullptr;}

    if(hitsInCPU != nullptr)
    {
        delete[] hitsInCPU->idxs;
        delete[] hitsInCPU->xs;
        delete[] hitsInCPU->ys;
        delete[] hitsInCPU->zs;
        delete[] hitsInCPU->moduleIndices;
        delete hitsInCPU->nHits;
        delete hitsInCPU;
        hitsInCPU = nullptr;
    }
    if(rangesInCPU != nullptr)
    {
        delete[] rangesInCPU->hitRanges;
        delete[] rangesInCPU->quintupletModuleIndices;
        delete rangesInCPU;
        rangesInCPU = nullptr;
    }

    if(mdsInCPU != nullptr)
    {
        delete[] mdsInCPU->anchorHitIndices;
        delete[] mdsInCPU->nMDs;
        delete[] mdsInCPU->totOccupancyMDs;
        delete mdsInCPU;
        mdsInCPU = nullptr;
    }

    if(segmentsInCPU != nullptr)
    {
        delete[] segmentsInCPU->mdIndices;
        delete[] segmentsInCPU->nSegments;
        delete[] segmentsInCPU->totOccupancySegments;
        delete[] segmentsInCPU->innerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->outerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->ptIn;
        delete[] segmentsInCPU->eta;
        delete[] segmentsInCPU->phi;
        delete segmentsInCPU;
        segmentsInCPU = nullptr;
    }

    if(tripletsInCPU != nullptr)
    {
        delete[] tripletsInCPU->segmentIndices;
        delete[] tripletsInCPU->nTriplets;
        delete[] tripletsInCPU->totOccupancyTriplets;
        delete[] tripletsInCPU->betaIn;
        delete[] tripletsInCPU->betaOut;
        delete[] tripletsInCPU->pt_beta;
        delete[] tripletsInCPU->logicalLayers;
        delete[] tripletsInCPU->lowerModuleIndices;
        delete[] tripletsInCPU->hitIndices;
        delete tripletsInCPU;
        tripletsInCPU = nullptr;
    }

#ifdef FINAL_T5
    if(quintupletsInCPU != nullptr)
    {
        delete[] quintupletsInCPU->tripletIndices;
        delete[] quintupletsInCPU->nQuintuplets;
        delete[] quintupletsInCPU->totOccupancyQuintuplets;
        delete[] quintupletsInCPU->lowerModuleIndices;
        delete[] quintupletsInCPU->innerRadius;
        delete[] quintupletsInCPU->outerRadius;
        delete[] quintupletsInCPU->regressionRadius;
        delete quintupletsInCPU;
        quintupletsInCPU = nullptr;
    }
#endif

    if(pixelTripletsInCPU != nullptr)
    {
        delete[] pixelTripletsInCPU->tripletIndices;
        delete[] pixelTripletsInCPU->pixelSegmentIndices;
        delete[] pixelTripletsInCPU->pixelRadius;
        delete[] pixelTripletsInCPU->tripletRadius;
        delete pixelTripletsInCPU->nPixelTriplets;
        delete pixelTripletsInCPU->totOccupancyPixelTriplets;
        delete pixelTripletsInCPU;
        pixelTripletsInCPU = nullptr;
    }

    if(pixelQuintupletsInCPU != nullptr)
    {
        delete[] pixelQuintupletsInCPU->pixelIndices;
        delete[] pixelQuintupletsInCPU->T5Indices;
        delete[] pixelQuintupletsInCPU->isDup;
        delete[] pixelQuintupletsInCPU->score;
        delete pixelQuintupletsInCPU->nPixelQuintuplets;
        delete pixelQuintupletsInCPU->totOccupancyPixelQuintuplets;
        delete pixelQuintupletsInCPU;
        pixelQuintupletsInCPU = nullptr;
    }

    if(trackCandidatesInCPU != nullptr)
    {
        delete[] trackCandidatesInCPU->objectIndices;
        delete[] trackCandidatesInCPU->trackCandidateType;
        delete[] trackCandidatesInCPU->nTrackCandidates;
        delete[] trackCandidatesInCPU->logicalLayers;
        delete[] trackCandidatesInCPU->hitIndices;
        delete[] trackCandidatesInCPU->lowerModuleIndices;
        delete trackCandidatesInCPU;
        trackCandidatesInCPU = nullptr;
    }

    if(trackExtensionsInCPU != nullptr)
    {
        delete[] trackExtensionsInCPU->nTrackExtensions;
        delete[] trackExtensionsInCPU->totOccupancyTrackExtensions;
        delete[] trackExtensionsInCPU->constituentTCTypes;
        delete[] trackExtensionsInCPU->constituentTCIndices;
        delete[] trackExtensionsInCPU->nLayerOverlaps;
        delete[] trackExtensionsInCPU->nHitOverlaps;
        delete[] trackExtensionsInCPU->isDup;
        delete[] trackExtensionsInCPU->regressionRadius;

        delete trackExtensionsInCPU;
        trackExtensionsInCPU = nullptr;
    }

    if(modulesInCPU != nullptr)
    {
        delete[] modulesInCPU->nLowerModules;
        delete[] modulesInCPU->nModules;
        delete[] modulesInCPU->detIds;
        delete[] modulesInCPU->isLower;
        delete[] modulesInCPU->layers;
        delete[] modulesInCPU->subdets;
        delete[] modulesInCPU->rings;
        delete[] modulesInCPU->rods;
        delete[] modulesInCPU->modules;
        delete[] modulesInCPU->sides;
        delete[] modulesInCPU->eta;
        delete[] modulesInCPU->r;
        delete[] modulesInCPU;
        modulesInCPU = nullptr;
    }
    if(modulesInCPUFull != nullptr)
    {
        delete[] modulesInCPUFull->detIds;
        delete[] modulesInCPUFull->moduleMap;
        delete[] modulesInCPUFull->nConnectedModules;
        delete[] modulesInCPUFull->drdzs;
        delete[] modulesInCPUFull->slopes;
        delete[] modulesInCPUFull->nModules;
        delete[] modulesInCPUFull->nLowerModules;
        delete[] modulesInCPUFull->layers;
        delete[] modulesInCPUFull->rings;
        delete[] modulesInCPUFull->modules;
        delete[] modulesInCPUFull->rods;
        delete[] modulesInCPUFull->sides;
        delete[] modulesInCPUFull->subdets;
        delete[] modulesInCPUFull->eta;
        delete[] modulesInCPUFull->r;
        delete[] modulesInCPUFull->isInverted;
        delete[] modulesInCPUFull->isLower;


        delete[] modulesInCPUFull->moduleType;
        delete[] modulesInCPUFull->moduleLayerType;
        delete[] modulesInCPUFull;
        modulesInCPUFull = nullptr;
    }

}

void SDL::initModules(const char* moduleMetaDataFilePath)
{
    hipStream_t default_stream = 0;
    if(modulesInGPU == nullptr)
    {
        //modulesInGPU = (SDL::modules*)cms::cuda::allocate_host(sizeof(struct SDL::modules), default_stream);
        hipHostMalloc(&modulesInGPU, sizeof(struct SDL::modules));
        //pixelMapping = new pixelMap;
        hipHostMalloc(&pixelMapping, sizeof(struct SDL::pixelMap));
        //pixelMapping = (SDL::pixelMap*)cms::cuda::allocate_host(sizeof(struct SDL::pixelMap), default_stream);
        //nModules gets filled here
        loadModulesFromFile(*modulesInGPU,nModules,nLowerModules, *pixelMapping, default_stream, moduleMetaDataFilePath);
        hipStreamSynchronize(default_stream);
    }
    //resetObjectRanges(*modulesInGPU,nModules, default_stream);
}


void SDL::cleanModules()
{
  //hipStream_t default_stream = 0;
  //#ifdef CACHE_ALLOC
  //freeModulesCache(*modulesInGPU,*pixelMapping); //bug in freeing cached modules. Decided to remove module caching since it doesn't change by event.
  //#else
    freeModules(*modulesInGPU, *pixelMapping);
  //#endif
    hipHostFree(modulesInGPU);
    hipHostFree(pixelMapping);
    //cms::cuda::free_host(modulesInGPU);
    //cms::cuda::free_host(pixelMapping);
}

void SDL::Event::resetObjectsInModule()
{
    resetObjectRanges(*rangesInGPU,nModules,stream);
}

__device__ int binary_search(
                            unsigned int *data, // Array that we are searching over
                            unsigned int search_val, // Value we want to find in data array
                            unsigned int ndata) // Number of elements in data array
{
    unsigned int low = 0;
    unsigned int high = ndata - 1;

    while(low <= high)
    {
        unsigned int mid = (low + high)/2;
        unsigned int test_val = data[mid];
        if (test_val == search_val)
            return mid;
        else if (test_val > search_val)
            high = mid - 1;
        else
            low = mid + 1;
    }
    // Couldn't find search value in array.
    return -1;
}

__global__ void moduleRangesKernel(uint16_t nLower, struct SDL::modules *modulesInGPU, struct SDL::hits *hitsInGPU)
{
    for ( int lowerIndex = blockIdx.x * blockDim.x + threadIdx.x; lowerIndex < nLower; lowerIndex += blockDim.x*gridDim.x )
    {
        uint16_t upperIndex = modulesInGPU->partnerModuleIndices[lowerIndex];
        if (hitsInGPU->hitRanges[lowerIndex * 2] != -1 && hitsInGPU->hitRanges[upperIndex * 2] != -1)
        {
            hitsInGPU->hitRangesLower[lowerIndex] =  hitsInGPU->hitRanges[lowerIndex * 2]; 
            hitsInGPU->hitRangesUpper[lowerIndex] =  hitsInGPU->hitRanges[upperIndex * 2];
            hitsInGPU->hitRangesnLower[lowerIndex] = hitsInGPU->hitRanges[lowerIndex * 2 + 1] - hitsInGPU->hitRanges[lowerIndex * 2] + 1;
            hitsInGPU->hitRangesnUpper[lowerIndex] = hitsInGPU->hitRanges[upperIndex * 2 + 1] - hitsInGPU->hitRanges[upperIndex * 2] + 1;
        }
    }
}

__global__ void hitLoopKernel(
                            uint16_t Endcap, // Integer corresponding to endcap in module subdets
                            uint16_t TwoS, // Integer corresponding to TwoS in moduleType
                            int nHits, // Total number of hits in event
                            unsigned int nModules, // Number of modules
                            unsigned int nEndCapMap, // Number of elements in endcap map
                            unsigned int* geoMapDetId, // DetId's from endcap map
                            float* geoMapPhi, // Phi values from endcap map
                            struct SDL::modules *modulesInGPU,
                            struct SDL::hits *hitsInGPU)
{
    for( int ihit = blockIdx.x * blockDim.x + threadIdx.x; ihit < nHits; ihit += blockDim.x * gridDim.x )
    {
        float ihit_x = hitsInGPU->xs[ihit];
        float ihit_y = hitsInGPU->ys[ihit];
        float ihit_z = hitsInGPU->zs[ihit];
        int iDetId = hitsInGPU->detid[ihit];

        hitsInGPU->rts[ihit] = sqrt(ihit_x*ihit_x + ihit_y*ihit_y);
        hitsInGPU->phis[ihit] = SDL::phi(ihit_x,ihit_y,ihit_z);
        hitsInGPU->etas[ihit] = ((ihit_z>0)-(ihit_z<0))*acosh(sqrt(ihit_x*ihit_x+ihit_y*ihit_y+ihit_z*ihit_z)/hitsInGPU->rts[ihit]);

        int found_index = binary_search(modulesInGPU->mapdetId, iDetId, nModules);
        uint16_t lastModuleIndex = modulesInGPU->mapIdx[found_index];

        hitsInGPU->moduleIndices[ihit] = lastModuleIndex;

        if(modulesInGPU->subdets[lastModuleIndex] == Endcap && modulesInGPU->moduleType[lastModuleIndex] == TwoS)
        {
            int found_index = binary_search(geoMapDetId, iDetId, nEndCapMap);
            float phi = 0;
            // Unclear why these are not in map, but CPU map returns phi = 0 for all exceptions.
            if (found_index != -1)
                phi = geoMapPhi[found_index];
            float cos_phi = cosf(phi);
            hitsInGPU->highEdgeXs[ihit] = ihit_x + 2.5f * cos_phi;
            hitsInGPU->lowEdgeXs[ihit] = ihit_x - 2.5f * cos_phi;
            float sin_phi = sinf(phi);
            hitsInGPU->highEdgeYs[ihit] = ihit_y + 2.5f * sin_phi;
            hitsInGPU->lowEdgeYs[ihit] = ihit_y - 2.5f * sin_phi;
        }
        // Need to set initial value if index hasn't been seen before.
        int old = atomicCAS(&(hitsInGPU->hitRanges[lastModuleIndex * 2]), -1, ihit);
        // For subsequent visits, stores the min value.
        if (old != -1)
            atomicMin(&hitsInGPU->hitRanges[lastModuleIndex * 2], ihit);

        atomicMax(&hitsInGPU->hitRanges[lastModuleIndex * 2 + 1], ihit);
    }
}

void SDL::Event::addHitToEvent(std::vector<float> x, std::vector<float> y, std::vector<float> z, std::vector<unsigned int> detId, std::vector<unsigned int> idxInNtuple)
{
    // Use the actual number of hits instead of a max.
    const int nHits = x.size();

//    for (int i=0; i<25; i++) printf("%f \n",x[i]);

    // Get current device for future use.
    hipGetDevice(&dev);

    // Initialize space on device/host for next event.
    if (hitsInGPU == nullptr)
    {
        hitsInGPU = (SDL::hits*)cms::cuda::allocate_host(sizeof(SDL::hits), stream);
        // Unclear why but this has to be 2*nHits to avoid crashing.
        createHitsInExplicitMemory(*hitsInGPU, nModules, 2*nHits, stream, 1);
    }
    if (rangesInGPU == nullptr)
    {
        rangesInGPU = (SDL::objectRanges*)cms::cuda::allocate_host(sizeof(SDL::objectRanges), stream);
    	createRangesInExplicitMemory(*rangesInGPU, nModules, stream, nLowerModules);
        resetObjectsInModule();
    }
    hipStreamSynchronize(stream);
    // Copy the host arrays to the GPU.
    hipMemcpyAsync(hitsInGPU->xs, &x[0], nHits*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitsInGPU->ys, &y[0], nHits*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitsInGPU->zs, &z[0], nHits*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitsInGPU->detid, &detId[0], nHits*sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitsInGPU->idxs, &idxInNtuple[0], nHits*sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitsInGPU->nHits, &nHits, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);
    // Calculate secondary variables on the GPU.
    hitLoopKernel<<<MAX_BLOCKS,256,0,stream>>>(
                                            Endcap,
                                            TwoS,
                                            nHits,
                                            nModules,
                                            SDL::endcapGeometry.nEndCapMap,
                                            SDL::endcapGeometry.geoMapDetId,
                                            SDL::endcapGeometry.geoMapPhi,
                                            modulesInGPU,
                                            hitsInGPU);
    //std::cout << hipGetLastError() << std::endl;
    hipStreamSynchronize(stream);

    // No stream synchronize needed after second kernel call. Saves ~100 us.
    // This is because addPixelSegmentToEvent (which is run next) doesn't rely on hitsinGPU->hitrange variables.
    // Also, modulesInGPU->partnerModuleIndices is not alterned in addPixelSegmentToEvent.
    moduleRangesKernel<<<MAX_BLOCKS,256,0,stream>>>(nLowerModules, modulesInGPU, hitsInGPU);
    //std::cout << hipGetLastError() << std::endl;
}

__global__ void addPixelSegmentToEventKernel(unsigned int* hitIndices0,unsigned int* hitIndices1,unsigned int* hitIndices2,unsigned int* hitIndices3, float* dPhiChange, float* ptIn, float* ptErr, float* px, float* py, float* pz, float* eta, float* etaErr,float* phi, float* charge, uint16_t pixelModuleIndex, struct SDL::modules& modulesInGPU, struct SDL::objectRanges& rangesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU,const int size, int* superbin, int8_t* pixelType, short* isQuad)
{
    for( int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < size; tid += blockDim.x*gridDim.x)
    {

      unsigned int innerMDIndex = rangesInGPU.miniDoubletModuleIndices[pixelModuleIndex] + 2*(tid);
      unsigned int outerMDIndex = rangesInGPU.miniDoubletModuleIndices[pixelModuleIndex] + 2*(tid) +1;
      unsigned int pixelSegmentIndex = rangesInGPU.segmentModuleIndices[pixelModuleIndex] + tid;

      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,outerMDIndex);

    //in outer hits - pt, eta, phi
    float slope = sinhf(hitsInGPU.ys[mdsInGPU.outerHitIndices[innerMDIndex]]);
    float intercept = hitsInGPU.zs[mdsInGPU.anchorHitIndices[innerMDIndex]] - slope * hitsInGPU.rts[mdsInGPU.anchorHitIndices[innerMDIndex]];
    float score_lsq=(hitsInGPU.rts[mdsInGPU.anchorHitIndices[outerMDIndex]] * slope + intercept) - (hitsInGPU.zs[mdsInGPU.anchorHitIndices[outerMDIndex]]);
    score_lsq = score_lsq * score_lsq;

    unsigned int hits1[4];
    hits1[0] = hitsInGPU.idxs[mdsInGPU.anchorHitIndices[innerMDIndex]];
    hits1[1] = hitsInGPU.idxs[mdsInGPU.anchorHitIndices[outerMDIndex]];
    hits1[2] = hitsInGPU.idxs[mdsInGPU.outerHitIndices[innerMDIndex]];
    hits1[3] = hitsInGPU.idxs[mdsInGPU.outerHitIndices[outerMDIndex]];
    addPixelSegmentToMemory(segmentsInGPU, mdsInGPU, modulesInGPU, innerMDIndex, outerMDIndex, pixelModuleIndex, hits1, hitIndices0[tid], hitIndices2[tid], dPhiChange[tid], ptIn[tid], ptErr[tid], px[tid], py[tid], pz[tid], etaErr[tid], eta[tid], phi[tid], charge[tid], pixelSegmentIndex, tid, superbin[tid], pixelType[tid],isQuad[tid],score_lsq);
    }
}
void SDL::Event::addPixelSegmentToEvent(std::vector<unsigned int> hitIndices0,std::vector<unsigned int> hitIndices1,std::vector<unsigned int> hitIndices2,std::vector<unsigned int> hitIndices3, std::vector<float> dPhiChange, std::vector<float> ptIn, std::vector<float> ptErr, std::vector<float> px, std::vector<float> py, std::vector<float> pz, std::vector<float> eta, std::vector<float> etaErr, std::vector<float> phi, std::vector<float> charge, std::vector<int> superbin, std::vector<int8_t> pixelType, std::vector<short> isQuad)
{
    if(mdsInGPU == nullptr)
    {
        mdsInGPU = (SDL::miniDoublets*)cms::cuda::allocate_host(sizeof(SDL::miniDoublets), stream);
        //hardcoded range numbers for this will come from studies!
        unsigned int nTotalMDs;
        createMDArrayRanges(*modulesInGPU, *rangesInGPU, nLowerModules, nTotalMDs, stream, N_MAX_MD_PER_MODULES, N_MAX_PIXEL_MD_PER_MODULES);

    	createMDsInExplicitMemory(*mdsInGPU, nTotalMDs, nLowerModules, N_MAX_PIXEL_MD_PER_MODULES,stream);

        hipMemcpyAsync(mdsInGPU->nMemoryLocations, &nTotalMDs, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream);

    }
    if(segmentsInGPU == nullptr)
    {
        segmentsInGPU = (SDL::segments*)cms::cuda::allocate_host(sizeof(SDL::segments), stream);
        //hardcoded range numbers for this will come from studies!
        unsigned int nTotalSegments;
        createSegmentArrayRanges(*modulesInGPU, *rangesInGPU, *mdsInGPU, nLowerModules, nTotalSegments, stream, N_MAX_SEGMENTS_PER_MODULE, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
//        cout<<"nTotalSegments: "<<nTotalSegments<<std::endl; // for memory usage

        createSegmentsInExplicitMemory(*segmentsInGPU, nTotalSegments, nLowerModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE,stream);

        hipMemcpyAsync(segmentsInGPU->nMemoryLocations, &nTotalSegments, sizeof(unsigned int), hipMemcpyHostToDevice, stream);;
        hipStreamSynchronize(stream);

    }
    hipStreamSynchronize(stream);
    const int size = ptIn.size();
    uint16_t pixelModuleIndex = (*detIdToIndex)[1];
    unsigned int* hitIndices0_host = &hitIndices0[0];
    unsigned int* hitIndices1_host = &hitIndices1[0];
    unsigned int* hitIndices2_host = &hitIndices2[0];
    unsigned int* hitIndices3_host = &hitIndices3[0];
    float* dPhiChange_host = &dPhiChange[0];
    float* ptIn_host = &ptIn[0];
    float* ptErr_host = &ptErr[0];
    float* px_host = &px[0];
    float* py_host = &py[0];
    float* pz_host = &pz[0];
    float* etaErr_host = &etaErr[0];
    float* eta_host = &eta[0];
    float* phi_host = &phi[0];
    float* charge_host = &charge[0];
    int* superbin_host = &superbin[0];
    int8_t* pixelType_host = &pixelType[0];
    short* isQuad_host = &isQuad[0];

    unsigned int* hitIndices0_dev;
    unsigned int* hitIndices1_dev;
    unsigned int* hitIndices2_dev;
    unsigned int* hitIndices3_dev;
    float* dPhiChange_dev;
    float* ptIn_dev;
    float* ptErr_dev;
    float* px_dev;
    float* py_dev;
    float* pz_dev;
    float* etaErr_dev;
    float* eta_dev;
    float* phi_dev;
    float* charge_dev;
    int* superbin_dev;
    int8_t* pixelType_dev;
    short* isQuad_dev;
    hitIndices0_dev = (unsigned int*)cms::cuda::allocate_device(dev, size*sizeof(unsigned int), stream);
    hitIndices1_dev = (unsigned int*)cms::cuda::allocate_device(dev, size*sizeof(unsigned int), stream);
    hitIndices2_dev = (unsigned int*)cms::cuda::allocate_device(dev, size*sizeof(unsigned int), stream);
    hitIndices3_dev = (unsigned int*)cms::cuda::allocate_device(dev, size*sizeof(unsigned int), stream);
    dPhiChange_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);
    ptIn_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);
    ptErr_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);
    px_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);
    py_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);
    pz_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);
    etaErr_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);
    eta_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);
    phi_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);
    charge_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);
    superbin_dev = (int*)cms::cuda::allocate_device(dev, size*sizeof(int), stream);
    pixelType_dev = (int8_t*)cms::cuda::allocate_device(dev, size*sizeof(int8_t), stream);
    isQuad_dev = (short*)cms::cuda::allocate_device(dev, size*sizeof(short), stream);

    hipMemcpyAsync(hitIndices0_dev,hitIndices0_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitIndices1_dev,hitIndices1_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitIndices2_dev,hitIndices2_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitIndices3_dev,hitIndices3_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(dPhiChange_dev,dPhiChange_host,size*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(ptIn_dev,ptIn_host,size*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(ptErr_dev,ptErr_host,size*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(px_dev,px_host,size*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(py_dev,py_host,size*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(pz_dev,pz_host,size*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(etaErr_dev,etaErr_host,size*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(eta_dev, eta_host, size*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(phi_dev, phi_host, size*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(charge_dev, charge_host, size*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(superbin_dev,superbin_host,size*sizeof(int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(pixelType_dev,pixelType_host,size*sizeof(int8_t),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(isQuad_dev,isQuad_host,size*sizeof(short),hipMemcpyHostToDevice,stream);

    hipStreamSynchronize(stream);
    unsigned int nThreads = 256;
    unsigned int nBlocks =  MAX_BLOCKS;//size % nThreads == 0 ? size/nThreads : size/nThreads + 1;

    addPixelSegmentToEventKernel<<<nBlocks,nThreads,0,stream>>>(hitIndices0_dev,hitIndices1_dev,hitIndices2_dev,hitIndices3_dev,dPhiChange_dev,ptIn_dev,ptErr_dev,px_dev,py_dev,pz_dev,eta_dev, etaErr_dev, phi_dev, charge_dev, pixelModuleIndex, *modulesInGPU, *rangesInGPU, *hitsInGPU,*mdsInGPU,*segmentsInGPU,size, superbin_dev, pixelType_dev,isQuad_dev);

   //hipDeviceSynchronize();
   hipStreamSynchronize(stream);
   hipMemcpyAsync(&(segmentsInGPU->nSegments)[pixelModuleIndex], &size, sizeof(unsigned int), hipMemcpyHostToDevice,stream);
   hipMemcpyAsync(&(segmentsInGPU->totOccupancySegments)[pixelModuleIndex], &size, sizeof(unsigned int), hipMemcpyHostToDevice,stream);
   unsigned int mdSize = 2 * size;
   hipMemcpyAsync(&(mdsInGPU->nMDs)[pixelModuleIndex], &mdSize, sizeof(unsigned int), hipMemcpyHostToDevice,stream);
   hipMemcpyAsync(&(mdsInGPU->totOccupancyMDs)[pixelModuleIndex], &mdSize, sizeof(unsigned int), hipMemcpyHostToDevice,stream);
   hipStreamSynchronize(stream);
  
    cms::cuda::free_device(dev, hitIndices0_dev);
    cms::cuda::free_device(dev, hitIndices1_dev);
    cms::cuda::free_device(dev, hitIndices2_dev);
    cms::cuda::free_device(dev, hitIndices3_dev);
    cms::cuda::free_device(dev, dPhiChange_dev);
    cms::cuda::free_device(dev, ptIn_dev);
    cms::cuda::free_device(dev, ptErr_dev);
    cms::cuda::free_device(dev, px_dev);
    cms::cuda::free_device(dev, py_dev);
    cms::cuda::free_device(dev, pz_dev);
    cms::cuda::free_device(dev, etaErr_dev);
    cms::cuda::free_device(dev, eta_dev);
    cms::cuda::free_device(dev, phi_dev);
    cms::cuda::free_device(dev, superbin_dev);
    cms::cuda::free_device(dev, pixelType_dev);
    cms::cuda::free_device(dev, isQuad_dev);
    hipStreamSynchronize(stream);
}

void SDL::Event::addMiniDoubletsToEvent()
{
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        if(mdsInGPU->nMDs[i] == 0 or hitsInGPU->hitRanges[i * 2] == -1)
        {
            rangesInGPU->mdRanges[i * 2] = -1;
            rangesInGPU->mdRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU->mdRanges[i * 2] = rangesInGPU->miniDoubletModuleIndices[i];
            rangesInGPU->mdRanges[i * 2 + 1] = rangesInGPU->miniDoubletModuleIndices[i] + mdsInGPU->nMDs[i] - 1;

            if(modulesInGPU->subdets[i] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[modulesInGPU->layers[i] -1] += mdsInGPU->nMDs[i];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[modulesInGPU->layers[i] - 1] += mdsInGPU->nMDs[i];
            }

        }
    }
}
void SDL::Event::addMiniDoubletsToEventExplicit()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    unsigned int* nMDsCPU;
    nMDsCPU = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(nMDsCPU,mdsInGPU->nMDs,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    short* module_subdets;
    module_subdets = (short*)cms::cuda::allocate_host(nLowerModules* sizeof(short), stream);
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_mdRanges;
    module_mdRanges = (int*)cms::cuda::allocate_host(nLowerModules* 2*sizeof(int), stream);
    hipMemcpyAsync(module_mdRanges,rangesInGPU->mdRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    module_layers = (short*)cms::cuda::allocate_host(nLowerModules * sizeof(short), stream);
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_hitRanges;
    module_hitRanges = (int*)cms::cuda::allocate_host(nLowerModules* 2*sizeof(int), stream);
    hipMemcpyAsync(module_hitRanges,hitsInGPU->hitRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);

    int* module_miniDoubletModuleIndices;
    module_miniDoubletModuleIndices = (int*)cms::cuda::allocate_host(nLowerModules * sizeof(int), stream);
    hipMemcpyAsync(module_miniDoubletModuleIndices, rangesInGPU->miniDoubletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        if(nMDsCPU[i] == 0 or module_hitRanges[i * 2] == -1)
        {
            module_mdRanges[i * 2] = -1;
            module_mdRanges[i * 2 + 1] = -1;
        }
        else
        {
            module_mdRanges[i * 2] = module_miniDoubletModuleIndices[i] ;
            module_mdRanges[i * 2 + 1] = module_miniDoubletModuleIndices[i] + nMDsCPU[i] - 1;

            if(module_subdets[i] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[module_layers[i] -1] += nMDsCPU[i];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[module_layers[i] - 1] += nMDsCPU[i];
            }

        }
    }
    hipMemcpyAsync(rangesInGPU->mdRanges,module_mdRanges,nLowerModules*2*sizeof(int),hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);
    cms::cuda::free_host(nMDsCPU);
    cms::cuda::free_host(module_subdets);
    cms::cuda::free_host(module_mdRanges);
    cms::cuda::free_host(module_layers);
    cms::cuda::free_host(module_hitRanges);
    cms::cuda::free_host(module_miniDoubletModuleIndices);
}
void SDL::Event::addSegmentsToEvent()
{
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        if(segmentsInGPU->nSegments[i] == 0)
        {
            rangesInGPU->segmentRanges[i * 2] = -1;
            rangesInGPU->segmentRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU->segmentRanges[i * 2] = rangesInGPU->segmentModuleIndices[i];
            rangesInGPU->segmentRanges[i * 2 + 1] = rangesInGPU->segmentModuleIndices[i] + segmentsInGPU->nSegments[i] - 1;

            if(modulesInGPU->subdets[i] == Barrel)
            {

                n_segments_by_layer_barrel_[modulesInGPU->layers[i] - 1] += segmentsInGPU->nSegments[i];
            }
            else
            {
                n_segments_by_layer_endcap_[modulesInGPU->layers[i] -1] += segmentsInGPU->nSegments[i];
            }
        }
    }
}
void SDL::Event::addSegmentsToEventExplicit()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    unsigned int* nSegmentsCPU;
    nSegmentsCPU = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(nSegmentsCPU,segmentsInGPU->nSegments,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);

    short* module_subdets;
    module_subdets = (short*)cms::cuda::allocate_host(nLowerModules* sizeof(short), stream);
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_segmentRanges;
    module_segmentRanges = (int*)cms::cuda::allocate_host(nLowerModules* 2*sizeof(int), stream);
    hipMemcpyAsync(module_segmentRanges,rangesInGPU->segmentRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    module_layers = (short*)cms::cuda::allocate_host(nLowerModules * sizeof(short), stream);
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);

    int* module_segmentModuleIndices;
    module_segmentModuleIndices = (int*)cms::cuda::allocate_host(nLowerModules * sizeof(int), stream);
    hipMemcpyAsync(module_segmentModuleIndices, rangesInGPU->segmentModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        if(nSegmentsCPU[i] == 0)
        {
            module_segmentRanges[i * 2] = -1;
            module_segmentRanges[i * 2 + 1] = -1;
        }
        else
        {
            module_segmentRanges[i * 2] = module_segmentModuleIndices[i];
            module_segmentRanges[i * 2 + 1] = module_segmentModuleIndices[i] + nSegmentsCPU[i] - 1;

            if(module_subdets[i] == Barrel)
            {
                n_segments_by_layer_barrel_[module_layers[i] - 1] += nSegmentsCPU[i];
            }
            else
            {
                n_segments_by_layer_endcap_[module_layers[i] -1] += nSegmentsCPU[i];
            }
        }
    }
    hipMemcpyAsync(rangesInGPU->segmentRanges, module_segmentRanges, nLowerModules * 2 * sizeof(int), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);

    cms::cuda::free_host(nSegmentsCPU);
    cms::cuda::free_host(module_subdets);
    cms::cuda::free_host(module_segmentRanges);
    cms::cuda::free_host(module_layers);
    cms::cuda::free_host(module_segmentModuleIndices);
}

void SDL::Event::createMiniDoublets()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    //hardcoded range numbers for this will come from studies!
    unsigned int nTotalMDs;
    createMDArrayRanges(*modulesInGPU, *rangesInGPU, nLowerModules, nTotalMDs, stream, N_MAX_MD_PER_MODULES, N_MAX_PIXEL_MD_PER_MODULES);
//    cout<<"nTotalMDs: "<<nTotalMDs<<std::endl; // for memory usage

    if(mdsInGPU == nullptr)
    {
        mdsInGPU = (SDL::miniDoublets*)cms::cuda::allocate_host(sizeof(SDL::miniDoublets), stream);

        //FIXME: Add memory locations for pixel MDs
        createMDsInExplicitMemory(*mdsInGPU, nTotalMDs, nLowerModules, N_MAX_PIXEL_MD_PER_MODULES, stream);

    }
    hipStreamSynchronize(stream);

    int maxThreadsPerModule=0;
    int* module_hitRanges;
    module_hitRanges = (int*)cms::cuda::allocate_host(nModules* 2*sizeof(int), stream);
    hipMemcpyAsync(module_hitRanges,hitsInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    bool* module_isLower;
    module_isLower = (bool*)cms::cuda::allocate_host(nModules*sizeof(bool), stream);
    hipMemcpyAsync(module_isLower,modulesInGPU->isLower,nModules*sizeof(bool),hipMemcpyDeviceToHost,stream);
    bool* module_isInverted;
    module_isInverted = (bool*)cms::cuda::allocate_host(nModules*sizeof(bool), stream);
    hipMemcpyAsync(module_isInverted,modulesInGPU->isInverted,nModules*sizeof(bool),hipMemcpyDeviceToHost,stream);
    int* module_partnerModuleIndices;
    module_partnerModuleIndices = (int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(module_partnerModuleIndices, modulesInGPU->partnerModuleIndices, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    for (uint16_t lowerModuleIndex=0; lowerModuleIndex<nLowerModules; lowerModuleIndex++) 
    {
        uint16_t upperModuleIndex = module_partnerModuleIndices[lowerModuleIndex];
        int lowerHitRanges = module_hitRanges[lowerModuleIndex*2];
        int upperHitRanges = module_hitRanges[upperModuleIndex*2];
        if(lowerHitRanges!=-1 && upperHitRanges!=-1) 
        {
            int nLowerHits = module_hitRanges[lowerModuleIndex * 2 + 1] - lowerHitRanges + 1;
            int nUpperHits = module_hitRanges[upperModuleIndex * 2 + 1] - upperHitRanges + 1;
            maxThreadsPerModule = maxThreadsPerModule > (nLowerHits*nUpperHits) ? maxThreadsPerModule : nLowerHits*nUpperHits;
        }
    }
    cms::cuda::free_host(module_hitRanges);
    cms::cuda::free_host(module_partnerModuleIndices);
    cms::cuda::free_host(module_isLower);
    cms::cuda::free_host(module_isInverted);

    dim3 nThreads(32,16,1);
    //dim3 nThreads(64,16,1);
    dim3 nBlocks(1,MAX_BLOCKS,1);

    SDL::createMiniDoubletsInGPUv2<<<nBlocks,nThreads,64*4*16*sizeof(float),stream>>>(*modulesInGPU,*hitsInGPU,*mdsInGPU,*rangesInGPU);

    hipError_t cudaerr = hipGetLastError(); 
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
    //hipDeviceSynchronize();
    hipStreamSynchronize(stream);

#if defined(AddObjects)
addMiniDoubletsToEventExplicit();
#endif

}

void SDL::Event::createSegmentsWithModuleMap()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    if(segmentsInGPU == nullptr)
    {
        segmentsInGPU = (SDL::segments*)cms::cuda::allocate_host(sizeof(SDL::segments), stream);
        createSegmentsInExplicitMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nLowerModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE,stream);
    }

//HERE
    dim3 cSnThreads(64,1,1);
    uint32_t blks = nLowerModules;
//printf("HERE Num nLowerModules=%d Blks=%d\n",nLowerModules,blks);
    dim3 cSnBlocks(blks,1,1);
    SDL::createSegmentsInGPUv2<<<cSnBlocks,cSnThreads,0,stream>>>(*modulesInGPU, *mdsInGPU, *segmentsInGPU, *rangesInGPU);
    hipError_t cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
    hipStreamSynchronize(stream);
#if defined(AddObjects)
    addSegmentsToEventExplicit();
#endif

}


void SDL::Event::createTriplets()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    if(tripletsInGPU == nullptr)
    {
        tripletsInGPU = (SDL::triplets*)cms::cuda::allocate_host(sizeof(SDL::triplets), stream);
        unsigned int maxTriplets;
        createTripletArrayRanges(*modulesInGPU, *rangesInGPU, *segmentsInGPU, nLowerModules, maxTriplets, stream);
//        cout<<"nTotalTriplets: "<<maxTriplets<<std::endl; // for memory usage
        createTripletsInExplicitMemory(*tripletsInGPU, maxTriplets, nLowerModules,stream);

        hipMemcpyAsync(tripletsInGPU->nMemoryLocations, &maxTriplets, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream);

    }
    //TODO:Move this also inside the ranges function
    uint16_t nonZeroModules=0;
    unsigned int max_InnerSeg=0;
    uint16_t *index = (uint16_t*)malloc(nLowerModules*sizeof(unsigned int));
    uint16_t *index_gpu;
    index_gpu = (uint16_t*)cms::cuda::allocate_device(dev, nLowerModules*sizeof(uint16_t), stream);
    unsigned int *nSegments = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    hipMemcpyAsync((void *)nSegments, segmentsInGPU->nSegments, nLowerModules*sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    uint16_t* module_nConnectedModules;
    module_nConnectedModules = (uint16_t*)cms::cuda::allocate_host(nLowerModules* sizeof(uint16_t), stream);
    hipMemcpyAsync(module_nConnectedModules,modulesInGPU->nConnectedModules,nLowerModules*sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    for (uint16_t innerLowerModuleIndex = 0; innerLowerModuleIndex <nLowerModules; innerLowerModuleIndex++) 
    {
        uint16_t nConnectedModules = module_nConnectedModules[innerLowerModuleIndex];
        unsigned int nInnerSegments = nSegments[innerLowerModuleIndex];
        if (nConnectedModules != 0 and nInnerSegments != 0) 
        {
            index[nonZeroModules] = innerLowerModuleIndex;
            nonZeroModules++;
        }
        max_InnerSeg = max(max_InnerSeg, nInnerSegments);
    }
    cms::cuda::free_host(module_nConnectedModules);
    hipMemcpyAsync(index_gpu, index, nonZeroModules*sizeof(uint16_t), hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);

    dim3 nThreads(16,16,1);
    dim3 nBlocks(1,1,MAX_BLOCKS);
    //createTripletsInGPU<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *rangesInGPU, index_gpu,nonZeroModules);
    SDL::createTripletsInGPUv2<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *rangesInGPU, index_gpu,nonZeroModules);
    hipError_t cudaerr =hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    } 
    hipStreamSynchronize(stream);
    free(nSegments);
    free(index);
    cms::cuda::free_device(dev, index_gpu);

#if defined(AddObjects)
    addTripletsToEventExplicit();
#endif
}

void SDL::Event::createTrackCandidates()
{
    uint16_t nEligibleModules;
    hipMemcpyAsync(&nEligibleModules,rangesInGPU->nEligibleT5Modules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    if(trackCandidatesInGPU == nullptr)
    {
        trackCandidatesInGPU = (SDL::trackCandidates*)cms::cuda::allocate_host(sizeof(SDL::trackCandidates), stream);
        createTrackCandidatesInExplicitMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES + N_MAX_PIXEL_TRACK_CANDIDATES,stream);
    }

#ifdef FINAL_pT3
    //printf("running final state pT3\n");
    dim3 nThreadsT3(64,16,1);
    dim3 nBlocksT3(20,4,1);
#ifdef Crossclean_pT3
    SDL::crossCleanpT3<<<nBlocksT3, nThreadsT3,0,stream>>>(*modulesInGPU, *rangesInGPU, *pixelTripletsInGPU, *segmentsInGPU, *pixelQuintupletsInGPU);
    hipError_t cudaerr_pT3 = hipGetLastError();
    if(cudaerr_pT3 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pT3)<<std::endl;
    }hipStreamSynchronize(stream);

#endif
    //adding objects
    SDL::addpT3asTrackCandidatesInGPU<<<1,512,0,stream>>>(*pixelTripletsInGPU, *trackCandidatesInGPU);
    hipError_t cudaerr_pT3TC = hipGetLastError();
    if(cudaerr_pT3TC != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pT3TC)<<std::endl;
    }hipStreamSynchronize(stream);

#endif

#ifdef FINAL_T5
    //dim3 dupThreads(32,16,2);
    //dim3 dupBlocks(1,1,MAX_BLOCKS);
    dim3 dupThreads(32,16,1);
    dim3 dupBlocks(max(nEligibleModules/32,1),max(nEligibleModules/16,1),1);

    removeDupQuintupletsInGPUBeforeTC<<<dupBlocks,dupThreads,0,stream>>>(*quintupletsInGPU,*rangesInGPU);
    hipStreamSynchronize(stream);

    dim3 nThreads(32,1,32);
    dim3 nBlocks(MAX_BLOCKS,1,(13296/32) + 1);
    crossCleanT5<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *quintupletsInGPU, *pixelQuintupletsInGPU,*pixelTripletsInGPU,*rangesInGPU);
    hipError_t cudaerr_T5 =hipGetLastError(); 
    if(cudaerr_T5 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_T5)<<std::endl;
    }hipStreamSynchronize(stream);

    dim3 nThreadsAddT5(128,8,1);
    dim3 nBlocksAddT5(10,8,1);
    addT5asTrackCandidateInGPU<<<nBlocksAddT5, nThreadsAddT5, 0, stream>>>(*modulesInGPU, *rangesInGPU, *quintupletsInGPU, *trackCandidatesInGPU);
    hipError_t cudaerr_T5TC =hipGetLastError(); 
    if(cudaerr_T5TC != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_T5TC)<<std::endl;
    }hipStreamSynchronize(stream);
#endif // final state T5


#ifdef FINAL_pLS
#ifdef DUP_pLS
    dim3 nThreadspLS(32,32,1);
    dim3 nBlockspLS(MAX_BLOCKS/4, MAX_BLOCKS*4, 1);
    checkHitspLS<<<nBlockspLS, nThreadspLS, 0,stream>>>(*modulesInGPU, *rangesInGPU, *mdsInGPU, *segmentsInGPU, *hitsInGPU, true);
    hipError_t cudaerrpix = hipGetLastError();
    if(cudaerrpix != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerrpix)<<std::endl;

    }hipStreamSynchronize(stream);
#endif  

    dim3 nThreads_pLS(32,16,1);
    dim3 nBlocks_pLS(20,4,1);
    SDL::crossCleanpLS<<<nBlocks_pLS, nThreads_pLS, 0, stream>>>(*modulesInGPU, *rangesInGPU, *pixelTripletsInGPU, *trackCandidatesInGPU, *segmentsInGPU, *mdsInGPU,*hitsInGPU, *quintupletsInGPU);
    hipError_t cudaerr_pLS = hipGetLastError();
    if(cudaerr_pLS != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pLS)<<std::endl;
    }hipStreamSynchronize(stream);

    unsigned int nThreadsx_pLS = 384;
    unsigned int nBlocksx_pLS = MAX_BLOCKS;//(20000) % nThreadsx_pLS == 0 ? 20000 / nThreadsx_pLS : 20000 / nThreadsx_pLS + 1;
    SDL::addpLSasTrackCandidateInGPU<<<nBlocksx_pLS, nThreadsx_pLS, 0, stream>>>(*modulesInGPU, *trackCandidatesInGPU, *segmentsInGPU);
    hipError_t cudaerr_pLSTC = hipGetLastError();
    if(cudaerr_pLSTC != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pLS)<<std::endl;
    }hipStreamSynchronize(stream);
#endif
}

void SDL::Event::createExtendedTracks()
{
    if(trackExtensionsInGPU == nullptr)
    {
        trackExtensionsInGPU = (SDL::trackExtensions*)cms::cuda::allocate_host(sizeof(SDL::trackExtensions), stream);
    }

    unsigned int nTrackCandidates;
    hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);

#ifdef T3T3_EXTENSIONS
    createTrackExtensionsInExplicitMemory(*trackExtensionsInGPU, nTrackCandidates * N_MAX_TRACK_EXTENSIONS_PER_TC + N_MAX_T3T3_TRACK_EXTENSIONS, nTrackCandidates + 1, stream); 
#else
    createTrackExtensionsInExplicitMemory(*trackExtensionsInGPU, nTrackCandidates * N_MAX_TRACK_EXTENSIONS_PER_TC, nTrackCandidates, stream); 

    dim3 nThreads(16,1,1);
    dim3 nBlocks(80,1,nTrackCandidates); 
    //createExtendedTracksInGPU<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *rangesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *pixelTripletsInGPU, *quintupletsInGPU, *pixelQuintupletsInGPU, *trackCandidatesInGPU, *trackExtensionsInGPU);
    SDL::createExtendedTracksInGPUv2<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *rangesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *pixelTripletsInGPU, *quintupletsInGPU, *pixelQuintupletsInGPU, *trackCandidatesInGPU, *trackExtensionsInGPU);

    hipError_t cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }hipStreamSynchronize(stream);

#ifdef T3T3_EXTENSIONS
    dim3 nThreadsT3T3(1,16,16);
    dim3 nBlocksT3T3(nLowerModules % nThreadsT3T3.x == 0 ? nLowerModules / nThreadsT3T3.x: nLowerModules / nThreadsT3T3.x + 1, maxT3s % nThreadsT3T3.y == 0 ? maxT3s / nThreadsT3T3.y : maxT3s / nThreadsT3T3.y + 1, maxT3s % nThreadsT3T3.z == 0 ? maxT3s / nThreadsT3T3.z : maxT3s / nThreadsT3T3.z + 1);

    createT3T3ExtendedTracksInGPU<<<nBlocksT3T3, nThreadsT3T3,0,stream>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, *pixelTripletsInGPU, *pixelQuintupletsInGPU, *trackCandidatesInGPU, *trackExtensionsInGPU, nTrackCandidates);

    cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#endif

    int nThreadsDupCleaning = 512;
    int nBlocksDupCleaning = (nTrackCandidates % nThreadsDupCleaning == 0) ? nTrackCandidates / nThreadsDupCleaning : nTrackCandidates / nThreadsDupCleaning + 1;

    cleanDuplicateExtendedTracks<<<nThreadsDupCleaning, nBlocksDupCleaning,0,stream>>>(*trackExtensionsInGPU, nTrackCandidates);

    cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }hipStreamSynchronize(stream);

//    hipDeviceSynchronize();
}
#endif

void SDL::Event::createPixelTriplets()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    if(pixelTripletsInGPU == nullptr)
    {
        pixelTripletsInGPU = (SDL::pixelTriplets*)cms::cuda::allocate_host(sizeof(SDL::pixelTriplets), stream);
    }

    createPixelTripletsInExplicitMemory(*pixelTripletsInGPU, N_MAX_PIXEL_TRIPLETS,stream);

    unsigned int pixelModuleIndex = nLowerModules;
    int* superbins;
    int8_t* pixelTypes;
    unsigned int *nTriplets;
    unsigned int nInnerSegments = 0;
    hipMemcpyAsync(&nInnerSegments, &(segmentsInGPU->nSegments[pixelModuleIndex]), sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    nTriplets = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(nTriplets, tripletsInGPU->nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    superbins = (int*)cms::cuda::allocate_host(N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int), stream);
    pixelTypes = (int8_t*)cms::cuda::allocate_host(N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t), stream);

    hipMemcpyAsync(superbins,segmentsInGPU->superbin,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(pixelTypes,segmentsInGPU->pixelType,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t),hipMemcpyDeviceToHost,stream);

    unsigned int* connectedPixelSize_host;
    unsigned int* connectedPixelIndex_host;
    connectedPixelSize_host = (unsigned int*)cms::cuda::allocate_host(nInnerSegments* sizeof(unsigned int), stream);
    connectedPixelIndex_host = (unsigned int*)cms::cuda::allocate_host(nInnerSegments* sizeof(unsigned int), stream);
    unsigned int* connectedPixelSize_dev;
    unsigned int* connectedPixelIndex_dev;
    connectedPixelSize_dev = (unsigned int*)cms::cuda::allocate_device(dev, nInnerSegments*sizeof(unsigned int), stream);
    connectedPixelIndex_dev = (unsigned int*)cms::cuda::allocate_device(dev, nInnerSegments*sizeof(unsigned int), stream);

    // unsigned int max_size =0;
    hipStreamSynchronize(stream);
    int pixelIndexOffsetPos = pixelMapping->connectedPixelsIndex[44999] + pixelMapping->connectedPixelsSizes[44999];
    int pixelIndexOffsetNeg = pixelMapping->connectedPixelsIndexPos[44999] + pixelMapping->connectedPixelsSizes[44999] + pixelIndexOffsetPos;

    // TODO: check if a map/reduction to just eligible pLSs would speed up the kernel
    //   the current selection still leaves a significant fraction of unmatchable pLSs
    for (unsigned int i = 0; i < nInnerSegments; i++)
    {// loop over # pLS
        int8_t pixelType = pixelTypes[i];// get pixel type for this pLS
        int superbin = superbins[i]; //get superbin for this pixel
        if((superbin < 0) or (superbin >= 45000) or (pixelType > 2) or (pixelType < 0))
        {
            connectedPixelSize_host[i] = 0;
            connectedPixelIndex_host[i] = 0;
            continue;
        }

        if(pixelType ==0)
        { // used pixel type to select correct size-index arrays
            connectedPixelSize_host[i]  = pixelMapping->connectedPixelsSizes[superbin]; //number of connected modules to this pixel
            auto connectedIdxBase = pixelMapping->connectedPixelsIndex[superbin];
            connectedPixelIndex_host[i] = connectedIdxBase;// index to get start of connected modules for this superbin in map
            // printf("i %d out of nInnerSegments %d type %d superbin %d connectedPixelIndex %d connectedPixelSize %d\n",
            //        i, nInnerSegments, pixelType, superbin, connectedPixelIndex_host[i], connectedPixelSize_host[i]);
        }
        else if(pixelType ==1)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesPos[superbin]; //number of pixel connected modules
            auto connectedIdxBase = pixelMapping->connectedPixelsIndexPos[superbin]+pixelIndexOffsetPos;
            connectedPixelIndex_host[i] = connectedIdxBase;// index to get start of connected pixel modules
        }
        else if(pixelType ==2)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesNeg[superbin]; //number of pixel connected modules
            auto connectedIdxBase = pixelMapping->connectedPixelsIndexNeg[superbin] + pixelIndexOffsetNeg;
            connectedPixelIndex_host[i] = connectedIdxBase;// index to get start of connected pixel modules
        }
    }

    hipMemcpyAsync(connectedPixelSize_dev, connectedPixelSize_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(connectedPixelIndex_dev, connectedPixelIndex_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);

    cms::cuda::free_host(connectedPixelSize_host);
    cms::cuda::free_host(connectedPixelIndex_host);
    cms::cuda::free_host(superbins);
    cms::cuda::free_host(pixelTypes);
    cms::cuda::free_host(nTriplets);

    dim3 nThreads(32,4,1);
    dim3 nBlocks(1,4096,16 /* above median of connected modules*/);

    SDL::createPixelTripletsInGPUFromMapv2<<<nBlocks, nThreads,0,stream>>>(*modulesInGPU, *rangesInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *pixelTripletsInGPU, connectedPixelSize_dev,connectedPixelIndex_dev,nInnerSegments);

    hipError_t cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
    hipStreamSynchronize(stream);
    //}hipDeviceSynchronize();
    cms::cuda::free_device(dev, connectedPixelSize_dev);
    cms::cuda::free_device(dev, connectedPixelIndex_dev);


#ifdef Warnings
    unsigned int nPixelTriplets;
    hipMemcpyAsync(&nPixelTriplets, pixelTripletsInGPU->nPixelTriplets,  sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    std::cout<<"number of pixel triplets = "<<nPixelTriplets<<std::endl;
#endif

    //pT3s can be cleaned here because they're not used in making pT5s!
#ifdef DUP_pT3
    //dim3 nThreads_dup(160,1,1);
    dim3 nThreads_dup(32,32,1);
    dim3 nBlocks_dup(1,40,1); //seems like more blocks lead to conflicting writes
    removeDupPixelTripletsInGPUFromMap<<<nBlocks_dup,nThreads_dup,0,stream>>>(*pixelTripletsInGPU,false);
hipStreamSynchronize(stream);
#endif

}

void SDL::Event::createQuintuplets()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    uint16_t nEligibleT5Modules = 0;
    //uint16_t *indicesOfEligibleModules = (uint16_t*)malloc(nLowerModules*sizeof(uint16_t));

#ifdef CACHE_ALLOC
        rangesInGPU->indicesOfEligibleT5Modules = (uint16_t*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(uint16_t), stream);
#else
        hipMalloc(&(rangesInGPU->indicesOfEligibleT5Modules), nLowerModules * sizeof(uint16_t));
#endif
    hipMemsetAsync(rangesInGPU->quintupletModuleIndices, -1, sizeof(int) * (nLowerModules),stream);
hipStreamSynchronize(stream);
    unsigned int nTotalQuintuplets;
    unsigned int *device_nTotalQuintuplets;
    hipMalloc((void **)&device_nTotalQuintuplets, sizeof(unsigned int));
    createEligibleModulesListForQuintupletsGPU<<<1,1024,0,stream>>>(*modulesInGPU, *tripletsInGPU, device_nTotalQuintuplets, stream, *rangesInGPU);
hipStreamSynchronize(stream);
    hipMemcpyAsync(&nEligibleT5Modules,rangesInGPU->nEligibleT5Modules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(&nTotalQuintuplets,device_nTotalQuintuplets,sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipFree(device_nTotalQuintuplets);
hipStreamSynchronize(stream);

    if(quintupletsInGPU == nullptr)
    {
        quintupletsInGPU = (SDL::quintuplets*)cms::cuda::allocate_host(sizeof(SDL::quintuplets), stream);
        createQuintupletsInExplicitMemory(*quintupletsInGPU, nTotalQuintuplets, nLowerModules, nEligibleT5Modules,stream);
        hipMemcpyAsync(quintupletsInGPU->nMemoryLocations, &nTotalQuintuplets, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream);

    }
hipStreamSynchronize(stream);



    dim3 nThreads(32, 8, 1);
    dim3 nBlocks(1,1,max(nEligibleT5Modules,1));

    SDL::createQuintupletsInGPUv2<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, *rangesInGPU,nEligibleT5Modules);
    hipError_t cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
    hipStreamSynchronize(stream);
    //free(indicesOfEligibleModules);

#ifdef DUP_T5
    dim3 dupThreads(32,32,1);
    dim3 dupBlocks(1,1,MAX_BLOCKS);
    removeDupQuintupletsInGPUAfterBuild<<<dupBlocks,dupThreads,0,stream>>>(*modulesInGPU, *quintupletsInGPU,*rangesInGPU);
    hipStreamSynchronize(stream);
#endif

#if defined(AddObjects)
    addQuintupletsToEventExplicit();
#endif

}
void SDL::Event::pixelLineSegmentCleaning()
{
#ifdef DUP_pLS
    //printf("cleaning pixels\n");
    dim3 nThreadspLS(32,32,1);
    dim3 nBlockspLS(MAX_BLOCKS/4, MAX_BLOCKS*4, 1);

    checkHitspLS<<<nBlockspLS, nThreadspLS, 0,stream>>>(*modulesInGPU, *rangesInGPU, *mdsInGPU, *segmentsInGPU, *hitsInGPU, false);
    hipError_t cudaerrpix = hipGetLastError();
    if(cudaerrpix != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerrpix)<<std::endl;

    }hipStreamSynchronize(stream);
    //}hipDeviceSynchronize();
#endif  

}
void SDL::Event::createPixelQuintuplets()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);

    if(pixelQuintupletsInGPU == nullptr)
    {
        pixelQuintupletsInGPU = (SDL::pixelQuintuplets*)cms::cuda::allocate_host(sizeof(SDL::pixelQuintuplets), stream);
        createPixelQuintupletsInExplicitMemory(*pixelQuintupletsInGPU, N_MAX_PIXEL_QUINTUPLETS,stream);
    }
   if(trackCandidatesInGPU == nullptr)
    {
        trackCandidatesInGPU = (SDL::trackCandidates*)cms::cuda::allocate_host(sizeof(SDL::trackCandidates), stream);
        createTrackCandidatesInExplicitMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES + N_MAX_PIXEL_TRACK_CANDIDATES,stream);
    } 

    unsigned int pixelModuleIndex;
    int* superbins;
    int8_t* pixelTypes;
    unsigned int *nQuintuplets;

    unsigned int* connectedPixelSize_host;
    unsigned int* connectedPixelIndex_host;
    unsigned int* connectedPixelSize_dev;
    unsigned int* connectedPixelIndex_dev;

    nQuintuplets = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(nQuintuplets, quintupletsInGPU->nQuintuplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);

    superbins = (int*)cms::cuda::allocate_host(N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int), stream);
    pixelTypes = (int8_t*)cms::cuda::allocate_host(N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t), stream);

    hipMemcpyAsync(superbins,segmentsInGPU->superbin,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(pixelTypes,segmentsInGPU->pixelType,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t),hipMemcpyDeviceToHost,stream);
    
    hipStreamSynchronize(stream);
    pixelModuleIndex = nLowerModules;
    unsigned int nInnerSegments = 0;
    hipMemcpyAsync(&nInnerSegments, &(segmentsInGPU->nSegments[pixelModuleIndex]), sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    connectedPixelSize_host = (unsigned int*)cms::cuda::allocate_host(nInnerSegments* sizeof(unsigned int), stream);
    connectedPixelIndex_host = (unsigned int*)cms::cuda::allocate_host(nInnerSegments* sizeof(unsigned int), stream);
    connectedPixelSize_dev = (unsigned int*)cms::cuda::allocate_device(dev,nInnerSegments* sizeof(unsigned int),stream);
    connectedPixelIndex_dev = (unsigned int*)cms::cuda::allocate_device(dev,nInnerSegments* sizeof(unsigned int),stream);
    hipStreamSynchronize(stream);

    int pixelIndexOffsetPos = pixelMapping->connectedPixelsIndex[44999] + pixelMapping->connectedPixelsSizes[44999];
    int pixelIndexOffsetNeg = pixelMapping->connectedPixelsIndexPos[44999] + pixelMapping->connectedPixelsSizes[44999] + pixelIndexOffsetPos;

    for (unsigned int i = 0; i < nInnerSegments; i++)
    {// loop over # pLS
        int8_t pixelType = pixelTypes[i];// get pixel type for this pLS
        int superbin = superbins[i]; //get superbin for this pixel
        if((superbin < 0) or (superbin >= 45000) or (pixelType > 2) or (pixelType < 0))
        {
            connectedPixelIndex_host[i] = 0;
            connectedPixelSize_host[i] = 0;
            continue;
        }

        if(pixelType ==0)
        { // used pixel type to select correct size-index arrays
            connectedPixelSize_host[i]  = pixelMapping->connectedPixelsSizes[superbin]; //number of connected modules to this pixel
            unsigned int connectedIdxBase = pixelMapping->connectedPixelsIndex[superbin];
            connectedPixelIndex_host[i] = connectedIdxBase;
        }
        else if(pixelType ==1)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesPos[superbin]; //number of pixel connected modules
            unsigned int connectedIdxBase = pixelMapping->connectedPixelsIndexPos[superbin]+pixelIndexOffsetPos;
            connectedPixelIndex_host[i] = connectedIdxBase;
        }
        else if(pixelType ==2)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesNeg[superbin]; //number of pixel connected modules
            unsigned int connectedIdxBase = pixelMapping->connectedPixelsIndexNeg[superbin] + pixelIndexOffsetNeg;
            connectedPixelIndex_host[i] = connectedIdxBase;
        }
    }

    hipMemcpyAsync(connectedPixelSize_dev, connectedPixelSize_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(connectedPixelIndex_dev, connectedPixelIndex_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
hipStreamSynchronize(stream);

    //less cheap method to estimate max_size for y axis
    unsigned int max_size = *std::max_element(nQuintuplets, nQuintuplets + nLowerModules);
    dim3 nThreads(16,16,1);
    dim3 nBlocks(1,MAX_BLOCKS,16);
                  
    SDL::createPixelQuintupletsInGPUFromMapv2<<<nBlocks, nThreads,0,stream>>>(*modulesInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, *pixelQuintupletsInGPU, connectedPixelSize_dev, connectedPixelIndex_dev, nInnerSegments,*rangesInGPU);

    hipError_t cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
    hipStreamSynchronize(stream);
    cms::cuda::free_host(connectedPixelSize_host);
    cms::cuda::free_host(connectedPixelIndex_host);
    cms::cuda::free_device(dev, connectedPixelSize_dev);
    cms::cuda::free_device(dev, connectedPixelIndex_dev);
    cms::cuda::free_host(superbins);
    cms::cuda::free_host(pixelTypes);
    cms::cuda::free_host(nQuintuplets);
    //free(segs_pix);
    //hipFree(segs_pix_gpu);

    dim3 nThreads_dup(32,32,1);
    dim3 nBlocks_dup(1,MAX_BLOCKS,1);
#ifdef DUP_pT5
    //printf("run dup pT5\n");
    removeDupPixelQuintupletsInGPUFromMap<<<nBlocks_dup,nThreads_dup,0,stream>>>(*pixelQuintupletsInGPU, false);
    hipError_t cudaerr2 = hipGetLastError(); 
    if(cudaerr2 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr2)<<std::endl;
    }hipStreamSynchronize(stream);
    //}hipDeviceSynchronize();
#endif
#ifdef FINAL_pT5
    //printf("Adding pT5s to TC collection\n");
    unsigned int nThreadsx_pT5 = 256;
    unsigned int nBlocksx_pT5 = 1;//(N_MAX_PIXEL_QUINTUPLETS) % nThreadsx_pT5 == 0 ? N_MAX_PIXEL_QUINTUPLETS / nThreadsx_pT5 : N_MAX_PIXEL_QUINTUPLETS / nThreadsx_pT5 + 1;
    SDL::addpT5asTrackCandidateInGPU<<<nBlocksx_pT5, nThreadsx_pT5,0,stream>>>(*modulesInGPU, *pixelQuintupletsInGPU, *trackCandidatesInGPU, *segmentsInGPU, *tripletsInGPU,*quintupletsInGPU);

    hipError_t cudaerr_pT5 = hipGetLastError();
    if(cudaerr_pT5 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pT5)<<std::endl;
    }
    hipStreamSynchronize(stream);
#endif
#ifdef Warnings
    unsigned int nPixelQuintuplets;
    hipMemcpyAsync(&nPixelQuintuplets, &(pixelQuintupletsInGPU->nPixelQuintuplets), sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    std::cout<<"number of pixel quintuplets = "<<nPixelQuintuplets<<std::endl;
#endif   
}

void SDL::Event::addQuintupletsToEvent()
{
    for(uint16_t i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        if(quintupletsInGPU->nQuintuplets[i] == 0)
        {
            rangesInGPU->quintupletRanges[i * 2] = -1;
            rangesInGPU->quintupletRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU->quintupletRanges[i * 2] = rangesInGPU->quintupletModuleIndices[i];
            rangesInGPU->quintupletRanges[i * 2 + 1] = rangesInGPU->quintupletModuleIndices[i] + quintupletsInGPU->nQuintuplets[i] - 1;

            if(modulesInGPU->subdets[i] == Barrel)
            {
                n_quintuplets_by_layer_barrel_[modulesInGPU->layers[i] - 1] += quintupletsInGPU->nQuintuplets[i];
            }
            else
            {
                n_quintuplets_by_layer_endcap_[modulesInGPU->layers[i] - 1] += quintupletsInGPU->nQuintuplets[i];
            }
        }
    }
}

void SDL::Event::addQuintupletsToEventExplicit()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    unsigned int* nQuintupletsCPU;
    nQuintupletsCPU = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);

    hipMemcpyAsync(nQuintupletsCPU,quintupletsInGPU->nQuintuplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);

    short* module_subdets;
    module_subdets = (short*)cms::cuda::allocate_host(nModules* sizeof(short), stream);
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_quintupletRanges;
    module_quintupletRanges = (int*)cms::cuda::allocate_host(nLowerModules* 2*sizeof(int), stream);
    hipMemcpyAsync(module_quintupletRanges,rangesInGPU->quintupletRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    module_layers = (short*)cms::cuda::allocate_host(nLowerModules * sizeof(short), stream);
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_quintupletModuleIndices;
    module_quintupletModuleIndices = (int*)cms::cuda::allocate_host(nLowerModules * sizeof(int), stream);
    hipMemcpyAsync(module_quintupletModuleIndices, rangesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    for(uint16_t i = 0; i<nLowerModules; i++)
    {
        if(nQuintupletsCPU[i] == 0 or module_quintupletModuleIndices[i] == -1)
        {
            module_quintupletRanges[i * 2] = -1;
            module_quintupletRanges[i * 2 + 1] = -1;
        }
       else
        {
            module_quintupletRanges[i * 2] = module_quintupletModuleIndices[i];
            module_quintupletRanges[i * 2 + 1] = module_quintupletModuleIndices[i] + nQuintupletsCPU[i] - 1;

            if(module_subdets[i] == Barrel)
            {
                n_quintuplets_by_layer_barrel_[module_layers[i] - 1] += nQuintupletsCPU[i];
            }
            else
            {
                n_quintuplets_by_layer_endcap_[module_layers[i] - 1] += nQuintupletsCPU[i];
            }
        }
    }
    cms::cuda::free_host(nQuintupletsCPU);
    cms::cuda::free_host(module_quintupletRanges);
    cms::cuda::free_host(module_layers);
    cms::cuda::free_host(module_subdets);
    cms::cuda::free_host(module_quintupletModuleIndices);

}

void SDL::Event::addTripletsToEvent()
{
    for(uint16_t i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        //tracklets run only on lower modules!!!!!!
        if(tripletsInGPU->nTriplets[i] == 0)
        {
            rangesInGPU->tripletRanges[i * 2] = -1;
            rangesInGPU->tripletRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU->tripletRanges[i * 2] = rangesInGPU->tripletModuleIndices[i];
            rangesInGPU->tripletRanges[i * 2 + 1] = rangesInGPU->tripletModuleIndices[i] + tripletsInGPU->nTriplets[i] - 1;

            if(modulesInGPU->subdets[i] == Barrel)
            {
                n_triplets_by_layer_barrel_[modulesInGPU->layers[i] - 1] += tripletsInGPU->nTriplets[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[modulesInGPU->layers[i] - 1] += tripletsInGPU->nTriplets[i];
            }
        }
    }
}
void SDL::Event::addTripletsToEventExplicit()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    unsigned int* nTripletsCPU;
    nTripletsCPU = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(nTripletsCPU,tripletsInGPU->nTriplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);

    short* module_subdets;
    module_subdets = (short*)cms::cuda::allocate_host(nLowerModules* sizeof(short), stream);
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_tripletRanges;
    module_tripletRanges = (int*)cms::cuda::allocate_host(nLowerModules* 2*sizeof(int), stream);
    hipMemcpyAsync(module_tripletRanges,rangesInGPU->tripletRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    module_layers = (short*)cms::cuda::allocate_host(nLowerModules * sizeof(short), stream);
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);

    int* module_tripletModuleIndices;
    module_tripletModuleIndices = (int*)cms::cuda::allocate_host(nLowerModules * sizeof(int), stream);
    hipMemcpyAsync(module_tripletModuleIndices, rangesInGPU->tripletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);
    for(uint16_t i = 0; i<nLowerModules; i++)
    {
        if(nTripletsCPU[i] == 0)
        {
            module_tripletRanges[i * 2] = -1;
            module_tripletRanges[i * 2 + 1] = -1;
        }
        else
        {
            module_tripletRanges[i * 2] = module_tripletModuleIndices[i];
            module_tripletRanges[i * 2 + 1] = module_tripletModuleIndices[i] +  nTripletsCPU[i] - 1;

            if(module_subdets[i] == Barrel)
            {
                n_triplets_by_layer_barrel_[module_layers[i] - 1] += nTripletsCPU[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[module_layers[i] - 1] += nTripletsCPU[i];
            }
        }
    }

    hipMemcpyAsync(rangesInGPU->tripletRanges, module_tripletRanges, nLowerModules * 2 * sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    cms::cuda::free_host(nTripletsCPU);
    cms::cuda::free_host(module_tripletRanges);
    cms::cuda::free_host(module_layers);
    cms::cuda::free_host(module_subdets);
    cms::cuda::free_host(module_tripletModuleIndices);
}

unsigned int SDL::Event::getNumberOfHits()
{
    unsigned int hits = 0;
    for(auto &it:n_hits_by_layer_barrel_)
    {
        hits += it;
    }
    for(auto& it:n_hits_by_layer_endcap_)
    {
        hits += it;
    }

    return hits;
}

unsigned int SDL::Event::getNumberOfHitsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_hits_by_layer_barrel_[layer];
    else
        return n_hits_by_layer_barrel_[layer] + n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerBarrel(unsigned int layer)
{
    return n_hits_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerEndcap(unsigned int layer)
{
    return n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoublets()
{
     unsigned int miniDoublets = 0;
    for(auto &it:n_minidoublets_by_layer_barrel_)
    {
        miniDoublets += it;
    }
    for(auto &it:n_minidoublets_by_layer_endcap_)
    {
        miniDoublets += it;
    }

    return miniDoublets;

}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_minidoublets_by_layer_barrel_[layer];
    else
        return n_minidoublets_by_layer_barrel_[layer] + n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerBarrel(unsigned int layer)
{
    return n_minidoublets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerEndcap(unsigned int layer)
{
    return n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegments()
{
    unsigned int segments = 0;
    for(auto &it:n_segments_by_layer_barrel_)
    {
        segments += it;
    }
    for(auto &it:n_segments_by_layer_endcap_)
    {
        segments += it;
    }

    return segments;

}

unsigned int SDL::Event::getNumberOfSegmentsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_segments_by_layer_barrel_[layer];
    else
        return n_segments_by_layer_barrel_[layer] + n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerBarrel(unsigned int layer)
{
    return n_segments_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerEndcap(unsigned int layer)
{
    return n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTriplets()
{
    unsigned int triplets = 0;
    for(auto &it:n_triplets_by_layer_barrel_)
    {
        triplets += it;
    }
    for(auto &it:n_triplets_by_layer_endcap_)
    {
        triplets += it;
    }

    return triplets;

}

unsigned int SDL::Event::getNumberOfTripletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_triplets_by_layer_barrel_[layer];
    else
        return n_triplets_by_layer_barrel_[layer] + n_triplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerBarrel(unsigned int layer)
{
    return n_triplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerEndcap(unsigned int layer)
{
    return n_triplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfPixelTriplets()
{
    unsigned int nPixelTriplets;
    hipMemcpyAsync(&nPixelTriplets, pixelTripletsInGPU->nPixelTriplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    return nPixelTriplets;
}


unsigned int SDL::Event::getNumberOfExtendedTracks()
{
    unsigned int nTrackCandidates;
    hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);

    unsigned int *nTrackExtensionsCPU = new unsigned int[nTrackCandidates];
    hipMemcpyAsync(nTrackExtensionsCPU, trackExtensionsInGPU->nTrackExtensions, (nTrackCandidates)* sizeof(unsigned int), hipMemcpyDeviceToHost,stream);

    unsigned int nTrackExtensions = 0;
    for(size_t it = 0; it < nTrackCandidates; it++)    
    {
        nTrackExtensions += nTrackExtensionsCPU[it];

    }
#ifdef T3T3_EXTENSIONS
    unsigned int nT3T3Extensions;
    hipMemcpyAsync(&nT3T3Extensions,&(trackExtensionsInGPU->nTrackExtensions[nTrackCandidates]), sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    nTrackExtensions += nT3T3Extensions;
#endif
    delete[] nTrackExtensionsCPU;
    return nTrackExtensions;
}

unsigned int SDL::Event::getNumberOfT3T3ExtendedTracks()
{
    unsigned int nTrackCandidates;
    hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);

    unsigned int nT3T3Extensions;
    hipMemcpyAsync(&nT3T3Extensions, trackExtensionsInGPU->nTrackExtensions + nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    return nT3T3Extensions;
}

unsigned int SDL::Event::getNumberOfPixelQuintuplets()
{
    unsigned int nPixelQuintuplets;
    hipMemcpyAsync(&nPixelQuintuplets, pixelQuintupletsInGPU->nPixelQuintuplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    return nPixelQuintuplets;
}
unsigned int SDL::Event::getNumberOfQuintuplets()
{
    unsigned int quintuplets = 0;
    for(auto &it:n_quintuplets_by_layer_barrel_)
    {
        quintuplets += it;
    }
    for(auto &it:n_quintuplets_by_layer_endcap_)
    {
        quintuplets += it;
    }

    return quintuplets;

}

unsigned int SDL::Event::getNumberOfQuintupletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_quintuplets_by_layer_barrel_[layer];
    else
        return n_quintuplets_by_layer_barrel_[layer] + n_quintuplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayerBarrel(unsigned int layer)
{
    return n_quintuplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayerEndcap(unsigned int layer)
{
    return n_quintuplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidates()
{    
    unsigned int nTrackCandidates;
    hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    return nTrackCandidates;
}

unsigned int SDL::Event::getNumberOfPT5TrackCandidates()
{
    unsigned int nTrackCandidatesPT5;
    hipMemcpyAsync(&nTrackCandidatesPT5, trackCandidatesInGPU->nTrackCandidatespT5, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    return nTrackCandidatesPT5;
}

unsigned int SDL::Event::getNumberOfPT3TrackCandidates()
{
    unsigned int nTrackCandidatesPT3;
    hipMemcpyAsync(&nTrackCandidatesPT3, trackCandidatesInGPU->nTrackCandidatespT3, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    return nTrackCandidatesPT3;
}

unsigned int SDL::Event::getNumberOfPLSTrackCandidates()
{
    unsigned int nTrackCandidatesPLS;
    hipMemcpyAsync(&nTrackCandidatesPLS, trackCandidatesInGPU->nTrackCandidatespLS, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    return nTrackCandidatesPLS;
}

unsigned int SDL::Event::getNumberOfPixelTrackCandidates()
{
    unsigned int nTrackCandidates;
    unsigned int nTrackCandidatesT5;
    hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(&nTrackCandidatesT5, trackCandidatesInGPU->nTrackCandidatesT5, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    return nTrackCandidates - nTrackCandidatesT5;
}

unsigned int SDL::Event::getNumberOfT5TrackCandidates()
{
    unsigned int nTrackCandidatesT5;
    hipMemcpyAsync(&nTrackCandidatesT5, trackCandidatesInGPU->nTrackCandidatesT5, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    return nTrackCandidatesT5; 
}

SDL::hits* SDL::Event::getHits() //std::shared_ptr should take care of garbage collection
{
    if(hitsInCPU == nullptr)
    {
        hitsInCPU = new SDL::hits;
        hitsInCPU->nHits = new unsigned int;
        unsigned int nHits;
        hipMemcpyAsync(&nHits, hitsInGPU->nHits, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        *(hitsInCPU->nHits) = nHits;
        hitsInCPU->idxs = new unsigned int[nHits];
        hitsInCPU->xs = new float[nHits];
        hitsInCPU->ys = new float[nHits];
        hitsInCPU->zs = new float[nHits];
        hitsInCPU->moduleIndices = new uint16_t[nHits];
        hipMemcpyAsync(hitsInCPU->idxs, hitsInGPU->idxs,sizeof(unsigned int) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->xs, hitsInGPU->xs, sizeof(float) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->ys, hitsInGPU->ys, sizeof(float) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->zs, hitsInGPU->zs, sizeof(float) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->moduleIndices, hitsInGPU->moduleIndices, sizeof(uint16_t) * nHits, hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return hitsInCPU;
}
SDL::objectRanges* SDL::Event::getRanges()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    if(rangesInCPU == nullptr)
    {
        rangesInCPU = new SDL::objectRanges;
        rangesInCPU->hitRanges = new int[2*nModules];
        rangesInCPU->quintupletModuleIndices = new int[nLowerModules];
        hipMemcpyAsync(rangesInCPU->hitRanges, hitsInGPU->hitRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost,stream);
        rangesInCPU->miniDoubletModuleIndices = new int[nLowerModules+1];
        rangesInCPU->segmentModuleIndices = new int[nLowerModules + 1];
        rangesInCPU->tripletModuleIndices = new int[nLowerModules];
        hipMemcpyAsync(rangesInCPU->quintupletModuleIndices, rangesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(rangesInCPU->miniDoubletModuleIndices, rangesInGPU->miniDoubletModuleIndices, (nLowerModules + 1) * sizeof(int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(rangesInCPU->segmentModuleIndices, rangesInGPU->segmentModuleIndices, (nLowerModules + 1) * sizeof(int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(rangesInCPU->tripletModuleIndices, rangesInGPU->tripletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost, stream);
hipStreamSynchronize(stream);
    }
    return rangesInCPU;
}

SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    if(mdsInCPU == nullptr)
    {
        mdsInCPU = new SDL::miniDoublets;
        mdsInCPU->nMDs = new unsigned int[nLowerModules+1];

        //compute memory locations
        mdsInCPU->nMemoryLocations = new unsigned int;
        hipMemcpyAsync(mdsInCPU->nMemoryLocations, mdsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
        mdsInCPU->totOccupancyMDs = new unsigned int[nLowerModules+1];

        mdsInCPU->anchorHitIndices = new unsigned int[*(mdsInCPU->nMemoryLocations)];
        mdsInCPU->outerHitIndices = new unsigned int[*(mdsInCPU->nMemoryLocations)];
        hipMemcpyAsync(mdsInCPU->anchorHitIndices, mdsInGPU->anchorHitIndices, *(mdsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(mdsInCPU->outerHitIndices, mdsInGPU->outerHitIndices, *(mdsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(mdsInCPU->nMDs, mdsInGPU->nMDs, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(mdsInCPU->totOccupancyMDs, mdsInGPU->totOccupancyMDs, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return mdsInCPU;
}

SDL::segments* SDL::Event::getSegments()
{
    if(segmentsInCPU == nullptr)
    {
        segmentsInCPU = new SDL::segments;
        
        segmentsInCPU->nSegments = new unsigned int[nLowerModules+1];
        hipMemcpyAsync(segmentsInCPU->nSegments, segmentsInGPU->nSegments, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        
        segmentsInCPU->nMemoryLocations = new unsigned int;
        hipMemcpyAsync(segmentsInCPU->nMemoryLocations, segmentsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        segmentsInCPU->mdIndices = new unsigned int[2 * *(segmentsInCPU->nMemoryLocations)];
        segmentsInCPU->innerMiniDoubletAnchorHitIndices = new unsigned int[*(segmentsInCPU->nMemoryLocations)];
        segmentsInCPU->outerMiniDoubletAnchorHitIndices = new unsigned int[*(segmentsInCPU->nMemoryLocations)];
        segmentsInCPU->totOccupancySegments = new unsigned int[nLowerModules+1];

        segmentsInCPU->ptIn = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->eta = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->phi = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->isDup = new bool[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->isQuad = new bool[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->score = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];

        hipMemcpyAsync(segmentsInCPU->mdIndices, segmentsInGPU->mdIndices, 2 * *(segmentsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->innerMiniDoubletAnchorHitIndices, segmentsInGPU->innerMiniDoubletAnchorHitIndices, *(segmentsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->outerMiniDoubletAnchorHitIndices, segmentsInGPU->outerMiniDoubletAnchorHitIndices, *(segmentsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->totOccupancySegments, segmentsInGPU->totOccupancySegments, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->ptIn, segmentsInGPU->ptIn, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->eta, segmentsInGPU->eta, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->phi, segmentsInGPU->phi, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->isDup, segmentsInGPU->isDup, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->isQuad, segmentsInGPU->isQuad, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->score, segmentsInGPU->score, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return segmentsInCPU;
}

SDL::triplets* SDL::Event::getTriplets()
{
    if(tripletsInCPU == nullptr)
    {
        uint16_t nLowerModules;
        tripletsInCPU = new SDL::triplets;
        hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        tripletsInCPU->nMemoryLocations = new unsigned int;
        hipMemcpyAsync(tripletsInCPU->nMemoryLocations, tripletsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        tripletsInCPU->segmentIndices = new unsigned[2 * *(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->nTriplets = new unsigned int[nLowerModules];
        tripletsInCPU->betaIn  = new FPX[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->betaOut = new FPX[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->pt_beta = new FPX[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->hitIndices = new unsigned int[6 * *(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->logicalLayers = new uint8_t[3 * *(tripletsInCPU->nMemoryLocations)];
#ifdef CUT_VALUE_DEBUG

        tripletsInCPU->zOut = new float[4 * *(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->zLo = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->zHi = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->zLoPointed = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->zHiPointed = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->sdlCut = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->betaInCut = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->betaOutCut = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->deltaBetaCut = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->rtLo = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->rtHi = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->kZ = new float[*(tripletsInCPU->nMemoryLocations)];

        tripletsInCPU->rtOut = tripletsInCPU->zOut + *(tripletsInCPU->nMemoryLocations);
        tripletsInCPU->deltaPhiPos = tripletsInCPU->zOut + 2 * *(tripletsInCPU->nMemoryLocations);
        tripletsInCPU->deltaPhi = tripletsInCPU->zOut + 3 * *(tripletsInCPU->nMemoryLocations);

        hipMemcpyAsync(tripletsInCPU->zOut, tripletsInGPU->zOut, 4 * * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->zLo, tripletsInGPU->zLo, * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->zHi, tripletsInGPU->zHi, * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->zLoPointed, tripletsInGPU->zLoPointed, 4 * * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->zHiPointed, tripletsInGPU->zHiPointed, * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->sdlCut, tripletsInGPU->sdlCut, *(tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->betaInCut, tripletsInGPU->betaInCut,  * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->betaOutCut, tripletsInGPU->betaOutCut,  * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->deltaBetaCut, tripletsInGPU->deltaBetaCut, *(tripletsInCPU->nMemoryLocations)*sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpyAsync(tripletsInCPU->rtLo, tripletsInGPU->rtLo,  * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->rtHi, tripletsInGPU->rtHi,  * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->kZ, tripletsInGPU->kZ,  * (tripletsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
#endif

        hipMemcpyAsync(tripletsInCPU->hitIndices, tripletsInGPU->hitIndices, 6 * *(tripletsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->logicalLayers, tripletsInGPU->logicalLayers, 3 * *(tripletsInCPU->nMemoryLocations) * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->segmentIndices, tripletsInGPU->segmentIndices, 2 * *(tripletsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->betaIn, tripletsInGPU->betaIn,   *(tripletsInCPU->nMemoryLocations) * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->betaOut, tripletsInGPU->betaOut, *(tripletsInCPU->nMemoryLocations) * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->pt_beta, tripletsInGPU->pt_beta, *(tripletsInCPU->nMemoryLocations) * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        tripletsInCPU->totOccupancyTriplets = new unsigned int[nLowerModules];
        hipMemcpyAsync(tripletsInCPU->nTriplets, tripletsInGPU->nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->totOccupancyTriplets, tripletsInGPU->totOccupancyTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return tripletsInCPU;
}

SDL::quintuplets* SDL::Event::getQuintuplets()
{
    if(quintupletsInCPU == nullptr)
    {
        quintupletsInCPU = new SDL::quintuplets;
        uint16_t nLowerModules;
        hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        uint16_t nEligibleT5Modules;
        hipMemcpyAsync(&nEligibleT5Modules, rangesInGPU->nEligibleT5Modules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        unsigned int nMemoryLocations;
        hipMemcpyAsync(&nMemoryLocations, quintupletsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        quintupletsInCPU->nQuintuplets = new unsigned int[nLowerModules];
        quintupletsInCPU->totOccupancyQuintuplets = new unsigned int[nLowerModules];
        quintupletsInCPU->tripletIndices = new unsigned int[2 * nMemoryLocations];
        quintupletsInCPU->lowerModuleIndices = new uint16_t[5 * nMemoryLocations];
        quintupletsInCPU->innerRadius = new FPX[nMemoryLocations];
        quintupletsInCPU->outerRadius = new FPX[nMemoryLocations];
        quintupletsInCPU->isDup = new bool[nMemoryLocations];
        quintupletsInCPU->score_rphisum = new FPX[nMemoryLocations];
        quintupletsInCPU->eta = new FPX[nMemoryLocations];
        quintupletsInCPU->phi = new FPX[nMemoryLocations];
        quintupletsInCPU->regressionRadius = new float[nMemoryLocations];
#ifdef CUT_VALUE_DEBUG
        quintupletsInCPU->innerRadiusMin = new float[nMemoryLocations];
        quintupletsInCPU->innerRadiusMin2S = new float[nMemoryLocations];
        quintupletsInCPU->innerRadiusMax = new float[nMemoryLocations];
        quintupletsInCPU->innerRadiusMax2S = new float[nMemoryLocations];
        quintupletsInCPU->bridgeRadius = new float[nMemoryLocations];
        quintupletsInCPU->bridgeRadiusMin = new float[nMemoryLocations];
        quintupletsInCPU->bridgeRadiusMin2S = new float[nMemoryLocations];
        quintupletsInCPU->bridgeRadiusMax = new float[nMemoryLocations];
        quintupletsInCPU->bridgeRadiusMax2S = new float[nMemoryLocations];
        quintupletsInCPU->outerRadiusMin = new float[nMemoryLocations];
        quintupletsInCPU->outerRadiusMin2S = new float[nMemoryLocations];
        quintupletsInCPU->outerRadiusMax = new float[nMemoryLocations];
        quintupletsInCPU->outerRadiusMax2S = new float[nMemoryLocations];
        quintupletsInCPU->chiSquared = new float[nMemoryLocations];
        quintupletsInCPU->nonAnchorChiSquared = new float[nMemoryLocations];

        hipMemcpyAsync(quintupletsInCPU->innerRadiusMin, quintupletsInGPU->innerRadiusMin, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->innerRadiusMax, quintupletsInGPU->innerRadiusMax, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->bridgeRadius, quintupletsInGPU->bridgeRadius, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->bridgeRadiusMin, quintupletsInGPU->bridgeRadiusMin, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->bridgeRadiusMin2S, quintupletsInGPU->bridgeRadiusMin2S, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->bridgeRadiusMax, quintupletsInGPU->bridgeRadiusMax, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->bridgeRadiusMax2S, quintupletsInGPU->bridgeRadiusMax2S, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->outerRadiusMin, quintupletsInGPU->outerRadiusMin, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->outerRadiusMin2S, quintupletsInGPU->outerRadiusMin2S, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->outerRadiusMax, quintupletsInGPU->outerRadiusMax, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->outerRadiusMax2S, quintupletsInGPU->outerRadiusMax2S, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->chiSquared, quintupletsInGPU->chiSquared, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->nonAnchorChiSquared, quintupletsInGPU->nonAnchorChiSquared, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
#endif
        hipMemcpyAsync(quintupletsInCPU->nQuintuplets, quintupletsInGPU->nQuintuplets,  nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->totOccupancyQuintuplets, quintupletsInGPU->totOccupancyQuintuplets,  nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->tripletIndices, quintupletsInGPU->tripletIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->lowerModuleIndices, quintupletsInGPU->lowerModuleIndices, 5 * nMemoryLocations * sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->innerRadius, quintupletsInGPU->innerRadius, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->outerRadius, quintupletsInGPU->outerRadius, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->isDup, quintupletsInGPU->isDup, nMemoryLocations * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->score_rphisum, quintupletsInGPU->score_rphisum, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->eta, quintupletsInGPU->eta, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->phi, quintupletsInGPU->phi, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->regressionRadius, quintupletsInGPU->regressionRadius, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    }

    return quintupletsInCPU;
}

SDL::pixelTriplets* SDL::Event::getPixelTriplets()
{
    if(pixelTripletsInCPU == nullptr)
    {
        pixelTripletsInCPU = new SDL::pixelTriplets;

        pixelTripletsInCPU->nPixelTriplets = new unsigned int;
        pixelTripletsInCPU->totOccupancyPixelTriplets = new unsigned int;
        hipMemcpyAsync(pixelTripletsInCPU->nPixelTriplets, pixelTripletsInGPU->nPixelTriplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->totOccupancyPixelTriplets, pixelTripletsInGPU->totOccupancyPixelTriplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
        unsigned int nPixelTriplets = *(pixelTripletsInCPU->nPixelTriplets);
        pixelTripletsInCPU->tripletIndices = new unsigned int[nPixelTriplets];
        pixelTripletsInCPU->pixelSegmentIndices = new unsigned int[nPixelTriplets];
        pixelTripletsInCPU->pixelRadius = new FPX[nPixelTriplets];
        pixelTripletsInCPU->tripletRadius = new FPX[nPixelTriplets];
        pixelTripletsInCPU->isDup = new bool[nPixelTriplets];
        pixelTripletsInCPU->eta = new  FPX[nPixelTriplets];
        pixelTripletsInCPU->phi = new  FPX[nPixelTriplets];
        pixelTripletsInCPU->score =new FPX[nPixelTriplets];
#ifdef CUT_VALUE_DEBUG
        pixelTripletsInCPU->pixelRadiusError = new float[nPixelTriplets];
        pixelTripletsInCPU->rzChiSquared = new float[nPixelTriplets];
        pixelTripletsInCPU->rPhiChiSquared = new float[nPixelTriplets];
        pixelTripletsInCPU->rPhiChiSquaredInwards = new float[nPixelTriplets];

        hipMemcpyAsync(pixelTripletsInCPU->pixelRadiusError, pixelTripletsInGPU->pixelRadiusError, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(pixelTripletsInCPU->rzChiSquared, pixelTripletsInGPU->rzChiSquared, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(pixelTripletsInCPU->rPhiChiSquared, pixelTripletsInGPU->rPhiChiSquared, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(pixelTripletsInCPU->rPhiChiSquaredInwards, pixelTripletsInGPU->rPhiChiSquaredInwards, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost, stream);
#endif

        hipMemcpyAsync(pixelTripletsInCPU->tripletIndices, pixelTripletsInGPU->tripletIndices, nPixelTriplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->pixelSegmentIndices, pixelTripletsInGPU->pixelSegmentIndices, nPixelTriplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->pixelRadius, pixelTripletsInGPU->pixelRadius, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->tripletRadius, pixelTripletsInGPU->tripletRadius, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->isDup, pixelTripletsInGPU->isDup, nPixelTriplets * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->eta, pixelTripletsInGPU->eta, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->phi, pixelTripletsInGPU->phi, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->score, pixelTripletsInGPU->score, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    }
    return pixelTripletsInCPU;
}

SDL::pixelQuintuplets* SDL::Event::getPixelQuintuplets()
{
    if(pixelQuintupletsInCPU == nullptr)
    {
        pixelQuintupletsInCPU = new SDL::pixelQuintuplets;

        pixelQuintupletsInCPU->nPixelQuintuplets = new unsigned int;
        pixelQuintupletsInCPU->totOccupancyPixelQuintuplets = new unsigned int;
        hipMemcpyAsync(pixelQuintupletsInCPU->nPixelQuintuplets, pixelQuintupletsInGPU->nPixelQuintuplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->totOccupancyPixelQuintuplets, pixelQuintupletsInGPU->totOccupancyPixelQuintuplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
        unsigned int nPixelQuintuplets = *(pixelQuintupletsInCPU->nPixelQuintuplets);

        pixelQuintupletsInCPU->pixelIndices = new unsigned int[nPixelQuintuplets];
        pixelQuintupletsInCPU->T5Indices = new unsigned int[nPixelQuintuplets];
        pixelQuintupletsInCPU->isDup = new bool[nPixelQuintuplets];
        pixelQuintupletsInCPU->score = new FPX[nPixelQuintuplets];
#ifdef CUT_VALUE_DEBUG
        pixelQuintupletsInCPU->rzChiSquared = new float[nPixelQuintuplets];
        pixelQuintupletsInCPU->rPhiChiSquared = new float[nPixelQuintuplets];
        pixelQuintupletsInCPU->rPhiChiSquaredInwards = new float[nPixelQuintuplets];

        hipMemcpyAsync(pixelQuintupletsInCPU->rzChiSquared, pixelQuintupletsInGPU->rzChiSquared, nPixelQuintuplets * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->rPhiChiSquared, pixelQuintupletsInGPU->rPhiChiSquared, nPixelQuintuplets * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->rPhiChiSquaredInwards, pixelQuintupletsInGPU->rPhiChiSquaredInwards, nPixelQuintuplets * sizeof(float), hipMemcpyDeviceToHost, stream);

#endif
        hipMemcpyAsync(pixelQuintupletsInCPU->pixelIndices, pixelQuintupletsInGPU->pixelIndices, nPixelQuintuplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->T5Indices, pixelQuintupletsInGPU->T5Indices, nPixelQuintuplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->isDup, pixelQuintupletsInGPU->isDup, nPixelQuintuplets * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->score, pixelQuintupletsInGPU->score, nPixelQuintuplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return pixelQuintupletsInCPU;
}

SDL::trackCandidates* SDL::Event::getTrackCandidates()
{
    if(trackCandidatesInCPU == nullptr)
    {
        trackCandidatesInCPU = new SDL::trackCandidates;
        trackCandidatesInCPU->nTrackCandidates = new unsigned int;
        hipMemcpyAsync(trackCandidatesInCPU->nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        unsigned int nTrackCandidates = *(trackCandidatesInCPU->nTrackCandidates);

        trackCandidatesInCPU->objectIndices = new unsigned int[2 * nTrackCandidates];
        trackCandidatesInCPU->trackCandidateType = new short[nTrackCandidates];
        trackCandidatesInCPU->partOfExtension = new bool[nTrackCandidates];
        trackCandidatesInCPU->hitIndices = new unsigned int[14 * nTrackCandidates];
        trackCandidatesInCPU->logicalLayers = new uint8_t[7 * nTrackCandidates];

        hipMemcpyAsync(trackCandidatesInCPU->partOfExtension, trackCandidatesInGPU->partOfExtension, nTrackCandidates * sizeof(bool), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(trackCandidatesInCPU->hitIndices, trackCandidatesInGPU->hitIndices, 14 * nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(trackCandidatesInCPU->logicalLayers, trackCandidatesInGPU->logicalLayers, 7 * nTrackCandidates * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(trackCandidatesInCPU->objectIndices, trackCandidatesInGPU->objectIndices, 2 * nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);                                                                                    
        hipMemcpyAsync(trackCandidatesInCPU->trackCandidateType, trackCandidatesInGPU->trackCandidateType, nTrackCandidates * sizeof(short), hipMemcpyDeviceToHost,stream);                                                                                                                
        hipStreamSynchronize(stream);
    }
    return trackCandidatesInCPU;
}

SDL::modules* SDL::Event::getFullModules()
{
    if(modulesInCPUFull == nullptr)
    {
        modulesInCPUFull = new SDL::modules;
        uint16_t nLowerModules;
        hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    modulesInCPUFull->detIds = new unsigned int[nModules];
    modulesInCPUFull->moduleMap = new uint16_t[40*nModules];
    modulesInCPUFull->nConnectedModules = new uint16_t[nModules];
    modulesInCPUFull->drdzs = new float[nModules];
    modulesInCPUFull->slopes = new float[nModules];
    modulesInCPUFull->nModules = new uint16_t[1];
    modulesInCPUFull->nLowerModules = new uint16_t[1];
    modulesInCPUFull->layers = new short[nModules];
    modulesInCPUFull->rings = new short[nModules];
    modulesInCPUFull->modules = new short[nModules];
    modulesInCPUFull->rods = new short[nModules];
    modulesInCPUFull->subdets = new short[nModules];
    modulesInCPUFull->sides = new short[nModules];
    modulesInCPUFull->isInverted = new bool[nModules];
    modulesInCPUFull->isLower = new bool[nModules];


    modulesInCPUFull->moduleType = new ModuleType[nModules];
    modulesInCPUFull->moduleLayerType = new ModuleLayerType[nModules];
    hipMemcpyAsync(modulesInCPUFull->detIds,modulesInGPU->detIds,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->moduleMap,modulesInGPU->moduleMap,40*nModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->drdzs,modulesInGPU->drdzs,sizeof(float)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->slopes,modulesInGPU->slopes,sizeof(float)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->rings,modulesInGPU->rings,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->modules,modulesInGPU->modules,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->rods,modulesInGPU->rods,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->subdets,modulesInGPU->subdets,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->sides,modulesInGPU->sides,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->isInverted,modulesInGPU->isInverted,sizeof(bool)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->isLower,modulesInGPU->isLower,sizeof(bool)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->moduleType,modulesInGPU->moduleType,sizeof(ModuleType)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->moduleLayerType,modulesInGPU->moduleLayerType,sizeof(ModuleLayerType)*nModules,hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    }
    return modulesInCPUFull;
}
SDL::modules* SDL::Event::getModules()
{
    if(modulesInCPU == nullptr)
    {
        modulesInCPU = new SDL::modules;
        uint16_t nLowerModules;
        hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        modulesInCPU->nLowerModules = new uint16_t[1];
        modulesInCPU->nModules = new uint16_t[1];
        modulesInCPU->detIds = new unsigned int[nModules];
        modulesInCPU->isLower = new bool[nModules];
        modulesInCPU->layers = new short[nModules];
        modulesInCPU->subdets = new short[nModules];
        modulesInCPU->rings = new short[nModules];
        modulesInCPU->rods = new short[nModules];
        modulesInCPU->modules = new short[nModules];
        modulesInCPU->sides = new short[nModules];
        modulesInCPU->eta = new float[nModules];
        modulesInCPU->r = new float[nModules];
        modulesInCPU->moduleType = new ModuleType[nModules];

        hipMemcpyAsync(modulesInCPU->nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->nModules, modulesInGPU->nModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->detIds, modulesInGPU->detIds, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->isLower, modulesInGPU->isLower, nModules * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->layers, modulesInGPU->layers, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->subdets, modulesInGPU->subdets, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->rings, modulesInGPU->rings, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->rods, modulesInGPU->rods, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->modules, modulesInGPU->modules, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->sides, modulesInGPU->sides, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->eta, modulesInGPU->eta, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->r, modulesInGPU->r, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->moduleType, modulesInGPU->moduleType, nModules * sizeof(ModuleType), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
    }
    return modulesInCPU;
}

SDL::trackExtensions* SDL::Event::getTrackExtensions()
{
   if(trackExtensionsInCPU == nullptr)
   {
       trackExtensionsInCPU = new SDL::trackExtensions;
       unsigned int nTrackCandidates;
       hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
       hipStreamSynchronize(stream);
       unsigned int maxTrackExtensions = nTrackCandidates * N_MAX_TRACK_EXTENSIONS_PER_TC;
#ifdef T3T3_EXTENSIONS
       maxTrackExtensions += N_MAX_T3T3_TRACK_EXTENSIONS;
       nTrackCandidates++;
#endif
       std::cout<<"nTrackCandidates = "<<nTrackCandidates<<std::endl;
       trackExtensionsInCPU->nTrackExtensions = new unsigned int[nTrackCandidates];
       trackExtensionsInCPU->totOccupancyTrackExtensions = new unsigned int[nTrackCandidates];
       trackExtensionsInCPU->constituentTCTypes = new short[3 * maxTrackExtensions];
       trackExtensionsInCPU->constituentTCIndices = new unsigned int[3 * maxTrackExtensions];
       trackExtensionsInCPU->nLayerOverlaps = new uint8_t[2 * maxTrackExtensions];
       trackExtensionsInCPU->nHitOverlaps = new uint8_t[2 * maxTrackExtensions];
       trackExtensionsInCPU->isDup = new bool[maxTrackExtensions];
       trackExtensionsInCPU->regressionRadius = new FPX[maxTrackExtensions];
#ifdef CUT_VALUE_DEBUG
       trackExtensionsInCPU->rPhiChiSquared = new FPX[maxTrackExtensions];
       trackExtensionsInCPU->rzChiSquared = new FPX[maxTrackExtensions];
       trackExtensionsInCPU->innerRadius = new float[maxTrackExtensions];
       trackExtensionsInCPU->outerRadius = new float[maxTrackExtensions];
       
       hipMemcpyAsync(trackExtensionsInCPU->rPhiChiSquared, trackExtensionsInGPU->rPhiChiSquared, maxTrackExtensions * sizeof(FPX), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->rzChiSquared, trackExtensionsInGPU->rzChiSquared, maxTrackExtensions * sizeof(FPX), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->innerRadius, trackExtensionsInGPU->innerRadius, maxTrackExtensions * sizeof(float), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->outerRadius, trackExtensionsInGPU->outerRadius, maxTrackExtensions * sizeof(float), hipMemcpyDeviceToHost, stream);

#endif

       hipMemcpyAsync(trackExtensionsInCPU->nTrackExtensions, trackExtensionsInGPU->nTrackExtensions, nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->totOccupancyTrackExtensions, trackExtensionsInGPU->totOccupancyTrackExtensions, nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->constituentTCTypes, trackExtensionsInGPU->constituentTCTypes, 3 * maxTrackExtensions * sizeof(short), hipMemcpyDeviceToHost,stream);
       hipMemcpyAsync(trackExtensionsInCPU->constituentTCIndices, trackExtensionsInGPU->constituentTCIndices, 3 * maxTrackExtensions * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);

       hipMemcpyAsync(trackExtensionsInCPU->nLayerOverlaps, trackExtensionsInGPU->nLayerOverlaps, 2 * maxTrackExtensions * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->nHitOverlaps, trackExtensionsInGPU->nHitOverlaps, 2 * maxTrackExtensions * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->isDup, trackExtensionsInGPU->isDup, maxTrackExtensions * sizeof(bool), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->regressionRadius, trackExtensionsInGPU->regressionRadius, maxTrackExtensions * sizeof(FPX), hipMemcpyDeviceToHost, stream);
       hipStreamSynchronize(stream);
   }

   return trackExtensionsInCPU;
}

