#include "hip/hip_runtime.h"
# include "Event.cuh"
#include "allocate.h"

struct SDL::modules* SDL::modulesInGPU = nullptr;
struct SDL::pixelMap* SDL::pixelMapping = nullptr;
uint16_t SDL::nModules;
uint16_t SDL::nLowerModules;

SDL::Event::Event(hipStream_t estream)
{
    int version;
    int driver;
    hipRuntimeGetVersion(&version);
    hipDriverGetVersion(&driver);
    //printf("version: %d Driver %d\n",version, driver);
    stream = estream;
    hitsInGPU = nullptr;
    mdsInGPU = nullptr;
    segmentsInGPU = nullptr;
    tripletsInGPU = nullptr;
    tripletsInwardInGPU = nullptr;
    quintupletsInGPU = nullptr;
    trackCandidatesInGPU = nullptr;
    pixelTripletsInGPU = nullptr;
    pixelQuintupletsInGPU = nullptr;
    trackExtensionsInGPU = nullptr;
    rangesInGPU = nullptr;

    hitsInCPU = nullptr;
    rangesInCPU = nullptr;
    mdsInCPU = nullptr;
    segmentsInCPU = nullptr;
    tripletsInCPU = nullptr;
    trackCandidatesInCPU = nullptr;
    modulesInCPU = nullptr;
    modulesInCPUFull = nullptr;
    quintupletsInCPU = nullptr;
    pixelTripletsInCPU = nullptr;
    pixelQuintupletsInCPU = nullptr;
    trackExtensionsInCPU = nullptr;

    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_triplets_by_layer_barrel_[i] = 0;
        n_trackCandidates_by_layer_barrel_[i] = 0;
        n_quintuplets_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
            n_segments_by_layer_endcap_[i] = 0;
            n_triplets_by_layer_endcap_[i] = 0;
            n_trackCandidates_by_layer_endcap_[i] = 0;
            n_quintuplets_by_layer_endcap_[i] = 0;
        }
    }
    //resetObjectsInModule();

}

SDL::Event::~Event()
{
#ifdef CACHE_ALLOC
    if(rangesInGPU){rangesInGPU->freeMemoryCache();}
    if(hitsInGPU){hitsInGPU->freeMemoryCache();}
    if(mdsInGPU){mdsInGPU->freeMemoryCache();}
    if(segmentsInGPU){segmentsInGPU->freeMemoryCache();}
    if(tripletsInGPU){tripletsInGPU->freeMemoryCache();}
    if(quintupletsInGPU){quintupletsInGPU->freeMemoryCache();}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemoryCache();}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemoryCache();}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemoryCache();}
    if(trackExtensionsInGPU){trackExtensionsInGPU->freeMemoryCache();}
#else

    if(rangesInGPU){rangesInGPU->freeMemory();}
    if(hitsInGPU){hitsInGPU->freeMemory(stream);}
    if(mdsInGPU){mdsInGPU->freeMemory(stream);}
    if(segmentsInGPU){segmentsInGPU->freeMemory(stream);}
    if(tripletsInGPU){tripletsInGPU->freeMemory(stream);}
    if(quintupletsInGPU){quintupletsInGPU->freeMemory(stream);}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemory(stream);}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemory(stream);}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemory(stream);}
    if(trackExtensionsInGPU){trackExtensionsInGPU->freeMemory(stream);}
#endif
    if(rangesInGPU != nullptr){hipHostFree(rangesInGPU);}
    if(mdsInGPU != nullptr){hipHostFree(mdsInGPU);}
    if(segmentsInGPU!= nullptr){hipHostFree(segmentsInGPU);}
    if(tripletsInGPU!= nullptr){hipHostFree(tripletsInGPU);}
    if(tripletsInwardInGPU != nullptr){hipHostFree(tripletsInwardInGPU);}
    if(trackCandidatesInGPU!= nullptr){hipHostFree(trackCandidatesInGPU);}
    if(hitsInGPU!= nullptr){hipHostFree(hitsInGPU);}

    if(pixelTripletsInGPU!= nullptr){hipHostFree(pixelTripletsInGPU);}
    if(pixelQuintupletsInGPU!= nullptr){hipHostFree(pixelQuintupletsInGPU);}

    if(quintupletsInGPU!= nullptr){hipHostFree(quintupletsInGPU);}
    if(trackExtensionsInGPU != nullptr){hipHostFree(trackExtensionsInGPU);}

#ifdef Explicit_Hit
    if(hitsInCPU != nullptr)
    {
        delete[] hitsInCPU->idxs;
        delete[] hitsInCPU->xs;
        delete[] hitsInCPU->ys;
        delete[] hitsInCPU->zs;
        delete[] hitsInCPU->moduleIndices;
        delete hitsInCPU->nHits;
        delete hitsInCPU;
    }
    if(rangesInCPU != nullptr)
    {
        delete[] rangesInCPU->quintupletModuleIndices;
        delete rangesInCPU;
    }
#endif
#ifdef Explicit_MD
    if(mdsInCPU != nullptr)
    {
        delete[] mdsInCPU->anchorHitIndices;
        delete[] mdsInCPU->nMDs;
        delete mdsInCPU->nMemoryLocations;
        delete[] mdsInCPU->totOccupancyMDs;
        delete mdsInCPU;
    }
#endif
#ifdef Explicit_Seg
    if(segmentsInCPU != nullptr)
    {
        delete[] segmentsInCPU->mdIndices;
        delete[] segmentsInCPU->nSegments;
        delete[] segmentsInCPU->totOccupancySegments;
        delete[] segmentsInCPU->innerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->outerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->ptIn;
        delete[] segmentsInCPU->eta;
        delete[] segmentsInCPU->phi;
        delete segmentsInCPU->nMemoryLocations;
        delete segmentsInCPU;
    }
#endif
#ifdef Explicit_Trips
    if(tripletsInCPU != nullptr)
    {
        delete[] tripletsInCPU->segmentIndices;
        delete[] tripletsInCPU->nTriplets;
        delete[] tripletsInCPU->totOccupancyTriplets;
        delete[] tripletsInCPU->betaIn;
        delete[] tripletsInCPU->betaOut;
        delete[] tripletsInCPU->pt_beta;
        delete[] tripletsInCPU->hitIndices;
        delete[] tripletsInCPU->logicalLayers;
        delete[] tripletsInCPU->lowerModuleIndices;
        delete tripletsInCPU->nMemoryLocations;
        delete tripletsInCPU;
    }
#endif
#ifdef Explicit_T5
#ifdef FINAL_T5
    if(quintupletsInCPU != nullptr)
    {
        delete[] quintupletsInCPU->tripletIndices;
        delete[] quintupletsInCPU->nQuintuplets;
        delete[] quintupletsInCPU->totOccupancyQuintuplets;
        delete[] quintupletsInCPU->lowerModuleIndices;
        delete[] quintupletsInCPU->innerRadius;
        delete[] quintupletsInCPU->outerRadius;
        delete[] quintupletsInCPU->regressionRadius;
        delete quintupletsInCPU;
    }
#endif
#endif

#ifdef Explicit_PT3
    if(pixelTripletsInCPU != nullptr)
    {
        delete[] pixelTripletsInCPU->tripletIndices;
        delete[] pixelTripletsInCPU->pixelSegmentIndices;
        delete[] pixelTripletsInCPU->pixelRadius;
        delete[] pixelTripletsInCPU->tripletRadius;
        delete pixelTripletsInCPU->nPixelTriplets;
        delete pixelTripletsInCPU->totOccupancyPixelTriplets;
        delete pixelTripletsInCPU;
    }
#endif
#ifdef Explicit_PT5
    if(pixelQuintupletsInCPU != nullptr)
    {
        delete[] pixelQuintupletsInCPU->pixelIndices;
        delete[] pixelQuintupletsInCPU->T5Indices;
        delete[] pixelQuintupletsInCPU->isDup;
        delete[] pixelQuintupletsInCPU->score;
        delete pixelQuintupletsInCPU->nPixelQuintuplets;
        delete pixelQuintupletsInCPU->totOccupancyPixelQuintuplets;
        delete pixelQuintupletsInCPU;
    }
#endif

#ifdef Explicit_Track
    if(trackCandidatesInCPU != nullptr)
    {
        delete[] trackCandidatesInCPU->objectIndices;
        delete[] trackCandidatesInCPU->trackCandidateType;
        delete[] trackCandidatesInCPU->nTrackCandidates;
        delete[] trackCandidatesInCPU->hitIndices;
        delete[] trackCandidatesInCPU->logicalLayers;
        delete[] trackCandidatesInCPU->partOfExtension;
        delete trackCandidatesInCPU;
    }
#endif
#ifdef Explicit_Extensions
    if(trackExtensionsInCPU != nullptr)
    {
        delete[] trackExtensionsInCPU->nTrackExtensions;
        delete[] trackExtensionsInCPU->totOccupancyTrackExtensions;
        delete[] trackExtensionsInCPU->constituentTCTypes;
        delete[] trackExtensionsInCPU->constituentTCIndices;
        delete[] trackExtensionsInCPU->nLayerOverlaps;
        delete[] trackExtensionsInCPU->nHitOverlaps;
        delete[] trackExtensionsInCPU->isDup;
        delete[] trackExtensionsInCPU->regressionRadius;

        delete trackExtensionsInCPU;
    }
#endif
#ifdef Explicit_Module
    if(modulesInCPU != nullptr)
    {
        delete[] modulesInCPU->nLowerModules;
        delete[] modulesInCPU->nModules;
        delete[] modulesInCPU->detIds;
        delete[] modulesInCPU->isLower;
        delete[] modulesInCPU->layers;
        delete[] modulesInCPU->subdets;
        delete[] modulesInCPU->rings;
        delete[] modulesInCPU;
    }
    if(modulesInCPUFull != nullptr)
    {
        delete[] modulesInCPUFull->detIds;
        delete[] modulesInCPUFull->moduleMap;
        delete[] modulesInCPUFull->nConnectedModules;
        delete[] modulesInCPUFull->drdzs;
        delete[] modulesInCPUFull->slopes;
        delete[] modulesInCPUFull->nModules;
        delete[] modulesInCPUFull->nLowerModules;
        delete[] modulesInCPUFull->layers;
        delete[] modulesInCPUFull->rings;
        delete[] modulesInCPUFull->modules;
        delete[] modulesInCPUFull->rods;
        delete[] modulesInCPUFull->subdets;
        delete[] modulesInCPUFull->sides;
        delete[] modulesInCPUFull->isInverted;
        delete[] modulesInCPUFull->isLower;


        delete[] modulesInCPUFull->moduleType;
        delete[] modulesInCPUFull->moduleLayerType;
        delete[] modulesInCPUFull;
    }
#endif
}
void SDL::Event::resetEvent()
{
#ifdef CACHE_ALLOC
    if(hitsInGPU){hitsInGPU->freeMemoryCache();}
    if(mdsInGPU){mdsInGPU->freeMemoryCache();}
    if(quintupletsInGPU){quintupletsInGPU->freeMemoryCache();}
    if(rangesInGPU){rangesInGPU->freeMemoryCache();}
    if(segmentsInGPU){segmentsInGPU->freeMemoryCache();}
    if(tripletsInGPU){tripletsInGPU->freeMemoryCache();}
    if(tripletsInwardInGPU){tripletsInwardInGPU->freeMemoryCache();}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemoryCache();}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemoryCache();}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemoryCache();}
    if(trackExtensionsInGPU){trackExtensionsInGPU->freeMemoryCache();}

#else
    if(hitsInGPU){hitsInGPU->freeMemory(stream);}
    if(quintupletsInGPU){quintupletsInGPU->freeMemory(stream);}
    if(rangesInGPU){rangesInGPU->freeMemory();}
    if(mdsInGPU){mdsInGPU->freeMemory(stream);}
    if(segmentsInGPU){segmentsInGPU->freeMemory(stream);}
    if(tripletsInGPU){tripletsInGPU->freeMemory(stream);}
    if(tripletsInwardInGPU){tripletsInwardInGPU->freeMemory(stream);}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemory(stream);}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemory(stream);}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemory(stream);}
    if(trackExtensionsInGPU){trackExtensionsInGPU->freeMemory(stream);}
#endif
    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_triplets_by_layer_barrel_[i] = 0;
        n_trackCandidates_by_layer_barrel_[i] = 0;
        n_quintuplets_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
            n_segments_by_layer_endcap_[i] = 0;
            n_triplets_by_layer_endcap_[i] = 0;
            n_trackCandidates_by_layer_endcap_[i] = 0;
            n_quintuplets_by_layer_endcap_[i] = 0;
        }
    }
    if(hitsInGPU){hipHostFree(hitsInGPU);
    hitsInGPU = nullptr;}
    if(mdsInGPU){hipHostFree(mdsInGPU);
    mdsInGPU = nullptr;}
    if(rangesInGPU){hipHostFree(rangesInGPU);
    rangesInGPU = nullptr;}
    if(segmentsInGPU){hipHostFree(segmentsInGPU);
    segmentsInGPU = nullptr;}
    if(tripletsInGPU){hipHostFree(tripletsInGPU);
    tripletsInGPU = nullptr;}
      if(quintupletsInGPU){hipHostFree(quintupletsInGPU);
      quintupletsInGPU = nullptr;}
    if(trackCandidatesInGPU){hipHostFree(trackCandidatesInGPU);
    trackCandidatesInGPU = nullptr;}
    if(pixelTripletsInGPU){hipHostFree(pixelTripletsInGPU);
    pixelTripletsInGPU = nullptr;}
    if(pixelQuintupletsInGPU){hipHostFree(pixelQuintupletsInGPU);
    pixelQuintupletsInGPU = nullptr;}
    if(trackExtensionsInGPU){hipHostFree(trackExtensionsInGPU);
    trackExtensionsInGPU = nullptr;}
#ifdef Explicit_Hit
    if(hitsInCPU != nullptr)
    {
        delete[] hitsInCPU->idxs;
        delete[] hitsInCPU->xs;
        delete[] hitsInCPU->ys;
        delete[] hitsInCPU->zs;
        delete[] hitsInCPU->moduleIndices;
        delete hitsInCPU->nHits;
        delete hitsInCPU;
        hitsInCPU = nullptr;
    }
    if(rangesInCPU != nullptr)
    {
        delete[] rangesInCPU->hitRanges;
        delete[] rangesInCPU->quintupletModuleIndices;
        delete rangesInCPU;
        rangesInCPU = nullptr;
    }
#endif
#ifdef Explicit_MD
    if(mdsInCPU != nullptr)
    {
        delete[] mdsInCPU->anchorHitIndices;
        delete[] mdsInCPU->nMDs;
        delete[] mdsInCPU->totOccupancyMDs;
        delete mdsInCPU;
        mdsInCPU = nullptr;
    }
#endif
#ifdef Explicit_Seg
    if(segmentsInCPU != nullptr)
    {
        delete[] segmentsInCPU->mdIndices;
        delete[] segmentsInCPU->nSegments;
        delete[] segmentsInCPU->totOccupancySegments;
        delete[] segmentsInCPU->innerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->outerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->ptIn;
        delete[] segmentsInCPU->eta;
        delete[] segmentsInCPU->phi;
        delete segmentsInCPU;
        segmentsInCPU = nullptr;
    }
#endif
#ifdef Explicit_Trips
    if(tripletsInCPU != nullptr)
    {
        delete[] tripletsInCPU->segmentIndices;
        delete[] tripletsInCPU->nTriplets;
        delete[] tripletsInCPU->totOccupancyTriplets;
        delete[] tripletsInCPU->betaIn;
        delete[] tripletsInCPU->betaOut;
        delete[] tripletsInCPU->pt_beta;
        delete[] tripletsInCPU->logicalLayers;
        delete[] tripletsInCPU->lowerModuleIndices;
        delete[] tripletsInCPU->hitIndices;
        delete tripletsInCPU;
        tripletsInCPU = nullptr;
    }
#endif
#ifdef Explicit_T5
#ifdef FINAL_T5
    if(quintupletsInCPU != nullptr)
    {
        delete[] quintupletsInCPU->tripletIndices;
        delete[] quintupletsInCPU->nQuintuplets;
        delete[] quintupletsInCPU->totOccupancyQuintuplets;
        delete[] quintupletsInCPU->lowerModuleIndices;
        delete[] quintupletsInCPU->innerRadius;
        delete[] quintupletsInCPU->outerRadius;
        delete[] quintupletsInCPU->regressionRadius;
        delete quintupletsInCPU;
        quintupletsInCPU = nullptr;
    }
#endif
#endif

#ifdef Explicit_PT3
    if(pixelTripletsInCPU != nullptr)
    {
        delete[] pixelTripletsInCPU->tripletIndices;
        delete[] pixelTripletsInCPU->pixelSegmentIndices;
        delete[] pixelTripletsInCPU->pixelRadius;
        delete[] pixelTripletsInCPU->tripletRadius;
        delete pixelTripletsInCPU->nPixelTriplets;
        delete pixelTripletsInCPU->totOccupancyPixelTriplets;
        delete pixelTripletsInCPU;
        pixelTripletsInCPU = nullptr;
    }
#endif
#ifdef Explicit_PT5
    if(pixelQuintupletsInCPU != nullptr)
    {
        delete[] pixelQuintupletsInCPU->pixelIndices;
        delete[] pixelQuintupletsInCPU->T5Indices;
        delete[] pixelQuintupletsInCPU->isDup;
        delete[] pixelQuintupletsInCPU->score;
        delete pixelQuintupletsInCPU->nPixelQuintuplets;
        delete pixelQuintupletsInCPU->totOccupancyPixelQuintuplets;
        delete pixelQuintupletsInCPU;
        pixelQuintupletsInCPU = nullptr;
    }
#endif
#ifdef Explicit_Track
    if(trackCandidatesInCPU != nullptr)
    {
        delete[] trackCandidatesInCPU->objectIndices;
        delete[] trackCandidatesInCPU->trackCandidateType;
        delete[] trackCandidatesInCPU->nTrackCandidates;
        delete[] trackCandidatesInCPU->logicalLayers;
        delete[] trackCandidatesInCPU->hitIndices;
        delete[] trackCandidatesInCPU->lowerModuleIndices;
        delete trackCandidatesInCPU;
        trackCandidatesInCPU = nullptr;
    }
#endif
#ifdef Explicit_Extensions
    if(trackExtensionsInCPU != nullptr)
    {
        delete[] trackExtensionsInCPU->nTrackExtensions;
        delete[] trackExtensionsInCPU->totOccupancyTrackExtensions;
        delete[] trackExtensionsInCPU->constituentTCTypes;
        delete[] trackExtensionsInCPU->constituentTCIndices;
        delete[] trackExtensionsInCPU->nLayerOverlaps;
        delete[] trackExtensionsInCPU->nHitOverlaps;
        delete[] trackExtensionsInCPU->isDup;
        delete[] trackExtensionsInCPU->regressionRadius;

        delete trackExtensionsInCPU;
        trackExtensionsInCPU = nullptr;
    }
#endif
#ifdef Explicit_Module
    if(modulesInCPU != nullptr)
    {
        delete[] modulesInCPU->nLowerModules;
        delete[] modulesInCPU->nModules;
        delete[] modulesInCPU->detIds;
        delete[] modulesInCPU->isLower;
        delete[] modulesInCPU->layers;
        delete[] modulesInCPU->subdets;
        delete[] modulesInCPU->rings;
        delete[] modulesInCPU;
        modulesInCPU = nullptr;
    }
    if(modulesInCPUFull != nullptr)
    {
        delete[] modulesInCPUFull->detIds;
        delete[] modulesInCPUFull->moduleMap;
        delete[] modulesInCPUFull->nConnectedModules;
        delete[] modulesInCPUFull->drdzs;
        delete[] modulesInCPUFull->slopes;
        delete[] modulesInCPUFull->nModules;
        delete[] modulesInCPUFull->nLowerModules;
        delete[] modulesInCPUFull->layers;
        delete[] modulesInCPUFull->rings;
        delete[] modulesInCPUFull->modules;
        delete[] modulesInCPUFull->rods;
        delete[] modulesInCPUFull->subdets;
        delete[] modulesInCPUFull->sides;
        delete[] modulesInCPUFull->isInverted;
        delete[] modulesInCPUFull->isLower;


        delete[] modulesInCPUFull->moduleType;
        delete[] modulesInCPUFull->moduleLayerType;
        delete[] modulesInCPUFull;
        modulesInCPUFull = nullptr;
    }
#endif


}

void SDL::initModules(const char* moduleMetaDataFilePath)
{
    hipStream_t modStream;
    hipStreamCreate(&modStream);
    if(modulesInGPU == nullptr)
    {
        hipHostMalloc(&modulesInGPU, sizeof(struct SDL::modules));
        //pixelMapping = new pixelMap;
        hipHostMalloc(&pixelMapping, sizeof(struct SDL::pixelMap));
        loadModulesFromFile(*modulesInGPU,nModules,nLowerModules, *pixelMapping,modStream,moduleMetaDataFilePath); //nModules gets filled here
    }
    //resetObjectRanges(*modulesInGPU,nModules,modStream);
    hipStreamSynchronize(modStream);
    hipStreamDestroy(modStream);
}

void SDL::cleanModules()
{
  //#ifdef CACHE_ALLOC
  //freeModulesCache(*modulesInGPU,*pixelMapping); //bug in freeing cached modules. Decided to remove module caching since it doesn't change by event.
  //#else
    hipStream_t modStream;
    hipStreamCreate(&modStream);
    freeModules(*modulesInGPU,*pixelMapping,modStream);
    hipStreamSynchronize(modStream);
    hipStreamDestroy(modStream);
  //#endif
    hipHostFree(modulesInGPU);
    hipHostFree(pixelMapping);
//    hipDeviceReset(); // uncomment for leak check "cuda-memcheck --leak-check full --show-backtrace yes" does not work with caching.
}

void SDL::Event::resetObjectsInModule()
{
    resetObjectRanges(*rangesInGPU,nModules,stream);
}

// Best working hit loading method. Previously named OMP
void SDL::Event::addHitToEvent(std::vector<float> x, std::vector<float> y, std::vector<float> z, std::vector<unsigned int> detId, std::vector<unsigned int> idxInNtuple)
{
    const int loopsize = x.size();// use the actual number of hits instead of a "max"
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    //printf("loopsize %i\n",loopsize);

    if(rangesInGPU == nullptr)
    {

        hipHostMalloc(&rangesInGPU, sizeof(SDL::objectRanges));
        #ifdef Explicit_Hit
    	  createRangesInExplicitMemory(*rangesInGPU, nModules,stream,nLowerModules); //unclear why but this has to be 2*loopsize to avoid crashing later (reported in tracklet allocation). seems to do with nHits values as well. this allows nhits to be set to the correct value of loopsize to get correct results without crashing. still beats the "max hits" so i think this is fine.
        #else
        createRangesInUnifiedMemory(*rangesInGPU,nModules,stream,nLowerModules);
        #endif
    resetObjectsInModule();
    }
    if(hitsInGPU == nullptr)
    {

        hipHostMalloc(&hitsInGPU, sizeof(SDL::hits));
        #ifdef Explicit_Hit
    	  createHitsInExplicitMemory(*hitsInGPU, 2*loopsize,stream); //unclear why but this has to be 2*loopsize to avoid crashing later (reported in tracklet allocation). seems to do with nHits values as well. this allows nhits to be set to the correct value of loopsize to get correct results without crashing. still beats the "max hits" so i think this is fine.
        #else
        createHitsInUnifiedMemory(*hitsInGPU,2*loopsize,0,stream);
        #endif
    }
hipStreamSynchronize(stream);


    float* host_x;// = &x[0]; // convert from std::vector to host array easily since vectors are ordered
    float* host_y;// = &y[0];
    float* host_z;// = &z[0];
    unsigned int* host_detId;// = &detId[0];
    unsigned int* host_idxs;// = &idxInNtuple[0];
    hipHostMalloc(&host_x,sizeof(float)*loopsize);
    hipHostMalloc(&host_y,sizeof(float)*loopsize);
    hipHostMalloc(&host_z,sizeof(float)*loopsize);
    hipHostMalloc(&host_detId,sizeof(unsigned int)*loopsize);
    hipHostMalloc(&host_idxs,sizeof(unsigned int)*loopsize);

    //float* host_x = &x[0]; // convert from std::vector to host array easily since vectors are ordered
    //float* host_y = &y[0];
    //float* host_z = &z[0];
    //unsigned int* host_detId = &detId[0];
    //unsigned int* host_idxs = &idxInNtuple[0];

    float* host_phis;
    float* host_etas;
    unsigned int* host_moduleIndex;
    float* host_rts;
    //float* host_idxs;
    float* host_highEdgeXs;
    float* host_highEdgeYs;
    float* host_lowEdgeXs;
    float* host_lowEdgeYs;
    hipHostMalloc(&host_moduleIndex, sizeof(float)*loopsize);
    hipHostMalloc(&host_phis,sizeof(float)*loopsize);
    hipHostMalloc(&host_etas,sizeof(float)*loopsize);
    hipHostMalloc(&host_rts,sizeof(float)*loopsize);
    //hipHostMalloc(&host_idxs,sizeof(unsigned int)*loopsize);
    hipHostMalloc(&host_highEdgeXs,sizeof(float)*loopsize);
    hipHostMalloc(&host_highEdgeYs,sizeof(float)*loopsize);
    hipHostMalloc(&host_lowEdgeXs,sizeof(float)*loopsize);
    hipHostMalloc(&host_lowEdgeYs,sizeof(float)*loopsize);


    short* module_layers;
    short* module_subdet;
    uint16_t* module_partnerModuleIndices;
    int* module_hitRanges;
    int* module_hitRangesUpper;
    int* module_hitRangesLower;
    int8_t* module_hitRangesnUpper;
    int8_t* module_hitRangesnLower;
    ModuleType* module_moduleType;
    hipHostMalloc(&module_layers,sizeof(short)*nModules);
    hipHostMalloc(&module_subdet,sizeof(short)*nModules);
    hipHostMalloc(&module_partnerModuleIndices, sizeof(uint16_t) * nModules);
    hipHostMalloc(&module_hitRanges,sizeof(int)*2*nModules);
    hipHostMalloc(&module_hitRangesUpper,sizeof(int)*nModules);
    hipHostMalloc(&module_hitRangesLower,sizeof(int)*nModules);
    hipHostMalloc(&module_hitRangesnUpper,sizeof(int8_t)*nModules);
    hipHostMalloc(&module_hitRangesnLower,sizeof(int8_t)*nModules);
    hipHostMalloc(&module_moduleType,sizeof(ModuleType)*nModules);

    hipMemcpyAsync(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_subdet,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_partnerModuleIndices, modulesInGPU->partnerModuleIndices, nModules * sizeof(uint16_t), hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(module_hitRanges,rangesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_hitRangesLower,rangesInGPU->hitRangesLower,nModules*sizeof(int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_hitRangesUpper,rangesInGPU->hitRangesUpper,nModules*sizeof(int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_hitRangesnLower,rangesInGPU->hitRangesnLower,nModules*sizeof(int8_t),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_hitRangesnUpper,rangesInGPU->hitRangesnUpper,nModules*sizeof(int8_t),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_moduleType,modulesInGPU->moduleType,nModules*sizeof(ModuleType),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);


  for (int ihit=0; ihit<loopsize;ihit++){
    host_x[ihit] = x.at(ihit); // convert from std::vector to host array easily since vectors are ordered
    host_y[ihit] = y.at(ihit);
    host_z[ihit] = z.at(ihit);
    host_detId[ihit] = detId.at(ihit);
    host_idxs[ihit] = idxInNtuple.at(ihit);

    unsigned int moduleLayer = module_layers[(*detIdToIndex)[host_detId[ihit]]];
    unsigned int subdet = module_subdet[(*detIdToIndex)[host_detId[ihit]]];
    host_moduleIndex[ihit] = (*detIdToIndex)[host_detId[ihit]]; //module indices appropriately done


      host_rts[ihit] = sqrt(host_x[ihit]*host_x[ihit] + host_y[ihit]*host_y[ihit]);
      host_phis[ihit] = phi(host_x[ihit],host_y[ihit],host_z[ihit]);
      host_etas[ihit] = ((host_z[ihit]>0)-(host_z[ihit]<0))* std::acosh(sqrt(host_x[ihit]*host_x[ihit]+host_y[ihit]*host_y[ihit]+host_z[ihit]*host_z[ihit])/host_rts[ihit]);
//// This part i think has a race condition. so this is not run in parallel.
      unsigned int this_index = host_moduleIndex[ihit];
      if(module_subdet[this_index] == Endcap && module_moduleType[this_index] == TwoS)
      {
          float xhigh, yhigh, xlow, ylow;
          getEdgeHits(host_detId[ihit],host_x[ihit],host_y[ihit],xhigh,yhigh,xlow,ylow);
          host_highEdgeXs[ihit] = xhigh;
          host_highEdgeYs[ihit] = yhigh;
          host_lowEdgeXs[ihit] = xlow;
          host_lowEdgeYs[ihit] = ylow;

      }

      //set the hit ranges appropriately in the modules struct

      ////start the index rolling if the module is encountered for the first time
      ////always update the end index
      //modulesInGPU->hitRanges[this_index * 2 + 1] = ihit;
      //start the index rolling if the module is encountered for the first time
      if(module_hitRanges[this_index * 2] == -1)
      {
          module_hitRanges[this_index * 2] = ihit;
      }
      //always update the end index
      module_hitRanges[this_index * 2 + 1] = ihit;
      //printf("ranges: %u %u %u\n",this_index,module_hitRanges[this_index * 2],module_hitRanges[this_index * 2+1]);

  }
//range testing
    for(uint16_t lowerModuleIndex = 0; lowerModuleIndex< nLowerModules; lowerModuleIndex++)
    {

        uint16_t upperModuleIndex = module_partnerModuleIndices[lowerModuleIndex];

        int lowerHitRanges = module_hitRanges[lowerModuleIndex*2];
        int upperHitRanges = module_hitRanges[upperModuleIndex*2];

        if(module_hitRanges[lowerModuleIndex * 2] == -1) continue; //return;
        if(module_hitRanges[upperModuleIndex * 2] == -1) continue; //return;
        module_hitRangesLower[lowerModuleIndex] =  module_hitRanges[lowerModuleIndex * 2]; 
        module_hitRangesUpper[lowerModuleIndex] =  module_hitRanges[upperModuleIndex * 2];
        module_hitRangesnLower[lowerModuleIndex] = module_hitRanges[lowerModuleIndex * 2 + 1] - module_hitRanges[lowerModuleIndex * 2] + 1;
        module_hitRangesnUpper[lowerModuleIndex] = module_hitRanges[upperModuleIndex * 2 + 1] - module_hitRanges[upperModuleIndex * 2] + 1;
        //printf("hits %d %d %d\n",lowerModuleArrayIndex,module_hitRangesLower[lowerModuleArrayIndex],module_hitRangesUpper[lowerModuleArrayIndex]);
    }
//simply copy the host arrays to the hitsInGPU struct
    hipMemcpyAsync(hitsInGPU->xs,host_x,loopsize*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->ys,host_y,loopsize*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->zs,host_z,loopsize*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->rts,host_rts,loopsize*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->phis,host_phis,loopsize*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->etas,host_etas,loopsize*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->idxs,host_idxs,loopsize*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->moduleIndices,host_moduleIndex,loopsize*sizeof(uint16_t),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->highEdgeXs,host_highEdgeXs,loopsize*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->highEdgeYs,host_highEdgeYs,loopsize*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->lowEdgeXs,host_lowEdgeXs,loopsize*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->lowEdgeYs,host_lowEdgeYs,loopsize*sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitsInGPU->nHits,&loopsize,sizeof(unsigned int),hipMemcpyHostToDevice,stream);// value can't correctly be set in hit allocation
    hipMemcpyAsync(rangesInGPU->hitRanges,module_hitRanges,nModules*2*sizeof(int),hipMemcpyHostToDevice,stream);// value can't correctly be set in hit allocation
    hipMemcpyAsync(rangesInGPU->hitRangesLower,module_hitRangesLower,nModules*sizeof(int),hipMemcpyHostToDevice,stream);// value can't correctly be set in hit allocation
    hipMemcpyAsync(rangesInGPU->hitRangesUpper,module_hitRangesUpper,nModules*sizeof(int),hipMemcpyHostToDevice,stream);// value can't correctly be set in hit allocation
    hipMemcpyAsync(rangesInGPU->hitRangesnLower,module_hitRangesnLower,nModules*sizeof(int8_t),hipMemcpyHostToDevice,stream);// value can't correctly be set in hit allocation
    hipMemcpyAsync(rangesInGPU->hitRangesnUpper,module_hitRangesnUpper,nModules*sizeof(int8_t),hipMemcpyHostToDevice,stream);// value can't correctly be set in hit allocation
hipStreamSynchronize(stream);

    hipHostFree(host_rts);
    hipHostFree(host_phis);
    hipHostFree(host_etas);
    hipHostFree(host_moduleIndex);
    hipHostFree(host_highEdgeXs);
    hipHostFree(host_highEdgeYs);
    hipHostFree(host_lowEdgeXs);
    hipHostFree(host_lowEdgeYs);
    hipHostFree(module_layers);
    hipHostFree(module_subdet);
    hipHostFree(module_partnerModuleIndices);
    hipHostFree(module_hitRanges);
    hipHostFree(module_hitRangesLower);
    hipHostFree(module_hitRangesUpper);
    hipHostFree(module_hitRangesnLower);
    hipHostFree(module_hitRangesnUpper);
    hipHostFree(module_moduleType);
    hipHostFree(host_x);
    hipHostFree(host_y);
    hipHostFree(host_z);
    hipHostFree(host_detId);
    hipHostFree(host_idxs);

}
__global__ void addPixelSegmentToEventKernel(unsigned int* hitIndices0,unsigned int* hitIndices1,unsigned int* hitIndices2,unsigned int* hitIndices3, float* dPhiChange, float* ptIn, float* ptErr, float* px, float* py, float* pz, float* eta, float* etaErr,float* phi, uint16_t pixelModuleIndex, struct SDL::modules& modulesInGPU, struct SDL::objectRanges& rangesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU,const int size, int* superbin, int8_t* pixelType, short* isQuad)
{

    for( int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < size; tid += blockDim.x*gridDim.x)
    {

      unsigned int innerMDIndex = rangesInGPU.miniDoubletModuleIndices[pixelModuleIndex] + 2*(tid);
      unsigned int outerMDIndex = rangesInGPU.miniDoubletModuleIndices[pixelModuleIndex] + 2*(tid) +1;
      unsigned int pixelSegmentIndex = rangesInGPU.segmentModuleIndices[pixelModuleIndex] + tid;

#ifdef CUT_VALUE_DEBUG
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,outerMDIndex);
#else
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,outerMDIndex);
#endif

    int hits1[4];
    hits1[0] = hitsInGPU.idxs[mdsInGPU.anchorHitIndices[innerMDIndex]];
    hits1[1] = hitsInGPU.idxs[mdsInGPU.anchorHitIndices[outerMDIndex]];
    hits1[2] = hitsInGPU.idxs[mdsInGPU.outerHitIndices[innerMDIndex]];
    hits1[3] = hitsInGPU.idxs[mdsInGPU.outerHitIndices[outerMDIndex]];
    float rsum=0, zsum=0, r2sum=0,rzsum=0;
    for(int i =0; i < 4; i++)
    {
        rsum += hitsInGPU.rts[hits1[i]];
        zsum += hitsInGPU.zs[hits1[i]];
        r2sum += hitsInGPU.rts[hits1[i]]*hitsInGPU.rts[hits1[i]];
        rzsum += hitsInGPU.rts[hits1[i]]*hitsInGPU.zs[hits1[i]];
    }
    float slope_lsq = (4*rzsum - rsum*zsum)/(4*r2sum-rsum*rsum);
    float b = (r2sum*zsum-rsum*rzsum)/(r2sum*4-rsum*rsum);
    float score_lsq=0;
    for( int i=0; i <4; i++)
    {
        float z = hitsInGPU.zs[hits1[i]];
        float r = hitsInGPU.rts[hits1[i]];
        float var_lsq = slope_lsq*(r)+b - z;
        score_lsq += abs(var_lsq);//(var_lsq*var_lsq) / (err*err);
    }
    addPixelSegmentToMemory(segmentsInGPU, mdsInGPU, modulesInGPU, innerMDIndex, outerMDIndex, pixelModuleIndex, hitIndices0[tid], hitIndices2[tid], dPhiChange[tid], ptIn[tid], ptErr[tid], px[tid], py[tid], pz[tid], etaErr[tid], eta[tid], phi[tid], pixelSegmentIndex, tid, superbin[tid], pixelType[tid],isQuad[tid],score_lsq);
    }
}
void SDL::Event::addPixelSegmentToEvent(std::vector<unsigned int> hitIndices0,std::vector<unsigned int> hitIndices1,std::vector<unsigned int> hitIndices2,std::vector<unsigned int> hitIndices3, std::vector<float> dPhiChange, std::vector<float> ptIn, std::vector<float> ptErr, std::vector<float> px, std::vector<float> py, std::vector<float> pz, std::vector<float> eta, std::vector<float> etaErr, std::vector<float> phi, std::vector<int> superbin, std::vector<int8_t> pixelType, std::vector<short> isQuad)
{
    if(mdsInGPU == nullptr)
    {
        hipHostMalloc(&mdsInGPU, sizeof(SDL::miniDoublets));
        //hardcoded range numbers for this will come from studies!
        unsigned int nTotalMDs;
        createMDArrayRanges(*modulesInGPU, *rangesInGPU, nLowerModules, nTotalMDs, stream, N_MAX_MD_PER_MODULES, N_MAX_PIXEL_MD_PER_MODULES);

#ifdef Explicit_MD
    	createMDsInExplicitMemory(*mdsInGPU, nTotalMDs, nLowerModules, N_MAX_PIXEL_MD_PER_MODULES,stream);
#else
    	createMDsInUnifiedMemory(*mdsInGPU, nTotalMDs, nLowerModules, N_MAX_PIXEL_MD_PER_MODULES,stream);
#endif
        hipMemcpyAsync(mdsInGPU->nMemoryLocations, &nTotalMDs, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
         hipStreamSynchronize(stream);

    }
    if(segmentsInGPU == nullptr)
    {
        hipHostMalloc(&segmentsInGPU, sizeof(SDL::segments));
        //hardcoded range numbers for this will come from studies!
        unsigned int nTotalSegments;
        createSegmentArrayRanges(*modulesInGPU, *rangesInGPU, *mdsInGPU, nLowerModules, nTotalSegments, stream, N_MAX_SEGMENTS_PER_MODULE, N_MAX_PIXEL_SEGMENTS_PER_MODULE);

#ifdef Explicit_Seg
        createSegmentsInExplicitMemory(*segmentsInGPU, nTotalSegments, nLowerModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE,stream);
#else
        createSegmentsInUnifiedMemory(*segmentsInGPU, nTotalSegments,  nLowerModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE,stream);
#endif
        hipMemcpyAsync(segmentsInGPU->nMemoryLocations, &nTotalSegments, sizeof(unsigned int), hipMemcpyHostToDevice, stream);;
        hipStreamSynchronize(stream);


    }
    hipStreamSynchronize(stream);
    const int size = ptIn.size();
    uint16_t pixelModuleIndex = (*detIdToIndex)[1];
    unsigned int* hitIndices0_host = &hitIndices0[0];
    unsigned int* hitIndices1_host = &hitIndices1[0];
    unsigned int* hitIndices2_host = &hitIndices2[0];
    unsigned int* hitIndices3_host = &hitIndices3[0];
    float* dPhiChange_host = &dPhiChange[0];
    float* ptIn_host = &ptIn[0];
    float* ptErr_host = &ptErr[0];
    float* px_host = &px[0];
    float* py_host = &py[0];
    float* pz_host = &pz[0];
    float* etaErr_host = &etaErr[0];
    float* eta_host = &eta[0];
    float* phi_host = &phi[0];
    int* superbin_host = &superbin[0];
    int8_t* pixelType_host = &pixelType[0];
    short* isQuad_host = &isQuad[0];

    unsigned int* hitIndices0_dev;
    unsigned int* hitIndices1_dev;
    unsigned int* hitIndices2_dev;
    unsigned int* hitIndices3_dev;
    float* dPhiChange_dev;
    float* ptIn_dev;
    float* ptErr_dev;
    float* px_dev;
    float* py_dev;
    float* pz_dev;
    float* etaErr_dev;
    float* eta_dev;
    float* phi_dev;
    int* superbin_dev;
    int8_t* pixelType_dev;
    short* isQuad_dev;

    hipMalloc(&hitIndices0_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices1_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices2_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices3_dev,size*sizeof(unsigned int));
    hipMalloc(&dPhiChange_dev,size*sizeof(unsigned int));
    hipMalloc(&ptIn_dev,size*sizeof(unsigned int));
    hipMalloc(&ptErr_dev,size*sizeof(unsigned int));
    hipMalloc(&px_dev,size*sizeof(unsigned int));
    hipMalloc(&py_dev,size*sizeof(unsigned int));
    hipMalloc(&pz_dev,size*sizeof(unsigned int));
    hipMalloc(&etaErr_dev,size*sizeof(unsigned int));
    hipMalloc(&eta_dev, size*sizeof(unsigned int));
    hipMalloc(&phi_dev, size*sizeof(unsigned int));
    hipMalloc(&superbin_dev,size*sizeof(int));
    hipMalloc(&pixelType_dev,size*sizeof(int8_t));
    hipMalloc(&isQuad_dev,size*sizeof(short));

    hipMemcpyAsync(hitIndices0_dev,hitIndices0_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitIndices1_dev,hitIndices1_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitIndices2_dev,hitIndices2_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(hitIndices3_dev,hitIndices3_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(dPhiChange_dev,dPhiChange_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(ptIn_dev,ptIn_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(ptErr_dev,ptErr_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(px_dev,px_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(py_dev,py_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(pz_dev,pz_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(etaErr_dev,etaErr_host,size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(eta_dev, eta_host, size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(phi_dev, phi_host, size*sizeof(unsigned int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(superbin_dev,superbin_host,size*sizeof(int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(pixelType_dev,pixelType_host,size*sizeof(int8_t),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(isQuad_dev,isQuad_host,size*sizeof(short),hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);

    unsigned int nThreads = 256;
    unsigned int nBlocks =  MAX_BLOCKS;//size % nThreads == 0 ? size/nThreads : size/nThreads + 1;

    addPixelSegmentToEventKernel<<<nBlocks,nThreads,0,stream>>>(hitIndices0_dev,hitIndices1_dev,hitIndices2_dev,hitIndices3_dev,dPhiChange_dev,ptIn_dev,ptErr_dev,px_dev,py_dev,pz_dev,eta_dev, etaErr_dev, phi_dev, pixelModuleIndex, *modulesInGPU, *rangesInGPU, *hitsInGPU,*mdsInGPU,*segmentsInGPU,size, superbin_dev, pixelType_dev,isQuad_dev);

   //hipDeviceSynchronize();
   hipStreamSynchronize(stream);
   hipMemcpyAsync(&(segmentsInGPU->nSegments)[pixelModuleIndex], &size, sizeof(unsigned int), hipMemcpyHostToDevice,stream);
   hipMemcpyAsync(&(segmentsInGPU->totOccupancySegments)[pixelModuleIndex], &size, sizeof(unsigned int), hipMemcpyHostToDevice,stream);
   unsigned int mdSize = 2 * size;
   hipMemcpyAsync(&(mdsInGPU->nMDs)[pixelModuleIndex], &mdSize, sizeof(unsigned int), hipMemcpyHostToDevice,stream);
   hipMemcpyAsync(&(mdsInGPU->totOccupancyMDs)[pixelModuleIndex], &mdSize, sizeof(unsigned int), hipMemcpyHostToDevice,stream);
   hipStreamSynchronize(stream);

    //hipFreeAsync(hitIndices0_dev,stream);
    //hipFreeAsync(hitIndices1_dev,stream);
    //hipFreeAsync(hitIndices2_dev,stream);
    //hipFreeAsync(hitIndices3_dev,stream);
    //hipFreeAsync(dPhiChange_dev,stream);
    //hipFreeAsync(ptIn_dev,stream);
    //hipFreeAsync(ptErr_dev,stream);
    //hipFreeAsync(px_dev,stream);
    //hipFreeAsync(py_dev,stream);
    //hipFreeAsync(pz_dev,stream);
    //hipFreeAsync(etaErr_dev,stream);
    //hipFreeAsync(eta_dev,stream);
    //hipFreeAsync(phi_dev,stream);
    //hipFreeAsync(superbin_dev,stream);
    //hipFreeAsync(pixelType_dev,stream);
    //hipFreeAsync(isQuad_dev,stream);
  
    hipFree(hitIndices0_dev);
    hipFree(hitIndices1_dev);
    hipFree(hitIndices2_dev);
    hipFree(hitIndices3_dev);
    hipFree(dPhiChange_dev);
    hipFree(ptIn_dev);
    hipFree(ptErr_dev);
    hipFree(px_dev);
    hipFree(py_dev);
    hipFree(pz_dev);
    hipFree(etaErr_dev);
    hipFree(eta_dev);
    hipFree(phi_dev);
    hipFree(superbin_dev);
    hipFree(pixelType_dev);
    hipFree(isQuad_dev);
  
hipStreamSynchronize(stream);
}

void SDL::Event::addMiniDoubletsToEvent()
{
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        if(mdsInGPU->nMDs[i] == 0 or rangesInGPU->hitRanges[i * 2] == -1)
        {
            rangesInGPU->mdRanges[i * 2] = -1;
            rangesInGPU->mdRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU->mdRanges[i * 2] = rangesInGPU->miniDoubletModuleIndices[i];
            rangesInGPU->mdRanges[i * 2 + 1] = rangesInGPU->miniDoubletModuleIndices[i] + mdsInGPU->nMDs[i] - 1;

            if(modulesInGPU->subdets[i] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[modulesInGPU->layers[i] -1] += mdsInGPU->nMDs[i];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[modulesInGPU->layers[i] - 1] += mdsInGPU->nMDs[i];
            }

        }
    }
}
void SDL::Event::addMiniDoubletsToEventExplicit()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    unsigned int* nMDsCPU;
    hipHostMalloc(&nMDsCPU, nLowerModules * sizeof(unsigned int));
    hipMemcpyAsync(nMDsCPU,mdsInGPU->nMDs,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    short* module_subdets;
    hipHostMalloc(&module_subdets, nLowerModules* sizeof(short));
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_mdRanges;
    hipHostMalloc(&module_mdRanges, nLowerModules* 2*sizeof(int));
    hipMemcpyAsync(module_mdRanges,rangesInGPU->mdRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    hipHostMalloc(&module_layers, nLowerModules * sizeof(short));
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_hitRanges;
    hipHostMalloc(&module_hitRanges, nLowerModules* 2*sizeof(int));
    hipMemcpyAsync(module_hitRanges,rangesInGPU->hitRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);

    int* module_miniDoubletModuleIndices;
    hipHostMalloc(&module_miniDoubletModuleIndices, nLowerModules * sizeof(int));
    hipMemcpyAsync(module_miniDoubletModuleIndices, rangesInGPU->miniDoubletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        if(nMDsCPU[i] == 0 or module_hitRanges[i * 2] == -1)
        {
            module_mdRanges[i * 2] = -1;
            module_mdRanges[i * 2 + 1] = -1;
        }
        else
        {
            module_mdRanges[i * 2] = module_miniDoubletModuleIndices[i] ;
            module_mdRanges[i * 2 + 1] = module_miniDoubletModuleIndices[i] + nMDsCPU[i] - 1;

            if(module_subdets[i] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[module_layers[i] -1] += nMDsCPU[i];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[module_layers[i] - 1] += nMDsCPU[i];
            }

        }
    }
    hipMemcpyAsync(rangesInGPU->mdRanges,module_mdRanges,nLowerModules*2*sizeof(int),hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);
    hipHostFree(nMDsCPU);
    hipHostFree(module_subdets);
    hipHostFree(module_mdRanges);
    hipHostFree(module_layers);
    hipHostFree(module_hitRanges);
    hipHostFree(module_miniDoubletModuleIndices);
}
void SDL::Event::addSegmentsToEvent()
{
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        if(segmentsInGPU->nSegments[i] == 0)
        {
            rangesInGPU->segmentRanges[i * 2] = -1;
            rangesInGPU->segmentRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU->segmentRanges[i * 2] = rangesInGPU->segmentModuleIndices[i];
            rangesInGPU->segmentRanges[i * 2 + 1] = rangesInGPU->segmentModuleIndices[i] + segmentsInGPU->nSegments[i] - 1;

            if(modulesInGPU->subdets[i] == Barrel)
            {

                n_segments_by_layer_barrel_[modulesInGPU->layers[i] - 1] += segmentsInGPU->nSegments[i];
            }
            else
            {
                n_segments_by_layer_endcap_[modulesInGPU->layers[i] -1] += segmentsInGPU->nSegments[i];
            }
        }
    }
}
void SDL::Event::addSegmentsToEventExplicit()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    unsigned int* nSegmentsCPU;
    hipHostMalloc(&nSegmentsCPU, nLowerModules * sizeof(unsigned int));
    hipMemcpyAsync(nSegmentsCPU,segmentsInGPU->nSegments,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);

    short* module_subdets;
    hipHostMalloc(&module_subdets, nLowerModules* sizeof(short));
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_segmentRanges;
    hipHostMalloc(&module_segmentRanges, nLowerModules* 2*sizeof(int));
    hipMemcpyAsync(module_segmentRanges,rangesInGPU->segmentRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    hipHostMalloc(&module_layers, nLowerModules * sizeof(short));
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);

    int* module_segmentModuleIndices;
    hipHostMalloc(&module_segmentModuleIndices, nLowerModules * sizeof(int));
    hipMemcpyAsync(module_segmentModuleIndices, rangesInGPU->segmentModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        if(nSegmentsCPU[i] == 0)
        {
            module_segmentRanges[i * 2] = -1;
            module_segmentRanges[i * 2 + 1] = -1;
        }
        else
        {
            module_segmentRanges[i * 2] = module_segmentModuleIndices[i];
            module_segmentRanges[i * 2 + 1] = module_segmentModuleIndices[i] + nSegmentsCPU[i] - 1;

            if(module_subdets[i] == Barrel)
            {
                n_segments_by_layer_barrel_[module_layers[i] - 1] += nSegmentsCPU[i];
            }
            else
            {
                n_segments_by_layer_endcap_[module_layers[i] -1] += nSegmentsCPU[i];
            }
        }
    }
    hipMemcpyAsync(rangesInGPU->segmentRanges, module_segmentRanges, nLowerModules * 2 * sizeof(int), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);

    hipHostFree(nSegmentsCPU);
    hipHostFree(module_subdets);
    hipHostFree(module_segmentRanges);
    hipHostFree(module_layers);
    hipHostFree(module_segmentModuleIndices);
}

void SDL::Event::createMiniDoublets()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    //hardcoded range numbers for this will come from studies!
    unsigned int nTotalMDs;
    createMDArrayRanges(*modulesInGPU, *rangesInGPU, nLowerModules, nTotalMDs, stream, N_MAX_MD_PER_MODULES, N_MAX_PIXEL_MD_PER_MODULES);

    if(mdsInGPU == nullptr)
    {
        hipHostMalloc(&mdsInGPU, sizeof(SDL::miniDoublets));
#ifdef Explicit_MD
        //FIXME: Add memory locations for pixel MDs
        createMDsInExplicitMemory(*mdsInGPU, nTotalMDs, nLowerModules, N_MAX_PIXEL_MD_PER_MODULES, stream);

#else
        createMDsInUnifiedMemory(*mdsInGPU, nTotalMDs, nLowerModules, N_MAX_PIXEL_MD_PER_MODULES, stream);
#endif

    }
    hipStreamSynchronize(stream);

    int maxThreadsPerModule=0;
#ifdef Explicit_Module
    int* module_hitRanges;
    hipHostMalloc(&module_hitRanges, nModules* 2*sizeof(int));
    hipMemcpyAsync(module_hitRanges,rangesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    bool* module_isLower;
    hipHostMalloc(&module_isLower, nModules*sizeof(bool));
    hipMemcpyAsync(module_isLower,modulesInGPU->isLower,nModules*sizeof(bool),hipMemcpyDeviceToHost,stream);
    bool* module_isInverted;
    hipHostMalloc(&module_isInverted, nModules*sizeof(bool));
    hipMemcpyAsync(module_isInverted,modulesInGPU->isInverted,nModules*sizeof(bool),hipMemcpyDeviceToHost,stream);
    int* module_partnerModuleIndices;
    hipHostMalloc(&module_partnerModuleIndices, nLowerModules * sizeof(unsigned int));
    hipMemcpyAsync(module_partnerModuleIndices, modulesInGPU->partnerModuleIndices, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    for (uint16_t lowerModuleIndex=0; lowerModuleIndex<nLowerModules; lowerModuleIndex++) 
    {
        uint16_t upperModuleIndex = module_partnerModuleIndices[lowerModuleIndex];
        int lowerHitRanges = module_hitRanges[lowerModuleIndex*2];
        int upperHitRanges = module_hitRanges[upperModuleIndex*2];
        if(lowerHitRanges!=-1 && upperHitRanges!=-1) 
        {
            unsigned int nLowerHits = module_hitRanges[lowerModuleIndex * 2 + 1] - lowerHitRanges + 1;
            unsigned int nUpperHits = module_hitRanges[upperModuleIndex * 2 + 1] - upperHitRanges + 1;
            maxThreadsPerModule = maxThreadsPerModule > (nLowerHits*nUpperHits) ? maxThreadsPerModule : nLowerHits*nUpperHits;
        }
    }
    hipHostFree(module_hitRanges);
    hipHostFree(module_partnerModuleIndices);
    hipHostFree(module_isLower);
    hipHostFree(module_isInverted);
#else
    for (int i=0; i<nLowerModules; i++) 
    {
        int lowerModuleIndex = i;
        int upperModuleIndex = modulesInGPU->partnerModuleIndices[i];
        int lowerHitRanges = rangesInGPU->hitRanges[lowerModuleIndex*2];
        int upperHitRanges = rangesInGPU->hitRanges[upperModuleIndex*2];
        if(lowerHitRanges!=-1&&upperHitRanges!=-1) 
        {
            unsigned int nLowerHits = rangesInGPU->hitRanges[lowerModuleIndex * 2 + 1] - lowerHitRanges + 1;
            unsigned int nUpperHits = rangesInGPU->hitRanges[upperModuleIndex * 2 + 1] - upperHitRanges + 1;
            maxThreadsPerModule = maxThreadsPerModule > (nLowerHits*nUpperHits) ? maxThreadsPerModule : nLowerHits*nUpperHits;
        }
    }
#endif
    //dim3 nThreads(1,128);
    //dim3 nBlocks((nLowerModules % nThreads.x == 0 ? nLowerModules/nThreads.x : nLowerModules/nThreads.x + 1), (maxThreadsPerModule % nThreads.y == 0 ? maxThreadsPerModule/nThreads.y : maxThreadsPerModule/nThreads.y + 1));
    //dim3 nThreads(16,16,4);
    //dim3 nThreads(32,32,1);
    dim3 nThreads(64,16,1);
    dim3 nBlocks(1,MAX_BLOCKS,1);
    //dim3 nBlocks(1,1,MAX_BLOCKS);
    //dim3 nBlocks(1,1,1);

    createMiniDoubletsInGPU<<<nBlocks,nThreads,64*4*16*sizeof(float),stream>>>(*modulesInGPU,*hitsInGPU,*mdsInGPU,*rangesInGPU);

    hipError_t cudaerr = hipGetLastError(); 
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
    //hipDeviceSynchronize();
    hipStreamSynchronize(stream);

#if defined(AddObjects)
#ifdef Explicit_MD
    addMiniDoubletsToEventExplicit();
#else
    addMiniDoubletsToEvent();
#endif
#endif

}

void SDL::Event::createSegmentsWithModuleMap()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    if(segmentsInGPU == nullptr)
    {
        hipHostMalloc(&segmentsInGPU, sizeof(SDL::segments));
#ifdef Explicit_Seg
        createSegmentsInExplicitMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nLowerModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE,stream);
#else
        createSegmentsInUnifiedMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nLowerModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE,stream);
#endif
    }
//    hipStreamSynchronize(stream);
    int max_cModules=0;
    int sq_max_nMDs = 0;
    int nonZeroModules = 0;
    unsigned int* nMDs;
    hipHostMalloc(&nMDs, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpyAsync((void *)nMDs, mdsInGPU->nMDs, nLowerModules*sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

#ifdef Explicit_Module
    uint16_t* module_nConnectedModules;
    hipHostMalloc(&module_nConnectedModules, nLowerModules* sizeof(uint16_t));
    hipMemcpyAsync(module_nConnectedModules,modulesInGPU->nConnectedModules,nLowerModules*sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    uint16_t* module_moduleMap;
    hipHostMalloc(&module_moduleMap, nLowerModules*40* sizeof(uint16_t));
    hipMemcpyAsync(module_moduleMap,modulesInGPU->moduleMap,nLowerModules*40*sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    for (uint16_t innerLowerModuleIndex = 0; innerLowerModuleIndex <nLowerModules; innerLowerModuleIndex++) 
    {
        uint16_t nConnectedModules = module_nConnectedModules[innerLowerModuleIndex];
        unsigned int nInnerMDs = nMDs[innerLowerModuleIndex];
        max_cModules = max(max_cModules, nConnectedModules);
        int limit_local = 0;
        if (nConnectedModules!=0) nonZeroModules++;
        for (uint16_t j=0; j<nConnectedModules; j++) 
        {
            uint16_t outerLowerModuleIndex = module_moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + j];
            int nOuterMDs = nMDs[outerLowerModuleIndex];
            int total = nInnerMDs*nOuterMDs;
            limit_local = max(limit_local,  total);
        }
        sq_max_nMDs = max(sq_max_nMDs, limit_local);
    }
    hipHostFree(module_nConnectedModules);
    hipHostFree(module_moduleMap);
#else
    for (uint16_t innerLowerModuleIndex =0; innerLowerModuleIndex <nLowerModules; innerLowerModuleIndex++) 
    {
        uint16_t nConnectedModules = modulesInGPU->nConnectedModules[innerLowerModuleIndex];
        unsigned int nInnerMDs = nMDs[innerLowerModuleIndex];
        max_cModules = max(max_cModules, nConnectedModules); 
        int limit_local = 0;
        if (nConnectedModules!=0) nonZeroModules++;
        for (uint16_t j=0; j<nConnectedModules; j++) 
        {
            uint16_t outerLowerModuleIndex = modulesInGPU->moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + j];
            int nOuterMDs = nMDs[outerLowerModuleIndex];
            int total = nInnerMDs*nOuterMDs;
            limit_local = limit_local > total ? limit_local : total;
        }
        sq_max_nMDs = sq_max_nMDs > limit_local ? sq_max_nMDs : limit_local;
    }
  #endif
    dim3 nThreads(32,32,1);
    dim3 nBlocks(1,1,MAX_BLOCKS);

    createSegmentsInGPU<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *mdsInGPU, *segmentsInGPU, *rangesInGPU);
    hipHostFree(nMDs);
    hipError_t cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
    hipStreamSynchronize(stream);
#if defined(AddObjects)
#ifdef Explicit_Seg
    addSegmentsToEventExplicit();
#else
    addSegmentsToEvent();
#endif
#endif

}


void SDL::Event::createTriplets()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    if(tripletsInGPU == nullptr)
    {
        hipHostMalloc(&tripletsInGPU, sizeof(SDL::triplets));
        unsigned int maxTriplets;
        createTripletArrayRanges(*modulesInGPU, *rangesInGPU, *segmentsInGPU, nLowerModules, maxTriplets, stream, N_MAX_TRIPLETS_PER_MODULE);
#ifdef Explicit_Trips
        createTripletsInExplicitMemory(*tripletsInGPU, maxTriplets, nLowerModules,stream);
#else
        createTripletsInUnifiedMemory(*tripletsInGPU, maxTriplets, nLowerModules,stream);
#endif
        hipMemcpyAsync(tripletsInGPU->nMemoryLocations, &maxTriplets, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream);

    }
    if(tripletsInwardInGPU == nullptr)
    {
        hipHostMalloc(&tripletsInwardInGPU, sizeof(SDL::triplets));
        unsigned int maxTriplets;
        createInwardTripletArrayRanges(*modulesInGPU, *rangesInGPU, *segmentsInGPU, nLowerModules, maxTriplets, stream, N_MAX_INWARD_TRIPLETS_PER_MODULE);
#ifdef Explicit_Trips
        createTripletsInExcplicitMemory(*tripletsInwardInGPU, maxTriplets, nLowerModules, stream, true);
#else
        createTripletsInUnifiedMemory(*tripletsInwardInGPU, maxTriplets, nLowerModules, stream, true);
#endif
        hipMemcpyAsync(tripletsInwardInGPU->nMemoryLocations, &maxTriplets, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream);
    }

    //TODO:Move this also inside the ranges function
    uint16_t nonZeroModules=0;
    unsigned int max_InnerSeg=0;
    uint16_t *index = (uint16_t*)malloc(nLowerModules*sizeof(unsigned int));
    uint16_t *index_gpu;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(uint16_t));
    unsigned int *nSegments = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    hipMemcpyAsync((void *)nSegments, segmentsInGPU->nSegments, nLowerModules*sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

#ifdef Explicit_Module
    uint16_t* module_nConnectedModules;
    hipHostMalloc(&module_nConnectedModules, nLowerModules* sizeof(uint16_t));
    hipMemcpyAsync(module_nConnectedModules,modulesInGPU->nConnectedModules,nLowerModules*sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    for (uint16_t innerLowerModuleIndex = 0; innerLowerModuleIndex <nLowerModules; innerLowerModuleIndex++) 
    {
        uint16_t nConnectedModules = module_nConnectedModules[innerLowerModuleIndex];
        unsigned int nInnerSegments = nSegments[innerLowerModuleIndex];
        if (nConnectedModules != 0 and nInnerSegments != 0) 
        {
            index[nonZeroModules] = innerLowerModuleIndex;
            nonZeroModules++;
        }
        max_InnerSeg = max(max_InnerSeg, nInnerSegments);
    }
    hipHostFree(module_nConnectedModules);
#else
    for (uint16_t innerLowerModuleIndex = 0; innerLowerModuleIndex < nLowerModules; innerLowerModuleIndex++) 
    {
        uint16_t nConnectedModules = modulesInGPU->nConnectedModules[innerLowerModuleIndex];
        unsigned int nInnerSegments = nSegments[innerLowerModuleIndex];
        if (nConnectedModules != 0 and nInnerSegments != 0) 
        {
            index[nonZeroModules] = innerLowerModuleIndex;
            nonZeroModules++;
        }
        max_InnerSeg = max(max_InnerSeg, nInnerSegments);
    }
#endif
    hipMemcpyAsync(index_gpu, index, nonZeroModules*sizeof(uint16_t), hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);

    dim3 nThreads(16,64,1);
    dim3 nBlocks(1,1,MAX_BLOCKS);
    createTripletsInGPU<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *tripletsInwardInGPU, *rangesInGPU, index_gpu,nonZeroModules);
    hipError_t cudaerr =hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    } 
    hipStreamSynchronize(stream);
    free(nSegments);
    free(index);
    hipFree(index_gpu);

#if defined(AddObjects)
#ifdef Explicit_Trips
    addTripletsToEventExplicit();
#else
    addTripletsToEvent();
#endif
#endif
}

void SDL::Event::createTrackCandidates()
{
    if(trackCandidatesInGPU == nullptr)
    {
        //printf("did this run twice?\n");
        hipHostMalloc(&trackCandidatesInGPU, sizeof(SDL::trackCandidates));
#ifdef Explicit_Track
        createTrackCandidatesInExplicitMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES + N_MAX_PIXEL_TRACK_CANDIDATES,stream);
#else
        createTrackCandidatesInUnifiedMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES + N_MAX_PIXEL_TRACK_CANDIDATES,stream);

#endif
    }

#ifdef FINAL_pT3
    //printf("running final state pT3\n");
    unsigned int nThreadsx = 1024;
    unsigned int nBlocksx = MAX_BLOCKS;//(N_MAX_PIXEL_TRIPLETS) % nThreadsx == 0 ? N_MAX_PIXEL_TRIPLETS / nThreadsx : N_MAX_PIXEL_TRIPLETS / nThreadsx + 1;
    addpT3asTrackCandidateInGPU<<<nBlocksx, nThreadsx,0,stream>>>(*modulesInGPU, *rangesInGPU, *pixelTripletsInGPU, *trackCandidatesInGPU, *segmentsInGPU, *pixelQuintupletsInGPU);
    hipError_t cudaerr_pT3 = hipGetLastError();
    if(cudaerr_pT3 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pT3)<<std::endl;
    }hipStreamSynchronize(stream);
#endif

#ifdef FINAL_T5
    //dim3 dupThreads(64,16,1);
    //dim3 dupBlocks(1,MAX_BLOCKS,1);
    dim3 dupThreads(32,32,1);
    dim3 dupBlocks(1,1,MAX_BLOCKS);
    dim3 nThreads(32,32,1);
    dim3 nBlocks(1,MAX_BLOCKS,1);
    removeDupQuintupletsInGPUv2<<<dupBlocks,dupThreads,0,stream>>>(*modulesInGPU, *quintupletsInGPU,true,*rangesInGPU);
    //hipDeviceSynchronize();
    hipStreamSynchronize(stream);
    addT5asTrackCandidateInGPU<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *quintupletsInGPU,*trackCandidatesInGPU,*pixelQuintupletsInGPU,*pixelTripletsInGPU,*rangesInGPU);

    hipError_t cudaerr_T5 =hipGetLastError(); 
    if(cudaerr_T5 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_T5)<<std::endl;
    }hipStreamSynchronize(stream);
#endif // final state T5
#ifdef FINAL_pLS
    //printf("Adding pLSs to TC collection\n");
#ifdef DUP_pLS
    //printf("cleaning pixels\n");
    checkHitspLS<<<MAX_BLOCKS,1024,0,stream>>>(*modulesInGPU, *rangesInGPU, *mdsInGPU, *segmentsInGPU, *hitsInGPU, true);
    hipError_t cudaerrpix = hipGetLastError();
    if(cudaerrpix != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerrpix)<<std::endl;

    }hipStreamSynchronize(stream);
#endif  
    unsigned int nThreadsx_pLS = 1024;
    unsigned int nBlocksx_pLS = MAX_BLOCKS;//(20000) % nThreadsx_pLS == 0 ? 20000 / nThreadsx_pLS : 20000 / nThreadsx_pLS + 1;
    addpLSasTrackCandidateInGPU<<<nBlocksx, nThreadsx,0,stream>>>(*modulesInGPU, *rangesInGPU, *pixelTripletsInGPU, *trackCandidatesInGPU, *segmentsInGPU, *pixelQuintupletsInGPU,*mdsInGPU,*hitsInGPU,*quintupletsInGPU);
    hipError_t cudaerr_pLS = hipGetLastError();
    if(cudaerr_pLS != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pLS)<<std::endl;
    }hipStreamSynchronize(stream);
#endif
}

void SDL::Event::createExtendedTracks()
{
    if(trackExtensionsInGPU == nullptr)
    {
        hipHostMalloc(&trackExtensionsInGPU, sizeof(SDL::trackExtensions));
    }

    unsigned int nTrackCandidates;
    hipMemcpy(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost);

#ifdef T3T3_EXTENSIONS
#ifdef Explicit_Extensions
    createTrackExtensionsInExplicitMemory(*trackExtensionsInGPU, nTrackCandidates * N_MAX_TRACK_EXTENSIONS_PER_TC + N_MAX_T3T3_TRACK_EXTENSIONS, nTrackCandidates + 1, stream); 
#else
    createTrackExtensionsInUnifiedMemory(*trackExtensionsInGPU, nTrackCandidates * N_MAX_TRACK_EXTENSIONS_PER_TC + N_MAX_T3T3_TRACK_EXTENSIONS, nTrackCandidates + 1, stream);
#endif
#else
#ifdef Explicit_Extensions
    createTrackExtensionsInExplicitMemory(*trackExtensionsInGPU, nTrackCandidates * N_MAX_TRACK_EXTENSIONS_PER_TC, nTrackCandidates, stream); 
#else
    createTrackExtensionsInUnifiedMemory(*trackExtensionsInGPU, nTrackCandidates * N_MAX_TRACK_EXTENSIONS_PER_TC, nTrackCandidates, stream);
#endif

    dim3 nThreads(32,1,16);
    dim3 nBlocks(80,1,200); 
    createExtendedTracksInGPU<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *rangesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *pixelTripletsInGPU, *quintupletsInGPU, *pixelQuintupletsInGPU, *trackCandidatesInGPU, *trackExtensionsInGPU);

    hipError_t cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }hipStreamSynchronize(stream);

#ifdef T3T3_EXTENSIONS
    dim3 nThreadsT3T3(1,16,16);
    dim3 nBlocksT3T3(nLowerModules % nThreadsT3T3.x == 0 ? nLowerModules / nThreadsT3T3.x: nLowerModules / nThreadsT3T3.x + 1, maxT3s % nThreadsT3T3.y == 0 ? maxT3s / nThreadsT3T3.y : maxT3s / nThreadsT3T3.y + 1, maxT3s % nThreadsT3T3.z == 0 ? maxT3s / nThreadsT3T3.z : maxT3s / nThreadsT3T3.z + 1);

    createT3T3ExtendedTracksInGPU<<<nBlocksT3T3, nThreadsT3T3,0,stream>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, *pixelTripletsInGPU, *pixelQuintupletsInGPU, *trackCandidatesInGPU, *trackExtensionsInGPU, nTrackCandidates);

    cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#endif

    int nThreadsDupCleaning = 512;
    int nBlocksDupCleaning = (nTrackCandidates % nThreadsDupCleaning == 0) ? nTrackCandidates / nThreadsDupCleaning : nTrackCandidates / nThreadsDupCleaning + 1;

    cleanDuplicateExtendedTracks<<<nThreadsDupCleaning, nBlocksDupCleaning,0,stream>>>(*trackExtensionsInGPU, nTrackCandidates);

    cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }hipStreamSynchronize(stream);

//    hipDeviceSynchronize();
}
#endif

void SDL::Event::createPixelTriplets()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    if(pixelTripletsInGPU == nullptr)
    {
        hipHostMalloc(&pixelTripletsInGPU, sizeof(SDL::pixelTriplets));
    }
#ifdef Explicit_PT3
    createPixelTripletsInExplicitMemory(*pixelTripletsInGPU, N_MAX_PIXEL_TRIPLETS,stream);
#else
    createPixelTripletsInUnifiedMemory(*pixelTripletsInGPU, N_MAX_PIXEL_TRIPLETS,stream);
#endif

    unsigned int pixelModuleIndex = nLowerModules;
    int* superbins;
    int8_t* pixelTypes;
    unsigned int *nTriplets;
    unsigned int nInnerSegments = 0;
    hipMemcpyAsync(&nInnerSegments, &(segmentsInGPU->nSegments[pixelModuleIndex]), sizeof(unsigned int), hipMemcpyDeviceToHost,stream);

    hipHostMalloc(&nTriplets, nLowerModules * sizeof(unsigned int));
    hipMemcpyAsync(nTriplets, tripletsInGPU->nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);

    hipHostMalloc(&superbins,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int));
    hipHostMalloc(&pixelTypes,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t));

    hipMemcpyAsync(superbins,segmentsInGPU->superbin,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(pixelTypes,segmentsInGPU->pixelType,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t),hipMemcpyDeviceToHost,stream);

    unsigned int* connectedPixelSize_host;
    unsigned int* connectedPixelIndex_host;
    hipHostMalloc(&connectedPixelSize_host, nInnerSegments* sizeof(unsigned int));
    hipHostMalloc(&connectedPixelIndex_host, nInnerSegments* sizeof(unsigned int));
    unsigned int* connectedPixelSize_dev;
    unsigned int* connectedPixelIndex_dev;
    hipMalloc(&connectedPixelSize_dev, nInnerSegments* sizeof(unsigned int));
    hipMalloc(&connectedPixelIndex_dev, nInnerSegments* sizeof(unsigned int));
    unsigned int max_size =0;
    int threadSize = 1000000;
    unsigned int *segs_pix = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *segs_pix_offset = segs_pix+threadSize;
    unsigned int *segs_pix_gpu;
    unsigned int *segs_pix_gpu_offset;
    hipMalloc((void **)&segs_pix_gpu, 2*threadSize*sizeof(unsigned int));
    //hipMallocAsync((void **)&segs_pix_gpu, 2*threadSize*sizeof(unsigned int),stream);
    segs_pix_gpu_offset = segs_pix_gpu + threadSize;
    hipMemsetAsync(segs_pix_gpu, nInnerSegments, threadSize*sizeof(unsigned int),stream); // so if not set, it will pass in the kernel
hipStreamSynchronize(stream);
    unsigned int totalSegs=0;
    int pixelIndexOffsetPos = pixelMapping->connectedPixelsIndex[44999] + pixelMapping->connectedPixelsSizes[44999];
    int pixelIndexOffsetNeg = pixelMapping->connectedPixelsIndexPos[44999] + pixelMapping->connectedPixelsSizes[44999] + pixelIndexOffsetPos;

    for (int i = 0; i < nInnerSegments; i++)
    {// loop over # pLS
        int8_t pixelType = pixelTypes[i];// get pixel type for this pLS
        int superbin = superbins[i]; //get superbin for this pixel
        if((superbin < 0) or (superbin >= 45000) or (pixelType > 2) or (pixelType < 0))
        {
            continue;
        }

        if(pixelType ==0)
        { // used pixel type to select correct size-index arrays
            connectedPixelSize_host[i]  = pixelMapping->connectedPixelsSizes[superbin]; //number of connected modules to this pixel
            connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndex[superbin];// index to get start of connected modules for this superbin in map
            for (int j=0; j < pixelMapping->connectedPixelsSizes[superbin]; j++)
            { // loop over modules from the size
                segs_pix[totalSegs+j] = i; // save the pixel index in array to be transfered to kernel
                segs_pix_offset[totalSegs+j] = j; // save this segment in array to be transfered to kernel
            }
            totalSegs += connectedPixelSize_host[i]; // increment counter
        }
        else if(pixelType ==1)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesPos[superbin]; //number of pixel connected modules
            connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndexPos[superbin]+pixelIndexOffsetPos;// index to get start of connected pixel modules
            for (int j=0; j < pixelMapping->connectedPixelsSizesPos[superbin]; j++)
            {
                segs_pix[totalSegs+j] = i;
                segs_pix_offset[totalSegs+j] = j;
            }
            totalSegs += connectedPixelSize_host[i];
        }
        else if(pixelType ==2)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesNeg[superbin]; //number of pixel connected modules
            connectedPixelIndex_host[i] =pixelMapping->connectedPixelsIndexNeg[superbin] + pixelIndexOffsetNeg;// index to get start of connected pixel modules
            for (int j=0; j < pixelMapping->connectedPixelsSizesNeg[superbin]; j++)
            {
                segs_pix[totalSegs+j] = i;
                segs_pix_offset[totalSegs+j] = j;
            }
            totalSegs += connectedPixelSize_host[i];
        }
    }

    hipMemcpyAsync(connectedPixelSize_dev, connectedPixelSize_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(connectedPixelIndex_dev, connectedPixelIndex_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(segs_pix_gpu,segs_pix,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(segs_pix_gpu_offset,segs_pix_offset,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
hipStreamSynchronize(stream);

    //less cheap method to estimate max_size for y axis
    max_size = *std::max_element(nTriplets, nTriplets + nLowerModules);
    //dim3 nThreads(16,16,1);
    //dim3 nBlocks((totalSegs % nThreads.x == 0 ? totalSegs / nThreads.x : totalSegs / nThreads.x + 1),
    //              (max_size % nThreads.y == 0 ? max_size/nThreads.y : max_size/nThreads.y + 1),1);
    //printf("%d %d\n",totalSegs,max_size);
    dim3 nThreads(16,64,1);
    dim3 nBlocks(1,MAX_BLOCKS,1);
    createPixelTripletsInGPUFromMap<<<nBlocks, nThreads,0,stream>>>(*modulesInGPU, *rangesInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *pixelTripletsInGPU, connectedPixelSize_dev,connectedPixelIndex_dev,nInnerSegments,segs_pix_gpu,segs_pix_gpu_offset, totalSegs);

    hipError_t cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }hipStreamSynchronize(stream);
    //}hipDeviceSynchronize();
    hipHostFree(connectedPixelSize_host);
    hipHostFree(connectedPixelIndex_host);
    hipFree(connectedPixelSize_dev);
    hipFree(connectedPixelIndex_dev);
    //hipFreeAsync(connectedPixelSize_dev,stream);
    //hipFreeAsync(connectedPixelIndex_dev,stream);
    hipHostFree(superbins);
    hipHostFree(pixelTypes);
    hipHostFree(nTriplets);
    free(segs_pix);
    //hipFreeAsync(segs_pix_gpu,stream);
    hipFree(segs_pix_gpu);
#ifdef Warnings
    unsigned int nPixelTriplets;
    hipMemcpyAsync(&nPixelTriplets, &(pixelTripletsInGPU->nPixelTriplets),  sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    cudaStreamSynhronize(stream);
    std::cout<<"number of pixel triplets = "<<nPixelTriplets<<std::endl;
#endif

    //pT3s can be cleaned here because they're not used in making pT5s!
#ifdef DUP_pT3
    dim3 nThreads_dup(512,1,1);
    dim3 nBlocks_dup(MAX_BLOCKS,1,1);
    removeDupPixelTripletsInGPUFromMap<<<nBlocks_dup,nThreads_dup,0,stream>>>(*pixelTripletsInGPU,false);
hipStreamSynchronize(stream);
#endif

}

void SDL::Event::createQuintuplets()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    uint16_t nEligibleT5Modules = 0;
    uint16_t *indicesOfEligibleModules = (uint16_t*)malloc(nLowerModules*sizeof(uint16_t));

    unsigned int maxTriplets;
    createEligibleModulesListForQuintuplets(*modulesInGPU, *tripletsInGPU, nEligibleT5Modules, indicesOfEligibleModules, N_MAX_QUINTUPLETS_PER_MODULE, maxTriplets,stream,*rangesInGPU);

    if(quintupletsInGPU == nullptr)
    {
        hipHostMalloc(&quintupletsInGPU, sizeof(SDL::quintuplets));
#ifdef Explicit_T5
        createQuintupletsInExplicitMemory(*quintupletsInGPU, N_MAX_QUINTUPLETS_PER_MODULE, nLowerModules, nEligibleT5Modules,stream);
#else
        createQuintupletsInUnifiedMemory(*quintupletsInGPU, N_MAX_QUINTUPLETS_PER_MODULE, nLowerModules, nEligibleT5Modules,stream);
#endif
    }
    hipStreamSynchronize(stream);


    int threadSize=N_MAX_TOTAL_TRIPLETS;
    unsigned int *threadIdx = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *threadIdx_offset = threadIdx+threadSize;
    unsigned int *threadIdx_gpu;
    unsigned int *threadIdx_gpu_offset;
    hipMalloc((void **)&threadIdx_gpu, 2*threadSize*sizeof(unsigned int));
    //hipMallocAsync((void **)&threadIdx_gpu, 2*threadSize*sizeof(unsigned int),stream);
    hipMemsetAsync(threadIdx_gpu, nLowerModules, threadSize*sizeof(unsigned int),stream);

    unsigned int *nTriplets = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    hipMemcpyAsync(nTriplets, tripletsInGPU->nTriplets, nLowerModules*sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    threadIdx_gpu_offset = threadIdx_gpu + threadSize;

    int nTotalTriplets = 0;
    for (int i=0; i<nEligibleT5Modules; i++) 
    {
        int index = indicesOfEligibleModules[i];
        unsigned int nInnerTriplets = nTriplets[index];
        if (nInnerTriplets !=0) 
        {
            for (int j=0; j<nInnerTriplets; j++) 
            {
                threadIdx[nTotalTriplets + j] = index;
                threadIdx_offset[nTotalTriplets + j] = j;
            }
            nTotalTriplets += nInnerTriplets;
        }
    }
    //printf("T5: nTotalTriplets=%d nEligibleT5Modules=%d\n", nTotalTriplets, nEligibleT5Modules);
    if (threadSize < nTotalTriplets) 
    {
        printf("threadSize=%d nTotalTriplets=%d: Increase buffer size for threadIdx in createQuintuplets\n", threadSize, nTotalTriplets);
        exit(1);
    }
    hipMemcpyAsync(threadIdx_gpu, threadIdx, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(threadIdx_gpu_offset, threadIdx_offset, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
hipStreamSynchronize(stream);

    dim3 nThreads(32, 32, 1);
    dim3 nBlocks(1,MAX_BLOCKS,1);

    createQuintupletsInGPU<<<nBlocks,nThreads,0,stream>>>(*modulesInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, threadIdx_gpu, threadIdx_gpu_offset, nTotalTriplets,*rangesInGPU);
    hipError_t cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
    hipStreamSynchronize(stream);
    free(threadIdx);
    free(nTriplets);
    hipFree(threadIdx_gpu);
    free(indicesOfEligibleModules);

#ifdef DUP_T5
  //dim3 dupThreads(64,16,1);
    //dim3 dupBlocks(1,MAX_BLOCKS,1);
    dim3 dupThreads(32,32,1);
    dim3 dupBlocks(1,1,MAX_BLOCKS);
    removeDupQuintupletsInGPU<<<dupBlocks,dupThreads,0,stream>>>(*modulesInGPU, *quintupletsInGPU,false,*rangesInGPU);
    //hipDeviceSynchronize();
    hipStreamSynchronize(stream);
#endif

#if defined(AddObjects)
#ifdef Explicit_T5
    addQuintupletsToEventExplicit();
#else
    addQuintupletsToEvent();
#endif
#endif

}
void SDL::Event::pixelLineSegmentCleaning()
{
#ifdef DUP_pLS
    //printf("cleaning pixels\n");
    checkHitspLS<<<MAX_BLOCKS,1024,0,stream>>>(*modulesInGPU, *rangesInGPU, *mdsInGPU, *segmentsInGPU, *hitsInGPU, false);
    hipError_t cudaerrpix = hipGetLastError();
    if(cudaerrpix != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerrpix)<<std::endl;

    }hipStreamSynchronize(stream);
    //}hipDeviceSynchronize();
#endif  

}
void SDL::Event::createPixelQuintuplets()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);

    if(pixelQuintupletsInGPU == nullptr)
    {
        hipHostMalloc(&pixelQuintupletsInGPU, sizeof(SDL::pixelQuintuplets));
#ifdef Explicit_PT5
    createPixelQuintupletsInExplicitMemory(*pixelQuintupletsInGPU, N_MAX_PIXEL_QUINTUPLETS,stream);
#else
    createPixelQuintupletsInUnifiedMemory(*pixelQuintupletsInGPU, N_MAX_PIXEL_QUINTUPLETS,stream);
#endif  
    }
   if(trackCandidatesInGPU == nullptr)
    {
        hipHostMalloc(&trackCandidatesInGPU, sizeof(SDL::trackCandidates));
#ifdef Explicit_Track
        createTrackCandidatesInExplicitMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES + N_MAX_PIXEL_TRACK_CANDIDATES,stream);
#else
        createTrackCandidatesInUnifiedMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES + N_MAX_PIXEL_TRACK_CANDIDATES,stream);

#endif
    } 

    unsigned int pixelModuleIndex;
    int* superbins;
    int8_t* pixelTypes;
    unsigned int *nQuintuplets;

    unsigned int* connectedPixelSize_host;
    unsigned int* connectedPixelIndex_host;
    unsigned int* connectedPixelSize_dev;
    unsigned int* connectedPixelIndex_dev;

    hipHostMalloc(&nQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMemcpyAsync(nQuintuplets, quintupletsInGPU->nQuintuplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);

    hipHostMalloc(&superbins,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int));
    hipHostMalloc(&pixelTypes,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t));

    hipMemcpyAsync(superbins,segmentsInGPU->superbin,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(pixelTypes,segmentsInGPU->pixelType,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t),hipMemcpyDeviceToHost,stream);
    
    hipStreamSynchronize(stream);
    pixelModuleIndex = nLowerModules;
    unsigned int nInnerSegments = 0;
    hipMemcpyAsync(&nInnerSegments, &(segmentsInGPU->nSegments[pixelModuleIndex]), sizeof(unsigned int), hipMemcpyDeviceToHost,stream);


    hipHostMalloc(&connectedPixelSize_host, nInnerSegments* sizeof(unsigned int));
    hipHostMalloc(&connectedPixelIndex_host, nInnerSegments* sizeof(unsigned int));
    hipMalloc(&connectedPixelSize_dev, nInnerSegments* sizeof(unsigned int));
    hipMalloc(&connectedPixelIndex_dev, nInnerSegments* sizeof(unsigned int));

    unsigned int max_size =0;
    int threadSize = 1000000;
    unsigned int *segs_pix = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *segs_pix_offset = segs_pix+threadSize;
    unsigned int *segs_pix_gpu;
    unsigned int *segs_pix_gpu_offset;
    hipMalloc((void **)&segs_pix_gpu, 2*threadSize*sizeof(unsigned int));
    hipStreamSynchronize(stream);

    segs_pix_gpu_offset = segs_pix_gpu + threadSize;
    hipMemsetAsync(segs_pix_gpu, nInnerSegments, threadSize*sizeof(unsigned int),stream); // so if not set, it will pass in the kernel
    unsigned int totalSegs=0;


    int pixelIndexOffsetPos = pixelMapping->connectedPixelsIndex[44999] + pixelMapping->connectedPixelsSizes[44999];
    int pixelIndexOffsetNeg = pixelMapping->connectedPixelsIndexPos[44999] + pixelMapping->connectedPixelsSizes[44999] + pixelIndexOffsetPos;

    for (int i = 0; i < nInnerSegments; i++)
    {// loop over # pLS
        int8_t pixelType = pixelTypes[i];// get pixel type for this pLS
        int superbin = superbins[i]; //get superbin for this pixel
        if((superbin < 0) or (superbin >= 45000) or (pixelType > 2) or (pixelType < 0))
        {
            continue;
        }

        if(pixelType ==0)
        { // used pixel type to select correct size-index arrays
            connectedPixelSize_host[i]  = pixelMapping->connectedPixelsSizes[superbin]; //number of connected modules to this pixel
            connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndex[superbin];// index to get start of connected modules for this superbin in map
            for (int j=0; j < pixelMapping->connectedPixelsSizes[superbin]; j++)
            { // loop over modules from the size
                segs_pix[totalSegs+j] = i; // save the pixel index in array to be transfered to kernel
                segs_pix_offset[totalSegs+j] = j; // save this segment in array to be transfered to kernel
            }
            totalSegs += connectedPixelSize_host[i]; // increment counter
        }
        else if(pixelType ==1)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesPos[superbin]; //number of pixel connected modules
            connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndexPos[superbin]+pixelIndexOffsetPos;// index to get start of connected pixel modules
            for (int j=0; j < pixelMapping->connectedPixelsSizesPos[superbin]; j++)
            {
                segs_pix[totalSegs+j] = i;
                segs_pix_offset[totalSegs+j] = j;
            }
            totalSegs += connectedPixelSize_host[i];
        }
        else if(pixelType ==2)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesNeg[superbin]; //number of pixel connected modules
            connectedPixelIndex_host[i] =pixelMapping->connectedPixelsIndexNeg[superbin] + pixelIndexOffsetNeg;// index to get start of connected pixel modules
            for (int j=0; j < pixelMapping->connectedPixelsSizesNeg[superbin]; j++)
            {
                segs_pix[totalSegs+j] = i;
                segs_pix_offset[totalSegs+j] = j;
            }
            totalSegs += connectedPixelSize_host[i];
        }
    }

    hipMemcpyAsync(connectedPixelSize_dev, connectedPixelSize_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(connectedPixelIndex_dev, connectedPixelIndex_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(segs_pix_gpu,segs_pix,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(segs_pix_gpu_offset,segs_pix_offset,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
hipStreamSynchronize(stream);

    //less cheap method to estimate max_size for y axis
    max_size = *std::max_element(nQuintuplets, nQuintuplets + nLowerModules);
    dim3 nThreads(16,64,1);
    dim3 nBlocks(1,MAX_BLOCKS,1);
                  
    createPixelQuintupletsInGPUFromMap<<<nBlocks, nThreads,0,stream>>>(*modulesInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, *pixelQuintupletsInGPU, connectedPixelSize_dev, connectedPixelIndex_dev, nInnerSegments, segs_pix_gpu, segs_pix_gpu_offset, totalSegs,*rangesInGPU);

    hipError_t cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
    hipStreamSynchronize(stream);
    hipHostFree(connectedPixelSize_host);
    hipHostFree(connectedPixelIndex_host);
    hipFree(connectedPixelSize_dev);
    hipFree(connectedPixelIndex_dev);
    hipHostFree(superbins);
    hipHostFree(pixelTypes);
    hipHostFree(nQuintuplets);
    free(segs_pix);
    hipFree(segs_pix_gpu);

    dim3 nThreads_dup(512,1,1);
    dim3 nBlocks_dup(MAX_BLOCKS,1,1);
#ifdef DUP_pT5
    //printf("run dup pT5\n");
    removeDupPixelQuintupletsInGPUFromMap<<<nBlocks_dup,nThreads_dup,0,stream>>>(*pixelQuintupletsInGPU, false);
    hipError_t cudaerr2 = hipGetLastError(); 
    if(cudaerr2 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr2)<<std::endl;
    }hipStreamSynchronize(stream);
    //}hipDeviceSynchronize();
#endif
#ifdef FINAL_pT5
    //printf("Adding pT5s to TC collection\n");
    unsigned int nThreadsx_pT5 = 256;
    unsigned int nBlocksx_pT5 = 1;//(N_MAX_PIXEL_QUINTUPLETS) % nThreadsx_pT5 == 0 ? N_MAX_PIXEL_QUINTUPLETS / nThreadsx_pT5 : N_MAX_PIXEL_QUINTUPLETS / nThreadsx_pT5 + 1;
    addpT5asTrackCandidateInGPU<<<nBlocksx_pT5, nThreadsx_pT5,0,stream>>>(*modulesInGPU, *pixelQuintupletsInGPU, *trackCandidatesInGPU, *segmentsInGPU, *tripletsInGPU,*quintupletsInGPU);

    hipError_t cudaerr_pT5 = hipGetLastError();
    if(cudaerr_pT5 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pT5)<<std::endl;
    }
    hipStreamSynchronize(stream);
#endif
#ifdef Warnings
    unsigned int nPixelQuintuplets;
    hipMemcpyAsync(&nPixelQuintuplets, &(pixelQuintupletsInGPU->nPixelQuintuplets), sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    std::cout<<"number of pixel quintuplets = "<<nPixelQuintuplets<<std::endl;
#endif   
}

void SDL::Event::addQuintupletsToEvent()
{
    for(uint16_t i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        if(quintupletsInGPU->nQuintuplets[i] == 0)
        {
            rangesInGPU->quintupletRanges[i * 2] = -1;
            rangesInGPU->quintupletRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU->quintupletRanges[i * 2] = rangesInGPU->quintupletModuleIndices[i];
            rangesInGPU->quintupletRanges[i * 2 + 1] = rangesInGPU->quintupletModuleIndices[i] + quintupletsInGPU->nQuintuplets[i] - 1;

            if(modulesInGPU->subdets[i] == Barrel)
            {
                n_quintuplets_by_layer_barrel_[modulesInGPU->layers[i] - 1] += quintupletsInGPU->nQuintuplets[i];
            }
            else
            {
                n_quintuplets_by_layer_endcap_[modulesInGPU->layers[i] - 1] += quintupletsInGPU->nQuintuplets[i];
            }
        }
    }
}

void SDL::Event::addQuintupletsToEventExplicit()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    unsigned int* nQuintupletsCPU;
    hipHostMalloc(&nQuintupletsCPU, nLowerModules * sizeof(unsigned int));
    hipMemcpyAsync(nQuintupletsCPU,quintupletsInGPU->nQuintuplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);

    short* module_subdets;
    hipHostMalloc(&module_subdets, nModules* sizeof(short));
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_quintupletRanges;
    hipHostMalloc(&module_quintupletRanges, nLowerModules* 2*sizeof(int));
    hipMemcpyAsync(module_quintupletRanges,rangesInGPU->quintupletRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    hipHostMalloc(&module_layers, nLowerModules * sizeof(short));
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_quintupletModuleIndices;
    hipHostMalloc(&module_quintupletModuleIndices, nLowerModules * sizeof(int));
    hipMemcpyAsync(module_quintupletModuleIndices, rangesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    for(uint16_t i = 0; i<nLowerModules; i++)
    {
        if(nQuintupletsCPU[i] == 0 or module_quintupletModuleIndices[i] == -1)
        {
            module_quintupletRanges[i * 2] = -1;
            module_quintupletRanges[i * 2 + 1] = -1;
        }
       else
        {
            module_quintupletRanges[i * 2] = module_quintupletModuleIndices[i];
            module_quintupletRanges[i * 2 + 1] = module_quintupletModuleIndices[i] + nQuintupletsCPU[i] - 1;

            if(module_subdets[i] == Barrel)
            {
                n_quintuplets_by_layer_barrel_[module_layers[i] - 1] += nQuintupletsCPU[i];
            }
            else
            {
                n_quintuplets_by_layer_endcap_[module_layers[i] - 1] += nQuintupletsCPU[i];
            }
        }
    }
    hipHostFree(nQuintupletsCPU);
    hipHostFree(module_quintupletRanges);
    hipHostFree(module_layers);
    hipHostFree(module_subdets);
    hipHostFree(module_quintupletModuleIndices);

}

void SDL::Event::addTripletsToEvent()
{
    for(uint16_t i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        //tracklets run only on lower modules!!!!!!
        if(tripletsInGPU->nTriplets[i] == 0)
        {
            rangesInGPU->tripletRanges[i * 2] = -1;
            rangesInGPU->tripletRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU->tripletRanges[i * 2] = rangesInGPU->tripletModuleIndices[i];
            rangesInGPU->tripletRanges[i * 2 + 1] = rangesInGPU->tripletModuleIndices[i] + tripletsInGPU->nTriplets[i] - 1;

            if(modulesInGPU->subdets[i] == Barrel)
            {
                n_triplets_by_layer_barrel_[modulesInGPU->layers[i] - 1] += tripletsInGPU->nTriplets[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[modulesInGPU->layers[i] - 1] += tripletsInGPU->nTriplets[i];
            }
        }
    }
}
void SDL::Event::addTripletsToEventExplicit()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules,modulesInGPU->nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    unsigned int* nTripletsCPU;
    hipHostMalloc(&nTripletsCPU, nLowerModules * sizeof(unsigned int));
    hipMemcpyAsync(nTripletsCPU,tripletsInGPU->nTriplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);

    short* module_subdets;
    hipHostMalloc(&module_subdets, nLowerModules* sizeof(short));
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_tripletRanges;
    hipHostMalloc(&module_tripletRanges, nLowerModules* 2*sizeof(int));
    hipMemcpyAsync(module_tripletRanges,rangesInGPU->tripletRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    hipHostMalloc(&module_layers, nLowerModules * sizeof(short));
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);

    int* module_tripletModuleIndices;
    hipHostMalloc(&module_tripletModuleIndices, nLowerModules * sizeof(int));
    hipMemcpyAsync(module_tripletModuleIndices, rangesInGPU->tripletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);
    for(uint16_t i = 0; i<nLowerModules; i++)
    {
        if(nTripletsCPU[i] == 0)
        {
            module_tripletRanges[i * 2] = -1;
            module_tripletRanges[i * 2 + 1] = -1;
        }
        else
        {
            module_tripletRanges[i * 2] = module_tripletModuleIndices[i];
            module_tripletRanges[i * 2 + 1] = module_tripletModuleIndices[i] +  nTripletsCPU[i] - 1;

            if(module_subdets[i] == Barrel)
            {
                n_triplets_by_layer_barrel_[module_layers[i] - 1] += nTripletsCPU[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[module_layers[i] - 1] += nTripletsCPU[i];
            }
        }
    }

    hipMemcpyAsync(rangesInGPU->tripletRanges, module_tripletRanges, nLowerModules * 2 * sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    hipHostFree(nTripletsCPU);
    hipHostFree(module_tripletRanges);
    hipHostFree(module_layers);
    hipHostFree(module_subdets);
    hipHostFree(module_tripletModuleIndices);
}

unsigned int SDL::Event::getNumberOfHits()
{
    unsigned int hits = 0;
    for(auto &it:n_hits_by_layer_barrel_)
    {
        hits += it;
    }
    for(auto& it:n_hits_by_layer_endcap_)
    {
        hits += it;
    }

    return hits;
}

unsigned int SDL::Event::getNumberOfHitsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_hits_by_layer_barrel_[layer];
    else
        return n_hits_by_layer_barrel_[layer] + n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerBarrel(unsigned int layer)
{
    return n_hits_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerEndcap(unsigned int layer)
{
    return n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoublets()
{
     unsigned int miniDoublets = 0;
    for(auto &it:n_minidoublets_by_layer_barrel_)
    {
        miniDoublets += it;
    }
    for(auto &it:n_minidoublets_by_layer_endcap_)
    {
        miniDoublets += it;
    }

    return miniDoublets;

}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_minidoublets_by_layer_barrel_[layer];
    else
        return n_minidoublets_by_layer_barrel_[layer] + n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerBarrel(unsigned int layer)
{
    return n_minidoublets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerEndcap(unsigned int layer)
{
    return n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegments()
{
    unsigned int segments = 0;
    for(auto &it:n_segments_by_layer_barrel_)
    {
        segments += it;
    }
    for(auto &it:n_segments_by_layer_endcap_)
    {
        segments += it;
    }

    return segments;

}

unsigned int SDL::Event::getNumberOfSegmentsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_segments_by_layer_barrel_[layer];
    else
        return n_segments_by_layer_barrel_[layer] + n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerBarrel(unsigned int layer)
{
    return n_segments_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerEndcap(unsigned int layer)
{
    return n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTriplets()
{
    unsigned int triplets = 0;
    for(auto &it:n_triplets_by_layer_barrel_)
    {
        triplets += it;
    }
    for(auto &it:n_triplets_by_layer_endcap_)
    {
        triplets += it;
    }

    return triplets;

}

unsigned int SDL::Event::getNumberOfTripletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_triplets_by_layer_barrel_[layer];
    else
        return n_triplets_by_layer_barrel_[layer] + n_triplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerBarrel(unsigned int layer)
{
    return n_triplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerEndcap(unsigned int layer)
{
    return n_triplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfPixelTriplets()
{
#ifdef Explicit_PT3
    unsigned int nPixelTriplets;
    hipMemcpyAsync(&nPixelTriplets, pixelTripletsInGPU->nPixelTriplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    return nPixelTriplets;
#else
    return *(pixelTripletsInGPU->nPixelTriplets);
#endif
}


unsigned int SDL::Event::getNumberOfExtendedTracks()
{
    unsigned int nTrackCandidates;
    hipMemcpy(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost);

    unsigned int *nTrackExtensionsCPU = new unsigned int[nTrackCandidates];
    hipMemcpy(nTrackExtensionsCPU, trackExtensionsInGPU->nTrackExtensions, (nTrackCandidates)* sizeof(unsigned int), hipMemcpyDeviceToHost);

    unsigned int nTrackExtensions = 0;
    for(size_t it = 0; it < nTrackCandidates; it++)    
    {
        nTrackExtensions += nTrackExtensionsCPU[it];

    }
#ifdef T3T3_EXTENSIONS
    unsigned int nT3T3Extensions;
    hipMemcpy(&nT3T3Extensions,&(trackExtensionsInGPU->nTrackExtensions[nTrackCandidates]), sizeof(unsigned int), hipMemcpyDeviceToHost);
    nTrackExtensions += nT3T3Extensions;
#endif
    delete[] nTrackExtensionsCPU;
    return nTrackExtensions;
}

unsigned int SDL::Event::getNumberOfT3T3ExtendedTracks()
{
    unsigned int nTrackCandidates;
    hipMemcpy(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost);

    unsigned int nT3T3Extensions;
    hipMemcpy(&nT3T3Extensions, trackExtensionsInGPU->nTrackExtensions + nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost);
    return nT3T3Extensions;
}

unsigned int SDL::Event::getNumberOfPixelQuintuplets()
{
#ifdef Explicit_PT5
    unsigned int nPixelQuintuplets;
    hipMemcpyAsync(&nPixelQuintuplets, pixelQuintupletsInGPU->nPixelQuintuplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    return nPixelQuintuplets;

#else
    return *(pixelQuintupletsInGPU->nPixelQuintuplets);
#endif
}
unsigned int SDL::Event::getNumberOfQuintuplets()
{
    unsigned int quintuplets = 0;
    for(auto &it:n_quintuplets_by_layer_barrel_)
    {
        quintuplets += it;
    }
    for(auto &it:n_quintuplets_by_layer_endcap_)
    {
        quintuplets += it;
    }

    return quintuplets;

}

unsigned int SDL::Event::getNumberOfQuintupletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_quintuplets_by_layer_barrel_[layer];
    else
        return n_quintuplets_by_layer_barrel_[layer] + n_quintuplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayerBarrel(unsigned int layer)
{
    return n_quintuplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayerEndcap(unsigned int layer)
{
    return n_quintuplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidates()
{    
    unsigned int nTrackCandidates;
    hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    return nTrackCandidates;
}

unsigned int SDL::Event::getNumberOfPT5TrackCandidates()
{
    unsigned int nTrackCandidatesPT5;
    hipMemcpyAsync(&nTrackCandidatesPT5, trackCandidatesInGPU->nTrackCandidatespT5, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    return nTrackCandidatesPT5;
}

unsigned int SDL::Event::getNumberOfPT3TrackCandidates()
{
    unsigned int nTrackCandidatesPT3;
    hipMemcpyAsync(&nTrackCandidatesPT3, trackCandidatesInGPU->nTrackCandidatespT3, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    return nTrackCandidatesPT3;
}

unsigned int SDL::Event::getNumberOfPLSTrackCandidates()
{
    unsigned int nTrackCandidatesPLS;
    hipMemcpyAsync(&nTrackCandidatesPLS, trackCandidatesInGPU->nTrackCandidatespLS, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    return nTrackCandidatesPLS;
}

unsigned int SDL::Event::getNumberOfPixelTrackCandidates()
{
    unsigned int nTrackCandidates;
    unsigned int nTrackCandidatesT5;
    hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(&nTrackCandidatesT5, trackCandidatesInGPU->nTrackCandidatesT5, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    return nTrackCandidates - nTrackCandidatesT5;
}

unsigned int SDL::Event::getNumberOfT5TrackCandidates()
{
    unsigned int nTrackCandidatesT5;
    hipMemcpyAsync(&nTrackCandidatesT5, trackCandidatesInGPU->nTrackCandidatesT5, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    return nTrackCandidatesT5; 
}
#ifdef Explicit_Hit
SDL::hits* SDL::Event::getHits() //std::shared_ptr should take care of garbage collection
{
    if(hitsInCPU == nullptr)
    {
        hitsInCPU = new SDL::hits;
        hitsInCPU->nHits = new unsigned int;
        unsigned int nHits;
        hipMemcpyAsync(&nHits, hitsInGPU->nHits, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
        *(hitsInCPU->nHits) = nHits;
        hitsInCPU->idxs = new unsigned int[nHits];
        hitsInCPU->xs = new float[nHits];
        hitsInCPU->ys = new float[nHits];
        hitsInCPU->zs = new float[nHits];
        hitsInCPU->moduleIndices = new uint16_t[nHits];
        hipMemcpyAsync(hitsInCPU->idxs, hitsInGPU->idxs,sizeof(unsigned int) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->xs, hitsInGPU->xs, sizeof(float) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->ys, hitsInGPU->ys, sizeof(float) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->zs, hitsInGPU->zs, sizeof(float) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->moduleIndices, hitsInGPU->moduleIndices, sizeof(uint16_t) * nHits, hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    }
    return hitsInCPU;
}
SDL::objectRanges* SDL::Event::getRanges()
{
    uint16_t nLowerModules;
    hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    if(rangesInCPU == nullptr)
    {
        rangesInCPU = new SDL::objectRanges;
        rangesInCPU->hitRanges = new int[2*nModules];
        rangesInCPU->quintupletModuleIndices = new int[nLowerModules];
        rangesInCPU->miniDoubletModuleIndices = new int[nLowerModules+1];
        rangesInCPU->segmentModuleIndices = new int[nLowerModules + 1];
        rangesInCPU->tripletModuleIndices = new int[nLowerModules];
        hipMemcpyAsync(rangesInCPU->hitRanges, rangesInGPU->hitRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(rangesInCPU->quintupletModuleIndices, rangesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(rangesInCPU->miniDoubletModuleIndices, rangesInGPU->miniDoubletModuleIndices, (nLowerModules + 1) * sizeof(int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(rangesInCPU->segmentModuleIndices, rangesInGPU->segmentModuleIndices, (nLowerModules + 1) * sizeof(int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(rangesInCPU->tripletModuleIndices, rangesInGPU->tripletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost, stream);
hipStreamSynchronize(stream);
    }
    return rangesInCPU;
}
#else
SDL::hits* SDL::Event::getHits() //std::shared_ptr should take care of garbage collection
{
    return hitsInGPU;
}
SDL::objectRanges* SDL::Event::getRanges()
{
    return rangesInGPU;
}
#endif


#ifdef Explicit_MD
SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    if(mdsInCPU == nullptr)
    {
        mdsInCPU = new SDL::miniDoublets;
        mdsInCPU->nMDs = new unsigned int[nLowerModules+1];

        //compute memory locations
        mdsInCPU->nMemoryLocations = new unsigned int;
        hipMemcpyAsync(mdsInCPU->nMemoryLocations, mdsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
        mdsInCPU->totOccupancyMDs = new unsigned int[nLowerModules+1];

        mdsInCPU->anchorHitIndices = new unsigned int[*(mdsInCPU->nMemoryLocations)];
        mdsInCPU->outerHitIndices = new unsigned int[*(mdsInCPU->nMemoryLocations)];
        hipMemcpyAsync(mdsInCPU->anchorHitIndices, mdsInGPU->anchorHitIndices, *(mdsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(mdsInCPU->outerHitIndices, mdsInGPU->outerHitIndices, *(mdsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(mdsInCPU->nMDs, mdsInGPU->nMDs, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(mdsInCPU->totOccupancyMDs, mdsInGPU->totOccupancyMDs, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return mdsInCPU;
}
#else
SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    return mdsInGPU;
}
#endif


#ifdef Explicit_Seg
SDL::segments* SDL::Event::getSegments()
{
    if(segmentsInCPU == nullptr)
    {
        segmentsInCPU = new SDL::segments;
        
        segmentsInCPU->nSegments = new unsigned int[nLowerModules+1];
        hipMemcpyAsync(segmentsInCPU->nSegments, segmentsInGPU->nSegments, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        
        segmentsInCPU->nMemoryLocations = new unsigned int;
        hipMemcpyAsync(segmentsInCPU->nMemoryLocations, segmentsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        segmentsInCPU->mdIndices = new unsigned int[2 * *(segmentsInCPU->nMemoryLocations)];
        segmentsInCPU->innerMiniDoubletAnchorHitIndices = new unsigned int[*(segmentsInCPU->nMemoryLocations)];
        segmentsInCPU->outerMiniDoubletAnchorHitIndices = new unsigned int[*(segmentsInCPU->nMemoryLocations)];
        segmentsInCPU->totOccupancySegments = new unsigned int[nLowerModules+1];

        segmentsInCPU->ptIn = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->eta = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->phi = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->isDup = new bool[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->isQuad = new bool[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->score = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];

        hipMemcpyAsync(segmentsInCPU->mdIndices, segmentsInGPU->mdIndices, 2 * *(segmentsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->innerMiniDoubletAnchorHitIndices, segmentsInGPU->innerMiniDoubletAnchorHitIndices, *(segmentsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->outerMiniDoubletAnchorHitIndices, segmentsInGPU->outerMiniDoubletAnchorHitIndices, *(segmentsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->totOccupancySegments, segmentsInGPU->totOccupancySegments, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->ptIn, segmentsInGPU->ptIn, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->eta, segmentsInGPU->eta, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->phi, segmentsInGPU->phi, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->isDup, segmentsInGPU->isDup, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->isQuad, segmentsInGPU->isQuad, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->score, segmentsInGPU->score, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return segmentsInCPU;
}
#else
SDL::segments* SDL::Event::getSegments()
{
    return segmentsInGPU;
}
#endif

#ifdef Explicit_Trips
SDL::triplets* SDL::Event::getTriplets()
{
    if(tripletsInCPU == nullptr)
    {
        uint16_t nLowerModules;
        tripletsInCPU = new SDL::triplets;
        hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        tripletsInCPU->nMemoryLocations = new unsigned int;
        hipMemcpyAsync(tripletsInCPU->nMemoryLocations, tripletsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        tripletsInCPU->segmentIndices = new unsigned[2 * *(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->nTriplets = new unsigned int[nLowerModules];
        tripletsInCPU->betaIn  = new FPX[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->betaOut = new FPX[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->pt_beta = new FPX[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->hitIndices = new unsigned int[6 * *(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->logicalLayers = new uint8_t[3 * *(tripletsInCPU->nMemoryLocations)];

        hipMemcpyAsync(tripletsInCPU->hitIndices, tripletsInGPU->hitIndices, 6 * *(tripletsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->logicalLayers, tripletsInGPU->logicalLayers, 3 * *(tripletsInCPU->nMemoryLocations) * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->segmentIndices, tripletsInGPU->segmentIndices, 2 * *(tripletsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->betaIn, tripletsInGPU->betaIn,   *(tripletsInCPU->nMemoryLocations) * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->betaOut, tripletsInGPU->betaOut, *(tripletsInCPU->nMemoryLocations) * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->pt_beta, tripletsInGPU->pt_beta, *(tripletsInCPU->nMemoryLocations) * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        tripletsInCPU->totOccupancyTriplets = new unsigned int[nLowerModules];
        hipMemcpyAsync(tripletsInCPU->nTriplets, tripletsInGPU->nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->totOccupancyTriplets, tripletsInGPU->totOccupancyTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);

        hipStreamSynchronize(stream);
    }
    return tripletsInCPU;
}
#else
SDL::triplets* SDL::Event::getTriplets()
{
    return tripletsInGPU;
}
#endif

#ifdef Explicit_T5
SDL::quintuplets* SDL::Event::getQuintuplets()
{
    if(quintupletsInCPU == nullptr)
    {
        quintupletsInCPU = new SDL::quintuplets;
        uint16_t nLowerModules;
        hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        uint16_t nEligibleT5Modules;
        hipMemcpyAsync(&nEligibleT5Modules, rangesInGPU->nEligibleT5Modules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
        unsigned int nMemoryLocations = nEligibleT5Modules * N_MAX_QUINTUPLETS_PER_MODULE;

        quintupletsInCPU->nQuintuplets = new unsigned int[nLowerModules];
        quintupletsInCPU->totOccupancyQuintuplets = new unsigned int[nLowerModules];
        quintupletsInCPU->tripletIndices = new unsigned int[2 * nMemoryLocations];
        quintupletsInCPU->lowerModuleIndices = new uint16_t[5 * nMemoryLocations];
        quintupletsInCPU->innerRadius = new FPX[nMemoryLocations];
        quintupletsInCPU->outerRadius = new FPX[nMemoryLocations];
        quintupletsInCPU->isDup = new bool[nMemoryLocations];
        quintupletsInCPU->score_rphisum = new FPX[nMemoryLocations];
        quintupletsInCPU->eta = new FPX[nMemoryLocations];
        quintupletsInCPU->phi = new FPX[nMemoryLocations];
        quintupletsInCPU->regressionRadius = new float[nMemoryLocations];
        hipMemcpyAsync(quintupletsInCPU->nQuintuplets, quintupletsInGPU->nQuintuplets,  nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->totOccupancyQuintuplets, quintupletsInGPU->totOccupancyQuintuplets,  nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->tripletIndices, quintupletsInGPU->tripletIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->lowerModuleIndices, quintupletsInGPU->lowerModuleIndices, 5 * nMemoryLocations * sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->innerRadius, quintupletsInGPU->innerRadius, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->outerRadius, quintupletsInGPU->outerRadius, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->isDup, quintupletsInGPU->isDup, nMemoryLocations * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->score_rphisum, quintupletsInGPU->score_rphisum, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->eta, quintupletsInGPU->eta, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->phi, quintupletsInGPU->phi, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->regressionRadius, quintupletsInGPU->regressionRadius, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    }

    return quintupletsInCPU;
}
#else
SDL::quintuplets* SDL::Event::getQuintuplets()
{
    return quintupletsInGPU;
}
#endif

#ifdef Explicit_PT3
SDL::pixelTriplets* SDL::Event::getPixelTriplets()
{
    if(pixelTripletsInCPU == nullptr)
    {
        pixelTripletsInCPU = new SDL::pixelTriplets;

        pixelTripletsInCPU->nPixelTriplets = new unsigned int;
        pixelTripletsInCPU->totOccupancyPixelTriplets = new unsigned int;
        hipMemcpyAsync(pixelTripletsInCPU->nPixelTriplets, pixelTripletsInGPU->nPixelTriplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->totOccupancyPixelTriplets, pixelTripletsInGPU->totOccupancyPixelTriplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
        unsigned int nPixelTriplets = *(pixelTripletsInCPU->nPixelTriplets);
        pixelTripletsInCPU->tripletIndices = new unsigned int[nPixelTriplets];
        pixelTripletsInCPU->pixelSegmentIndices = new unsigned int[nPixelTriplets];
        pixelTripletsInCPU->pixelRadius = new FPX[nPixelTriplets];
        pixelTripletsInCPU->tripletRadius = new FPX[nPixelTriplets];
        pixelTripletsInCPU->isDup = new bool[nPixelTriplets];
        pixelTripletsInCPU->eta = new  FPX[nPixelTriplets];
        pixelTripletsInCPU->phi = new  FPX[nPixelTriplets];
        pixelTripletsInCPU->score =new FPX[nPixelTriplets];

        hipMemcpyAsync(pixelTripletsInCPU->tripletIndices, pixelTripletsInGPU->tripletIndices, nPixelTriplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->pixelSegmentIndices, pixelTripletsInGPU->pixelSegmentIndices, nPixelTriplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->pixelRadius, pixelTripletsInGPU->pixelRadius, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->tripletRadius, pixelTripletsInGPU->tripletRadius, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->isDup, pixelTripletsInGPU->isDup, nPixelTriplets * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->eta, pixelTripletsInGPU->eta, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->phi, pixelTripletsInGPU->phi, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->score, pixelTripletsInGPU->score, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    }
    return pixelTripletsInCPU;
}
#else
SDL::pixelTriplets* SDL::Event::getPixelTriplets()
{
    return pixelTripletsInGPU;
}
#endif

#ifdef Explicit_PT5
SDL::pixelQuintuplets* SDL::Event::getPixelQuintuplets()
{
    if(pixelQuintupletsInCPU == nullptr)
    {
        pixelQuintupletsInCPU = new SDL::pixelQuintuplets;

        pixelQuintupletsInCPU->nPixelQuintuplets = new unsigned int;
        pixelQuintupletsInCPU->totOccupancyPixelQuintuplets = new unsigned int;
        hipMemcpyAsync(pixelQuintupletsInCPU->nPixelQuintuplets, pixelQuintupletsInGPU->nPixelQuintuplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->totOccupancyPixelQuintuplets, pixelQuintupletsInGPU->totOccupancyPixelQuintuplets, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
        unsigned int nPixelQuintuplets = *(pixelQuintupletsInCPU->nPixelQuintuplets);

        pixelQuintupletsInCPU->pixelIndices = new unsigned int[nPixelQuintuplets];
        pixelQuintupletsInCPU->T5Indices = new unsigned int[nPixelQuintuplets];
        pixelQuintupletsInCPU->isDup = new bool[nPixelQuintuplets];
        pixelQuintupletsInCPU->score = new FPX[nPixelQuintuplets];

        hipMemcpyAsync(pixelQuintupletsInCPU->pixelIndices, pixelQuintupletsInGPU->pixelIndices, nPixelQuintuplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->T5Indices, pixelQuintupletsInGPU->T5Indices, nPixelQuintuplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->isDup, pixelQuintupletsInGPU->isDup, nPixelQuintuplets * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->score, pixelQuintupletsInGPU->score, nPixelQuintuplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    }
    return pixelQuintupletsInCPU;
}
#else
SDL::pixelQuintuplets* SDL::Event::getPixelQuintuplets()
{
    return pixelQuintupletsInGPU;
}
#endif

#ifdef Explicit_Track
SDL::trackCandidates* SDL::Event::getTrackCandidates()
{
    if(trackCandidatesInCPU == nullptr)
    {
        trackCandidatesInCPU = new SDL::trackCandidates;
        trackCandidatesInCPU->nTrackCandidates = new unsigned int;
        hipMemcpyAsync(trackCandidatesInCPU->nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
        unsigned int nTrackCandidates = *(trackCandidatesInCPU->nTrackCandidates);

        trackCandidatesInCPU->objectIndices = new unsigned int[2 * nTrackCandidates];
        trackCandidatesInCPU->trackCandidateType = new short[nTrackCandidates];
        trackCandidatesInCPU->partOfExtension = new bool[nTrackCandidates];
        trackCandidatesInCPU->hitIndices = new unsigned int[14 * nTrackCandidates];
        trackCandidatesInCPU->logicalLayers = new uint8_t[7 * nTrackCandidates];

        hipMemcpyAsync(trackCandidatesInCPU->partOfExtension, trackCandidatesInGPU->partOfExtension, nTrackCandidates * sizeof(bool), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(trackCandidatesInCPU->hitIndices, trackCandidatesInGPU->hitIndices, 14 * nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(trackCandidatesInCPU->logicalLayers, trackCandidatesInGPU->logicalLayers, 7 * nTrackCandidates * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(trackCandidatesInCPU->objectIndices, trackCandidatesInGPU->objectIndices, 2 * nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);                                                                                    
        hipMemcpyAsync(trackCandidatesInCPU->trackCandidateType, trackCandidatesInGPU->trackCandidateType, nTrackCandidates * sizeof(short), hipMemcpyDeviceToHost,stream);                                                                                                                
hipStreamSynchronize(stream);
    }
    return trackCandidatesInCPU;
}
#else
SDL::trackCandidates* SDL::Event::getTrackCandidates()
{
    return trackCandidatesInGPU;
}
#endif
#ifdef Explicit_Module
SDL::modules* SDL::Event::getFullModules()
{
    if(modulesInCPUFull == nullptr)
    {
        modulesInCPUFull = new SDL::modules;
        uint16_t nLowerModules;
        hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    modulesInCPUFull->detIds = new unsigned int[nModules];
    modulesInCPUFull->moduleMap = new uint16_t[40*nModules];
    modulesInCPUFull->nConnectedModules = new uint16_t[nModules];
    modulesInCPUFull->drdzs = new float[nModules];
    modulesInCPUFull->slopes = new float[nModules];
    modulesInCPUFull->nModules = new uint16_t[1];
    modulesInCPUFull->nLowerModules = new uint16_t[1];
    modulesInCPUFull->layers = new short[nModules];
    modulesInCPUFull->rings = new short[nModules];
    modulesInCPUFull->modules = new short[nModules];
    modulesInCPUFull->rods = new short[nModules];
    modulesInCPUFull->subdets = new short[nModules];
    modulesInCPUFull->sides = new short[nModules];
    modulesInCPUFull->isInverted = new bool[nModules];
    modulesInCPUFull->isLower = new bool[nModules];


    modulesInCPUFull->moduleType = new ModuleType[nModules];
    modulesInCPUFull->moduleLayerType = new ModuleLayerType[nModules];
    hipMemcpyAsync(modulesInCPUFull->detIds,modulesInGPU->detIds,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->moduleMap,modulesInGPU->moduleMap,40*nModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->drdzs,modulesInGPU->drdzs,sizeof(float)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->slopes,modulesInGPU->slopes,sizeof(float)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->rings,modulesInGPU->rings,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->modules,modulesInGPU->modules,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->rods,modulesInGPU->rods,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->subdets,modulesInGPU->subdets,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->sides,modulesInGPU->sides,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->isInverted,modulesInGPU->isInverted,sizeof(bool)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->isLower,modulesInGPU->isLower,sizeof(bool)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->moduleType,modulesInGPU->moduleType,sizeof(ModuleType)*nModules,hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(modulesInCPUFull->moduleLayerType,modulesInGPU->moduleLayerType,sizeof(ModuleLayerType)*nModules,hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    }
    return modulesInCPUFull;
}
SDL::modules* SDL::Event::getModules()
{
    if(modulesInCPU == nullptr)
    {
        modulesInCPU = new SDL::modules;
        uint16_t nLowerModules;
        hipMemcpyAsync(&nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
        modulesInCPU->nLowerModules = new uint16_t[1];
        modulesInCPU->nModules = new uint16_t[1];
        modulesInCPU->detIds = new unsigned int[nModules];
        modulesInCPU->isLower = new bool[nModules];
        modulesInCPU->layers = new short[nModules];
        modulesInCPU->subdets = new short[nModules];
        modulesInCPU->rings = new short[nModules];


        hipMemcpyAsync(modulesInCPU->nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->nModules, modulesInGPU->nModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->detIds, modulesInGPU->detIds, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->isLower, modulesInGPU->isLower, nModules * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->layers, modulesInGPU->layers, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->subdets, modulesInGPU->subdets, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->rings, modulesInGPU->rings, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);
    }
    return modulesInCPU;
}
#else
SDL::modules* SDL::Event::getModules()
{
    return modulesInGPU;
}
SDL::modules* SDL::Event::getFullModules()
{
    return modulesInGPU;
}
#endif

#ifdef Explicit_Extensions
SDL::trackExtensions* SDL::Event::getTrackExtensions()
{
   if(trackExtensionsInCPU == nullptr)
   {
       trackExtensionsInCPU = new SDL::trackExtensions;
       unsigned int nTrackCandidates;
       hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
       hipStreamSynchronize(stream);
       unsigned int maxTrackExtensions = nTrackCandidates * N_MAX_TRACK_EXTENSIONS_PER_TC;
#ifdef T3T3_EXTENSIONS
       maxTrackExtensions += N_MAX_T3T3_TRACK_EXTENSIONS;
       nTrackCandidates++;
#endif
       std::cout<<"nTrackCandidates = "<<nTrackCandidates<<std::endl;
       trackExtensionsInCPU->nTrackExtensions = new unsigned int[nTrackCandidates];
       trackExtensionsInCPU->totOccupancyTrackExtensions = new unsigned int[nTrackCandidates];
       trackExtensionsInCPU->constituentTCTypes = new short[3 * maxTrackExtensions];
       trackExtensionsInCPU->constituentTCIndices = new unsigned int[3 * maxTrackExtensions];
       trackExtensionsInCPU->nLayerOverlaps = new uint8_t[2 * maxTrackExtensions];
       trackExtensionsInCPU->nHitOverlaps = new uint8_t[2 * maxTrackExtensions];
       trackExtensionsInCPU->isDup = new bool[maxTrackExtensions];
       trackExtensionsInCPU->regressionRadius = new FPX[maxTrackExtensions];

       hipMemcpyAsync(trackExtensionsInCPU->nTrackExtensions, trackExtensionsInGPU->nTrackExtensions, nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->totOccupancyTrackExtensions, trackExtensionsInGPU->totOccupancyTrackExtensions, nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
       hipMemcpy(trackExtensionsInCPU->constituentTCTypes, trackExtensionsInGPU->constituentTCTypes, 3 * maxTrackExtensions * sizeof(short), hipMemcpyDeviceToHost);
       hipMemcpyAsync(trackExtensionsInCPU->constituentTCIndices, trackExtensionsInGPU->constituentTCIndices, 3 * maxTrackExtensions * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);

       hipMemcpyAsync(trackExtensionsInCPU->nLayerOverlaps, trackExtensionsInGPU->nLayerOverlaps, 2 * maxTrackExtensions * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->nHitOverlaps, trackExtensionsInGPU->nHitOverlaps, 2 * maxTrackExtensions * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->isDup, trackExtensionsInGPU->isDup, maxTrackExtensions * sizeof(bool), hipMemcpyDeviceToHost, stream);
       hipMemcpyAsync(trackExtensionsInCPU->regressionRadius, trackExtensionsInGPU->regressionRadius, maxTrackExtensions * sizeof(FPX), hipMemcpyDeviceToHost, stream);
       hipStreamSynchronize(stream);
   }

   return trackExtensionsInCPU;
}
#else
SDL::trackExtensions* SDL::Event::getTrackExtensions()
{
    return trackExtensionsInGPU;
}
#endif

