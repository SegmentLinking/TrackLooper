#include "hip/hip_runtime.h"
#include "Event.cuh"

struct SDL::modules* SDL::modulesInGPU = nullptr;
struct SDL::pixelMap* SDL::pixelMapping = nullptr;
uint16_t SDL::nModules;
uint16_t SDL::nLowerModules;

// Temporary alpaka statements
using Dim = alpaka::DimInt<3u>;
using Idx = std::size_t;
using Vec = alpaka::Vec<Dim,Idx>;
using WorkDiv = alpaka::WorkDivMembers<Dim, Idx>;
using Acc = alpaka::AccGpuCudaRt<Dim, Idx>;
using QueueProperty = alpaka::NonBlocking;
using QueueAcc = alpaka::Queue<Acc, QueueProperty>;
auto devAcc = alpaka::getDevByIdx<Acc>(0u);
Vec const elementsPerThread(Vec::all(static_cast<Idx>(1)));

SDL::Event::Event(hipStream_t estream,bool verbose)
{
    int version;
    int driver;
    hipRuntimeGetVersion(&version);
    hipDriverGetVersion(&driver);
    //printf("version: %d Driver %d\n",version, driver);
    stream = estream;
    addObjects = verbose;
    hitsInGPU = nullptr;
    mdsInGPU = nullptr;
    segmentsInGPU = nullptr;
    tripletsInGPU = nullptr;
    quintupletsInGPU = nullptr;
    trackCandidatesInGPU = nullptr;
    pixelTripletsInGPU = nullptr;
    pixelQuintupletsInGPU = nullptr;
    rangesInGPU = nullptr;

    hitsInCPU = nullptr;
    rangesInCPU = nullptr;
    mdsInCPU = nullptr;
    segmentsInCPU = nullptr;
    tripletsInCPU = nullptr;
    trackCandidatesInCPU = nullptr;
    modulesInCPU = nullptr;
    modulesInCPUFull = nullptr;
    quintupletsInCPU = nullptr;
    pixelTripletsInCPU = nullptr;
    pixelQuintupletsInCPU = nullptr;

    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_triplets_by_layer_barrel_[i] = 0;
        n_trackCandidates_by_layer_barrel_[i] = 0;
        n_quintuplets_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
            n_segments_by_layer_endcap_[i] = 0;
            n_triplets_by_layer_endcap_[i] = 0;
            n_trackCandidates_by_layer_endcap_[i] = 0;
            n_quintuplets_by_layer_endcap_[i] = 0;
        }
    }
}

SDL::Event::~Event()
{
#ifdef CACHE_ALLOC
    if(rangesInGPU){rangesInGPU->freeMemoryCache();}
    if(hitsInGPU){hitsInGPU->freeMemoryCache();}
    if(mdsInGPU){mdsInGPU->freeMemoryCache();}
    if(segmentsInGPU){segmentsInGPU->freeMemoryCache();}
    if(tripletsInGPU){tripletsInGPU->freeMemoryCache();}
    if(quintupletsInGPU){quintupletsInGPU->freeMemoryCache();}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemoryCache();}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemoryCache();}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemoryCache();}
#else

    if(rangesInGPU){rangesInGPU->freeMemory();}
    if(hitsInGPU){hitsInGPU->freeMemory();}
    if(mdsInGPU){mdsInGPU->freeMemory(stream);}
    if(segmentsInGPU){segmentsInGPU->freeMemory(stream);}
    if(tripletsInGPU){tripletsInGPU->freeMemory(stream);}
    if(quintupletsInGPU){quintupletsInGPU->freeMemory(stream);}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemory(stream);}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemory(stream);}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemory(stream);}
#endif
    if(rangesInGPU != nullptr){cms::cuda::free_host(rangesInGPU);}
    if(mdsInGPU != nullptr){cms::cuda::free_host(mdsInGPU);}
    if(segmentsInGPU!= nullptr){cms::cuda::free_host(segmentsInGPU);}
    if(tripletsInGPU!= nullptr){cms::cuda::free_host(tripletsInGPU);}
    if(trackCandidatesInGPU!= nullptr){cms::cuda::free_host(trackCandidatesInGPU);}
    if(hitsInGPU!= nullptr){cms::cuda::free_host(hitsInGPU);}
    if(pixelTripletsInGPU!= nullptr){cms::cuda::free_host(pixelTripletsInGPU);}
    if(pixelQuintupletsInGPU!= nullptr){cms::cuda::free_host(pixelQuintupletsInGPU);}
    if(quintupletsInGPU!= nullptr){cms::cuda::free_host(quintupletsInGPU);}

    if(hitsInCPU != nullptr)
    {
        delete[] hitsInCPU->idxs;
        delete[] hitsInCPU->xs;
        delete[] hitsInCPU->ys;
        delete[] hitsInCPU->zs;
        delete[] hitsInCPU->moduleIndices;
        delete hitsInCPU->nHits;
        delete hitsInCPU;
    }
    if(rangesInCPU != nullptr)
    {
        delete[] rangesInCPU->quintupletModuleIndices;
        delete rangesInCPU;
    }

    if(mdsInCPU != nullptr)
    {
        delete[] mdsInCPU->anchorHitIndices;
        delete[] mdsInCPU->nMDs;
        delete mdsInCPU->nMemoryLocations;
        delete[] mdsInCPU->totOccupancyMDs;
        delete mdsInCPU;
    }

    if(segmentsInCPU != nullptr)
    {
        delete[] segmentsInCPU->mdIndices;
        delete[] segmentsInCPU->nSegments;
        delete[] segmentsInCPU->totOccupancySegments;
        delete[] segmentsInCPU->innerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->outerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->ptIn;
        delete[] segmentsInCPU->eta;
        delete[] segmentsInCPU->phi;
        delete segmentsInCPU->nMemoryLocations;
        delete segmentsInCPU;
    }

    if(tripletsInCPU != nullptr)
    {
        delete[] tripletsInCPU->segmentIndices;
        delete[] tripletsInCPU->nTriplets;
        delete[] tripletsInCPU->totOccupancyTriplets;
        delete[] tripletsInCPU->betaIn;
        delete[] tripletsInCPU->betaOut;
        delete[] tripletsInCPU->pt_beta;
        delete[] tripletsInCPU->hitIndices;
        delete[] tripletsInCPU->logicalLayers;
        delete[] tripletsInCPU->lowerModuleIndices;
        delete tripletsInCPU->nMemoryLocations;
#ifdef CUT_VALUE_DEBUG
        delete[] tripletsInCPU->zOut;
        delete[] tripletsInCPU->zLo;
        delete[] tripletsInCPU->zHi;
        delete[] tripletsInCPU->zLoPointed;
        delete[] tripletsInCPU->zHiPointed;
        delete[] tripletsInCPU->sdlCut;
        delete[] tripletsInCPU->betaInCut;
        delete[] tripletsInCPU->betaOutCut;
        delete[] tripletsInCPU->deltaBetaCut;
        delete[] tripletsInCPU->rtLo;
        delete[] tripletsInCPU->rtHi;
        delete[] tripletsInCPU->kZ;
#endif
        delete tripletsInCPU;
    }
    if(quintupletsInCPU != nullptr)
    {
        delete[] quintupletsInCPU->tripletIndices;
        delete[] quintupletsInCPU->nQuintuplets;
        delete[] quintupletsInCPU->totOccupancyQuintuplets;
        delete[] quintupletsInCPU->lowerModuleIndices;
        delete[] quintupletsInCPU->innerRadius;
        delete[] quintupletsInCPU->outerRadius;
        delete[] quintupletsInCPU->regressionRadius;
        delete[] quintupletsInCPU->bridgeRadius;
        delete[] quintupletsInCPU->chiSquared;
        delete[] quintupletsInCPU->rzChiSquared;
        delete[] quintupletsInCPU->nonAnchorChiSquared;
        delete quintupletsInCPU;
    }

    if(pixelTripletsInCPU != nullptr)
    {
        delete[] pixelTripletsInCPU->tripletIndices;
        delete[] pixelTripletsInCPU->pixelSegmentIndices;
        delete[] pixelTripletsInCPU->pixelRadius;
        delete[] pixelTripletsInCPU->tripletRadius;
        delete pixelTripletsInCPU->nPixelTriplets;
        delete pixelTripletsInCPU->totOccupancyPixelTriplets;
        delete[] pixelTripletsInCPU->rzChiSquared;
        delete[] pixelTripletsInCPU->rPhiChiSquared;
        delete[] pixelTripletsInCPU->rPhiChiSquaredInwards;
        delete pixelTripletsInCPU;
    }

    if(pixelQuintupletsInCPU != nullptr)
    {
        delete[] pixelQuintupletsInCPU->pixelIndices;
        delete[] pixelQuintupletsInCPU->T5Indices;
        delete[] pixelQuintupletsInCPU->isDup;
        delete[] pixelQuintupletsInCPU->score;
        delete pixelQuintupletsInCPU->nPixelQuintuplets;
        delete pixelQuintupletsInCPU->totOccupancyPixelQuintuplets;
        delete[] pixelQuintupletsInCPU->rzChiSquared;
        delete[] pixelQuintupletsInCPU->rPhiChiSquared;
        delete[] pixelQuintupletsInCPU->rPhiChiSquaredInwards;
        delete pixelQuintupletsInCPU;
    }

    if(trackCandidatesInCPU != nullptr)
    {
        delete[] trackCandidatesInCPU->objectIndices;
        delete[] trackCandidatesInCPU->trackCandidateType;
        delete[] trackCandidatesInCPU->nTrackCandidates;
        delete[] trackCandidatesInCPU->hitIndices;
        delete[] trackCandidatesInCPU->logicalLayers;
        delete trackCandidatesInCPU;
    }


    if(modulesInCPU != nullptr)
    {
        delete[] modulesInCPU->nLowerModules;
        delete[] modulesInCPU->nModules;
        delete[] modulesInCPU->detIds;
        delete[] modulesInCPU->isLower;
        delete[] modulesInCPU->layers;
        delete[] modulesInCPU->subdets;
        delete[] modulesInCPU->rings;
        delete[] modulesInCPU->rods;
        delete[] modulesInCPU->modules;
        delete[] modulesInCPU->sides;
        delete[] modulesInCPU->eta;
        delete[] modulesInCPU->r;
        delete[] modulesInCPU;
    }
    if(modulesInCPUFull != nullptr)
    {
        delete[] modulesInCPUFull->detIds;
        delete[] modulesInCPUFull->moduleMap;
        delete[] modulesInCPUFull->nConnectedModules;
        delete[] modulesInCPUFull->drdzs;
        delete[] modulesInCPUFull->slopes;
        delete[] modulesInCPUFull->nModules;
        delete[] modulesInCPUFull->nLowerModules;
        delete[] modulesInCPUFull->layers;
        delete[] modulesInCPUFull->rings;
        delete[] modulesInCPUFull->modules;
        delete[] modulesInCPUFull->rods;
        delete[] modulesInCPUFull->subdets;
        delete[] modulesInCPUFull->sides;
        delete[] modulesInCPUFull->eta;
        delete[] modulesInCPUFull->r;
        delete[] modulesInCPUFull->isInverted;
        delete[] modulesInCPUFull->isLower;


        delete[] modulesInCPUFull->moduleType;
        delete[] modulesInCPUFull->moduleLayerType;
        delete[] modulesInCPUFull;
    }
    SDL::freeEndCapMapMemory();
}

void SDL::Event::resetEvent()
{
#ifdef CACHE_ALLOC
    if(hitsInGPU){hitsInGPU->freeMemoryCache();}
    if(mdsInGPU){mdsInGPU->freeMemoryCache();}
    if(quintupletsInGPU){quintupletsInGPU->freeMemoryCache();}
    if(rangesInGPU){rangesInGPU->freeMemoryCache();}
    if(segmentsInGPU){segmentsInGPU->freeMemoryCache();}
    if(tripletsInGPU){tripletsInGPU->freeMemoryCache();}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemoryCache();}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemoryCache();}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemoryCache();}

#else
    if(hitsInGPU){hitsInGPU->freeMemory();}
    if(quintupletsInGPU){quintupletsInGPU->freeMemory(stream);}
    if(rangesInGPU){rangesInGPU->freeMemory();}
    if(mdsInGPU){mdsInGPU->freeMemory(stream);}
    if(segmentsInGPU){segmentsInGPU->freeMemory(stream);}
    if(tripletsInGPU){tripletsInGPU->freeMemory(stream);}
    if(pixelQuintupletsInGPU){pixelQuintupletsInGPU->freeMemory(stream);}
    if(pixelTripletsInGPU){pixelTripletsInGPU->freeMemory(stream);}
    if(trackCandidatesInGPU){trackCandidatesInGPU->freeMemory(stream);}
#endif
    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_triplets_by_layer_barrel_[i] = 0;
        n_trackCandidates_by_layer_barrel_[i] = 0;
        n_quintuplets_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
            n_segments_by_layer_endcap_[i] = 0;
            n_triplets_by_layer_endcap_[i] = 0;
            n_trackCandidates_by_layer_endcap_[i] = 0;
            n_quintuplets_by_layer_endcap_[i] = 0;
        }
    }
    if(hitsInGPU){cms::cuda::free_host(hitsInGPU);
      hitsInGPU = nullptr;}
    if(mdsInGPU){cms::cuda::free_host(mdsInGPU);
      mdsInGPU = nullptr;}
    if(rangesInGPU){cms::cuda::free_host(rangesInGPU);
      rangesInGPU = nullptr;}
    if(segmentsInGPU){cms::cuda::free_host(segmentsInGPU);
      segmentsInGPU = nullptr;}
    if(tripletsInGPU){cms::cuda::free_host(tripletsInGPU);
      tripletsInGPU = nullptr;}
    if(quintupletsInGPU){cms::cuda::free_host(quintupletsInGPU);
      quintupletsInGPU = nullptr;}
    if(trackCandidatesInGPU){cms::cuda::free_host(trackCandidatesInGPU);
      trackCandidatesInGPU = nullptr;}
    if(pixelTripletsInGPU){cms::cuda::free_host(pixelTripletsInGPU);
      pixelTripletsInGPU = nullptr;}
    if(pixelQuintupletsInGPU){cms::cuda::free_host(pixelQuintupletsInGPU);
      pixelQuintupletsInGPU = nullptr;}

    if(hitsInCPU != nullptr)
    {
        delete[] hitsInCPU->idxs;
        delete[] hitsInCPU->xs;
        delete[] hitsInCPU->ys;
        delete[] hitsInCPU->zs;
        delete[] hitsInCPU->moduleIndices;
        delete hitsInCPU->nHits;
        delete hitsInCPU;
        hitsInCPU = nullptr;
    }
    if(rangesInCPU != nullptr)
    {
        delete[] rangesInCPU->hitRanges;
        delete[] rangesInCPU->quintupletModuleIndices;
        delete rangesInCPU;
        rangesInCPU = nullptr;
    }

    if(mdsInCPU != nullptr)
    {
        delete[] mdsInCPU->anchorHitIndices;
        delete[] mdsInCPU->nMDs;
        delete[] mdsInCPU->totOccupancyMDs;
        delete mdsInCPU;
        mdsInCPU = nullptr;
    }

    if(segmentsInCPU != nullptr)
    {
        delete[] segmentsInCPU->mdIndices;
        delete[] segmentsInCPU->nSegments;
        delete[] segmentsInCPU->totOccupancySegments;
        delete[] segmentsInCPU->innerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->outerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->ptIn;
        delete[] segmentsInCPU->eta;
        delete[] segmentsInCPU->phi;
        delete segmentsInCPU;
        segmentsInCPU = nullptr;
    }

    if(tripletsInCPU != nullptr)
    {
        delete[] tripletsInCPU->segmentIndices;
        delete[] tripletsInCPU->nTriplets;
        delete[] tripletsInCPU->totOccupancyTriplets;
        delete[] tripletsInCPU->betaIn;
        delete[] tripletsInCPU->betaOut;
        delete[] tripletsInCPU->pt_beta;
        delete[] tripletsInCPU->logicalLayers;
        delete[] tripletsInCPU->lowerModuleIndices;
        delete[] tripletsInCPU->hitIndices;
        delete tripletsInCPU;
        tripletsInCPU = nullptr;
    }
    if(quintupletsInCPU != nullptr)
    {
        delete[] quintupletsInCPU->tripletIndices;
        delete[] quintupletsInCPU->nQuintuplets;
        delete[] quintupletsInCPU->totOccupancyQuintuplets;
        delete[] quintupletsInCPU->lowerModuleIndices;
        delete[] quintupletsInCPU->innerRadius;
        delete[] quintupletsInCPU->outerRadius;
        delete[] quintupletsInCPU->regressionRadius;
        delete[] quintupletsInCPU->bridgeRadius;
        delete[] quintupletsInCPU->chiSquared;
        delete[] quintupletsInCPU->rzChiSquared;
        delete[] quintupletsInCPU->nonAnchorChiSquared;
        delete quintupletsInCPU;
        quintupletsInCPU = nullptr;
    }
    if(pixelTripletsInCPU != nullptr)
    {
        delete[] pixelTripletsInCPU->tripletIndices;
        delete[] pixelTripletsInCPU->pixelSegmentIndices;
        delete[] pixelTripletsInCPU->pixelRadius;
        delete[] pixelTripletsInCPU->tripletRadius;
        delete pixelTripletsInCPU->nPixelTriplets;
        delete pixelTripletsInCPU->totOccupancyPixelTriplets;
        delete[] pixelTripletsInCPU->rzChiSquared;
        delete[] pixelTripletsInCPU->rPhiChiSquared;
        delete[] pixelTripletsInCPU->rPhiChiSquaredInwards;
        delete pixelTripletsInCPU;
        pixelTripletsInCPU = nullptr;
    }

    if(pixelQuintupletsInCPU != nullptr)
    {
        delete[] pixelQuintupletsInCPU->pixelIndices;
        delete[] pixelQuintupletsInCPU->T5Indices;
        delete[] pixelQuintupletsInCPU->isDup;
        delete[] pixelQuintupletsInCPU->score;
        delete pixelQuintupletsInCPU->nPixelQuintuplets;
        delete pixelQuintupletsInCPU->totOccupancyPixelQuintuplets;
        delete[] pixelQuintupletsInCPU->rzChiSquared;
        delete[] pixelQuintupletsInCPU->rPhiChiSquared;
        delete[] pixelQuintupletsInCPU->rPhiChiSquaredInwards;
        delete pixelQuintupletsInCPU;
        pixelQuintupletsInCPU = nullptr;
    }
    if(trackCandidatesInCPU != nullptr)
    {
        delete[] trackCandidatesInCPU->objectIndices;
        delete[] trackCandidatesInCPU->trackCandidateType;
        delete[] trackCandidatesInCPU->nTrackCandidates;
        delete[] trackCandidatesInCPU->logicalLayers;
        delete[] trackCandidatesInCPU->hitIndices;
        delete[] trackCandidatesInCPU->lowerModuleIndices;
        delete trackCandidatesInCPU;
        trackCandidatesInCPU = nullptr;
    }

    if(modulesInCPU != nullptr)
    {
        delete[] modulesInCPU->nLowerModules;
        delete[] modulesInCPU->nModules;
        delete[] modulesInCPU->detIds;
        delete[] modulesInCPU->isLower;
        delete[] modulesInCPU->layers;
        delete[] modulesInCPU->subdets;
        delete[] modulesInCPU->rings;
        delete[] modulesInCPU->rods;
        delete[] modulesInCPU->modules;
        delete[] modulesInCPU->sides;
        delete[] modulesInCPU->eta;
        delete[] modulesInCPU->r;
        delete[] modulesInCPU;
        modulesInCPU = nullptr;
    }
    if(modulesInCPUFull != nullptr)
    {
        delete[] modulesInCPUFull->detIds;
        delete[] modulesInCPUFull->moduleMap;
        delete[] modulesInCPUFull->nConnectedModules;
        delete[] modulesInCPUFull->drdzs;
        delete[] modulesInCPUFull->slopes;
        delete[] modulesInCPUFull->nModules;
        delete[] modulesInCPUFull->nLowerModules;
        delete[] modulesInCPUFull->layers;
        delete[] modulesInCPUFull->rings;
        delete[] modulesInCPUFull->modules;
        delete[] modulesInCPUFull->rods;
        delete[] modulesInCPUFull->sides;
        delete[] modulesInCPUFull->subdets;
        delete[] modulesInCPUFull->eta;
        delete[] modulesInCPUFull->r;
        delete[] modulesInCPUFull->isInverted;
        delete[] modulesInCPUFull->isLower;


        delete[] modulesInCPUFull->moduleType;
        delete[] modulesInCPUFull->moduleLayerType;
        delete[] modulesInCPUFull;
        modulesInCPUFull = nullptr;
    }

}

void SDL::initModules(const char* moduleMetaDataFilePath)
{
    hipStream_t default_stream = 0;
    if(modulesInGPU == nullptr)
    {
        hipHostMalloc(&modulesInGPU, sizeof(struct SDL::modules));
        hipHostMalloc(&pixelMapping, sizeof(struct SDL::pixelMap));
        //nModules gets filled here
        loadModulesFromFile(*modulesInGPU,nModules,nLowerModules, *pixelMapping, default_stream, moduleMetaDataFilePath);
        hipStreamSynchronize(default_stream);
    }
    //resetObjectRanges(*modulesInGPU,nModules, default_stream);
}

void SDL::cleanModules()
{
    freeModules(*modulesInGPU, *pixelMapping);
    hipHostFree(modulesInGPU);
    hipHostFree(pixelMapping);
}

void SDL::Event::resetObjectsInModule()
{
    resetObjectRanges(*rangesInGPU,nModules,stream);
}

ALPAKA_FN_HOST_ACC ALPAKA_FN_INLINE int binary_search(
    unsigned int *data, // Array that we are searching over
    unsigned int search_val, // Value we want to find in data array
    unsigned int ndata) // Number of elements in data array
{
    unsigned int low = 0;
    unsigned int high = ndata - 1;

    while(low <= high)
    {
        unsigned int mid = (low + high)/2;
        unsigned int test_val = data[mid];
        if (test_val == search_val)
            return mid;
        else if (test_val > search_val)
            high = mid - 1;
        else
            low = mid + 1;
    }
    // Couldn't find search value in array.
    return -1;
}

struct moduleRangesKernel
{
    ALPAKA_NO_HOST_ACC_WARNING
    template<typename TAcc>
    ALPAKA_FN_ACC void operator()(
        TAcc const & acc,
        struct SDL::modules *modulesInGPU,
        struct SDL::hits *hitsInGPU,
        int const & nLowerModules) const
    {
        using Dim = alpaka::Dim<TAcc>;
        using Idx = alpaka::Idx<TAcc>;
        using Vec = alpaka::Vec<Dim, Idx>;

        Vec const globalThreadIdx = alpaka::getIdx<alpaka::Grid, alpaka::Threads>(acc);
        Vec const gridThreadExtent = alpaka::getWorkDiv<alpaka::Grid, alpaka::Threads>(acc);

        for(int lowerIndex = globalThreadIdx[2]; lowerIndex < nLowerModules; lowerIndex += gridThreadExtent[2])
        {
            uint16_t upperIndex = modulesInGPU->partnerModuleIndices[lowerIndex];
            if (hitsInGPU->hitRanges[lowerIndex * 2] != -1 && hitsInGPU->hitRanges[upperIndex * 2] != -1)
            {
                hitsInGPU->hitRangesLower[lowerIndex] =  hitsInGPU->hitRanges[lowerIndex * 2]; 
                hitsInGPU->hitRangesUpper[lowerIndex] =  hitsInGPU->hitRanges[upperIndex * 2];
                hitsInGPU->hitRangesnLower[lowerIndex] = hitsInGPU->hitRanges[lowerIndex * 2 + 1] - hitsInGPU->hitRanges[lowerIndex * 2] + 1;
                hitsInGPU->hitRangesnUpper[lowerIndex] = hitsInGPU->hitRanges[upperIndex * 2 + 1] - hitsInGPU->hitRanges[upperIndex * 2] + 1;
            }
        }
    }
};

struct hitLoopKernel
{
    ALPAKA_NO_HOST_ACC_WARNING
    template<typename TAcc>
    ALPAKA_FN_ACC void operator()(
        TAcc const & acc,
        uint16_t Endcap, // Integer corresponding to endcap in module subdets
        uint16_t TwoS, // Integer corresponding to TwoS in moduleType
        unsigned int nModules, // Number of modules
        unsigned int nEndCapMap, // Number of elements in endcap map
        unsigned int* geoMapDetId, // DetId's from endcap map
        float* geoMapPhi, // Phi values from endcap map
        struct SDL::modules *modulesInGPU,
        struct SDL::hits *hitsInGPU,
        int const & nHits) const // Total number of hits in event
    {
        using Dim = alpaka::Dim<TAcc>;
        using Idx = alpaka::Idx<TAcc>;
        using Vec = alpaka::Vec<Dim, Idx>;

        Vec const globalThreadIdx = alpaka::getIdx<alpaka::Grid, alpaka::Threads>(acc);
        Vec const gridThreadExtent = alpaka::getWorkDiv<alpaka::Grid, alpaka::Threads>(acc);

        for(int ihit = globalThreadIdx[2]; ihit < nHits; ihit += gridThreadExtent[2])
        {
            float ihit_x = hitsInGPU->xs[ihit];
            float ihit_y = hitsInGPU->ys[ihit];
            float ihit_z = hitsInGPU->zs[ihit];
            int iDetId = hitsInGPU->detid[ihit];
    
            hitsInGPU->rts[ihit] = alpaka::math::sqrt(acc, ihit_x*ihit_x + ihit_y*ihit_y);
            hitsInGPU->phis[ihit] = SDL::phi(acc, ihit_x,ihit_y);
            // Acosh has no supported implementation in Alpaka right now.
            hitsInGPU->etas[ihit] = ((ihit_z>0)-(ihit_z<0)) * SDL::temp_acosh(acc, alpaka::math::sqrt(acc, ihit_x*ihit_x+ihit_y*ihit_y+ihit_z*ihit_z)/hitsInGPU->rts[ihit]);
            int found_index = binary_search(modulesInGPU->mapdetId, iDetId, nModules);
            uint16_t lastModuleIndex = modulesInGPU->mapIdx[found_index];
    
            hitsInGPU->moduleIndices[ihit] = lastModuleIndex;
    
            if(modulesInGPU->subdets[lastModuleIndex] == Endcap && modulesInGPU->moduleType[lastModuleIndex] == TwoS)
            {
                int found_index = binary_search(geoMapDetId, iDetId, nEndCapMap);
                float phi = 0;
                // Unclear why these are not in map, but CPU map returns phi = 0 for all exceptions.
                if (found_index != -1)
                    phi = geoMapPhi[found_index];
                float cos_phi = alpaka::math::cos(acc, phi);
                hitsInGPU->highEdgeXs[ihit] = ihit_x + 2.5f * cos_phi;
                hitsInGPU->lowEdgeXs[ihit] = ihit_x - 2.5f * cos_phi;
                float sin_phi = alpaka::math::sin(acc, phi);
                hitsInGPU->highEdgeYs[ihit] = ihit_y + 2.5f * sin_phi;
                hitsInGPU->lowEdgeYs[ihit] = ihit_y - 2.5f * sin_phi;
            }
            // Need to set initial value if index hasn't been seen before.
            int old = alpaka::atomicOp<alpaka::AtomicCas>(acc, &(hitsInGPU->hitRanges[lastModuleIndex * 2]), -1, ihit);
            // For subsequent visits, stores the min value.
            if (old != -1)
                alpaka::atomicOp<alpaka::AtomicMin>(acc, &hitsInGPU->hitRanges[lastModuleIndex * 2], ihit);

            alpaka::atomicOp<alpaka::AtomicMax>(acc, &hitsInGPU->hitRanges[lastModuleIndex * 2 + 1], ihit);
        }
    }
};

void SDL::Event::addHitToEvent(std::vector<float> x, std::vector<float> y, std::vector<float> z, std::vector<unsigned int> detId, std::vector<unsigned int> idxInNtuple)
{
    // Use the actual number of hits instead of a max.
    const int nHits = x.size();

    // Get current device for future use.
    hipGetDevice(&dev);

    // Initialize space on device/host for next event.
    if (hitsInGPU == nullptr)
    {
        hitsInGPU = (SDL::hits*)cms::cuda::allocate_host(sizeof(SDL::hits), stream);
        // Unclear why but this has to be 2*nHits to avoid crashing.
        createHitsInExplicitMemory(*hitsInGPU, nModules, 2*nHits, stream, 1);
    }
    if (rangesInGPU == nullptr)
    {
        rangesInGPU = (SDL::objectRanges*)cms::cuda::allocate_host(sizeof(SDL::objectRanges), stream);
    	createRangesInExplicitMemory(*rangesInGPU, nModules, stream, nLowerModules);
        resetObjectsInModule();
    }
    hipStreamSynchronize(stream);
    // Copy the host arrays to the GPU.
    hipMemcpyAsync(hitsInGPU->xs, &x[0], nHits*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitsInGPU->ys, &y[0], nHits*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitsInGPU->zs, &z[0], nHits*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitsInGPU->detid, &detId[0], nHits*sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitsInGPU->idxs, &idxInNtuple[0], nHits*sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitsInGPU->nHits, &nHits, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);

    // Temporary fix for queue initialization.
    QueueAcc queue(devAcc);

    Vec const threadsPerBlock1(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(256));
    Vec const blocksPerGrid1(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(MAX_BLOCKS));
    WorkDiv const hit_loop_workdiv(blocksPerGrid1, threadsPerBlock1, elementsPerThread);

    hitLoopKernel hit_loop_kernel;
    auto const hit_loop_task(alpaka::createTaskKernel<Acc>(
        hit_loop_workdiv,
        hit_loop_kernel,
        Endcap,
        TwoS,
        nModules,
        SDL::endcapGeometry.nEndCapMap,
        SDL::endcapGeometry.geoMapDetId,
        SDL::endcapGeometry.geoMapPhi,
        modulesInGPU,
        hitsInGPU,
        nHits));

    alpaka::enqueue(queue, hit_loop_task);
    alpaka::wait(queue);

    Vec const threadsPerBlock2(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(256));
    Vec const blocksPerGrid2(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(MAX_BLOCKS));
    WorkDiv const module_ranges_workdiv(blocksPerGrid2, threadsPerBlock2, elementsPerThread);

    moduleRangesKernel module_ranges_kernel;
    auto const module_ranges_task(alpaka::createTaskKernel<Acc>(
        module_ranges_workdiv,
        module_ranges_kernel,
        modulesInGPU,
        hitsInGPU,
        nLowerModules));

    // Waiting isn't needed after second kernel call. Saves ~100 us.
    // This is because addPixelSegmentToEvent (which is run next) doesn't rely on hitsinGPU->hitrange variables.
    // Also, modulesInGPU->partnerModuleIndices is not alterned in addPixelSegmentToEvent.
    alpaka::enqueue(queue, module_ranges_task);
}

struct addPixelSegmentToEventKernel
{
    ALPAKA_NO_HOST_ACC_WARNING
    template<typename TAcc>
    ALPAKA_FN_ACC void operator()(
        TAcc const & acc,
        struct SDL::modules& modulesInGPU,
        struct SDL::objectRanges& rangesInGPU,
        struct SDL::hits& hitsInGPU,
        struct SDL::miniDoublets& mdsInGPU,
        struct SDL::segments& segmentsInGPU,
        unsigned int* hitIndices0,
        unsigned int* hitIndices1,
        unsigned int* hitIndices2,
        unsigned int* hitIndices3,
        float* dPhiChange,
        uint16_t pixelModuleIndex,
        const int size) const
    {
        using Dim = alpaka::Dim<TAcc>;
        using Idx = alpaka::Idx<TAcc>;
        using Vec = alpaka::Vec<Dim, Idx>;

        Vec const globalThreadIdx = alpaka::getIdx<alpaka::Grid, alpaka::Threads>(acc);
        Vec const gridThreadExtent = alpaka::getWorkDiv<alpaka::Grid, alpaka::Threads>(acc);

        for(int tid = globalThreadIdx[2]; tid < size; tid += gridThreadExtent[2])
        {
            unsigned int innerMDIndex = rangesInGPU.miniDoubletModuleIndices[pixelModuleIndex] + 2*(tid);
            unsigned int outerMDIndex = rangesInGPU.miniDoubletModuleIndices[pixelModuleIndex] + 2*(tid) +1;
            unsigned int pixelSegmentIndex = rangesInGPU.segmentModuleIndices[pixelModuleIndex] + tid;

            addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,innerMDIndex);
            addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,outerMDIndex);

            //in outer hits - pt, eta, phi
            float slope = SDL::temp_sinh(acc, hitsInGPU.ys[mdsInGPU.outerHitIndices[innerMDIndex]]);
            float intercept = hitsInGPU.zs[mdsInGPU.anchorHitIndices[innerMDIndex]] - slope * hitsInGPU.rts[mdsInGPU.anchorHitIndices[innerMDIndex]];
            float score_lsq=(hitsInGPU.rts[mdsInGPU.anchorHitIndices[outerMDIndex]] * slope + intercept) - (hitsInGPU.zs[mdsInGPU.anchorHitIndices[outerMDIndex]]);
            score_lsq = score_lsq * score_lsq;

            unsigned int hits1[4];
            hits1[0] = hitsInGPU.idxs[mdsInGPU.anchorHitIndices[innerMDIndex]];
            hits1[1] = hitsInGPU.idxs[mdsInGPU.anchorHitIndices[outerMDIndex]];
            hits1[2] = hitsInGPU.idxs[mdsInGPU.outerHitIndices[innerMDIndex]];
            hits1[3] = hitsInGPU.idxs[mdsInGPU.outerHitIndices[outerMDIndex]];
            addPixelSegmentToMemory(acc, segmentsInGPU, mdsInGPU, innerMDIndex, outerMDIndex, pixelModuleIndex, hits1, hitIndices0[tid], hitIndices2[tid], dPhiChange[tid], pixelSegmentIndex, tid, score_lsq);
        }
    }
};

void SDL::Event::addPixelSegmentToEvent(std::vector<unsigned int> hitIndices0,std::vector<unsigned int> hitIndices1,std::vector<unsigned int> hitIndices2,std::vector<unsigned int> hitIndices3, std::vector<float> dPhiChange, std::vector<float> ptIn, std::vector<float> ptErr, std::vector<float> px, std::vector<float> py, std::vector<float> pz, std::vector<float> eta, std::vector<float> etaErr, std::vector<float> phi, std::vector<int> charge, std::vector<unsigned int> seedIdx, std::vector<int> superbin, std::vector<int8_t> pixelType, std::vector<char> isQuad)
{
    const int size = ptIn.size();
    unsigned int mdSize = 2 * size;
    uint16_t pixelModuleIndex = (*detIdToIndex)[1];

    if(mdsInGPU == nullptr)
    {
        mdsInGPU = (SDL::miniDoublets*)cms::cuda::allocate_host(sizeof(SDL::miniDoublets), stream);
        unsigned int nTotalMDs;
        hipMemsetAsync(&rangesInGPU->miniDoubletModuleOccupancy[nLowerModules],N_MAX_PIXEL_MD_PER_MODULES, sizeof(unsigned int),stream);
        createMDArrayRangesGPU<<<1,1024,0,stream>>>(*modulesInGPU, *rangesInGPU);
        hipMemcpyAsync(&nTotalMDs,rangesInGPU->device_nTotalMDs,sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        nTotalMDs+= N_MAX_PIXEL_MD_PER_MODULES;
        createMDsInExplicitMemory(*mdsInGPU, nTotalMDs, nLowerModules, N_MAX_PIXEL_MD_PER_MODULES,stream);
        hipMemcpyAsync(mdsInGPU->nMemoryLocations, &nTotalMDs, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream);
    }
    if(segmentsInGPU == nullptr)
    {
        segmentsInGPU = (SDL::segments*)cms::cuda::allocate_host(sizeof(SDL::segments), stream);
        // can be optimized here: because we didn't distinguish pixel segments and outer-tracker segments and call them both "segments", so they use the index continuously.
        // If we want to further study the memory footprint in detail, we can separate the two and allocate different memories to them
        createSegmentArrayRanges<<<1,1024,0,stream>>>(*modulesInGPU, *rangesInGPU, *mdsInGPU);
        hipMemcpyAsync(&nTotalSegments,rangesInGPU->device_nTotalSegs,sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        nTotalSegments += N_MAX_PIXEL_SEGMENTS_PER_MODULE;
        createSegmentsInExplicitMemory(*segmentsInGPU, nTotalSegments, nLowerModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE,stream);

        hipMemcpyAsync(segmentsInGPU->nMemoryLocations, &nTotalSegments, sizeof(unsigned int), hipMemcpyHostToDevice, stream);;
        hipStreamSynchronize(stream);
    }

    unsigned int* hitIndices0_dev = (unsigned int*)cms::cuda::allocate_device(dev, size*sizeof(unsigned int), stream);
    unsigned int* hitIndices1_dev = (unsigned int*)cms::cuda::allocate_device(dev, size*sizeof(unsigned int), stream);
    unsigned int* hitIndices2_dev = (unsigned int*)cms::cuda::allocate_device(dev, size*sizeof(unsigned int), stream);
    unsigned int* hitIndices3_dev = (unsigned int*)cms::cuda::allocate_device(dev, size*sizeof(unsigned int), stream);
    float* dPhiChange_dev = (float*)cms::cuda::allocate_device(dev, size*sizeof(float), stream);

    hipMemcpyAsync(hitIndices0_dev, &hitIndices0[0], size*sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitIndices1_dev, &hitIndices1[0], size*sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitIndices2_dev, &hitIndices2[0], size*sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(hitIndices3_dev, &hitIndices3[0], size*sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(dPhiChange_dev, &dPhiChange[0], size*sizeof(float), hipMemcpyHostToDevice, stream);

    hipMemcpyAsync(segmentsInGPU->isQuad, &isQuad[0], size*sizeof(char), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->ptIn, &ptIn[0], size*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->ptErr, &ptErr[0], size*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->px, &px[0], size*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->py, &py[0], size*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->pz, &pz[0], size*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->etaErr, &etaErr[0], size*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->eta, &eta[0], size*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->phi, &phi[0], size*sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->charge, &charge[0], size*sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->seedIdx, &seedIdx[0], size*sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->superbin, &superbin[0], size*sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(segmentsInGPU->pixelType, &pixelType[0], size*sizeof(int8_t), hipMemcpyHostToDevice, stream);

    hipMemcpyAsync(&(segmentsInGPU->nSegments)[pixelModuleIndex], &size, sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(&(segmentsInGPU->totOccupancySegments)[pixelModuleIndex], &size, sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(&(mdsInGPU->nMDs)[pixelModuleIndex], &mdSize, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(&(mdsInGPU->totOccupancyMDs)[pixelModuleIndex], &mdSize, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);

    // Temporary fix for queue initialization.
    QueueAcc queue(devAcc);

    Vec const threadsPerBlock(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(256));
    Vec const blocksPerGrid(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(MAX_BLOCKS));
    WorkDiv const addPixelSegmentToEvent_workdiv(blocksPerGrid, threadsPerBlock, elementsPerThread);

    addPixelSegmentToEventKernel addPixelSegmentToEvent_kernel;
    auto const addPixelSegmentToEvent_task(alpaka::createTaskKernel<Acc>(
        addPixelSegmentToEvent_workdiv,
        addPixelSegmentToEvent_kernel,
        *modulesInGPU,
        *rangesInGPU,
        *hitsInGPU,
        *mdsInGPU,
        *segmentsInGPU,
        hitIndices0_dev,
        hitIndices1_dev,
        hitIndices2_dev,
        hitIndices3_dev,
        dPhiChange_dev,
        pixelModuleIndex,
        size));

    alpaka::enqueue(queue, addPixelSegmentToEvent_task);
    alpaka::wait(queue);

    cms::cuda::free_device(dev, hitIndices0_dev);
    cms::cuda::free_device(dev, hitIndices1_dev);
    cms::cuda::free_device(dev, hitIndices2_dev);
    cms::cuda::free_device(dev, hitIndices3_dev);
    cms::cuda::free_device(dev, dPhiChange_dev);
    hipStreamSynchronize(stream);
}

void SDL::Event::addMiniDoubletsToEventExplicit()
{
    unsigned int* nMDsCPU;
    nMDsCPU = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(nMDsCPU,mdsInGPU->nMDs,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    short* module_subdets;
    module_subdets = (short*)cms::cuda::allocate_host(nLowerModules* sizeof(short), stream);
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    module_layers = (short*)cms::cuda::allocate_host(nLowerModules * sizeof(short), stream);
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_hitRanges;
    module_hitRanges = (int*)cms::cuda::allocate_host(nLowerModules* 2*sizeof(int), stream);
    hipMemcpyAsync(module_hitRanges,hitsInGPU->hitRanges,nLowerModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);

    hipStreamSynchronize(stream);

    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        if(!(nMDsCPU[i] == 0 or module_hitRanges[i * 2] == -1))
        {
            if(module_subdets[i] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[module_layers[i] -1] += nMDsCPU[i];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[module_layers[i] - 1] += nMDsCPU[i];
            }

        }
    }
    cms::cuda::free_host(nMDsCPU);
    cms::cuda::free_host(module_subdets);
    cms::cuda::free_host(module_layers);
    cms::cuda::free_host(module_hitRanges);
}

void SDL::Event::addSegmentsToEventExplicit()
{
    unsigned int* nSegmentsCPU;
    nSegmentsCPU = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(nSegmentsCPU,segmentsInGPU->nSegments,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);

    short* module_subdets;
    module_subdets = (short*)cms::cuda::allocate_host(nLowerModules* sizeof(short), stream);
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    module_layers = (short*)cms::cuda::allocate_host(nLowerModules * sizeof(short), stream);
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);

    hipStreamSynchronize(stream);
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        if(!(nSegmentsCPU[i] == 0))
        {
            if(module_subdets[i] == Barrel)
            {
                n_segments_by_layer_barrel_[module_layers[i] - 1] += nSegmentsCPU[i];
            }
            else
            {
                n_segments_by_layer_endcap_[module_layers[i] -1] += nSegmentsCPU[i];
            }
        }
    }
    cms::cuda::free_host(nSegmentsCPU);
    cms::cuda::free_host(module_subdets);
    cms::cuda::free_host(module_layers);
}

void SDL::Event::createMiniDoublets()
{
    //hardcoded range numbers for this will come from studies!
    unsigned int nTotalMDs;
    hipMemsetAsync(&rangesInGPU->miniDoubletModuleOccupancy[nLowerModules],N_MAX_PIXEL_MD_PER_MODULES, sizeof(unsigned int),stream);
    createMDArrayRangesGPU<<<1,1024,0,stream>>>(*modulesInGPU, *rangesInGPU); 
    hipMemcpyAsync(&nTotalMDs,rangesInGPU->device_nTotalMDs,sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    nTotalMDs+=N_MAX_PIXEL_MD_PER_MODULES;

    if(mdsInGPU == nullptr)
    {
        mdsInGPU = (SDL::miniDoublets*)cms::cuda::allocate_host(sizeof(SDL::miniDoublets), stream);

        //FIXME: Add memory locations for pixel MDs
        createMDsInExplicitMemory(*mdsInGPU, nTotalMDs, nLowerModules, N_MAX_PIXEL_MD_PER_MODULES, stream);

    }
    hipStreamSynchronize(stream);

    int maxThreadsPerModule=0;
    int* module_hitRanges;
    module_hitRanges = (int*)cms::cuda::allocate_host(nModules* 2*sizeof(int), stream);
    hipMemcpyAsync(module_hitRanges,hitsInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost,stream);
    bool* module_isLower;
    module_isLower = (bool*)cms::cuda::allocate_host(nModules*sizeof(bool), stream);
    hipMemcpyAsync(module_isLower,modulesInGPU->isLower,nModules*sizeof(bool),hipMemcpyDeviceToHost,stream);
    bool* module_isInverted;
    module_isInverted = (bool*)cms::cuda::allocate_host(nModules*sizeof(bool), stream);
    hipMemcpyAsync(module_isInverted,modulesInGPU->isInverted,nModules*sizeof(bool),hipMemcpyDeviceToHost,stream);
    int* module_partnerModuleIndices;
    module_partnerModuleIndices = (int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(module_partnerModuleIndices, modulesInGPU->partnerModuleIndices, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    for (uint16_t lowerModuleIndex=0; lowerModuleIndex<nLowerModules; lowerModuleIndex++) 
    {
        uint16_t upperModuleIndex = module_partnerModuleIndices[lowerModuleIndex];
        int lowerHitRanges = module_hitRanges[lowerModuleIndex*2];
        int upperHitRanges = module_hitRanges[upperModuleIndex*2];
        if(lowerHitRanges!=-1 && upperHitRanges!=-1) 
        {
            int nLowerHits = module_hitRanges[lowerModuleIndex * 2 + 1] - lowerHitRanges + 1;
            int nUpperHits = module_hitRanges[upperModuleIndex * 2 + 1] - upperHitRanges + 1;
            maxThreadsPerModule = maxThreadsPerModule > (nLowerHits*nUpperHits) ? maxThreadsPerModule : nLowerHits*nUpperHits;
        }
    }
    cms::cuda::free_host(module_hitRanges);
    cms::cuda::free_host(module_partnerModuleIndices);
    cms::cuda::free_host(module_isLower);
    cms::cuda::free_host(module_isInverted);

    // Temporary fix for queue initialization.
    QueueAcc queue(devAcc);

    Vec const threadsPerBlock(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(32));
    Vec const blocksPerGrid(static_cast<Idx>(1), static_cast<Idx>(MAX_BLOCKS), static_cast<Idx>(1));

    WorkDiv const createMiniDoubletsInGPUv2_workDiv(blocksPerGrid, threadsPerBlock, elementsPerThread);

    SDL::createMiniDoubletsInGPUv2 createMiniDoubletsInGPUv2_kernel;
    auto const createMiniDoubletsInGPUv2Task(alpaka::createTaskKernel<Acc>(
        createMiniDoubletsInGPUv2_workDiv,
        createMiniDoubletsInGPUv2_kernel,
        *modulesInGPU,
        *hitsInGPU,
        *mdsInGPU,
        *rangesInGPU));

    alpaka::enqueue(queue, createMiniDoubletsInGPUv2Task);
    alpaka::wait(queue);

    addMiniDoubletRangesToEventExplicit<<<1,1024,0,stream>>>(*modulesInGPU,*mdsInGPU, *rangesInGPU,*hitsInGPU);
    hipStreamSynchronize(stream);

    if(addObjects){
      addMiniDoubletsToEventExplicit();
    }

}

void SDL::Event::createSegmentsWithModuleMap()
{
    if(segmentsInGPU == nullptr)
    {
        segmentsInGPU = (SDL::segments*)cms::cuda::allocate_host(sizeof(SDL::segments), stream);
        createSegmentsInExplicitMemory(*segmentsInGPU, nTotalSegments, nLowerModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE,stream);
    }

    // Temporary fix for queue initialization.
    QueueAcc queue(devAcc);

    Vec const threadsPerBlock(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(64));
    Vec const blocksPerGrid(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(nLowerModules));

    WorkDiv const createSegmentsInGPUv2_workDiv(blocksPerGrid, threadsPerBlock, elementsPerThread);

    SDL::createSegmentsInGPUv2 createSegmentsInGPUv2_kernel;
    auto const createSegmentsInGPUv2Task(alpaka::createTaskKernel<Acc>(
        createSegmentsInGPUv2_workDiv,
        createSegmentsInGPUv2_kernel,
        *modulesInGPU,
        *mdsInGPU,
        *segmentsInGPU,
        *rangesInGPU));

    alpaka::enqueue(queue, createSegmentsInGPUv2Task);
    alpaka::wait(queue);

    addSegmentRangesToEventExplicit<<<1,1024,0,stream>>>(*modulesInGPU,*segmentsInGPU, *rangesInGPU);
    hipStreamSynchronize(stream);

    if(addObjects){
      addSegmentsToEventExplicit();
    }
}


void SDL::Event::createTriplets()
{
    if(tripletsInGPU == nullptr)
    {
        tripletsInGPU = (SDL::triplets*)cms::cuda::allocate_host(sizeof(SDL::triplets), stream);
        unsigned int maxTriplets;
        createTripletArrayRanges<<<1,1024,0,stream>>>(*modulesInGPU, *rangesInGPU, *segmentsInGPU);
        hipMemcpyAsync(&maxTriplets,rangesInGPU->device_nTotalTrips,sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        createTripletsInExplicitMemory(*tripletsInGPU, maxTriplets, nLowerModules,stream);

        hipMemcpyAsync(tripletsInGPU->nMemoryLocations, &maxTriplets, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream);
    }
    //TODO:Move this also inside the ranges function
    uint16_t nonZeroModules=0;
    unsigned int max_InnerSeg=0;
    uint16_t *index = (uint16_t*)malloc(nLowerModules*sizeof(unsigned int));
    uint16_t *index_gpu;
    index_gpu = (uint16_t*)cms::cuda::allocate_device(dev, nLowerModules*sizeof(uint16_t), stream);
    unsigned int *nSegments = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    hipMemcpyAsync((void *)nSegments, segmentsInGPU->nSegments, nLowerModules*sizeof(unsigned int), hipMemcpyDeviceToHost,stream); 
    hipStreamSynchronize(stream);

    uint16_t* module_nConnectedModules;
    module_nConnectedModules = (uint16_t*)cms::cuda::allocate_host(nLowerModules* sizeof(uint16_t), stream);
    hipMemcpyAsync(module_nConnectedModules,modulesInGPU->nConnectedModules,nLowerModules*sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    for (uint16_t innerLowerModuleIndex = 0; innerLowerModuleIndex <nLowerModules; innerLowerModuleIndex++) 
    {
        uint16_t nConnectedModules = module_nConnectedModules[innerLowerModuleIndex];
        unsigned int nInnerSegments = nSegments[innerLowerModuleIndex];
        if (nConnectedModules != 0 and nInnerSegments != 0) 
        {
            index[nonZeroModules] = innerLowerModuleIndex;
            nonZeroModules++;
        }
        max_InnerSeg = max(max_InnerSeg, nInnerSegments);
    }
    cms::cuda::free_host(module_nConnectedModules);
    hipMemcpyAsync(index_gpu, index, nonZeroModules*sizeof(uint16_t), hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);

    // Temporary fix for queue initialization.
    QueueAcc queue(devAcc);

    Vec const threadsPerBlock(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(16));
    Vec const blocksPerGrid(static_cast<Idx>(MAX_BLOCKS), static_cast<Idx>(1), static_cast<Idx>(1));

    WorkDiv const createTripletsInGPUv2_workDiv(blocksPerGrid, threadsPerBlock, elementsPerThread);

    SDL::createTripletsInGPUv2 createTripletsInGPUv2_kernel;
    auto const createTripletsInGPUv2Task(alpaka::createTaskKernel<Acc>(
        createTripletsInGPUv2_workDiv,
        createTripletsInGPUv2_kernel,
        *modulesInGPU,
        *mdsInGPU,
        *segmentsInGPU,
        *tripletsInGPU,
        *rangesInGPU,
        index_gpu,
        nonZeroModules));

    alpaka::enqueue(queue, createTripletsInGPUv2Task);
    alpaka::wait(queue);

    addTripletRangesToEventExplicit<<<1,1024,0,stream>>>(*modulesInGPU,*tripletsInGPU,*rangesInGPU);
    hipStreamSynchronize(stream);

    free(nSegments);
    free(index);
    cms::cuda::free_device(dev, index_gpu);

    if(addObjects){
      addTripletsToEventExplicit();
    }
}

void SDL::Event::createTrackCandidates()
{
    uint16_t nEligibleModules;
    hipMemcpyAsync(&nEligibleModules,rangesInGPU->nEligibleT5Modules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    if(trackCandidatesInGPU == nullptr)
    {
        trackCandidatesInGPU = (SDL::trackCandidates*)cms::cuda::allocate_host(sizeof(SDL::trackCandidates), stream);
        createTrackCandidatesInExplicitMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES + N_MAX_PIXEL_TRACK_CANDIDATES,stream);
    }

    // Temporary fix for queue initialization.
    QueueAcc queue(devAcc);

    Vec const threadsPerBlock_crossCleanpT3(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(64));
    Vec const blocksPerGrid_crossCleanpT3(static_cast<Idx>(1), static_cast<Idx>(4), static_cast<Idx>(20));

    WorkDiv const crossCleanpT3_workDiv(blocksPerGrid_crossCleanpT3, blocksPerGrid_crossCleanpT3, elementsPerThread);

    SDL::crossCleanpT3 crossCleanpT3_kernel;
    auto const crossCleanpT3Task(alpaka::createTaskKernel<Acc>(
        crossCleanpT3_workDiv,
        crossCleanpT3_kernel,
        *modulesInGPU,
        *rangesInGPU,
        *pixelTripletsInGPU,
        *segmentsInGPU,
        *pixelQuintupletsInGPU));

    alpaka::enqueue(queue, crossCleanpT3Task);
    alpaka::wait(queue);

    //adding objects
    Vec const threadsPerBlock_addpT3asTrackCandidatesInGPU(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(512));
    Vec const blocksPerGrid_addpT3asTrackCandidatesInGPU(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(1));

    WorkDiv const addpT3asTrackCandidatesInGPU_workDiv(blocksPerGrid_addpT3asTrackCandidatesInGPU, threadsPerBlock_addpT3asTrackCandidatesInGPU, elementsPerThread);

    SDL::addpT3asTrackCandidatesInGPU addpT3asTrackCandidatesInGPU_kernel;
    auto const addpT3asTrackCandidatesInGPUTask(alpaka::createTaskKernel<Acc>(
        addpT3asTrackCandidatesInGPU_workDiv,
        addpT3asTrackCandidatesInGPU_kernel,
        *pixelTripletsInGPU,
        *trackCandidatesInGPU));

    alpaka::enqueue(queue, addpT3asTrackCandidatesInGPUTask);
    alpaka::wait(queue);

    Vec const threadsPerBlockRemoveDupQuints(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(32));
    Vec const blocksPerGridRemoveDupQuints(static_cast<Idx>(1), static_cast<Idx>(max(nEligibleModules/16,1)), static_cast<Idx>(max(nEligibleModules/32,1)));

    WorkDiv const removeDupQuintupletsInGPUBeforeTC_workDiv(blocksPerGridRemoveDupQuints, threadsPerBlockRemoveDupQuints, elementsPerThread);

    SDL::removeDupQuintupletsInGPUBeforeTC removeDupQuintupletsInGPUBeforeTC_kernel;
    auto const removeDupQuintupletsInGPUBeforeTCTask(alpaka::createTaskKernel<Acc>(
        removeDupQuintupletsInGPUBeforeTC_workDiv,
        removeDupQuintupletsInGPUBeforeTC_kernel,
        *quintupletsInGPU,
        *rangesInGPU));

    alpaka::enqueue(queue, removeDupQuintupletsInGPUBeforeTCTask);
    alpaka::wait(queue);

    Vec const threadsPerBlock_crossCleanT5(static_cast<Idx>(32), static_cast<Idx>(1), static_cast<Idx>(32));
    Vec const blocksPerGrid_crossCleanT5(static_cast<Idx>((13296/32) + 1), static_cast<Idx>(1), static_cast<Idx>(MAX_BLOCKS));

    WorkDiv const crossCleanT5_workDiv(blocksPerGrid_crossCleanT5, threadsPerBlock_crossCleanT5, elementsPerThread);

    SDL::crossCleanT5 crossCleanT5_kernel;
    auto const crossCleanT5Task(alpaka::createTaskKernel<Acc>(
        crossCleanT5_workDiv,
        crossCleanT5_kernel,
        *modulesInGPU,
        *quintupletsInGPU,
        *pixelQuintupletsInGPU,
        *pixelTripletsInGPU,
        *rangesInGPU));

    alpaka::enqueue(queue, crossCleanT5Task);
    alpaka::wait(queue);

    Vec const threadsPerBlock_addT5asTrackCandidateInGPU(static_cast<Idx>(1), static_cast<Idx>(8), static_cast<Idx>(128));
    Vec const blocksPerGrid_addT5asTrackCandidateInGPU(static_cast<Idx>(1), static_cast<Idx>(8), static_cast<Idx>(10));

    WorkDiv const addT5asTrackCandidateInGPU_workDiv(blocksPerGrid_addT5asTrackCandidateInGPU, threadsPerBlock_addT5asTrackCandidateInGPU, elementsPerThread);

    SDL::addT5asTrackCandidateInGPU addT5asTrackCandidateInGPU_kernel;
    auto const addT5asTrackCandidateInGPUTask(alpaka::createTaskKernel<Acc>(
        addT5asTrackCandidateInGPU_workDiv,
        addT5asTrackCandidateInGPU_kernel,
        *modulesInGPU,
        *rangesInGPU,
        *quintupletsInGPU,
        *trackCandidatesInGPU));

    alpaka::enqueue(queue, addT5asTrackCandidateInGPUTask);
    alpaka::wait(queue);

    Vec const threadsPerBlockCheckHitspLS(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(16));
    Vec const blocksPerGridCheckHitspLS(static_cast<Idx>(1), static_cast<Idx>(MAX_BLOCKS*4), static_cast<Idx>(MAX_BLOCKS/4));

    WorkDiv const checkHitspLS_workDiv(blocksPerGridCheckHitspLS, threadsPerBlockCheckHitspLS, elementsPerThread);

    SDL::checkHitspLS checkHitspLS_kernel;
    auto const checkHitspLSTask(alpaka::createTaskKernel<Acc>(
        checkHitspLS_workDiv,
        checkHitspLS_kernel,
        *modulesInGPU,
        *segmentsInGPU,
        true));

    alpaka::enqueue(queue, checkHitspLSTask);
    alpaka::wait(queue);

    Vec const threadsPerBlock_crossCleanpLS(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(32));
    Vec const blocksPerGrid_crossCleanpLS(static_cast<Idx>(1), static_cast<Idx>(4), static_cast<Idx>(20));

    WorkDiv const crossCleanpLS_workDiv(blocksPerGrid_crossCleanpLS, threadsPerBlock_crossCleanpLS, elementsPerThread);

    SDL::crossCleanpLS crossCleanpLS_kernel;
    auto const crossCleanpLSTask(alpaka::createTaskKernel<Acc>(
        crossCleanpLS_workDiv,
        crossCleanpLS_kernel,
        *modulesInGPU,
        *rangesInGPU,
        *pixelTripletsInGPU,
        *trackCandidatesInGPU,
        *segmentsInGPU,
        *mdsInGPU,
        *hitsInGPU,
        *quintupletsInGPU));

    alpaka::enqueue(queue, crossCleanpLSTask);
    alpaka::wait(queue);

    Vec const threadsPerBlock_addpLSasTrackCandidateInGPU(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(384));
    Vec const blocksPerGrid_addpLSasTrackCandidateInGPU(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(MAX_BLOCKS));

    WorkDiv const addpLSasTrackCandidateInGPU_workDiv(blocksPerGrid_addpLSasTrackCandidateInGPU, threadsPerBlock_addpLSasTrackCandidateInGPU, elementsPerThread);

    SDL::addpLSasTrackCandidateInGPU addpLSasTrackCandidateInGPU_kernel;
    auto const addpLSasTrackCandidateInGPUTask(alpaka::createTaskKernel<Acc>(
        addpLSasTrackCandidateInGPU_workDiv,
        addpLSasTrackCandidateInGPU_kernel,
        *modulesInGPU,
        *trackCandidatesInGPU,
        *segmentsInGPU));

    alpaka::enqueue(queue, addpLSasTrackCandidateInGPUTask);
    alpaka::wait(queue);
}

void SDL::Event::createPixelTriplets()
{
    if(pixelTripletsInGPU == nullptr)
    {
        pixelTripletsInGPU = (SDL::pixelTriplets*)cms::cuda::allocate_host(sizeof(SDL::pixelTriplets), stream);
    }

    createPixelTripletsInExplicitMemory(*pixelTripletsInGPU, N_MAX_PIXEL_TRIPLETS,stream);

    unsigned int pixelModuleIndex = nLowerModules;
    int* superbins;
    int8_t* pixelTypes;
    unsigned int *nTriplets;
    unsigned int nInnerSegments = 0;
    hipMemcpyAsync(&nInnerSegments, &(segmentsInGPU->nSegments[pixelModuleIndex]), sizeof(int), hipMemcpyDeviceToHost,stream);
    nTriplets = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(nTriplets, tripletsInGPU->nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    superbins = (int*)cms::cuda::allocate_host(N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int), stream);
    pixelTypes = (int8_t*)cms::cuda::allocate_host(N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t), stream);

    hipMemcpyAsync(superbins,segmentsInGPU->superbin,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(pixelTypes,segmentsInGPU->pixelType,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t),hipMemcpyDeviceToHost,stream);

    unsigned int* connectedPixelSize_host;
    unsigned int* connectedPixelIndex_host;
    connectedPixelSize_host = (unsigned int*)cms::cuda::allocate_host(nInnerSegments* sizeof(unsigned int), stream);
    connectedPixelIndex_host = (unsigned int*)cms::cuda::allocate_host(nInnerSegments* sizeof(unsigned int), stream);
    unsigned int* connectedPixelSize_dev;
    unsigned int* connectedPixelIndex_dev;
    connectedPixelSize_dev = (unsigned int*)cms::cuda::allocate_device(dev, nInnerSegments*sizeof(unsigned int), stream);
    connectedPixelIndex_dev = (unsigned int*)cms::cuda::allocate_device(dev, nInnerSegments*sizeof(unsigned int), stream);

    hipStreamSynchronize(stream);
    int pixelIndexOffsetPos = pixelMapping->connectedPixelsIndex[44999] + pixelMapping->connectedPixelsSizes[44999];
    int pixelIndexOffsetNeg = pixelMapping->connectedPixelsIndexPos[44999] + pixelMapping->connectedPixelsSizes[44999] + pixelIndexOffsetPos;

    // TODO: check if a map/reduction to just eligible pLSs would speed up the kernel
    // the current selection still leaves a significant fraction of unmatchable pLSs
    for (unsigned int i = 0; i < nInnerSegments; i++)
    {// loop over # pLS
        int8_t pixelType = pixelTypes[i];// get pixel type for this pLS
        int superbin = superbins[i]; //get superbin for this pixel
        if((superbin < 0) or (superbin >= 45000) or (pixelType > 2) or (pixelType < 0))
        {
            connectedPixelSize_host[i] = 0;
            connectedPixelIndex_host[i] = 0;
            continue;
        }

        if(pixelType ==0)
        { // used pixel type to select correct size-index arrays
            connectedPixelSize_host[i]  = pixelMapping->connectedPixelsSizes[superbin]; //number of connected modules to this pixel
            auto connectedIdxBase = pixelMapping->connectedPixelsIndex[superbin];
            connectedPixelIndex_host[i] = connectedIdxBase;// index to get start of connected modules for this superbin in map
            // printf("i %d out of nInnerSegments %d type %d superbin %d connectedPixelIndex %d connectedPixelSize %d\n",
            //        i, nInnerSegments, pixelType, superbin, connectedPixelIndex_host[i], connectedPixelSize_host[i]);
        }
        else if(pixelType ==1)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesPos[superbin]; //number of pixel connected modules
            auto connectedIdxBase = pixelMapping->connectedPixelsIndexPos[superbin]+pixelIndexOffsetPos;
            connectedPixelIndex_host[i] = connectedIdxBase;// index to get start of connected pixel modules
        }
        else if(pixelType ==2)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesNeg[superbin]; //number of pixel connected modules
            auto connectedIdxBase = pixelMapping->connectedPixelsIndexNeg[superbin] + pixelIndexOffsetNeg;
            connectedPixelIndex_host[i] = connectedIdxBase;// index to get start of connected pixel modules
        }
    }

    hipMemcpyAsync(connectedPixelSize_dev, connectedPixelSize_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(connectedPixelIndex_dev, connectedPixelIndex_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);

    cms::cuda::free_host(connectedPixelSize_host);
    cms::cuda::free_host(connectedPixelIndex_host);
    cms::cuda::free_host(superbins);
    cms::cuda::free_host(pixelTypes);
    cms::cuda::free_host(nTriplets);

    // Temporary fix for queue initialization.
    QueueAcc queue(devAcc);

    Vec const threadsPerBlock(static_cast<Idx>(1), static_cast<Idx>(4), static_cast<Idx>(32));
    Vec const blocksPerGrid(static_cast<Idx>(16 /* above median of connected modules*/), static_cast<Idx>(4096), static_cast<Idx>(1));

    WorkDiv const createPixelTripletsInGPUFromMapv2_workDiv(blocksPerGrid, threadsPerBlock, elementsPerThread);

    SDL::createPixelTripletsInGPUFromMapv2 createPixelTripletsInGPUFromMapv2_kernel;
    auto const createPixelTripletsInGPUFromMapv2Task(alpaka::createTaskKernel<Acc>(
        createPixelTripletsInGPUFromMapv2_workDiv,
        createPixelTripletsInGPUFromMapv2_kernel,
        *modulesInGPU,
        *rangesInGPU,
        *mdsInGPU,
        *segmentsInGPU,
        *tripletsInGPU,
        *pixelTripletsInGPU,
        connectedPixelSize_dev,
        connectedPixelIndex_dev,
        nInnerSegments));

    alpaka::enqueue(queue, createPixelTripletsInGPUFromMapv2Task);
    alpaka::wait(queue);

    cms::cuda::free_device(dev, connectedPixelSize_dev);
    cms::cuda::free_device(dev, connectedPixelIndex_dev);


#ifdef Warnings
    int nPixelTriplets;
    hipMemcpyAsync(&nPixelTriplets, pixelTripletsInGPU->nPixelTriplets,  sizeof(int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    std::cout<<"number of pixel triplets = "<<nPixelTriplets<<std::endl;
#endif

    //pT3s can be cleaned here because they're not used in making pT5s!
    Vec const threadsPerBlockDupPixTrip(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(16));
    //seems like more blocks lead to conflicting writes
    Vec const blocksPerGridDupPixTrip(static_cast<Idx>(1), static_cast<Idx>(40), static_cast<Idx>(1));

    WorkDiv const removeDupPixelTripletsInGPUFromMap_workDiv(blocksPerGridDupPixTrip, threadsPerBlockDupPixTrip, elementsPerThread);

    SDL::removeDupPixelTripletsInGPUFromMap removeDupPixelTripletsInGPUFromMap_kernel;
    auto const removeDupPixelTripletsInGPUFromMapTask(alpaka::createTaskKernel<Acc>(
        removeDupPixelTripletsInGPUFromMap_workDiv,
        removeDupPixelTripletsInGPUFromMap_kernel,
        *pixelTripletsInGPU,
        false));

    alpaka::enqueue(queue, removeDupPixelTripletsInGPUFromMapTask);
    alpaka::wait(queue);
}

void SDL::Event::createQuintuplets()
{
    uint16_t nEligibleT5Modules = 0;

#ifdef CACHE_ALLOC
    rangesInGPU->indicesOfEligibleT5Modules = (uint16_t*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(uint16_t), stream);
#else
    hipMalloc(&(rangesInGPU->indicesOfEligibleT5Modules), nLowerModules * sizeof(uint16_t));
#endif
    hipMemsetAsync(rangesInGPU->quintupletModuleIndices, -1, sizeof(int) * (nLowerModules),stream);
    hipStreamSynchronize(stream);
    unsigned int nTotalQuintuplets;

    // Temporary fix for queue initialization.
    QueueAcc queue(devAcc);

    Vec const threadsPerBlockCreateQuints(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(1024));
    Vec const blocksPerGridCreateQuints(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(1));

    WorkDiv const createEligibleModulesListForQuintupletsGPU_workDiv(blocksPerGridCreateQuints, threadsPerBlockCreateQuints, elementsPerThread);

    SDL::createEligibleModulesListForQuintupletsGPU createEligibleModulesListForQuintupletsGPU_kernel;
    auto const createEligibleModulesListForQuintupletsGPUTask(alpaka::createTaskKernel<Acc>(
        createEligibleModulesListForQuintupletsGPU_workDiv,
        createEligibleModulesListForQuintupletsGPU_kernel,
        *modulesInGPU,
        *tripletsInGPU,
        *rangesInGPU,
        device_nTotalQuintuplets));

    alpaka::enqueue(queue, createEligibleModulesListForQuintupletsGPUTask);
    alpaka::wait(queue);

    hipMemcpyAsync(&nEligibleT5Modules,rangesInGPU->nEligibleT5Modules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(&nTotalQuintuplets,device_nTotalQuintuplets,sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    if(quintupletsInGPU == nullptr)
    {
        quintupletsInGPU = (SDL::quintuplets*)cms::cuda::allocate_host(sizeof(SDL::quintuplets), stream);
        createQuintupletsInExplicitMemory(*quintupletsInGPU, nTotalQuintuplets, nLowerModules, nEligibleT5Modules,stream);
        hipMemcpyAsync(quintupletsInGPU->nMemoryLocations, &nTotalQuintuplets, sizeof(unsigned int), hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream);
    }

    Vec const threadsPerBlockQuints(static_cast<Idx>(1), static_cast<Idx>(8), static_cast<Idx>(32));
    Vec const blocksPerGridQuints(static_cast<Idx>(max(nEligibleT5Modules,1)), static_cast<Idx>(1), static_cast<Idx>(1));

    WorkDiv const createQuintupletsInGPUv2_workDiv(blocksPerGridQuints, threadsPerBlockQuints, elementsPerThread);

    SDL::createQuintupletsInGPUv2 createQuintupletsInGPUv2_kernel;
    auto const createQuintupletsInGPUv2Task(alpaka::createTaskKernel<Acc>(
        createQuintupletsInGPUv2_workDiv,
        createQuintupletsInGPUv2_kernel,
        *modulesInGPU,
        *mdsInGPU,
        *segmentsInGPU,
        *tripletsInGPU,
        *quintupletsInGPU,
        *rangesInGPU,
        nEligibleT5Modules));

    alpaka::enqueue(queue, createQuintupletsInGPUv2Task);
    alpaka::wait(queue);

    Vec const threadsPerBlockDupQuint(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(16));
    Vec const blocksPerGridDupQuint(static_cast<Idx>(MAX_BLOCKS), static_cast<Idx>(1), static_cast<Idx>(1));

    WorkDiv const removeDupQuintupletsInGPUAfterBuild_workDiv(blocksPerGridDupQuint, threadsPerBlockDupQuint, elementsPerThread);

    SDL::removeDupQuintupletsInGPUAfterBuild removeDupQuintupletsInGPUAfterBuild_kernel;
    auto const removeDupQuintupletsInGPUAfterBuildTask(alpaka::createTaskKernel<Acc>(
        removeDupQuintupletsInGPUAfterBuild_workDiv,
        removeDupQuintupletsInGPUAfterBuild_kernel,
        *modulesInGPU,
        *quintupletsInGPU,
        *rangesInGPU));

    alpaka::enqueue(queue, removeDupQuintupletsInGPUAfterBuildTask);
    alpaka::wait(queue);

    addQuintupletRangesToEventExplicit<<<1,1024,0,stream>>>(*modulesInGPU, *quintupletsInGPU,*rangesInGPU);
    hipStreamSynchronize(stream);

    if(addObjects){
      addQuintupletsToEventExplicit();
    }

}

void SDL::Event::pixelLineSegmentCleaning()
{
    // Temporary fix for queue initialization.
    QueueAcc queue(devAcc);

    Vec const threadsPerBlockCheckHitspLS(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(16));
    Vec const blocksPerGridCheckHitspLS(static_cast<Idx>(1), static_cast<Idx>(MAX_BLOCKS*4), static_cast<Idx>(MAX_BLOCKS/4));

    WorkDiv const checkHitspLS_workDiv(blocksPerGridCheckHitspLS, threadsPerBlockCheckHitspLS, elementsPerThread);

    SDL::checkHitspLS checkHitspLS_kernel;
    auto const checkHitspLSTask(alpaka::createTaskKernel<Acc>(
        checkHitspLS_workDiv,
        checkHitspLS_kernel,
        *modulesInGPU,
        *segmentsInGPU,
        false));

    alpaka::enqueue(queue, checkHitspLSTask);
    alpaka::wait(queue);
}

void SDL::Event::createPixelQuintuplets()
{
    if(pixelQuintupletsInGPU == nullptr)
    {
        pixelQuintupletsInGPU = (SDL::pixelQuintuplets*)cms::cuda::allocate_host(sizeof(SDL::pixelQuintuplets), stream);
        createPixelQuintupletsInExplicitMemory(*pixelQuintupletsInGPU, N_MAX_PIXEL_QUINTUPLETS,stream);
    }
    if(trackCandidatesInGPU == nullptr)
    {
        trackCandidatesInGPU = (SDL::trackCandidates*)cms::cuda::allocate_host(sizeof(SDL::trackCandidates), stream);
        createTrackCandidatesInExplicitMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES + N_MAX_PIXEL_TRACK_CANDIDATES,stream);
    } 

    unsigned int pixelModuleIndex;
    int* superbins;
    int8_t* pixelTypes;
    int *nQuintuplets;

    unsigned int* connectedPixelSize_host;
    unsigned int* connectedPixelIndex_host;
    unsigned int* connectedPixelSize_dev;
    unsigned int* connectedPixelIndex_dev;

    nQuintuplets = (int*)cms::cuda::allocate_host(nLowerModules * sizeof(int), stream);
    hipMemcpyAsync(nQuintuplets, quintupletsInGPU->nQuintuplets, nLowerModules * sizeof(int), hipMemcpyDeviceToHost,stream);

    superbins = (int*)cms::cuda::allocate_host(N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int), stream);
    pixelTypes = (int8_t*)cms::cuda::allocate_host(N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t), stream);

    hipMemcpyAsync(superbins,segmentsInGPU->superbin,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(pixelTypes,segmentsInGPU->pixelType,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int8_t),hipMemcpyDeviceToHost,stream);

    hipStreamSynchronize(stream);
    pixelModuleIndex = nLowerModules;
    unsigned int nInnerSegments = 0;
    hipMemcpyAsync(&nInnerSegments, &(segmentsInGPU->nSegments[pixelModuleIndex]), sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
    connectedPixelSize_host = (unsigned int*)cms::cuda::allocate_host(nInnerSegments* sizeof(unsigned int), stream);
    connectedPixelIndex_host = (unsigned int*)cms::cuda::allocate_host(nInnerSegments* sizeof(unsigned int), stream);
    connectedPixelSize_dev = (unsigned int*)cms::cuda::allocate_device(dev,nInnerSegments* sizeof(unsigned int),stream);
    connectedPixelIndex_dev = (unsigned int*)cms::cuda::allocate_device(dev,nInnerSegments* sizeof(unsigned int),stream);
    hipStreamSynchronize(stream);

    int pixelIndexOffsetPos = pixelMapping->connectedPixelsIndex[44999] + pixelMapping->connectedPixelsSizes[44999];
    int pixelIndexOffsetNeg = pixelMapping->connectedPixelsIndexPos[44999] + pixelMapping->connectedPixelsSizes[44999] + pixelIndexOffsetPos;

    for (unsigned int i = 0; i < nInnerSegments; i++)
    {// loop over # pLS
        int8_t pixelType = pixelTypes[i];// get pixel type for this pLS
        int superbin = superbins[i]; //get superbin for this pixel
        if((superbin < 0) or (superbin >= 45000) or (pixelType > 2) or (pixelType < 0))
        {
            connectedPixelIndex_host[i] = 0;
            connectedPixelSize_host[i] = 0;
            continue;
        }

        if(pixelType ==0)
        { // used pixel type to select correct size-index arrays
            connectedPixelSize_host[i]  = pixelMapping->connectedPixelsSizes[superbin]; //number of connected modules to this pixel
            unsigned int connectedIdxBase = pixelMapping->connectedPixelsIndex[superbin];
            connectedPixelIndex_host[i] = connectedIdxBase;
        }
        else if(pixelType ==1)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesPos[superbin]; //number of pixel connected modules
            unsigned int connectedIdxBase = pixelMapping->connectedPixelsIndexPos[superbin]+pixelIndexOffsetPos;
            connectedPixelIndex_host[i] = connectedIdxBase;
        }
        else if(pixelType ==2)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesNeg[superbin]; //number of pixel connected modules
            unsigned int connectedIdxBase = pixelMapping->connectedPixelsIndexNeg[superbin] + pixelIndexOffsetNeg;
            connectedPixelIndex_host[i] = connectedIdxBase;
        }
    }

    hipMemcpyAsync(connectedPixelSize_dev, connectedPixelSize_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(connectedPixelIndex_dev, connectedPixelIndex_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice,stream);
    hipStreamSynchronize(stream);

    // Temporary fix for queue initialization.
    QueueAcc queue(devAcc);

    Vec const threadsPerBlockCreatePixQuints(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(16));
    Vec const blocksPerGridCreatePixQuints(static_cast<Idx>(16), static_cast<Idx>(MAX_BLOCKS), static_cast<Idx>(1));

    WorkDiv const createPixelQuintupletsInGPUFromMapv2_workDiv(blocksPerGridCreatePixQuints, threadsPerBlockCreatePixQuints, elementsPerThread);

    SDL::createPixelQuintupletsInGPUFromMapv2 createPixelQuintupletsInGPUFromMapv2_kernel;
    auto const createPixelQuintupletsInGPUFromMapv2Task(alpaka::createTaskKernel<Acc>(
        createPixelQuintupletsInGPUFromMapv2_workDiv,
        createPixelQuintupletsInGPUFromMapv2_kernel,
        *modulesInGPU,
        *mdsInGPU,
        *segmentsInGPU,
        *tripletsInGPU,
        *quintupletsInGPU,
        *pixelQuintupletsInGPU,
        connectedPixelSize_dev,
        connectedPixelIndex_dev,
        nInnerSegments,
        *rangesInGPU));

    alpaka::enqueue(queue, createPixelQuintupletsInGPUFromMapv2Task);
    alpaka::wait(queue);

    cms::cuda::free_host(superbins);
    cms::cuda::free_host(pixelTypes);
    cms::cuda::free_host(nQuintuplets);
    cms::cuda::free_host(connectedPixelSize_host);
    cms::cuda::free_host(connectedPixelIndex_host);
    cms::cuda::free_device(dev, connectedPixelSize_dev);
    cms::cuda::free_device(dev, connectedPixelIndex_dev);

    Vec const threadsPerBlockDupPix(static_cast<Idx>(1), static_cast<Idx>(16), static_cast<Idx>(16));
    Vec const blocksPerGridDupPix(static_cast<Idx>(1), static_cast<Idx>(MAX_BLOCKS), static_cast<Idx>(1));

    WorkDiv const removeDupPixelQuintupletsInGPUFromMap_workDiv(blocksPerGridDupPix, threadsPerBlockDupPix, elementsPerThread);

    SDL::removeDupPixelQuintupletsInGPUFromMap removeDupPixelQuintupletsInGPUFromMap_kernel;
    auto const removeDupPixelQuintupletsInGPUFromMapTask(alpaka::createTaskKernel<Acc>(
        removeDupPixelQuintupletsInGPUFromMap_workDiv,
        removeDupPixelQuintupletsInGPUFromMap_kernel,
        *pixelQuintupletsInGPU,
        false));

    alpaka::enqueue(queue, removeDupPixelQuintupletsInGPUFromMapTask);
    alpaka::wait(queue);

    Vec const threadsPerBlockAddpT5asTrackCan(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(256));
    Vec const blocksPerGridAddpT5asTrackCan(static_cast<Idx>(1), static_cast<Idx>(1), static_cast<Idx>(1));

    WorkDiv const addpT5asTrackCandidateInGPU_workDiv(blocksPerGridAddpT5asTrackCan, threadsPerBlockAddpT5asTrackCan, elementsPerThread);

    SDL::addpT5asTrackCandidateInGPU addpT5asTrackCandidateInGPU_kernel;
    auto const addpT5asTrackCandidateInGPUTask(alpaka::createTaskKernel<Acc>(
        addpT5asTrackCandidateInGPU_workDiv,
        addpT5asTrackCandidateInGPU_kernel,
        *pixelQuintupletsInGPU,
        *trackCandidatesInGPU,
        *quintupletsInGPU));

    alpaka::enqueue(queue, addpT5asTrackCandidateInGPUTask);
    alpaka::wait(queue);
#ifdef Warnings
    int nPixelQuintuplets;
    hipMemcpyAsync(&nPixelQuintuplets, &(pixelQuintupletsInGPU->nPixelQuintuplets), sizeof(int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    std::cout<<"number of pixel quintuplets = "<<nPixelQuintuplets<<std::endl;
#endif   
}


void SDL::Event::addQuintupletsToEventExplicit()
{
    unsigned int* nQuintupletsCPU;
    nQuintupletsCPU = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(nQuintupletsCPU,quintupletsInGPU->nQuintuplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);

    short* module_subdets;
    module_subdets = (short*)cms::cuda::allocate_host(nModules* sizeof(short), stream);
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    module_layers = (short*)cms::cuda::allocate_host(nLowerModules * sizeof(short), stream);
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    int* module_quintupletModuleIndices;
    module_quintupletModuleIndices = (int*)cms::cuda::allocate_host(nLowerModules * sizeof(int), stream);
    hipMemcpyAsync(module_quintupletModuleIndices, rangesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    for(uint16_t i = 0; i<nLowerModules; i++)
    {
        if(!(nQuintupletsCPU[i] == 0 or module_quintupletModuleIndices[i] == -1))
        {
            if(module_subdets[i] == Barrel)
            {
                n_quintuplets_by_layer_barrel_[module_layers[i] - 1] += nQuintupletsCPU[i];
            }
            else
            {
                n_quintuplets_by_layer_endcap_[module_layers[i] - 1] += nQuintupletsCPU[i];
            }
        }
    }
    cms::cuda::free_host(nQuintupletsCPU);
    cms::cuda::free_host(module_layers);
    cms::cuda::free_host(module_subdets);
    cms::cuda::free_host(module_quintupletModuleIndices);
}

void SDL::Event::addTripletsToEventExplicit()
{
    unsigned int* nTripletsCPU;
    nTripletsCPU = (unsigned int*)cms::cuda::allocate_host(nLowerModules * sizeof(unsigned int), stream);
    hipMemcpyAsync(nTripletsCPU,tripletsInGPU->nTriplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);

    short* module_subdets;
    module_subdets = (short*)cms::cuda::allocate_host(nLowerModules* sizeof(short), stream);
    hipMemcpyAsync(module_subdets,modulesInGPU->subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    module_layers = (short*)cms::cuda::allocate_host(nLowerModules * sizeof(short), stream);
    hipMemcpyAsync(module_layers,modulesInGPU->layers,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);

    hipStreamSynchronize(stream);
    for(uint16_t i = 0; i<nLowerModules; i++)
    {
        if(nTripletsCPU[i] != 0)
        {
            if(module_subdets[i] == Barrel)
            {
                n_triplets_by_layer_barrel_[module_layers[i] - 1] += nTripletsCPU[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[module_layers[i] - 1] += nTripletsCPU[i];
            }
        }
    }

    cms::cuda::free_host(nTripletsCPU);
    cms::cuda::free_host(module_layers);
    cms::cuda::free_host(module_subdets);
}

unsigned int SDL::Event::getNumberOfHits()
{
    unsigned int hits = 0;
    for(auto &it:n_hits_by_layer_barrel_)
    {
        hits += it;
    }
    for(auto& it:n_hits_by_layer_endcap_)
    {
        hits += it;
    }

    return hits;
}

unsigned int SDL::Event::getNumberOfHitsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_hits_by_layer_barrel_[layer];
    else
        return n_hits_by_layer_barrel_[layer] + n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerBarrel(unsigned int layer)
{
    return n_hits_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerEndcap(unsigned int layer)
{
    return n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoublets()
{
     unsigned int miniDoublets = 0;
    for(auto &it:n_minidoublets_by_layer_barrel_)
    {
        miniDoublets += it;
    }
    for(auto &it:n_minidoublets_by_layer_endcap_)
    {
        miniDoublets += it;
    }

    return miniDoublets;
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_minidoublets_by_layer_barrel_[layer];
    else
        return n_minidoublets_by_layer_barrel_[layer] + n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerBarrel(unsigned int layer)
{
    return n_minidoublets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerEndcap(unsigned int layer)
{
    return n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegments()
{
    unsigned int segments = 0;
    for(auto &it:n_segments_by_layer_barrel_)
    {
        segments += it;
    }
    for(auto &it:n_segments_by_layer_endcap_)
    {
        segments += it;
    }

    return segments;
}

unsigned int SDL::Event::getNumberOfSegmentsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_segments_by_layer_barrel_[layer];
    else
        return n_segments_by_layer_barrel_[layer] + n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerBarrel(unsigned int layer)
{
    return n_segments_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerEndcap(unsigned int layer)
{
    return n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTriplets()
{
    unsigned int triplets = 0;
    for(auto &it:n_triplets_by_layer_barrel_)
    {
        triplets += it;
    }
    for(auto &it:n_triplets_by_layer_endcap_)
    {
        triplets += it;
    }

    return triplets;
}

unsigned int SDL::Event::getNumberOfTripletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_triplets_by_layer_barrel_[layer];
    else
        return n_triplets_by_layer_barrel_[layer] + n_triplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerBarrel(unsigned int layer)
{
    return n_triplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerEndcap(unsigned int layer)
{
    return n_triplets_by_layer_endcap_[layer];
}

int SDL::Event::getNumberOfPixelTriplets()
{
    int nPixelTriplets;
    hipMemcpyAsync(&nPixelTriplets, pixelTripletsInGPU->nPixelTriplets, sizeof(int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    return nPixelTriplets;
}

int SDL::Event::getNumberOfPixelQuintuplets()
{
    int nPixelQuintuplets;
    hipMemcpyAsync(&nPixelQuintuplets, pixelQuintupletsInGPU->nPixelQuintuplets, sizeof(int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);
    return nPixelQuintuplets;
}
unsigned int SDL::Event::getNumberOfQuintuplets()
{
    unsigned int quintuplets = 0;
    for(auto &it:n_quintuplets_by_layer_barrel_)
    {
        quintuplets += it;
    }
    for(auto &it:n_quintuplets_by_layer_endcap_)
    {
        quintuplets += it;
    }

    return quintuplets;
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_quintuplets_by_layer_barrel_[layer];
    else
        return n_quintuplets_by_layer_barrel_[layer] + n_quintuplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayerBarrel(unsigned int layer)
{
    return n_quintuplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayerEndcap(unsigned int layer)
{
    return n_quintuplets_by_layer_endcap_[layer];
}

int SDL::Event::getNumberOfTrackCandidates()
{    
    int nTrackCandidates;
    hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    return nTrackCandidates;
}

int SDL::Event::getNumberOfPT5TrackCandidates()
{
    int nTrackCandidatesPT5;
    hipMemcpyAsync(&nTrackCandidatesPT5, trackCandidatesInGPU->nTrackCandidatespT5, sizeof(int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    return nTrackCandidatesPT5;
}

int SDL::Event::getNumberOfPT3TrackCandidates()
{
    int nTrackCandidatesPT3;
    hipMemcpyAsync(&nTrackCandidatesPT3, trackCandidatesInGPU->nTrackCandidatespT3, sizeof(int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    return nTrackCandidatesPT3;
}

int SDL::Event::getNumberOfPLSTrackCandidates()
{
    unsigned int nTrackCandidatesPLS;
    hipMemcpyAsync(&nTrackCandidatesPLS, trackCandidatesInGPU->nTrackCandidatespLS, sizeof(int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    return nTrackCandidatesPLS;
}

int SDL::Event::getNumberOfPixelTrackCandidates()
{
    int nTrackCandidates;
    int nTrackCandidatesT5;
    hipMemcpyAsync(&nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(int), hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(&nTrackCandidatesT5, trackCandidatesInGPU->nTrackCandidatesT5, sizeof(int), hipMemcpyDeviceToHost,stream);
    hipStreamSynchronize(stream);

    return nTrackCandidates - nTrackCandidatesT5;
}

int SDL::Event::getNumberOfT5TrackCandidates()
{
    int nTrackCandidatesT5;
    hipMemcpyAsync(&nTrackCandidatesT5, trackCandidatesInGPU->nTrackCandidatesT5, sizeof(int), hipMemcpyDeviceToHost,stream);
    return nTrackCandidatesT5; 
}

SDL::hits* SDL::Event::getHits() //std::shared_ptr should take care of garbage collection
{
    if(hitsInCPU == nullptr)
    {
        hitsInCPU = new SDL::hits;
        hitsInCPU->nHits = new unsigned int;
        unsigned int nHits;
        hipMemcpyAsync(&nHits, hitsInGPU->nHits, sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        *(hitsInCPU->nHits) = nHits;
        hitsInCPU->idxs = new unsigned int[nHits];
        hitsInCPU->detid = new unsigned int[nHits];
        hitsInCPU->xs = new float[nHits];
        hitsInCPU->ys = new float[nHits];
        hitsInCPU->zs = new float[nHits];
        hitsInCPU->moduleIndices = new uint16_t[nHits];
        hipMemcpyAsync(hitsInCPU->idxs, hitsInGPU->idxs,sizeof(unsigned int) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->detid, hitsInGPU->detid, sizeof(unsigned int) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->xs, hitsInGPU->xs, sizeof(float) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->ys, hitsInGPU->ys, sizeof(float) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->zs, hitsInGPU->zs, sizeof(float) * nHits, hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(hitsInCPU->moduleIndices, hitsInGPU->moduleIndices, sizeof(uint16_t) * nHits, hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return hitsInCPU;
}

SDL::objectRanges* SDL::Event::getRanges()
{
    if(rangesInCPU == nullptr)
    {
        rangesInCPU = new SDL::objectRanges;
        rangesInCPU->hitRanges = new int[2*nModules];
        rangesInCPU->quintupletModuleIndices = new int[nLowerModules];
        hipMemcpyAsync(rangesInCPU->hitRanges, hitsInGPU->hitRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost,stream);
        rangesInCPU->miniDoubletModuleIndices = new int[nLowerModules+1];
        rangesInCPU->segmentModuleIndices = new int[nLowerModules + 1];
        rangesInCPU->tripletModuleIndices = new int[nLowerModules];
        hipMemcpyAsync(rangesInCPU->quintupletModuleIndices, rangesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(rangesInCPU->miniDoubletModuleIndices, rangesInGPU->miniDoubletModuleIndices, (nLowerModules + 1) * sizeof(int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(rangesInCPU->segmentModuleIndices, rangesInGPU->segmentModuleIndices, (nLowerModules + 1) * sizeof(int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(rangesInCPU->tripletModuleIndices, rangesInGPU->tripletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
    }
    return rangesInCPU;
}

SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    if(mdsInCPU == nullptr)
    {
        mdsInCPU = new SDL::miniDoublets;
        mdsInCPU->nMDs = new int[nLowerModules+1];

        //compute memory locations
        mdsInCPU->nMemoryLocations = new unsigned int;
        hipMemcpyAsync(mdsInCPU->nMemoryLocations, mdsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
        mdsInCPU->totOccupancyMDs = new int[nLowerModules+1];

        mdsInCPU->anchorHitIndices = new unsigned int[*(mdsInCPU->nMemoryLocations)];
        mdsInCPU->outerHitIndices = new unsigned int[*(mdsInCPU->nMemoryLocations)];
        mdsInCPU->dphichanges = new float[*(mdsInCPU->nMemoryLocations)];
        hipMemcpyAsync(mdsInCPU->anchorHitIndices, mdsInGPU->anchorHitIndices, *(mdsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(mdsInCPU->outerHitIndices, mdsInGPU->outerHitIndices, *(mdsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(mdsInCPU->dphichanges, mdsInGPU->dphichanges, *(mdsInCPU->nMemoryLocations) * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(mdsInCPU->nMDs, mdsInGPU->nMDs, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(mdsInCPU->totOccupancyMDs, mdsInGPU->totOccupancyMDs, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return mdsInCPU;
}

SDL::segments* SDL::Event::getSegments()
{
    if(segmentsInCPU == nullptr)
    {
        segmentsInCPU = new SDL::segments;
        
        segmentsInCPU->nSegments = new int[nLowerModules+1];
        hipMemcpyAsync(segmentsInCPU->nSegments, segmentsInGPU->nSegments, (nLowerModules+1) * sizeof(int), hipMemcpyDeviceToHost,stream);
        
        segmentsInCPU->nMemoryLocations = new unsigned int;
        hipMemcpyAsync(segmentsInCPU->nMemoryLocations, segmentsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        segmentsInCPU->mdIndices = new unsigned int[2 * *(segmentsInCPU->nMemoryLocations)];
        segmentsInCPU->innerMiniDoubletAnchorHitIndices = new unsigned int[*(segmentsInCPU->nMemoryLocations)];
        segmentsInCPU->outerMiniDoubletAnchorHitIndices = new unsigned int[*(segmentsInCPU->nMemoryLocations)];
        segmentsInCPU->totOccupancySegments = new int[nLowerModules+1];

        segmentsInCPU->ptIn = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->eta = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->phi = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->seedIdx = new unsigned int[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->isDup = new bool[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->isQuad = new char[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->score = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];

        hipMemcpyAsync(segmentsInCPU->mdIndices, segmentsInGPU->mdIndices, 2 * *(segmentsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->innerMiniDoubletAnchorHitIndices, segmentsInGPU->innerMiniDoubletAnchorHitIndices, *(segmentsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->outerMiniDoubletAnchorHitIndices, segmentsInGPU->outerMiniDoubletAnchorHitIndices, *(segmentsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->totOccupancySegments, segmentsInGPU->totOccupancySegments, (nLowerModules+1) * sizeof(int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->ptIn, segmentsInGPU->ptIn, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->eta, segmentsInGPU->eta, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->phi, segmentsInGPU->phi, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->seedIdx, segmentsInGPU->seedIdx, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->isDup, segmentsInGPU->isDup, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->isQuad, segmentsInGPU->isQuad, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(char), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(segmentsInCPU->score, segmentsInGPU->score, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return segmentsInCPU;
}

SDL::triplets* SDL::Event::getTriplets()
{
    if(tripletsInCPU == nullptr)
    {
        tripletsInCPU = new SDL::triplets;
        tripletsInCPU->nMemoryLocations = new unsigned int;
        hipMemcpyAsync(tripletsInCPU->nMemoryLocations, tripletsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        tripletsInCPU->segmentIndices = new unsigned[2 * *(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->nTriplets = new int[nLowerModules];
        tripletsInCPU->betaIn  = new FPX[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->betaOut = new FPX[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->pt_beta = new FPX[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->hitIndices = new unsigned int[6 * *(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->logicalLayers = new uint8_t[3 * *(tripletsInCPU->nMemoryLocations)];
#ifdef CUT_VALUE_DEBUG
        tripletsInCPU->zOut = new float[4 * *(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->zLo = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->zHi = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->zLoPointed = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->zHiPointed = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->sdlCut = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->betaInCut = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->betaOutCut = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->deltaBetaCut = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->rtLo = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->rtHi = new float[*(tripletsInCPU->nMemoryLocations)];
        tripletsInCPU->kZ = new float[*(tripletsInCPU->nMemoryLocations)];

        tripletsInCPU->rtOut = tripletsInCPU->zOut + *(tripletsInCPU->nMemoryLocations);
        tripletsInCPU->deltaPhiPos = tripletsInCPU->zOut + 2 * *(tripletsInCPU->nMemoryLocations);
        tripletsInCPU->deltaPhi = tripletsInCPU->zOut + 3 * *(tripletsInCPU->nMemoryLocations);

        hipMemcpyAsync(tripletsInCPU->zOut, tripletsInGPU->zOut, 4 * * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->zLo, tripletsInGPU->zLo, * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->zHi, tripletsInGPU->zHi, * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->zLoPointed, tripletsInGPU->zLoPointed, 4 * * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->zHiPointed, tripletsInGPU->zHiPointed, * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->sdlCut, tripletsInGPU->sdlCut, *(tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->betaInCut, tripletsInGPU->betaInCut,  * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->betaOutCut, tripletsInGPU->betaOutCut,  * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->deltaBetaCut, tripletsInGPU->deltaBetaCut, *(tripletsInCPU->nMemoryLocations)*sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpyAsync(tripletsInCPU->rtLo, tripletsInGPU->rtLo,  * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->rtHi, tripletsInGPU->rtHi,  * (tripletsInCPU->nMemoryLocations)* sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->kZ, tripletsInGPU->kZ,  * (tripletsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
#endif
        hipMemcpyAsync(tripletsInCPU->hitIndices, tripletsInGPU->hitIndices, 6 * *(tripletsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->logicalLayers, tripletsInGPU->logicalLayers, 3 * *(tripletsInCPU->nMemoryLocations) * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(tripletsInCPU->segmentIndices, tripletsInGPU->segmentIndices, 2 * *(tripletsInCPU->nMemoryLocations) * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->betaIn, tripletsInGPU->betaIn,   *(tripletsInCPU->nMemoryLocations) * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->betaOut, tripletsInGPU->betaOut, *(tripletsInCPU->nMemoryLocations) * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->pt_beta, tripletsInGPU->pt_beta, *(tripletsInCPU->nMemoryLocations) * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        tripletsInCPU->totOccupancyTriplets = new int[nLowerModules];
        hipMemcpyAsync(tripletsInCPU->nTriplets, tripletsInGPU->nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(tripletsInCPU->totOccupancyTriplets, tripletsInGPU->totOccupancyTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return tripletsInCPU;
}

SDL::quintuplets* SDL::Event::getQuintuplets()
{
    if(quintupletsInCPU == nullptr)
    {
        quintupletsInCPU = new SDL::quintuplets;
        uint16_t nEligibleT5Modules;
        hipMemcpyAsync(&nEligibleT5Modules, rangesInGPU->nEligibleT5Modules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        unsigned int nMemoryLocations;
        hipMemcpyAsync(&nMemoryLocations, quintupletsInGPU->nMemoryLocations, sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        quintupletsInCPU->nQuintuplets = new int[nLowerModules];
        quintupletsInCPU->totOccupancyQuintuplets = new int[nLowerModules];
        quintupletsInCPU->tripletIndices = new unsigned int[2 * nMemoryLocations];
        quintupletsInCPU->lowerModuleIndices = new uint16_t[5 * nMemoryLocations];
        quintupletsInCPU->innerRadius = new FPX[nMemoryLocations];
        quintupletsInCPU->outerRadius = new FPX[nMemoryLocations];
        quintupletsInCPU->bridgeRadius = new FPX[nMemoryLocations];

        quintupletsInCPU->isDup = new bool[nMemoryLocations];
        quintupletsInCPU->score_rphisum = new FPX[nMemoryLocations];
        quintupletsInCPU->eta = new FPX[nMemoryLocations];
        quintupletsInCPU->phi = new FPX[nMemoryLocations];

        quintupletsInCPU->rzChiSquared = new float[nMemoryLocations];
        quintupletsInCPU->chiSquared = new float[nMemoryLocations];
        quintupletsInCPU->nonAnchorChiSquared = new float[nMemoryLocations];

        hipMemcpyAsync(quintupletsInCPU->nQuintuplets, quintupletsInGPU->nQuintuplets,  nLowerModules * sizeof(int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->totOccupancyQuintuplets, quintupletsInGPU->totOccupancyQuintuplets,  nLowerModules * sizeof(int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->tripletIndices, quintupletsInGPU->tripletIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->lowerModuleIndices, quintupletsInGPU->lowerModuleIndices, 5 * nMemoryLocations * sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->innerRadius, quintupletsInGPU->innerRadius, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->bridgeRadius, quintupletsInGPU->bridgeRadius, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->outerRadius, quintupletsInGPU->outerRadius, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->isDup, quintupletsInGPU->isDup, nMemoryLocations * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->score_rphisum, quintupletsInGPU->score_rphisum, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->eta, quintupletsInGPU->eta, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->phi, quintupletsInGPU->phi, nMemoryLocations * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(quintupletsInCPU->chiSquared, quintupletsInGPU->chiSquared, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->rzChiSquared, quintupletsInGPU->rzChiSquared, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(quintupletsInCPU->nonAnchorChiSquared, quintupletsInGPU->nonAnchorChiSquared, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost, stream);

        hipStreamSynchronize(stream);
    }
    return quintupletsInCPU;
}

SDL::pixelTriplets* SDL::Event::getPixelTriplets()
{
    if(pixelTripletsInCPU == nullptr)
    {
        pixelTripletsInCPU = new SDL::pixelTriplets;

        pixelTripletsInCPU->nPixelTriplets = new int;
        pixelTripletsInCPU->totOccupancyPixelTriplets = new int;
        hipMemcpyAsync(pixelTripletsInCPU->nPixelTriplets, pixelTripletsInGPU->nPixelTriplets, sizeof(int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->totOccupancyPixelTriplets, pixelTripletsInGPU->totOccupancyPixelTriplets, sizeof(int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        unsigned int nPixelTriplets = *(pixelTripletsInCPU->nPixelTriplets);
        pixelTripletsInCPU->tripletIndices = new unsigned int[nPixelTriplets];
        pixelTripletsInCPU->pixelSegmentIndices = new unsigned int[nPixelTriplets];
        pixelTripletsInCPU->pixelRadius = new FPX[nPixelTriplets];
        pixelTripletsInCPU->tripletRadius = new FPX[nPixelTriplets];
        pixelTripletsInCPU->isDup = new bool[nPixelTriplets];
        pixelTripletsInCPU->eta = new  FPX[nPixelTriplets];
        pixelTripletsInCPU->phi = new  FPX[nPixelTriplets];
        pixelTripletsInCPU->score =new FPX[nPixelTriplets];
        pixelTripletsInCPU->rzChiSquared = new float[nPixelTriplets];
        pixelTripletsInCPU->rPhiChiSquared = new float[nPixelTriplets];
        pixelTripletsInCPU->rPhiChiSquaredInwards = new float[nPixelTriplets];

        hipMemcpyAsync(pixelTripletsInCPU->rzChiSquared, pixelTripletsInGPU->rzChiSquared, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(pixelTripletsInCPU->rPhiChiSquared, pixelTripletsInGPU->rPhiChiSquared, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(pixelTripletsInCPU->rPhiChiSquaredInwards, pixelTripletsInGPU->rPhiChiSquaredInwards, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost, stream);

        hipMemcpyAsync(pixelTripletsInCPU->tripletIndices, pixelTripletsInGPU->tripletIndices, nPixelTriplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->pixelSegmentIndices, pixelTripletsInGPU->pixelSegmentIndices, nPixelTriplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->pixelRadius, pixelTripletsInGPU->pixelRadius, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->tripletRadius, pixelTripletsInGPU->tripletRadius, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->isDup, pixelTripletsInGPU->isDup, nPixelTriplets * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->eta, pixelTripletsInGPU->eta, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->phi, pixelTripletsInGPU->phi, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelTripletsInCPU->score, pixelTripletsInGPU->score, nPixelTriplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return pixelTripletsInCPU;
}

SDL::pixelQuintuplets* SDL::Event::getPixelQuintuplets()
{
    if(pixelQuintupletsInCPU == nullptr)
    {
        pixelQuintupletsInCPU = new SDL::pixelQuintuplets;

        pixelQuintupletsInCPU->nPixelQuintuplets = new int;
        pixelQuintupletsInCPU->totOccupancyPixelQuintuplets = new int;
        hipMemcpyAsync(pixelQuintupletsInCPU->nPixelQuintuplets, pixelQuintupletsInGPU->nPixelQuintuplets, sizeof(int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->totOccupancyPixelQuintuplets, pixelQuintupletsInGPU->totOccupancyPixelQuintuplets, sizeof(int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        int nPixelQuintuplets = *(pixelQuintupletsInCPU->nPixelQuintuplets);

        pixelQuintupletsInCPU->pixelIndices = new unsigned int[nPixelQuintuplets];
        pixelQuintupletsInCPU->T5Indices = new unsigned int[nPixelQuintuplets];
        pixelQuintupletsInCPU->isDup = new bool[nPixelQuintuplets];
        pixelQuintupletsInCPU->score = new FPX[nPixelQuintuplets];
        pixelQuintupletsInCPU->rzChiSquared = new float[nPixelQuintuplets];
        pixelQuintupletsInCPU->rPhiChiSquared = new float[nPixelQuintuplets];
        pixelQuintupletsInCPU->rPhiChiSquaredInwards = new float[nPixelQuintuplets];

        hipMemcpyAsync(pixelQuintupletsInCPU->rzChiSquared, pixelQuintupletsInGPU->rzChiSquared, nPixelQuintuplets * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->rPhiChiSquared, pixelQuintupletsInGPU->rPhiChiSquared, nPixelQuintuplets * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->rPhiChiSquaredInwards, pixelQuintupletsInGPU->rPhiChiSquaredInwards, nPixelQuintuplets * sizeof(float), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->pixelIndices, pixelQuintupletsInGPU->pixelIndices, nPixelQuintuplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->T5Indices, pixelQuintupletsInGPU->T5Indices, nPixelQuintuplets * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->isDup, pixelQuintupletsInGPU->isDup, nPixelQuintuplets * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(pixelQuintupletsInCPU->score, pixelQuintupletsInGPU->score, nPixelQuintuplets * sizeof(FPX), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return pixelQuintupletsInCPU;
}

SDL::trackCandidates* SDL::Event::getTrackCandidates()
{
    if(trackCandidatesInCPU == nullptr)
    {
        trackCandidatesInCPU = new SDL::trackCandidates;
        trackCandidatesInCPU->nTrackCandidates = new int;
        hipMemcpyAsync(trackCandidatesInCPU->nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, sizeof(int), hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
        int nTrackCandidates = *(trackCandidatesInCPU->nTrackCandidates);

        trackCandidatesInCPU->directObjectIndices = new unsigned int[nTrackCandidates];
        trackCandidatesInCPU->objectIndices = new unsigned int[2 * nTrackCandidates];
        trackCandidatesInCPU->trackCandidateType = new short[nTrackCandidates];
        trackCandidatesInCPU->hitIndices = new unsigned int[14 * nTrackCandidates];
        trackCandidatesInCPU->logicalLayers = new uint8_t[7 * nTrackCandidates];

        hipMemcpyAsync(trackCandidatesInCPU->hitIndices, trackCandidatesInGPU->hitIndices, 14 * nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(trackCandidatesInCPU->logicalLayers, trackCandidatesInGPU->logicalLayers, 7 * nTrackCandidates * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(trackCandidatesInCPU->directObjectIndices, trackCandidatesInGPU->directObjectIndices, nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);                                                                                    
        hipMemcpyAsync(trackCandidatesInCPU->objectIndices, trackCandidatesInGPU->objectIndices, 2 * nTrackCandidates * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);                                                                                    
        hipMemcpyAsync(trackCandidatesInCPU->trackCandidateType, trackCandidatesInGPU->trackCandidateType, nTrackCandidates * sizeof(short), hipMemcpyDeviceToHost,stream);                                                                                                                
        hipStreamSynchronize(stream);
    }
    return trackCandidatesInCPU;
}

SDL::modules* SDL::Event::getFullModules()
{
    if(modulesInCPUFull == nullptr)
    {
        modulesInCPUFull = new SDL::modules;

        modulesInCPUFull->detIds = new unsigned int[nModules];
        modulesInCPUFull->moduleMap = new uint16_t[40*nModules];
        modulesInCPUFull->nConnectedModules = new uint16_t[nModules];
        modulesInCPUFull->drdzs = new float[nModules];
        modulesInCPUFull->slopes = new float[nModules];
        modulesInCPUFull->nModules = new uint16_t[1];
        modulesInCPUFull->nLowerModules = new uint16_t[1];
        modulesInCPUFull->layers = new short[nModules];
        modulesInCPUFull->rings = new short[nModules];
        modulesInCPUFull->modules = new short[nModules];
        modulesInCPUFull->rods = new short[nModules];
        modulesInCPUFull->subdets = new short[nModules];
        modulesInCPUFull->sides = new short[nModules];
        modulesInCPUFull->isInverted = new bool[nModules];
        modulesInCPUFull->isLower = new bool[nModules];

        modulesInCPUFull->moduleType = new ModuleType[nModules];
        modulesInCPUFull->moduleLayerType = new ModuleLayerType[nModules];
        hipMemcpyAsync(modulesInCPUFull->detIds,modulesInGPU->detIds,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->moduleMap,modulesInGPU->moduleMap,40*nModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->drdzs,modulesInGPU->drdzs,sizeof(float)*nModules,hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->slopes,modulesInGPU->slopes,sizeof(float)*nModules,hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->rings,modulesInGPU->rings,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->modules,modulesInGPU->modules,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->rods,modulesInGPU->rods,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->subdets,modulesInGPU->subdets,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->sides,modulesInGPU->sides,sizeof(short)*nModules,hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->isInverted,modulesInGPU->isInverted,sizeof(bool)*nModules,hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->isLower,modulesInGPU->isLower,sizeof(bool)*nModules,hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->moduleType,modulesInGPU->moduleType,sizeof(ModuleType)*nModules,hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPUFull->moduleLayerType,modulesInGPU->moduleLayerType,sizeof(ModuleLayerType)*nModules,hipMemcpyDeviceToHost,stream);
        hipStreamSynchronize(stream);
    }
    return modulesInCPUFull;
}

SDL::modules* SDL::Event::getModules()
{
    if(modulesInCPU == nullptr)
    {
        modulesInCPU = new SDL::modules;
        modulesInCPU->nLowerModules = new uint16_t[1];
        modulesInCPU->nModules = new uint16_t[1];
        modulesInCPU->detIds = new unsigned int[nModules];
        modulesInCPU->isLower = new bool[nModules];
        modulesInCPU->layers = new short[nModules];
        modulesInCPU->subdets = new short[nModules];
        modulesInCPU->rings = new short[nModules];
        modulesInCPU->rods = new short[nModules];
        modulesInCPU->modules = new short[nModules];
        modulesInCPU->sides = new short[nModules];
        modulesInCPU->eta = new float[nModules];
        modulesInCPU->r = new float[nModules];
        modulesInCPU->moduleType = new ModuleType[nModules];

        hipMemcpyAsync(modulesInCPU->nLowerModules, modulesInGPU->nLowerModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->nModules, modulesInGPU->nModules, sizeof(uint16_t), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->detIds, modulesInGPU->detIds, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->isLower, modulesInGPU->isLower, nModules * sizeof(bool), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->layers, modulesInGPU->layers, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->subdets, modulesInGPU->subdets, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->rings, modulesInGPU->rings, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->rods, modulesInGPU->rods, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->modules, modulesInGPU->modules, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->sides, modulesInGPU->sides, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->eta, modulesInGPU->eta, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->r, modulesInGPU->r, nModules * sizeof(short), hipMemcpyDeviceToHost,stream);
        hipMemcpyAsync(modulesInCPU->moduleType, modulesInGPU->moduleType, nModules * sizeof(ModuleType), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
    }
    return modulesInCPU;
}
