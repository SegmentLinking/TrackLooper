#include "hip/hip_runtime.h"
# include "Event.cuh"
#include "allocate.h"

unsigned int N_MAX_HITS_PER_MODULE = 100;
struct SDL::modules* SDL::modulesInGPU = nullptr;
struct SDL::pixelMap* SDL::pixelMapping = nullptr;
unsigned int SDL::nModules;

SDL::Event::Event()
{
    hitsInGPU = nullptr;
    mdsInGPU = nullptr;
    segmentsInGPU = nullptr;
    trackletsInGPU = nullptr;
    pixelTrackletsInGPU = nullptr;
    tripletsInGPU = nullptr;
    quintupletsInGPU = nullptr;
    trackCandidatesInGPU = nullptr;
    pixelTripletsInGPU = nullptr;
    pixelQuintupletsInGPU = nullptr;

    hitsInCPU = nullptr;
    mdsInCPU = nullptr;
    segmentsInCPU = nullptr;
    trackletsInCPU = nullptr;
    pixelTrackletsInCPU = nullptr;
    tripletsInCPU = nullptr;
    trackCandidatesInCPU = nullptr;
    modulesInCPU = nullptr;
    modulesInCPUFull = nullptr;
    quintupletsInCPU = nullptr;
    pixelTripletsInCPU = nullptr;
    pixelQuintupletsInCPU = nullptr;

    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_tracklets_by_layer_barrel_[i] = 0;
        n_triplets_by_layer_barrel_[i] = 0;
        n_trackCandidates_by_layer_barrel_[i] = 0;
        n_quintuplets_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
            n_segments_by_layer_endcap_[i] = 0;
            n_tracklets_by_layer_endcap_[i] = 0;
            n_triplets_by_layer_endcap_[i] = 0;
            n_trackCandidates_by_layer_endcap_[i] = 0;
            n_quintuplets_by_layer_endcap_[i] = 0;
        }
    }
    resetObjectsInModule();

}

SDL::Event::~Event()
{

#ifdef CACHE_ALLOC
    mdsInGPU->freeMemoryCache();
    segmentsInGPU->freeMemoryCache();
    tripletsInGPU->freeMemoryCache();
    pixelTrackletsInGPU->freeMemoryCache();
    trackCandidatesInGPU->freeMemoryCache();
#ifdef FINAL_T5
    quintupletsInGPU->freeMemoryCache();
#endif
#ifdef FINAL_T3T4
    trackletsInGPU->freeMemoryCache();
#endif
#else
    mdsInGPU->freeMemory();
    segmentsInGPU->freeMemory();
    tripletsInGPU->freeMemory();
    pixelTrackletsInGPU->freeMemory();
    trackCandidatesInGPU->freeMemory();
#ifdef FINAL_T5
    quintupletsInGPU->freeMemory();
#endif
#ifdef FINAL_T3T4
    trackletsInGPU->freeMemory();
#endif
#endif
    hipHostFree(mdsInGPU);
    hipHostFree(segmentsInGPU);
    hipHostFree(tripletsInGPU);
    hipHostFree(pixelTrackletsInGPU);
    hipHostFree(trackCandidatesInGPU);
    hitsInGPU->freeMemory();
    hipHostFree(hitsInGPU);

    pixelTripletsInGPU->freeMemory();
    hipHostFree(pixelTripletsInGPU);

#ifdef FINAL_T5
    hipHostFree(quintupletsInGPU);
#endif
#ifdef FINAL_T3T4
    hipHostFree(trackletsInGPU);
#endif

#ifdef Explicit_Hit
    if(hitsInCPU != nullptr)
    {
        delete[] hitsInCPU->idxs;
        delete[] hitsInCPU->xs;
        delete[] hitsInCPU->ys;
        delete[] hitsInCPU->zs;
        delete[] hitsInCPU->moduleIndices;
        delete hitsInCPU->nHits;
        delete hitsInCPU;
    }
#endif
#ifdef Explicit_MD
    if(mdsInCPU != nullptr)
    {
        delete[] mdsInCPU->hitIndices;
        delete[] mdsInCPU->nMDs;
        delete mdsInCPU;
    }
#endif
#ifdef Explicit_Seg
    if(segmentsInCPU != nullptr)
    {
        delete[] segmentsInCPU->mdIndices;
        delete[] segmentsInCPU->nSegments;
        delete[] segmentsInCPU->innerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->outerMiniDoubletAnchorHitIndices;
        delete[] segmentsInCPU->ptIn;
        delete[] segmentsInCPU->eta;
        delete[] segmentsInCPU->phi;
        delete segmentsInCPU;
    }
#endif
#ifdef Explicit_Tracklet
    if(trackletsInCPU != nullptr)
    {
        delete[] trackletsInCPU->segmentIndices;
        delete[] trackletsInCPU->nTracklets;
        delete[] trackletsInCPU->betaIn;
        delete[] trackletsInCPU->betaOut;
        delete[] trackletsInCPU->pt_beta;
        delete trackletsInCPU;
    }
    if(pixelTrackletsInCPU != nullptr)
    {
        delete[] pixelTrackletsInCPU->segmentIndices;
        delete pixelTrackletsInCPU->nPixelTracklets;
        delete[] pixelTrackletsInCPU->betaIn;
        delete[] pixelTrackletsInCPU->betaOut;
        delete[] pixelTrackletsInCPU->pt_beta;
        delete pixelTrackletsInCPU;
    }
#endif
#ifdef Explicit_Trips
    if(tripletsInCPU != nullptr)
    {
        delete[] tripletsInCPU->segmentIndices;
        delete[] tripletsInCPU->nTriplets;
        delete[] tripletsInCPU->betaIn;
        delete[] tripletsInCPU->betaOut;
        delete[] tripletsInCPU->pt_beta;
        delete tripletsInCPU;
    }
#endif
#ifdef Explicit_T5
#ifdef FINAL_T5
    if(quintupletsInCPU != nullptr)
    {
        delete[] quintupletsInCPU->tripletIndices;
        delete[] quintupletsInCPU->nQuintuplets;
        delete[] quintupletsInCPU->lowerModuleIndices;
        delete[] quintupletsInCPU->innerRadius;
        delete[] quintupletsInCPU->outerRadius;
        delete[] quintupletsInCPU->regressionRadius;
        delete quintupletsInCPU;
    }
#endif
#endif

#ifdef Explicit_PT3
    if(pixelTripletsInCPU != nullptr)
    {
        delete[] pixelTripletsInCPU->tripletIndices;
        delete[] pixelTripletsInCPU->pixelSegmentIndices;
        delete[] pixelTripletsInCPU->pixelRadius;
        delete[] pixelTripletsInCPU->tripletRadius;
        delete pixelTripletsInCPU->nPixelTriplets;
        delete pixelTripletsInCPU;
    }
#endif

#ifdef Explicit_Track
    if(trackCandidatesInCPU != nullptr)
    {
        delete[] trackCandidatesInCPU->objectIndices;
        delete[] trackCandidatesInCPU->trackCandidateType;
        delete[] trackCandidatesInCPU->nTrackCandidates;
        delete trackCandidatesInCPU;
    }
#endif
#ifdef Explicit_Module
    if(modulesInCPU != nullptr)
    {
        delete[] modulesInCPU->nLowerModules;
        delete[] modulesInCPU->nModules;
        delete[] modulesInCPU->lowerModuleIndices;
        delete[] modulesInCPU->detIds;
        delete[] modulesInCPU->hitRanges;
        delete[] modulesInCPU->isLower;
        delete[] modulesInCPU->trackCandidateModuleIndices;
        delete[] modulesInCPU->quintupletModuleIndices;
        delete[] modulesInCPU->layers;
        delete[] modulesInCPU->subdets;
        delete[] modulesInCPU->rings;
        delete[] modulesInCPU;
    }
    if(modulesInCPUFull != nullptr)
    {
        delete[] modulesInCPUFull->detIds;
        delete[] modulesInCPUFull->moduleMap;
        delete[] modulesInCPUFull->nConnectedModules;
        delete[] modulesInCPUFull->drdzs;
        delete[] modulesInCPUFull->slopes;
        delete[] modulesInCPUFull->nModules;
        delete[] modulesInCPUFull->nLowerModules;
        delete[] modulesInCPUFull->layers;
        delete[] modulesInCPUFull->rings;
        delete[] modulesInCPUFull->modules;
        delete[] modulesInCPUFull->rods;
        delete[] modulesInCPUFull->subdets;
        delete[] modulesInCPUFull->sides;
        delete[] modulesInCPUFull->isInverted;
        delete[] modulesInCPUFull->isLower;

        delete[] modulesInCPUFull->hitRanges;
        delete[] modulesInCPUFull->mdRanges;
        delete[] modulesInCPUFull->segmentRanges;
        delete[] modulesInCPUFull->trackletRanges;
        delete[] modulesInCPUFull->tripletRanges;
        delete[] modulesInCPUFull->trackCandidateRanges;

        delete[] modulesInCPUFull->moduleType;
        delete[] modulesInCPUFull->moduleLayerType;

        delete[] modulesInCPUFull->lowerModuleIndices;
        delete[] modulesInCPUFull->reverseLookupLowerModuleIndices;
        delete[] modulesInCPUFull->trackCandidateModuleIndices;
        delete[] modulesInCPUFull->quintupletModuleIndices;
        delete[] modulesInCPUFull;
    }
#endif
}

void SDL::initModules(const char* moduleMetaDataFilePath)
{
    if(modulesInGPU == nullptr)
    {
        hipHostMalloc(&modulesInGPU, sizeof(struct SDL::modules));
        //pixelMapping = new pixelMap;
        hipHostMalloc(&pixelMapping, sizeof(struct SDL::pixelMap));
        loadModulesFromFile(*modulesInGPU,nModules,*pixelMapping,moduleMetaDataFilePath); //nModules gets filled here
    }
    resetObjectRanges(*modulesInGPU,nModules);
}

void SDL::cleanModules()
{
  #ifdef CACHE_ALLOC
  freeModulesCache(*modulesInGPU,*pixelMapping);
  #else
  freeModules(*modulesInGPU,*pixelMapping);
  #endif
  hipHostFree(modulesInGPU);
  hipHostFree(pixelMapping);
}

void SDL::Event::resetObjectsInModule()
{
    resetObjectRanges(*modulesInGPU,nModules);
}

// Best working hit loading method. Previously named OMP
void SDL::Event::addHitToEvent(std::vector<float> x, std::vector<float> y, std::vector<float> z, std::vector<unsigned int> detId, std::vector<unsigned int> idxInNtuple)
{
    const int loopsize = x.size();// use the actual number of hits instead of a "max"

    if(hitsInGPU == nullptr)
    {

        hipHostMalloc(&hitsInGPU, sizeof(SDL::hits));
        #ifdef Explicit_Hit
    	  createHitsInExplicitMemory(*hitsInGPU, 2*loopsize); //unclear why but this has to be 2*loopsize to avoid crashing later (reported in tracklet allocation). seems to do with nHits values as well. this allows nhits to be set to the correct value of loopsize to get correct results without crashing. still beats the "max hits" so i think this is fine.
        #else
        createHitsInUnifiedMemory(*hitsInGPU,2*loopsize,0);
        #endif
    }


    float* host_x = &x[0]; // convert from std::vector to host array easily since vectors are ordered
    float* host_y = &y[0];
    float* host_z = &z[0];
    float* host_phis;
    float* host_etas;
    unsigned int* host_detId = &detId[0];
    unsigned int* host_idxs = &idxInNtuple[0];
    unsigned int* host_moduleIndex;
    float* host_rts;
    //float* host_idxs;
    float* host_highEdgeXs;
    float* host_highEdgeYs;
    float* host_lowEdgeXs;
    float* host_lowEdgeYs;
    hipHostMalloc(&host_moduleIndex,sizeof(unsigned int)*loopsize);
    hipHostMalloc(&host_phis,sizeof(float)*loopsize);
    hipHostMalloc(&host_etas,sizeof(float)*loopsize);
    hipHostMalloc(&host_rts,sizeof(float)*loopsize);
    //hipHostMalloc(&host_idxs,sizeof(unsigned int)*loopsize);
    hipHostMalloc(&host_highEdgeXs,sizeof(float)*loopsize);
    hipHostMalloc(&host_highEdgeYs,sizeof(float)*loopsize);
    hipHostMalloc(&host_lowEdgeXs,sizeof(float)*loopsize);
    hipHostMalloc(&host_lowEdgeYs,sizeof(float)*loopsize);


    short* module_layers;
    short* module_subdet;
    int* module_hitRanges;
    ModuleType* module_moduleType;
    hipHostMalloc(&module_layers,sizeof(short)*nModules);
    hipHostMalloc(&module_subdet,sizeof(short)*nModules);
    hipHostMalloc(&module_hitRanges,sizeof(int)*2*nModules);
    hipHostMalloc(&module_moduleType,sizeof(ModuleType)*nModules);
    hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    hipMemcpy(module_subdet,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
    hipMemcpy(module_hitRanges,modulesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(module_moduleType,modulesInGPU->moduleType,nModules*sizeof(ModuleType),hipMemcpyDeviceToHost);


  for (int ihit=0; ihit<loopsize;ihit++){
    unsigned int moduleLayer = module_layers[(*detIdToIndex)[host_detId[ihit]]];
    unsigned int subdet = module_subdet[(*detIdToIndex)[host_detId[ihit]]];
    host_moduleIndex[ihit] = (*detIdToIndex)[host_detId[ihit]];


      host_rts[ihit] = sqrt(host_x[ihit]*host_x[ihit] + host_y[ihit]*host_y[ihit]);
      host_phis[ihit] = phi(host_x[ihit],host_y[ihit],host_z[ihit]);
      host_etas[ihit] = ((host_z[ihit]>0)-(host_z[ihit]<0))* std::acosh(sqrt(host_x[ihit]*host_x[ihit]+host_y[ihit]*host_y[ihit]+host_z[ihit]*host_z[ihit])/host_rts[ihit]);
//// This part i think has a race condition. so this is not run in parallel.
      unsigned int this_index = host_moduleIndex[ihit];
      if(module_subdet[this_index] == Endcap && module_moduleType[this_index] == TwoS)
      {
          float xhigh, yhigh, xlow, ylow;
          getEdgeHits(host_detId[ihit],host_x[ihit],host_y[ihit],xhigh,yhigh,xlow,ylow);
          host_highEdgeXs[ihit] = xhigh;
          host_highEdgeYs[ihit] = yhigh;
          host_lowEdgeXs[ihit] = xlow;
          host_lowEdgeYs[ihit] = ylow;

      }

      //set the hit ranges appropriately in the modules struct

      ////start the index rolling if the module is encountered for the first time
      ////always update the end index
      //modulesInGPU->hitRanges[this_index * 2 + 1] = ihit;
      //start the index rolling if the module is encountered for the first time
      if(module_hitRanges[this_index * 2] == -1)
      {
          module_hitRanges[this_index * 2] = ihit;
      }
      //always update the end index
      module_hitRanges[this_index * 2 + 1] = ihit;

  }
//simply copy the host arrays to the hitsInGPU struct
    hipMemcpy(hitsInGPU->xs,host_x,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->ys,host_y,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->zs,host_z,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->rts,host_rts,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->phis,host_phis,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->etas,host_etas,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->idxs,host_idxs,loopsize*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->moduleIndices,host_moduleIndex,loopsize*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->highEdgeXs,host_highEdgeXs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->highEdgeYs,host_highEdgeYs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->lowEdgeXs,host_lowEdgeXs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->lowEdgeYs,host_lowEdgeYs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->nHits,&loopsize,sizeof(unsigned int),hipMemcpyHostToDevice);// value can't correctly be set in hit allocation
    hipMemcpy(modulesInGPU->hitRanges,module_hitRanges,nModules*2*sizeof(int),hipMemcpyHostToDevice);// value can't correctly be set in hit allocation
    hipDeviceSynchronize(); //doesn't seem to make a difference

    hipHostFree(host_rts);
    //hipHostFree(host_idxs);
    hipHostFree(host_phis);
    hipHostFree(host_etas);
    hipHostFree(host_moduleIndex);
    hipHostFree(host_highEdgeXs);
    hipHostFree(host_highEdgeYs);
    hipHostFree(host_lowEdgeXs);
    hipHostFree(host_lowEdgeYs);
    hipHostFree(module_layers);
    hipHostFree(module_subdet);
    hipHostFree(module_hitRanges);
    hipHostFree(module_moduleType);

}
__global__ void addPixelSegmentToEventKernel(unsigned int* hitIndices0,unsigned int* hitIndices1,unsigned int* hitIndices2,unsigned int* hitIndices3, float* dPhiChange, float* ptIn, float* ptErr, float* px, float* py, float* pz, float* eta, float* etaErr,float* phi, unsigned int pixelModuleIndex, struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU,const int size, int* superbin, int* pixelType)
{

    for( int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < size; tid += blockDim.x*gridDim.x)
    {

      unsigned int innerMDIndex = pixelModuleIndex * N_MAX_MD_PER_MODULES + 2*(tid);
      unsigned int outerMDIndex = pixelModuleIndex * N_MAX_MD_PER_MODULES + 2*(tid) +1;
      unsigned int pixelSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + tid;

#ifdef CUT_VALUE_DEBUG
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,outerMDIndex);
#else
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,outerMDIndex);
#endif
      addPixelSegmentToMemory(segmentsInGPU, mdsInGPU, hitsInGPU, modulesInGPU, innerMDIndex, outerMDIndex, pixelModuleIndex, hitIndices0[tid], hitIndices2[tid], dPhiChange[tid], ptIn[tid], ptErr[tid], px[tid], py[tid], pz[tid], etaErr[tid], eta[tid], phi[tid], pixelSegmentIndex, tid, superbin[tid], pixelType[tid]);
    }
}
void SDL::Event::addPixelSegmentToEvent(std::vector<unsigned int> hitIndices0,std::vector<unsigned int> hitIndices1,std::vector<unsigned int> hitIndices2,std::vector<unsigned int> hitIndices3, std::vector<float> dPhiChange, std::vector<float> ptIn, std::vector<float> ptErr, std::vector<float> px, std::vector<float> py, std::vector<float> pz, std::vector<float> eta, std::vector<float> etaErr, std::vector<float> phi, std::vector<int> superbin, std::vector<int> pixelType)
{
    if(mdsInGPU == nullptr)
    {
        hipHostMalloc(&mdsInGPU, sizeof(SDL::miniDoublets));
#ifdef Explicit_MD
    	createMDsInExplicitMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#else
    	createMDsInUnifiedMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#endif
    }
    if(segmentsInGPU == nullptr)
    {
        hipHostMalloc(&segmentsInGPU, sizeof(SDL::segments));
#ifdef Explicit_Seg
        createSegmentsInExplicitMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#else
        createSegmentsInUnifiedMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#endif
    }
    const int size = ptIn.size();
    unsigned int pixelModuleIndex = (*detIdToIndex)[1];
    unsigned int* hitIndices0_host = &hitIndices0[0];
    unsigned int* hitIndices1_host = &hitIndices1[0];
    unsigned int* hitIndices2_host = &hitIndices2[0];
    unsigned int* hitIndices3_host = &hitIndices3[0];
    float* dPhiChange_host = &dPhiChange[0];
    float* ptIn_host = &ptIn[0];
    float* ptErr_host = &ptErr[0];
    float* px_host = &px[0];
    float* py_host = &py[0];
    float* pz_host = &pz[0];
    float* etaErr_host = &etaErr[0];
    float* eta_host = &eta[0];
    float* phi_host = &phi[0];
    int* superbin_host = &superbin[0];
    int* pixelType_host = &pixelType[0];

    unsigned int* hitIndices0_dev;
    unsigned int* hitIndices1_dev;
    unsigned int* hitIndices2_dev;
    unsigned int* hitIndices3_dev;
    float* dPhiChange_dev;
    float* ptIn_dev;
    float* ptErr_dev;
    float* px_dev;
    float* py_dev;
    float* pz_dev;
    float* etaErr_dev;
    float* eta_dev;
    float* phi_dev;
    int* superbin_dev;
    int* pixelType_dev;

    hipMalloc(&hitIndices0_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices1_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices2_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices3_dev,size*sizeof(unsigned int));
    hipMalloc(&dPhiChange_dev,size*sizeof(unsigned int));
    hipMalloc(&ptIn_dev,size*sizeof(unsigned int));
    hipMalloc(&ptErr_dev,size*sizeof(unsigned int));
    hipMalloc(&px_dev,size*sizeof(unsigned int));
    hipMalloc(&py_dev,size*sizeof(unsigned int));
    hipMalloc(&pz_dev,size*sizeof(unsigned int));
    hipMalloc(&etaErr_dev,size*sizeof(unsigned int));
    hipMalloc(&eta_dev, size*sizeof(unsigned int));
    hipMalloc(&phi_dev, size*sizeof(unsigned int));
    hipMalloc(&superbin_dev,size*sizeof(int));
    hipMalloc(&pixelType_dev,size*sizeof(int));

    hipMemcpy(hitIndices0_dev,hitIndices0_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitIndices1_dev,hitIndices1_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitIndices2_dev,hitIndices2_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitIndices3_dev,hitIndices3_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(dPhiChange_dev,dPhiChange_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(ptIn_dev,ptIn_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(ptErr_dev,ptErr_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(px_dev,px_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(py_dev,py_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(pz_dev,pz_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(etaErr_dev,etaErr_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(eta_dev, eta_host, size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(phi_dev, phi_host, size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(superbin_dev,superbin_host,size*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(pixelType_dev,pixelType_host,size*sizeof(int),hipMemcpyHostToDevice);

    unsigned int nThreads = 256;
    unsigned int nBlocks =  size % nThreads == 0 ? size/nThreads : size/nThreads + 1;

    addPixelSegmentToEventKernel<<<nBlocks,nThreads>>>(hitIndices0_dev,hitIndices1_dev,hitIndices2_dev,hitIndices3_dev,dPhiChange_dev,ptIn_dev,ptErr_dev,px_dev,py_dev,pz_dev,eta_dev, etaErr_dev, phi_dev, pixelModuleIndex, *modulesInGPU,*hitsInGPU,*mdsInGPU,*segmentsInGPU,size, superbin_dev, pixelType_dev);
   //std::cout<<"Number of pixel segments = "<<size<<std::endl;
   hipDeviceSynchronize();
   hipMemcpy(&(segmentsInGPU->nSegments)[pixelModuleIndex], &size, sizeof(unsigned int), hipMemcpyHostToDevice);
   unsigned int mdSize = 2 * size;
   hipMemcpy(&(mdsInGPU->nMDs)[pixelModuleIndex], &mdSize, sizeof(unsigned int), hipMemcpyHostToDevice);

    hipFree(hitIndices0_dev);
    hipFree(hitIndices1_dev);
    hipFree(hitIndices2_dev);
    hipFree(hitIndices3_dev);
    hipFree(dPhiChange_dev);
    hipFree(ptIn_dev);
    hipFree(ptErr_dev);
    hipFree(px_dev);
    hipFree(py_dev);
    hipFree(pz_dev);
    hipFree(etaErr_dev);
    hipFree(eta_dev);
    hipFree(phi_dev);
    hipFree(superbin_dev);
    hipFree(pixelType_dev);
}

void SDL::Event::addMiniDoubletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        if(mdsInGPU->nMDs[idx] == 0 or modulesInGPU->hitRanges[idx * 2] == -1)
        {
            modulesInGPU->mdRanges[idx * 2] = -1;
            modulesInGPU->mdRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->mdRanges[idx * 2] = idx * N_MAX_MD_PER_MODULES;
            modulesInGPU->mdRanges[idx * 2 + 1] = (idx * N_MAX_MD_PER_MODULES) + mdsInGPU->nMDs[idx] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[modulesInGPU->layers[idx] -1] += mdsInGPU->nMDs[idx];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += mdsInGPU->nMDs[idx];
            }

        }
    }
}
void SDL::Event::addMiniDoubletsToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
unsigned int* nMDsCPU;
hipHostMalloc(&nMDsCPU, nModules * sizeof(unsigned int));
hipMemcpy(nMDsCPU,mdsInGPU->nMDs,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_mdRanges;
hipHostMalloc(&module_mdRanges, nModules* 2*sizeof(int));
hipMemcpy(module_mdRanges,modulesInGPU->mdRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
int* module_hitRanges;
hipHostMalloc(&module_hitRanges, nModules* 2*sizeof(int));
hipMemcpy(module_hitRanges,modulesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);

    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        if(nMDsCPU[idx] == 0 or module_hitRanges[idx * 2] == -1)
        {
            module_mdRanges[idx * 2] = -1;
            module_mdRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_mdRanges[idx * 2] = idx * N_MAX_MD_PER_MODULES;
            module_mdRanges[idx * 2 + 1] = (idx * N_MAX_MD_PER_MODULES) + nMDsCPU[idx] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[module_layers[idx] -1] += nMDsCPU[idx];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[module_layers[idx] - 1] += nMDsCPU[idx];
            }

        }
    }
hipMemcpy(modulesInGPU->mdRanges,module_mdRanges,nModules*2*sizeof(int),hipMemcpyHostToDevice);
hipHostFree(nMDsCPU);
hipHostFree(module_subdets);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_mdRanges);
hipHostFree(module_layers);
hipHostFree(module_hitRanges);
}

void SDL::Event::addSegmentsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        if(segmentsInGPU->nSegments[idx] == 0)
        {
            modulesInGPU->segmentRanges[idx * 2] = -1;
            modulesInGPU->segmentRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->segmentRanges[idx * 2] = idx * N_MAX_SEGMENTS_PER_MODULE;
            modulesInGPU->segmentRanges[idx * 2 + 1] = idx * N_MAX_SEGMENTS_PER_MODULE + segmentsInGPU->nSegments[idx] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {

                n_segments_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += segmentsInGPU->nSegments[idx];
            }
            else
            {
                n_segments_by_layer_endcap_[modulesInGPU->layers[idx] -1] += segmentsInGPU->nSegments[idx];
            }
        }
    }
}
void SDL::Event::addSegmentsToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

unsigned int* nSegmentsCPU;
hipHostMalloc(&nSegmentsCPU, nModules * sizeof(unsigned int));
hipMemcpy(nSegmentsCPU,segmentsInGPU->nSegments,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_segmentRanges;
hipHostMalloc(&module_segmentRanges, nModules* 2*sizeof(int));
hipMemcpy(module_segmentRanges,modulesInGPU->segmentRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        if(nSegmentsCPU[idx] == 0)
        {
            module_segmentRanges[idx * 2] = -1;
            module_segmentRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_segmentRanges[idx * 2] = idx * N_MAX_SEGMENTS_PER_MODULE;
            module_segmentRanges[idx * 2 + 1] = idx * N_MAX_SEGMENTS_PER_MODULE + nSegmentsCPU[idx] - 1;

            if(module_subdets[idx] == Barrel)
            {

                n_segments_by_layer_barrel_[module_layers[idx] - 1] += nSegmentsCPU[idx];
            }
            else
            {
                n_segments_by_layer_endcap_[module_layers[idx] -1] += nSegmentsCPU[idx];
            }
        }
    }
hipHostFree(nSegmentsCPU);
hipHostFree(module_subdets);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_segmentRanges);
hipHostFree(module_layers);
}

void SDL::Event::createMiniDoublets()
{
    hipDeviceSynchronize();
    auto memStart = std::chrono::high_resolution_clock::now();
    if(mdsInGPU == nullptr)
    {
        hipHostMalloc(&mdsInGPU, sizeof(SDL::miniDoublets));
#ifdef Explicit_MD
        //FIXME: Add memory locations for pixel MDs
    	createMDsInExplicitMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#else
    	createMDsInUnifiedMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#endif
    }
    hipDeviceSynchronize();
    auto memStop = std::chrono::high_resolution_clock::now();
    auto memDuration = std::chrono::duration_cast<std::chrono::milliseconds>(memStop - memStart); //in milliseconds

    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

#ifdef NESTED_PARA
    int nThreads = 1;
    int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;
#else
#ifdef NEWGRID_MD
    int maxThreadsPerModule=0;
    #ifdef Explicit_Module
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    int* module_hitRanges;
    hipHostMalloc(&module_hitRanges, nModules* 2*sizeof(int));
    hipMemcpy(module_hitRanges,modulesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    bool* module_isLower;
    hipHostMalloc(&module_isLower, nModules*sizeof(bool));
    hipMemcpy(module_isLower,modulesInGPU->isLower,nModules*sizeof(bool),hipMemcpyDeviceToHost);
    bool* module_isInverted;
    hipHostMalloc(&module_isInverted, nModules*sizeof(bool));
    hipMemcpy(module_isInverted,modulesInGPU->isInverted,nModules*sizeof(bool),hipMemcpyDeviceToHost);

    for (int i=0; i<nLowerModules; i++) {
      int lowerModuleIndex = module_lowerModuleIndices[i];
      int upperModuleIndex = modulesInGPU->partnerModuleIndexExplicit(lowerModuleIndex,module_isLower[lowerModuleIndex],module_isInverted[lowerModuleIndex]);
      int lowerHitRanges = module_hitRanges[lowerModuleIndex*2];
      int upperHitRanges = module_hitRanges[upperModuleIndex*2];
      if(lowerHitRanges!=-1&&upperHitRanges!=-1) {
        unsigned int nLowerHits = module_hitRanges[lowerModuleIndex * 2 + 1] - lowerHitRanges + 1;
        unsigned int nUpperHits = module_hitRanges[upperModuleIndex * 2 + 1] - upperHitRanges + 1;
        maxThreadsPerModule = maxThreadsPerModule > (nLowerHits*nUpperHits) ? maxThreadsPerModule : nLowerHits*nUpperHits;
      }
    }
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_hitRanges);
    hipHostFree(module_isLower);
    hipHostFree(module_isInverted);
    #else
    //int maxThreadsPerModule=0;
    for (int i=0; i<nLowerModules; i++) {
      int lowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      int upperModuleIndex = modulesInGPU->partnerModuleIndex(lowerModuleIndex);
      int lowerHitRanges = modulesInGPU->hitRanges[lowerModuleIndex*2];
      int upperHitRanges = modulesInGPU->hitRanges[upperModuleIndex*2];
      if(lowerHitRanges!=-1&&upperHitRanges!=-1) {
        unsigned int nLowerHits = modulesInGPU->hitRanges[lowerModuleIndex * 2 + 1] - lowerHitRanges + 1;
        unsigned int nUpperHits = modulesInGPU->hitRanges[upperModuleIndex * 2 + 1] - upperHitRanges + 1;
        maxThreadsPerModule = maxThreadsPerModule > (nLowerHits*nUpperHits) ? maxThreadsPerModule : nLowerHits*nUpperHits;
      }
    }
    #endif
    //printf("maxThreadsPerModule=%d\n", maxThreadsPerModule);
    dim3 nThreads(1,128);
    dim3 nBlocks((nLowerModules % nThreads.x == 0 ? nLowerModules/nThreads.x : nLowerModules/nThreads.x + 1), (maxThreadsPerModule % nThreads.y == 0 ? maxThreadsPerModule/nThreads.y : maxThreadsPerModule/nThreads.y + 1));
#else
    dim3 nThreads(1,16,16);
    dim3 nBlocks((nLowerModules % nThreads.x == 0 ? nLowerModules/nThreads.x : nLowerModules/nThreads.x + 1),(N_MAX_HITS_PER_MODULE % nThreads.y == 0 ? N_MAX_HITS_PER_MODULE/nThreads.y : N_MAX_HITS_PER_MODULE/nThreads.y + 1), (N_MAX_HITS_PER_MODULE % nThreads.z == 0 ? N_MAX_HITS_PER_MODULE/nThreads.z : N_MAX_HITS_PER_MODULE/nThreads.z + 1));
#endif
#endif

    hipDeviceSynchronize();
    auto syncStart = std::chrono::high_resolution_clock::now();

    createMiniDoubletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU,*hitsInGPU,*mdsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    auto syncStop = std::chrono::high_resolution_clock::now();

    auto syncDuration =  std::chrono::duration_cast<std::chrono::milliseconds>(syncStop - syncStart);

    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }

#if defined(AddObjects)
#ifdef Explicit_MD
    addMiniDoubletsToEventExplicit();
#else
    addMiniDoubletsToEvent();
#endif
#endif


}

void SDL::Event::createSegmentsWithModuleMap()
{
    if(segmentsInGPU == nullptr)
    {
        hipHostMalloc(&segmentsInGPU, sizeof(SDL::segments));
#ifdef Explicit_Seg
        createSegmentsInExplicitMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#else
        createSegmentsInUnifiedMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#endif
    }
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;
#else
#ifdef NEWGRID_Seg
    int max_cModules=0;
    int sq_max_nMDs = 0;
    int nonZeroModules = 0;
  #ifdef Explicit_Module
    unsigned int nModules;
    hipMemcpy(&nModules,modulesInGPU->nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* nMDs = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nMDs, mdsInGPU->nMDs, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);

    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_nConnectedModules;
    hipHostMalloc(&module_nConnectedModules, nModules* sizeof(unsigned int));
    hipMemcpy(module_nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_moduleMap;
    hipHostMalloc(&module_moduleMap, nModules*40* sizeof(unsigned int));
    hipMemcpy(module_moduleMap,modulesInGPU->moduleMap,nModules*40*sizeof(unsigned int),hipMemcpyDeviceToHost);

    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = module_lowerModuleIndices[i];
      unsigned int nConnectedModules = module_nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerMDs = nMDs[innerLowerModuleIndex];
      max_cModules = max_cModules > nConnectedModules ? max_cModules : nConnectedModules;
      int limit_local = 0;
      if (nConnectedModules!=0) nonZeroModules++;
      for (int j=0; j<nConnectedModules; j++) {
        int outerLowerModuleIndex = module_moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + j];
        int nOuterMDs = nMDs[outerLowerModuleIndex];
        int total = nInnerMDs*nOuterMDs;
        limit_local = limit_local > total ? limit_local : total;
      }
      sq_max_nMDs = sq_max_nMDs > limit_local ? sq_max_nMDs : limit_local;
    }
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_nConnectedModules);
    hipHostFree(module_moduleMap);
  #else

    unsigned int nModules = *modulesInGPU->nModules;
    unsigned int* nMDs = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nMDs, mdsInGPU->nMDs, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      unsigned int nConnectedModules = modulesInGPU->nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerMDs = nMDs[innerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : nMDs[innerLowerModuleIndex];
      max_cModules = max_cModules > nConnectedModules ? max_cModules : nConnectedModules;
      int limit_local = 0;
      if (nConnectedModules!=0) nonZeroModules++;
      for (int j=0; j<nConnectedModules; j++) {
        int outerLowerModuleIndex = modulesInGPU->moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + j];
        int nOuterMDs = nMDs[outerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : nMDs[outerLowerModuleIndex];
        int total = nInnerMDs*nOuterMDs;
        limit_local = limit_local > total ? limit_local : total;
      }
      sq_max_nMDs = sq_max_nMDs > limit_local ? sq_max_nMDs : limit_local;
    }
  #endif
    //printf("max nConnectedModules=%d nonZeroModules=%d max sq_max_nMDs=%d\n", max_cModules, nonZeroModules, sq_max_nMDs);
    dim3 nThreads(256,1,1);
    dim3 nBlocks((sq_max_nMDs%nThreads.x==0 ? sq_max_nMDs/nThreads.x : sq_max_nMDs/nThreads.x + 1), (max_cModules%nThreads.y==0 ? max_cModules/nThreads.y : max_cModules/nThreads.y + 1), (nLowerModules%nThreads.z==0 ? nLowerModules/nThreads.z : nLowerModules/nThreads.z + 1));
    free(nMDs);
#else
    dim3 nThreads(1,16,16);
    dim3 nBlocks(((nLowerModules * MAX_CONNECTED_MODULES)  % nThreads.x == 0 ? (nLowerModules * MAX_CONNECTED_MODULES)/nThreads.x : (nLowerModules * MAX_CONNECTED_MODULES)/nThreads.x + 1),(N_MAX_MD_PER_MODULES % nThreads.y == 0 ? N_MAX_MD_PER_MODULES/nThreads.y : N_MAX_MD_PER_MODULES/nThreads.y + 1), (N_MAX_MD_PER_MODULES % nThreads.z == 0  ? N_MAX_MD_PER_MODULES/nThreads.z : N_MAX_MD_PER_MODULES/nThreads.z + 1));
#endif
#endif

    createSegmentsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#if defined(AddObjects)
#ifdef Explicit_Seg
    addSegmentsToEventExplicit();
#else
    addSegmentsToEvent();
#endif
#endif

}


void SDL::Event::createTriplets()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    if(tripletsInGPU == nullptr)
    {
        hipHostMalloc(&tripletsInGPU, sizeof(SDL::triplets));
#ifdef Explicit_Trips
        createTripletsInExplicitMemory(*tripletsInGPU, N_MAX_TRIPLETS_PER_MODULE, nLowerModules);
#else
        createTripletsInUnifiedMemory(*tripletsInGPU, N_MAX_TRIPLETS_PER_MODULE, nLowerModules);
#endif
    }

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTripletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
#else
#ifdef NEWGRID_Trips
  #ifdef Explicit_Module
    unsigned int nonZeroModules=0;
    unsigned int max_InnerSeg=0;
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    //unsigned int nModules = *modulesInGPU->nModules;
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_nConnectedModules;
    hipHostMalloc(&module_nConnectedModules, nModules* sizeof(unsigned int));
    hipMemcpy(module_nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = module_lowerModuleIndices[i];
      unsigned int nConnectedModules = module_nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerSegments = nSegments[innerLowerModuleIndex];
      if (nConnectedModules!=0&&nInnerSegments!=0) {
        index[nonZeroModules] = i;
        nonZeroModules++;
      }
      max_InnerSeg = max_InnerSeg > nInnerSegments ? max_InnerSeg : nInnerSegments;
    }
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_nConnectedModules);
  #else
    unsigned int nonZeroModules=0;
    unsigned int max_InnerSeg=0;
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    unsigned int nModules = *modulesInGPU->nModules;
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      unsigned int nConnectedModules = modulesInGPU->nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerSegments = nSegments[innerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : nSegments[innerLowerModuleIndex];
      if (nConnectedModules!=0&&nInnerSegments!=0) {
        index[nonZeroModules] = i;
        nonZeroModules++;
      }
      max_InnerSeg = max_InnerSeg > nInnerSegments ? max_InnerSeg : nInnerSegments;
    }
  #endif
    hipMemcpy(index_gpu, index, nonZeroModules*sizeof(unsigned int), hipMemcpyHostToDevice);
    int max_OuterSeg = 0;
    max_OuterSeg = N_MAX_SEGMENTS_PER_MODULE;
    dim3 nThreads(16,16,1);
    dim3 nBlocks((max_OuterSeg % nThreads.x == 0 ? max_OuterSeg / nThreads.x : max_OuterSeg / nThreads.x + 1),(max_InnerSeg % nThreads.y == 0 ? max_InnerSeg/nThreads.y : max_InnerSeg/nThreads.y + 1), (nonZeroModules % nThreads.z == 0 ? nonZeroModules/nThreads.z : nonZeroModules/nThreads.z + 1));
    createTripletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, index_gpu);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
    free(nSegments);
    free(index);
    hipFree(index_gpu);
#else
    printf("original 3D grid launching in createTriplets does not exist");
    exit(1);
#endif
#endif

#if defined(AddObjects)
#ifdef Explicit_Trips
    addTripletsToEventExplicit();
#else
    addTripletsToEvent();
#endif
#endif
}

void SDL::Event::createTrackletsWithModuleMap()
{
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    //TRCAKLETS - To conserve memory, we shall be only declaring nLowerModules amount of memory!!!!!!!
    if(trackletsInGPU == nullptr)
    {
        hipHostMalloc(&trackletsInGPU, sizeof(SDL::tracklets));
#ifdef Explicit_Tracklet
        //FIXME:Add memory locations for pixel tracklets
        createTrackletsInExplicitMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE, nLowerModules);
#else
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE, nLowerModules);
#endif
    }

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    #ifdef T4FromT3
      createTrackletsFromTriplets<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *trackletsInGPU);
    #else
      createTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);
    #endif

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
#else
#ifdef NEWGRID_Tracklet
  #ifdef T4FromT3
    int threadSize=230000;
    unsigned int *nTriplets = (unsigned int*)malloc((nLowerModules-1)*sizeof(unsigned int));
    unsigned int *threadIdx = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *threadIdx_offset = threadIdx+threadSize;
    unsigned int *threadIdx_gpu;
    unsigned int *threadIdx_gpu_offset;
    hipMalloc((void **)&threadIdx_gpu, 2*threadSize*sizeof(unsigned int));
    threadIdx_gpu_offset = threadIdx_gpu + threadSize;
    hipMemset(threadIdx_gpu, nLowerModules, threadSize*sizeof(unsigned int));
    hipMemcpy(nTriplets, tripletsInGPU->nTriplets, (nLowerModules-1)*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int totalCand=0;
    for (int i=0; i< nLowerModules-1; i++) {
      unsigned int nInnerTriplets = nTriplets[i];
      if(nInnerTriplets > N_MAX_TRIPLETS_PER_MODULE)
        nInnerTriplets = N_MAX_TRIPLETS_PER_MODULE;
      if (nInnerTriplets !=0) {
        for (int k=0; k<nInnerTriplets; k++) {
          threadIdx[totalCand+k] = i;
          threadIdx_offset[totalCand+k] = k;
        }
        totalCand += nInnerTriplets;
      }
    }
    hipMemcpy(threadIdx_gpu, threadIdx, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(threadIdx_gpu_offset, threadIdx_offset, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(16, 32, 1);
    dim3 nBlocks((N_MAX_TRIPLETS_PER_MODULE % nThreads.x == 0 ? N_MAX_TRIPLETS_PER_MODULE/nThreads.x : N_MAX_TRIPLETS_PER_MODULE/nThreads.x + 1), (totalCand % nThreads.y == 0 ? totalCand/nThreads.y : totalCand/nThreads.y + 1), 1);

    createTrackletsFromTriplets<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *trackletsInGPU,threadIdx_gpu,threadIdx_gpu_offset);
    free(threadIdx);
    hipFree(threadIdx_gpu);
    free(nTriplets);

  #else
      int max_cModules = 0;
      int sq_max_segments = 0;
      int nonZeroSegModules = 0;
      int inner_max_segments = 0;
      int outer_max_segments = 0;
      unsigned int *index_gpu;
      unsigned int *outerLowerModuleIndices = (unsigned int*)malloc(nModules*N_MAX_SEGMENTS_PER_MODULE*sizeof(unsigned int));
      unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
      unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
      hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    #ifdef Explicit_Module
      hipMemcpy((void *)outerLowerModuleIndices, segmentsInGPU->outerLowerModuleIndices, nModules*N_MAX_SEGMENTS_PER_MODULE*sizeof(unsigned int), hipMemcpyDeviceToHost);
      hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
      unsigned int* module_lowerModuleIndices;
      hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
      hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
      unsigned int* module_nConnectedModules;
      hipHostMalloc(&module_nConnectedModules, nModules* sizeof(unsigned int));
      hipMemcpy(module_nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
      unsigned int* module_moduleMap;
      hipHostMalloc(&module_moduleMap, nModules*40* sizeof(unsigned int));
      hipMemcpy(module_moduleMap,modulesInGPU->moduleMap,nModules*40*sizeof(unsigned int),hipMemcpyDeviceToHost);
      for (int i=0; i<nLowerModules; i++) {
        unsigned int innerInnerLowerModuleIndex = module_lowerModuleIndices[i];
        unsigned int nInnerSegments = nSegments[innerInnerLowerModuleIndex];
        if (nInnerSegments!=0) {
          index[nonZeroSegModules] = i;
          nonZeroSegModules++;
        }
        inner_max_segments = inner_max_segments > nInnerSegments ? inner_max_segments : nInnerSegments;

        for (int j=0; j<nInnerSegments; j++) {
          unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + j;
          unsigned int innerOuterLowerModuleIndex = outerLowerModuleIndices[innerSegmentIndex];
          unsigned int nOuterInnerLowerModules = module_nConnectedModules[innerOuterLowerModuleIndex];
          max_cModules = max_cModules > nOuterInnerLowerModules ? max_cModules : nOuterInnerLowerModules;
          for (int k=0; k<nOuterInnerLowerModules; k++) {
            unsigned int outerInnerLowerModuleIndex = module_moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + k];
            unsigned int nOuterSegments = nSegments[outerInnerLowerModuleIndex];
            sq_max_segments = sq_max_segments > nInnerSegments*nOuterSegments ? sq_max_segments : nInnerSegments*nOuterSegments;
          }
        }
      }
      hipHostFree(module_lowerModuleIndices);
      hipHostFree(module_nConnectedModules);
      hipHostFree(module_moduleMap);
    #else
      //unsigned int nModules = *modulesInGPU->nModules;
      hipMemcpy((void *)outerLowerModuleIndices, segmentsInGPU->outerLowerModuleIndices, nModules*N_MAX_SEGMENTS_PER_MODULE*sizeof(unsigned int), hipMemcpyDeviceToHost);
      hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
      for (int i=0; i<nLowerModules; i++) {
        unsigned int innerInnerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
        unsigned int nInnerSegments = nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE  ? N_MAX_SEGMENTS_PER_MODULE : nSegments[innerInnerLowerModuleIndex];
        if (nInnerSegments!=0) {
          index[nonZeroSegModules] = i;
          nonZeroSegModules++;
        }
        inner_max_segments = inner_max_segments > nInnerSegments ? inner_max_segments : nInnerSegments;

        for (int j=0; j<nInnerSegments; j++) {
          unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + j;
          unsigned int innerOuterLowerModuleIndex = outerLowerModuleIndices[innerSegmentIndex];
          unsigned int nOuterInnerLowerModules = modulesInGPU->nConnectedModules[innerOuterLowerModuleIndex];
          max_cModules = max_cModules > nOuterInnerLowerModules ? max_cModules : nOuterInnerLowerModules;
          for (int k=0; k<nOuterInnerLowerModules; k++) {
            unsigned int outerInnerLowerModuleIndex = modulesInGPU->moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + k];
            unsigned int nOuterSegments = nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : nSegments[outerInnerLowerModuleIndex];
            sq_max_segments = sq_max_segments > nInnerSegments*nOuterSegments ? sq_max_segments : nInnerSegments*nOuterSegments;
          }
        }
      }
    #endif
    hipMemcpy(index_gpu, index, nonZeroSegModules*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(128,1,1);
    dim3 nBlocks((sq_max_segments%nThreads.x==0 ? sq_max_segments/nThreads.x : sq_max_segments/nThreads.x + 1), (max_cModules%nThreads.y==0 ? max_cModules/nThreads.y : max_cModules/nThreads.y + 1), (nonZeroSegModules%nThreads.z==0 ? nonZeroSegModules/nThreads.z : nonZeroSegModules/nThreads.z + 1));

    createTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU, index_gpu);
    free(outerLowerModuleIndices);
    free(nSegments);
    free(index);
    hipFree(index_gpu);
  #endif
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }


#else
    printf("original 3D grid launching in createTracklets does not exist");
    exit(1);
#endif
#endif
#if defined(AddObjects)
#ifdef Explicit_Tracklet
    addTrackletsToEventExplicit();
#else
    addTrackletsToEvent();
#endif
#endif

}

void SDL::Event::createPixelTrackletsWithMap()
{
    if(pixelTrackletsInGPU == nullptr)
    {
        hipHostMalloc(&pixelTrackletsInGPU, sizeof(SDL::pixelTracklets));
#ifdef Explicit_Tracklet
        createPixelTrackletsInExplicitMemory(*pixelTrackletsInGPU, N_MAX_PIXEL_TRACKLETS_PER_MODULE);
#else
        createPixelTrackletsInUnifiedMemory(*pixelTrackletsInGPU, N_MAX_PIXEL_TRACKLETS_PER_MODULE);
#endif
    }
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createPixelTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
#else
#ifdef NEWGRID_Pixel
    unsigned int pixelModuleIndex;
    unsigned int nInnerSegments;
    int* superbins;
    int* pixelTypes;
#ifdef Explicit_Module
    unsigned int nModules;
    hipMemcpy(&nModules,modulesInGPU->nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    pixelModuleIndex = nModules-1;
    unsigned int* nSegments;
    hipHostMalloc(& nSegments,nModules*sizeof(unsigned int));
    hipMemcpy(nSegments,segmentsInGPU->nSegments,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    nInnerSegments = nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : nSegments[pixelModuleIndex]; // number of pLS
    hipHostMalloc(& superbins,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int));
    hipHostMalloc(& pixelTypes,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int));
    hipMemcpy(superbins,segmentsInGPU->superbin,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(pixelTypes,segmentsInGPU->pixelType,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost);

#else
    pixelModuleIndex = *modulesInGPU->nModules - 1; // pixel module index
    nInnerSegments = segmentsInGPU->nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU->nSegments[pixelModuleIndex]; // number of pLS
    superbins = segmentsInGPU->superbin;
    pixelTypes = segmentsInGPU->pixelType;
#endif
    unsigned int* connectedPixelSize_host;
    unsigned int* connectedPixelIndex_host;
    hipHostMalloc(&connectedPixelSize_host, nInnerSegments* sizeof(unsigned int));
    hipHostMalloc(&connectedPixelIndex_host, nInnerSegments* sizeof(unsigned int));
    unsigned int* connectedPixelSize_dev;
    unsigned int* connectedPixelIndex_dev;
    hipMalloc(&connectedPixelSize_dev, nInnerSegments* sizeof(unsigned int));
    hipMalloc(&connectedPixelIndex_dev, nInnerSegments* sizeof(unsigned int));
    unsigned int max_size =0;
    int threadSize = 1000000;
    unsigned int *segs_pix = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *segs_pix_offset = segs_pix+threadSize;
    unsigned int *segs_pix_gpu;
    unsigned int *segs_pix_gpu_offset;
    hipMalloc((void **)&segs_pix_gpu, 2*threadSize*sizeof(unsigned int));
    segs_pix_gpu_offset = segs_pix_gpu + threadSize;
    hipMemset(segs_pix_gpu, nInnerSegments, threadSize*sizeof(unsigned int)); // so if not set, it will pass in the kernel
    unsigned int totalSegs=0;
    int pixelIndexOffsetPos = pixelMapping->connectedPixelsIndex[44999] + pixelMapping->connectedPixelsSizes[44999];
    int pixelIndexOffsetNeg = pixelMapping->connectedPixelsIndexPos[44999] + pixelMapping->connectedPixelsSizes[44999] + pixelIndexOffsetPos;
    int i =-1;
    for (int ix=0; ix < nInnerSegments;ix++){// loop over # pLS
      int pixelType = pixelTypes[ix];// get pixel type for this pLS
      int superbin = superbins[ix]; //get superbin for this pixel
      if(superbin <0) {/*printf("bad neg %d\n",ix);*/continue;}
      if(superbin >=45000) {/*printf("bad pos %d %d %d\n",ix,superbin,pixelType);*/continue;}// skip any weird out of range values
      if(pixelType >2 || pixelType < 0){/*printf("bad pixel type %d %d\n",ix,pixelType);*/continue;}
      i++;
      if(pixelType ==0){ // used pixel type to select correct size-index arrays
        connectedPixelSize_host[i]  = pixelMapping->connectedPixelsSizes[superbin]; //number of connected modules to this pixel
        connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndex[superbin];// index to get start of connected modules for this superbin in map
        for (int j=0; j < pixelMapping->connectedPixelsSizes[superbin]; j++){ // loop over modules from the size
          segs_pix[totalSegs+j] = i; // save the pixel index in array to be transfered to kernel
          segs_pix_offset[totalSegs+j] = j; // save this segment in array to be transfered to kernel
        }
        totalSegs += connectedPixelSize_host[i]; // increment counter
      }
      else if(pixelType ==1){
        connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesPos[superbin]; //number of pixel connected modules
        connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndexPos[superbin]+pixelIndexOffsetPos;// index to get start of connected pixel modules
        for (int j=0; j < pixelMapping->connectedPixelsSizesPos[superbin]; j++){
          segs_pix[totalSegs+j] = i;
          segs_pix_offset[totalSegs+j] = j;
        }
        totalSegs += connectedPixelSize_host[i];
      }
      else if(pixelType ==2){
        connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesNeg[superbin]; //number of pixel connected modules
        connectedPixelIndex_host[i] =pixelMapping->connectedPixelsIndexNeg[superbin] + pixelIndexOffsetNeg;// index to get start of connected pixel modules
        for (int j=0; j < pixelMapping->connectedPixelsSizesNeg[superbin]; j++){
          segs_pix[totalSegs+j] = i;
          segs_pix_offset[totalSegs+j] = j;
        }
        totalSegs += connectedPixelSize_host[i];
      }
      else{continue;}
    }

    hipMemcpy(connectedPixelSize_dev, connectedPixelSize_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(connectedPixelIndex_dev, connectedPixelIndex_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(segs_pix_gpu,segs_pix,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(segs_pix_gpu_offset,segs_pix_offset,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    //nuking max_size
    max_size = N_MAX_SEGMENTS_PER_MODULE;

    dim3 nThreads(32,16,1);
    dim3 nBlocks((totalSegs % nThreads.x == 0 ? totalSegs / nThreads.x : totalSegs / nThreads.x + 1),
                  (max_size % nThreads.y == 0 ? max_size/nThreads.y : max_size/nThreads.y + 1),1);
    createPixelTrackletsInGPUFromMap<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU,
    connectedPixelSize_dev,connectedPixelIndex_dev,nInnerSegments,segs_pix_gpu,segs_pix_gpu_offset, totalSegs);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

      }

    hipHostFree(connectedPixelSize_host);
    hipHostFree(connectedPixelIndex_host);
    hipFree(connectedPixelSize_dev);
    hipFree(connectedPixelIndex_dev);
#ifdef Explicit_Module
    hipHostFree(nSegments);
    hipHostFree(superbins);
    hipHostFree(pixelTypes);
#endif
    free(segs_pix);
    hipFree(segs_pix_gpu);

#else
    printf("original 3D grid launching in createPixelTracklets does not exist");
    exit(2);
#endif
#endif

    unsigned int nPixelTracklets;
    hipMemcpy(&nPixelTracklets, &(pixelTrackletsInGPU->nPixelTracklets), sizeof(unsigned int), hipMemcpyDeviceToHost);
}

void SDL::Event::createPixelTracklets()
{
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    if(pixelTrackletsInGPU == nullptr)
    {
        hipHostMalloc(&pixelTrackletsInGPU, sizeof(SDL::pixelTracklets));
#ifdef Explicit_Tracklet
        createPixelTrackletsInExplicitMemory(*pixelTrackletsInGPU, N_MAX_PIXEL_TRACKLETS_PER_MODULE);
#else
        createPixelTrackletsInUnifiedMemory(*pixelTrackletsInGPU, N_MAX_PIXEL_TRACKLETS_PER_MODULE);
#endif
    }

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createPixelTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
#else
#ifdef NEWGRID_Pixel
#ifdef Explicit_Module
    unsigned int nModules; //= *modulesInGPU->nModules;
    hipMemcpy(&nModules,modulesInGPU->nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
#else
    unsigned int nModules = *modulesInGPU->nModules;
#endif
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int pixelModuleIndex = nModules - 1;
    unsigned int nInnerSegments = nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : nSegments[pixelModuleIndex];
#ifdef Explicit_Module
    unsigned int* lowerModuleIndices;
    hipHostMalloc(&lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
#endif
    int threadSize = 100000;
    unsigned int *threadIdx = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *threadIdx_offset = threadIdx+threadSize;
    unsigned int *threadIdx_gpu;
    unsigned int *threadIdx_gpu_offset;
    hipMalloc((void **)&threadIdx_gpu, 2*threadSize*sizeof(unsigned int));
    threadIdx_gpu_offset = threadIdx_gpu + threadSize;
    hipMemset(threadIdx_gpu, nLowerModules, threadSize*sizeof(unsigned int));
    unsigned int totalCand=0;
    for (int i=0; i<nLowerModules; i++) {
#ifdef Explicit_Module
      unsigned int outerInnerLowerModuleIndex = lowerModuleIndices[i];
#else
      unsigned int outerInnerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
#endif
      unsigned int nOuterSegments = nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : nSegments[outerInnerLowerModuleIndex];
      if (nOuterSegments!=0) {
	for (int k=0; k<nOuterSegments; k++) {
          threadIdx[totalCand+k] = i;
          threadIdx_offset[totalCand+k] = k;
        }
	totalCand += nOuterSegments;
      }
    }

    if (threadSize < totalCand) {
      printf("threadSize=%d totalCand=%d: increase buffer size for threadIdx in createPixelTracklets\n", threadSize, totalCand);
      exit(1);
    }

    hipMemcpy(threadIdx_gpu, threadIdx, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(threadIdx_gpu_offset, threadIdx_offset, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(16,32,1);
    dim3 nBlocks((nInnerSegments % nThreads.x == 0 ? nInnerSegments / nThreads.x : nInnerSegments / nThreads.x + 1),(totalCand % nThreads.y == 0 ? totalCand/nThreads.y : totalCand/nThreads.y + 1), 1);
    createPixelTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU, threadIdx_gpu, threadIdx_gpu_offset);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
    	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }

    free(nSegments);
    free(threadIdx);
    hipFree(threadIdx_gpu);
#ifdef Explicit_Module
    hipHostFree(lowerModuleIndices);
#endif

#else
    printf("original 3D grid launching in createPixelTracklets does not exist");
    exit(2);
#endif
#endif

    unsigned int nPixelTracklets;
    hipMemcpy(&nPixelTracklets, &(pixelTrackletsInGPU->nPixelTracklets), sizeof(unsigned int), hipMemcpyDeviceToHost);
#ifdef Warnings
    std::cout<<"number of pixel tracklets = "<<nPixelTracklets<<std::endl;
#endif
}

void SDL::Event::createTrackCandidates()
{
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules + 1; //including the pixel module
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    nLowerModules += 1;// include the pixel module

    //construct the list of eligible modules
    unsigned int nEligibleModules = 0;
    createEligibleModulesListForTrackCandidates(*modulesInGPU, nEligibleModules, N_MAX_TRACK_CANDIDATES_PER_MODULE);

    if(trackCandidatesInGPU == nullptr)
    {
        hipHostMalloc(&trackCandidatesInGPU, sizeof(SDL::trackCandidates));
#ifdef Explicit_Track
        createTrackCandidatesInExplicitMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES_PER_MODULE, N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE, nLowerModules, nEligibleModules);
#else
        createTrackCandidatesInUnifiedMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES_PER_MODULE, N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE, nLowerModules, nEligibleModules);
#endif
    }

#ifdef FINAL_pT5
    printf("Adding pT5s to TC collection\n");
    unsigned int nThreadsx_pT5 = 1;
    unsigned int nBlocksx_pT5 = (N_MAX_PIXEL_QUINTUPLETS) % nThreadsx_pT5 == 0 ? N_MAX_PIXEL_QUINTUPLETS / nThreadsx_pT5 : N_MAX_PIXEL_QUINTUPLETS / nThreadsx_pT5 + 1;
    addpT5asTrackCandidateInGPU<<<nBlocksx_pT5, nThreadsx_pT5>>>(*modulesInGPU, *pixelQuintupletsInGPU, *trackCandidatesInGPU);
    hipError_t cudaerr_pT5 = hipDeviceSynchronize();
    if(cudaerr_pT5 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pT5)<<std::endl;
    }
#endif

#ifdef FINAL_T5
    printf("running final state T5\n");
    dim3 nThreads(32,16,1);
    dim3 nBlocks(((nLowerModules-1) % nThreads.x == 0 ? (nLowerModules-1)/nThreads.x : (nLowerModules-1)/nThreads.x + 1),((N_MAX_QUINTUPLETS_PER_MODULE-1) % nThreads.y == 0 ? (N_MAX_QUINTUPLETS_PER_MODULE-1)/nThreads.y : (N_MAX_QUINTUPLETS_PER_MODULE-1)/nThreads.y + 1),1);
    addT5asTrackCandidateInGPU<<<nBlocks,nThreads>>>(*modulesInGPU,*quintupletsInGPU,*trackCandidatesInGPU);

    hipError_t cudaerr_T5 = hipDeviceSynchronize();
    if(cudaerr_T5 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_T5)<<std::endl;
    }
#endif // final state T5
#ifdef FINAL_pT2
    printf("running final state pT2\n");
    unsigned int nThreadsx = 1;
    unsigned int nBlocksx = ( N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE) % nThreadsx == 0 ? N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE/nThreadsx : N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE/nThreadsx + 1;
    addpT2asTrackCandidateInGPU<<<nBlocksx,nThreadsx>>>(*modulesInGPU,*pixelTrackletsInGPU,*trackCandidatesInGPU);
    hipError_t cudaerr_pT2 = hipDeviceSynchronize();
    if(cudaerr_pT2 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pT2)<<std::endl;
    }
#elif FINAL_pT3
    printf("running final state pT3\n");
    unsigned int nThreadsx = 1;
    unsigned int nBlocksx = (N_MAX_PIXEL_TRIPLETS) % nThreadsx == 0 ? N_MAX_PIXEL_TRIPLETS / nThreadsx : N_MAX_PIXEL_TRIPLETS / nThreadsx + 1;
    addpT3asTrackCandidateInGPU<<<nBlocksx, nThreadsx>>>(*modulesInGPU, *pixelTripletsInGPU, *trackCandidatesInGPU, *segmentsInGPU, *pixelQuintupletsInGPU);
    hipError_t cudaerr_pT3 = hipDeviceSynchronize();
    if(cudaerr_pT3 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr_pT3)<<std::endl;
    }
#endif // final state pT2 and pT3


#ifdef FINAL_T3T4
    printf("running final state T3T4\n");
#ifdef NESTED_PARA
    //auto t0 = std::chrono::high_resolution_clock::now();
    unsigned int nThreads = 1;
    unsigned int nBlocks = (nLowerModules-1) % nThreads == 0 ? (nLowerModules-1)/nThreads : (nLowerModules-1)/nThreads + 1;

    createTrackCandidatesInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU, *tripletsInGPU, *trackCandidatesInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }

    //Pixel Track Candidates created separately
    nThreads = 1;
    nBlocks = (nLowerModules - 1) % nThreads == 0 ? (nLowerModules - 1)/nThreads : (nLowerModules - 1)/nThreads + 1;

    createPixelTrackCandidatesInGPU<<<nBlocks, nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU, *trackletsInGPU, *tripletsInGPU, *trackCandidatesInGPU);

    cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#else
#ifdef NEWGRID_Track
    //auto t0 = std::chrono::high_resolution_clock::now();
    int maxOuterTr = max(N_MAX_TRACKLETS_PER_MODULE, N_MAX_TRIPLETS_PER_MODULE);
    unsigned int *nTriplets = (unsigned int*)malloc((2*nLowerModules-1)*sizeof(unsigned int));
    //unsigned int *nTracklets = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *nTracklets = nTriplets + nLowerModules -1;
    //int threadSize=2300000;
    int threadSize=10000000;
    unsigned int *threadIdx = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *threadIdx_offset = threadIdx+threadSize;
    unsigned int *threadIdx_gpu;
    unsigned int *threadIdx_gpu_offset;
    hipMalloc((void **)&threadIdx_gpu, 2*threadSize*sizeof(unsigned int));
    threadIdx_gpu_offset = threadIdx_gpu + threadSize;
    hipMemset(threadIdx_gpu, nLowerModules, threadSize*sizeof(unsigned int));
    hipMemcpy(nTriplets, tripletsInGPU->nTriplets, (nLowerModules-1)*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(nTracklets, trackletsInGPU->nTracklets, nLowerModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    int nPixelTracklets;
    hipMemcpy(&nPixelTracklets, pixelTrackletsInGPU->nPixelTracklets, sizeof(unsigned int), hipMemcpyDeviceToHost);
    if(nPixelTracklets > N_MAX_PIXEL_TRACKLETS_PER_MODULE)
      nPixelTracklets = N_MAX_PIXEL_TRACKLETS_PER_MODULE;
    unsigned int totalCand=0;
    for (int i=0; i< nLowerModules-1; i++) {
      unsigned int nInnerTracklets = nTracklets[i];
      if(nInnerTracklets > N_MAX_TRACKLETS_PER_MODULE)
	nInnerTracklets = N_MAX_TRACKLETS_PER_MODULE;
      unsigned int nInnerTriplets = nTriplets[i];
      if(nInnerTriplets > N_MAX_TRIPLETS_PER_MODULE)
        nInnerTriplets = N_MAX_TRIPLETS_PER_MODULE;
      unsigned int temp = max(nInnerTracklets, nInnerTriplets);
      if (temp !=0) {
        for (int k=0; k<temp; k++) {
          threadIdx[totalCand+k] = i;
          //printf("totalCand+k: %d\n",totalCand+k);
          threadIdx_offset[totalCand+k] = k;
        }
	totalCand += temp;
      }
    }
    if (threadSize < totalCand) {
      printf("threadSize=%d totalCand=%d: Increase buffer size for threadIdx in createTrackCandidates\n", threadSize, totalCand);
      exit(2);
    }
    hipMemcpy(threadIdx_gpu, threadIdx, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(threadIdx_gpu_offset, threadIdx_offset, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(16, 32, 1);
    dim3 nBlocks((maxOuterTr % nThreads.x == 0 ? maxOuterTr/nThreads.x : maxOuterTr/nThreads.x + 1), (totalCand % nThreads.y == 0 ? totalCand/nThreads.y : totalCand/nThreads.y + 1), 1);
    createTrackCandidatesInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU, *tripletsInGPU, *trackCandidatesInGPU, threadIdx_gpu, threadIdx_gpu_offset);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
    dim3 nThreads_p(16,16,1);
    dim3 nBlocks_p((nPixelTracklets % nThreads_p.x == 0 ? nPixelTracklets/nThreads_p.x : nPixelTracklets/nThreads_p.x + 1), (totalCand % nThreads_p.y == 0 ? totalCand/nThreads_p.y : totalCand/nThreads_p.y + 1), 1);
    createPixelTrackCandidatesInGPU<<<nBlocks_p, nThreads_p>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTrackletsInGPU, *trackletsInGPU, *tripletsInGPU, *trackCandidatesInGPU, threadIdx_gpu, threadIdx_gpu_offset);
    cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }


    free(threadIdx);
    free(nTriplets);
    hipFree(threadIdx_gpu);
#else
    printf("original 3D grid launching in createTrackCandidates does not exist");
    exit(3);
#endif
#endif
#endif // Final state T3+T4
#if defined(AddObjects)
#ifdef Explicit_Track
    addTrackCandidatesToEventExplicit();
#else
    addTrackCandidatesToEvent();
#endif
#endif

}

void SDL::Event::createPixelTriplets()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);

    if(pixelTripletsInGPU == nullptr)
    {
        hipHostMalloc(&pixelTripletsInGPU, sizeof(SDL::pixelTriplets));
    }
#ifdef Explicit_PT3
    createPixelTripletsInExplicitMemory(*pixelTripletsInGPU, N_MAX_PIXEL_TRIPLETS);
#else
    createPixelTripletsInUnifiedMemory(*pixelTripletsInGPU, N_MAX_PIXEL_TRIPLETS);
#endif

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules / nThreads : nLowerModules / nThreads + 1;

    createPixelTripletsInGPU<<<nBlocks, nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *pixelTripletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
#else
#ifdef NEWGRID_pT3
    unsigned int pixelModuleIndex;
    int* superbins;
    int* pixelTypes;
    unsigned int *nTriplets;
    unsigned int nModules;
    hipMemcpy(&nModules,modulesInGPU->nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    pixelModuleIndex = nModules-1;
    unsigned int nInnerSegments = 0;
    hipMemcpy(&nInnerSegments, &(segmentsInGPU->nSegments[pixelModuleIndex]), sizeof(unsigned int), hipMemcpyDeviceToHost);
    nInnerSegments = std::min(nInnerSegments, N_MAX_PIXEL_SEGMENTS_PER_MODULE);

    hipHostMalloc(&nTriplets, nLowerModules * sizeof(unsigned int));
    hipMemcpy(nTriplets, tripletsInGPU->nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipHostMalloc(&superbins,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int));
    hipHostMalloc(&pixelTypes,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int));

    hipMemcpy(superbins,segmentsInGPU->superbin,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(pixelTypes,segmentsInGPU->pixelType,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost);

    unsigned int* connectedPixelSize_host;
    unsigned int* connectedPixelIndex_host;
    hipHostMalloc(&connectedPixelSize_host, nInnerSegments* sizeof(unsigned int));
    hipHostMalloc(&connectedPixelIndex_host, nInnerSegments* sizeof(unsigned int));
    unsigned int* connectedPixelSize_dev;
    unsigned int* connectedPixelIndex_dev;
    hipMalloc(&connectedPixelSize_dev, nInnerSegments* sizeof(unsigned int));
    hipMalloc(&connectedPixelIndex_dev, nInnerSegments* sizeof(unsigned int));
    unsigned int max_size =0;
    int threadSize = 1000000;
    unsigned int *segs_pix = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *segs_pix_offset = segs_pix+threadSize;
    unsigned int *segs_pix_gpu;
    unsigned int *segs_pix_gpu_offset;
    hipMalloc((void **)&segs_pix_gpu, 2*threadSize*sizeof(unsigned int));
    segs_pix_gpu_offset = segs_pix_gpu + threadSize;
    hipMemset(segs_pix_gpu, nInnerSegments, threadSize*sizeof(unsigned int)); // so if not set, it will pass in the kernel
    unsigned int totalSegs=0;
    int pixelIndexOffsetPos = pixelMapping->connectedPixelsIndex[44999] + pixelMapping->connectedPixelsSizes[44999];
    int pixelIndexOffsetNeg = pixelMapping->connectedPixelsIndexPos[44999] + pixelMapping->connectedPixelsSizes[44999] + pixelIndexOffsetPos;

    for (int i = 0; i < nInnerSegments; i++)
    {// loop over # pLS
        int pixelType = pixelTypes[i];// get pixel type for this pLS
        int superbin = superbins[i]; //get superbin for this pixel
        if((superbin < 0) or (superbin >= 45000) or (pixelType > 2) or (pixelType < 0))
        {
            continue;
        }

        if(pixelType ==0)
        { // used pixel type to select correct size-index arrays
            connectedPixelSize_host[i]  = pixelMapping->connectedPixelsSizes[superbin]; //number of connected modules to this pixel
            connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndex[superbin];// index to get start of connected modules for this superbin in map
            for (int j=0; j < pixelMapping->connectedPixelsSizes[superbin]; j++)
            { // loop over modules from the size
                segs_pix[totalSegs+j] = i; // save the pixel index in array to be transfered to kernel
              segs_pix_offset[totalSegs+j] = j; // save this segment in array to be transfered to kernel
            }
            totalSegs += connectedPixelSize_host[i]; // increment counter
        }
        else if(pixelType ==1)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesPos[superbin]; //number of pixel connected modules
            connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndexPos[superbin]+pixelIndexOffsetPos;// index to get start of connected pixel modules
            for (int j=0; j < pixelMapping->connectedPixelsSizesPos[superbin]; j++)
            {
                segs_pix[totalSegs+j] = i;
                segs_pix_offset[totalSegs+j] = j;
            }
            totalSegs += connectedPixelSize_host[i];
        }
        else if(pixelType ==2)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesNeg[superbin]; //number of pixel connected modules
            connectedPixelIndex_host[i] =pixelMapping->connectedPixelsIndexNeg[superbin] + pixelIndexOffsetNeg;// index to get start of connected pixel modules
            for (int j=0; j < pixelMapping->connectedPixelsSizesNeg[superbin]; j++)
            {
                segs_pix[totalSegs+j] = i;
                segs_pix_offset[totalSegs+j] = j;
            }
            totalSegs += connectedPixelSize_host[i];
        }
    }

    hipMemcpy(connectedPixelSize_dev, connectedPixelSize_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(connectedPixelIndex_dev, connectedPixelIndex_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(segs_pix_gpu,segs_pix,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(segs_pix_gpu_offset,segs_pix_offset,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    //less cheap method to estimate max_size for y axis
    max_size = *std::max_element(nTriplets, nTriplets + nLowerModules);
    dim3 nThreads(16,16,1);
    dim3 nBlocks((totalSegs % nThreads.x == 0 ? totalSegs / nThreads.x : totalSegs / nThreads.x + 1),
                  (max_size % nThreads.y == 0 ? max_size/nThreads.y : max_size/nThreads.y + 1),1);
    createPixelTripletsInGPUFromMap<<<nBlocks, nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *pixelTripletsInGPU, connectedPixelSize_dev,connectedPixelIndex_dev,nInnerSegments,segs_pix_gpu,segs_pix_gpu_offset, totalSegs);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
    hipHostFree(connectedPixelSize_host);
    hipHostFree(connectedPixelIndex_host);
    hipFree(connectedPixelSize_dev);
    hipFree(connectedPixelIndex_dev);
    hipHostFree(superbins);
    hipHostFree(pixelTypes);
    hipHostFree(nTriplets);
    free(segs_pix);
    hipFree(segs_pix_gpu);
#else
    printf("original 3D grid launching in createPixelTriplets does not exist");
    exit(2);
#endif
#endif
    unsigned int nPixelTriplets;
    hipMemcpy(&nPixelTriplets, &(pixelTripletsInGPU->nPixelTriplets),  sizeof(unsigned int), hipMemcpyDeviceToHost);
#ifdef Warnings
    std::cout<<"number of pixel triplets = "<<nPixelTriplets<<std::endl;
#endif

    //pT3s can be cleaned here because they're not used in making pT5s!
#ifdef DUP_pT3
    printf("run dup pT3\n");
    dim3 nThreads_dup(1024,1,1);
    dim3 nBlocks_dup(64,1,1);
    removeDupPixelTripletsInGPUFromMap<<<nBlocks_dup,nThreads_dup>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTripletsInGPU,*tripletsInGPU);
#endif

}


void SDL::Event::createQuintuplets()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int nEligibleT5Modules = 0;
    unsigned int *indicesOfEligibleModules = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));

    unsigned int maxTriplets;
    createEligibleModulesListForQuintuplets(*modulesInGPU, *tripletsInGPU, nEligibleT5Modules, indicesOfEligibleModules, N_MAX_QUINTUPLETS_PER_MODULE, maxTriplets);

    if(quintupletsInGPU == nullptr)
    {
        hipHostMalloc(&quintupletsInGPU, sizeof(SDL::quintuplets));
#ifdef Explicit_T5
        createQuintupletsInExplicitMemory(*quintupletsInGPU, N_MAX_QUINTUPLETS_PER_MODULE, nLowerModules, nEligibleT5Modules);
#else
        createQuintupletsInUnifiedMemory(*quintupletsInGPU, N_MAX_QUINTUPLETS_PER_MODULE, nLowerModules, nEligibleT5Modules);
#endif
    }


#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;
    createQuintupletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU);


    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#else
#ifdef NEWGRID_T5
    int threadSize=N_MAX_TOTAL_TRIPLETS;
    unsigned int *threadIdx = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *threadIdx_offset = threadIdx+threadSize;
    unsigned int *threadIdx_gpu;
    unsigned int *threadIdx_gpu_offset;
    hipMalloc((void **)&threadIdx_gpu, 2*threadSize*sizeof(unsigned int));
    threadIdx_gpu_offset = threadIdx_gpu + threadSize;
    hipMemset(threadIdx_gpu, nLowerModules, threadSize*sizeof(unsigned int));

    unsigned int *nTriplets = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    hipMemcpy(nTriplets, tripletsInGPU->nTriplets, nLowerModules*sizeof(unsigned int), hipMemcpyDeviceToHost);

    int nTotalTriplets = 0;
    for (int i=0; i<nEligibleT5Modules; i++) {
      int index = indicesOfEligibleModules[i];
      unsigned int nInnerTriplets = nTriplets[index];
      if (nInnerTriplets > N_MAX_TRIPLETS_PER_MODULE) nInnerTriplets = N_MAX_TRIPLETS_PER_MODULE;
      if (nInnerTriplets !=0) {
        for (int j=0; j<nInnerTriplets; j++) {
          threadIdx[nTotalTriplets + j] = index;
          threadIdx_offset[nTotalTriplets + j] = j;
        }
        nTotalTriplets += nInnerTriplets;
      }
    }
    printf("T5: nTotalTriplets=%d nEligibleT5Modules=%d\n", nTotalTriplets, nEligibleT5Modules);
    if (threadSize < nTotalTriplets) {
      printf("threadSize=%d nTotalTriplets=%d: Increase buffer size for threadIdx in createQuintuplets\n", threadSize, nTotalTriplets);
      exit(1);
    }
    hipMemcpy(threadIdx_gpu, threadIdx, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(threadIdx_gpu_offset, threadIdx_offset, threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 nThreads(16, 16, 1);
    int max_outerTriplets = N_MAX_TRIPLETS_PER_MODULE;

    int mp;
    hipDeviceGetAttribute(&mp, hipDeviceAttributeMultiprocessorCount, 0);
    int m = (nTotalTriplets + nThreads.y*mp - 1)/(nThreads.y*mp);
    int mPerThread=16;
    m = (m + mPerThread -1)/mPerThread;
    int nblocksY = mp*m;
    printf("cuda multiprocessor #:%d mPerThreads=%d nBlocksY=%d\n", mp, mPerThread, nblocksY);
    dim3 nBlocks((max_outerTriplets % nThreads.x == 0 ? max_outerTriplets/nThreads.x : max_outerTriplets/nThreads.x + 1), nblocksY, 1);

    //    dim3 nBlocks((max_outerTriplets % nThreads.x == 0 ? max_outerTriplets/nThreads.x : max_outerTriplets/nThreads.x + 1), (nTotalTriplets % nThreads.y == 0 ? nTotalTriplets/nThreads.y : nTotalTriplets/nThreads.y + 1), 1);
    createQuintupletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, threadIdx_gpu, threadIdx_gpu_offset, nTotalTriplets);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
    free(threadIdx);
    free(nTriplets);
    hipFree(threadIdx_gpu);
#else
    printf("original 3D grid launching in createQuintuplets does not exist");
    exit(3);
#endif
#endif
    free(indicesOfEligibleModules);

#ifdef DUP_T5
  printf("run dup T5\n");
    dim3 dupThreads(32,32,1);
    dim3 dupBlocks(16,16,1);
    //dim3 dupThreads(1,1,1);
    //dim3 dupBlocks(1,1,1);
    removeDupQuintupletsInGPU<<<dupBlocks,dupThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU,false);
    hipDeviceSynchronize();
    //removeDupQuintupletsInGPU<<<dupBlocks,dupThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU,true);
    //hipDeviceSynchronize();
#endif

#if defined(AddObjects)
#ifdef Explicit_T5
    addQuintupletsToEventExplicit();
#else
    addQuintupletsToEvent();
#endif
#endif

}

void SDL::Event::createPixelQuintuplets()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);

    if(pixelQuintupletsInGPU == nullptr)
    {
        hipHostMalloc(&pixelQuintupletsInGPU, sizeof(SDL::pixelQuintuplets));
    }
#ifdef Explicit_PT5
    createPixelQuintupletsInExplicitMemory(*pixelQuintupletsInGPU, N_MAX_PIXEL_QUINTUPLETS);
#else
    createPixelQuintupletsInUnifiedMemory(*pixelQuintupletsInGPU, N_MAX_PIXEL_QUINTUPLETS);
#endif  
    
#ifdef DUP_pLS
    checkHitspLS<<<64,1024>>>(*modulesInGPU,*mdsInGPU, *segmentsInGPU, *hitsInGPU);
    hipError_t cudaerrpix = hipDeviceSynchronize();
    if(cudaerrpix != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerrpix)<<std::endl;

    }
#endif  

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads/ + 1;

    createPixelQuintupletsInGPU<<<nBlocks, nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, *pixelQuintupletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#else
#ifdef NEWGRID_pT5
    unsigned int pixelModuleIndex;
    int* superbins;
    int* pixelTypes;
    unsigned int *nQuintuplets;
    unsigned int nModules;
    hipMemcpy(&nModules,modulesInGPU->nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    pixelModuleIndex = nModules-1;
    unsigned int nInnerSegments = 0;
    hipMemcpy(&nInnerSegments, &(segmentsInGPU->nSegments[pixelModuleIndex]), sizeof(unsigned int), hipMemcpyDeviceToHost);
    nInnerSegments = std::min(nInnerSegments, N_MAX_PIXEL_SEGMENTS_PER_MODULE);

    hipHostMalloc(&nQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMemcpy(nQuintuplets, quintupletsInGPU->nQuintuplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipHostMalloc(&superbins,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int));
    hipHostMalloc(&pixelTypes,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int));

    hipMemcpy(superbins,segmentsInGPU->superbin,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(pixelTypes,segmentsInGPU->pixelType,N_MAX_PIXEL_SEGMENTS_PER_MODULE*sizeof(int),hipMemcpyDeviceToHost);

    unsigned int* connectedPixelSize_host;
    unsigned int* connectedPixelIndex_host;
    hipHostMalloc(&connectedPixelSize_host, nInnerSegments* sizeof(unsigned int));
    hipHostMalloc(&connectedPixelIndex_host, nInnerSegments* sizeof(unsigned int));
    unsigned int* connectedPixelSize_dev;
    unsigned int* connectedPixelIndex_dev;
    hipMalloc(&connectedPixelSize_dev, nInnerSegments* sizeof(unsigned int));
    hipMalloc(&connectedPixelIndex_dev, nInnerSegments* sizeof(unsigned int));
    unsigned int max_size =0;
    int threadSize = 1000000;
    unsigned int *segs_pix = (unsigned int*)malloc(2*threadSize*sizeof(unsigned int));
    unsigned int *segs_pix_offset = segs_pix+threadSize;
    unsigned int *segs_pix_gpu;
    unsigned int *segs_pix_gpu_offset;
    hipMalloc((void **)&segs_pix_gpu, 2*threadSize*sizeof(unsigned int));
    segs_pix_gpu_offset = segs_pix_gpu + threadSize;
    hipMemset(segs_pix_gpu, nInnerSegments, threadSize*sizeof(unsigned int)); // so if not set, it will pass in the kernel
    unsigned int totalSegs=0;
    int pixelIndexOffsetPos = pixelMapping->connectedPixelsIndex[44999] + pixelMapping->connectedPixelsSizes[44999];
    int pixelIndexOffsetNeg = pixelMapping->connectedPixelsIndexPos[44999] + pixelMapping->connectedPixelsSizes[44999] + pixelIndexOffsetPos;

    for (int i = 0; i < nInnerSegments; i++)
    {// loop over # pLS
        int pixelType = pixelTypes[i];// get pixel type for this pLS
        int superbin = superbins[i]; //get superbin for this pixel
        if((superbin < 0) or (superbin >= 45000) or (pixelType > 2) or (pixelType < 0))
        {
            continue;
        }

        if(pixelType ==0)
        { // used pixel type to select correct size-index arrays
            connectedPixelSize_host[i]  = pixelMapping->connectedPixelsSizes[superbin]; //number of connected modules to this pixel
            connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndex[superbin];// index to get start of connected modules for this superbin in map
            for (int j=0; j < pixelMapping->connectedPixelsSizes[superbin]; j++)
            { // loop over modules from the size
                segs_pix[totalSegs+j] = i; // save the pixel index in array to be transfered to kernel
                segs_pix_offset[totalSegs+j] = j; // save this segment in array to be transfered to kernel
            }
            totalSegs += connectedPixelSize_host[i]; // increment counter
        }
        else if(pixelType ==1)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesPos[superbin]; //number of pixel connected modules
            connectedPixelIndex_host[i] = pixelMapping->connectedPixelsIndexPos[superbin]+pixelIndexOffsetPos;// index to get start of connected pixel modules
            for (int j=0; j < pixelMapping->connectedPixelsSizesPos[superbin]; j++)
            {
                segs_pix[totalSegs+j] = i;
                segs_pix_offset[totalSegs+j] = j;
            }
            totalSegs += connectedPixelSize_host[i];
        }
        else if(pixelType ==2)
        {
            connectedPixelSize_host[i] = pixelMapping->connectedPixelsSizesNeg[superbin]; //number of pixel connected modules
            connectedPixelIndex_host[i] =pixelMapping->connectedPixelsIndexNeg[superbin] + pixelIndexOffsetNeg;// index to get start of connected pixel modules
            for (int j=0; j < pixelMapping->connectedPixelsSizesNeg[superbin]; j++)
            {
                segs_pix[totalSegs+j] = i;
                segs_pix_offset[totalSegs+j] = j;
            }
            totalSegs += connectedPixelSize_host[i];
        }
    }

    hipMemcpy(connectedPixelSize_dev, connectedPixelSize_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(connectedPixelIndex_dev, connectedPixelIndex_host, nInnerSegments*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(segs_pix_gpu,segs_pix,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(segs_pix_gpu_offset,segs_pix_offset,threadSize*sizeof(unsigned int), hipMemcpyHostToDevice);

    //less cheap method to estimate max_size for y axis
    max_size = *std::max_element(nQuintuplets, nQuintuplets + nLowerModules);
    dim3 nThreads(16,16,1);
    dim3 nBlocks((totalSegs % nThreads.x == 0 ? totalSegs / nThreads.x : totalSegs / nThreads.x + 1),
                  (max_size % nThreads.y == 0 ? max_size/nThreads.y : max_size/nThreads.y + 1),1);
    createPixelQuintupletsInGPUFromMap<<<nBlocks, nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, *quintupletsInGPU, *pixelQuintupletsInGPU, connectedPixelSize_dev, connectedPixelIndex_dev, nInnerSegments, segs_pix_gpu, segs_pix_gpu_offset, totalSegs);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
	    std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
    hipHostFree(connectedPixelSize_host);
    hipHostFree(connectedPixelIndex_host);
    hipFree(connectedPixelSize_dev);
    hipFree(connectedPixelIndex_dev);
    hipHostFree(superbins);
    hipHostFree(pixelTypes);
    hipHostFree(nQuintuplets);
    free(segs_pix);
    hipFree(segs_pix_gpu);

#else
    printf("original 3D grid launching in createPixelQuintuplets does not exist");
    exit(2);
#endif
#endif
    unsigned int nPixelQuintuplets;
    hipMemcpy(&nPixelQuintuplets, &(pixelQuintupletsInGPU->nPixelQuintuplets), sizeof(unsigned int), hipMemcpyDeviceToHost);
    dim3 nThreads_dup(1024,1,1);
    dim3 nBlocks_dup(64,1,1);
#ifdef DUP_pT5
    printf("run dup pT5\n");
    removeDupPixelQuintupletsInGPUFromMap<<<nBlocks_dup,nThreads_dup>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *pixelTripletsInGPU,*tripletsInGPU, *pixelQuintupletsInGPU, *quintupletsInGPU);
    hipError_t cudaerr2 = hipDeviceSynchronize();
    if(cudaerr2 != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr2)<<std::endl;
    }
#endif
    markUsedObjects<<<nBlocks_dup,nThreads_dup>>>(*modulesInGPU, *segmentsInGPU, *tripletsInGPU, *pixelQuintupletsInGPU, *quintupletsInGPU);
#ifdef Warnings
    std::cout<<"number of pixel quintuplets = "<<nPixelQuintuplets<<std::endl;
#endif   
}


void SDL::Event::createTrackletsWithAGapWithModuleMap()
{
    //use the same trackletsInGPU as before if it exists
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    //TRCAKLETS - To conserve memory, we shall be only declaring nLowerModules amount of memory!!!!!!!
    if(trackletsInGPU == nullptr)
    {
        hipHostMalloc(&trackletsInGPU, sizeof(SDL::tracklets));
#ifdef Explicit_Tracklet
        createTrackletsInExplicitMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
#else
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
#endif
    }

    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTrackletsWithAGapInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }

}


void SDL::Event::addTrackletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(trackletsInGPU->nTracklets[i] == 0)
        {
            modulesInGPU->trackletRanges[idx * 2] = -1;
            modulesInGPU->trackletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->trackletRanges[idx * 2] = idx * N_MAX_TRACKLETS_PER_MODULE;
            modulesInGPU->trackletRanges[idx * 2 + 1] = idx * N_MAX_TRACKLETS_PER_MODULE + trackletsInGPU->nTracklets[i] - 1;


            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_tracklets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += trackletsInGPU->nTracklets[i];
            }
            else
            {
                n_tracklets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += trackletsInGPU->nTracklets[i];
            }
        }
    }
}
void SDL::Event::addTrackletsToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

unsigned int* nTrackletsCPU;
hipHostMalloc(&nTrackletsCPU, nLowerModules * sizeof(unsigned int));
hipMemcpy(nTrackletsCPU,trackletsInGPU->nTracklets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_trackletRanges;
hipHostMalloc(&module_trackletRanges, nModules* 2*sizeof(int));
hipMemcpy(module_trackletRanges,modulesInGPU->trackletRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(nTrackletsCPU[i] == 0)
        {
            module_trackletRanges[idx * 2] = -1;
            module_trackletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_trackletRanges[idx * 2] = idx * N_MAX_TRACKLETS_PER_MODULE;
            module_trackletRanges[idx * 2 + 1] = idx * N_MAX_TRACKLETS_PER_MODULE + nTrackletsCPU[i] - 1;


            if(module_subdets[idx] == Barrel)
            {
                n_tracklets_by_layer_barrel_[module_layers[idx] - 1] += nTrackletsCPU[i];
            }
            else
            {
                n_tracklets_by_layer_endcap_[module_layers[idx] - 1] += nTrackletsCPU[i];
            }
        }
    }
hipHostFree(nTrackletsCPU);
hipHostFree(module_subdets);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_trackletRanges);
hipHostFree(module_layers);
}

void SDL::Event::addTrackCandidatesToEventExplicit()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int* nTrackCandidatesCPU;
    hipHostMalloc(&nTrackCandidatesCPU, (nLowerModules )* sizeof(unsigned int));
    hipMemcpy(nTrackCandidatesCPU,trackCandidatesInGPU->nTrackCandidates,(nLowerModules)*sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    int* module_trackCandidateRanges;
    hipHostMalloc(&module_trackCandidateRanges, nModules* 2*sizeof(int));
    hipMemcpy(module_trackCandidateRanges,modulesInGPU->trackCandidateRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    short* module_layers;
    hipHostMalloc(&module_layers, nModules * sizeof(short));
    hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    short* module_subdets;
    hipHostMalloc(&module_subdets, nModules* sizeof(short));
    hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);

    int* module_trackCandidateModuleIndices;
    hipHostMalloc(&module_trackCandidateModuleIndices, (nLowerModules + 1) * sizeof(int));
    hipMemcpy(module_trackCandidateModuleIndices, modulesInGPU->trackCandidateModuleIndices, sizeof(int) * (nLowerModules + 1), hipMemcpyDeviceToHost);

    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];


        if(nTrackCandidatesCPU[i] == 0)
        {
            module_trackCandidateRanges[idx * 2] = -1;
            module_trackCandidateRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_trackCandidateRanges[idx * 2] = module_trackCandidateModuleIndices[i];
            module_trackCandidateRanges[idx * 2 + 1] = module_trackCandidateModuleIndices[i] + nTrackCandidatesCPU[i] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_trackCandidates_by_layer_barrel_[module_layers[idx] - 1] += nTrackCandidatesCPU[i];
            }
            else
            {
                n_trackCandidates_by_layer_endcap_[module_layers[idx] - 1] += nTrackCandidatesCPU[i];
            }
        }
    }
    hipHostFree(nTrackCandidatesCPU);
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_trackCandidateRanges);
    hipHostFree(module_layers);
    hipHostFree(module_subdets);
    hipHostFree(module_trackCandidateModuleIndices);
}
void SDL::Event::addTrackCandidatesToEvent()
{

    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];


        if(trackCandidatesInGPU->nTrackCandidates[i] == 0 or SDL::modulesInGPU->trackCandidateModuleIndices[i] == -1)
        {
            modulesInGPU->trackCandidateRanges[idx * 2] = -1;
            modulesInGPU->trackCandidateRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->trackCandidateRanges[idx * 2] = SDL::modulesInGPU->trackCandidateModuleIndices[i];
            modulesInGPU->trackCandidateRanges[idx * 2 + 1] = SDL::modulesInGPU->trackCandidateModuleIndices[i] +  trackCandidatesInGPU->nTrackCandidates[i] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_trackCandidates_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += trackCandidatesInGPU->nTrackCandidates[i];
            }
            else
            {
                n_trackCandidates_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += trackCandidatesInGPU->nTrackCandidates[i];
            }
        }
    }
}

void SDL::Event::addQuintupletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(quintupletsInGPU->nQuintuplets[i] == 0)
        {
            modulesInGPU->quintupletRanges[idx * 2] = -1;
            modulesInGPU->quintupletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->quintupletRanges[idx * 2] = SDL::modulesInGPU->quintupletModuleIndices[i];
            modulesInGPU->quintupletRanges[idx * 2 + 1] = SDL::modulesInGPU->quintupletModuleIndices[i] + quintupletsInGPU->nQuintuplets[i] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_quintuplets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += quintupletsInGPU->nQuintuplets[i];
            }
            else
            {
                n_quintuplets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += quintupletsInGPU->nQuintuplets[i];
            }
        }
    }
}

void SDL::Event::addQuintupletsToEventExplicit()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int* nQuintupletsCPU;
    hipHostMalloc(&nQuintupletsCPU, nLowerModules * sizeof(unsigned int));
    hipMemcpy(nQuintupletsCPU,quintupletsInGPU->nQuintuplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

    short* module_subdets;
    hipHostMalloc(&module_subdets, nModules* sizeof(short));
    hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);

    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);

    int* module_quintupletRanges;
    hipHostMalloc(&module_quintupletRanges, nModules* 2*sizeof(int));
    hipMemcpy(module_quintupletRanges,modulesInGPU->quintupletRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    short* module_layers;
    hipHostMalloc(&module_layers, nModules * sizeof(short));
    hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    int* module_quintupletModuleIndices;
    hipHostMalloc(&module_quintupletModuleIndices, nLowerModules * sizeof(int));
    hipMemcpy(module_quintupletModuleIndices, modulesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        if(nQuintupletsCPU[i] == 0 or module_quintupletModuleIndices[i] == -1)
        {
            module_quintupletRanges[idx * 2] = -1;
            module_quintupletRanges[idx * 2 + 1] = -1;
        }
       else
        {
            module_quintupletRanges[idx * 2] = module_quintupletModuleIndices[i];
            module_quintupletRanges[idx * 2 + 1] = module_quintupletModuleIndices[i] + nQuintupletsCPU[i] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_quintuplets_by_layer_barrel_[module_layers[idx] - 1] += nQuintupletsCPU[i];
            }
            else
            {
                n_quintuplets_by_layer_endcap_[module_layers[idx] - 1] += nQuintupletsCPU[i];
            }
        }
    }
    hipHostFree(nQuintupletsCPU);
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_quintupletRanges);
    hipHostFree(module_layers);
    hipHostFree(module_subdets);
    hipHostFree(module_quintupletModuleIndices);

}

void SDL::Event::addTripletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(tripletsInGPU->nTriplets[i] == 0)
        {
            modulesInGPU->tripletRanges[idx * 2] = -1;
            modulesInGPU->tripletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->tripletRanges[idx * 2] = idx * N_MAX_TRIPLETS_PER_MODULE;
            modulesInGPU->tripletRanges[idx * 2 + 1] = idx * N_MAX_TRIPLETS_PER_MODULE + tripletsInGPU->nTriplets[i] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_triplets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += tripletsInGPU->nTriplets[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += tripletsInGPU->nTriplets[i];
            }
        }
    }
}
void SDL::Event::addTripletsToEventExplicit()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int* nTripletsCPU;
    hipHostMalloc(&nTripletsCPU, nLowerModules * sizeof(unsigned int));
    hipMemcpy(nTripletsCPU,tripletsInGPU->nTriplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

    short* module_subdets;
    hipHostMalloc(&module_subdets, nModules* sizeof(short));
    hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    int* module_tripletRanges;
    hipHostMalloc(&module_tripletRanges, nModules* 2*sizeof(int));
    hipMemcpy(module_tripletRanges,modulesInGPU->tripletRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    short* module_layers;
    hipHostMalloc(&module_layers, nModules * sizeof(short));
    hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(nTripletsCPU[i] == 0)
        {
            module_tripletRanges[idx * 2] = -1;
            module_tripletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_tripletRanges[idx * 2] = idx * N_MAX_TRIPLETS_PER_MODULE;
            module_tripletRanges[idx * 2 + 1] = idx * N_MAX_TRIPLETS_PER_MODULE + nTripletsCPU[i] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_triplets_by_layer_barrel_[module_layers[idx] - 1] += nTripletsCPU[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[module_layers[idx] - 1] += nTripletsCPU[i];
            }
        }
    }
    hipHostFree(nTripletsCPU);
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_tripletRanges);
    hipHostFree(module_layers);
    hipHostFree(module_subdets);
}


unsigned int SDL::Event::getNumberOfHits()
{
    unsigned int hits = 0;
    for(auto &it:n_hits_by_layer_barrel_)
    {
        hits += it;
    }
    for(auto& it:n_hits_by_layer_endcap_)
    {
        hits += it;
    }

    return hits;
}

unsigned int SDL::Event::getNumberOfHitsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_hits_by_layer_barrel_[layer];
    else
        return n_hits_by_layer_barrel_[layer] + n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerBarrel(unsigned int layer)
{
    return n_hits_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerEndcap(unsigned int layer)
{
    return n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoublets()
{
     unsigned int miniDoublets = 0;
    for(auto &it:n_minidoublets_by_layer_barrel_)
    {
        miniDoublets += it;
    }
    for(auto &it:n_minidoublets_by_layer_endcap_)
    {
        miniDoublets += it;
    }

    return miniDoublets;

}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_minidoublets_by_layer_barrel_[layer];
    else
        return n_minidoublets_by_layer_barrel_[layer] + n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerBarrel(unsigned int layer)
{
    return n_minidoublets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerEndcap(unsigned int layer)
{
    return n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegments()
{
    unsigned int segments = 0;
    for(auto &it:n_segments_by_layer_barrel_)
    {
        segments += it;
    }
    for(auto &it:n_segments_by_layer_endcap_)
    {
        segments += it;
    }

    return segments;

}

unsigned int SDL::Event::getNumberOfSegmentsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_segments_by_layer_barrel_[layer];
    else
        return n_segments_by_layer_barrel_[layer] + n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerBarrel(unsigned int layer)
{
    return n_segments_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerEndcap(unsigned int layer)
{
    return n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfPixelTracklets()
{
#ifdef Explicit_Tracklet
    unsigned int nLowerModules;// = *(SDL::modulesInGPU->nLowerModules);
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int nTrackletsInPixelModule;
    hipMemcpy(&nTrackletsInPixelModule,pixelTrackletsInGPU->nPixelTracklets,sizeof(unsigned int),hipMemcpyDeviceToHost);
    return nTrackletsInPixelModule;
#else
    return *(pixelTrackletsInGPU->nPixelTracklets);
#endif

}

unsigned int SDL::Event::getNumberOfTracklets()
{
    unsigned int tracklets = 0;
    for(auto &it:n_tracklets_by_layer_barrel_)
    {
        tracklets += it;
    }
    for(auto &it:n_tracklets_by_layer_endcap_)
    {
        tracklets += it;
    }

    return tracklets;

}

unsigned int SDL::Event::getNumberOfTrackletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_tracklets_by_layer_barrel_[layer];
    else
        return n_tracklets_by_layer_barrel_[layer] + n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackletsByLayerBarrel(unsigned int layer)
{
    return n_tracklets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTrackletsByLayerEndcap(unsigned int layer)
{
    return n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTriplets()
{
    unsigned int triplets = 0;
    for(auto &it:n_triplets_by_layer_barrel_)
    {
        triplets += it;
    }
    for(auto &it:n_triplets_by_layer_endcap_)
    {
        triplets += it;
    }

    return triplets;

}


unsigned int SDL::Event::getNumberOfTripletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_triplets_by_layer_barrel_[layer];
    else
        return n_triplets_by_layer_barrel_[layer] + n_triplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerBarrel(unsigned int layer)
{
    return n_triplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerEndcap(unsigned int layer)
{
    return n_triplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfPixelTriplets()
{
#ifdef Explicit_PT3
    unsigned int nPixelTriplets;
    hipMemcpy(&nPixelTriplets, pixelTripletsInGPU->nPixelTriplets, sizeof(unsigned int), hipMemcpyDeviceToHost);
    return nPixelTriplets;
#else
    return *(pixelTripletsInGPU->nPixelTriplets);
#endif
}


unsigned int SDL::Event::getNumberOfPixelQuintuplets()
{
#ifdef Explicit_PT5
    unsigned int nPixelQuintuplets;
    hipMemcpy(&nPixelQuintuplets, pixelQuintupletsInGPU->nPixelQuintuplets, sizeof(unsigned int), hipMemcpyDeviceToHost);
    return nPixelQuintuplets;

#else
    return *(pixelQuintupletsInGPU->nPixelQuintuplets);
#endif
}
unsigned int SDL::Event::getNumberOfQuintuplets()
{
    unsigned int quintuplets = 0;
    for(auto &it:n_quintuplets_by_layer_barrel_)
    {
        quintuplets += it;
    }
    for(auto &it:n_quintuplets_by_layer_endcap_)
    {
        quintuplets += it;
    }

    return quintuplets;

}

unsigned int SDL::Event::getNumberOfQuintupletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_quintuplets_by_layer_barrel_[layer];
    else
        return n_quintuplets_by_layer_barrel_[layer] + n_quintuplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayerBarrel(unsigned int layer)
{
    return n_quintuplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfQuintupletsByLayerEndcap(unsigned int layer)
{
    return n_quintuplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidates()
{
    unsigned int trackCandidates = 0;
    for(auto &it:n_trackCandidates_by_layer_barrel_)
    {
        trackCandidates += it;
    }
    for(auto &it:n_trackCandidates_by_layer_endcap_)
    {
        trackCandidates += it;
    }

    //hack - add pixel track candidate multiplicity
    trackCandidates += getNumberOfPixelTrackCandidates();

    return trackCandidates;

}

unsigned int SDL::Event::getNumberOfPixelTrackCandidates()
{
#ifdef Explicit_Track
    unsigned int nLowerModules;// = *(SDL::modulesInGPU->nLowerModules);
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int nTrackCandidatesInPixelModule;
    hipMemcpy(&nTrackCandidatesInPixelModule,&trackCandidatesInGPU->nTrackCandidates[nLowerModules],sizeof(unsigned int),hipMemcpyDeviceToHost);
    return nTrackCandidatesInPixelModule;
#else
    return trackCandidatesInGPU->nTrackCandidates[*(modulesInGPU->nLowerModules)];
#endif

}
unsigned int SDL::Event::getNumberOfTrackCandidatesByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_trackCandidates_by_layer_barrel_[layer];
    else
        return n_trackCandidates_by_layer_barrel_[layer] + n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidatesByLayerBarrel(unsigned int layer)
{
    return n_trackCandidates_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidatesByLayerEndcap(unsigned int layer)
{
    return n_trackCandidates_by_layer_endcap_[layer];
}

#ifdef Explicit_Hit
SDL::hits* SDL::Event::getHits() //std::shared_ptr should take care of garbage collection
{
    if(hitsInCPU == nullptr)
    {
        hitsInCPU = new SDL::hits;
        hitsInCPU->nHits = new unsigned int;
        unsigned int nHits;
        hipMemcpy(&nHits, hitsInGPU->nHits, sizeof(unsigned int), hipMemcpyDeviceToHost);
        *(hitsInCPU->nHits) = nHits;
        hitsInCPU->idxs = new unsigned int[nHits];
        hitsInCPU->xs = new float[nHits];
        hitsInCPU->ys = new float[nHits];
        hitsInCPU->zs = new float[nHits];
        hitsInCPU->moduleIndices = new unsigned int[nHits];
        hipMemcpy(hitsInCPU->idxs, hitsInGPU->idxs,sizeof(unsigned int) * nHits, hipMemcpyDeviceToHost);
        hipMemcpy(hitsInCPU->xs, hitsInGPU->xs, sizeof(float) * nHits, hipMemcpyDeviceToHost);
        hipMemcpy(hitsInCPU->ys, hitsInGPU->ys, sizeof(float) * nHits, hipMemcpyDeviceToHost);
        hipMemcpy(hitsInCPU->zs, hitsInGPU->zs, sizeof(float) * nHits, hipMemcpyDeviceToHost);
        hipMemcpy(hitsInCPU->moduleIndices, hitsInGPU->moduleIndices, sizeof(unsigned int) * nHits, hipMemcpyDeviceToHost);
    }
    return hitsInCPU;
}
#else
SDL::hits* SDL::Event::getHits() //std::shared_ptr should take care of garbage collection
{
    return hitsInGPU;
}
#endif


#ifdef Explicit_MD
SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    if(mdsInCPU == nullptr)
    {
        mdsInCPU = new SDL::miniDoublets;
        unsigned int nMemoryLocations = (N_MAX_MD_PER_MODULES * (nModules - 1) + N_MAX_PIXEL_MD_PER_MODULES);
        mdsInCPU->hitIndices = new unsigned int[2 * nMemoryLocations];
        mdsInCPU->nMDs = new unsigned int[nModules];
        hipMemcpy(mdsInCPU->hitIndices, mdsInGPU->hitIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(mdsInCPU->nMDs, mdsInGPU->nMDs, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return mdsInCPU;
}
#else
SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    return mdsInGPU;
}
#endif


#ifdef Explicit_Seg
SDL::segments* SDL::Event::getSegments()
{
    if(segmentsInCPU == nullptr)
    {
        segmentsInCPU = new SDL::segments;
        unsigned int nMemoryLocations = (N_MAX_SEGMENTS_PER_MODULE) * (nModules - 1) + N_MAX_PIXEL_SEGMENTS_PER_MODULE;
        segmentsInCPU->mdIndices = new unsigned int[2 * nMemoryLocations];
        segmentsInCPU->nSegments = new unsigned int[nModules];
        segmentsInCPU->innerMiniDoubletAnchorHitIndices = new unsigned int[nMemoryLocations];
        segmentsInCPU->outerMiniDoubletAnchorHitIndices = new unsigned int[nMemoryLocations];
        segmentsInCPU->ptIn = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->eta = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->phi = new float[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        segmentsInCPU->isDup = new bool[N_MAX_PIXEL_SEGMENTS_PER_MODULE];
        hipMemcpy(segmentsInCPU->mdIndices, segmentsInGPU->mdIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->nSegments, segmentsInGPU->nSegments, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->innerMiniDoubletAnchorHitIndices, segmentsInGPU->innerMiniDoubletAnchorHitIndices, nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->outerMiniDoubletAnchorHitIndices, segmentsInGPU->outerMiniDoubletAnchorHitIndices, nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->ptIn, segmentsInGPU->ptIn, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->eta, segmentsInGPU->eta, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->phi, segmentsInGPU->phi, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->isDup, segmentsInGPU->isDup, N_MAX_PIXEL_SEGMENTS_PER_MODULE * sizeof(bool), hipMemcpyDeviceToHost);


    }
    return segmentsInCPU;
}
#else
SDL::segments* SDL::Event::getSegments()
{
    return segmentsInGPU;
}
#endif

#ifdef Explicit_Tracklet
SDL::tracklets* SDL::Event::getTracklets()
{
#ifdef FINAL_T3T4
    if(trackletsInCPU == nullptr)
    {
        unsigned int nLowerModules;
        trackletsInCPU = new SDL::tracklets;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = (N_MAX_TRACKLETS_PER_MODULE) * nLowerModules;
        trackletsInCPU->segmentIndices = new unsigned int[2 * nMemoryLocations];
        trackletsInCPU->nTracklets = new unsigned int[nLowerModules];
        trackletsInCPU->betaIn = new float[nMemoryLocations];
        trackletsInCPU->betaOut = new float[nMemoryLocations];
        trackletsInCPU->pt_beta = new float[nMemoryLocations];
        hipMemcpy(trackletsInCPU->segmentIndices, trackletsInGPU->segmentIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(trackletsInCPU->betaIn, trackletsInGPU->betaIn, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(trackletsInCPU->betaOut, trackletsInGPU->betaOut, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(trackletsInCPU->pt_beta, trackletsInGPU->pt_beta, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(trackletsInCPU->nTracklets, trackletsInGPU->nTracklets, (nLowerModules)* sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
#endif
    return trackletsInCPU;
}

SDL::pixelTracklets* SDL::Event::getPixelTracklets()
{
    if(pixelTrackletsInCPU == nullptr)
    {
        pixelTrackletsInCPU = new SDL::pixelTracklets;
        pixelTrackletsInCPU->segmentIndices = new unsigned int[2 * N_MAX_PIXEL_TRACKLETS_PER_MODULE];
        pixelTrackletsInCPU->nPixelTracklets = new unsigned int;
        pixelTrackletsInCPU->betaIn = new float[N_MAX_PIXEL_TRACKLETS_PER_MODULE];
        pixelTrackletsInCPU->betaOut = new float[N_MAX_PIXEL_TRACKLETS_PER_MODULE];
        pixelTrackletsInCPU->pt_beta = new float[N_MAX_PIXEL_TRACKLETS_PER_MODULE];

        hipMemcpy(pixelTrackletsInCPU->segmentIndices, pixelTrackletsInGPU->segmentIndices, 2 * N_MAX_PIXEL_TRACKLETS_PER_MODULE * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTrackletsInCPU->nPixelTracklets, pixelTrackletsInGPU->nPixelTracklets, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTrackletsInCPU->betaIn, pixelTrackletsInGPU->betaIn, N_MAX_PIXEL_TRACKLETS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTrackletsInCPU->betaOut, pixelTrackletsInGPU->betaOut, N_MAX_PIXEL_TRACKLETS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTrackletsInCPU->pt_beta, pixelTrackletsInGPU->pt_beta, N_MAX_PIXEL_TRACKLETS_PER_MODULE * sizeof(float), hipMemcpyDeviceToHost);
    }
    return pixelTrackletsInCPU;
}

#else
SDL::tracklets* SDL::Event::getTracklets()
{
    return trackletsInGPU;
}

SDL::pixelTracklets* SDL::Event::getPixelTracklets()
{
    return pixelTrackletsInGPU;
}
#endif

#ifdef Explicit_Trips
SDL::triplets* SDL::Event::getTriplets()
{
    if(tripletsInCPU == nullptr)
    {
        unsigned int nLowerModules;
        tripletsInCPU = new SDL::triplets;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = (N_MAX_TRIPLETS_PER_MODULE) * (nLowerModules);
        tripletsInCPU->segmentIndices = new unsigned[2 * nMemoryLocations];
        tripletsInCPU->nTriplets = new unsigned int[nLowerModules];
        tripletsInCPU->betaIn = new float[nMemoryLocations];
        tripletsInCPU->betaOut = new float[nMemoryLocations];
        tripletsInCPU->pt_beta = new float[nMemoryLocations];
        hipMemcpy(tripletsInCPU->segmentIndices, tripletsInGPU->segmentIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(tripletsInCPU->betaIn, tripletsInGPU->betaIn, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(tripletsInCPU->betaOut, tripletsInGPU->betaOut, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(tripletsInCPU->pt_beta, tripletsInGPU->pt_beta, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(tripletsInCPU->nTriplets, tripletsInGPU->nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return tripletsInCPU;
}
#else
SDL::triplets* SDL::Event::getTriplets()
{
    return tripletsInGPU;
}
#endif

#ifdef Explicit_T5
SDL::quintuplets* SDL::Event::getQuintuplets()
{
    if(quintupletsInCPU == nullptr)
    {
        quintupletsInCPU = new SDL::quintuplets;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nEligibleT5Modules;
        hipMemcpy(&nEligibleT5Modules, modulesInGPU->nEligibleT5Modules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = nEligibleT5Modules * N_MAX_QUINTUPLETS_PER_MODULE;

        quintupletsInCPU->nQuintuplets = new unsigned int[nLowerModules];
        quintupletsInCPU->tripletIndices = new unsigned int[2 * nMemoryLocations];
        quintupletsInCPU->lowerModuleIndices = new unsigned int[5 * nMemoryLocations];
        quintupletsInCPU->innerRadius = new float[nMemoryLocations];
        quintupletsInCPU->outerRadius = new float[nMemoryLocations];
        quintupletsInCPU->isDup = new bool[nMemoryLocations];
        quintupletsInCPU->score_rphi = new float[nMemoryLocations];
        quintupletsInCPU->score_rz = new float[nMemoryLocations];
        quintupletsInCPU->score_rzlsq = new float[nMemoryLocations];
        quintupletsInCPU->score_rphisum = new float[nMemoryLocations];
        quintupletsInCPU->eta = new float[nMemoryLocations];
        quintupletsInCPU->phi = new float[nMemoryLocations];
        quintupletsInCPU->regressionRadius = new float[nMemoryLocations];
        hipMemcpy(quintupletsInCPU->nQuintuplets, quintupletsInGPU->nQuintuplets,  nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->tripletIndices, quintupletsInGPU->tripletIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->lowerModuleIndices, quintupletsInGPU->lowerModuleIndices, 5 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->innerRadius, quintupletsInGPU->innerRadius, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->outerRadius, quintupletsInGPU->outerRadius, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->isDup, quintupletsInGPU->isDup, nMemoryLocations * sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->score_rphi, quintupletsInGPU->score_rphi, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->score_rphisum, quintupletsInGPU->score_rphisum, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->score_rz, quintupletsInGPU->score_rz, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->score_rzlsq, quintupletsInGPU->score_rzlsq, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->eta, quintupletsInGPU->eta, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->phi, quintupletsInGPU->phi, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(quintupletsInCPU->regressionRadius, quintupletsInGPU->regressionRadius, nMemoryLocations * sizeof(float), hipMemcpyDeviceToHost);
    }

    return quintupletsInCPU;
}
#else
SDL::quintuplets* SDL::Event::getQuintuplets()
{
    return quintupletsInGPU;
}
#endif

#ifdef Explicit_PT3
SDL::pixelTriplets* SDL::Event::getPixelTriplets()
{
    if(pixelTripletsInCPU == nullptr)
    {
        pixelTripletsInCPU = new SDL::pixelTriplets;

        pixelTripletsInCPU->nPixelTriplets = new unsigned int;
        hipMemcpy(pixelTripletsInCPU->nPixelTriplets, pixelTripletsInGPU->nPixelTriplets, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nPixelTriplets = *(pixelTripletsInCPU->nPixelTriplets);
        pixelTripletsInCPU->tripletIndices = new unsigned int[nPixelTriplets];
        pixelTripletsInCPU->pixelSegmentIndices = new unsigned int[nPixelTriplets];
        pixelTripletsInCPU->pixelRadius = new float[nPixelTriplets];
        pixelTripletsInCPU->pixelRadiusError = new float[nPixelTriplets];
        pixelTripletsInCPU->tripletRadius = new float[nPixelTriplets];
        pixelTripletsInCPU->isDup = new bool[nPixelTriplets];
        pixelTripletsInCPU->eta = new float[nPixelTriplets];
        pixelTripletsInCPU->phi = new float[nPixelTriplets];
        pixelTripletsInCPU->score = new float[nPixelTriplets];

        hipMemcpy(pixelTripletsInCPU->tripletIndices, pixelTripletsInGPU->tripletIndices, nPixelTriplets * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTripletsInCPU->pixelSegmentIndices, pixelTripletsInGPU->pixelSegmentIndices, nPixelTriplets * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTripletsInCPU->pixelRadius, pixelTripletsInGPU->pixelRadius, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTripletsInCPU->tripletRadius, pixelTripletsInGPU->tripletRadius, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTripletsInCPU->isDup, pixelTripletsInGPU->isDup, nPixelTriplets * sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTripletsInCPU->eta, pixelTripletsInGPU->eta, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTripletsInCPU->phi, pixelTripletsInGPU->phi, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(pixelTripletsInCPU->score, pixelTripletsInGPU->score, nPixelTriplets * sizeof(float), hipMemcpyDeviceToHost);
    }
    return pixelTripletsInCPU;
}
#else
SDL::pixelTriplets* SDL::Event::getPixelTriplets()
{
    return pixelTripletsInGPU;
}
#endif

#ifdef Explicit_PT5
SDL::pixelQuintuplets* SDL::Event::getPixelQuintuplets()
{
    if(pixelQuintupletsInCPU == nullptr)
    {
        pixelQuintupletsInCPU = new SDL::pixelQuintuplets;

        pixelQuintupletsInCPU->nPixelQuintuplets = new unsigned int;
        hipMemcpy(pixelQuintupletsInCPU->nPixelQuintuplets, pixelQuintupletsInGPU->nPixelQuintuplets, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nPixelQuintuplets = *(pixelQuintupletsInCPU->nPixelQuintuplets);

        pixelQuintupletsInCPU->pixelIndices = new unsigned int[nPixelQuintuplets];
        pixelQuintupletsInCPU->T5Indices = new unsigned int[nPixelQuintuplets];
        pixelQuintupletsInCPU->isDup = new bool[nPixelQuintuplets];

        hipMemcpy(pixelQuintupletsInCPU->pixelIndices, pixelQuintupletsInGPU->pixelIndices, nPixelQuintuplets * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(pixelQuintupletsInCPU->T5Indices, pixelQuintupletsInGPU->T5Indices, nPixelQuintuplets * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(pixelQuintupletsInCPU->isDup, pixelQuintupletsInGPU->isDup, nPixelQuintuplets * sizeof(bool), hipMemcpyDeviceToHost);
    }
    return pixelQuintupletsInCPU;
}
#else
SDL::pixelQuintuplets* SDL::Event::getPixelQuintuplets()
{
    return pixelQuintupletsInGPU;
}
#endif

#ifdef Explicit_Track
SDL::trackCandidates* SDL::Event::getTrackCandidates()
{
    if(trackCandidatesInCPU == nullptr)
    {
        trackCandidatesInCPU = new SDL::trackCandidates;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nEligibleModules;
        hipMemcpy(&nEligibleModules, modulesInGPU->nEligibleModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = (N_MAX_TRACK_CANDIDATES_PER_MODULE) * (nEligibleModules -1) + (N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE);

        trackCandidatesInCPU->objectIndices = new unsigned int[2 * nMemoryLocations];
        trackCandidatesInCPU->trackCandidateType = new short[nMemoryLocations];
        trackCandidatesInCPU->nTrackCandidates = new unsigned int[nLowerModules+1];
        hipMemcpy(trackCandidatesInCPU->objectIndices, trackCandidatesInGPU->objectIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(trackCandidatesInCPU->trackCandidateType, trackCandidatesInGPU->trackCandidateType, nMemoryLocations * sizeof(short), hipMemcpyDeviceToHost);
        hipMemcpy(trackCandidatesInCPU->nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, (nLowerModules + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return trackCandidatesInCPU;
}
#else
SDL::trackCandidates* SDL::Event::getTrackCandidates()
{
    return trackCandidatesInGPU;
}
#endif
#ifdef Explicit_Module
SDL::modules* SDL::Event::getFullModules()
{
    if(modulesInCPUFull == nullptr)
    {
        modulesInCPUFull = new SDL::modules;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);

    modulesInCPUFull->detIds = new unsigned int[nModules];
    modulesInCPUFull->moduleMap = new unsigned int[40*nModules];
    modulesInCPUFull->nConnectedModules = new unsigned int[nModules];
    modulesInCPUFull->drdzs = new float[nModules];
    modulesInCPUFull->slopes = new float[nModules];
    modulesInCPUFull->nModules = new unsigned int[1];
    modulesInCPUFull->nLowerModules = new unsigned int[1];
    modulesInCPUFull->layers = new short[nModules];
    modulesInCPUFull->rings = new short[nModules];
    modulesInCPUFull->modules = new short[nModules];
    modulesInCPUFull->rods = new short[nModules];
    modulesInCPUFull->subdets = new short[nModules];
    modulesInCPUFull->sides = new short[nModules];
    modulesInCPUFull->isInverted = new bool[nModules];
    modulesInCPUFull->isLower = new bool[nModules];

    modulesInCPUFull->hitRanges = new int[2*nModules];
    modulesInCPUFull->mdRanges = new int[2*nModules];
    modulesInCPUFull->segmentRanges = new int[2*nModules];
    modulesInCPUFull->trackletRanges = new int[2*nModules];
    modulesInCPUFull->tripletRanges = new int[2*nModules];
    modulesInCPUFull->trackCandidateRanges = new int[2*nModules];

    modulesInCPUFull->moduleType = new ModuleType[nModules];
    modulesInCPUFull->moduleLayerType = new ModuleLayerType[nModules];

    modulesInCPUFull->lowerModuleIndices = new unsigned int[nLowerModules+1];
    modulesInCPUFull->reverseLookupLowerModuleIndices = new int[nModules];
    modulesInCPUFull->trackCandidateModuleIndices = new int[nLowerModules+1];
    modulesInCPUFull->quintupletModuleIndices = new int[nLowerModules];

    hipMemcpy(modulesInCPUFull->detIds,modulesInGPU->detIds,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->moduleMap,modulesInGPU->moduleMap,40*nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->drdzs,modulesInGPU->drdzs,sizeof(float)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->slopes,modulesInGPU->slopes,sizeof(float)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->rings,modulesInGPU->rings,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->modules,modulesInGPU->modules,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->rods,modulesInGPU->rods,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->subdets,modulesInGPU->subdets,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->sides,modulesInGPU->sides,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->isInverted,modulesInGPU->isInverted,sizeof(bool)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->isLower,modulesInGPU->isLower,sizeof(bool)*nModules,hipMemcpyDeviceToHost);

    hipMemcpy(modulesInCPUFull->hitRanges, modulesInGPU->hitRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->mdRanges, modulesInGPU->mdRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->segmentRanges, modulesInGPU->segmentRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->trackletRanges, modulesInGPU->trackletRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->tripletRanges, modulesInGPU->tripletRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->trackCandidateRanges, modulesInGPU->trackCandidateRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(modulesInCPUFull->reverseLookupLowerModuleIndices, modulesInGPU->reverseLookupLowerModuleIndices, nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->lowerModuleIndices, modulesInGPU->lowerModuleIndices, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->trackCandidateModuleIndices, modulesInGPU->trackCandidateModuleIndices, (nLowerModules+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->quintupletModuleIndices, modulesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(modulesInCPUFull->moduleType,modulesInGPU->moduleType,sizeof(ModuleType)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->moduleLayerType,modulesInGPU->moduleLayerType,sizeof(ModuleLayerType)*nModules,hipMemcpyDeviceToHost);
    }
    return modulesInCPUFull;
}
SDL::modules* SDL::Event::getModules()
{
    //if(modulesInCPU == nullptr)
    //{
        modulesInCPU = new SDL::modules;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        modulesInCPU->nLowerModules = new unsigned int[1];
        modulesInCPU->nModules = new unsigned int[1];
        modulesInCPU->lowerModuleIndices = new unsigned int[nLowerModules+1];
        modulesInCPU->detIds = new unsigned int[nModules];
        modulesInCPU->hitRanges = new int[2*nModules];
        modulesInCPU->isLower = new bool[nModules];
        modulesInCPU->trackCandidateModuleIndices = new int[nLowerModules+1];
        modulesInCPU->quintupletModuleIndices = new int[nLowerModules];
        modulesInCPU->layers = new short[nModules];
        modulesInCPU->subdets = new short[nModules];
        modulesInCPU->rings = new short[nModules];


        hipMemcpy(modulesInCPU->nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->nModules, modulesInGPU->nModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->lowerModuleIndices, modulesInGPU->lowerModuleIndices, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->detIds, modulesInGPU->detIds, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->hitRanges, modulesInGPU->hitRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->isLower, modulesInGPU->isLower, nModules * sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->trackCandidateModuleIndices, modulesInGPU->trackCandidateModuleIndices, (nLowerModules+1) * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->quintupletModuleIndices, modulesInGPU->quintupletModuleIndices, nLowerModules * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->layers, modulesInGPU->layers, nModules * sizeof(short), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->subdets, modulesInGPU->subdets, nModules * sizeof(short), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->rings, modulesInGPU->rings, nModules * sizeof(short), hipMemcpyDeviceToHost);
    //}
    return modulesInCPU;
}
#else
SDL::modules* SDL::Event::getModules()
{
    return modulesInGPU;
}
SDL::modules* SDL::Event::getFullModules()
{
    return modulesInGPU;
}
#endif

