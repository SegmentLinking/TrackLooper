#include "hip/hip_runtime.h"
# include "Event.cuh"
#include "allocate.h"


unsigned int N_MAX_HITS_PER_MODULE = 100;
const unsigned int N_MAX_MD_PER_MODULES = 100;
const unsigned int N_MAX_SEGMENTS_PER_MODULE = 600; //WHY!
const unsigned int MAX_CONNECTED_MODULES = 40;
const unsigned int N_MAX_TRACKLETS_PER_MODULE = 8000;//temporary
const unsigned int N_MAX_TRIPLETS_PER_MODULE = 5000;
const unsigned int N_MAX_TRACK_CANDIDATES_PER_MODULE = 50000;
const unsigned int N_MAX_PIXEL_MD_PER_MODULES = 100000;
const unsigned int N_MAX_PIXEL_SEGMENTS_PER_MODULE = 50000;
const unsigned int N_MAX_PIXEL_TRACKLETS_PER_MODULE = 3000000;
const unsigned int N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE = 5000000;


struct SDL::modules* SDL::modulesInGPU = nullptr;
unsigned int SDL::nModules;

SDL::Event::Event()
{
    hitsInGPU = nullptr;
    mdsInGPU = nullptr;
    segmentsInGPU = nullptr;
    trackletsInGPU = nullptr;
    tripletsInGPU = nullptr;
    trackCandidatesInGPU = nullptr;


    hitsInCPU = nullptr;
    mdsInCPU = nullptr;
    segmentsInCPU = nullptr;
    trackletsInCPU = nullptr;
    tripletsInCPU = nullptr;
    trackCandidatesInCPU = nullptr;
    modulesInCPU = nullptr;
    modulesInCPUFull = nullptr;
    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_tracklets_by_layer_barrel_[i] = 0;
        n_triplets_by_layer_barrel_[i] = 0;
        n_trackCandidates_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
            n_segments_by_layer_endcap_[i] = 0;
            n_tracklets_by_layer_endcap_[i] = 0;
            n_triplets_by_layer_endcap_[i] = 0;
            n_trackCandidates_by_layer_endcap_[i] = 0;
        }
    }
    resetObjectsInModule();

}

SDL::Event::~Event()
{

#ifdef CACHE_ALLOC
    mdsInGPU->freeMemoryCache();
    segmentsInGPU->freeMemoryCache();
    tripletsInGPU->freeMemoryCache();
    trackletsInGPU->freeMemoryCache();
    trackCandidatesInGPU->freeMemoryCache();
#else
    mdsInGPU->freeMemory();
    segmentsInGPU->freeMemory();
    tripletsInGPU->freeMemory();
    trackletsInGPU->freeMemory();
    trackCandidatesInGPU->freeMemory();
#endif
    hipHostFree(mdsInGPU);
    hipHostFree(segmentsInGPU);
    hipHostFree(tripletsInGPU);
    hipHostFree(trackletsInGPU);
    hipHostFree(trackCandidatesInGPU);
    hitsInGPU->freeMemory();
    hipHostFree(hitsInGPU);

#ifdef Explicit_Hit
    if(hitsInCPU != nullptr)
    {
        delete[] hitsInCPU->idxs;
        delete hitsInCPU->nHits;
        delete hitsInCPU;
    }
#endif
#ifdef Explicit_MD
    if(mdsInCPU != nullptr)
    {
        delete[] mdsInCPU->hitIndices;
        delete[] mdsInCPU->nMDs;
        delete mdsInCPU;
    }
#endif
#ifdef Explicit_Seg
    if(segmentsInCPU != nullptr)
    {
        delete[] segmentsInCPU->mdIndices;
        delete[] segmentsInCPU->nSegments;
        delete segmentsInCPU;
    }
#endif
#ifdef Explicit_Tracklet
    if(trackletsInCPU != nullptr)
    {
        delete[] trackletsInCPU->segmentIndices;
        delete[] trackletsInCPU->nTracklets;
        delete trackletsInCPU;
    }
#endif
#ifdef Explicit_Trips
    if(tripletsInCPU != nullptr)
    {
        delete[] tripletsInCPU->segmentIndices;
        delete[] tripletsInCPU->nTriplets;
        delete tripletsInCPU;
    }
#endif
#ifdef Explicit_Track
    if(trackCandidatesInCPU != nullptr)
    {
        delete[] trackCandidatesInCPU->objectIndices;
        delete[] trackCandidatesInCPU->trackCandidateType;
        delete[] trackCandidatesInCPU->nTrackCandidates;
        delete trackCandidatesInCPU;
    }
#endif
#ifdef Explicit_Module
    if(modulesInCPU != nullptr)
    {
        delete[] modulesInCPU->nLowerModules;
        delete[] modulesInCPU->nModules;
        delete[] modulesInCPU->lowerModuleIndices;
        delete[] modulesInCPU->detIds;
        delete[] modulesInCPU->hitRanges;
        delete[] modulesInCPU->isLower;
        delete[] modulesInCPU->trackCandidateModuleIndices;
        delete[] modulesInCPU;
    }
    if(modulesInCPUFull != nullptr)
    {
        delete[] modulesInCPUFull->detIds;
        delete[] modulesInCPUFull->moduleMap;
        delete[] modulesInCPUFull->nConnectedModules;
        delete[] modulesInCPUFull->drdzs;
        delete[] modulesInCPUFull->slopes;
        delete[] modulesInCPUFull->nModules;
        delete[] modulesInCPUFull->nLowerModules;
        delete[] modulesInCPUFull->layers;
        delete[] modulesInCPUFull->rings;
        delete[] modulesInCPUFull->modules;
        delete[] modulesInCPUFull->rods;
        delete[] modulesInCPUFull->subdets;
        delete[] modulesInCPUFull->sides;
        delete[] modulesInCPUFull->isInverted;
        delete[] modulesInCPUFull->isLower;

        delete[] modulesInCPUFull->hitRanges;
        delete[] modulesInCPUFull->mdRanges;
        delete[] modulesInCPUFull->segmentRanges;
        delete[] modulesInCPUFull->trackletRanges;
        delete[] modulesInCPUFull->tripletRanges;
        delete[] modulesInCPUFull->trackCandidateRanges;

        delete[] modulesInCPUFull->moduleType;
        delete[] modulesInCPUFull->moduleLayerType;

        delete[] modulesInCPUFull->lowerModuleIndices;
        delete[] modulesInCPUFull->reverseLookupLowerModuleIndices;
        delete[] modulesInCPUFull->trackCandidateModuleIndices;
        delete[] modulesInCPUFull;
    }
#endif
}

void SDL::initModules(const char* moduleMetaDataFilePath)
{
    if(modulesInGPU == nullptr)
    {
        hipHostMalloc(&modulesInGPU, sizeof(struct SDL::modules));
        loadModulesFromFile(*modulesInGPU,nModules,moduleMetaDataFilePath); //nModules gets filled here
    }
    resetObjectRanges(*modulesInGPU,nModules);
}

void SDL::cleanModules()
{
  #ifdef CACHE_ALLOC
  freeModulesCache(*modulesInGPU);
  #else
  freeModules(*modulesInGPU);
  #endif
  hipHostFree(modulesInGPU);
}

void SDL::Event::resetObjectsInModule()
{
    resetObjectRanges(*modulesInGPU,nModules);
}
// add hits via kernel method.
void SDL::Event::addHitToEventGPU(std::vector<float> x, std::vector<float> y, std::vector<float> z, std::vector<unsigned int> detId)
{

    const int loopsize = x.size();
    if(hitsInGPU == nullptr)
    {

        hipHostMalloc(&hitsInGPU, sizeof(SDL::hits));
    	  createHitsInExplicitMemory(*hitsInGPU, 2*loopsize);
    }

    //calls the addHitToMemory function
    ////Explicit
    unsigned int nThreads = 256;
    unsigned int nBlocks =  loopsize % nThreads == 0 ? loopsize/nThreads : loopsize/nThreads + 1;

    float* dev_x;
    float* dev_y;
    float* dev_z;
    float* dev_phi;
    float* host_x = &x[0];
    float* host_y = &y[0];
    float* host_z = &z[0];
    float* host_phi;
    float* host_highEdgeXs;
    float* host_highEdgeYs;
    float* host_lowEdgeXs;
    float* host_lowEdgeYs;
    hipHostMalloc(&host_highEdgeXs,sizeof(float)*loopsize);
    hipHostMalloc(&host_highEdgeYs,sizeof(float)*loopsize);
    hipHostMalloc(&host_lowEdgeXs,sizeof(float)*loopsize);
    hipHostMalloc(&host_lowEdgeYs,sizeof(float)*loopsize);
    unsigned int* host_detId = &detId[0];
    unsigned int* host_moduleIndex;
    unsigned int* dev_moduleIndex;
    hipMalloc(&dev_x,loopsize*sizeof(float));
    hipMalloc(&dev_y,loopsize*sizeof(float));
    hipMalloc(&dev_z,loopsize*sizeof(float));
    hipMalloc(&dev_moduleIndex,sizeof(unsigned int)*loopsize);
    hipHostMalloc(&host_moduleIndex,sizeof(unsigned int)*loopsize);
    hipMalloc(&dev_phi,sizeof(float)*loopsize);
    hipHostMalloc(&host_phi,sizeof(float)*loopsize);
  for (int ihit=0; ihit<loopsize;ihit++){
    unsigned int moduleLayer = modulesInGPU->layers[(*detIdToIndex)[host_detId[ihit]]]; // I think detIdToIndex needs to be handled on host. this can be run in parallel otherwise
    unsigned int subdet = modulesInGPU->subdets[(*detIdToIndex)[host_detId[ihit]]];
    host_moduleIndex[ihit] = (*detIdToIndex)[host_detId[ihit]];
    host_phi[ihit] = endcapGeometry.getCentroidPhi(host_detId[ihit]);

    if(subdet == Barrel)
    {
        n_hits_by_layer_barrel_[moduleLayer-1]++;
    }
    else
    {
        n_hits_by_layer_endcap_[moduleLayer-1]++;
    }
      unsigned int this_index = host_moduleIndex[ihit];
      if(modulesInGPU->subdets[this_index] == Endcap && modulesInGPU->moduleType[this_index] == TwoS) // cannot be run in parallel
      {
          float xhigh, yhigh, xlow, ylow;
          getEdgeHits(host_detId[ihit],host_x[ihit],host_y[ihit],xhigh,yhigh,xlow,ylow);
          host_highEdgeXs[ihit] = xhigh;
          host_highEdgeYs[ihit] = yhigh;
          host_lowEdgeXs[ihit] = xlow;
          host_lowEdgeYs[ihit] = ylow;

      }

      //set the hit ranges appropriately in the modules struct

      //start the index rolling if the module is encountered for the first time
      if(modulesInGPU->hitRanges[this_index * 2] == -1) // cannot be run in parallel
      {
          modulesInGPU->hitRanges[this_index * 2] = ihit;
      }
      //always update the end index
      modulesInGPU->hitRanges[this_index * 2 + 1] = ihit;
  }
    hipMemcpy(dev_x,host_x,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_y,host_y,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_z,host_z,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(dev_moduleIndex,host_moduleIndex,loopsize*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(dev_phi,host_phi,loopsize*sizeof(float),hipMemcpyHostToDevice);
    //hipDeviceSynchronize();
    addHitToMemoryKernel<<<nBlocks,nThreads>>>(*hitsInGPU, *modulesInGPU, dev_x, dev_y, dev_z, dev_moduleIndex,dev_phi,loopsize);
    hipMemcpy(hitsInGPU->highEdgeXs,host_highEdgeXs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->highEdgeYs,host_highEdgeYs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->lowEdgeXs,host_lowEdgeXs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->lowEdgeYs,host_lowEdgeYs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->nHits,&loopsize,sizeof(unsigned int),hipMemcpyHostToDevice);// value can't correctly be set in hit allocation
    hipDeviceSynchronize();
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_z);
    hipFree(dev_moduleIndex);
    hipFree(dev_phi);
    hipHostFree(host_phi);
    hipHostFree(host_moduleIndex);
    hipHostFree(host_highEdgeXs);
    hipHostFree(host_highEdgeYs);
    hipHostFree(host_lowEdgeXs);
    hipHostFree(host_lowEdgeYs);

}
//explicit method using omp
void SDL::Event::addHitToEventOMP(std::vector<float> x, std::vector<float> y, std::vector<float> z, std::vector<unsigned int> detId, std::vector<unsigned int> idxInNtuple)
{
    const int loopsize = x.size();// use the actual number of hits instead of a "max"

    if(hitsInGPU == nullptr)
    {

        hipHostMalloc(&hitsInGPU, sizeof(SDL::hits));
        #ifdef Explicit_Hit
    	  createHitsInExplicitMemory(*hitsInGPU, 2*loopsize); //unclear why but this has to be 2*loopsize to avoid crashing later (reported in tracklet allocation). seems to do with nHits values as well. this allows nhits to be set to the correct value of loopsize to get correct results without crashing. still beats the "max hits" so i think this is fine.
        #else
        createHitsInUnifiedMemory(*hitsInGPU,2*loopsize,0);
        #endif
    }


    float* host_x = &x[0]; // convert from std::vector to host array easily since vectors are ordered
    float* host_y = &y[0];
    float* host_z = &z[0];
    float* host_phis;
    unsigned int* host_detId = &detId[0];
    unsigned int* host_idxs = &idxInNtuple[0];
    unsigned int* host_moduleIndex;
    float* host_rts;
    //float* host_idxs;
    float* host_highEdgeXs;
    float* host_highEdgeYs;
    float* host_lowEdgeXs;
    float* host_lowEdgeYs;
    hipHostMalloc(&host_moduleIndex,sizeof(unsigned int)*loopsize);
    hipHostMalloc(&host_phis,sizeof(float)*loopsize);
    hipHostMalloc(&host_rts,sizeof(float)*loopsize);
    //hipHostMalloc(&host_idxs,sizeof(unsigned int)*loopsize);
    hipHostMalloc(&host_highEdgeXs,sizeof(float)*loopsize);
    hipHostMalloc(&host_highEdgeYs,sizeof(float)*loopsize);
    hipHostMalloc(&host_lowEdgeXs,sizeof(float)*loopsize);
    hipHostMalloc(&host_lowEdgeYs,sizeof(float)*loopsize);


    short* module_layers;
    short* module_subdet;
    int* module_hitRanges;
    ModuleType* module_moduleType;
    hipHostMalloc(&module_layers,sizeof(short)*nModules);
    hipHostMalloc(&module_subdet,sizeof(short)*nModules);
    hipHostMalloc(&module_hitRanges,sizeof(int)*2*nModules);
    hipHostMalloc(&module_moduleType,sizeof(ModuleType)*nModules);
    hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    hipMemcpy(module_subdet,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
    hipMemcpy(module_hitRanges,modulesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(module_moduleType,modulesInGPU->moduleType,nModules*sizeof(ModuleType),hipMemcpyDeviceToHost);


//#pragma omp parallel for  // this part can be run in parallel.
  for (int ihit=0; ihit<loopsize;ihit++){
    unsigned int moduleLayer = module_layers[(*detIdToIndex)[host_detId[ihit]]];
    unsigned int subdet = module_subdet[(*detIdToIndex)[host_detId[ihit]]];
    //unsigned int moduleLayer = modulesInGPU->layers[(*detIdToIndex)[host_detId[ihit]]];
    //unsigned int subdet = modulesInGPU->subdets[(*detIdToIndex)[host_detId[ihit]]];
    host_moduleIndex[ihit] = (*detIdToIndex)[host_detId[ihit]];

//    if(subdet == Barrel) // this doesn't seem useful anymore
//    {
//        n_hits_by_layer_barrel_[moduleLayer-1]++;
//    }
//    else
//    {
//        n_hits_by_layer_endcap_[moduleLayer-1]++;
//    }


      host_rts[ihit] = sqrt(host_x[ihit]*host_x[ihit] + host_y[ihit]*host_y[ihit]);
      host_phis[ihit] = phi(host_x[ihit],host_y[ihit],host_z[ihit]);
      //host_idxs[ihit] = ihit;
//  }
//// This part i think has a race condition. so this is not run in parallel.
////#pragma omp parallel for
//  for (int ihit=0; ihit<loopsize;ihit++){
      unsigned int this_index = host_moduleIndex[ihit];
      //if(modulesInGPU->subdets[this_index] == Endcap && modulesInGPU->moduleType[this_index] == TwoS)
      if(module_subdet[this_index] == Endcap && module_moduleType[this_index] == TwoS)
      {
          float xhigh, yhigh, xlow, ylow;
          getEdgeHits(host_detId[ihit],host_x[ihit],host_y[ihit],xhigh,yhigh,xlow,ylow);
          host_highEdgeXs[ihit] = xhigh;
          host_highEdgeYs[ihit] = yhigh;
          host_lowEdgeXs[ihit] = xlow;
          host_lowEdgeYs[ihit] = ylow;

      }

      //set the hit ranges appropriately in the modules struct

      ////start the index rolling if the module is encountered for the first time
      //if(modulesInGPU->hitRanges[this_index * 2] == -1)
      //{
      //    modulesInGPU->hitRanges[this_index * 2] = ihit;
      //}
      ////always update the end index
      //modulesInGPU->hitRanges[this_index * 2 + 1] = ihit;
      //start the index rolling if the module is encountered for the first time
      if(module_hitRanges[this_index * 2] == -1)
      {
          module_hitRanges[this_index * 2] = ihit;
      }
      //always update the end index
      module_hitRanges[this_index * 2 + 1] = ihit;

  }
//simply copy the host arrays to the hitsInGPU struct
    hipMemcpy(hitsInGPU->xs,host_x,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->ys,host_y,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->zs,host_z,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->rts,host_rts,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->idxs,host_idxs,loopsize*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->moduleIndices,host_moduleIndex,loopsize*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->highEdgeXs,host_highEdgeXs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->highEdgeYs,host_highEdgeYs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->lowEdgeXs,host_lowEdgeXs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->lowEdgeYs,host_lowEdgeYs,loopsize*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(hitsInGPU->nHits,&loopsize,sizeof(unsigned int),hipMemcpyHostToDevice);// value can't correctly be set in hit allocation
    hipMemcpy(modulesInGPU->hitRanges,module_hitRanges,nModules*2*sizeof(int),hipMemcpyHostToDevice);// value can't correctly be set in hit allocation
    hipDeviceSynchronize(); //doesn't seem to make a difference

    hipHostFree(host_rts);
    //hipHostFree(host_idxs);
    hipHostFree(host_phis);
    hipHostFree(host_moduleIndex);
    hipHostFree(host_highEdgeXs);
    hipHostFree(host_highEdgeYs);
    hipHostFree(host_lowEdgeXs);
    hipHostFree(host_lowEdgeYs);
    hipHostFree(module_layers);
    hipHostFree(module_subdet);
    hipHostFree(module_hitRanges);
    hipHostFree(module_moduleType);

}
// old method using unified memory
void SDL::Event::addHitToEvent(float x, float y, float z, unsigned int detId, unsigned int idx)
{
    const int HIT_MAX = 1000000;
    const int HIT_2S_MAX = 100000;

    if(hitsInGPU == nullptr)
    {

        hipHostMalloc(&hitsInGPU, sizeof(SDL::hits));
        #ifdef Explicit_Hit
        createHitsInExplicitMemory(*hitsInGPU,HIT_MAX);
        #else
        createHitsInUnifiedMemory(*hitsInGPU,HIT_MAX,HIT_2S_MAX);
        #endif
    }

    //calls the addHitToMemory function
    addHitToMemory(*hitsInGPU, *modulesInGPU, x, y, z, detId, idx);

    //short* module_layers;
    //short* module_subdet;
    //hipHostMalloc(&module_layers,sizeof(short)*nModules);
    //hipHostMalloc(&module_subdet,sizeof(short)*nModules);
    //hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    //hipMemcpy(module_subdet,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
    //unsigned int moduleLayer = module_layers[(*detIdToIndex)[detId]];
    //unsigned int subdet = module_subdet[(*detIdToIndex)[detId]];

    //if(subdet == Barrel)
    //{
    //    n_hits_by_layer_barrel_[moduleLayer-1]++;
    //}
    //else if(subdet == Endcap)
    //{
    //    n_hits_by_layer_endcap_[moduleLayer-1]++;
    //}

}
void /*unsigned int*/ SDL::Event::addPixToEvent(float x, float y, float z, unsigned int detId, unsigned int idx)
{
    const int HIT_MAX = 1000000;
    const int HIT_2S_MAX = 100000;

    if(hitsInGPU == nullptr)
    {

        hipHostMalloc(&hitsInGPU, sizeof(SDL::hits));
        #ifdef Explicit_Hit
        createHitsInExplicitMemory(*hitsInGPU,HIT_MAX);
        #else
        createHitsInUnifiedMemory(*hitsInGPU,HIT_MAX,HIT_2S_MAX);
        #endif
    }

    //calls the addHitToMemory function
    unsigned int moduleIndex = (*detIdToIndex)[detId];
    float phis = phi(x,y,z);
    addHitToMemoryGPU<<<1,1>>>(*hitsInGPU, *modulesInGPU, x, y, z, detId, idx, moduleIndex,phis);
    //addHitToMemory(*hitsInGPU, *modulesInGPU, x, y, z, detId, idx);

    unsigned int moduleLayer = modulesInGPU->layers[(*detIdToIndex)[detId]];
    unsigned int subdet = modulesInGPU->subdets[(*detIdToIndex)[detId]];

    if(subdet == Barrel)
    {
        n_hits_by_layer_barrel_[moduleLayer-1]++;
    }
    else if(subdet == Endcap)
    {
        n_hits_by_layer_endcap_[moduleLayer-1]++;
    }
//    unsigned int* hitIdx;
//    hipHostMalloc(&hitIdx,sizeof(unsigned int));
//    hipMemcpy(&hitIdx,hitsInGPU->nHits,sizeof(unsigned int),hipMemcpyDeviceToHost);
//    printf("hit %u\n",hitIdx[0]);
//   // unsigned int dummy = *hitIdx;//so i can return the value and still free?
//    //hipHostFree(hitIdx);
//    return 0;
}

__global__ void /*SDL::Event::*/addPixelSegmentToEventKernel(unsigned int* hitIndices, float dPhiChange, float ptIn, float ptErr, float px, float py, float pz, float etaErr,unsigned int pixelModuleIndex, struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{

    //step 1 : Add pixel MDs
    unsigned int innerMDIndex = pixelModuleIndex * N_MAX_MD_PER_MODULES + mdsInGPU.nMDs[pixelModuleIndex];

    //FIXME:Fake Pixel MDs are being added to MD unified memory!
#ifdef CUT_VALUE_DEBUG
    addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices[0], hitIndices[1], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,innerMDIndex);

#else
    addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices[0], hitIndices[1], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,innerMDIndex);
#endif
    mdsInGPU.nMDs[pixelModuleIndex]++;
    unsigned int outerMDIndex = pixelModuleIndex * N_MAX_MD_PER_MODULES + mdsInGPU.nMDs[pixelModuleIndex];
#ifdef CUT_VALUE_DEBUG
    addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices[2], hitIndices[3], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,outerMDIndex);
#else
    addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices[2], hitIndices[3], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,outerMDIndex);
#endif
    mdsInGPU.nMDs[pixelModuleIndex]++;

    //step 2 : Add pixel segment
    unsigned int pixelSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentsInGPU.nSegments[pixelModuleIndex];
    //FIXME:Fake Pixel Segment gets added to Segment unified memory in a convoluted fashion!
#ifdef CUT_VALUE_DEBUG
#else
    addPixelSegmentToMemory(segmentsInGPU, mdsInGPU, hitsInGPU, modulesInGPU, innerMDIndex, outerMDIndex, pixelModuleIndex, hitIndices[0], hitIndices[2], dPhiChange, ptIn, ptErr, px, py, pz, etaErr, pixelSegmentIndex, segmentsInGPU.nSegments[pixelModuleIndex]);
#endif
    segmentsInGPU.nSegments[pixelModuleIndex]++;
}
void SDL::Event::addPixelSegmentToEvent(std::vector<unsigned int> hitIndices, float dPhiChange, float ptIn, float ptErr, float px, float py, float pz, float etaErr)
{
    assert(hitIndices.size() == 4);
    unsigned int pixelModuleIndex = (*detIdToIndex)[1] -1;
  unsigned int* hitIndices_host = &hitIndices[0];
  unsigned int * hitIndices_dev;
  hipMalloc(&hitIndices_dev,4*sizeof(unsigned int));
  hipMemcpy(hitIndices_dev,hitIndices_host,4*sizeof(unsigned int),hipMemcpyHostToDevice);

  addPixelSegmentToEventKernel<<<1,1>>>(hitIndices_dev,dPhiChange,ptIn,ptErr,px,py,pz,etaErr,pixelModuleIndex, *modulesInGPU,*hitsInGPU,*mdsInGPU,*segmentsInGPU);
  hipDeviceSynchronize();
  hipFree(hitIndices_dev);
}
__global__ void addPixelSegmentToEventKernelV2(unsigned int* hitIndices0,unsigned int* hitIndices1,unsigned int* hitIndices2,unsigned int* hitIndices3, float* dPhiChange, float* ptIn, float* ptErr, float* px, float* py, float* pz, float* etaErr,unsigned int pixelModuleIndex, struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU,const int size)
{

    for( int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < size; tid += blockDim.x*gridDim.x)
    {

      unsigned int innerMDIndex = pixelModuleIndex * N_MAX_MD_PER_MODULES + 2*(tid);
      unsigned int outerMDIndex = pixelModuleIndex * N_MAX_MD_PER_MODULES + 2*(tid) +1;
      unsigned int pixelSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + tid;

#ifdef CUT_VALUE_DEBUG
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,0,0,0,0,outerMDIndex);
#else
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices0[tid], hitIndices1[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,innerMDIndex);
      addMDToMemory(mdsInGPU, hitsInGPU, modulesInGPU, hitIndices2[tid], hitIndices3[tid], pixelModuleIndex, 0,0,0,0,0,0,0,0,0,outerMDIndex);
#endif
      addPixelSegmentToMemory(segmentsInGPU, mdsInGPU, hitsInGPU, modulesInGPU, innerMDIndex, outerMDIndex, pixelModuleIndex, hitIndices0[tid], hitIndices2[tid], dPhiChange[tid], ptIn[tid], ptErr[tid], px[tid], py[tid], pz[tid], etaErr[tid], pixelSegmentIndex, tid);
    }
}
void SDL::Event::addPixelSegmentToEventV2(std::vector<unsigned int> hitIndices0,std::vector<unsigned int> hitIndices1,std::vector<unsigned int> hitIndices2,std::vector<unsigned int> hitIndices3, std::vector<float> dPhiChange, std::vector<float> ptIn, std::vector<float> ptErr, std::vector<float> px, std::vector<float> py, std::vector<float> pz, std::vector<float> etaErr)
{
    if(mdsInGPU == nullptr)
    {
        hipHostMalloc(&mdsInGPU, sizeof(SDL::miniDoublets));
#ifdef Explicit_MD
    	createMDsInExplicitMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#else
    	createMDsInUnifiedMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#endif
    }
    if(segmentsInGPU == nullptr)
    {
        hipHostMalloc(&segmentsInGPU, sizeof(SDL::segments));
#ifdef Explicit_Seg
        createSegmentsInExplicitMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#else
        createSegmentsInUnifiedMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#endif
    }
    const int size = ptIn.size();
    unsigned int pixelModuleIndex = (*detIdToIndex)[1];
    unsigned int* hitIndices0_host = &hitIndices0[0];
    unsigned int* hitIndices1_host = &hitIndices1[0];
    unsigned int* hitIndices2_host = &hitIndices2[0];
    unsigned int* hitIndices3_host = &hitIndices3[0];
    float* dPhiChange_host = &dPhiChange[0];
    float* ptIn_host = &ptIn[0];
    float* ptErr_host = &ptErr[0];
    float* px_host = &px[0];
    float* py_host = &py[0];
    float* pz_host = &pz[0];
    float* etaErr_host = &etaErr[0];

    unsigned int* hitIndices0_dev;
    unsigned int* hitIndices1_dev;
    unsigned int* hitIndices2_dev;
    unsigned int* hitIndices3_dev;
    float* dPhiChange_dev;
    float* ptIn_dev;
    float* ptErr_dev;
    float* px_dev;
    float* py_dev;
    float* pz_dev;
    float* etaErr_dev;

    hipMalloc(&hitIndices0_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices1_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices2_dev,size*sizeof(unsigned int));
    hipMalloc(&hitIndices3_dev,size*sizeof(unsigned int));
    hipMalloc(&dPhiChange_dev,size*sizeof(unsigned int));
    hipMalloc(&ptIn_dev,size*sizeof(unsigned int));
    hipMalloc(&ptErr_dev,size*sizeof(unsigned int));
    hipMalloc(&px_dev,size*sizeof(unsigned int));
    hipMalloc(&py_dev,size*sizeof(unsigned int));
    hipMalloc(&pz_dev,size*sizeof(unsigned int));
    hipMalloc(&etaErr_dev,size*sizeof(unsigned int));

    hipMemcpy(hitIndices0_dev,hitIndices0_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitIndices1_dev,hitIndices1_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitIndices2_dev,hitIndices2_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(hitIndices3_dev,hitIndices3_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(dPhiChange_dev,dPhiChange_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(ptIn_dev,ptIn_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(ptErr_dev,ptErr_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(px_dev,px_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(py_dev,py_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(pz_dev,pz_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(etaErr_dev,etaErr_host,size*sizeof(unsigned int),hipMemcpyHostToDevice);

    unsigned int nThreads = 256;
    unsigned int nBlocks =  size % nThreads == 0 ? size/nThreads : size/nThreads + 1;
  addPixelSegmentToEventKernelV2<<<nBlocks,nThreads>>>(hitIndices0_dev,hitIndices1_dev,hitIndices2_dev,hitIndices3_dev,dPhiChange_dev,ptIn_dev,ptErr_dev,px_dev,py_dev,pz_dev,etaErr_dev,pixelModuleIndex, *modulesInGPU,*hitsInGPU,*mdsInGPU,*segmentsInGPU,size);
   std::cout<<"Number of pixel segments = "<<size<<std::endl;
   hipDeviceSynchronize();
   hipMemcpy(&(segmentsInGPU->nSegments)[pixelModuleIndex], &size, sizeof(unsigned int), hipMemcpyHostToDevice);
   unsigned int mdSize = 2 * size;
   hipMemcpy(&(mdsInGPU->nMDs)[pixelModuleIndex], &mdSize, sizeof(unsigned int), hipMemcpyHostToDevice);

  hipFree(hitIndices0_dev);
  hipFree(hitIndices1_dev);
  hipFree(hitIndices2_dev);
  hipFree(hitIndices3_dev);
  hipFree(dPhiChange_dev);
  hipFree(ptIn_dev);
  hipFree(ptErr_dev);
  hipFree(px_dev);
  hipFree(py_dev);
  hipFree(pz_dev);
  hipFree(etaErr_dev);
}


void SDL::Event::addMiniDoubletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        if(mdsInGPU->nMDs[idx] == 0 or modulesInGPU->hitRanges[idx * 2] == -1)
        {
            modulesInGPU->mdRanges[idx * 2] = -1;
            modulesInGPU->mdRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->mdRanges[idx * 2] = idx * N_MAX_MD_PER_MODULES;
            modulesInGPU->mdRanges[idx * 2 + 1] = (idx * N_MAX_MD_PER_MODULES) + mdsInGPU->nMDs[idx] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[modulesInGPU->layers[idx] -1] += mdsInGPU->nMDs[idx];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += mdsInGPU->nMDs[idx];
            }

        }
    }
}
void SDL::Event::addMiniDoubletsToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
unsigned int* nMDsCPU;
hipHostMalloc(&nMDsCPU, nModules * sizeof(unsigned int));
hipMemcpy(nMDsCPU,mdsInGPU->nMDs,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_mdRanges;
hipHostMalloc(&module_mdRanges, nModules* 2*sizeof(int));
hipMemcpy(module_mdRanges,modulesInGPU->mdRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
int* module_hitRanges;
hipHostMalloc(&module_hitRanges, nModules* 2*sizeof(int));
hipMemcpy(module_hitRanges,modulesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);

    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        if(nMDsCPU[idx] == 0 or module_hitRanges[idx * 2] == -1)
        {
            module_mdRanges[idx * 2] = -1;
            module_mdRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_mdRanges[idx * 2] = idx * N_MAX_MD_PER_MODULES;
            module_mdRanges[idx * 2 + 1] = (idx * N_MAX_MD_PER_MODULES) + nMDsCPU[idx] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[module_layers[idx] -1] += nMDsCPU[idx];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[module_layers[idx] - 1] += nMDsCPU[idx];
            }

        }
    }
hipMemcpy(modulesInGPU->mdRanges,module_mdRanges,nModules*2*sizeof(int),hipMemcpyHostToDevice);
hipHostFree(nMDsCPU);
hipHostFree(module_subdets);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_mdRanges);
hipHostFree(module_layers);
hipHostFree(module_hitRanges);
}

void SDL::Event::addSegmentsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        if(segmentsInGPU->nSegments[idx] == 0)
        {
            modulesInGPU->segmentRanges[idx * 2] = -1;
            modulesInGPU->segmentRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->segmentRanges[idx * 2] = idx * N_MAX_SEGMENTS_PER_MODULE;
            modulesInGPU->segmentRanges[idx * 2 + 1] = idx * N_MAX_SEGMENTS_PER_MODULE + segmentsInGPU->nSegments[idx] - 1;

            //for(unsigned int jdx = 0; jdx < segmentsInGPU->nSegments[idx]; jdx++)
            //    printSegment(*segmentsInGPU, *mdsInGPU, *hitsInGPU, *modulesInGPU, idx * N_MAX_SEGMENTS_PER_MODULE + jdx);

            if(modulesInGPU->subdets[idx] == Barrel)
            {

                n_segments_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += segmentsInGPU->nSegments[idx];
            }
            else
            {
                n_segments_by_layer_endcap_[modulesInGPU->layers[idx] -1] += segmentsInGPU->nSegments[idx];
            }
        }
    }
}
void SDL::Event::addSegmentsToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

unsigned int* nSegmentsCPU;
hipHostMalloc(&nSegmentsCPU, nModules * sizeof(unsigned int));
hipMemcpy(nSegmentsCPU,segmentsInGPU->nSegments,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_segmentRanges;
hipHostMalloc(&module_segmentRanges, nModules* 2*sizeof(int));
hipMemcpy(module_segmentRanges,modulesInGPU->segmentRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        if(nSegmentsCPU[idx] == 0)
        {
            module_segmentRanges[idx * 2] = -1;
            module_segmentRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_segmentRanges[idx * 2] = idx * N_MAX_SEGMENTS_PER_MODULE;
            module_segmentRanges[idx * 2 + 1] = idx * N_MAX_SEGMENTS_PER_MODULE + nSegmentsCPU[idx] - 1;

            //for(unsigned int jdx = 0; jdx < segmentsInGPU->nSegments[idx]; jdx++)
            //    printSegment(*segmentsInGPU, *mdsInGPU, *hitsInGPU, *modulesInGPU, idx * N_MAX_SEGMENTS_PER_MODULE + jdx);

            if(module_subdets[idx] == Barrel)
            {

                n_segments_by_layer_barrel_[module_layers[idx] - 1] += nSegmentsCPU[idx];
            }
            else
            {
                n_segments_by_layer_endcap_[module_layers[idx] -1] += nSegmentsCPU[idx];
            }
        }
    }
hipHostFree(nSegmentsCPU);
hipHostFree(module_subdets);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_segmentRanges);
hipHostFree(module_layers);
}

void SDL::Event::createMiniDoublets()
{
    hipDeviceSynchronize();
    auto memStart = std::chrono::high_resolution_clock::now();
    if(mdsInGPU == nullptr)
    {
        hipHostMalloc(&mdsInGPU, sizeof(SDL::miniDoublets));
#ifdef Explicit_MD
        //FIXME: Add memory locations for pixel MDs
    	createMDsInExplicitMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#else
    	createMDsInUnifiedMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules, N_MAX_PIXEL_MD_PER_MODULES);
#endif
    }
    hipDeviceSynchronize();
    auto memStop = std::chrono::high_resolution_clock::now();
    auto memDuration = std::chrono::duration_cast<std::chrono::milliseconds>(memStop - memStart); //in milliseconds

    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

#ifdef NESTED_PARA
    int nThreads = 1;
    int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;
#else
#ifdef NEWGRID_MD
    int maxThreadsPerModule=0;
    #ifdef Explicit_Module
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    int* module_hitRanges;
    hipHostMalloc(&module_hitRanges, nModules* 2*sizeof(int));
    hipMemcpy(module_hitRanges,modulesInGPU->hitRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
    bool* module_isLower;
    hipHostMalloc(&module_isLower, nModules*sizeof(bool));
    hipMemcpy(module_isLower,modulesInGPU->isLower,nModules*sizeof(bool),hipMemcpyDeviceToHost);
    bool* module_isInverted;
    hipHostMalloc(&module_isInverted, nModules*sizeof(bool));
    hipMemcpy(module_isInverted,modulesInGPU->isInverted,nModules*sizeof(bool),hipMemcpyDeviceToHost);

    for (int i=0; i<nLowerModules; i++) {
      int lowerModuleIndex = module_lowerModuleIndices[i];
      int upperModuleIndex = modulesInGPU->partnerModuleIndexExplicit(lowerModuleIndex,module_isLower[lowerModuleIndex],module_isInverted[lowerModuleIndex]);
      int lowerHitRanges = module_hitRanges[lowerModuleIndex*2];
      int upperHitRanges = module_hitRanges[upperModuleIndex*2];
      if(lowerHitRanges!=-1&&upperHitRanges!=-1) {
        unsigned int nLowerHits = module_hitRanges[lowerModuleIndex * 2 + 1] - lowerHitRanges + 1;
        unsigned int nUpperHits = module_hitRanges[upperModuleIndex * 2 + 1] - upperHitRanges + 1;
        maxThreadsPerModule = maxThreadsPerModule > (nLowerHits*nUpperHits) ? maxThreadsPerModule : nLowerHits*nUpperHits;
      }
    }
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_hitRanges);
    hipHostFree(module_isLower);
    hipHostFree(module_isInverted);
    #else
    //int maxThreadsPerModule=0;
    for (int i=0; i<nLowerModules; i++) {
      int lowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      int upperModuleIndex = modulesInGPU->partnerModuleIndex(lowerModuleIndex);
      int lowerHitRanges = modulesInGPU->hitRanges[lowerModuleIndex*2];
      int upperHitRanges = modulesInGPU->hitRanges[upperModuleIndex*2];
      if(lowerHitRanges!=-1&&upperHitRanges!=-1) {
        unsigned int nLowerHits = modulesInGPU->hitRanges[lowerModuleIndex * 2 + 1] - lowerHitRanges + 1;
        unsigned int nUpperHits = modulesInGPU->hitRanges[upperModuleIndex * 2 + 1] - upperHitRanges + 1;
        maxThreadsPerModule = maxThreadsPerModule > (nLowerHits*nUpperHits) ? maxThreadsPerModule : nLowerHits*nUpperHits;
      }
    }
    #endif
    printf("maxThreadsPerModule=%d\n", maxThreadsPerModule);
    dim3 nThreads(1,128);
    dim3 nBlocks((nLowerModules % nThreads.x == 0 ? nLowerModules/nThreads.x : nLowerModules/nThreads.x + 1), (maxThreadsPerModule % nThreads.y == 0 ? maxThreadsPerModule/nThreads.y : maxThreadsPerModule/nThreads.y + 1));
#else
    dim3 nThreads(1,16,16);
    dim3 nBlocks((nLowerModules % nThreads.x == 0 ? nLowerModules/nThreads.x : nLowerModules/nThreads.x + 1),(N_MAX_HITS_PER_MODULE % nThreads.y == 0 ? N_MAX_HITS_PER_MODULE/nThreads.y : N_MAX_HITS_PER_MODULE/nThreads.y + 1), (N_MAX_HITS_PER_MODULE % nThreads.z == 0 ? N_MAX_HITS_PER_MODULE/nThreads.z : N_MAX_HITS_PER_MODULE/nThreads.z + 1));
    //std::cout<<nBlocks.x<<" "<<nBlocks.y<<" "<<nBlocks.z<<" "<<std::endl;
#endif
#endif

    hipDeviceSynchronize();
    auto syncStart = std::chrono::high_resolution_clock::now();

    createMiniDoubletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU,*hitsInGPU,*mdsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    auto syncStop = std::chrono::high_resolution_clock::now();

    auto syncDuration =  std::chrono::duration_cast<std::chrono::milliseconds>(syncStop - syncStart);

    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }

#if defined(AddObjects)
#ifdef Explicit_MD
    addMiniDoubletsToEventExplicit();
#else
    addMiniDoubletsToEvent();
#endif
#endif


}

void SDL::Event::createSegmentsWithModuleMap()
{
    if(segmentsInGPU == nullptr)
    {
        hipHostMalloc(&segmentsInGPU, sizeof(SDL::segments));
#ifdef Explicit_Seg
        //FIXME:Add memory locations for pixel segments
        //createSegmentsInExplicitMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules);
        createSegmentsInExplicitMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#else
        createSegmentsInUnifiedMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules, N_MAX_PIXEL_SEGMENTS_PER_MODULE);
#endif
    }
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;
#else
#ifdef NEWGRID_Seg
    int max_cModules=0;
    int sq_max_nMDs = 0;
    int nonZeroModules = 0;
  #ifdef Explicit_Module
    unsigned int nModules;
    hipMemcpy(&nModules,modulesInGPU->nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* nMDs = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nMDs, mdsInGPU->nMDs, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);

    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_nConnectedModules;
    hipHostMalloc(&module_nConnectedModules, nModules* sizeof(unsigned int));
    hipMemcpy(module_nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_moduleMap;
    hipHostMalloc(&module_moduleMap, nModules*40* sizeof(unsigned int));
    hipMemcpy(module_moduleMap,modulesInGPU->moduleMap,nModules*40*sizeof(unsigned int),hipMemcpyDeviceToHost);

    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = module_lowerModuleIndices[i];
      unsigned int nConnectedModules = module_nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerMDs = nMDs[innerLowerModuleIndex];
      max_cModules = max_cModules > nConnectedModules ? max_cModules : nConnectedModules;
      int limit_local = 0;
      if (nConnectedModules!=0) nonZeroModules++;
      for (int j=0; j<nConnectedModules; j++) {
        int outerLowerModuleIndex = module_moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + j];
        int nOuterMDs = nMDs[outerLowerModuleIndex];
        int total = nInnerMDs*nOuterMDs;
        limit_local = limit_local > total ? limit_local : total;
      }
      sq_max_nMDs = sq_max_nMDs > limit_local ? sq_max_nMDs : limit_local;
    }
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_nConnectedModules);
    hipHostFree(module_moduleMap);
  #else
    //int max_cModules=0;
    //int sq_max_nMDs = 0;
    //int nonZeroModules = 0;
    unsigned int nModules = *modulesInGPU->nModules;
    unsigned int* nMDs = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nMDs, mdsInGPU->nMDs, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      unsigned int nConnectedModules = modulesInGPU->nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerMDs = nMDs[innerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : nMDs[innerLowerModuleIndex];
      max_cModules = max_cModules > nConnectedModules ? max_cModules : nConnectedModules;
      int limit_local = 0;
      if (nConnectedModules!=0) nonZeroModules++;
      for (int j=0; j<nConnectedModules; j++) {
        int outerLowerModuleIndex = modulesInGPU->moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + j];
        int nOuterMDs = nMDs[outerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : nMDs[outerLowerModuleIndex];
        int total = nInnerMDs*nOuterMDs;
        limit_local = limit_local > total ? limit_local : total;
      }
      sq_max_nMDs = sq_max_nMDs > limit_local ? sq_max_nMDs : limit_local;
    }
  #endif
    printf("max nConnectedModules=%d nonZeroModules=%d max sq_max_nMDs=%d\n", max_cModules, nonZeroModules, sq_max_nMDs);
    dim3 nThreads(256,1,1);
    dim3 nBlocks((sq_max_nMDs%nThreads.x==0 ? sq_max_nMDs/nThreads.x : sq_max_nMDs/nThreads.x + 1), (max_cModules%nThreads.y==0 ? max_cModules/nThreads.y : max_cModules/nThreads.y + 1), (nLowerModules%nThreads.z==0 ? nLowerModules/nThreads.z : nLowerModules/nThreads.z + 1));
    free(nMDs);
#else
    dim3 nThreads(1,16,16);
    dim3 nBlocks(((nLowerModules * MAX_CONNECTED_MODULES)  % nThreads.x == 0 ? (nLowerModules * MAX_CONNECTED_MODULES)/nThreads.x : (nLowerModules * MAX_CONNECTED_MODULES)/nThreads.x + 1),(N_MAX_MD_PER_MODULES % nThreads.y == 0 ? N_MAX_MD_PER_MODULES/nThreads.y : N_MAX_MD_PER_MODULES/nThreads.y + 1), (N_MAX_MD_PER_MODULES % nThreads.z == 0  ? N_MAX_MD_PER_MODULES/nThreads.z : N_MAX_MD_PER_MODULES/nThreads.z + 1));
#endif
#endif

    createSegmentsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#if defined(AddObjects)
#ifdef Explicit_Seg
    addSegmentsToEventExplicit();
#else
    addSegmentsToEvent();
#endif
#endif

}


void SDL::Event::createTriplets()
{
    unsigned int nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    if(tripletsInGPU == nullptr)
    {
        hipHostMalloc(&tripletsInGPU, sizeof(SDL::triplets));
#ifdef Explicit_Trips
        createTripletsInExplicitMemory(*tripletsInGPU, N_MAX_TRIPLETS_PER_MODULE, nLowerModules);
#else
        createTripletsInUnifiedMemory(*tripletsInGPU, N_MAX_TRIPLETS_PER_MODULE, nLowerModules);
#endif
    }

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTripletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
#else
#ifdef NEWGRID_Trips
  #ifdef Explicit_Module
    unsigned int nonZeroModules=0;
    unsigned int max_InnerSeg=0;
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    //unsigned int nModules = *modulesInGPU->nModules;
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_nConnectedModules;
    hipHostMalloc(&module_nConnectedModules, nModules* sizeof(unsigned int));
    hipMemcpy(module_nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = module_lowerModuleIndices[i];
      unsigned int nConnectedModules = module_nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerSegments = nSegments[innerLowerModuleIndex];
      if (nConnectedModules!=0&&nInnerSegments!=0) {
        index[nonZeroModules] = i;
        nonZeroModules++;
      }
      max_InnerSeg = max_InnerSeg > nInnerSegments ? max_InnerSeg : nInnerSegments;
    }
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_nConnectedModules);
  #else
    unsigned int nonZeroModules=0;
    unsigned int max_InnerSeg=0;
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    unsigned int nModules = *modulesInGPU->nModules;
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      unsigned int nConnectedModules = modulesInGPU->nConnectedModules[innerLowerModuleIndex];
      unsigned int nInnerSegments = nSegments[innerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : nSegments[innerLowerModuleIndex];
      if (nConnectedModules!=0&&nInnerSegments!=0) {
        index[nonZeroModules] = i;
        nonZeroModules++;
      }
      max_InnerSeg = max_InnerSeg > nInnerSegments ? max_InnerSeg : nInnerSegments;
    }
  #endif
    hipMemcpy(index_gpu, index, nonZeroModules*sizeof(unsigned int), hipMemcpyHostToDevice);
    int max_OuterSeg = 0;
    /*
    for (int i=0; i<nModules; i++) {
      int nSeg = nSegments[i];
      max_OuterSeg = max_OuterSeg > nSeg ? max_OuterSeg : nSeg;
    }
    */
    max_OuterSeg = N_MAX_SEGMENTS_PER_MODULE;
    printf("nonZeroModules=%d max_InnerSeg=%d max_OuterSeg=%d\n", nonZeroModules, max_InnerSeg, max_OuterSeg);
    dim3 nThreads(32,16,1);
    dim3 nBlocks((max_OuterSeg % nThreads.x == 0 ? max_OuterSeg / nThreads.x : max_OuterSeg / nThreads.x + 1),(max_InnerSeg % nThreads.y == 0 ? max_InnerSeg/nThreads.y : max_InnerSeg/nThreads.y + 1), (nonZeroModules % nThreads.z == 0 ? nonZeroModules/nThreads.z : nonZeroModules/nThreads.z + 1));
    createTripletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU, index_gpu);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
    free(nSegments);
    free(index);
    hipFree(index_gpu);
#else
    printf("original 3D grid launching in createTriplets does not exist");
    exit(1);
#endif
#endif

#if defined(AddObjects)
#ifdef Explicit_Trips
    addTripletsToEventExplicit();
#else
    addTripletsToEvent();
#endif
#endif
}

void SDL::Event::createTrackletsWithModuleMap()
{
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    //TRCAKLETS - To conserve memory, we shall be only declaring nLowerModules amount of memory!!!!!!!
    if(trackletsInGPU == nullptr)
    {
        hipHostMalloc(&trackletsInGPU, sizeof(SDL::tracklets));
#ifdef Explicit_Tracklet
        //FIXME:Add memory locations for pixel tracklets
        createTrackletsInExplicitMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , N_MAX_PIXEL_TRACKLETS_PER_MODULE, nLowerModules);
#else
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , N_MAX_PIXEL_TRACKLETS_PER_MODULE, nLowerModules);
#endif
    }

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
#else
#ifdef NEWGRID_Tracklet
  #ifdef Explicit_Module
    //unsigned int nModules = *modulesInGPU->nModules;
    unsigned int *outerLowerModuleIndices = (unsigned int*)malloc(nModules*N_MAX_SEGMENTS_PER_MODULE*sizeof(unsigned int));
    hipMemcpy((void *)outerLowerModuleIndices, segmentsInGPU->outerLowerModuleIndices, nModules*N_MAX_SEGMENTS_PER_MODULE*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    int max_cModules = 0;
    int sq_max_segments = 0;
    int nonZeroSegModules = 0;
    int inner_max_segments = 0;
    int outer_max_segments = 0;
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_nConnectedModules;
    hipHostMalloc(&module_nConnectedModules, nModules* sizeof(unsigned int));
    hipMemcpy(module_nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int* module_moduleMap;
    hipHostMalloc(&module_moduleMap, nModules*40* sizeof(unsigned int));
    hipMemcpy(module_moduleMap,modulesInGPU->moduleMap,nModules*40*sizeof(unsigned int),hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerInnerLowerModuleIndex = module_lowerModuleIndices[i];
      unsigned int nInnerSegments = nSegments[innerInnerLowerModuleIndex];
      if (nInnerSegments!=0) {
        index[nonZeroSegModules] = i;
        nonZeroSegModules++;
      }
      inner_max_segments = inner_max_segments > nInnerSegments ? inner_max_segments : nInnerSegments;

      for (int j=0; j<nInnerSegments; j++) {
        unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + j;
        unsigned int innerOuterLowerModuleIndex = outerLowerModuleIndices[innerSegmentIndex];
        unsigned int nOuterInnerLowerModules = module_nConnectedModules[innerOuterLowerModuleIndex];
        max_cModules = max_cModules > nOuterInnerLowerModules ? max_cModules : nOuterInnerLowerModules;
        for (int k=0; k<nOuterInnerLowerModules; k++) {
          unsigned int outerInnerLowerModuleIndex = module_moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + k];
          unsigned int nOuterSegments = nSegments[outerInnerLowerModuleIndex];
          sq_max_segments = sq_max_segments > nInnerSegments*nOuterSegments ? sq_max_segments : nInnerSegments*nOuterSegments;
        }
      }
    }
    hipHostFree(module_lowerModuleIndices);
    hipHostFree(module_nConnectedModules);
    hipHostFree(module_moduleMap);
  #else
    unsigned int nModules = *modulesInGPU->nModules;
    unsigned int *outerLowerModuleIndices = (unsigned int*)malloc(nModules*N_MAX_SEGMENTS_PER_MODULE*sizeof(unsigned int));
    hipMemcpy((void *)outerLowerModuleIndices, segmentsInGPU->outerLowerModuleIndices, nModules*N_MAX_SEGMENTS_PER_MODULE*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    int max_cModules = 0;
    int sq_max_segments = 0;
    int nonZeroSegModules = 0;
    int inner_max_segments = 0;
    int outer_max_segments = 0;
    for (int i=0; i<nLowerModules; i++) {
      unsigned int innerInnerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      unsigned int nInnerSegments = nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE  ? N_MAX_SEGMENTS_PER_MODULE : nSegments[innerInnerLowerModuleIndex];
      if (nInnerSegments!=0) {
        index[nonZeroSegModules] = i;
        nonZeroSegModules++;
      }
      inner_max_segments = inner_max_segments > nInnerSegments ? inner_max_segments : nInnerSegments;

      for (int j=0; j<nInnerSegments; j++) {
        unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + j;
        unsigned int innerOuterLowerModuleIndex = outerLowerModuleIndices[innerSegmentIndex];
        unsigned int nOuterInnerLowerModules = modulesInGPU->nConnectedModules[innerOuterLowerModuleIndex];
        max_cModules = max_cModules > nOuterInnerLowerModules ? max_cModules : nOuterInnerLowerModules;
        for (int k=0; k<nOuterInnerLowerModules; k++) {
          unsigned int outerInnerLowerModuleIndex = modulesInGPU->moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + k];
          unsigned int nOuterSegments = nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : nSegments[outerInnerLowerModuleIndex];
          sq_max_segments = sq_max_segments > nInnerSegments*nOuterSegments ? sq_max_segments : nInnerSegments*nOuterSegments;
        }
      }
    }
  #endif
    hipMemcpy(index_gpu, index, nonZeroSegModules*sizeof(unsigned int), hipMemcpyHostToDevice);
    printf("max_cModules=%d sq_max_segments=%d nonZeroSegModules=%d\n", max_cModules, sq_max_segments, nonZeroSegModules);

    dim3 nThreads(128,1,1);
    dim3 nBlocks((sq_max_segments%nThreads.x==0 ? sq_max_segments/nThreads.x : sq_max_segments/nThreads.x + 1), (max_cModules%nThreads.y==0 ? max_cModules/nThreads.y : max_cModules/nThreads.y + 1), (nonZeroSegModules%nThreads.z==0 ? nonZeroSegModules/nThreads.z : nonZeroSegModules/nThreads.z + 1));

    createTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU, index_gpu);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }

    free(outerLowerModuleIndices);
    free(nSegments);
    free(index);
    hipFree(index_gpu);

#else
    printf("original 3D grid launching in createTracklets does not exist");
    exit(1);
#endif
#endif
    /*addTrackletsToEvent will be called in the createTrackletsWithAGapWithModuleMap function*/

#if defined(AddObjects)
#ifdef Explicit_Tracklet
    addTrackletsToEventExplicit();
#else
    addTrackletsToEvent();
#endif
#endif

}

void SDL::Event::createPixelTracklets()
{
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    if(trackletsInGPU == nullptr)
    {
        hipHostMalloc(&trackletsInGPU, sizeof(SDL::tracklets));
#ifdef Explicit_Tracklet
//        //FIXME:Change this to look like the unified allocator below after pixels have been incorporated!
        //createTrackletsInExplicitMemory(*trackletsInGPU,N_MAX_TRACKLETS_PER_MODULE, nLowerModules);
        createTrackletsInExplicitMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , N_MAX_PIXEL_TRACKLETS_PER_MODULE, nLowerModules);
#else
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , N_MAX_PIXEL_TRACKLETS_PER_MODULE, nLowerModules);
#endif
    }

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createPixelTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }
#else
#ifdef NEWGRID_Pixel
  #ifdef Explicit_Module
    unsigned int nModules; //= *modulesInGPU->nModules;
    hipMemcpy(&nModules,modulesInGPU->nModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int pixelModuleIndex = nModules - 1;
    unsigned int pixelLowerModuleArrayIndex;
    hipMemcpy(&pixelLowerModuleArrayIndex, modulesInGPU->reverseLookupLowerModuleIndices+pixelModuleIndex-1; sizeof(unsigned int), cudaMemcpyDeviceHost);
    unsigned int nonZeroModules=0;
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    unsigned int max_InnerSeg = 0;
    unsigned int max_OuterSeg = 0;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    unsigned int* module_lowerModuleIndices;
    hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
    hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
    for (int i=0; i<nLowerModules; i++) {
      unsigned int outerInnerLowerModuleIndex = module_lowerModuleIndices[i];
      unsigned int nInnerSegments = nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : nSegments[pixelModuleIndex];
      unsigned int nOuterSegments = nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : nSegments[outerInnerLowerModuleIndex];
      max_InnerSeg = max_InnerSeg > nInnerSegments ? max_InnerSeg : nInnerSegments;
      max_OuterSeg = max_OuterSeg > nOuterSegments ? max_OuterSeg : nOuterSegments;
      if (nInnerSegments!=0&&nOuterSegments!=0)
        index[nonZeroModules++] = i;
    }
    hipHostFree(module_lowerModuleIndices);
#else
    unsigned int nModules = *modulesInGPU->nModules;
    unsigned int *nSegments = (unsigned int*)malloc(nModules*sizeof(unsigned int));
    hipMemcpy((void *)nSegments, segmentsInGPU->nSegments, nModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    unsigned int pixelModuleIndex = nModules - 1;
    unsigned int pixelLowerModuleArrayIndex = modulesInGPU->reverseLookupLowerModuleIndices[pixelModuleIndex];
    unsigned int nonZeroModules=0;
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    unsigned int max_InnerSeg = 0;
    unsigned int max_OuterSeg = 0;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    for (int i=0; i<nLowerModules; i++) {
      unsigned int outerInnerLowerModuleIndex = modulesInGPU->lowerModuleIndices[i];
      unsigned int nInnerSegments = nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : nSegments[pixelModuleIndex];
      unsigned int nOuterSegments = nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : nSegments[outerInnerLowerModuleIndex];
      max_InnerSeg = max_InnerSeg > nInnerSegments ? max_InnerSeg : nInnerSegments;
      max_OuterSeg = max_OuterSeg > nOuterSegments ? max_OuterSeg : nOuterSegments;
      if (nInnerSegments!=0&&nOuterSegments!=0)
        index[nonZeroModules++] = i;
    }
#endif
    hipMemcpy(index_gpu, index, nonZeroModules*sizeof(unsigned int), hipMemcpyHostToDevice);
    printf("createPixelTracklets: nonZeroModules=%d max_InnerSeg=%d max_OuterSeg=%d\n", nonZeroModules, max_InnerSeg, max_OuterSeg);

    dim3 nThreads(16,32,1);
    dim3 nBlocks((max_OuterSeg % nThreads.x == 0 ? max_OuterSeg / nThreads.x : max_OuterSeg / nThreads.x + 1),(max_InnerSeg % nThreads.y == 0 ? max_InnerSeg/nThreads.y : max_InnerSeg/nThreads.y + 1), (nonZeroModules % nThreads.z == 0 ? nonZeroModules/nThreads.z : nonZeroModules/nThreads.z + 1));
    createPixelTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU, index_gpu);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

      }
    free(nSegments);
    free(index);
    hipFree(index_gpu);
#else
    printf("original 3D grid launching in createPixelTracklets does not exist");
    exit(1);
#endif
#endif

//#if defined(AddObjects) && !defined(Full_Explicit)
 //   std::cout<<"Number of pixel tracklets = "<<trackletsInGPU->nTracklets[nLowerModules]<<std::endl;
//#endif
    unsigned int nPixelTracklets;
    hipMemcpy(&nPixelTracklets, &(trackletsInGPU->nTracklets[nLowerModules]), sizeof(unsigned int), hipMemcpyDeviceToHost);
    std::cout<<"number of pixel tracklets = "<<nPixelTracklets<<std::endl;
}

void SDL::Event::createTrackCandidates()
{
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules + 1; //including the pixel module
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    nLowerModules += 1;// include the pixel module

    //construct the list of eligible modules
    unsigned int nEligibleModules = 0;
    createEligibleModulesListForTrackCandidates(*modulesInGPU, nEligibleModules, N_MAX_TRACK_CANDIDATES_PER_MODULE);

    if(trackCandidatesInGPU == nullptr)
    {
        hipHostMalloc(&trackCandidatesInGPU, sizeof(SDL::trackCandidates));
#ifdef Explicit_Track
        createTrackCandidatesInExplicitMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES_PER_MODULE, N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE, nLowerModules, nEligibleModules);
#else
        createTrackCandidatesInUnifiedMemory(*trackCandidatesInGPU, N_MAX_TRACK_CANDIDATES_PER_MODULE, N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE, nLowerModules, nEligibleModules);
#endif
    }

#ifdef NESTED_PARA
    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTrackCandidatesInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU, *tripletsInGPU, *trackCandidatesInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#else
    int maxInnerTr = 0;
    int maxOuterTr = max(N_MAX_TRACKLETS_PER_MODULE, N_MAX_TRIPLETS_PER_MODULE);
    int nonZeroModules=0;
    unsigned int *index = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *nTriplets = (unsigned int*)malloc((nLowerModules-1)*sizeof(unsigned int));
    unsigned int *nTracklets = (unsigned int*)malloc(nLowerModules*sizeof(unsigned int));
    unsigned int *index_gpu;
    hipMalloc((void **)&index_gpu, nLowerModules*sizeof(unsigned int));
    hipMemcpy(nTriplets, tripletsInGPU->nTriplets, (nLowerModules-1)*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(nTracklets, trackletsInGPU->nTracklets, nLowerModules*sizeof(unsigned int), hipMemcpyDeviceToHost);
    for (int i=0; i< nLowerModules; i++) {
      unsigned int nInnerTracklets = nTracklets[i];
      if (i == nLowerModules-1) {
        if(nInnerTracklets > N_MAX_PIXEL_TRACKLETS_PER_MODULE)
          nInnerTracklets = N_MAX_PIXEL_TRACKLETS_PER_MODULE;
      }
      else {
        if(nInnerTracklets > N_MAX_TRACKLETS_PER_MODULE)
          nInnerTracklets = N_MAX_TRACKLETS_PER_MODULE;
      }
      unsigned int nInnerTriplets = 0;
      if (i != nLowerModules-1)
        nInnerTriplets = nTriplets[i];
      if(nInnerTriplets > N_MAX_TRIPLETS_PER_MODULE)
        nInnerTriplets = N_MAX_TRIPLETS_PER_MODULE;
      unsigned int temp = max(nInnerTracklets, nInnerTriplets);
      if (temp !=0) {
        index[nonZeroModules] = i;
        nonZeroModules++;
      }
      maxInnerTr = maxInnerTr > temp ? maxInnerTr : temp;
    }
    hipMemcpy(index_gpu, index, nonZeroModules*sizeof(unsigned int), hipMemcpyHostToDevice);
    printf("createTrackCandidates: nonZeroModules=%d maxInnerTr=%d maxOuterTr=%d\n", nonZeroModules, maxInnerTr, maxOuterTr);

    dim3 nThreads(32, 32, 1);
    dim3 nBlocks(maxOuterTr % nThreads.x == 0 ? maxOuterTr/nThreads.x : maxOuterTr/nThreads.x + 1,(maxInnerTr % nThreads.y == 0 ? maxInnerTr/nThreads.y : maxInnerTr/nThreads.y + 1), (nonZeroModules % nThreads.z == 0 ? nonZeroModules/nThreads.z : nonZeroModules/nThreads.z + 1));

    createTrackCandidatesInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU, *tripletsInGPU, *trackCandidatesInGPU, index_gpu);
    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
      {
	std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
      }
    free(index);
    free(nTriplets);
    free(nTracklets);
    hipFree(index_gpu);
#endif

#if defined(AddObjects)
#ifdef Explicit_Track
    addTrackCandidatesToEventExplicit();
#else
    addTrackCandidatesToEvent();
#endif
#endif

}

void SDL::Event::createTrackletsWithAGapWithModuleMap()
{
    //use the same trackletsInGPU as before if it exists
    unsigned int nLowerModules;// = *modulesInGPU->nLowerModules;
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

    //TRCAKLETS - To conserve memory, we shall be only declaring nLowerModules amount of memory!!!!!!!
    if(trackletsInGPU == nullptr)
    {
        hipHostMalloc(&trackletsInGPU, sizeof(SDL::tracklets));
#ifdef Explicit_Tracklet
        //createTrackletsInExplicitMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
        createTrackletsInExplicitMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , N_MAX_PIXEL_TRACKLETS_PER_MODULE, nLowerModules);
#else
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , N_MAX_PIXEL_TRACKLETS_PER_MODULE, nLowerModules);
#endif
    }

    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTrackletsWithAGapInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;

    }

}


void SDL::Event::addTrackletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(trackletsInGPU->nTracklets[i] == 0)
        {
            modulesInGPU->trackletRanges[idx * 2] = -1;
            modulesInGPU->trackletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->trackletRanges[idx * 2] = idx * N_MAX_TRACKLETS_PER_MODULE;
            modulesInGPU->trackletRanges[idx * 2 + 1] = idx * N_MAX_TRACKLETS_PER_MODULE + trackletsInGPU->nTracklets[i] - 1;


            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_tracklets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += trackletsInGPU->nTracklets[i];
            }
            else
            {
                n_tracklets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += trackletsInGPU->nTracklets[i];
            }
        }
    }
}
void SDL::Event::addTrackletsToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

unsigned int* nTrackletsCPU;
hipHostMalloc(&nTrackletsCPU, nLowerModules * sizeof(unsigned int));
hipMemcpy(nTrackletsCPU,trackletsInGPU->nTracklets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_trackletRanges;
hipHostMalloc(&module_trackletRanges, nModules* 2*sizeof(int));
hipMemcpy(module_trackletRanges,modulesInGPU->trackletRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(nTrackletsCPU[i] == 0)
        {
            module_trackletRanges[idx * 2] = -1;
            module_trackletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_trackletRanges[idx * 2] = idx * N_MAX_TRACKLETS_PER_MODULE;
            module_trackletRanges[idx * 2 + 1] = idx * N_MAX_TRACKLETS_PER_MODULE + nTrackletsCPU[i] - 1;


            if(module_subdets[idx] == Barrel)
            {
                n_tracklets_by_layer_barrel_[module_layers[idx] - 1] += nTrackletsCPU[i];
            }
            else
            {
                n_tracklets_by_layer_endcap_[module_layers[idx] - 1] += nTrackletsCPU[i];
            }
        }
    }
hipHostFree(nTrackletsCPU);
hipHostFree(module_subdets);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_trackletRanges);
hipHostFree(module_layers);
}

void SDL::Event::addTrackCandidatesToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

unsigned int* nTrackCandidatesCPU;
hipHostMalloc(&nTrackCandidatesCPU, (nLowerModules )* sizeof(unsigned int));
hipMemcpy(nTrackCandidatesCPU,trackCandidatesInGPU->nTrackCandidates,(nLowerModules)*sizeof(unsigned int),hipMemcpyDeviceToHost);

unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_trackCandidateRanges;
hipHostMalloc(&module_trackCandidateRanges, nModules* 2*sizeof(int));
hipMemcpy(module_trackCandidateRanges,modulesInGPU->trackCandidateRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);

    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];


        if(nTrackCandidatesCPU[i] == 0)
        {
            module_trackCandidateRanges[idx * 2] = -1;
            module_trackCandidateRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_trackCandidateRanges[idx * 2] = idx * N_MAX_TRACK_CANDIDATES_PER_MODULE;
            module_trackCandidateRanges[idx * 2 + 1] = idx * N_MAX_TRACK_CANDIDATES_PER_MODULE + nTrackCandidatesCPU[i] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_trackCandidates_by_layer_barrel_[module_layers[idx] - 1] += nTrackCandidatesCPU[i];
            }
            else
            {
                n_trackCandidates_by_layer_endcap_[module_layers[idx] - 1] += nTrackCandidatesCPU[i];
            }
        }
    }
hipHostFree(nTrackCandidatesCPU);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_trackCandidateRanges);
hipHostFree(module_layers);
hipHostFree(module_subdets);
}
void SDL::Event::addTrackCandidatesToEvent()
{

    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];


        if(trackCandidatesInGPU->nTrackCandidates[i] == 0)
        {
            modulesInGPU->trackCandidateRanges[idx * 2] = -1;
            modulesInGPU->trackCandidateRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->trackCandidateRanges[idx * 2] = idx * N_MAX_TRACK_CANDIDATES_PER_MODULE;
            modulesInGPU->trackCandidateRanges[idx * 2 + 1] = idx * N_MAX_TRACK_CANDIDATES_PER_MODULE + trackCandidatesInGPU->nTrackCandidates[i] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_trackCandidates_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += trackCandidatesInGPU->nTrackCandidates[i];
            }
            else
            {
                n_trackCandidates_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += trackCandidatesInGPU->nTrackCandidates[i];
            }
        }
    }
}


void SDL::Event::addTripletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(tripletsInGPU->nTriplets[i] == 0)
        {
            modulesInGPU->tripletRanges[idx * 2] = -1;
            modulesInGPU->tripletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->tripletRanges[idx * 2] = idx * N_MAX_TRIPLETS_PER_MODULE;
            modulesInGPU->tripletRanges[idx * 2 + 1] = idx * N_MAX_TRIPLETS_PER_MODULE + tripletsInGPU->nTriplets[i] - 1;

            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_triplets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += tripletsInGPU->nTriplets[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += tripletsInGPU->nTriplets[i];
            }
        }
    }
}
void SDL::Event::addTripletsToEventExplicit()
{
unsigned int nLowerModules;
hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);

unsigned int* nTripletsCPU;
hipHostMalloc(&nTripletsCPU, nLowerModules * sizeof(unsigned int));
hipMemcpy(nTripletsCPU,tripletsInGPU->nTriplets,nLowerModules*sizeof(unsigned int),hipMemcpyDeviceToHost);

short* module_subdets;
hipHostMalloc(&module_subdets, nModules* sizeof(short));
hipMemcpy(module_subdets,modulesInGPU->subdets,nModules*sizeof(short),hipMemcpyDeviceToHost);
unsigned int* module_lowerModuleIndices;
hipHostMalloc(&module_lowerModuleIndices, (nLowerModules +1)* sizeof(unsigned int));
hipMemcpy(module_lowerModuleIndices,modulesInGPU->lowerModuleIndices,(nLowerModules+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
int* module_tripletRanges;
hipHostMalloc(&module_tripletRanges, nModules* 2*sizeof(int));
hipMemcpy(module_tripletRanges,modulesInGPU->tripletRanges,nModules*2*sizeof(int),hipMemcpyDeviceToHost);
short* module_layers;
hipHostMalloc(&module_layers, nModules * sizeof(short));
hipMemcpy(module_layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    unsigned int idx;
    for(unsigned int i = 0; i<nLowerModules; i++)
    {
        idx = module_lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(nTripletsCPU[i] == 0)
        {
            module_tripletRanges[idx * 2] = -1;
            module_tripletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            module_tripletRanges[idx * 2] = idx * N_MAX_TRIPLETS_PER_MODULE;
            module_tripletRanges[idx * 2 + 1] = idx * N_MAX_TRIPLETS_PER_MODULE + nTripletsCPU[i] - 1;

            if(module_subdets[idx] == Barrel)
            {
                n_triplets_by_layer_barrel_[module_layers[idx] - 1] += nTripletsCPU[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[module_layers[idx] - 1] += nTripletsCPU[i];
            }
        }
    }
hipHostFree(nTripletsCPU);
hipHostFree(module_lowerModuleIndices);
hipHostFree(module_tripletRanges);
hipHostFree(module_layers);
hipHostFree(module_subdets);
}
#ifndef NESTED_PARA
__global__ void createMiniDoubletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU)
{
    int lowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    //int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;
    if(lowerModuleArrayIndex >= (*modulesInGPU.nLowerModules)) return; //extra precaution

    int lowerModuleIndex = modulesInGPU.lowerModuleIndices[lowerModuleArrayIndex];
    int upperModuleIndex = modulesInGPU.partnerModuleIndex(lowerModuleIndex);

    if(modulesInGPU.hitRanges[lowerModuleIndex * 2] == -1) return;
    if(modulesInGPU.hitRanges[upperModuleIndex * 2] == -1) return;
    unsigned int nLowerHits = modulesInGPU.hitRanges[lowerModuleIndex * 2 + 1] - modulesInGPU.hitRanges[lowerModuleIndex * 2] + 1;
    unsigned int nUpperHits = modulesInGPU.hitRanges[upperModuleIndex * 2 + 1] - modulesInGPU.hitRanges[upperModuleIndex * 2] + 1;

#ifdef NEWGRID_MD
    int lowerHitIndex =  (blockIdx.y * blockDim.y + threadIdx.y) / nUpperHits;
    int upperHitIndex =  (blockIdx.y * blockDim.y + threadIdx.y) % nUpperHits;
#else
    int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;
#endif

    //consider assigining a dummy computation function for these
    if(lowerHitIndex >= nLowerHits) return;
    if(upperHitIndex >= nUpperHits) return;

    unsigned int lowerHitArrayIndex = modulesInGPU.hitRanges[lowerModuleIndex * 2] + lowerHitIndex;
    unsigned int upperHitArrayIndex = modulesInGPU.hitRanges[upperModuleIndex * 2] + upperHitIndex;

    float dz, drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;

#ifdef CUT_VALUE_DEBUG
    float dzCut, drtCut, miniCut;
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz,  drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut);
#else
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange);
#endif

    if(success)
    {
        unsigned int mdModuleIndex = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);
        if(mdModuleIndex >= N_MAX_MD_PER_MODULES)
        {
            if(mdModuleIndex == N_MAX_MD_PER_MODULES)
                printf("Mini-doublet excess alert! Module index =  %d\n",lowerModuleIndex);
        }
        else
        {
            unsigned int mdIndex = lowerModuleIndex * N_MAX_MD_PER_MODULES + mdModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz,drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut, mdIndex);
#else
        addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIndex);
#endif

        }

    }
}
#else
__global__ void createMiniDoubletsFromLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, unsigned int lowerModuleIndex, unsigned int upperModuleIndex, unsigned int nLowerHits, unsigned int nUpperHits)
{
    unsigned int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;

    //consider assigining a dummy computation function for these
    if(lowerHitIndex >= nLowerHits) return;
    if(upperHitIndex >= nUpperHits) return;

    unsigned int lowerHitArrayIndex = modulesInGPU.hitRanges[lowerModuleIndex * 2] + lowerHitIndex;
    unsigned int upperHitArrayIndex = modulesInGPU.hitRanges[upperModuleIndex * 2] + upperHitIndex;

    float dz, drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;

#ifdef CUT_VALUE_DEBUG
    float dzCut, drtCut, miniCut;
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz,  drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut);
#else
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange);
#endif

    if(success)
    {
        unsigned int mdModuleIndex = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);

        if(mdModuleIndex >= N_MAX_MD_PER_MODULES)
        {
            if(mdModuleIndex == N_MAX_MD_PER_MODULES)
                printf("Mini-doublet excess alert! Module index = %d\n",lowerModuleIndex);
        }
        else
        {
            unsigned int mdIndex = lowerModuleIndex * N_MAX_MD_PER_MODULES + mdModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz,drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut, mdIndex);
#else
            addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIndex);
#endif
        }

    }
}


__global__ void createMiniDoubletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU)
{
    int lowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(lowerModuleArrayIndex >= (*modulesInGPU.nLowerModules)) return; //extra precaution

    int lowerModuleIndex = modulesInGPU.lowerModuleIndices[lowerModuleArrayIndex];
    int upperModuleIndex = modulesInGPU.partnerModuleIndex(lowerModuleIndex);

    if(modulesInGPU.hitRanges[lowerModuleIndex * 2] == -1) return;
    if(modulesInGPU.hitRanges[upperModuleIndex * 2] == -1) return;

    unsigned int nLowerHits = modulesInGPU.hitRanges[lowerModuleIndex * 2 + 1] - modulesInGPU.hitRanges[lowerModuleIndex * 2] + 1;
    unsigned int nUpperHits = modulesInGPU.hitRanges[upperModuleIndex * 2 + 1] - modulesInGPU.hitRanges[upperModuleIndex * 2] + 1;

    dim3 nThreads(1,16,16);
    dim3 nBlocks(1,nLowerHits % nThreads.y == 0 ? nLowerHits/nThreads.y : nLowerHits/nThreads.y + 1, nUpperHits % nThreads.z == 0 ? nUpperHits/nThreads.z : nUpperHits/nThreads.z + 1);

    createMiniDoubletsFromLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, lowerModuleIndex, upperModuleIndex, nLowerHits, nUpperHits);


}
#endif

#ifndef NESTED_PARA
__global__ void createSegmentsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{
#ifdef NEWGRID_Seg
    int innerLowerModuleArrayIdx = blockIdx.z * blockDim.z + threadIdx.z;
    int outerLowerModuleArrayIdx = blockIdx.y * blockDim.y + threadIdx.y;
#else
    int xAxisIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int innerMDArrayIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int outerMDArrayIdx = blockIdx.z * blockDim.z + threadIdx.z;

    int innerLowerModuleArrayIdx = xAxisIdx/MAX_CONNECTED_MODULES;
    int outerLowerModuleArrayIdx = xAxisIdx % MAX_CONNECTED_MODULES; //need this index from the connected module array
#endif
    if(innerLowerModuleArrayIdx >= *modulesInGPU.nLowerModules) return;

    unsigned int innerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerLowerModuleArrayIdx];

    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerLowerModuleIndex];

    if(outerLowerModuleArrayIdx >= nConnectedModules) return;

    unsigned int outerLowerModuleIndex = modulesInGPU.moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + outerLowerModuleArrayIdx];

    unsigned int nInnerMDs = mdsInGPU.nMDs[innerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[innerLowerModuleIndex];
    unsigned int nOuterMDs = mdsInGPU.nMDs[outerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[outerLowerModuleIndex];

#ifdef NEWGRID_Seg
    if (nInnerMDs*nOuterMDs == 0) return;
    int innerMDArrayIdx = (blockIdx.x * blockDim.x + threadIdx.x) / nOuterMDs;
    int outerMDArrayIdx = (blockIdx.x * blockDim.x + threadIdx.x) % nOuterMDs;
#endif

    if(innerMDArrayIdx >= nInnerMDs) return;
    if(outerMDArrayIdx >= nOuterMDs) return;

    unsigned int innerMDIndex = modulesInGPU.mdRanges[innerLowerModuleIndex * 2] + innerMDArrayIdx;
    unsigned int outerMDIndex = modulesInGPU.mdRanges[outerLowerModuleIndex * 2] + outerMDArrayIdx;

    float zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD;

    unsigned int innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex;

    dPhiMin = 0;
    dPhiMax = 0;
    dPhiChangeMin = 0;
    dPhiChangeMax = 0;
#ifdef CUT_VALUE_DEBUG
    float zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold, dAlphaInnerMDOuterMDThreshold;

    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
            dAlphaInnerMDOuterMDThreshold, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);

#else
    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);
#endif

    if(success)
    {
        unsigned int segmentModuleIdx = atomicAdd(&segmentsInGPU.nSegments[innerLowerModuleIndex],1);
        if(segmentModuleIdx >= N_MAX_SEGMENTS_PER_MODULE)
        {
            if(segmentModuleIdx == N_MAX_SEGMENTS_PER_MODULE)
                printf("Segment excess alert! Module index = %d\n",innerLowerModuleIndex);
        }
        else
        {
            unsigned int segmentIdx = innerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentModuleIdx;
#ifdef CUT_VALUE_DEBUG
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
                dAlphaInnerMDOuterMDThreshold, segmentIdx);
#else
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD,segmentIdx);
#endif

        }
    }
}
#else

__global__ void createSegmentsFromInnerLowerModule(struct SDL::modules&modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerLowerModuleIndex, unsigned int nInnerMDs)
{
    unsigned int outerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int innerMDArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int outerMDArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    unsigned int outerLowerModuleIndex = modulesInGPU.moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + outerLowerModuleArrayIndex];

    unsigned int nOuterMDs = mdsInGPU.nMDs[outerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[outerLowerModuleIndex];
    if(innerMDArrayIndex >= nInnerMDs) return;
    if(outerMDArrayIndex >= nOuterMDs) return;

    unsigned int innerMDIndex = innerLowerModuleIndex * N_MAX_MD_PER_MODULES + innerMDArrayIndex;
    unsigned int outerMDIndex = outerLowerModuleIndex * N_MAX_MD_PER_MODULES + outerMDArrayIndex;

    float zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD;

    unsigned int innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex;

    dPhiMin = 0;
    dPhiMax = 0;
    dPhiChangeMin = 0;
    dPhiChangeMax = 0;
#ifdef CUT_VALUE_DEBUG
    float zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold, dAlphaInnerMDOuterMDThreshold;

    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
            dAlphaInnerMDOuterMDThreshold, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);

#else
    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);
#endif

    if(success)
    {
        unsigned int segmentModuleIdx = atomicAdd(&segmentsInGPU.nSegments[innerLowerModuleIndex],1);
        if(segmentModuleIdx >= N_MAX_SEGMENTS_PER_MODULE)
        {
            if(segmentModuleIdx == N_MAX_SEGMENTS_PER_MODULE)
                printf("Segment excess alert! Module index = %d\n",innerLowerModuleIndex);
        }
        else
        {
            unsigned int segmentIdx = innerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentModuleIdx;
#ifdef CUT_VALUE_DEBUG
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, zLo, zHi, rtLo, rtHi, sdCut, dAlphaInnerMDSegmentThreshold, dAlphaOuterMDSegmentThreshold,
                dAlphaInnerMDOuterMDThreshold, segmentIdx);
#else
            addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD,segmentIdx);
#endif

        }

    }

}

__global__ void createSegmentsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{
    int innerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerLowerModuleArrayIndex];
    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerLowerModuleIndex];
    unsigned int nInnerMDs = mdsInGPU.nMDs[innerLowerModuleIndex] > N_MAX_MD_PER_MODULES ? N_MAX_MD_PER_MODULES : mdsInGPU.nMDs[innerLowerModuleIndex];

    if(nConnectedModules == 0) return;

    if(nInnerMDs == 0) return;
    dim3 nThreads(1,16,16);
    dim3 nBlocks((nConnectedModules % nThreads.x == 0 ? nConnectedModules/nThreads.x : nConnectedModules/nThreads.x + 1), (nInnerMDs % nThreads.y == 0 ? nInnerMDs/nThreads.y : nInnerMDs/nThreads.y + 1), (N_MAX_MD_PER_MODULES % nThreads.z == 0 ? N_MAX_MD_PER_MODULES/nThreads.z : N_MAX_MD_PER_MODULES/nThreads.z + 1));

    createSegmentsFromInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerLowerModuleIndex,nInnerMDs);

}
#endif

#ifndef NESTED_PARA
__global__ void createTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int *index_gpu)
{
  //int innerInnerLowerModuleArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;
  int innerInnerLowerModuleArrayIndex = index_gpu[blockIdx.z * blockDim.z + threadIdx.z];
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
  unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];

  if(nInnerSegments == 0) return;

  int outerInnerLowerModuleArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  int innerSegmentArrayIndex = (blockIdx.x * blockDim.x + threadIdx.x) % nInnerSegments;
  int outerSegmentArrayIndex = (blockIdx.x * blockDim.x + threadIdx.x) / nInnerSegments;

  if(innerSegmentArrayIndex >= nInnerSegments) return;

  //outer inner lower module array indices should be obtained from the partner module of the inner segment's outer lower module
  unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

  unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

  //number of possible outer segment inner MD lower modules
  unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
  if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

  unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

  unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
  if(outerSegmentArrayIndex >= nOuterSegments) return;

  unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

  //for completeness - outerOuterLowerModuleIndex
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

  //with both segment indices obtained, run the tracklet algorithm
  float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;

#ifdef CUT_VALUE_DEBUG
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses

#else

  bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut); //might want to send the other two module indices and the anchor hits also to save memory accesses
#endif

  if(success)
    {
      unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
      if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
      {
          if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
              printf("Tracklet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
      }
      else
      {
          unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
          addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
          addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex);

#endif

      }
    }
}
#else
__global__ void createTrackletsFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    if(innerSegmentArrayIndex >= nInnerSegments) return;
        //outer inner lower module array indices should be obtained from the partner module of the inner segment's outer lower module
    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;


    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    //number of possible outer segment inner MD lower modules
    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm

   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;

#ifdef CUT_VALUE_DEBUG
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses

#else

  bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut); //might want to send the other two module indices and the anchor hits also to save memory accesses
#endif

   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
        {
            if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
                printf("Tracklet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
        }
        else
        {
            unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex);

#endif
        }
   }



}

__global__ void createTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
  int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
  unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
  unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
  if(nInnerSegments == 0) return;

  dim3 nThreads(1,16,16);
  dim3 nBlocks(MAX_CONNECTED_MODULES % nThreads.x  == 0 ? MAX_CONNECTED_MODULES / nThreads.x : MAX_CONNECTED_MODULES / nThreads.x + 1 ,nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,N_MAX_SEGMENTS_PER_MODULE % nThreads.z == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.z : N_MAX_SEGMENTS_PER_MODULE/nThreads.z + 1);

  createTrackletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex,nInnerSegments,innerInnerLowerModuleArrayIndex);

}
#endif

#ifndef NESTED_PARA
 __global__ void createPixelTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int *index_gpu)
 {
   int outerInnerLowerModuleArrayIndex = index_gpu[blockIdx.z * blockDim.z + threadIdx.z];
   if(outerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

   unsigned int outerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerInnerLowerModuleArrayIndex];
   unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1; //last dude
   unsigned int pixelLowerModuleArrayIndex = modulesInGPU.reverseLookupLowerModuleIndices[pixelModuleIndex]; //should be the same as nLowerModules
  unsigned int nInnerSegments = segmentsInGPU.nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[pixelModuleIndex];
  unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
  int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  int outerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if(innerSegmentArrayIndex >= nInnerSegments) return;
  if(outerSegmentArrayIndex >= nOuterSegments) return;

  unsigned int innerSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;
  unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];
  float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut;
#ifdef CUT_VALUE_DEBUG
  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
  bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
#else
   bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
#endif

 if(success)
   {
     unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[pixelLowerModuleArrayIndex], 1);
     if(trackletModuleIndex >= N_MAX_PIXEL_TRACKLETS_PER_MODULE)
       {
	 if(trackletModuleIndex == N_MAX_PIXEL_TRACKLETS_PER_MODULE)
	   printf("Pixel Tracklet excess alert! Module index = %d\n",pixelModuleIndex);
       }
     else
       {
	 unsigned int trackletIndex = pixelLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
	 addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
	 addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex);
#endif
       }
   }

 }
#else
__global__ void createPixelTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(outerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[outerInnerLowerModuleArrayIndex];
    unsigned int pixelModuleIndex = *modulesInGPU.nModules - 1; //last dude
    unsigned int pixelLowerModuleArrayIndex = modulesInGPU.reverseLookupLowerModuleIndices[pixelModuleIndex]; //should be the same as nLowerModules
    unsigned int nInnerSegments = segmentsInGPU.nSegments[pixelModuleIndex] > N_MAX_PIXEL_SEGMENTS_PER_MODULE ? N_MAX_PIXEL_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[pixelModuleIndex];
    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(nOuterSegments == 0) return;
    if(nInnerSegments == 0) return;
    dim3 nThreads(16,16,1);
    dim3 nBlocks(nInnerSegments % nThreads.x == 0 ? nInnerSegments / nThreads.x : nInnerSegments / nThreads.x + 1, nOuterSegments % nThreads.y == 0 ? nOuterSegments / nThreads.y : nOuterSegments / nThreads.y + 1, 1);

    createPixelTrackletsFromOuterInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, trackletsInGPU, outerInnerLowerModuleIndex, nInnerSegments, nOuterSegments, pixelModuleIndex, pixelLowerModuleArrayIndex);

}

__global__ void createPixelTrackletsFromOuterInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int outerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int nOuterSegments, unsigned int pixelModuleIndex, unsigned int pixelLowerModuleArrayIndex)
{
    int innerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if(innerSegmentArrayIndex >= nInnerSegments) return;
    if(outerSegmentArrayIndex >= nOuterSegments) return;
    unsigned int innerSegmentIndex = pixelModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;
    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut;
#ifdef CUT_VALUE_DEBUG
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
#else
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, pixelModuleIndex, pixelModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
#endif

    if(success)
    {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[pixelLowerModuleArrayIndex], 1);
        if(trackletModuleIndex >= N_MAX_PIXEL_TRACKLETS_PER_MODULE)
        {
            if(trackletModuleIndex == N_MAX_PIXEL_TRACKLETS_PER_MODULE)
                printf("Pixel Tracklet excess alert! Module index = %d\n",pixelModuleIndex);
        }
        else
        {
            unsigned int trackletIndex = pixelLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
                addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,pixelModuleIndex,pixelModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex);
#endif
        }



    }
}
#endif

__global__ void createTrackletsWithAGapFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    //Proposal 1 : Inner kernel takes care of both loops
    int xAxisIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex =  blockIdx.y * blockDim.y + threadIdx.y;
    int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    if(innerSegmentArrayIndex >= nInnerSegments) return;

    int middleLowerModuleArrayIndex = xAxisIndex / MAX_CONNECTED_MODULES;
    int outerInnerLowerModuleArrayIndex = xAxisIndex % MAX_CONNECTED_MODULES;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;
    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    //first check for middle modules
    unsigned int nMiddleLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
    if(middleLowerModuleArrayIndex >= nMiddleLowerModules) return;

    unsigned int middleLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + middleLowerModuleArrayIndex];

    //second check for outerInnerLowerMoules
    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[middleLowerModuleIndex];
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[middleLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm

   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;
#ifdef CUT_VALUE_DEBUG
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;
    bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, N_MAX_SEGMENTS_PER_MODULE); //might want to send the other two module indices and the anchor hits also to save memory accesses
#else
   bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut); //might want to send the other two module indices and the anchor hits also to save memory accesses
#endif
   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        if(trackletModuleIndex >= N_MAX_TRACKLETS_PER_MODULE)
        {
            if(trackletModuleIndex == N_MAX_TRACKLETS_PER_MODULE)
                 printf("T4x excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
        }
        else
        {

            unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;
#ifdef CUT_VALUE_DEBUG
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, trackletIndex);

#else
            addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex);
#endif

        }
   }
}

__global__ void createTrackletsWithAGapInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    //outer kernel for proposal 1
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    dim3 nThreads(1,16,16);
    dim3 nBlocks((MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) % nThreads.x  == 0 ? (MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) / nThreads.x : (MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) / nThreads.x + 1 ,nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,N_MAX_SEGMENTS_PER_MODULE % nThreads.z == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.z : N_MAX_SEGMENTS_PER_MODULE/nThreads.z + 1);

    createTrackletsWithAGapFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex,nInnerSegments,innerInnerLowerModuleArrayIndex);

}

/*__global__ void createTrackletsWithAGapFromMiddleLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int middleLowerModuleIndex, unsigned int innerSegmentIndex, unsigned int innerInnerLowerModuleArrayIndex, unsigned int nOuterInnerLowerModules,unsigned int innerOuterLowerModuleIndex)
{
    //Inner kernel of Proposal 2 : Inner kernel does middle->outer modoule mapping
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //check for outerInnerLowerModules
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;


    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[middleLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex and innerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm

   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;

   bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut); //might want to send the other two module indices and the anchor hits also to save memory accesses
   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;

        addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex);
   }
}

__global__ void createTrackletsWithAGapFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    //Middle kernel of Proposal 2 : middle kernel does the inner->middle module mapping

    int middleLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if(innerSegmentArrayIndex >= nInnerSegments) return;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

    //middle lower module - modules that are connected to outer lower module of inner segment
    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];
    unsigned int nMiddleLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];

    if(middleLowerModuleArrayIndex >= nMiddleLowerModules) return;

    unsigned int middleLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + middleLowerModuleArrayIndex];

    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[middleLowerModuleIndex];

    dim3 nThreads(1,32,1);
    dim3 nBlocks(nOuterInnerLowerModules % nThreads.x  == 0 ? nOuterInnerLowerModules / nThreads.x : nOuterInnerLowerModules / nThreads.x + 1 ,N_MAX_SEGMENTS_PER_MODULE % nThreads.y == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.y : N_MAX_SEGMENTS_PER_MODULE/nThreads.y + 1,1);

    createTrackletsWithAGapFromMiddleLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex, middleLowerModuleIndex,innerSegmentIndex,innerInnerLowerModuleArrayIndex,nOuterInnerLowerModules,innerOuterLowerModuleIndex);


}


__global__ void createTrackletsWithAGapInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    //outer kernel for proposal 2
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    dim3 nThreads(1,1,1);
    dim3 nBlocks(MAX_CONNECTED_MODULES % nThreads.x  == 0 ? MAX_CONNECTED_MODULES / nThreads.x : MAX_CONNECTED_MODULES / nThreads.x + 1 , nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,1);

    createTrackletsWithAGapFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, trackletsInGPU, innerInnerLowerModuleIndex, nInnerSegments, innerInnerLowerModuleArrayIndex);

}*/

#ifndef NESTED_PARA
#ifdef NEWGRID_Trips
__global__ void createTripletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, unsigned int *index_gpu)
{
//int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int innerInnerLowerModuleArrayIndex = index_gpu[blockIdx.z * blockDim.z + threadIdx.z];
  if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;

  unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
  unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerInnerLowerModuleIndex];
  if(nConnectedModules == 0) return;

  unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex];

  int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  int outerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if(innerSegmentArrayIndex >= nInnerSegments) return;

  unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

  //middle lower module - outer lower module of inner segment
  unsigned int middleLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

  unsigned int nOuterSegments = segmentsInGPU.nSegments[middleLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[middleLowerModuleIndex];
  if(outerSegmentArrayIndex >= nOuterSegments) return;

  unsigned int outerSegmentIndex = middleLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
  unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

  float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;
#ifdef CUT_VALUE_DEBUG
  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    bool success = runTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
#else

  bool success = runTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut);
#endif

  if(success)
    {
      unsigned int tripletModuleIndex = atomicAdd(&tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex], 1);
      if(tripletModuleIndex >= N_MAX_TRIPLETS_PER_MODULE)
      {
          if(tripletModuleIndex == N_MAX_TRIPLETS_PER_MODULE)
              printf("Triplet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
      }
      unsigned int tripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + tripletModuleIndex;
#ifdef CUT_VALUE_DEBUG

        addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, zLo,zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, tripletIndex);

#else
      addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, tripletIndex);
#endif
    }
}
#endif
#else
__global__ void createTripletsFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int nConnectedModules, unsigned int innerInnerLowerModuleArrayIndex)
{
    int innerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if(innerSegmentArrayIndex >= nInnerSegments) return;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

    //middle lower module - outer lower module of inner segment
    unsigned int middleLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[middleLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[middleLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;
    unsigned int outerSegmentIndex = middleLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;
#ifdef CUT_VALUE_DEBUG
  float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    bool success = runTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
#else

  bool success = runTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut);
#endif

    if(success)
    {
        unsigned int tripletModuleIndex = atomicAdd(&tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex], 1);
        if(tripletModuleIndex >= N_MAX_TRIPLETS_PER_MODULE)
        {
            if(tripletModuleIndex == N_MAX_TRIPLETS_PER_MODULE)
                printf("Triplet excess alert! Module index = %d\n",innerInnerLowerModuleIndex);
        }
        else
        {
            unsigned int tripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + tripletModuleIndex;
#ifdef CUT_VALUE_DEBUG

            addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, zLo,zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ, tripletIndex);

#else
        addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, tripletIndex);
#endif

        }
    }
}

__global__ void createTripletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU)
{
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex] > N_MAX_SEGMENTS_PER_MODULE ? N_MAX_SEGMENTS_PER_MODULE : segmentsInGPU.nSegments[innerInnerLowerModuleIndex] ;
    if(nInnerSegments == 0) return;

    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerInnerLowerModuleIndex];
    if(nConnectedModules == 0) return;

    dim3 nThreads(16,16,1);
    dim3 nBlocks(nInnerSegments % nThreads.x == 0 ? nInnerSegments / nThreads.x : nInnerSegments / nThreads.x + 1, N_MAX_SEGMENTS_PER_MODULE % nThreads.y == 0 ? N_MAX_SEGMENTS_PER_MODULE / nThreads.y : N_MAX_SEGMENTS_PER_MODULE / nThreads.y + 1);

    createTripletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, innerInnerLowerModuleIndex, nInnerSegments, nConnectedModules, innerInnerLowerModuleArrayIndex);
}
#endif

#ifndef NESTED_PARA
__global__ void createTrackCandidatesInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, unsigned int* index_gpu)
{
  //inner tracklet/triplet inner segment inner MD lower module
  int innerInnerInnerLowerModuleArrayIndex = index_gpu[blockIdx.z * blockDim.z + threadIdx.z];
  //hack to include pixel detector
  if(innerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules + 1) return;

  unsigned int nTracklets = trackletsInGPU.nTracklets[innerInnerInnerLowerModuleArrayIndex];
  if(innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules)
    {
      if(nTracklets > N_MAX_PIXEL_TRACKLETS_PER_MODULE)
        nTracklets = N_MAX_PIXEL_TRACKLETS_PER_MODULE;
    }
  else
    {
      if(nTracklets > N_MAX_TRACKLETS_PER_MODULE)
        nTracklets = N_MAX_TRACKLETS_PER_MODULE;
    }
  unsigned int nTriplets = innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules ? 0 : tripletsInGPU.nTriplets[innerInnerInnerLowerModuleArrayIndex]; // should be zero for the pixels
  if(nTriplets > N_MAX_TRIPLETS_PER_MODULE)
    nTriplets = N_MAX_TRIPLETS_PER_MODULE;

  unsigned int temp = max(nTracklets,nTriplets);
  unsigned int MAX_OBJECTS = max(N_MAX_TRACKLETS_PER_MODULE, N_MAX_TRIPLETS_PER_MODULE);

  if(temp == 0) return;

  int innerObjectArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
  int outerObjectArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;

  int innerObjectIndex = 0;
  int outerObjectIndex = 0;
  short trackCandidateType;
  bool success;
  //step 1 tracklet-tracklet
  if(innerObjectArrayIndex < nTracklets)
    {
      innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
      unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];/*same as innerOuterInnerLowerModuleIndex*/

      if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {

          outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;

          success = runTrackCandidateDefaultAlgoTwoTracklets(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);

          if(success)
            {
              unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
              atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[innerInnerInnerLowerModuleArrayIndex],1);
              if((innerInnerInnerLowerModuleArrayIndex < *modulesInGPU.nLowerModules  && trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE) || (innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE))
                {
                  if((innerInnerInnerLowerModuleArrayIndex < *modulesInGPU.nLowerModules  && trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE) || (innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE))

                    printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                }
              else
                {
                  //                  unsigned int trackCandidateIdx = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACK_CANDIDATES_PER_MODULE + trackCandidateModuleIdx;
                  if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                      printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);

                    }
                  else
                    {
                      unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
                      addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);

                    }

                }
            }

        }
    }

  //step 2 tracklet-triplet
  if(innerObjectArrayIndex < nTracklets)
    {
      innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
      unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];//same as innerOuterInnerLowerModuleIndex
  if(outerObjectArrayIndex < fminf(tripletsInGPU.nTriplets[outerInnerInnerLowerModuleIndex],N_MAX_TRIPLETS_PER_MODULE))
    {
      outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRIPLETS_PER_MODULE + outerObjectArrayIndex;
      success = runTrackCandidateDefaultAlgoTrackletToTriplet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
      if(success)
	{
	  unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
	  atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T3[innerInnerInnerLowerModuleArrayIndex],1);
	  if((innerInnerInnerLowerModuleArrayIndex < *modulesInGPU.nLowerModules  && trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE) || (innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE))
	    {
	      if((innerInnerInnerLowerModuleArrayIndex < *modulesInGPU.nLowerModules  && trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE) || (innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE))
		printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
	    }
	  else
	    {

	      //                    unsigned int trackCandidateIdx = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACK_CANDIDATES_PER_MODULE + trackCandidateModuleIdx;
	      if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
		{
		  printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
		}
	      else
		{
		  unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;

		  addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);
		}
	    }
	}

    }
    }

  //step 3 triplet-tracklet
  if(innerObjectArrayIndex < nTriplets)
    {
      innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + innerObjectArrayIndex;
      unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletsInGPU.lowerModuleIndices[3 * innerObjectIndex + 1]];//sameas innerOuterInnerLowerModuleIndex

   if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
     {
       outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;
       success = runTrackCandidateDefaultAlgoTripletToTracklet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
       if(success)
	 {
	   unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
	   atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT3T4[innerInnerInnerLowerModuleArrayIndex],1);
	   if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
	     {
	       if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
		 printf("Track Candidate excess alert! Module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
	     }
	   else
	     {
	       unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
	       addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);

	     }
	 }
     }

    }
}

#else

__global__ void createTrackCandidatesInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU)
{
    //inner tracklet/triplet inner segment inner MD lower module
    int innerInnerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //hack to include pixel detector
    if(innerInnerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules + 1) return;

    unsigned int nTracklets = trackletsInGPU.nTracklets[innerInnerInnerLowerModuleArrayIndex];
    if(innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules)
    {
        if(nTracklets > N_MAX_PIXEL_TRACKLETS_PER_MODULE)
            nTracklets = N_MAX_PIXEL_TRACKLETS_PER_MODULE;
    }
    else
    {
        if(nTracklets > N_MAX_TRACKLETS_PER_MODULE)
            nTracklets = N_MAX_TRACKLETS_PER_MODULE;
    }
    unsigned int nTriplets = innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules ? 0 : tripletsInGPU.nTriplets[innerInnerInnerLowerModuleArrayIndex]; // should be zero for the pixels
    if(nTriplets > N_MAX_TRIPLETS_PER_MODULE)
        nTriplets = N_MAX_TRIPLETS_PER_MODULE;

    unsigned int temp = max(nTracklets,nTriplets);
    unsigned int MAX_OBJECTS = max(N_MAX_TRACKLETS_PER_MODULE, N_MAX_TRIPLETS_PER_MODULE);

    if(temp == 0) return;

    //triplets and tracklets are stored directly using lower module array index
    dim3 nThreads(16,16,1);
    dim3 nBlocks(temp % nThreads.x == 0 ? temp / nThreads.x : temp / nThreads.x + 1, MAX_OBJECTS % nThreads.y == 0 ? MAX_OBJECTS / nThreads.y : MAX_OBJECTS / nThreads.y + 1, 1);

    createTrackCandidatesFromInnerInnerInnerLowerModule<<<nBlocks, nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, trackletsInGPU, tripletsInGPU, trackCandidatesInGPU,innerInnerInnerLowerModuleArrayIndex,nTracklets,nTriplets);
}

__global__ void createTrackCandidatesFromInnerInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, unsigned int innerInnerInnerLowerModuleArrayIndex, unsigned int nInnerTracklets, unsigned int nInnerTriplets)
{
    int innerObjectArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerObjectArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

    int innerObjectIndex = 0;
    int outerObjectIndex = 0;
    short trackCandidateType;
    bool success;
    //step 1 tracklet-tracklet
    if(innerObjectArrayIndex < nInnerTracklets)
    {
        innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
        unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];/*same as innerOuterInnerLowerModuleIndex*/

        if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {

            outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;

            success = runTrackCandidateDefaultAlgoTwoTracklets(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);

            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T4[innerInnerInnerLowerModuleArrayIndex],1);
                if((innerInnerInnerLowerModuleArrayIndex < *modulesInGPU.nLowerModules  && trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE) || (innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE))
                {
                    if((innerInnerInnerLowerModuleArrayIndex < *modulesInGPU.nLowerModules  && trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE) || (innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE))

                        printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                }
                else
                {
//		    unsigned int trackCandidateIdx = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACK_CANDIDATES_PER_MODULE + trackCandidateModuleIdx;
                    if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                       printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);

                    }
                    else
                   {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
                        addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);
                    }

                }
            }

        }
    }
    //step 2 tracklet-triplet
    if(innerObjectArrayIndex < nInnerTracklets)
    {
        innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + innerObjectArrayIndex;
        unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[trackletsInGPU.lowerModuleIndices[4 * innerObjectIndex + 2]];//same as innerOuterInnerLowerModuleIndex
        if(outerObjectArrayIndex < fminf(tripletsInGPU.nTriplets[outerInnerInnerLowerModuleIndex],N_MAX_TRIPLETS_PER_MODULE))
        {
            outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRIPLETS_PER_MODULE + outerObjectArrayIndex;
            success = runTrackCandidateDefaultAlgoTrackletToTriplet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT4T3[innerInnerInnerLowerModuleArrayIndex],1);
                if((innerInnerInnerLowerModuleArrayIndex < *modulesInGPU.nLowerModules  && trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE) || (innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx >= N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE))
                {
                    if((innerInnerInnerLowerModuleArrayIndex < *modulesInGPU.nLowerModules  && trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE) || (innerInnerInnerLowerModuleArrayIndex == *modulesInGPU.nLowerModules && trackCandidateModuleIdx == N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE))
                        printf("Track Candidate excess alert! lower Module array index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                }
                else
                {

//                    unsigned int trackCandidateIdx = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACK_CANDIDATES_PER_MODULE + trackCandidateModuleIdx;
                    if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                        printf("Track candidates: no memory for module at module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                    }
                    else
                    {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;

                        addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);
                    }
                }
            }

        }
    }

    //step 3 triplet-tracklet
    if(innerObjectArrayIndex < nInnerTriplets)
    {
        innerObjectIndex = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + innerObjectArrayIndex;
        unsigned int outerInnerInnerLowerModuleIndex = modulesInGPU.reverseLookupLowerModuleIndices[tripletsInGPU.lowerModuleIndices[3 * innerObjectIndex + 1]];//same as innerOuterInnerLowerModuleIndex

        if(outerObjectArrayIndex < fminf(trackletsInGPU.nTracklets[outerInnerInnerLowerModuleIndex],N_MAX_TRACKLETS_PER_MODULE))
        {
            outerObjectIndex = outerInnerInnerLowerModuleIndex * N_MAX_TRACKLETS_PER_MODULE + outerObjectArrayIndex;
            success = runTrackCandidateDefaultAlgoTripletToTracklet(trackletsInGPU, tripletsInGPU, innerObjectIndex, outerObjectIndex,trackCandidateType);
            if(success)
            {
                unsigned int trackCandidateModuleIdx = atomicAdd(&trackCandidatesInGPU.nTrackCandidates[innerInnerInnerLowerModuleArrayIndex],1);
                atomicAdd(&trackCandidatesInGPU.nTrackCandidatesT3T4[innerInnerInnerLowerModuleArrayIndex],1);
	        if(trackCandidateModuleIdx >= N_MAX_TRACK_CANDIDATES_PER_MODULE)
                {
                   if(trackCandidateModuleIdx == N_MAX_TRACK_CANDIDATES_PER_MODULE)
                       printf("Track Candidate excess alert! Module index = %d\n",innerInnerInnerLowerModuleArrayIndex);
                }
                else
                {
//              	    unsigned int trackCandidateIdx = innerInnerInnerLowerModuleArrayIndex * N_MAX_TRACK_CANDIDATES_PER_MODULE + trackCandidateModuleIdx;
                    if(modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1)
                    {
                        printf("Track candidates: no memory for module at module index = %d, outer T4 module index = %d\n",innerInnerInnerLowerModuleArrayIndex, outerInnerInnerLowerModuleIndex);
                    }
                    else
                    {
                        unsigned int trackCandidateIdx = modulesInGPU.trackCandidateModuleIndices[innerInnerInnerLowerModuleArrayIndex] + trackCandidateModuleIdx;
                        addTrackCandidateToMemory(trackCandidatesInGPU, trackCandidateType, innerObjectIndex, outerObjectIndex, trackCandidateIdx);

                    }
                }
            }

        }
    }
}
#endif

unsigned int SDL::Event::getNumberOfHits()
{
    unsigned int hits = 0;
    for(auto &it:n_hits_by_layer_barrel_)
    {
        hits += it;
    }
    for(auto& it:n_hits_by_layer_endcap_)
    {
        hits += it;
    }

    return hits;
}

unsigned int SDL::Event::getNumberOfHitsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_hits_by_layer_barrel_[layer];
    else
        return n_hits_by_layer_barrel_[layer] + n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerBarrel(unsigned int layer)
{
    return n_hits_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerEndcap(unsigned int layer)
{
    return n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoublets()
{
     unsigned int miniDoublets = 0;
    for(auto &it:n_minidoublets_by_layer_barrel_)
    {
        miniDoublets += it;
    }
    for(auto &it:n_minidoublets_by_layer_endcap_)
    {
        miniDoublets += it;
    }

    return miniDoublets;

}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_minidoublets_by_layer_barrel_[layer];
    else
        return n_minidoublets_by_layer_barrel_[layer] + n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerBarrel(unsigned int layer)
{
    return n_minidoublets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerEndcap(unsigned int layer)
{
    return n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegments()
{
    unsigned int segments = 0;
    for(auto &it:n_segments_by_layer_barrel_)
    {
        segments += it;
    }
    for(auto &it:n_segments_by_layer_endcap_)
    {
        segments += it;
    }

    return segments;

}

unsigned int SDL::Event::getNumberOfSegmentsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_segments_by_layer_barrel_[layer];
    else
        return n_segments_by_layer_barrel_[layer] + n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerBarrel(unsigned int layer)
{
    return n_segments_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerEndcap(unsigned int layer)
{
    return n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTracklets()
{
    unsigned int tracklets = 0;
    for(auto &it:n_tracklets_by_layer_barrel_)
    {
        tracklets += it;
    }
    for(auto &it:n_tracklets_by_layer_endcap_)
    {
        tracklets += it;
    }

    return tracklets;

}

unsigned int SDL::Event::getNumberOfTrackletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_tracklets_by_layer_barrel_[layer];
    else
        return n_tracklets_by_layer_barrel_[layer] + n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackletsByLayerBarrel(unsigned int layer)
{
    return n_tracklets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTrackletsByLayerEndcap(unsigned int layer)
{
    return n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTriplets()
{
    unsigned int triplets = 0;
    for(auto &it:n_triplets_by_layer_barrel_)
    {
        triplets += it;
    }
    for(auto &it:n_triplets_by_layer_endcap_)
    {
        triplets += it;
    }

    return triplets;

}


unsigned int SDL::Event::getNumberOfTripletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_triplets_by_layer_barrel_[layer];
    else
        return n_triplets_by_layer_barrel_[layer] + n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerBarrel(unsigned int layer)
{
    return n_triplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerEndcap(unsigned int layer)
{
    return n_triplets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidates()
{
    unsigned int trackCandidates = 0;
    for(auto &it:n_trackCandidates_by_layer_barrel_)
    {
        trackCandidates += it;
    }
    for(auto &it:n_trackCandidates_by_layer_endcap_)
    {
        trackCandidates += it;
    }

    //hack - add pixel track candidate multiplicity
    trackCandidates += getNumberOfPixelTrackCandidates();

    return trackCandidates;

}

unsigned int SDL::Event::getNumberOfPixelTrackCandidates()
{
#ifdef Explicit_Track
    unsigned int nLowerModules;// = *(SDL::modulesInGPU->nLowerModules);
    hipMemcpy(&nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    unsigned int nTrackCandidatesInPixelModule;
    hipMemcpy(&nTrackCandidatesInPixelModule,&trackCandidatesInGPU->nTrackCandidates[nLowerModules],sizeof(unsigned int),hipMemcpyDeviceToHost);
    return nTrackCandidatesInPixelModule;
#else
    return trackCandidatesInGPU->nTrackCandidates[*(modulesInGPU->nLowerModules)];
#endif

}
unsigned int SDL::Event::getNumberOfTrackCandidatesByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_trackCandidates_by_layer_barrel_[layer];
    else
        return n_trackCandidates_by_layer_barrel_[layer] + n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidatesByLayerBarrel(unsigned int layer)
{
    return n_trackCandidates_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTrackCandidatesByLayerEndcap(unsigned int layer)
{
    return n_trackCandidates_by_layer_endcap_[layer];
}

#ifdef Explicit_Hit
SDL::hits* SDL::Event::getHits() //std::shared_ptr should take care of garbage collection
{
    if(hitsInCPU == nullptr)
    {
        hitsInCPU = new SDL::hits;
        hitsInCPU->nHits = new unsigned int;
        unsigned int nHits;
        hipMemcpy(&nHits, hitsInGPU->nHits, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hitsInCPU->idxs = new unsigned int[nHits];
        hipMemcpy(hitsInCPU->idxs, hitsInGPU->idxs,sizeof(unsigned int) * nHits, hipMemcpyDeviceToHost);
    }
    return hitsInCPU;
}
#else
SDL::hits* SDL::Event::getHits() //std::shared_ptr should take care of garbage collection
{
    return hitsInGPU;
}
#endif


#ifdef Explicit_MD
SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    if(mdsInCPU == nullptr)
    {
        mdsInCPU = new SDL::miniDoublets;
        unsigned int nMemoryLocations = (N_MAX_MD_PER_MODULES * (nModules - 1) + N_MAX_PIXEL_MD_PER_MODULES);
        mdsInCPU->hitIndices = new unsigned int[2 * nMemoryLocations];
        mdsInCPU->nMDs = new unsigned int[nModules];
        hipMemcpy(mdsInCPU->hitIndices, mdsInGPU->hitIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(mdsInCPU->nMDs, mdsInGPU->nMDs, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return mdsInCPU;
}
#else
SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    return mdsInGPU;
}
#endif


#ifdef Explicit_Seg
SDL::segments* SDL::Event::getSegments()
{
    if(segmentsInCPU == nullptr)
    {
        segmentsInCPU = new SDL::segments;
        unsigned int nMemoryLocations = (N_MAX_SEGMENTS_PER_MODULE) * (nModules - 1) + N_MAX_PIXEL_SEGMENTS_PER_MODULE;
        segmentsInCPU->mdIndices = new unsigned int[2 * nMemoryLocations];
        segmentsInCPU->nSegments = new unsigned int[nModules];
        hipMemcpy(segmentsInCPU->mdIndices, segmentsInGPU->mdIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(segmentsInCPU->nSegments, segmentsInGPU->nSegments, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return segmentsInCPU;
}
#else
SDL::segments* SDL::Event::getSegments()
{
    return segmentsInGPU;
}
#endif

#ifdef Explicit_Tracklet
SDL::tracklets* SDL::Event::getTracklets()
{
    if(trackletsInCPU == nullptr)
    {
        unsigned int nLowerModules;
        trackletsInCPU = new SDL::tracklets;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = (N_MAX_TRACKLETS_PER_MODULE) * nLowerModules + N_MAX_PIXEL_TRACKLETS_PER_MODULE;
        trackletsInCPU->segmentIndices = new unsigned int[2 * nMemoryLocations];
        trackletsInCPU->nTracklets = new unsigned int[nLowerModules];
        hipMemcpy(trackletsInCPU->segmentIndices, trackletsInGPU->segmentIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(trackletsInCPU->nTracklets, trackletsInGPU->nTracklets, (nLowerModules + 1)* sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return trackletsInCPU;
}
#else
SDL::tracklets* SDL::Event::getTracklets()
{
    return trackletsInGPU;
}
#endif

#ifdef Explicit_Trips
SDL::triplets* SDL::Event::getTriplets()
{
    if(tripletsInCPU == nullptr)
    {
        unsigned int nLowerModules;
        tripletsInCPU = new SDL::triplets;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = (N_MAX_TRIPLETS_PER_MODULE) * (nLowerModules);
        tripletsInCPU->segmentIndices = new unsigned[2 * nMemoryLocations];
        tripletsInCPU->nTriplets = new unsigned int[nLowerModules];
        hipMemcpy(tripletsInCPU->segmentIndices, tripletsInGPU->segmentIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(tripletsInCPU->nTriplets, tripletsInGPU->nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return tripletsInCPU;
}
#else
SDL::triplets* SDL::Event::getTriplets()
{
    return tripletsInGPU;
}
#endif

#ifdef Explicit_Track
SDL::trackCandidates* SDL::Event::getTrackCandidates()
{
    if(trackCandidatesInCPU == nullptr)
    {
        trackCandidatesInCPU = new SDL::trackCandidates;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nEligibleModules;
        hipMemcpy(&nEligibleModules, modulesInGPU->nEligibleModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        unsigned int nMemoryLocations = (N_MAX_TRACK_CANDIDATES_PER_MODULE) * (nEligibleModules -1) + (N_MAX_PIXEL_TRACK_CANDIDATES_PER_MODULE);

        trackCandidatesInCPU->objectIndices = new unsigned int[2 * nMemoryLocations];
        trackCandidatesInCPU->trackCandidateType = new short[nMemoryLocations];
        trackCandidatesInCPU->nTrackCandidates = new unsigned int[nLowerModules+1];
        hipMemcpy(trackCandidatesInCPU->objectIndices, trackCandidatesInGPU->objectIndices, 2 * nMemoryLocations * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(trackCandidatesInCPU->trackCandidateType, trackCandidatesInGPU->trackCandidateType, nMemoryLocations * sizeof(short), hipMemcpyDeviceToHost);
        hipMemcpy(trackCandidatesInCPU->nTrackCandidates, trackCandidatesInGPU->nTrackCandidates, (nLowerModules + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    return trackCandidatesInCPU;
}
#else
SDL::trackCandidates* SDL::Event::getTrackCandidates()
{
    return trackCandidatesInGPU;
}
#endif
#ifdef Explicit_Module
SDL::modules* SDL::Event::getFullModules()
{
    if(modulesInCPUFull == nullptr)
    {
        modulesInCPUFull = new SDL::modules;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);

    modulesInCPUFull->detIds = new unsigned int[nModules];
    modulesInCPUFull->moduleMap = new unsigned int[40*nModules];
    modulesInCPUFull->nConnectedModules = new unsigned int[nModules];
    modulesInCPUFull->drdzs = new float[nModules];
    modulesInCPUFull->slopes = new float[nModules];
    modulesInCPUFull->nModules = new unsigned int[1];
    modulesInCPUFull->nLowerModules = new unsigned int[1];
    modulesInCPUFull->layers = new short[nModules];
    modulesInCPUFull->rings = new short[nModules];
    modulesInCPUFull->modules = new short[nModules];
    modulesInCPUFull->rods = new short[nModules];
    modulesInCPUFull->subdets = new short[nModules];
    modulesInCPUFull->sides = new short[nModules];
    modulesInCPUFull->isInverted = new bool[nModules];
    modulesInCPUFull->isLower = new bool[nModules];

    modulesInCPUFull->hitRanges = new int[2*nModules];
    modulesInCPUFull->mdRanges = new int[2*nModules];
    modulesInCPUFull->segmentRanges = new int[2*nModules];
    modulesInCPUFull->trackletRanges = new int[2*nModules];
    modulesInCPUFull->tripletRanges = new int[2*nModules];
    modulesInCPUFull->trackCandidateRanges = new int[2*nModules];

    modulesInCPUFull->moduleType = new ModuleType[nModules];
    modulesInCPUFull->moduleLayerType = new ModuleLayerType[nModules];

    modulesInCPUFull->lowerModuleIndices = new unsigned int[nLowerModules+1];
    modulesInCPUFull->reverseLookupLowerModuleIndices = new int[nModules];
    modulesInCPUFull->trackCandidateModuleIndices = new int[nLowerModules+1];

    hipMemcpy(modulesInCPUFull->detIds,modulesInGPU->detIds,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->moduleMap,modulesInGPU->moduleMap,40*nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->nConnectedModules,modulesInGPU->nConnectedModules,nModules*sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->drdzs,modulesInGPU->drdzs,sizeof(float)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->slopes,modulesInGPU->slopes,sizeof(float)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->nLowerModules,modulesInGPU->nLowerModules,sizeof(unsigned int),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->layers,modulesInGPU->layers,nModules*sizeof(short),hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->rings,modulesInGPU->rings,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->modules,modulesInGPU->modules,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->rods,modulesInGPU->rods,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->subdets,modulesInGPU->subdets,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->sides,modulesInGPU->sides,sizeof(short)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->isInverted,modulesInGPU->isInverted,sizeof(bool)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->isLower,modulesInGPU->isLower,sizeof(bool)*nModules,hipMemcpyDeviceToHost);

    hipMemcpy(modulesInCPUFull->hitRanges, modulesInGPU->hitRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->mdRanges, modulesInGPU->mdRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->segmentRanges, modulesInGPU->segmentRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->trackletRanges, modulesInGPU->trackletRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->tripletRanges, modulesInGPU->tripletRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->trackCandidateRanges, modulesInGPU->trackCandidateRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(modulesInCPUFull->reverseLookupLowerModuleIndices, modulesInGPU->reverseLookupLowerModuleIndices, nModules * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->lowerModuleIndices, modulesInGPU->lowerModuleIndices, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->trackCandidateModuleIndices, modulesInGPU->trackCandidateModuleIndices, (nLowerModules+1) * sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(modulesInCPUFull->moduleType,modulesInGPU->moduleType,sizeof(ModuleType)*nModules,hipMemcpyDeviceToHost);
    hipMemcpy(modulesInCPUFull->moduleLayerType,modulesInGPU->moduleLayerType,sizeof(ModuleLayerType)*nModules,hipMemcpyDeviceToHost);
    }
    return modulesInCPUFull;
}
SDL::modules* SDL::Event::getModules()
{
    //if(modulesInCPU == nullptr)
    //{
        modulesInCPU = new SDL::modules;
        unsigned int nLowerModules;
        hipMemcpy(&nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);

        modulesInCPU->nLowerModules = new unsigned int[1];
        modulesInCPU->nModules = new unsigned int[1];
        modulesInCPU->lowerModuleIndices = new unsigned int[nLowerModules+1];
        modulesInCPU->detIds = new unsigned int[nModules];
        modulesInCPU->hitRanges = new int[2*nModules];
        modulesInCPU->isLower = new bool[nModules];
        modulesInCPU->trackCandidateModuleIndices = new int[nLowerModules+1];

        hipMemcpy(modulesInCPU->nLowerModules, modulesInGPU->nLowerModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->nModules, modulesInGPU->nModules, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->lowerModuleIndices, modulesInGPU->lowerModuleIndices, (nLowerModules+1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->detIds, modulesInGPU->detIds, nModules * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->hitRanges, modulesInGPU->hitRanges, 2*nModules * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->isLower, modulesInGPU->isLower, nModules * sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(modulesInCPU->trackCandidateModuleIndices, modulesInGPU->trackCandidateModuleIndices, (nLowerModules+1) * sizeof(int), hipMemcpyDeviceToHost);
    //}
    return modulesInCPU;
}
#else
SDL::modules* SDL::Event::getModules()
{
    return modulesInGPU;
}
SDL::modules* SDL::Event::getFullModules()
{
    return modulesInGPU;
}
#endif
