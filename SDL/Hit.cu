#include "hip/hip_runtime.h"
# include "Hit.cuh"
# include "allocate.h"

SDL::hits::hits()
{
    nHits = nullptr;
//    n2SHits = nullptr;
    xs = nullptr;
    ys = nullptr;
    zs = nullptr;
    moduleIndices = nullptr;
    detid = nullptr;
    rts = nullptr;
    phis = nullptr;
    etas = nullptr;
    highEdgeXs = nullptr;
    highEdgeYs = nullptr;
    lowEdgeXs = nullptr;
    lowEdgeYs = nullptr;
    hitRanges = nullptr;
    hitRangesLower = nullptr;
    hitRangesUpper = nullptr;
    hitRangesnLower = nullptr;
    hitRangesnUpper = nullptr;
}

SDL::hits::~hits()
{
}
void SDL::createHitsInExplicitMemory(struct hits& hitsInGPU, int nModules, unsigned int nMaxHits,hipStream_t stream,unsigned int evtnum)
{
//#ifdef CACHE_ALLOC
#if defined(CACHE_ALLOC)
 //   hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    hitsInGPU.xs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.ys = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.zs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);

    hitsInGPU.rts = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.phis = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.etas = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);

    hitsInGPU.moduleIndices = (uint16_t*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(uint16_t),stream);
    hitsInGPU.idxs = (unsigned int*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(unsigned int),stream);
    hitsInGPU.detid = (unsigned int*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(unsigned int),stream);

    hitsInGPU.highEdgeXs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.highEdgeYs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.lowEdgeXs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.lowEdgeYs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);

    hitsInGPU.nHits = (unsigned int*)cms::cuda::allocate_device(dev,evtnum*sizeof(unsigned int),stream);

    hitsInGPU.hitRanges =                  (int*)cms::cuda::allocate_device(dev,         evtnum*nModules * 2 * sizeof(int),stream);
    hitsInGPU.hitRangesLower =                  (int*)cms::cuda::allocate_device(dev,    evtnum*nModules * sizeof(int),stream);
    hitsInGPU.hitRangesUpper =                  (int*)cms::cuda::allocate_device(dev,    evtnum*nModules * sizeof(int),stream);
    hitsInGPU.hitRangesnLower =                  (int8_t*)cms::cuda::allocate_device(dev,evtnum*nModules * sizeof(int8_t),stream);
    hitsInGPU.hitRangesnUpper =                  (int8_t*)cms::cuda::allocate_device(dev,evtnum*nModules * sizeof(int8_t),stream);
#else
    hipMalloc(&hitsInGPU.xs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.ys, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.zs, nMaxHits * sizeof(float));

    hipMalloc(&hitsInGPU.moduleIndices, nMaxHits * sizeof(uint16_t));
    hipMalloc(&hitsInGPU.idxs, nMaxHits * sizeof(unsigned int));
    hipMalloc(&hitsInGPU.detid, nMaxHits * sizeof(unsigned int));

    hipMalloc(&hitsInGPU.rts, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.phis, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.etas, nMaxHits * sizeof(float));

    hipMalloc(&hitsInGPU.highEdgeXs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.highEdgeYs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.lowEdgeXs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.lowEdgeYs, nMaxHits * sizeof(float));

    //counters
    hipMalloc(&hitsInGPU.nHits,evtnum* sizeof(unsigned int));

    hipMalloc(&hitsInGPU.hitRanges,evtnum*nModules * 2 * sizeof(int));
    hipMalloc(&hitsInGPU.hitRangesLower,evtnum*nModules  * sizeof(int));
    hipMalloc(&hitsInGPU.hitRangesUpper,evtnum*nModules  * sizeof(int));
    hipMalloc(&hitsInGPU.hitRangesnLower,evtnum*nModules  * sizeof(int8_t));
    hipMalloc(&hitsInGPU.hitRangesnUpper,evtnum* nModules  * sizeof(int8_t));
#endif
    hipMemsetAsync(hitsInGPU.nHits,0,evtnum*sizeof(unsigned int),stream);
    hipMemsetAsync(hitsInGPU.hitRanges, -1,      evtnum*nModules*2*sizeof(int),stream);
    hipMemsetAsync(hitsInGPU.hitRangesLower, -1, evtnum*nModules*sizeof(int),stream);
    hipMemsetAsync(hitsInGPU.hitRangesUpper, -1, evtnum*nModules*sizeof(int),stream);
    hipMemsetAsync(hitsInGPU.hitRangesnLower, -1,evtnum*nModules*sizeof(int8_t),stream);
    hipMemsetAsync(hitsInGPU.hitRangesnUpper, -1,evtnum*nModules*sizeof(int8_t),stream);
    hipStreamSynchronize(stream);
}

__global__ void SDL::addHitToMemoryKernel(struct hits& hitsInGPU, struct modules& modulesInGPU,const float* x,const  float* y,const  float* z, const uint16_t* moduleIndex,const float* phis, const int loopsize)
{
  for (unsigned int ihit = blockIdx.x*blockDim.x + threadIdx.x; ihit <loopsize; ihit += blockDim.x*gridDim.x)
  //if(ihit < loopsize)
  {
      unsigned int idx = ihit;//*(hitsInGPU.nHits);

      hitsInGPU.xs[idx] = x[ihit];
      hitsInGPU.ys[idx] = y[ihit];
      hitsInGPU.zs[idx] = z[ihit];
      hitsInGPU.rts[idx] = sqrt(x[ihit]*x[ihit] + y[ihit]*y[ihit]);
      hitsInGPU.phis[idx] = phi(x[ihit],y[ihit],z[ihit]);
      hitsInGPU.moduleIndices[idx] = moduleIndex[ihit];
      hitsInGPU.idxs[idx] = ihit;
  }
}

ALPAKA_FN_ACC void SDL::getEdgeHitsK(float phi,float x, float y, float& xhigh, float& yhigh, float& xlow, float& ylow)
{
    xhigh = x + 2.5 * cos(phi);
    yhigh = y + 2.5 * sin(phi);
    xlow = x - 2.5 * cos(phi);
    ylow = y - 2.5 * sin(phi);
}
void SDL::getEdgeHits(unsigned int detId,float x, float y, float& xhigh, float& yhigh, float& xlow, float& ylow)
{
    float phi = endcapGeometry.getCentroidPhi(detId);
    xhigh = x + 2.5 * cos(phi);
    yhigh = y + 2.5 * sin(phi);
    xlow = x - 2.5 * cos(phi);
    ylow = y - 2.5 * sin(phi);
}

void SDL::printHit(struct hits& hitsInGPU, struct modules& modulesInGPU, unsigned int hitIndex)
{
    std::cout << "Hit(x=" << hitsInGPU.xs[hitIndex] << ", y=" << hitsInGPU.ys[hitIndex] << ", z=" << hitsInGPU.zs[hitIndex] << ", rt=" << hitsInGPU.rts[hitIndex] << ", phi=" << hitsInGPU.phis[hitIndex] <<", module subdet = "<<modulesInGPU.subdets[hitsInGPU.moduleIndices[hitIndex]]<<", module layer = "<< modulesInGPU.layers[hitsInGPU.moduleIndices[hitIndex]]<<", module ring = "<< modulesInGPU.rings[hitsInGPU.moduleIndices[hitIndex]]<<" )"<<std::endl;
}


void SDL::hits::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,nHits);
    cms::cuda::free_device(dev,xs);
    cms::cuda::free_device(dev,ys);
    cms::cuda::free_device(dev,zs);
    cms::cuda::free_device(dev,moduleIndices);
    cms::cuda::free_device(dev,rts);
    cms::cuda::free_device(dev,idxs);
    cms::cuda::free_device(dev,detid);
    cms::cuda::free_device(dev,phis);
    cms::cuda::free_device(dev,etas);

    cms::cuda::free_device(dev,highEdgeXs);
    cms::cuda::free_device(dev,highEdgeYs);
    cms::cuda::free_device(dev,lowEdgeXs);
    cms::cuda::free_device(dev,lowEdgeYs);
    
    cms::cuda::free_device(dev,hitRanges);
    cms::cuda::free_device(dev,hitRangesLower);
    cms::cuda::free_device(dev,hitRangesnLower);
    cms::cuda::free_device(dev,hitRangesUpper);
    cms::cuda::free_device(dev,hitRangesnUpper);
}
void SDL::hits::freeMemory()
//void SDL::hits::freeMemory(hipStream_t stream)
{
    hipFree(nHits);
    hipFree(xs);
    hipFree(ys);
    hipFree(zs);
    hipFree(moduleIndices);
    hipFree(rts);
    hipFree(idxs);
    hipFree(detid);
    hipFree(phis);
    hipFree(etas);

    hipFree(highEdgeXs);
    hipFree(highEdgeYs);
    hipFree(lowEdgeXs);
    hipFree(lowEdgeYs);
    
    hipFree(hitRanges);
    hipFree(hitRangesLower);
    hipFree(hitRangesnLower);
    hipFree(hitRangesUpper);
    hipFree(hitRangesnUpper);
//    hipStreamSynchronize(stream);
}
