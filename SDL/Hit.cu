#include "hip/hip_runtime.h"
# include "Hit.cuh"
# include "allocate.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif

SDL::hits::hits()
{
    nHits = nullptr;
//    n2SHits = nullptr;
    xs = nullptr;
    ys = nullptr;
    zs = nullptr;
    moduleIndices = nullptr;
    rts = nullptr;
    phis = nullptr;
    etas = nullptr;
    highEdgeXs = nullptr;
    highEdgeYs = nullptr;
    lowEdgeXs = nullptr;
    lowEdgeYs = nullptr;
    hitRanges = nullptr;
    hitRangesLower = nullptr;
    hitRangesUpper = nullptr;
    hitRangesnLower = nullptr;
    hitRangesnUpper = nullptr;
}

SDL::hits::~hits()
{
}
//FIXME:New array!
void SDL::createHitsInUnifiedMemory(struct hits& hitsInGPU,unsigned int nMaxHits,unsigned int nMax2SHits,hipStream_t stream, unsigned int evtnum)
{
    int nModules= 26593;
//#ifdef CACHE_ALLOC
#if defined(CACHE_ALLOC) && !defined(Preload_hits)
//    hipStream_t stream=0;
    hitsInGPU.xs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.ys = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.zs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);

    hitsInGPU.rts = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.phis = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.etas = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);

    hitsInGPU.moduleIndices = (uint16_t*)cms::cuda::allocate_managed(nMaxHits*sizeof(uint16_t),stream);
    hitsInGPU.idxs = (unsigned int*)cms::cuda::allocate_managed(nMaxHits*sizeof(unsigned int),stream);

    hitsInGPU.highEdgeXs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.highEdgeYs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.lowEdgeXs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);
    hitsInGPU.lowEdgeYs = (float*)cms::cuda::allocate_managed(nMaxHits*sizeof(float),stream);

    hitsInGPU.nHits = (unsigned int*)cms::cuda::allocate_managed(evtnum*sizeof(unsigned int),stream);
    
    hitsInGPU.hitRanges =                 (int*)cms::cuda::allocate_managed(evtnum*nModules * 2 * sizeof(int),stream);
    hitsInGPU.hitRangesLower =                 (int*)cms::cuda::allocate_managed(    evtnum*nModules * sizeof(int),stream);
    hitsInGPU.hitRangesUpper =                 (int*)cms::cuda::allocate_managed(    evtnum*nModules * sizeof(int),stream);
    hitsInGPU.hitRangesnLower =                 (int8_t*)cms::cuda::allocate_managed(evtnum*nModules * sizeof(int8_t),stream);
    hitsInGPU.hitRangesnUpper =                 (int8_t*)cms::cuda::allocate_managed(evtnum*nModules * sizeof(int8_t),stream);
#else
    //nMaxHits and nMax2SHits are the maximum possible numbers
    hipMallocManaged(&hitsInGPU.xs, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.ys, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.zs, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.moduleIndices, nMaxHits * sizeof(uint16_t));
    //TODO:This dude (idxs) is not used in the GPU at all. It is only used for simhit matching to make efficiency plots
    //We can even skip this one later
    hipMallocManaged(&hitsInGPU.idxs, nMaxHits * sizeof(unsigned int));

    hipMallocManaged(&hitsInGPU.rts, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.phis, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.etas, nMaxHits * sizeof(float));

    hipMallocManaged(&hitsInGPU.highEdgeXs, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.highEdgeYs, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.lowEdgeXs, nMaxHits * sizeof(float));
    hipMallocManaged(&hitsInGPU.lowEdgeYs, nMaxHits * sizeof(float));

    //counters
    hipMallocManaged(&hitsInGPU.nHits, evtnum*sizeof(unsigned int));

    hipMallocManaged(&hitsInGPU.hitRanges,      evtnum*nModules * 2 * sizeof(int));
    hipMallocManaged(&hitsInGPU.hitRangesLower, evtnum*nModules  * sizeof(int));
    hipMallocManaged(&hitsInGPU.hitRangesUpper, evtnum*nModules  * sizeof(int));
    hipMallocManaged(&hitsInGPU.hitRangesnLower,evtnum*nModules  * sizeof(int8_t));
    hipMallocManaged(&hitsInGPU.hitRangesnUpper,evtnum*nModules  * sizeof(int8_t));
#endif
    //*hitsInGPU.nHits = 0;
    hipMemsetAsync(hitsInGPU.nHits, 0,      evtnum*sizeof(unsigned int),stream);
    hipMemsetAsync(hitsInGPU.hitRanges, -1,      evtnum*nModules*2*sizeof(int),stream);
    hipMemsetAsync(hitsInGPU.hitRangesLower, -1, evtnum*nModules*sizeof(int),stream);
    hipMemsetAsync(hitsInGPU.hitRangesUpper, -1, evtnum*nModules*sizeof(int),stream);
    hipMemsetAsync(hitsInGPU.hitRangesnLower, -1,evtnum*nModules*sizeof(int8_t),stream);
    hipMemsetAsync(hitsInGPU.hitRangesnUpper, -1,evtnum*nModules*sizeof(int8_t),stream);
    hipStreamSynchronize(stream);
}
void SDL::createHitsInExplicitMemory(struct hits& hitsInGPU, unsigned int nMaxHits,hipStream_t stream,unsigned int evtnum)
{
    int nModules= 26593;
//#ifdef CACHE_ALLOC
#if defined(CACHE_ALLOC) && !defined(Preload_hits)
 //   hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    hitsInGPU.xs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.ys = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.zs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);

    hitsInGPU.rts = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.phis = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.etas = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);

    hitsInGPU.moduleIndices = (uint16_t*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(uint16_t),stream);
    hitsInGPU.idxs = (unsigned int*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(unsigned int),stream);

    hitsInGPU.highEdgeXs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.highEdgeYs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.lowEdgeXs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);
    hitsInGPU.lowEdgeYs = (float*)cms::cuda::allocate_device(dev,nMaxHits*sizeof(float),stream);

    hitsInGPU.nHits = (unsigned int*)cms::cuda::allocate_device(dev,evtnum*sizeof(unsigned int),stream);

    hitsInGPU.hitRanges =                  (int*)cms::cuda::allocate_device(dev,         evtnum*nModules * 2 * sizeof(int),stream);
    hitsInGPU.hitRangesLower =                  (int*)cms::cuda::allocate_device(dev,    evtnum*nModules * sizeof(int),stream);
    hitsInGPU.hitRangesUpper =                  (int*)cms::cuda::allocate_device(dev,    evtnum*nModules * sizeof(int),stream);
    hitsInGPU.hitRangesnLower =                  (int8_t*)cms::cuda::allocate_device(dev,evtnum*nModules * sizeof(int8_t),stream);
    hitsInGPU.hitRangesnUpper =                  (int8_t*)cms::cuda::allocate_device(dev,evtnum*nModules * sizeof(int8_t),stream);
#else
    hipMalloc(&hitsInGPU.xs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.ys, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.zs, nMaxHits * sizeof(float));

    hipMalloc(&hitsInGPU.moduleIndices, nMaxHits * sizeof(uint16_t));
    hipMalloc(&hitsInGPU.idxs, nMaxHits * sizeof(unsigned int));

    hipMalloc(&hitsInGPU.rts, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.phis, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.etas, nMaxHits * sizeof(float));

    hipMalloc(&hitsInGPU.highEdgeXs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.highEdgeYs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.lowEdgeXs, nMaxHits * sizeof(float));
    hipMalloc(&hitsInGPU.lowEdgeYs, nMaxHits * sizeof(float));

    //counters
    hipMalloc(&hitsInGPU.nHits,evtnum* sizeof(unsigned int));

    hipMalloc(&hitsInGPU.hitRanges,evtnum*nModules * 2 * sizeof(int));
    hipMalloc(&hitsInGPU.hitRangesLower,evtnum*nModules  * sizeof(int));
    hipMalloc(&hitsInGPU.hitRangesUpper,evtnum*nModules  * sizeof(int));
    hipMalloc(&hitsInGPU.hitRangesnLower,evtnum*nModules  * sizeof(int8_t));
    hipMalloc(&hitsInGPU.hitRangesnUpper,evtnum* nModules  * sizeof(int8_t));
#endif
    hipMemsetAsync(hitsInGPU.nHits,0,evtnum*sizeof(unsigned int),stream);
    hipMemsetAsync(hitsInGPU.hitRanges, -1,      evtnum*nModules*2*sizeof(int),stream);
    hipMemsetAsync(hitsInGPU.hitRangesLower, -1, evtnum*nModules*sizeof(int),stream);
    hipMemsetAsync(hitsInGPU.hitRangesUpper, -1, evtnum*nModules*sizeof(int),stream);
    hipMemsetAsync(hitsInGPU.hitRangesnLower, -1,evtnum*nModules*sizeof(int8_t),stream);
    hipMemsetAsync(hitsInGPU.hitRangesnUpper, -1,evtnum*nModules*sizeof(int8_t),stream);
    hipStreamSynchronize(stream);
}

__global__ void SDL::addHitToMemoryKernel(struct hits& hitsInGPU, struct modules& modulesInGPU,const float* x,const  float* y,const  float* z, const uint16_t* moduleIndex,const float* phis, const int loopsize)
{
  for (unsigned int ihit = blockIdx.x*blockDim.x + threadIdx.x; ihit <loopsize; ihit += blockDim.x*gridDim.x)
  //if(ihit < loopsize)
  {
      unsigned int idx = ihit;//*(hitsInGPU.nHits);

      hitsInGPU.xs[idx] = x[ihit];
      hitsInGPU.ys[idx] = y[ihit];
      hitsInGPU.zs[idx] = z[ihit];
      hitsInGPU.rts[idx] = sqrt(x[ihit]*x[ihit] + y[ihit]*y[ihit]);
      hitsInGPU.phis[idx] = phi(x[ihit],y[ihit],z[ihit]);
      hitsInGPU.moduleIndices[idx] = moduleIndex[ihit];
      hitsInGPU.idxs[idx] = ihit;
  }
}

__device__ void SDL::getEdgeHitsK(float phi,float x, float y, float& xhigh, float& yhigh, float& xlow, float& ylow)
{
    xhigh = x + 2.5 * cos(phi);
    yhigh = y + 2.5 * sin(phi);
    xlow = x - 2.5 * cos(phi);
    ylow = y - 2.5 * sin(phi);
}
void SDL::getEdgeHits(unsigned int detId,float x, float y, float& xhigh, float& yhigh, float& xlow, float& ylow)
{
    float phi = endcapGeometry.getCentroidPhi(detId);
    xhigh = x + 2.5 * cos(phi);
    yhigh = y + 2.5 * sin(phi);
    xlow = x - 2.5 * cos(phi);
    ylow = y - 2.5 * sin(phi);
}

void SDL::printHit(struct hits& hitsInGPU, struct modules& modulesInGPU, unsigned int hitIndex)
{
    std::cout << "Hit(x=" << hitsInGPU.xs[hitIndex] << ", y=" << hitsInGPU.ys[hitIndex] << ", z=" << hitsInGPU.zs[hitIndex] << ", rt=" << hitsInGPU.rts[hitIndex] << ", phi=" << hitsInGPU.phis[hitIndex] <<", module subdet = "<<modulesInGPU.subdets[hitsInGPU.moduleIndices[hitIndex]]<<", module layer = "<< modulesInGPU.layers[hitsInGPU.moduleIndices[hitIndex]]<<", module ring = "<< modulesInGPU.rings[hitsInGPU.moduleIndices[hitIndex]]<<" )"<<std::endl;
}


void SDL::hits::freeMemoryCache()
{
#ifdef Explicit_Hit
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,nHits);
    cms::cuda::free_device(dev,xs);
    cms::cuda::free_device(dev,ys);
    cms::cuda::free_device(dev,zs);
    cms::cuda::free_device(dev,moduleIndices);
    cms::cuda::free_device(dev,rts);
    cms::cuda::free_device(dev,idxs);
    cms::cuda::free_device(dev,phis);
    cms::cuda::free_device(dev,etas);

    cms::cuda::free_device(dev,highEdgeXs);
    cms::cuda::free_device(dev,highEdgeYs);
    cms::cuda::free_device(dev,lowEdgeXs);
    cms::cuda::free_device(dev,lowEdgeYs);
    
    cms::cuda::free_device(dev,hitRanges);
    cms::cuda::free_device(dev,hitRangesLower);
    cms::cuda::free_device(dev,hitRangesnLower);
    cms::cuda::free_device(dev,hitRangesUpper);
    cms::cuda::free_device(dev,hitRangesnUpper);
#else
    cms::cuda::free_managed(nHits);
    cms::cuda::free_managed(xs);
    cms::cuda::free_managed(ys);
    cms::cuda::free_managed(zs);
    cms::cuda::free_managed(moduleIndices);
    cms::cuda::free_managed(rts);
    cms::cuda::free_managed(idxs);
    cms::cuda::free_managed(phis);
    cms::cuda::free_managed(etas);

    cms::cuda::free_managed(highEdgeXs);
    cms::cuda::free_managed(highEdgeYs);
    cms::cuda::free_managed(lowEdgeXs);
    cms::cuda::free_managed(lowEdgeYs);
    
    cms::cuda::free_managed(hitRanges);
    cms::cuda::free_managed(hitRangesLower);
    cms::cuda::free_managed(hitRangesnLower);
    cms::cuda::free_managed(hitRangesUpper);
    cms::cuda::free_managed(hitRangesnUpper);
#endif
}
void SDL::hits::freeMemory()
//void SDL::hits::freeMemory(hipStream_t stream)
{
    hipFree(nHits);
    hipFree(xs);
    hipFree(ys);
    hipFree(zs);
    hipFree(moduleIndices);
    hipFree(rts);
    hipFree(idxs);
    hipFree(phis);
    hipFree(etas);

    hipFree(highEdgeXs);
    hipFree(highEdgeYs);
    hipFree(lowEdgeXs);
    hipFree(lowEdgeYs);
    
    hipFree(hitRanges);
    hipFree(hitRangesLower);
    hipFree(hitRangesnLower);
    hipFree(hitRangesUpper);
    hipFree(hitRangesnUpper);
//    hipStreamSynchronize(stream);
}
