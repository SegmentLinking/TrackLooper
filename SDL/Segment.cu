#include "hip/hip_runtime.h"
# include "Segment.cuh"

///FIXME:NOTICE THE NEW maxPixelSegments!

void SDL::segments::resetMemory(unsigned int nMemoryLocationsx, unsigned int nLowerModules, unsigned int maxPixelSegments,hipStream_t stream)
{
    // unsigned int nMemoryLocationsx = maxSegments * nLowerModules + maxPixelSegments;
    hipMemsetAsync(mdIndices,0, nMemoryLocationsx * 2 * sizeof(unsigned int),stream);
    hipMemsetAsync(innerLowerModuleIndices,0, nMemoryLocationsx * 2 * sizeof(uint16_t),stream);
    hipMemsetAsync(nSegments, 0,(nLowerModules+1) * sizeof(int),stream);
    hipMemsetAsync(totOccupancySegments, 0,(nLowerModules+1) * sizeof(int),stream);
    hipMemsetAsync(dPhis, 0,(nMemoryLocationsx * 6 )*sizeof(FPX),stream);
    hipMemsetAsync(ptIn, 0,(maxPixelSegments * 8)*sizeof(float),stream);
    hipMemsetAsync(superbin, 0,(maxPixelSegments )*sizeof(int),stream);
    hipMemsetAsync(pixelType, 0,(maxPixelSegments )*sizeof(int8_t),stream);
    hipMemsetAsync(isQuad, 0,(maxPixelSegments )*sizeof(bool),stream);
    hipMemsetAsync(isDup, 0,(maxPixelSegments )*sizeof(bool),stream);
    hipMemsetAsync(score, 0,(maxPixelSegments )*sizeof(float),stream);
    hipMemsetAsync(charge, 0,maxPixelSegments * sizeof(int),stream);
    hipMemsetAsync(seedIdx, 0,maxPixelSegments * sizeof(unsigned int),stream);
    hipMemsetAsync(circleCenterX, 0,maxPixelSegments * sizeof(float),stream);
    hipMemsetAsync(circleCenterY, 0,maxPixelSegments * sizeof(float),stream);
    hipMemsetAsync(circleRadius, 0,maxPixelSegments * sizeof(float),stream);
    hipMemsetAsync(partOfPT5, 0,maxPixelSegments * sizeof(bool),stream);
    hipMemsetAsync(pLSHitsIdxs, 0,maxPixelSegments * sizeof(uint4),stream);
}

void SDL::createSegmentArrayRanges(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU, uint16_t& nLowerModules, int& nTotalSegments, hipStream_t stream, const uint16_t& maxPixelSegments)
{
    /*
        write code here that will deal with importing module parameters to CPU, and get the relevant occupancies for a given module!*/

    int *module_segmentModuleIndices;
    short* module_subdets;
    short* module_layers;
    short* module_rings;
    float* module_eta;
    uint16_t* module_nConnectedModules;
    module_segmentModuleIndices = (int*)cms::cuda::allocate_host((nLowerModules + 1) * sizeof(unsigned int), stream);
    module_subdets = (short*)cms::cuda::allocate_host(nLowerModules* sizeof(short), stream);
    module_layers = (short*)cms::cuda::allocate_host(nLowerModules* sizeof(short), stream);
    module_rings = (short*)cms::cuda::allocate_host(nLowerModules* sizeof(short), stream);
    module_eta = (float*)cms::cuda::allocate_host(nLowerModules* sizeof(float), stream);
    module_nConnectedModules = (uint16_t*)cms::cuda::allocate_host(nLowerModules * sizeof(uint16_t), stream);
    hipMemcpyAsync(module_subdets,modulesInGPU.subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_layers,modulesInGPU.layers,nLowerModules * sizeof(short),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_rings,modulesInGPU.rings,nLowerModules * sizeof(short),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_eta,modulesInGPU.eta,nLowerModules * sizeof(float),hipMemcpyDeviceToHost,stream);
    hipMemcpyAsync(module_nConnectedModules,modulesInGPU.nConnectedModules,nLowerModules*sizeof(uint16_t),hipMemcpyDeviceToHost,stream);
 
    hipStreamSynchronize(stream);

    nTotalSegments = 0; //start!   
    for(uint16_t i = 0; i < nLowerModules; i++)
    {
        module_segmentModuleIndices[i] = nTotalSegments; //running counter - we start at the previous index!

        unsigned int occupancy;
        unsigned int category_number, eta_number;
        if (module_layers[i]<=3 && module_subdets[i]==5) category_number = 0;
        if (module_layers[i]>=4 && module_subdets[i]==5) category_number = 1;
        if (module_layers[i]<=2 && module_subdets[i]==4 && module_rings[i]>=11) category_number = 2;
        if (module_layers[i]>=3 && module_subdets[i]==4 && module_rings[i]>=8) category_number = 2;
        if (module_layers[i]<=2 && module_subdets[i]==4 && module_rings[i]<=10) category_number = 3;
        if (module_layers[i]>=3 && module_subdets[i]==4 && module_rings[i]<=7) category_number = 3;
        if (abs(module_eta[i])<0.75) eta_number=0;
        if (abs(module_eta[i])>0.75 && abs(module_eta[i])<1.5) eta_number=1;
        if (abs(module_eta[i])>1.5 && abs(module_eta[i])<2.25) eta_number=2;
        if (abs(module_eta[i])>2.25 && abs(module_eta[i])<3) eta_number=3;

        if (category_number == 0 && eta_number == 0) occupancy = 572;
        if (category_number == 0 && eta_number == 1) occupancy = 300;
        if (category_number == 0 && eta_number == 2) occupancy = 183;
        if (category_number == 0 && eta_number == 3) occupancy = 62;
        if (category_number == 1 && eta_number == 0) occupancy = 191;
        if (category_number == 1 && eta_number == 1) occupancy = 128;
        if (category_number == 2 && eta_number == 1) occupancy = 107;
        if (category_number == 2 && eta_number == 2) occupancy = 102;
        if (category_number == 3 && eta_number == 1) occupancy = 64;
        if (category_number == 3 && eta_number == 2) occupancy = 79;
        if (category_number == 3 && eta_number == 3) occupancy = 85;

        if(module_nConnectedModules[i] == 0) occupancy = 0;

        nTotalSegments += occupancy;
    }

    module_segmentModuleIndices[nLowerModules] = nTotalSegments;
    nTotalSegments += maxPixelSegments;

    hipMemcpyAsync(rangesInGPU.segmentModuleIndices, module_segmentModuleIndices,  (nLowerModules + 1) * sizeof(unsigned int), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);
    cms::cuda::free_host(module_segmentModuleIndices);
    cms::cuda::free_host(module_nConnectedModules);
    cms::cuda::free_host(module_subdets);
    cms::cuda::free_host(module_layers);
    cms::cuda::free_host(module_rings);
    cms::cuda::free_host(module_eta);
}

void SDL::createSegmentsInExplicitMemory(struct segments& segmentsInGPU, unsigned int nMemoryLocations, uint16_t nLowerModules, unsigned int maxPixelSegments, hipStream_t stream)
{
    //FIXME:Since the number of pixel segments is 10x the number of regular segments per module, we need to provide
    //extra memory to the pixel segments
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    segmentsInGPU.mdIndices = (unsigned int*)cms::cuda::allocate_device(dev,nMemoryLocations*4 *sizeof(unsigned int),stream);
    segmentsInGPU.innerLowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev,nMemoryLocations*2 *sizeof(uint16_t),stream);
    segmentsInGPU.nSegments = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(int),stream);
    segmentsInGPU.totOccupancySegments = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(unsigned int),stream);
    segmentsInGPU.dPhis = (FPX*)cms::cuda::allocate_device(dev,nMemoryLocations*6 *sizeof(FPX),stream);
    segmentsInGPU.ptIn = (float*)cms::cuda::allocate_device(dev, maxPixelSegments * 8 *sizeof(float),stream);
    segmentsInGPU.superbin = (int*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(int),stream);
    segmentsInGPU.pixelType = (int8_t*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(int8_t),stream);
    segmentsInGPU.isQuad = (bool*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(bool),stream);
    segmentsInGPU.isDup = (bool*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(bool),stream);
    segmentsInGPU.score = (float*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(float),stream);
    segmentsInGPU.charge = (int*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(int), stream);
    segmentsInGPU.seedIdx = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(unsigned int), stream);
    segmentsInGPU.circleCenterX = (float*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(float), stream);
    segmentsInGPU.circleCenterY = (float*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(float), stream);
    segmentsInGPU.circleRadius = (float*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(float), stream);
    segmentsInGPU.partOfPT5 = (bool*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(bool), stream);
    segmentsInGPU.pLSHitsIdxs = (uint4*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(uint4), stream);
    segmentsInGPU.nMemoryLocations = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);
#else
    hipMalloc(&segmentsInGPU.mdIndices, nMemoryLocations * 4 * sizeof(unsigned int));
    hipMalloc(&segmentsInGPU.innerLowerModuleIndices, nMemoryLocations * 2 * sizeof(uint16_t));
    hipMalloc(&segmentsInGPU.nSegments, (nLowerModules + 1) * sizeof(int));
    hipMalloc(&segmentsInGPU.totOccupancySegments, (nLowerModules + 1) * sizeof(int));
    hipMalloc(&segmentsInGPU.dPhis, nMemoryLocations * 6 *sizeof(FPX));
    hipMalloc(&segmentsInGPU.ptIn, maxPixelSegments * 8*sizeof(float));
    hipMalloc(&segmentsInGPU.superbin, (maxPixelSegments )*sizeof(int));
    hipMalloc(&segmentsInGPU.pixelType, (maxPixelSegments )*sizeof(int8_t));
    hipMalloc(&segmentsInGPU.isQuad, (maxPixelSegments )*sizeof(bool));
    hipMalloc(&segmentsInGPU.isDup, (maxPixelSegments )*sizeof(bool));
    hipMalloc(&segmentsInGPU.score, (maxPixelSegments )*sizeof(float));
    hipMalloc(&segmentsInGPU.charge, maxPixelSegments * sizeof(int));
    hipMalloc(&segmentsInGPU.seedIdx, maxPixelSegments * sizeof(unsigned int));
    hipMalloc(&segmentsInGPU.circleCenterX, maxPixelSegments * sizeof(float));
    hipMalloc(&segmentsInGPU.circleCenterY, maxPixelSegments * sizeof(float));
    hipMalloc(&segmentsInGPU.circleRadius, maxPixelSegments * sizeof(float));
    hipMalloc(&segmentsInGPU.partOfPT5, maxPixelSegments * sizeof(bool));
    hipMalloc(&segmentsInGPU.pLSHitsIdxs, maxPixelSegments * sizeof(uint4));
    hipMalloc(&segmentsInGPU.nMemoryLocations, sizeof(unsigned int));
#endif

    //segmentsInGPU.innerLowerModuleIndices = segmentsInGPU.mdIndices + nMemoryLocations * 2;
    segmentsInGPU.outerLowerModuleIndices = segmentsInGPU.innerLowerModuleIndices + nMemoryLocations;
    segmentsInGPU.innerMiniDoubletAnchorHitIndices = segmentsInGPU.mdIndices + nMemoryLocations * 2;
    segmentsInGPU.outerMiniDoubletAnchorHitIndices = segmentsInGPU.mdIndices + nMemoryLocations * 3;

    segmentsInGPU.dPhiMins = segmentsInGPU.dPhis + nMemoryLocations;
    segmentsInGPU.dPhiMaxs = segmentsInGPU.dPhis + nMemoryLocations * 2;
    segmentsInGPU.dPhiChanges = segmentsInGPU.dPhis + nMemoryLocations * 3;
    segmentsInGPU.dPhiChangeMins = segmentsInGPU.dPhis + nMemoryLocations * 4;
    segmentsInGPU.dPhiChangeMaxs = segmentsInGPU.dPhis + nMemoryLocations * 5;

    segmentsInGPU.ptErr  = segmentsInGPU.ptIn + maxPixelSegments;
    segmentsInGPU.px     = segmentsInGPU.ptIn + maxPixelSegments * 2;
    segmentsInGPU.py     = segmentsInGPU.ptIn + maxPixelSegments * 3;
    segmentsInGPU.pz     = segmentsInGPU.ptIn + maxPixelSegments * 4;
    segmentsInGPU.etaErr = segmentsInGPU.ptIn + maxPixelSegments * 5;
    segmentsInGPU.eta    = segmentsInGPU.ptIn + maxPixelSegments * 6;
    segmentsInGPU.phi    = segmentsInGPU.ptIn + maxPixelSegments * 7;

    hipMemsetAsync(segmentsInGPU.nSegments,0, (nLowerModules + 1) * sizeof(int),stream);
    hipMemsetAsync(segmentsInGPU.totOccupancySegments,0, (nLowerModules + 1) * sizeof(int),stream);
    hipMemsetAsync(segmentsInGPU.partOfPT5, false, maxPixelSegments * sizeof(bool),stream);
    hipMemsetAsync(segmentsInGPU.pLSHitsIdxs, 0, maxPixelSegments * sizeof(uint4),stream);
    hipMemsetAsync(segmentsInGPU.nMemoryLocations, nMemoryLocations, sizeof(unsigned int), stream);
    hipStreamSynchronize(stream);

}

SDL::segments::segments()
{
    superbin = nullptr;
    pixelType = nullptr;
    isQuad = nullptr;
    isDup = nullptr;
    score = nullptr;
    circleRadius = nullptr;
    charge = nullptr;
    seedIdx = nullptr;
    circleCenterX = nullptr;
    circleCenterY = nullptr;
    mdIndices = nullptr;
    innerLowerModuleIndices = nullptr;
    outerLowerModuleIndices = nullptr;
    innerMiniDoubletAnchorHitIndices = nullptr;
    outerMiniDoubletAnchorHitIndices = nullptr;

    nSegments = nullptr;
    totOccupancySegments = nullptr;
    dPhis = nullptr;
    dPhiMins = nullptr;
    dPhiMaxs = nullptr;
    dPhiChanges = nullptr;
    dPhiChangeMins = nullptr;
    dPhiChangeMaxs = nullptr;
    partOfPT5 = nullptr;
    pLSHitsIdxs = nullptr;

}

SDL::segments::~segments()
{
}

void SDL::segments::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,mdIndices);
    cms::cuda::free_device(dev,innerLowerModuleIndices);
    cms::cuda::free_device(dev,dPhis);
    cms::cuda::free_device(dev,ptIn);
    cms::cuda::free_device(dev,nSegments);
    cms::cuda::free_device(dev,totOccupancySegments);
    cms::cuda::free_device(dev, charge);
    cms::cuda::free_device(dev, seedIdx);
    cms::cuda::free_device(dev,superbin);
    cms::cuda::free_device(dev,pixelType);
    cms::cuda::free_device(dev,isQuad);
    cms::cuda::free_device(dev,isDup);
    cms::cuda::free_device(dev,score);
    cms::cuda::free_device(dev, circleCenterX);
    cms::cuda::free_device(dev, circleCenterY);
    cms::cuda::free_device(dev, circleRadius);
    cms::cuda::free_device(dev, partOfPT5);
    cms::cuda::free_device(dev, pLSHitsIdxs);
    cms::cuda::free_device(dev, nMemoryLocations);
}

void SDL::segments::freeMemory(hipStream_t stream)
{
    hipFree(mdIndices);
    hipFree(innerLowerModuleIndices);
    hipFree(nSegments);
    hipFree(totOccupancySegments);
    hipFree(dPhis);
    hipFree(ptIn);
    hipFree(superbin);
    hipFree(pixelType);
    hipFree(isQuad);
    hipFree(isDup);
    hipFree(score);
    hipFree(charge);
    hipFree(seedIdx);
    hipFree(circleCenterX);
    hipFree(circleCenterY);
    hipFree(circleRadius);
    hipFree(partOfPT5);
    hipFree(pLSHitsIdxs);
    hipFree(nMemoryLocations);
}

ALPAKA_FN_ACC void SDL::addSegmentToMemory(struct segments& segmentsInGPU, unsigned int lowerMDIndex, unsigned int upperMDIndex, uint16_t innerLowerModuleIndex, uint16_t outerLowerModuleIndex, unsigned int innerMDAnchorHitIndex, unsigned int outerMDAnchorHitIndex, float& dPhi, float& dPhiMin, float& dPhiMax, float& dPhiChange, float& dPhiChangeMin, float& dPhiChangeMax, unsigned int idx)
{
    //idx will be computed in the kernel, which is the index into which the 
    //segment will be written
    //nSegments will be incremented in the kernel
    //printf("seg: %u %u %u %u\n",lowerMDIndex, upperMDIndex,innerLowerModuleIndex,outerLowerModuleIndex);
    segmentsInGPU.mdIndices[idx * 2] = lowerMDIndex;
    segmentsInGPU.mdIndices[idx * 2 + 1] = upperMDIndex;
    segmentsInGPU.innerLowerModuleIndices[idx] = innerLowerModuleIndex;
    segmentsInGPU.outerLowerModuleIndices[idx] = outerLowerModuleIndex;
    segmentsInGPU.innerMiniDoubletAnchorHitIndices[idx] = innerMDAnchorHitIndex;
    segmentsInGPU.outerMiniDoubletAnchorHitIndices[idx] = outerMDAnchorHitIndex;

    segmentsInGPU.dPhis[idx]          = __F2H(dPhi);
    segmentsInGPU.dPhiMins[idx]       = __F2H(dPhiMin);
    segmentsInGPU.dPhiMaxs[idx]       = __F2H(dPhiMax);
    segmentsInGPU.dPhiChanges[idx]    = __F2H(dPhiChange);
    segmentsInGPU.dPhiChangeMins[idx] = __F2H(dPhiChangeMin);
    segmentsInGPU.dPhiChangeMaxs[idx] = __F2H(dPhiChangeMax);

}

ALPAKA_FN_ACC void SDL::addPixelSegmentToMemory(struct segments& segmentsInGPU, struct miniDoublets& mdsInGPU, struct modules& modulesInGPU, unsigned int innerMDIndex, unsigned int outerMDIndex, uint16_t pixelModuleIndex, unsigned int hitIdxs[4], unsigned int innerAnchorHitIndex, unsigned int outerAnchorHitIndex, float dPhiChange, float ptIn, float ptErr, float px, float py, float pz, float etaErr, float eta, float phi, int charge, unsigned int seedIdx, unsigned int idx, unsigned int pixelSegmentArrayIndex, int superbin, int8_t pixelType, short isQuad, float score)
{
    segmentsInGPU.mdIndices[idx * 2] = innerMDIndex;
    segmentsInGPU.mdIndices[idx * 2 + 1] = outerMDIndex;
    segmentsInGPU.innerLowerModuleIndices[idx] = pixelModuleIndex;
    segmentsInGPU.outerLowerModuleIndices[idx] = pixelModuleIndex;
    segmentsInGPU.innerMiniDoubletAnchorHitIndices[idx] = innerAnchorHitIndex;
    segmentsInGPU.outerMiniDoubletAnchorHitIndices[idx] = outerAnchorHitIndex;
    segmentsInGPU.dPhiChanges[idx] = __F2H(dPhiChange);
    segmentsInGPU.ptIn[pixelSegmentArrayIndex] = ptIn;
    segmentsInGPU.ptErr[pixelSegmentArrayIndex] = ptErr;
    segmentsInGPU.px[pixelSegmentArrayIndex] = px;
    segmentsInGPU.py[pixelSegmentArrayIndex] = py;
    segmentsInGPU.pz[pixelSegmentArrayIndex] = pz;
    segmentsInGPU.etaErr[pixelSegmentArrayIndex] = etaErr;
    segmentsInGPU.eta[pixelSegmentArrayIndex] = eta;
    segmentsInGPU.phi[pixelSegmentArrayIndex] = phi;
    segmentsInGPU.charge[pixelSegmentArrayIndex] = charge;
    segmentsInGPU.seedIdx[pixelSegmentArrayIndex] = seedIdx;

    segmentsInGPU.superbin[pixelSegmentArrayIndex] = superbin;
    segmentsInGPU.pixelType[pixelSegmentArrayIndex] = pixelType;
    segmentsInGPU.isQuad[pixelSegmentArrayIndex] = isQuad;
    segmentsInGPU.isDup[pixelSegmentArrayIndex] = false;
    segmentsInGPU.score[pixelSegmentArrayIndex] = score;

    segmentsInGPU.pLSHitsIdxs[pixelSegmentArrayIndex].x = hitIdxs[0];
    segmentsInGPU.pLSHitsIdxs[pixelSegmentArrayIndex].y = hitIdxs[1];
    segmentsInGPU.pLSHitsIdxs[pixelSegmentArrayIndex].z = hitIdxs[2];
    segmentsInGPU.pLSHitsIdxs[pixelSegmentArrayIndex].w = hitIdxs[3];

    //computing circle parameters
    /*
       The two anchor hits are r3PCA and r3LH. p3PCA pt, eta, phi is hitIndex1 x, y, z
    */
    float circleRadius = mdsInGPU.outerX[innerMDIndex] / (2 * k2Rinv1GeVf);
    float circlePhi = mdsInGPU.outerZ[innerMDIndex];
    float candidateCenterXs[] = {mdsInGPU.anchorX[innerMDIndex] + circleRadius * sinf(circlePhi), mdsInGPU.anchorX[innerMDIndex] - circleRadius * sinf(circlePhi)};
    float candidateCenterYs[] = {mdsInGPU.anchorY[innerMDIndex] - circleRadius * cosf(circlePhi), mdsInGPU.anchorY[innerMDIndex] + circleRadius * cosf(circlePhi)};

    //check which of the circles can accommodate r3LH better (we won't get perfect agreement)
    float bestChiSquared = 123456789.f;
    float chiSquared;
    size_t bestIndex;
    for(size_t i = 0; i < 2; i++)
    {
        chiSquared = fabsf(sqrtf((mdsInGPU.anchorX[outerMDIndex] - candidateCenterXs[i]) * (mdsInGPU.anchorX[outerMDIndex] - candidateCenterXs[i]) + (mdsInGPU.anchorY[outerMDIndex] - candidateCenterYs[i]) * (mdsInGPU.anchorY[outerMDIndex] - candidateCenterYs[i])) - circleRadius);
        if(chiSquared < bestChiSquared)
        {
            bestChiSquared = chiSquared;
            bestIndex = i;
        }
    }
    segmentsInGPU.circleCenterX[pixelSegmentArrayIndex] = candidateCenterXs[bestIndex];
    segmentsInGPU.circleCenterY[pixelSegmentArrayIndex] = candidateCenterYs[bestIndex];
    segmentsInGPU.circleRadius[pixelSegmentArrayIndex] = circleRadius;
}

void SDL::printSegment(struct SDL::segments& segmentsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::hits& hitsInGPU, struct SDL::modules& modulesInGPU, unsigned int segmentIndex)
{
    unsigned int innerMDIndex = segmentsInGPU.mdIndices[segmentIndex * 2];
    unsigned int outerMDIndex = segmentsInGPU.mdIndices[segmentIndex * 2 + 1];
    std::cout<<std::endl;
    std::cout<<"sg_dPhiChange : "<<__H2F(segmentsInGPU.dPhiChanges[segmentIndex]) << std::endl<<std::endl;

    std::cout << "Inner Mini-Doublet" << std::endl;
    std::cout << "------------------------------" << std::endl;
    {
        IndentingOStreambuf indent(std::cout);
        printMD(mdsInGPU, hitsInGPU, modulesInGPU, innerMDIndex);
    }
    std::cout<<std::endl<<" Outer Mini-Doublet" <<std::endl;
    std::cout << "------------------------------" << std::endl;
    {
        IndentingOStreambuf indent(std::cout);
        printMD(mdsInGPU, hitsInGPU, modulesInGPU, outerMDIndex);
    }
}