#include "hip/hip_runtime.h"
#include "Segment.cuh"

///FIXME:NOTICE THE NEW maxPixelSegments!

void SDL::segments::resetMemory(unsigned int nMemoryLocationsx, unsigned int nLowerModules, unsigned int maxPixelSegments,hipStream_t stream)
{
    hipMemsetAsync(mdIndices,0, nMemoryLocationsx * 2 * sizeof(unsigned int),stream);
    hipMemsetAsync(innerLowerModuleIndices,0, nMemoryLocationsx * 2 * sizeof(uint16_t),stream);
    hipMemsetAsync(nSegments, 0,(nLowerModules+1) * sizeof(int),stream);
    hipMemsetAsync(totOccupancySegments, 0,(nLowerModules+1) * sizeof(int),stream);
    hipMemsetAsync(dPhis, 0,(nMemoryLocationsx * 6 )*sizeof(FPX),stream);
    hipMemsetAsync(ptIn, 0,(maxPixelSegments * 8)*sizeof(float),stream);
    hipMemsetAsync(superbin, 0,(maxPixelSegments )*sizeof(int),stream);
    hipMemsetAsync(pixelType, 0,(maxPixelSegments )*sizeof(int8_t),stream);
    hipMemsetAsync(isQuad, 0,(maxPixelSegments )*sizeof(char),stream);
    hipMemsetAsync(isDup, 0,(maxPixelSegments )*sizeof(bool),stream);
    hipMemsetAsync(score, 0,(maxPixelSegments )*sizeof(float),stream);
    hipMemsetAsync(charge, 0,maxPixelSegments * sizeof(int),stream);
    hipMemsetAsync(seedIdx, 0,maxPixelSegments * sizeof(unsigned int),stream);
    hipMemsetAsync(circleCenterX, 0,maxPixelSegments * sizeof(float),stream);
    hipMemsetAsync(circleCenterY, 0,maxPixelSegments * sizeof(float),stream);
    hipMemsetAsync(circleRadius, 0,maxPixelSegments * sizeof(float),stream);
    hipMemsetAsync(partOfPT5, 0,maxPixelSegments * sizeof(bool),stream);
    hipMemsetAsync(pLSHitsIdxs, 0,maxPixelSegments * sizeof(uint4),stream);
}


__global__ void SDL::createSegmentArrayRanges(struct modules& modulesInGPU, struct objectRanges& rangesInGPU, struct miniDoublets& mdsInGPU)
{
    short module_subdets;
    short module_layers;
    short module_rings;
    float module_eta;

    __shared__ unsigned int nTotalSegments;
    nTotalSegments = 0; //start!
    __syncthreads(); 
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int np = gridDim.x * blockDim.x;
    for(uint16_t i = gid; i < *modulesInGPU.nLowerModules; i+= np)
    {
        if(modulesInGPU.nConnectedModules[i] == 0)
        {
          rangesInGPU.segmentModuleIndices[i] = nTotalSegments;
          rangesInGPU.segmentModuleOccupancy[i] = 0;
          continue;
        }
        module_subdets = modulesInGPU.subdets[i];
        module_layers = modulesInGPU.layers[i];
        module_rings = modulesInGPU.rings[i];
        module_eta = abs(modulesInGPU.eta[i]);
        unsigned int occupancy;
        unsigned int category_number, eta_number;
        if (module_layers<=3 && module_subdets==5) category_number = 0;
        else if (module_layers>=4 && module_subdets==5) category_number = 1;
        else if (module_layers<=2 && module_subdets==4 && module_rings>=11) category_number = 2;
        else if (module_layers>=3 && module_subdets==4 && module_rings>=8) category_number = 2;
        else if (module_layers<=2 && module_subdets==4 && module_rings<=10) category_number = 3;
        else if (module_layers>=3 && module_subdets==4 && module_rings<=7) category_number = 3;
        if (module_eta<0.75) eta_number=0;
        else if (module_eta>0.75 && module_eta<1.5) eta_number=1;
        else if (module_eta>1.5  && module_eta<2.25) eta_number=2;
        else if (module_eta>2.25 && module_eta<3) eta_number=3;

        if (category_number == 0 && eta_number == 0) occupancy = 572;
        else if (category_number == 0 && eta_number == 1) occupancy = 300;
        else if (category_number == 0 && eta_number == 2) occupancy = 183;
        else if (category_number == 0 && eta_number == 3) occupancy = 62;
        else if (category_number == 1 && eta_number == 0) occupancy = 191;
        else if (category_number == 1 && eta_number == 1) occupancy = 128;
        else if (category_number == 2 && eta_number == 1) occupancy = 107;
        else if (category_number == 2 && eta_number == 2) occupancy = 102;
        else if (category_number == 3 && eta_number == 1) occupancy = 64;
        else if (category_number == 3 && eta_number == 2) occupancy = 79;
        else if (category_number == 3 && eta_number == 3) occupancy = 85;


        unsigned int nTotSegs = atomicAdd(&nTotalSegments,occupancy);
        rangesInGPU.segmentModuleIndices[i] = nTotSegs;
        rangesInGPU.segmentModuleOccupancy[i] = occupancy;
    }

    __syncthreads();
    if(threadIdx.x==0){
      rangesInGPU.segmentModuleIndices[*modulesInGPU.nLowerModules] = nTotalSegments;
      *rangesInGPU.device_nTotalSegs = nTotalSegments;
    }
}


void SDL::createSegmentsInExplicitMemory(struct segments& segmentsInGPU, unsigned int nMemoryLocations, uint16_t nLowerModules, unsigned int maxPixelSegments, hipStream_t stream)
{
    //FIXME:Since the number of pixel segments is 10x the number of regular segments per module, we need to provide
    //extra memory to the pixel segments
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    segmentsInGPU.mdIndices = (unsigned int*)cms::cuda::allocate_device(dev,nMemoryLocations*4 *sizeof(unsigned int),stream);
    segmentsInGPU.innerLowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev,nMemoryLocations*2 *sizeof(uint16_t),stream);
    segmentsInGPU.nSegments = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(int),stream);
    segmentsInGPU.totOccupancySegments = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(unsigned int),stream);
    segmentsInGPU.dPhis = (FPX*)cms::cuda::allocate_device(dev,nMemoryLocations*6 *sizeof(FPX),stream);
    segmentsInGPU.ptIn = (float*)cms::cuda::allocate_device(dev, maxPixelSegments * 8 *sizeof(float),stream);
    segmentsInGPU.superbin = (int*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(int),stream);
    segmentsInGPU.pixelType = (int8_t*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(int8_t),stream);
    segmentsInGPU.isQuad = (char*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(char),stream);
    segmentsInGPU.isDup = (bool*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(bool),stream);
    segmentsInGPU.score = (float*)cms::cuda::allocate_device(dev,(maxPixelSegments) *sizeof(float),stream);
    segmentsInGPU.charge = (int*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(int), stream);
    segmentsInGPU.seedIdx = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(unsigned int), stream);
    segmentsInGPU.circleCenterX = (float*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(float), stream);
    segmentsInGPU.circleCenterY = (float*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(float), stream);
    segmentsInGPU.circleRadius = (float*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(float), stream);
    segmentsInGPU.partOfPT5 = (bool*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(bool), stream);
    segmentsInGPU.pLSHitsIdxs = (uint4*)cms::cuda::allocate_device(dev, maxPixelSegments * sizeof(uint4), stream);
    segmentsInGPU.nMemoryLocations = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);
#else
    hipMalloc(&segmentsInGPU.mdIndices, nMemoryLocations * 4 * sizeof(unsigned int));
    hipMalloc(&segmentsInGPU.innerLowerModuleIndices, nMemoryLocations * 2 * sizeof(uint16_t));
    hipMalloc(&segmentsInGPU.nSegments, (nLowerModules + 1) * sizeof(int));
    hipMalloc(&segmentsInGPU.totOccupancySegments, (nLowerModules + 1) * sizeof(int));
    hipMalloc(&segmentsInGPU.dPhis, nMemoryLocations * 6 *sizeof(FPX));
    hipMalloc(&segmentsInGPU.ptIn, maxPixelSegments * 8*sizeof(float));
    hipMalloc(&segmentsInGPU.superbin, (maxPixelSegments )*sizeof(int));
    hipMalloc(&segmentsInGPU.pixelType, (maxPixelSegments )*sizeof(int8_t));
    hipMalloc(&segmentsInGPU.isQuad, (maxPixelSegments )*sizeof(char));
    hipMalloc(&segmentsInGPU.isDup, (maxPixelSegments )*sizeof(bool));
    hipMalloc(&segmentsInGPU.score, (maxPixelSegments )*sizeof(float));
    hipMalloc(&segmentsInGPU.charge, maxPixelSegments * sizeof(int));
    hipMalloc(&segmentsInGPU.seedIdx, maxPixelSegments * sizeof(unsigned int));
    hipMalloc(&segmentsInGPU.circleCenterX, maxPixelSegments * sizeof(float));
    hipMalloc(&segmentsInGPU.circleCenterY, maxPixelSegments * sizeof(float));
    hipMalloc(&segmentsInGPU.circleRadius, maxPixelSegments * sizeof(float));
    hipMalloc(&segmentsInGPU.partOfPT5, maxPixelSegments * sizeof(bool));
    hipMalloc(&segmentsInGPU.pLSHitsIdxs, maxPixelSegments * sizeof(uint4));
    hipMalloc(&segmentsInGPU.nMemoryLocations, sizeof(unsigned int));
#endif
    segmentsInGPU.outerLowerModuleIndices = segmentsInGPU.innerLowerModuleIndices + nMemoryLocations;
    segmentsInGPU.innerMiniDoubletAnchorHitIndices = segmentsInGPU.mdIndices + nMemoryLocations * 2;
    segmentsInGPU.outerMiniDoubletAnchorHitIndices = segmentsInGPU.mdIndices + nMemoryLocations * 3;

    segmentsInGPU.dPhiMins = segmentsInGPU.dPhis + nMemoryLocations;
    segmentsInGPU.dPhiMaxs = segmentsInGPU.dPhis + nMemoryLocations * 2;
    segmentsInGPU.dPhiChanges = segmentsInGPU.dPhis + nMemoryLocations * 3;
    segmentsInGPU.dPhiChangeMins = segmentsInGPU.dPhis + nMemoryLocations * 4;
    segmentsInGPU.dPhiChangeMaxs = segmentsInGPU.dPhis + nMemoryLocations * 5;

    segmentsInGPU.ptErr  = segmentsInGPU.ptIn + maxPixelSegments;
    segmentsInGPU.px     = segmentsInGPU.ptIn + maxPixelSegments * 2;
    segmentsInGPU.py     = segmentsInGPU.ptIn + maxPixelSegments * 3;
    segmentsInGPU.pz     = segmentsInGPU.ptIn + maxPixelSegments * 4;
    segmentsInGPU.etaErr = segmentsInGPU.ptIn + maxPixelSegments * 5;
    segmentsInGPU.eta    = segmentsInGPU.ptIn + maxPixelSegments * 6;
    segmentsInGPU.phi    = segmentsInGPU.ptIn + maxPixelSegments * 7;

    hipMemsetAsync(segmentsInGPU.nSegments,0, (nLowerModules + 1) * sizeof(int),stream);
    hipMemsetAsync(segmentsInGPU.totOccupancySegments,0, (nLowerModules + 1) * sizeof(int),stream);
    hipMemsetAsync(segmentsInGPU.partOfPT5, false, maxPixelSegments * sizeof(bool),stream);
    hipMemsetAsync(segmentsInGPU.pLSHitsIdxs, 0, maxPixelSegments * sizeof(uint4),stream);
    hipMemsetAsync(segmentsInGPU.nMemoryLocations, nMemoryLocations, sizeof(unsigned int), stream);
    hipStreamSynchronize(stream);
}

SDL::segments::segments()
{
    superbin = nullptr;
    pixelType = nullptr;
    isQuad = nullptr;
    isDup = nullptr;
    score = nullptr;
    circleRadius = nullptr;
    charge = nullptr;
    seedIdx = nullptr;
    circleCenterX = nullptr;
    circleCenterY = nullptr;
    mdIndices = nullptr;
    innerLowerModuleIndices = nullptr;
    outerLowerModuleIndices = nullptr;
    innerMiniDoubletAnchorHitIndices = nullptr;
    outerMiniDoubletAnchorHitIndices = nullptr;

    nSegments = nullptr;
    totOccupancySegments = nullptr;
    dPhis = nullptr;
    dPhiMins = nullptr;
    dPhiMaxs = nullptr;
    dPhiChanges = nullptr;
    dPhiChangeMins = nullptr;
    dPhiChangeMaxs = nullptr;
    partOfPT5 = nullptr;
    pLSHitsIdxs = nullptr;
}

SDL::segments::~segments()
{
}

void SDL::segments::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,mdIndices);
    cms::cuda::free_device(dev,innerLowerModuleIndices);
    cms::cuda::free_device(dev,dPhis);
    cms::cuda::free_device(dev,ptIn);
    cms::cuda::free_device(dev,nSegments);
    cms::cuda::free_device(dev,totOccupancySegments);
    cms::cuda::free_device(dev, charge);
    cms::cuda::free_device(dev, seedIdx);
    cms::cuda::free_device(dev,superbin);
    cms::cuda::free_device(dev,pixelType);
    cms::cuda::free_device(dev,isQuad);
    cms::cuda::free_device(dev,isDup);
    cms::cuda::free_device(dev,score);
    cms::cuda::free_device(dev, circleCenterX);
    cms::cuda::free_device(dev, circleCenterY);
    cms::cuda::free_device(dev, circleRadius);
    cms::cuda::free_device(dev, partOfPT5);
    cms::cuda::free_device(dev, pLSHitsIdxs);
    cms::cuda::free_device(dev, nMemoryLocations);
}

void SDL::segments::freeMemory(hipStream_t stream)
{
    hipFree(mdIndices);
    hipFree(innerLowerModuleIndices);
    hipFree(nSegments);
    hipFree(totOccupancySegments);
    hipFree(dPhis);
    hipFree(ptIn);
    hipFree(superbin);
    hipFree(pixelType);
    hipFree(isQuad);
    hipFree(isDup);
    hipFree(score);
    hipFree(charge);
    hipFree(seedIdx);
    hipFree(circleCenterX);
    hipFree(circleCenterY);
    hipFree(circleRadius);
    hipFree(partOfPT5);
    hipFree(pLSHitsIdxs);
    hipFree(nMemoryLocations);
}

void SDL::printSegment(struct SDL::segments& segmentsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::hits& hitsInGPU, struct SDL::modules& modulesInGPU, unsigned int segmentIndex)
{
    unsigned int innerMDIndex = segmentsInGPU.mdIndices[segmentIndex * 2];
    unsigned int outerMDIndex = segmentsInGPU.mdIndices[segmentIndex * 2 + 1];
    std::cout<<std::endl;
    std::cout<<"sg_dPhiChange : "<<__H2F(segmentsInGPU.dPhiChanges[segmentIndex]) << std::endl<<std::endl;

    std::cout << "Inner Mini-Doublet" << std::endl;
    std::cout << "------------------------------" << std::endl;
    {
        IndentingOStreambuf indent(std::cout);
        printMD(mdsInGPU, hitsInGPU, modulesInGPU, innerMDIndex);
    }
    std::cout<<std::endl<<" Outer Mini-Doublet" <<std::endl;
    std::cout << "------------------------------" << std::endl;
    {
        IndentingOStreambuf indent(std::cout);
        printMD(mdsInGPU, hitsInGPU, modulesInGPU, outerMDIndex);
    }
}

__global__ void SDL::addSegmentRangesToEventExplicit(struct modules& modulesInGPU, struct segments& segmentsInGPU, struct objectRanges& rangesInGPU)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int np = gridDim.x * blockDim.x;
    for(uint16_t i = gid; i < *modulesInGPU.nLowerModules; i+= np)
    {
        if(segmentsInGPU.nSegments[i] == 0)
        {
            rangesInGPU.segmentRanges[i * 2] = -1;
            rangesInGPU.segmentRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU.segmentRanges[i * 2] = rangesInGPU.segmentModuleIndices[i];
            rangesInGPU.segmentRanges[i * 2 + 1] = rangesInGPU.segmentModuleIndices[i] + segmentsInGPU.nSegments[i] - 1;
        }
    }
}