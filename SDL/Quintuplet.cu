# include "Quintuplet.cuh"

SDL::quintuplets::quintuplets()
{
    tripletIndices = nullptr;
    lowerModuleIndices = nullptr;
    nQuintuplets = nullptr;
    totOccupancyQuintuplets = nullptr;
    innerRadius = nullptr;
    outerRadius = nullptr;
    regressionRadius = nullptr;
    isDup = nullptr;
    TightCutFlag = nullptr;
    partOfPT5 = nullptr;
    pt = nullptr;
    layer = nullptr;
    regressionG = nullptr;
    regressionF = nullptr;
    logicalLayers = nullptr;
    hitIndices = nullptr;
    bridgeRadius = nullptr;
    chiSquared = nullptr;
    rzChiSquared = nullptr;
    nonAnchorChiSquared = nullptr;
}

SDL::quintuplets::~quintuplets()
{
}

void SDL::quintuplets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev, tripletIndices);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, nQuintuplets);
    cms::cuda::free_device(dev, totOccupancyQuintuplets);
    cms::cuda::free_device(dev, innerRadius);
    cms::cuda::free_device(dev, outerRadius);
    cms::cuda::free_device(dev, partOfPT5);
    cms::cuda::free_device(dev, isDup);
    cms::cuda::free_device(dev, TightCutFlag);
    cms::cuda::free_device(dev, pt);
    cms::cuda::free_device(dev, layer);
    cms::cuda::free_device(dev, regressionG);
    cms::cuda::free_device(dev, regressionF);
    cms::cuda::free_device(dev, regressionRadius);
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, nMemoryLocations);
    cms::cuda::free_device(dev, bridgeRadius);
    cms::cuda::free_device(dev, rzChiSquared);
    cms::cuda::free_device(dev, chiSquared);
    cms::cuda::free_device(dev, nonAnchorChiSquared);
}

void SDL::quintuplets::freeMemory(hipStream_t stream)
{
    hipFree(tripletIndices);
    hipFree(lowerModuleIndices);
    hipFree(nQuintuplets);
    hipFree(totOccupancyQuintuplets);
    hipFree(innerRadius);
    hipFree(outerRadius);
    hipFree(regressionRadius);
    hipFree(partOfPT5);
    hipFree(isDup);
    hipFree(TightCutFlag);
    hipFree(pt);
    hipFree(layer);
    hipFree(regressionG);
    hipFree(regressionF);
    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(nMemoryLocations);
    hipFree(bridgeRadius);
    hipFree(rzChiSquared);
    hipFree(chiSquared);
    hipFree(nonAnchorChiSquared);
    hipStreamSynchronize(stream);
}

void SDL::createQuintupletsInExplicitMemory(struct SDL::quintuplets& quintupletsInGPU, const unsigned int& nTotalQuintuplets, const uint16_t& nLowerModules, const uint16_t& nEligibleModules,hipStream_t stream)
{
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    quintupletsInGPU.tripletIndices = (unsigned int*)cms::cuda::allocate_device(dev, 2 * nTotalQuintuplets * sizeof(unsigned int), stream);
    quintupletsInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev, 5 * nTotalQuintuplets * sizeof(uint16_t), stream);
    quintupletsInGPU.nQuintuplets = (int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(int), stream);
    quintupletsInGPU.totOccupancyQuintuplets = (int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(int), stream);
    quintupletsInGPU.innerRadius = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(FPX), stream);
    quintupletsInGPU.outerRadius = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(FPX), stream);
    quintupletsInGPU.bridgeRadius = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);

    quintupletsInGPU.pt = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets *4* sizeof(FPX), stream);
    quintupletsInGPU.layer = (uint8_t*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(uint8_t), stream);
    quintupletsInGPU.isDup = (bool*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(bool), stream);
    quintupletsInGPU.TightCutFlag = (bool*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(bool), stream);
    quintupletsInGPU.partOfPT5 = (bool*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(bool), stream);
    quintupletsInGPU.regressionRadius = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.regressionG = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.regressionF = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(uint8_t) * 5, stream);
    quintupletsInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(unsigned int) * 10, stream);
    quintupletsInGPU.nMemoryLocations = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);

    quintupletsInGPU.rzChiSquared = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.chiSquared = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.nonAnchorChiSquared = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
#else
    hipMalloc(&quintupletsInGPU.tripletIndices, 2 * nTotalQuintuplets * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.lowerModuleIndices, 5 * nTotalQuintuplets * sizeof(uint16_t));
    hipMalloc(&quintupletsInGPU.nQuintuplets, nLowerModules * sizeof(int));
    hipMalloc(&quintupletsInGPU.totOccupancyQuintuplets, nLowerModules * sizeof(int));
    hipMalloc(&quintupletsInGPU.innerRadius, nTotalQuintuplets * sizeof(FPX));
    hipMalloc(&quintupletsInGPU.outerRadius, nTotalQuintuplets * sizeof(FPX));
    hipMalloc(&quintupletsInGPU.pt, nTotalQuintuplets *4* sizeof(FPX));
    hipMalloc(&quintupletsInGPU.isDup, nTotalQuintuplets * sizeof(bool));
    hipMalloc(&quintupletsInGPU.TightCutFlag, nTotalQuintuplets * sizeof(bool));
    hipMalloc(&quintupletsInGPU.partOfPT5, nTotalQuintuplets * sizeof(bool));
    hipMalloc(&quintupletsInGPU.layer, nTotalQuintuplets * sizeof(uint8_t));
    hipMalloc(&quintupletsInGPU.regressionRadius, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.regressionG, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.regressionF, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.logicalLayers, nTotalQuintuplets * 5 * sizeof(uint8_t));
    hipMalloc(&quintupletsInGPU.hitIndices, nTotalQuintuplets * 10 * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.nMemoryLocations, sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.bridgeRadius, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.rzChiSquared, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.chiSquared, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.nonAnchorChiSquared, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.nMemoryLocations, sizeof(unsigned int));
#endif
    hipMemsetAsync(quintupletsInGPU.nQuintuplets,0,nLowerModules * sizeof(int),stream);
    hipMemsetAsync(quintupletsInGPU.totOccupancyQuintuplets,0,nLowerModules * sizeof(int),stream);
    hipMemsetAsync(quintupletsInGPU.isDup,0,nTotalQuintuplets * sizeof(bool),stream);
    hipMemsetAsync(quintupletsInGPU.TightCutFlag,0,nTotalQuintuplets * sizeof(bool),stream);
    hipMemsetAsync(quintupletsInGPU.partOfPT5,0,nTotalQuintuplets * sizeof(bool),stream);
    hipStreamSynchronize(stream);
    quintupletsInGPU.eta = quintupletsInGPU.pt + nTotalQuintuplets;
    quintupletsInGPU.phi = quintupletsInGPU.pt + 2*nTotalQuintuplets;
    quintupletsInGPU.score_rphisum = quintupletsInGPU.pt + 3*nTotalQuintuplets;
}