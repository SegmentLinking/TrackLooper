#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif
# include "Quintuplet.cuh"
#include "allocate.h"
#include "Kernels.cuh"

SDL::quintuplets::quintuplets()
{
    tripletIndices = nullptr;
    lowerModuleIndices = nullptr;
    nQuintuplets = nullptr;
    totOccupancyQuintuplets = nullptr;
    innerRadius = nullptr;
    outerRadius = nullptr;
    regressionRadius = nullptr;
    isDup = nullptr;
    partOfPT5 = nullptr;
    pt = nullptr;
    layer = nullptr;
    regressionG = nullptr;
    regressionF = nullptr;

    logicalLayers = nullptr;
    hitIndices = nullptr;
#ifdef CUT_VALUE_DEBUG
    innerRadiusMin = nullptr;
    innerRadiusMin2S = nullptr;
    innerRadiusMax = nullptr;
    innerRadiusMax2S = nullptr;
    bridgeRadius = nullptr;
    bridgeRadiusMin = nullptr;
    bridgeRadiusMin2S = nullptr;
    bridgeRadiusMax = nullptr;
    bridgeRadiusMax2S = nullptr;
    outerRadiusMin = nullptr;
    outerRadiusMin2S = nullptr;
    outerRadiusMax = nullptr;
    outerRadiusMax2S = nullptr;
    chiSquared = nullptr;
    nonAnchorChiSquared = nullptr;
#endif
}

SDL::quintuplets::~quintuplets()
{
}

void SDL::quintuplets::freeMemoryCache()
{
#ifdef Explicit_T5
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,tripletIndices);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, nQuintuplets);
    cms::cuda::free_device(dev, totOccupancyQuintuplets);
    cms::cuda::free_device(dev, innerRadius);
    cms::cuda::free_device(dev, outerRadius);
    cms::cuda::free_device(dev, partOfPT5);
    cms::cuda::free_device(dev, isDup);
    cms::cuda::free_device(dev, pt);
    cms::cuda::free_device(dev, layer);
    cms::cuda::free_device(dev, regressionG);
    cms::cuda::free_device(dev, regressionF);
    cms::cuda::free_device(dev, regressionRadius);
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, hitIndices);
#else
    cms::cuda::free_managed(tripletIndices);
    cms::cuda::free_managed(lowerModuleIndices);
    cms::cuda::free_managed(nQuintuplets);
    cms::cuda::free_managed(totOccupancyQuintuplets);
    cms::cuda::free_managed(innerRadius);
    cms::cuda::free_managed(outerRadius);
    cms::cuda::free_managed(partOfPT5);
    cms::cuda::free_managed(isDup);
    cms::cuda::free_managed(pt);
    cms::cuda::free_managed(layer);
    cms::cuda::free_managed(regressionG);
    cms::cuda::free_managed(regressionF);
    cms::cuda::free_managed(regressionRadius);

    cms::cuda::free_managed(logicalLayers);
    cms::cuda::free_managed(hitIndices);
#endif
}

void SDL::quintuplets::freeMemory(hipStream_t stream)
{
    hipFree(tripletIndices);
    hipFree(lowerModuleIndices);
    hipFree(nQuintuplets);
    hipFree(totOccupancyQuintuplets);
    hipFree(innerRadius);
    hipFree(outerRadius);
    hipFree(regressionRadius);
    hipFree(partOfPT5);
    hipFree(isDup);
    hipFree(pt);
    hipFree(layer);
    hipFree(regressionG);
    hipFree(regressionF);
    hipFree(logicalLayers);
    hipFree(hitIndices);
#ifdef CUT_VALUE_DEBUG
    hipFree(innerRadiusMin);
    hipFree(innerRadiusMin2S);
    hipFree(innerRadiusMax);
    hipFree(innerRadiusMax2S);
    hipFree(bridgeRadius);
    hipFree(bridgeRadiusMin);
    hipFree(bridgeRadiusMin2S);
    hipFree(bridgeRadiusMax);
    hipFree(bridgeRadiusMax2S);
    hipFree(outerRadiusMin);
    hipFree(outerRadiusMin2S);
    hipFree(outerRadiusMax);
    hipFree(outerRadiusMax2S);
    hipFree(chiSquared);
    hipFree(nonAnchorChiSquared);
#endif
hipStreamSynchronize(stream);
}

void SDL::createEligibleModulesListForQuintuplets(struct modules& modulesInGPU,struct triplets& tripletsInGPU, uint16_t& nEligibleModules, uint16_t* indicesOfEligibleModules, unsigned int maxQuintuplets, unsigned int& maxTriplets,hipStream_t stream,struct objectRanges& rangesInGPU)
{
    uint16_t nLowerModules;
    maxTriplets = 0;
    hipMemcpyAsync(&nLowerModules,modulesInGPU.nLowerModules,sizeof(uint16_t),hipMemcpyDeviceToHost,stream);

    hipMemsetAsync(rangesInGPU.quintupletModuleIndices, -1, sizeof(int) * (nLowerModules),stream);

    short* module_subdets;
    hipHostMalloc(&module_subdets, nLowerModules* sizeof(short));
    hipMemcpyAsync(module_subdets,modulesInGPU.subdets,nLowerModules*sizeof(short),hipMemcpyDeviceToHost,stream);
    short* module_layers;
    hipHostMalloc(&module_layers, nLowerModules * sizeof(short));
    hipMemcpyAsync(module_layers,modulesInGPU.layers,nLowerModules * sizeof(short),hipMemcpyDeviceToHost,stream);

    int* module_quintupletModuleIndices;
    hipHostMalloc(&module_quintupletModuleIndices, nLowerModules * sizeof(int));
    hipMemcpyAsync(module_quintupletModuleIndices,rangesInGPU.quintupletModuleIndices,nLowerModules *sizeof(int),hipMemcpyDeviceToHost,stream);

    unsigned int* nTriplets;
    hipHostMalloc(&nTriplets, nLowerModules * sizeof(unsigned int));
    hipMemcpyAsync(nTriplets, tripletsInGPU.nTriplets, nLowerModules * sizeof(unsigned int), hipMemcpyDeviceToHost,stream);
hipStreamSynchronize(stream);

    //start filling
    for(uint16_t i = 0; i < nLowerModules; i++)
    {
        //condition for a quintuple to exist for a module
        //TCs don't exist for layers 5 and 6 barrel, and layers 2,3,4,5 endcap
        if(((module_subdets[i] == SDL::Barrel and module_layers[i] < 3) or (module_subdets[i] == SDL::Endcap and module_layers[i] == 1)) and nTriplets[i] != 0)
        {
            module_quintupletModuleIndices[i] = nEligibleModules * maxQuintuplets; //for variable occupancy change this to module_quintupletModuleIndices[i-1] + blah
            indicesOfEligibleModules[nEligibleModules] = i;
            nEligibleModules++;
            maxTriplets = max(nTriplets[i], maxTriplets);
        }
    }
    hipMemcpyAsync(rangesInGPU.quintupletModuleIndices,module_quintupletModuleIndices,nLowerModules*sizeof(int),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(rangesInGPU.nEligibleT5Modules,&nEligibleModules,sizeof(uint16_t),hipMemcpyHostToDevice,stream);
hipStreamSynchronize(stream);
    hipHostFree(module_subdets);
    hipHostFree(module_layers);
    hipHostFree(module_quintupletModuleIndices);
    hipHostFree(nTriplets);
}


void SDL::createQuintupletsInUnifiedMemory(struct SDL::quintuplets& quintupletsInGPU, const unsigned int& maxQuintuplets, const uint16_t& nLowerModules, const uint16_t& nEligibleModules, hipStream_t stream)
{
    unsigned int nMemoryLocations = maxQuintuplets * nEligibleModules;
//    std::cout<<"Number of eligible T5 modules = "<<nEligibleModules<<std::endl;

#ifdef CACHE_ALLOC
//    hipStream_t stream = 0;
    quintupletsInGPU.tripletIndices = (unsigned int*)cms::cuda::allocate_managed(nMemoryLocations * 2 * sizeof(unsigned int), stream);
    quintupletsInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_managed(nMemoryLocations * 5 * sizeof(uint16_t), stream);
    quintupletsInGPU.nQuintuplets = (unsigned int*)cms::cuda::allocate_managed(nLowerModules * sizeof(unsigned int), stream);
    quintupletsInGPU.totOccupancyQuintuplets = (unsigned int*)cms::cuda::allocate_managed(nLowerModules * sizeof(unsigned int), stream);
    quintupletsInGPU.innerRadius = (FPX*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(FPX), stream);
    quintupletsInGPU.outerRadius = (FPX*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(FPX), stream);
    quintupletsInGPU.pt = (FPX*)cms::cuda::allocate_managed(nMemoryLocations *4* sizeof(FPX), stream);
    quintupletsInGPU.layer = (uint8_t*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(uint8_t), stream);
    quintupletsInGPU.isDup = (bool*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(bool), stream);
    quintupletsInGPU.partOfPT5 = (bool*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(bool), stream);
    quintupletsInGPU.regressionRadius = (float*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(float), stream);
    quintupletsInGPU.regressionG = (float*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(float), stream);
    quintupletsInGPU.regressionF = (float*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(float), stream);
    quintupletsInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(uint8_t) * 5, stream);
    quintupletsInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_managed(nMemoryLocations * sizeof(unsigned int) * 10, stream);
#else
    hipMallocManaged(&quintupletsInGPU.tripletIndices, 2 * nMemoryLocations * sizeof(unsigned int));
    hipMallocManaged(&quintupletsInGPU.lowerModuleIndices, 5 * nMemoryLocations * sizeof(uint16_t));

    hipMallocManaged(&quintupletsInGPU.nQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMallocManaged(&quintupletsInGPU.totOccupancyQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMallocManaged(&quintupletsInGPU.innerRadius, nMemoryLocations * sizeof(FPX));
    hipMallocManaged(&quintupletsInGPU.outerRadius, nMemoryLocations * sizeof(FPX));
    hipMallocManaged(&quintupletsInGPU.pt, nMemoryLocations *4* sizeof(FPX));
    hipMallocManaged(&quintupletsInGPU.layer, nMemoryLocations * sizeof(uint8_t));
    hipMallocManaged(&quintupletsInGPU.isDup, nMemoryLocations * sizeof(bool));
    hipMallocManaged(&quintupletsInGPU.partOfPT5, nMemoryLocations * sizeof(bool));
    hipMallocManaged(&quintupletsInGPU.regressionRadius, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.regressionG, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.regressionF, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.logicalLayers, nMemoryLocations * sizeof(uint8_t) * 5);
    hipMallocManaged(&quintupletsInGPU.hitIndices, nMemoryLocations * sizeof(unsigned int) * 10);
#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&quintupletsInGPU.innerRadiusMin, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMax, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadius, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMin, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMax, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMin, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMax, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMin2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.innerRadiusMax2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMin2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.bridgeRadiusMax2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMin2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.outerRadiusMax2S, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.chiSquared, nMemoryLocations * sizeof(float));
    hipMallocManaged(&quintupletsInGPU.nonAnchorChiSquared, nMemoryLocations * sizeof(float));
#endif
#endif
    quintupletsInGPU.eta = quintupletsInGPU.pt + nMemoryLocations;
    quintupletsInGPU.phi = quintupletsInGPU.pt + 2*nMemoryLocations;
    //quintupletsInGPU.score_rphi = quintupletsInGPU.pt + 3*nMemoryLocations;
    //quintupletsInGPU.score_rz = quintupletsInGPU.pt + 4*nMemoryLocations;
    quintupletsInGPU.score_rphisum = quintupletsInGPU.pt + 3*nMemoryLocations;
    //quintupletsInGPU.score_rzlsq = quintupletsInGPU.pt + 6*nMemoryLocations;
//#pragma omp parallel for
//    for(size_t i = 0; i<nLowerModules;i++)
//    {
//        quintupletsInGPU.nQuintuplets[i] = 0;
//    }

    hipMemsetAsync(quintupletsInGPU.nQuintuplets,0,nLowerModules * sizeof(unsigned int),stream);
    hipMemsetAsync(quintupletsInGPU.totOccupancyQuintuplets,0,nLowerModules * sizeof(unsigned int),stream);
    hipMemsetAsync(quintupletsInGPU.isDup,0,nMemoryLocations * sizeof(bool),stream);
    hipMemsetAsync(quintupletsInGPU.partOfPT5,0,nMemoryLocations * sizeof(bool),stream);
    hipStreamSynchronize(stream);
}

void SDL::createQuintupletsInExplicitMemory(struct SDL::quintuplets& quintupletsInGPU, const unsigned int& maxQuintuplets, const uint16_t& nLowerModules, const uint16_t& nEligibleModules,hipStream_t stream)
{
    unsigned int nMemoryLocations = nEligibleModules * maxQuintuplets;
#ifdef CACHE_ALLOC
 //   hipStream_t stream = 0;
    int dev;
    hipGetDevice(&dev);
    quintupletsInGPU.tripletIndices = (unsigned int*)cms::cuda::allocate_device(dev, 2 * nMemoryLocations * sizeof(unsigned int), stream);
    quintupletsInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev, 5 * nMemoryLocations * sizeof(uint16_t), stream);
    quintupletsInGPU.nQuintuplets = (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int), stream);
    quintupletsInGPU.totOccupancyQuintuplets = (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int), stream);
    quintupletsInGPU.innerRadius = (FPX*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(FPX), stream);
    quintupletsInGPU.outerRadius = (FPX*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(FPX), stream);
    quintupletsInGPU.pt = (FPX*)cms::cuda::allocate_device(dev, nMemoryLocations *4* sizeof(FPX), stream);
    quintupletsInGPU.layer = (uint8_t*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(uint8_t), stream);
    quintupletsInGPU.isDup = (bool*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(bool), stream);
    quintupletsInGPU.partOfPT5 = (bool*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(bool), stream);
    quintupletsInGPU.regressionRadius = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(float), stream);
    quintupletsInGPU.regressionG = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(float), stream);
    quintupletsInGPU.regressionF = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(float), stream);
    quintupletsInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(uint8_t) * 5, stream);
    quintupletsInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(unsigned int) * 10, stream);
#else
    hipMalloc(&quintupletsInGPU.tripletIndices, 2 * nMemoryLocations * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.lowerModuleIndices, 5 * nMemoryLocations * sizeof(uint16_t));
    hipMalloc(&quintupletsInGPU.nQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.totOccupancyQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.innerRadius, nMemoryLocations * sizeof(FPX));
    hipMalloc(&quintupletsInGPU.outerRadius, nMemoryLocations * sizeof(FPX));
    hipMalloc(&quintupletsInGPU.pt, nMemoryLocations *4* sizeof(FPX));
    hipMalloc(&quintupletsInGPU.isDup, nMemoryLocations * sizeof(bool));
    hipMalloc(&quintupletsInGPU.partOfPT5, nMemoryLocations * sizeof(bool));
    hipMalloc(&quintupletsInGPU.layer, nMemoryLocations * sizeof(uint8_t));
    hipMalloc(&quintupletsInGPU.regressionRadius, nMemoryLocations * sizeof(float));
    hipMalloc(&quintupletsInGPU.regressionG, nMemoryLocations * sizeof(float));
    hipMalloc(&quintupletsInGPU.regressionF, nMemoryLocations * sizeof(float));
    hipMalloc(&quintupletsInGPU.logicalLayers, nMemoryLocations * 5 * sizeof(uint8_t));
    hipMalloc(&quintupletsInGPU.hitIndices, nMemoryLocations * 10 * sizeof(unsigned int));
#endif
    hipMemsetAsync(quintupletsInGPU.nQuintuplets,0,nLowerModules * sizeof(unsigned int),stream);
    hipMemsetAsync(quintupletsInGPU.totOccupancyQuintuplets,0,nLowerModules * sizeof(unsigned int),stream);
    hipMemsetAsync(quintupletsInGPU.isDup,0,nMemoryLocations * sizeof(bool),stream);
    hipMemsetAsync(quintupletsInGPU.partOfPT5,0,nMemoryLocations * sizeof(bool),stream);
    hipStreamSynchronize(stream);
    quintupletsInGPU.eta = quintupletsInGPU.pt + nMemoryLocations;
    quintupletsInGPU.phi = quintupletsInGPU.pt + 2*nMemoryLocations;
    quintupletsInGPU.score_rphisum = quintupletsInGPU.pt + 3*nMemoryLocations;
}


#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addQuintupletToMemory(struct SDL::triplets& tripletsInGPU, struct SDL::triplets& tripletsInwardInGPU, struct SDL::quintuplets& quintupletsInGPU, unsigned int innerTripletInwardIndex, unsigned int outerTripletIndex, uint16_t& lowerModule1, uint16_t& lowerModule2, uint16_t& lowerModule3, uint16_t& lowerModule4, uint16_t& lowerModule5, float innerRadius, float innerRadiusMin, float innerRadiusMax, float outerRadius, float outerRadiusMin, float outerRadiusMax, float bridgeRadius, float bridgeRadiusMin, float bridgeRadiusMax,
        float innerRadiusMin2S, float innerRadiusMax2S, float bridgeRadiusMin2S, float bridgeRadiusMax2S, float outerRadiusMin2S, float outerRadiusMax2S, float regressionG, float regressionF, float regressionRadius, float chiSquared, float nonAnchorChiSquared, float pt, float eta, float phi, float scores, uint8_t layer, unsigned int quintupletIndex)
#else
__device__ void SDL::addQuintupletToMemory(struct SDL::triplets& tripletsInGPU, struct SDL::triplets& tripletsInwardInGPU, struct SDL::quintuplets& quintupletsInGPU, unsigned int innerTripletInwardIndex, unsigned int outerTripletIndex, uint16_t& lowerModule1, uint16_t& lowerModule2, uint16_t& lowerModule3, uint16_t& lowerModule4, uint16_t& lowerModule5, float innerRadius, float outerRadius, float regressionG, float regressionF, float regressionRadius, float pt, float eta, float phi, float scores, uint8_t layer, unsigned int quintupletIndex)
#endif

{
    //translate from inwards index to "regular" T3 index
    quintupletsInGPU.tripletIndices[2 * quintupletIndex] = tripletsInwardInGPU.outwardT3Indices[innerTripletInwardIndex];
    quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1] = outerTripletIndex;

    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex] = lowerModule1;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 1] = lowerModule2;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 2] = lowerModule3;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 3] = lowerModule4;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 4] = lowerModule5;
    quintupletsInGPU.innerRadius[quintupletIndex] = __F2H(innerRadius);
    quintupletsInGPU.outerRadius[quintupletIndex] = __F2H(outerRadius);
    quintupletsInGPU.pt[quintupletIndex] = __F2H(pt);
    quintupletsInGPU.eta[quintupletIndex] = __F2H(eta);
    quintupletsInGPU.phi[quintupletIndex] = __F2H(phi);
    quintupletsInGPU.score_rphisum[quintupletIndex] = __F2H(scores);
    quintupletsInGPU.layer[quintupletIndex] = layer;
    quintupletsInGPU.isDup[quintupletIndex] = false;
    quintupletsInGPU.regressionRadius[quintupletIndex] = regressionRadius;
    quintupletsInGPU.regressionG[quintupletIndex] = regressionG;
    quintupletsInGPU.regressionF[quintupletIndex] = regressionF;

    quintupletsInGPU.logicalLayers[5 * quintupletIndex] = tripletsInwardInGPU.logicalLayers[3 * innerTripletInwardIndex];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 1] = tripletsInwardInGPU.logicalLayers[3 * innerTripletInwardIndex + 1];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 2] = tripletsInwardInGPU.logicalLayers[3 * innerTripletInwardIndex + 2];

    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 3] = tripletsInGPU.logicalLayers[3 * outerTripletIndex + 1];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 4] = tripletsInGPU.logicalLayers[3 * outerTripletIndex + 2];
    //printf("logicalLayers %u %u %u %u %u\n",quintupletsInGPU.logicalLayers[5*quintupletIndex],quintupletsInGPU.logicalLayers[5*quintupletIndex+1],quintupletsInGPU.logicalLayers[5*quintupletIndex+2],quintupletsInGPU.logicalLayers[5*quintupletIndex+3],quintupletsInGPU.logicalLayers[5*quintupletIndex+4]);

    quintupletsInGPU.hitIndices[10 * quintupletIndex] = tripletsInwardInGPU.hitIndices[6 * innerTripletInwardIndex];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 1] = tripletsInwardInGPU.hitIndices[6 * innerTripletInwardIndex + 1];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 2] = tripletsInwardInGPU.hitIndices[6 * innerTripletInwardIndex + 2];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 3] = tripletsInwardInGPU.hitIndices[6 * innerTripletInwardIndex + 3];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 4] = tripletsInwardInGPU.hitIndices[6 * innerTripletInwardIndex + 4];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 5] = tripletsInwardInGPU.hitIndices[6 * innerTripletInwardIndex + 5];

    quintupletsInGPU.hitIndices[10 * quintupletIndex + 6] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 2];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 7] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 3];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 8] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 4];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 9] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 5];
#ifdef CUT_VALUE_DEBUG
    quintupletsInGPU.innerRadiusMin[quintupletIndex] = 1.0/innerInvRadiusMin;
    quintupletsInGPU.innerRadiusMax[quintupletIndex] = 1.0/innerInvRadiusMax;
    quintupletsInGPU.outerRadiusMin[quintupletIndex] = 1.0/outerInvRadiusMin;
    quintupletsInGPU.outerRadiusMax[quintupletIndex] = 1.0/outerInvRadiusMax;
    quintupletsInGPU.bridgeRadius[quintupletIndex] = bridgeRadius;
    quintupletsInGPU.bridgeRadiusMin[quintupletIndex] = 1.0/bridgeInvRadiusMin;
    quintupletsInGPU.bridgeRadiusMax[quintupletIndex] = 1.0/bridgeInvRadiusMax;
    quintupletsInGPU.innerRadiusMin2S[quintupletIndex] = innerRadiusMin2S;
    quintupletsInGPU.innerRadiusMax2S[quintupletIndex] = innerRadiusMax2S;
    quintupletsInGPU.bridgeRadiusMin2S[quintupletIndex] = bridgeRadiusMin2S;
    quintupletsInGPU.bridgeRadiusMax2S[quintupletIndex] = bridgeRadiusMax2S;
    quintupletsInGPU.outerRadiusMin2S[quintupletIndex] = outerRadiusMin2S;
    quintupletsInGPU.outerRadiusMax2S[quintupletIndex] = outerRadiusMax2S;
    quintupletsInGPU.chiSquared[quintupletIndex] = chiSquared;
    quintupletsInGPU.nonAnchorChiSquared[quintupletIndex] = nonAnchorChiSquared;
#endif

}
__device__ void SDL::rmQuintupletToMemory(struct SDL::quintuplets& quintupletsInGPU,unsigned int quintupletIndex)
{
    quintupletsInGPU.isDup[quintupletIndex] = true;

}

__device__ bool SDL::runQuintupletDefaultAlgo(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::triplets& tripletsInwardInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, unsigned int& innerTripletInwardIndex, unsigned int& outerTripletIndex, float& innerRadius, float& innerInvRadiusMin, float&
    innerInvRadiusMax, float& outerRadius, float& outerInvRadiusMin, float& outerInvRadiusMax, float& bridgeRadius, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& innerRadiusMin2S, float& innerRadiusMax2S, float& bridgeRadiusMin2S, float& bridgeRadiusMax2S, float& outerRadiusMin2S, float& outerRadiusMax2S, float& regressionG, float& regressionF, float& regressionRadius, float& chiSquared, float& nonAnchorChiSquared)
{
    bool pass = true;

    unsigned int firstSegmentIndex = tripletsInwardInGPU.segmentIndices[2 * innerTripletInwardIndex];
    unsigned int secondSegmentIndex = tripletsInwardInGPU.segmentIndices[2 * innerTripletInwardIndex + 1];
    unsigned int thirdSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int fourthSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex + 1];

    unsigned int innerOuterOuterMiniDoubletIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex + 1]; //inner triplet outer segment outer MD index
    unsigned int outerInnerInnerMiniDoubletIndex = segmentsInGPU.mdIndices[2 * thirdSegmentIndex]; //outer triplet inner segmnet inner MD index

    if (innerOuterOuterMiniDoubletIndex != outerInnerInnerMiniDoubletIndex) pass = false;


    //apply T4 criteria between segments 1 and 3
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    unsigned int firstMDIndex = segmentsInGPU.mdIndices[2 * firstSegmentIndex];
    unsigned int secondMDIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex];
    unsigned int thirdMDIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex + 1];
    unsigned int fourthMDIndex = segmentsInGPU.mdIndices[2 * thirdSegmentIndex + 1];
    unsigned int fifthMDIndex = segmentsInGPU.mdIndices[2 * fourthSegmentIndex + 1];

    pass = pass & runTrackletDefaultAlgo(modulesInGPU, mdsInGPU, segmentsInGPU, segmentsInGPU.innerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.outerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.innerLowerModuleIndices[thirdSegmentIndex], segmentsInGPU.outerLowerModuleIndices[thirdSegmentIndex], firstSegmentIndex, thirdSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    pass = pass & runTrackletDefaultAlgo(modulesInGPU, mdsInGPU, segmentsInGPU, segmentsInGPU.innerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.outerLowerModuleIndices[firstSegmentIndex], segmentsInGPU.innerLowerModuleIndices[fourthSegmentIndex], segmentsInGPU.outerLowerModuleIndices[fourthSegmentIndex], firstSegmentIndex, fourthSegmentIndex, firstMDIndex, secondMDIndex, fourthMDIndex, fifthMDIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    pass = pass & passT5RZConstraint(modulesInGPU, mdsInGPU, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, fifthMDIndex, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5);

    float x1 = mdsInGPU.anchorX[firstMDIndex];
    float x2 = mdsInGPU.anchorX[secondMDIndex];
    float x3 = mdsInGPU.anchorX[thirdMDIndex];
    float x4 = mdsInGPU.anchorX[fourthMDIndex];
    float x5 = mdsInGPU.anchorX[fifthMDIndex];
    
    float y1 = mdsInGPU.anchorY[firstMDIndex];
    float y2 = mdsInGPU.anchorY[secondMDIndex];
    float y3 = mdsInGPU.anchorY[thirdMDIndex];
    float y4 = mdsInGPU.anchorY[fourthMDIndex];
    float y5 = mdsInGPU.anchorY[fifthMDIndex];

    //non anchor is always shifted for tilted and endcap!
    float x1NonAnchor = mdsInGPU.outerX[firstMDIndex];
    float x2NonAnchor = mdsInGPU.outerX[secondMDIndex];
    float x3NonAnchor = mdsInGPU.outerX[thirdMDIndex];
    float x4NonAnchor = mdsInGPU.outerX[fourthMDIndex];
    float x5NonAnchor = mdsInGPU.outerX[fifthMDIndex];
    
    float y1NonAnchor = mdsInGPU.outerY[firstMDIndex];
    float y2NonAnchor = mdsInGPU.outerY[secondMDIndex];
    float y3NonAnchor = mdsInGPU.outerY[thirdMDIndex];
    float y4NonAnchor = mdsInGPU.outerY[fourthMDIndex];
    float y5NonAnchor = mdsInGPU.outerY[fifthMDIndex];


    //construct the arrays
    float x1Vec[] = {x1, x1, x1};
    float y1Vec[] = {y1, y1, y1};
    float x2Vec[] = {x2, x2, x2};
    float y2Vec[] = {y2, y2, y2};
    float x3Vec[] = {x3, x3, x3};
    float y3Vec[] = {y3, y3, y3};

    if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS)
    {
        x1Vec[1] = mdsInGPU.anchorLowEdgeX[firstMDIndex];
        x1Vec[2] = mdsInGPU.anchorHighEdgeX[firstMDIndex];

        y1Vec[1] = mdsInGPU.anchorLowEdgeY[firstMDIndex];
        y1Vec[2] = mdsInGPU.anchorHighEdgeY[firstMDIndex];
    }
    if(modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS)
    {
        x2Vec[1] = mdsInGPU.anchorLowEdgeX[secondMDIndex];
        x2Vec[2] = mdsInGPU.anchorHighEdgeX[secondMDIndex];

        y2Vec[1] = mdsInGPU.anchorLowEdgeY[secondMDIndex];
        y2Vec[2] = mdsInGPU.anchorHighEdgeY[secondMDIndex];
    }
    if(modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS)
    {
        x3Vec[1] = mdsInGPU.anchorLowEdgeX[thirdMDIndex];
        x3Vec[2] = mdsInGPU.anchorHighEdgeX[thirdMDIndex];

        y3Vec[1] = mdsInGPU.anchorLowEdgeY[thirdMDIndex];
        y3Vec[2] = mdsInGPU.anchorHighEdgeY[thirdMDIndex];
    }
    computeErrorInRadius(x1Vec, y1Vec, x2Vec, y2Vec, x3Vec, y3Vec, innerRadiusMin2S, innerRadiusMax2S);

    for (int i=0; i<3; i++) 
    {
      x1Vec[i] = x4;
      y1Vec[i] = y4;
    }
    if(modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS)
    {
        x1Vec[1] = mdsInGPU.anchorLowEdgeX[fourthMDIndex];
        x1Vec[2] = mdsInGPU.anchorHighEdgeX[fourthMDIndex];

        y1Vec[1] = mdsInGPU.anchorLowEdgeY[fourthMDIndex];
        y1Vec[2] = mdsInGPU.anchorHighEdgeY[fourthMDIndex];
    }
    computeErrorInRadius(x2Vec, y2Vec, x3Vec, y3Vec, x1Vec, y1Vec, bridgeRadiusMin2S, bridgeRadiusMax2S);

    for(int i=0; i<3; i++) 
    {
      x2Vec[i] = x5;
      y2Vec[i] = y5;
    }
    if(modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS)
    {
        x2Vec[1] = mdsInGPU.anchorLowEdgeX[fifthMDIndex];
        x2Vec[2] = mdsInGPU.anchorHighEdgeX[fifthMDIndex];

        y2Vec[1] = mdsInGPU.anchorLowEdgeY[fifthMDIndex];
        y2Vec[2] = mdsInGPU.anchorHighEdgeY[fifthMDIndex];
    }
    computeErrorInRadius(x3Vec, y3Vec, x1Vec, y1Vec, x2Vec, y2Vec, outerRadiusMin2S, outerRadiusMax2S);

    float g, f;
    innerRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3, g, f);
    outerRadius = computeRadiusFromThreeAnchorHits(x3, y3, x4, y4, x5, y5, g, f);
    bridgeRadius = computeRadiusFromThreeAnchorHits(x2, y2, x3, y3, x4, y4, g, f);


    pass = pass & (innerRadius >= 0.95f/(2.f * k2Rinv1GeVf));

    //split by category
    bool tempPass;
    if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Barrel)
    {
       tempPass = matchRadiiBBBBB(innerRadius, bridgeRadius, outerRadius, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBBBBE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        if(modulesInGPU.layers[lowerModuleIndex1] == 1)
        {
            tempPass = matchRadiiBBBEE12378(innerRadius, bridgeRadius, outerRadius,innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
        }
        else if(modulesInGPU.layers[lowerModuleIndex1] == 2)
        {
            tempPass = matchRadiiBBBEE23478(innerRadius, bridgeRadius, outerRadius,innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
        }
        else
        {
            tempPass = matchRadiiBBBEE34578(innerRadius, bridgeRadius, outerRadius,innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
        }
    }

    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBBEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBEEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
    }
    else
    {
        tempPass = matchRadiiEEEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S,innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
    }

    pass = pass & tempPass;


    //compute regression radius right here
    float xVec[] = {x1, x2, x3, x4, x5};
    float yVec[] = {y1, y2, y3, y4, y5};
    float sigmas[5], delta1[5], delta2[5], slopes[5];
    bool isFlat[5];
    //5 categories for sigmas
    const uint16_t lowerModuleIndices[] = {lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5};

    computeSigmasForRegression(modulesInGPU, lowerModuleIndices, delta1, delta2, slopes, isFlat);
    regressionRadius = computeRadiusUsingRegression(5,xVec, yVec, delta1, delta2, slopes, isFlat, regressionG, regressionF, sigmas, chiSquared);

    //extra chi squared cuts!
    if(regressionRadius < 5.0f/(2.f * k2Rinv1GeVf))
    {
        pass = pass & passChiSquaredConstraint(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, chiSquared);
    }
    //compute the other chisquared
    float nonAnchorDelta1[5], nonAnchorDelta2[5], nonAnchorSlopes[5];
    float nonAnchorxs[] = {x1NonAnchor, x2NonAnchor, x3NonAnchor, x4NonAnchor, x5NonAnchor};
    float nonAnchorys[] = {y1NonAnchor, y2NonAnchor, y3NonAnchor, y4NonAnchor, y5NonAnchor};

    computeSigmasForRegression(modulesInGPU, lowerModuleIndices, nonAnchorDelta1, nonAnchorDelta2, nonAnchorSlopes, isFlat, 5, false);
    nonAnchorChiSquared = computeChiSquared(5, nonAnchorxs, nonAnchorys, nonAnchorDelta1, nonAnchorDelta2, nonAnchorSlopes, isFlat, regressionG, regressionF, regressionRadius);
    return pass;
}

//90% constraint
__device__ bool SDL::passChiSquaredConstraint(struct SDL::modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& chiSquared)
{
    //following Philip's layer number prescription
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return chiSquared < 0.01788f;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return chiSquared < 0.04725f;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return chiSquared < 0.04725f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {       
            return chiSquared < 0.01788f;
        }   
        else if(layer4 == 9 and layer5 == 15)
        {
            return chiSquared < 0.08234f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 8 and layer5 == 9)
        {
            return chiSquared < 0.02360f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return chiSquared < 0.07167f;
        }
        else if(layer4 == 13 and layer5 == 14)
        {   
            return chiSquared < 0.08234f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 7 and layer5 == 8)
        {
            return chiSquared < 0.01026f;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return chiSquared < 0.06238f;
        }
        else if(layer4 == 12 and layer5 == 13)
        {
            return chiSquared < 0.06238f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4)
    {
        if(layer5 == 12)
        {
            return chiSquared < 0.09461f;
        }
        else if(layer5 == 5)
        {
            return chiSquared < 0.04725f;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return chiSquared < 0.00512f;
        }
        if(layer4 == 9 and layer5 == 15)
        {
            return chiSquared < 0.04112f;
        }
        else if(layer4 == 14 and layer5 == 15)
        {
            return chiSquared < 0.06238f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        if(layer4 == 8 and layer5 == 14)
        {
            return chiSquared < 0.07167f;
        }
        else if(layer4 == 13 and layer5 == 14)
        {
            return chiSquared < 0.06238f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return chiSquared < 0.10870f;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return chiSquared < 0.10870f;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return chiSquared < 0.08234f;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return chiSquared < 0.09461f;
    }
    else if(layer1 == 3 and layer2 == 4 and layer3 == 5 and layer4 == 12 and layer5 == 13)
    {
        return chiSquared < 0.09461f;
    }

    return true;
}

//bounds can be found at http://uaf-10.t2.ucsd.edu/~bsathian/SDL/T5_RZFix/t5_rz_thresholds.txt
__device__ bool SDL::passT5RZConstraint(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, unsigned int firstMDIndex, unsigned int secondMDIndex, unsigned int thirdMDIndex, unsigned int fourthMDIndex, unsigned int fifthMDIndex, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5) 
{
    const float& rt1 = mdsInGPU.anchorRt[firstMDIndex];
    const float& rt2 = mdsInGPU.anchorRt[secondMDIndex];
    const float& rt3 = mdsInGPU.anchorRt[thirdMDIndex];
    const float& rt4 = mdsInGPU.anchorRt[fourthMDIndex];
    const float& rt5 = mdsInGPU.anchorRt[fifthMDIndex];

    const float& z1 = mdsInGPU.anchorZ[firstMDIndex];
    const float& z2 = mdsInGPU.anchorZ[secondMDIndex];
    const float& z3 = mdsInGPU.anchorZ[thirdMDIndex];
    const float& z4 = mdsInGPU.anchorZ[fourthMDIndex];
    const float& z5 = mdsInGPU.anchorZ[fifthMDIndex];

    //following Philip's layer number prescription
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    //slope computed using the internal T3s
    const int moduleLayer1 = modulesInGPU.moduleType[lowerModuleIndex1];
    const int moduleLayer2 = modulesInGPU.moduleType[lowerModuleIndex2];
    const int moduleLayer3 = modulesInGPU.moduleType[lowerModuleIndex3];
    const int moduleLayer4 = modulesInGPU.moduleType[lowerModuleIndex4];
    const int moduleLayer5 = modulesInGPU.moduleType[lowerModuleIndex5];

    float slope;
    if(moduleLayer1 == 0 and moduleLayer2 == 0 and moduleLayer3 == 1) //PSPS2S
    {
        slope = (z2 -z1)/(rt2 - rt1);
    }
    else
    {
        slope = (z3 - z1)/(rt3 - rt1);
    }
    float residual4 = (layer4 <= 6)? ((z4 - z1) - slope * (rt4 - rt1)) : ((rt4 - rt1) - (z4 - z1)/slope);
    float residual5 = (layer4 <= 6) ? ((z5 - z1) - slope * (rt5 - rt1)) : ((rt5 - rt1) - (z5 - z1)/slope);

    // creating a chi squared type quantity
    // 0-> PS, 1->2S
    residual4 = (moduleLayer4 == 0) ? residual4/2.4f : residual4/5.0f;
    residual5 = (moduleLayer5 == 0) ? residual5/2.4f : residual5/5.0f;

    const float RMSE = sqrtf(0.5 * (residual4 * residual4 + residual5 * residual5));

    //categories!
    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 4 and layer5 == 5)
        {
            return RMSE < 0.545f; 
        }
        else if(layer4 == 4 and layer5 == 12)
        {
            return RMSE < 1.105f;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return RMSE < 0.775f;
        }
        else if(layer4 == 12 and layer5 == 13)
        {
            return RMSE < 0.625f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 8 and layer5 == 14)
        {
            return RMSE < 0.835f;
        }
        else if(layer4 == 13 and layer5 == 14)
        {
            return RMSE < 0.575f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9 and layer5 == 15)
    {
        return RMSE < 0.825f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 5 and layer5 == 6)
        {
            return RMSE < 0.845f;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return RMSE < 1.365f;
        }

        else if(layer4 == 12 and layer5 == 13)
        {
            return RMSE < 0.675f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7 and layer4 == 13 and layer5 == 14)
    {
            return RMSE < 0.495f;
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 12 and layer4 == 13 and layer5 == 14)
    {
        return RMSE < 0.695f; 
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 15)
        {
            return RMSE < 0.735f;
        }
        else if(layer4 == 14 and layer5 == 15)
        {
            return RMSE < 0.525f;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 13 and layer4 == 14 and layer5 == 15)
    {
        return RMSE < 0.665f;
    }
    else if(layer1 == 3 and layer2 == 4 and layer3 == 5 and layer4 == 12 and layer5 == 13)
    {
        return RMSE < 0.995f;
    }
    else if(layer1 == 3 and layer2 == 4 and layer3 == 12 and layer4 == 13 and layer5 == 14)
    {
        return RMSE < 0.525f;
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return RMSE < 0.525f;
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 13 and layer4 == 14 and layer5 == 15)
    {
        return RMSE < 0.745f;
    }
    else if(layer1 == 3 and layer2 == 12 and layer3 == 13 and layer4 == 14 and layer5 == 15)
    {
        return RMSE < 0.555f; 
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16)
    {
            return RMSE < 0.525f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 14 and layer4 == 15 and layer5 == 16)
    {
        return RMSE < 0.885f;
    }
    else if(layer1 == 7 and layer2 == 13 and layer3 == 14 and layer4 == 15 and layer5 == 16)
    {
        return RMSE < 0.845f;
    }

    return true;
}

__device__ bool SDL::checkIntervalOverlap(const float& firstMin, const float& firstMax, const float& secondMin, const float& secondMax)
{
    return ((firstMin <= secondMin) & (secondMin < firstMax)) |  ((secondMin < firstMin) & (firstMin < secondMax));
}

/*bounds for high Pt taken from : http://uaf-10.t2.ucsd.edu/~bsathian/SDL/T5_efficiency/efficiencies/new_efficiencies/efficiencies_20210513_T5_recovering_high_Pt_efficiencies/highE_radius_matching/highE_bounds.txt */

__device__ bool SDL::matchRadiiBBBBB(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound =  0.1512f;
    float bridgeInvRadiusErrorBound = 0.1781f;
    float outerInvRadiusErrorBound = 0.1840f;

    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 0.4449f;
        bridgeInvRadiusErrorBound = 0.4033f;
        outerInvRadiusErrorBound = 0.8016f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax);
}

__device__ bool SDL::matchRadiiBBBBE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{

    float innerInvRadiusErrorBound =  0.1781f;
    float bridgeInvRadiusErrorBound = 0.2167f;
    float outerInvRadiusErrorBound = 1.1116f;

    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 0.4750f;
        bridgeInvRadiusErrorBound = 0.3903f;
        outerInvRadiusErrorBound = 15.2120f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax);
}

__device__ bool SDL::matchRadiiBBBEE12378(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound = 0.178f;
    float bridgeInvRadiusErrorBound = 0.507f;
    float outerInvRadiusErrorBound = 7.655f;

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, fminf(bridgeInvRadiusMin, 1.0f/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0f/bridgeRadiusMin2S));
}

__device__ bool SDL::matchRadiiBBBEE23478(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound = 0.2097f;
    float bridgeInvRadiusErrorBound = 0.8557f;
    float outerInvRadiusErrorBound = 24.0450f;

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, fminf(bridgeInvRadiusMin, 1.0f/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0f/bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBBBEE34578(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound = 0.066f;
    float bridgeInvRadiusErrorBound = 0.617f;
    float outerInvRadiusErrorBound = 2.688f;

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, fminf(bridgeInvRadiusMin, 1.0f/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0f/bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBBBEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{

    float innerInvRadiusErrorBound =  0.1840f;
    float bridgeInvRadiusErrorBound = 0.5971f;
    float outerInvRadiusErrorBound = 11.7102f;

    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf)) //as good as no selections
    {
        innerInvRadiusErrorBound = 1.0412f;
        outerInvRadiusErrorBound = 32.2737f;
        bridgeInvRadiusErrorBound = 10.9688f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, fminf(bridgeInvRadiusMin, 1.0f/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0f/bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBBEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound =  0.6376f;
    float bridgeInvRadiusErrorBound = 2.1381f;
    float outerInvRadiusErrorBound = 20.4179f;

    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf)) //as good as no selections!
    {
        innerInvRadiusErrorBound = 12.9173f;
        outerInvRadiusErrorBound = 25.6702f;
        bridgeInvRadiusErrorBound = 5.1700f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, fminf(bridgeInvRadiusMin, 1.0f/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0f/bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBEEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{

    float innerInvRadiusErrorBound =  1.9382f;
    float bridgeInvRadiusErrorBound = 3.7280f;
    float outerInvRadiusErrorBound = 5.7030f;


    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 23.2713f;
        outerInvRadiusErrorBound = 24.0450f;
        bridgeInvRadiusErrorBound = 21.7980f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(fminf(innerInvRadiusMin, 1.0/innerRadiusMax2S), fmaxf(innerInvRadiusMax, 1.0/innerRadiusMin2S), fminf(bridgeInvRadiusMin, 1.0/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0/bridgeRadiusMin2S));
}

__device__ bool SDL::matchRadiiEEEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound =  1.9382f;
    float bridgeInvRadiusErrorBound = 2.2091f;
    float outerInvRadiusErrorBound = 7.4084f;

    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 22.5226f;
        bridgeInvRadiusErrorBound = 21.0966f;
        outerInvRadiusErrorBound = 19.1252f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(fminf(innerInvRadiusMin, 1.0/innerRadiusMax2S), fmaxf(innerInvRadiusMax, 1.0/innerRadiusMin2S), fminf(bridgeInvRadiusMin, 1.0/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0/bridgeRadiusMin2S));
}

__device__ void SDL::computeErrorInRadius(float* x1Vec, float* y1Vec, float* x2Vec, float* y2Vec, float* x3Vec, float* y3Vec, float& minimumRadius, float& maximumRadius)
{
    //brute force
    float candidateRadius;
    float g, f;
    minimumRadius = 123456789.f;
    maximumRadius = 0.f;
    for(size_t i = 0; i < 3; i++)
    {
        float x1 = x1Vec[i];
	float y1 = y1Vec[i];
        for(size_t j = 0; j < 3; j++)
        {
	    float x2 = x2Vec[j];
	    float y2 = y2Vec[j];
            for(size_t k = 0; k < 3; k++)
            {
	       float x3 = x3Vec[k];
               float y3 = y3Vec[k];
               candidateRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3, g, f);
               maximumRadius = fmaxf(candidateRadius, maximumRadius);
               minimumRadius = fminf(candidateRadius, minimumRadius);
            }
        }
    }
}
__device__ float SDL::computeRadiusFromThreeAnchorHits(float x1, float y1, float x2, float y2, float x3, float y3, float& g, float& f)
{
    float radius = 0.f;

    //writing manual code for computing radius, which obviously sucks
    //TODO:Use fancy inbuilt libraries like cuBLAS or cuSOLVE for this!
    //(g,f) -> center
    //first anchor hit - (x1,y1), second anchor hit - (x2,y2), third anchor hit - (x3, y3)

    /*
    if((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0)
    {
        return -1; //WTF man three collinear points!
    }
    */

    float denomInv = 1.0f/((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3));

    float xy1sqr = x1 * x1 + y1 * y1;

    float xy2sqr = x2 * x2 + y2 * y2;

    float xy3sqr = x3 * x3 + y3 * y3;

    g = 0.5f * ((y3 - y2) * xy1sqr + (y1 - y3) * xy2sqr + (y2 - y1) * xy3sqr) * denomInv;

    f = 0.5f * ((x2 - x3) * xy1sqr + (x3 - x1) * xy2sqr + (x1 - x2) * xy3sqr) * denomInv;

    float c = ((x2 * y3 - x3 * y2) * xy1sqr + (x3 * y1 - x1 * y3) * xy2sqr + (x1 * y2 - x2 * y1) * xy3sqr) * denomInv;

    if(((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0) || (g * g + f * f - c < 0))
    {
        printf("three collinear points or FATAL! r^2 < 0!\n");
	radius = -1.f;
    }
    else
      radius = sqrtf(g * g  + f * f - c);

    return radius;
}

__device__ bool SDL::T5HasCommonMiniDoublet(struct SDL::triplets& tripletsInGPU, struct SDL::triplets& tripletsInwardInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerTripletInwardIndex, unsigned int outerTripletIndex)
{
    unsigned int innerOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletInwardIndex + 1];
    unsigned int outerInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int innerOuterOuterMiniDoubletIndex = segmentsInGPU.mdIndices[2 * innerOuterSegmentIndex + 1]; //inner triplet outer segment outer MD index
    unsigned int outerInnerInnerMiniDoubletIndex = segmentsInGPU.mdIndices[2 * outerInnerSegmentIndex]; //outer triplet inner segmnet inner MD index


    return (innerOuterOuterMiniDoubletIndex == outerInnerInnerMiniDoubletIndex);
}

__device__ void SDL::computeSigmasForRegression(SDL::modules& modulesInGPU, const uint16_t* lowerModuleIndices, float* delta1, float* delta2, float* slopes, bool* isFlat, int nPoints, bool anchorHits) 
{
   /*bool anchorHits required to deal with a weird edge case wherein 
     the hits ultimately used in the regression are anchor hits, but the
     lower modules need not all be Pixel Modules (in case of PS). Similarly,
     when we compute the chi squared for the non-anchor hits, the "partner module"
     need not always be a PS strip module, but all non-anchor hits sit on strip 
     modules.
    */
    ModuleType moduleType;
    short moduleSubdet, moduleSide;
    float inv1 = 0.01f/0.009f;
    float inv2 = 0.15f/0.009f;
    float inv3 = 2.4f/0.009f;
    for(size_t i=0; i<nPoints; i++)
    {
        moduleType = modulesInGPU.moduleType[lowerModuleIndices[i]];
        moduleSubdet = modulesInGPU.subdets[lowerModuleIndices[i]];
        moduleSide = modulesInGPU.sides[lowerModuleIndices[i]];
        float& drdz = modulesInGPU.drdzs[lowerModuleIndices[i]];
        slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]]; 
        //category 1 - barrel PS flat
        if(moduleSubdet == Barrel and moduleType == PS and moduleSide == Center)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            delta2[i] = inv1;//1.1111f;//0.01;
            slopes[i] = -999.f;
            isFlat[i] = true;
        }

        //category 2 - barrel 2S
        else if(moduleSubdet == Barrel and moduleType == TwoS)
        {
            delta1[i] = 1.f;//0.009;
            delta2[i] = 1.f;//0.009;
            slopes[i] = -999.f;
            isFlat[i] = true;
        }

        //category 3 - barrel PS tilted
        else if(moduleSubdet == Barrel and moduleType == PS and moduleSide != Center)
        {

            //delta1[i] = 0.01;
            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            if(anchorHits)
            {
                delta2[i] = (inv2 * drdz/sqrtf(1 + drdz * drdz));
            }
            else
            {
                delta2[i] = (inv3 * drdz/sqrtf(1 + drdz * drdz));
            }
        }
        //category 4 - endcap PS
        else if(moduleSubdet == Endcap and moduleType == PS)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            /*despite the type of the module layer of the lower module index,
            all anchor hits are on the pixel side and all non-anchor hits are
            on the strip side!*/
            if(anchorHits)
            {
                delta2[i] = inv2;//16.6666f;//0.15f;
            }
            else
            {
                delta2[i] = inv3;//266.666f;//2.4f;
            }
        }

        //category 5 - endcap 2S
        else if(moduleSubdet == Endcap and moduleType == TwoS)
        {
            delta1[i] = 1.f;//0.009;
            delta2[i] = 500.f*inv1;//555.5555f;//5.f;
            isFlat[i] = false;
        }
        else
        {
            printf("ERROR!!!!! I SHOULDN'T BE HERE!!!! subdet = %d, type = %d, side = %d\n", moduleSubdet, moduleType, moduleSide);
        }
    }
}

__device__ float SDL::computeRadiusUsingRegression(int nPoints, float* xs, float* ys, float* delta1, float* delta2, float* slopes, bool* isFlat, float& g, float& f, float* sigmas, float& chiSquared)
{
    float radius = 0.f;

    //some extra variables
    //the two variables will be caled x1 and x2, and y (which is x^2 + y^2)

    float sigmaX1Squared = 0.f;
    float sigmaX2Squared = 0.f;
    float sigmaX1X2 = 0.f; 
    float sigmaX1y = 0.f; 
    float sigmaX2y = 0.f;
    float sigmaY = 0.f;
    float sigmaX1 = 0.f;
    float sigmaX2 = 0.f;
    float sigmaOne = 0.f;

    float xPrime, yPrime, absArctanSlope, angleM;
    for(size_t i = 0; i < nPoints; i++)
    {
        //computing sigmas is a very tricky affair
        //if the module is tilted or endcap, we need to use the slopes properly!

        absArctanSlope = ((slopes[i] != 123456789) ? fabs(atanf(slopes[i])) : 0.5f*float(M_PI)); // Since C++ can't represent infinity, SDL_INF = 123456789 was used to represent infinity in the data table

        if(xs[i] > 0 and ys[i] > 0)
        {
            angleM = 0.5f*float(M_PI) - absArctanSlope;
        }
        else if(xs[i] < 0 and ys[i] > 0)
        {
            angleM = absArctanSlope + 0.5f*float(M_PI);
        }
        else if(xs[i] < 0 and ys[i] < 0)
        {
            angleM = -(absArctanSlope + 0.5f*float(M_PI));
        }
        else if(xs[i] > 0 and ys[i] < 0)
        {
            angleM = -(0.5f*float(M_PI) - absArctanSlope);
        }

        if(not isFlat[i])
        {
            xPrime = xs[i] * cosf(angleM) + ys[i] * sinf(angleM);
            yPrime = ys[i] * cosf(angleM) - xs[i] * sinf(angleM);
        }
        else
        {
            xPrime = xs[i];
            yPrime = ys[i];
        }
        sigmas[i] = 2 * sqrtf((xPrime * delta1[i]) * (xPrime * delta1[i]) + (yPrime * delta2[i]) * (yPrime * delta2[i]));

        sigmaX1Squared += (xs[i] * xs[i])/(sigmas[i] * sigmas[i]);
        sigmaX2Squared += (ys[i] * ys[i])/(sigmas[i] * sigmas[i]);
        sigmaX1X2 += (xs[i] * ys[i])/(sigmas[i] * sigmas[i]);
        sigmaX1y += (xs[i] * (xs[i] * xs[i] + ys[i] * ys[i]))/(sigmas[i] * sigmas[i]);
        sigmaX2y += (ys[i] * (xs[i] * xs[i] + ys[i] * ys[i]))/(sigmas[i] * sigmas[i]);
        sigmaY += (xs[i] * xs[i] + ys[i] * ys[i])/(sigmas[i] * sigmas[i]);
        sigmaX1 += xs[i]/(sigmas[i] * sigmas[i]);
        sigmaX2 += ys[i]/(sigmas[i] * sigmas[i]);
        sigmaOne += 1.0f/(sigmas[i] * sigmas[i]);
    }
    float denominator = (sigmaX1X2 - sigmaX1 * sigmaX2) * (sigmaX1X2 - sigmaX1 * sigmaX2) - (sigmaX1Squared - sigmaX1 * sigmaX1) * (sigmaX2Squared - sigmaX2 * sigmaX2);

    float twoG = ((sigmaX2y - sigmaX2 * sigmaY) * (sigmaX1X2 - sigmaX1 * sigmaX2) - (sigmaX1y - sigmaX1 * sigmaY) * (sigmaX2Squared - sigmaX2 * sigmaX2)) / denominator;
    float twoF = ((sigmaX1y - sigmaX1 * sigmaY) * (sigmaX1X2 - sigmaX1 * sigmaX2) - (sigmaX2y - sigmaX2 * sigmaY) * (sigmaX1Squared - sigmaX1 * sigmaX1)) / denominator;

    float c = -(sigmaY - twoG * sigmaX1 - twoF * sigmaX2)/sigmaOne;
    g = 0.5f*twoG;
    f = 0.5f*twoF;
    if(g * g + f * f - c < 0)
    {
        printf("FATAL! r^2 < 0!\n");
        return -1;
    }
    
    radius = sqrtf(g * g  + f * f - c);
    //compute chi squared
    chiSquared = 0.f;
    for(size_t i = 0; i < nPoints; i++)
    {
       chiSquared +=  (xs[i] * xs[i] + ys[i] * ys[i] - twoG * xs[i] - twoF * ys[i] + c) * (xs[i] * xs[i] + ys[i] * ys[i] - twoG * xs[i] - twoF * ys[i] + c) / (sigmas[i] * sigmas[i]);
    }
    return radius;
}

__device__ float SDL::computeChiSquared(int nPoints, float* xs, float* ys, float* delta1, float* delta2, float* slopes, bool* isFlat, float g, float f, float radius)
{
    // given values of (g, f, radius) and a set of points (and its uncertainties)
    //compute chi squared
    float c = g*g + f*f - radius*radius;
    float chiSquared = 0.f;
    float absArctanSlope, angleM, xPrime, yPrime, sigma;
    for(size_t i = 0; i < nPoints; i++)
    {
        absArctanSlope = ((slopes[i] != 123456789) ? fabs(atanf(slopes[i])) : 0.5f*float(M_PI)); // Since C++ can't represent infinity, SDL_INF = 123456789 was used to represent infinity in the data table
        if(xs[i] > 0 and ys[i] > 0)
        {
            angleM = 0.5f*float(M_PI) - absArctanSlope;
        }
        else if(xs[i] < 0 and ys[i] > 0)
        {
            angleM = absArctanSlope + 0.5f*float(M_PI);
        }
        else if(xs[i] < 0 and ys[i] < 0)
        {
            angleM = -(absArctanSlope + 0.5f*float(M_PI));
        }
        else if(xs[i] > 0 and ys[i] < 0)
        {
            angleM = -(0.5f*float(M_PI) - absArctanSlope);
        }

        if(not isFlat[i])
        {
            xPrime = xs[i] * cosf(angleM) + ys[i] * sinf(angleM);
            yPrime = ys[i] * cosf(angleM) - xs[i] * sinf(angleM);
        }
        else
        {
            xPrime = xs[i];
            yPrime = ys[i];
        }
        sigma = 2 * sqrtf((xPrime * delta1[i]) * (xPrime * delta1[i]) + (yPrime * delta2[i]) * (yPrime * delta2[i]));
        chiSquared +=  (xs[i] * xs[i] + ys[i] * ys[i] - 2 * g * xs[i] - 2 * f * ys[i] + c) * (xs[i] * xs[i] + ys[i] * ys[i] - 2 * g * xs[i] - 2 * f * ys[i] + c) / (sigma * sigma);
    }
    return chiSquared; 
}
