#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif
# include "Quintuplet.cuh"
#include "allocate.h"
#include "Kernels.cuh"

SDL::quintuplets::quintuplets()
{
    tripletIndices = nullptr;
    lowerModuleIndices = nullptr;
    nQuintuplets = nullptr;
    totOccupancyQuintuplets = nullptr;
    innerRadius = nullptr;
    outerRadius = nullptr;
    regressionRadius = nullptr;
    isDup = nullptr;
    TightCutFlag = nullptr;
    partOfPT5 = nullptr;
    pt = nullptr;
    layer = nullptr;
    regressionG = nullptr;
    regressionF = nullptr;
    logicalLayers = nullptr;
    hitIndices = nullptr;
    bridgeRadius = nullptr;
    chiSquared = nullptr;
    rzChiSquared = nullptr;
    nonAnchorChiSquared = nullptr;
}

SDL::quintuplets::~quintuplets()
{
}

void SDL::quintuplets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,tripletIndices);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, nQuintuplets);
    cms::cuda::free_device(dev, totOccupancyQuintuplets);
    cms::cuda::free_device(dev, innerRadius);
    cms::cuda::free_device(dev, outerRadius);
    cms::cuda::free_device(dev, partOfPT5);
    cms::cuda::free_device(dev, isDup);
    cms::cuda::free_device(dev, TightCutFlag);
    cms::cuda::free_device(dev, pt);
    cms::cuda::free_device(dev, layer);
    cms::cuda::free_device(dev, regressionG);
    cms::cuda::free_device(dev, regressionF);
    cms::cuda::free_device(dev, regressionRadius);
    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, nMemoryLocations);
    cms::cuda::free_device(dev, bridgeRadius);
    cms::cuda::free_device(dev, rzChiSquared);
    cms::cuda::free_device(dev, chiSquared);
    cms::cuda::free_device(dev, nonAnchorChiSquared);
}

void SDL::quintuplets::freeMemory(hipStream_t stream)
{
    hipFree(tripletIndices);
    hipFree(lowerModuleIndices);
    hipFree(nQuintuplets);
    hipFree(totOccupancyQuintuplets);
    hipFree(innerRadius);
    hipFree(outerRadius);
    hipFree(regressionRadius);
    hipFree(partOfPT5);
    hipFree(isDup);
    hipFree(TightCutFlag);
    hipFree(pt);
    hipFree(layer);
    hipFree(regressionG);
    hipFree(regressionF);
    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(nMemoryLocations);
    hipFree(bridgeRadius);
    hipFree(rzChiSquared);
    hipFree(chiSquared);
    hipFree(nonAnchorChiSquared);
    hipStreamSynchronize(stream);
}
//TODO:Reuse the track candidate one instead of this!
__global__ void SDL::createEligibleModulesListForQuintupletsGPU(struct modules& modulesInGPU,struct triplets& tripletsInGPU, unsigned int* device_nTotalQuintuplets, hipStream_t stream,struct objectRanges& rangesInGPU)
{
    __shared__ int nEligibleT5Modulesx;
    __shared__ unsigned int nTotalQuintupletsx;
    nTotalQuintupletsx = 0; //start!
    nEligibleT5Modulesx = 0;
    __syncthreads();

    unsigned int occupancy;
    unsigned int category_number, eta_number;
    unsigned int layers, subdets, rings;
    float eta;
    //start filling
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int np = gridDim.x * blockDim.x;
    for(uint16_t i = gid; i < *modulesInGPU.nLowerModules; i+= np)
    {
        //condition for a quintuple to exist for a module
        //TCs don't exist for layers 5 and 6 barrel, and layers 2,3,4,5 endcap   
        layers = modulesInGPU.layers[i];
        subdets = modulesInGPU.subdets[i];
        rings = modulesInGPU.rings[i];
        eta = modulesInGPU.eta[i];  
        occupancy = 0;

        if (tripletsInGPU.nTriplets[i] == 0) continue;
        if (subdets == SDL::Barrel and layers >= 3) continue;
        if (subdets == SDL::Endcap and layers > 1) continue;

        int nEligibleT5Modules = atomicAdd(&nEligibleT5Modulesx,1);
        if (nEligibleT5Modules < 0) printf("%u\n",nEligibleT5Modules);
        if (layers<=3 && subdets==5) category_number = 0;
        if (layers>=4 && subdets==5) category_number = 1;
        if (layers<=2 && subdets==4 && rings>=11) category_number = 2;
        if (layers>=3 && subdets==4 && rings>=8) category_number = 2;
        if (layers<=2 && subdets==4 && rings<=10) category_number = 3;
        if (layers>=3 && subdets==4 && rings<=7) category_number = 3;
        if (abs(eta)<0.75) eta_number=0;
        if (abs(eta)>0.75 && abs(eta)<1.5) eta_number=1;
        if (abs(eta)>1.5 && abs(eta)<2.25) eta_number=2;
        if (abs(eta)>2.25 && abs(eta)<3) eta_number=3;

        if (category_number == 0 && eta_number == 0) occupancy = 336;
        if (category_number == 0 && eta_number == 1) occupancy = 414;
        if (category_number == 0 && eta_number == 2) occupancy = 231;
        if (category_number == 0 && eta_number == 3) occupancy = 146;
        if (category_number == 3 && eta_number == 1) occupancy = 0;
        if (category_number == 3 && eta_number == 2) occupancy = 191;
        if (category_number == 3 && eta_number == 3) occupancy = 106;

        unsigned int nTotQ = atomicAdd(&nTotalQuintupletsx,occupancy);
        rangesInGPU.quintupletModuleIndices[i] = nTotQ;
        rangesInGPU.indicesOfEligibleT5Modules[nEligibleT5Modules] = i;
    }
    __syncthreads();
    if(threadIdx.x==0){
        *rangesInGPU.nEligibleT5Modules = static_cast<uint16_t>(nEligibleT5Modulesx);
        *device_nTotalQuintuplets = nTotalQuintupletsx;
    }
}

void SDL::createQuintupletsInExplicitMemory(struct SDL::quintuplets& quintupletsInGPU, const unsigned int& nTotalQuintuplets, const uint16_t& nLowerModules, const uint16_t& nEligibleModules,hipStream_t stream)
{
    //unsigned int nMemoryLocations = nEligibleModules * maxQuintuplets;
#ifdef CACHE_ALLOC
 //   hipStream_t stream = 0;
    int dev;
    hipGetDevice(&dev);
    quintupletsInGPU.tripletIndices = (unsigned int*)cms::cuda::allocate_device(dev, 2 * nTotalQuintuplets * sizeof(unsigned int), stream);
    quintupletsInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev, 5 * nTotalQuintuplets * sizeof(uint16_t), stream);
    quintupletsInGPU.nQuintuplets = (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int), stream);
    quintupletsInGPU.totOccupancyQuintuplets = (unsigned int*)cms::cuda::allocate_device(dev, nLowerModules * sizeof(unsigned int), stream);
    quintupletsInGPU.innerRadius = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(FPX), stream);
    quintupletsInGPU.outerRadius = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(FPX), stream);
    quintupletsInGPU.bridgeRadius = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);

    quintupletsInGPU.pt = (FPX*)cms::cuda::allocate_device(dev, nTotalQuintuplets *4* sizeof(FPX), stream);
    quintupletsInGPU.layer = (uint8_t*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(uint8_t), stream);
    quintupletsInGPU.isDup = (bool*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(bool), stream);
    quintupletsInGPU.TightCutFlag = (bool*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(bool), stream);
    quintupletsInGPU.partOfPT5 = (bool*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(bool), stream);
    quintupletsInGPU.regressionRadius = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.regressionG = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.regressionF = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(uint8_t) * 5, stream);
    quintupletsInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(unsigned int) * 10, stream);
    quintupletsInGPU.nMemoryLocations = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);

    quintupletsInGPU.rzChiSquared = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.chiSquared = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
    quintupletsInGPU.nonAnchorChiSquared = (float*)cms::cuda::allocate_device(dev, nTotalQuintuplets * sizeof(float), stream);
#else
    hipMalloc(&quintupletsInGPU.tripletIndices, 2 * nTotalQuintuplets * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.lowerModuleIndices, 5 * nTotalQuintuplets * sizeof(uint16_t));
    hipMalloc(&quintupletsInGPU.nQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.totOccupancyQuintuplets, nLowerModules * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.innerRadius, nTotalQuintuplets * sizeof(FPX));
    hipMalloc(&quintupletsInGPU.outerRadius, nTotalQuintuplets * sizeof(FPX));
    hipMalloc(&quintupletsInGPU.pt, nTotalQuintuplets *4* sizeof(FPX));
    hipMalloc(&quintupletsInGPU.isDup, nTotalQuintuplets * sizeof(bool));
    hipMalloc(&quintupletsInGPU.TightCutFlag, nTotalQuintuplets * sizeof(bool));
    hipMalloc(&quintupletsInGPU.partOfPT5, nTotalQuintuplets * sizeof(bool));
    hipMalloc(&quintupletsInGPU.layer, nTotalQuintuplets * sizeof(uint8_t));
    hipMalloc(&quintupletsInGPU.regressionRadius, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.regressionG, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.regressionF, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.logicalLayers, nTotalQuintuplets * 5 * sizeof(uint8_t));
    hipMalloc(&quintupletsInGPU.hitIndices, nTotalQuintuplets * 10 * sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.nMemoryLocations, sizeof(unsigned int));
    hipMalloc(&quintupletsInGPU.bridgeRadius, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.rzChiSquared, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.chiSquared, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.nonAnchorChiSquared, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.rzChiSquared, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.residual_missing, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.residual4, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.residual5, nTotalQuintuplets * sizeof(float));
    hipMalloc(&quintupletsInGPU.nMemoryLocations, sizeof(unsigned int));
#endif
    hipMemsetAsync(quintupletsInGPU.nQuintuplets,0,nLowerModules * sizeof(unsigned int),stream);
    hipMemsetAsync(quintupletsInGPU.totOccupancyQuintuplets,0,nLowerModules * sizeof(unsigned int),stream);
    hipMemsetAsync(quintupletsInGPU.isDup,0,nTotalQuintuplets * sizeof(bool),stream);
    hipMemsetAsync(quintupletsInGPU.TightCutFlag,0,nTotalQuintuplets * sizeof(bool),stream);
    hipMemsetAsync(quintupletsInGPU.partOfPT5,0,nTotalQuintuplets * sizeof(bool),stream);
    hipStreamSynchronize(stream);
    quintupletsInGPU.eta = quintupletsInGPU.pt + nTotalQuintuplets;
    quintupletsInGPU.phi = quintupletsInGPU.pt + 2*nTotalQuintuplets;
    quintupletsInGPU.score_rphisum = quintupletsInGPU.pt + 3*nTotalQuintuplets;
}


__device__ void SDL::addQuintupletToMemory(struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex, uint16_t& lowerModule1, uint16_t& lowerModule2, uint16_t& lowerModule3, uint16_t& lowerModule4, uint16_t& lowerModule5, float& innerRadius, float& bridgeRadius, float& outerRadius, float& regressionG, float& regressionF, float& regressionRadius, float& rzChiSquared, float& rPhiChiSquared, float&
        nonAnchorChiSquared, float pt, float eta, float phi, float scores, uint8_t layer, unsigned int quintupletIndex, bool TightCutFlag)

{
    quintupletsInGPU.tripletIndices[2 * quintupletIndex] = innerTripletIndex;
    quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1] = outerTripletIndex;

    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex] = lowerModule1;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 1] = lowerModule2;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 2] = lowerModule3;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 3] = lowerModule4;
    quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 4] = lowerModule5;
    quintupletsInGPU.innerRadius[quintupletIndex] = __F2H(innerRadius);
    quintupletsInGPU.outerRadius[quintupletIndex] = __F2H(outerRadius);
    quintupletsInGPU.pt[quintupletIndex] = __F2H(pt);
    quintupletsInGPU.eta[quintupletIndex] = __F2H(eta);
    quintupletsInGPU.phi[quintupletIndex] = __F2H(phi);
    quintupletsInGPU.score_rphisum[quintupletIndex] = __F2H(scores);
    quintupletsInGPU.layer[quintupletIndex] = layer;
    quintupletsInGPU.isDup[quintupletIndex] = false;
    quintupletsInGPU.TightCutFlag[quintupletIndex] = TightCutFlag;
    quintupletsInGPU.regressionRadius[quintupletIndex] = regressionRadius;
    quintupletsInGPU.regressionG[quintupletIndex] = regressionG;
    quintupletsInGPU.regressionF[quintupletIndex] = regressionF;
    quintupletsInGPU.logicalLayers[5 * quintupletIndex] = tripletsInGPU.logicalLayers[3 * innerTripletIndex];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 1] = tripletsInGPU.logicalLayers[3 * innerTripletIndex + 1];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 2] = tripletsInGPU.logicalLayers[3 * innerTripletIndex + 2];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 3] = tripletsInGPU.logicalLayers[3 * outerTripletIndex + 1];
    quintupletsInGPU.logicalLayers[5 * quintupletIndex + 4] = tripletsInGPU.logicalLayers[3 * outerTripletIndex + 2];

    quintupletsInGPU.hitIndices[10 * quintupletIndex] = tripletsInGPU.hitIndices[6 * innerTripletIndex];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 1] = tripletsInGPU.hitIndices[6 * innerTripletIndex + 1];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 2] = tripletsInGPU.hitIndices[6 * innerTripletIndex + 2];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 3] = tripletsInGPU.hitIndices[6 * innerTripletIndex + 3];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 4] = tripletsInGPU.hitIndices[6 * innerTripletIndex + 4];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 5] = tripletsInGPU.hitIndices[6 * innerTripletIndex + 5];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 6] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 2];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 7] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 3];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 8] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 4];
    quintupletsInGPU.hitIndices[10 * quintupletIndex + 9] = tripletsInGPU.hitIndices[6 * outerTripletIndex + 5];
    quintupletsInGPU.bridgeRadius[quintupletIndex] = bridgeRadius;
    quintupletsInGPU.rzChiSquared[quintupletIndex] = rzChiSquared;
    quintupletsInGPU.chiSquared[quintupletIndex] = rPhiChiSquared;
    quintupletsInGPU.nonAnchorChiSquared[quintupletIndex] = nonAnchorChiSquared;

}

__device__ bool SDL::runQuintupletDefaultAlgo(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, unsigned int& innerTripletIndex, unsigned int& outerTripletIndex, float& innerRadius, float& outerRadius, float& bridgeRadius, float& regressionG, float& regressionF, float& regressionRadius, float& rzChiSquared, float& chiSquared, float& nonAnchorChiSquared, bool& TightCutFlag)
{
    bool pass = true;
    unsigned int firstSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex];
    unsigned int secondSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
    unsigned int thirdSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int fourthSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex + 1];

    unsigned int innerOuterOuterMiniDoubletIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex + 1]; //inner triplet outer segment outer MD index
    unsigned int outerInnerInnerMiniDoubletIndex = segmentsInGPU.mdIndices[2 * thirdSegmentIndex]; //outer triplet inner segmnet inner MD index

    //this cut reduces the number of candidates by a factor of 3, i.e., 2 out of 3 warps can end right here!
    if (innerOuterOuterMiniDoubletIndex != outerInnerInnerMiniDoubletIndex) return false;
    
    //apply T4 criteria between segments 1 and 3
    float zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta; //temp stuff
    float zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ;

    unsigned int firstMDIndex = segmentsInGPU.mdIndices[2 * firstSegmentIndex];
    unsigned int secondMDIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex];
    unsigned int thirdMDIndex = segmentsInGPU.mdIndices[2 * secondSegmentIndex + 1];
    unsigned int fourthMDIndex = segmentsInGPU.mdIndices[2 * thirdSegmentIndex + 1];
    unsigned int fifthMDIndex = segmentsInGPU.mdIndices[2 * fourthSegmentIndex + 1];

    pass = pass and runQuintupletDefaultAlgo(modulesInGPU, mdsInGPU, segmentsInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, firstSegmentIndex, thirdSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    if(not pass) return pass;

    pass = pass and runQuintupletDefaultAlgo(modulesInGPU, mdsInGPU, segmentsInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex4, lowerModuleIndex5, firstSegmentIndex, fourthSegmentIndex, firstMDIndex, secondMDIndex, fourthMDIndex, fifthMDIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    if(not pass) return pass;

    pass = pass and runQuintupletDefaultAlgo(modulesInGPU, mdsInGPU, segmentsInGPU, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex3, lowerModuleIndex4, secondSegmentIndex, thirdSegmentIndex, secondMDIndex, thirdMDIndex, thirdMDIndex, fourthMDIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    if(not pass) return pass;
    
    pass = pass and runQuintupletDefaultAlgo(modulesInGPU, mdsInGPU, segmentsInGPU, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, secondSegmentIndex, fourthSegmentIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, fifthMDIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, pt_beta, zLo, zHi, rtLo, rtHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    if(not pass) return pass;

    float x1 = mdsInGPU.anchorX[firstMDIndex];
    float x2 = mdsInGPU.anchorX[secondMDIndex];
    float x3 = mdsInGPU.anchorX[thirdMDIndex];
    float x4 = mdsInGPU.anchorX[fourthMDIndex];
    float x5 = mdsInGPU.anchorX[fifthMDIndex];
    
    float y1 = mdsInGPU.anchorY[firstMDIndex];
    float y2 = mdsInGPU.anchorY[secondMDIndex];
    float y3 = mdsInGPU.anchorY[thirdMDIndex];
    float y4 = mdsInGPU.anchorY[fourthMDIndex];
    float y5 = mdsInGPU.anchorY[fifthMDIndex];

    //construct the arrays
    float x1Vec[] = {x1, x1, x1};
    float y1Vec[] = {y1, y1, y1};
    float x2Vec[] = {x2, x2, x2};
    float y2Vec[] = {y2, y2, y2};
    float x3Vec[] = {x3, x3, x3};
    float y3Vec[] = {y3, y3, y3};

    if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS)
    {
        x1Vec[1] = mdsInGPU.anchorLowEdgeX[firstMDIndex];
        x1Vec[2] = mdsInGPU.anchorHighEdgeX[firstMDIndex];

        y1Vec[1] = mdsInGPU.anchorLowEdgeY[firstMDIndex];
        y1Vec[2] = mdsInGPU.anchorHighEdgeY[firstMDIndex];
    }
    if(modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS)
    {
        x2Vec[1] = mdsInGPU.anchorLowEdgeX[secondMDIndex];
        x2Vec[2] = mdsInGPU.anchorHighEdgeX[secondMDIndex];

        y2Vec[1] = mdsInGPU.anchorLowEdgeY[secondMDIndex];
        y2Vec[2] = mdsInGPU.anchorHighEdgeY[secondMDIndex];
    }
    if(modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS)
    {
        x3Vec[1] = mdsInGPU.anchorLowEdgeX[thirdMDIndex];
        x3Vec[2] = mdsInGPU.anchorHighEdgeX[thirdMDIndex];

        y3Vec[1] = mdsInGPU.anchorLowEdgeY[thirdMDIndex];
        y3Vec[2] = mdsInGPU.anchorHighEdgeY[thirdMDIndex];
    }

    float innerRadiusMin2S, innerRadiusMax2S;
    computeErrorInRadius(x1Vec, y1Vec, x2Vec, y2Vec, x3Vec, y3Vec, innerRadiusMin2S, innerRadiusMax2S);

    for (int i=0; i<3; i++) 
    {
      x1Vec[i] = x4;
      y1Vec[i] = y4;
    }
    if(modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS)
    {
        x1Vec[1] = mdsInGPU.anchorLowEdgeX[fourthMDIndex];
        x1Vec[2] = mdsInGPU.anchorHighEdgeX[fourthMDIndex];

        y1Vec[1] = mdsInGPU.anchorLowEdgeY[fourthMDIndex];
        y1Vec[2] = mdsInGPU.anchorHighEdgeY[fourthMDIndex];
    }

    float bridgeRadiusMin2S, bridgeRadiusMax2S;
    computeErrorInRadius(x2Vec, y2Vec, x3Vec, y3Vec, x1Vec, y1Vec, bridgeRadiusMin2S, bridgeRadiusMax2S);

    for(int i=0; i<3; i++) 
    {
      x2Vec[i] = x5;
      y2Vec[i] = y5;
    }
    if(modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS)
    {
        x2Vec[1] = mdsInGPU.anchorLowEdgeX[fifthMDIndex];
        x2Vec[2] = mdsInGPU.anchorHighEdgeX[fifthMDIndex];

        y2Vec[1] = mdsInGPU.anchorLowEdgeY[fifthMDIndex];
        y2Vec[2] = mdsInGPU.anchorHighEdgeY[fifthMDIndex];
    }

    float outerRadiusMin2S, outerRadiusMax2S;
    computeErrorInRadius(x3Vec, y3Vec, x1Vec, y1Vec, x2Vec, y2Vec, outerRadiusMin2S, outerRadiusMax2S);

    float g, f;
    outerRadius = computeRadiusFromThreeAnchorHits(x3, y3, x4, y4, x5, y5, g, f);
    bridgeRadius = computeRadiusFromThreeAnchorHits(x2, y2, x3, y3, x4, y4, g, f);
    innerRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3, g, f);

    float inner_pt = 2 * k2Rinv1GeVf * innerRadius;
//    pass = pass and passT5RZConstraint(modulesInGPU, mdsInGPU, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, fifthMDIndex, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rzChiSquared, residual_missing, residual4, residual5, inner_pt, innerRadius, g, f, TightCutFlag);
    passT5RZConstraint(modulesInGPU, mdsInGPU, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, fifthMDIndex, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rzChiSquared, residual_missing, residual4, residual5, inner_pt, innerRadius, g, f, TightCutFlag);

    if(not pass) return pass;

    pass = pass & (innerRadius >= 0.95f * ptCut/(2.f * k2Rinv1GeVf));

    float innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax;

    //split by category
    bool tempPass;
    if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Barrel)
    {
       tempPass = matchRadiiBBBBB(innerRadius, bridgeRadius, outerRadius, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBBBBE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        if(modulesInGPU.layers[lowerModuleIndex1] == 1)
        {
            tempPass = matchRadiiBBBEE12378(innerRadius, bridgeRadius, outerRadius,innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
        }
        else if(modulesInGPU.layers[lowerModuleIndex1] == 2)
        {
            tempPass = matchRadiiBBBEE23478(innerRadius, bridgeRadius, outerRadius,innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
        }
        else
        {
            tempPass = matchRadiiBBBEE34578(innerRadius, bridgeRadius, outerRadius,innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
        }
    }

    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBBEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
    }
    else if(modulesInGPU.subdets[lowerModuleIndex1] == SDL::Barrel and modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap)
    {
        tempPass = matchRadiiBEEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S, innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
    }
    else
    {
        tempPass = matchRadiiEEEEE(innerRadius, bridgeRadius, outerRadius, innerRadiusMin2S, innerRadiusMax2S, bridgeRadiusMin2S, bridgeRadiusMax2S, outerRadiusMin2S, outerRadiusMax2S,innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax, outerInvRadiusMin, outerInvRadiusMax);
    }

    //compute regression radius right here - this computation is expensive!!!
//    pass = pass and tempPass;
    if(not pass) return pass;

    float xVec[] = {x1, x2, x3, x4, x5};
    float yVec[] = {y1, y2, y3, y4, y5};
    float sigmas[5], delta1[5], delta2[5], slopes[5];
    bool isFlat[5];
    //5 categories for sigmas
    const uint16_t lowerModuleIndices[] = {lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5};

    computeSigmasForRegression(modulesInGPU, lowerModuleIndices, delta1, delta2, slopes, isFlat);
    regressionRadius = computeRadiusUsingRegression(5,xVec, yVec, delta1, delta2, slopes, isFlat, regressionG, regressionF, sigmas, chiSquared);

    //extra chi squared cuts!
    if(regressionRadius < 5.0f/(2.f * k2Rinv1GeVf))
    {
//        pass = pass and passChiSquaredConstraint(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, chiSquared);
        if(not pass) return pass;
    }

    //compute the other chisquared
    //non anchor is always shifted for tilted and endcap!
    float nonAnchorDelta1[5], nonAnchorDelta2[5], nonAnchorSlopes[5];
    float nonAnchorxs[] = { mdsInGPU.outerX[firstMDIndex], mdsInGPU.outerX[secondMDIndex], mdsInGPU.outerX[thirdMDIndex], mdsInGPU.outerX[fourthMDIndex], mdsInGPU.outerX[fifthMDIndex]};
    float nonAnchorys[] = { mdsInGPU.outerY[firstMDIndex], mdsInGPU.outerY[secondMDIndex], mdsInGPU.outerY[thirdMDIndex], mdsInGPU.outerY[fourthMDIndex], mdsInGPU.outerY[fifthMDIndex]};

    computeSigmasForRegression(modulesInGPU, lowerModuleIndices, nonAnchorDelta1, nonAnchorDelta2, nonAnchorSlopes, isFlat, 5, false);
    nonAnchorChiSquared = computeChiSquared(5, nonAnchorxs, nonAnchorys, nonAnchorDelta1, nonAnchorDelta2, nonAnchorSlopes, isFlat, regressionG, regressionF, regressionRadius);
    return pass;
}

//90% constraint
__device__ bool SDL::passChiSquaredConstraint(struct SDL::modules& modulesInGPU, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& chiSquared)
{
    //following Philip's layer number prescription
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return chiSquared < 0.01788f;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return chiSquared < 0.04725f;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return chiSquared < 0.04725f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {       
            return chiSquared < 0.01788f;
        }   
        else if(layer4 == 9 and layer5 == 15)
        {
            return chiSquared < 0.08234f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 8 and layer5 == 9)
        {
            return chiSquared < 0.02360f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return chiSquared < 0.07167f;
        }
        else if(layer4 == 13 and layer5 == 14)
        {   
            return chiSquared < 0.08234f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 7 and layer5 == 8)
        {
            return chiSquared < 0.01026f;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return chiSquared < 0.06238f;
        }
        else if(layer4 == 12 and layer5 == 13)
        {
            return chiSquared < 0.06238f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 4)
    {
        if(layer5 == 5)
        {
            return chiSquared < 0.04725f;
        }
        else if(layer5 == 12)
        {
            return chiSquared < 0.09461f;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return chiSquared < 0.00512f;
        }
        if(layer4 == 9 and layer5 == 15)
        {
            return chiSquared < 0.04112f;
        }
        else if(layer4 == 14 and layer5 == 15)
        {
            return chiSquared < 0.06238f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        if(layer4 == 8 and layer5 == 14)
        {
            return chiSquared < 0.07167f;
        }
        else if(layer4 == 13 and layer5 == 14)
        {
            return chiSquared < 0.06238f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 5 and layer5 == 6)
        {
            return chiSquared < 0.08234f;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return chiSquared < 0.10870f;
        }
        else if(layer4 == 12 and layer5 == 13)
        {
            return chiSquared < 0.10870f;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return chiSquared < 0.09461f;
    }
    else if(layer1 == 3 and layer2 == 4 and layer3 == 5 and layer4 == 12 and layer5 == 13)
    {
        return chiSquared < 0.09461f;
    }

    return true;
}

//bounds can be found at http://uaf-10.t2.ucsd.edu/~bsathian/SDL/T5_RZFix/t5_rz_thresholds.txt
__device__ bool SDL::passT5RZConstraint(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, unsigned int firstMDIndex, unsigned int secondMDIndex, unsigned int thirdMDIndex, unsigned int fourthMDIndex, unsigned int fifthMDIndex, uint16_t& lowerModuleIndex1, uint16_t& lowerModuleIndex2, uint16_t& lowerModuleIndex3, uint16_t& lowerModuleIndex4, uint16_t& lowerModuleIndex5, float& rzChiSquared, float& residual_missing, float& residual4, float& residual5, float inner_pt, float innerRadius, float g, float f, bool& TightCutFlag) 
{
    //(g,f) is the center of the circle fitted by the innermost 3 points on x,y coordinates
    const float& rt1 = mdsInGPU.anchorRt[firstMDIndex]/100; //in the unit of m instead of cm
    const float& rt2 = mdsInGPU.anchorRt[secondMDIndex]/100;
    const float& rt3 = mdsInGPU.anchorRt[thirdMDIndex]/100;
    const float& rt4 = mdsInGPU.anchorRt[fourthMDIndex]/100;
    const float& rt5 = mdsInGPU.anchorRt[fifthMDIndex]/100;

    const float& z1 = mdsInGPU.anchorZ[firstMDIndex]/100;
    const float& z2 = mdsInGPU.anchorZ[secondMDIndex]/100;
    const float& z3 = mdsInGPU.anchorZ[thirdMDIndex]/100;
    const float& z4 = mdsInGPU.anchorZ[fourthMDIndex]/100;
    const float& z5 = mdsInGPU.anchorZ[fifthMDIndex]/100;

    //following Philip's layer number prescription
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    //slope computed using the internal T3s
    const int moduleType1 = modulesInGPU.moduleType[lowerModuleIndex1]; //0 is ps, 1 is 2s
    const int moduleType2 = modulesInGPU.moduleType[lowerModuleIndex2];
    const int moduleType3 = modulesInGPU.moduleType[lowerModuleIndex3];
    const int moduleType4 = modulesInGPU.moduleType[lowerModuleIndex4];
    const int moduleType5 = modulesInGPU.moduleType[lowerModuleIndex5];

    const float& x1 = mdsInGPU.anchorX[firstMDIndex]/100;
    const float& x2 = mdsInGPU.anchorX[secondMDIndex]/100;
    const float& x3 = mdsInGPU.anchorX[thirdMDIndex]/100;
    const float& x4 = mdsInGPU.anchorX[fourthMDIndex]/100;
    const float& x5 = mdsInGPU.anchorX[fifthMDIndex]/100;
    const float& y1 = mdsInGPU.anchorY[firstMDIndex]/100;
    const float& y2 = mdsInGPU.anchorY[secondMDIndex]/100;
    const float& y3 = mdsInGPU.anchorY[thirdMDIndex]/100;
    const float& y4 = mdsInGPU.anchorY[fourthMDIndex]/100;
    const float& y5 = mdsInGPU.anchorY[fifthMDIndex]/100;

    float residual = 0;
    float error = 0;
    float x_center=g/100, y_center=f/100; 
    float x_init=mdsInGPU.anchorX[thirdMDIndex]/100;
    float y_init=mdsInGPU.anchorY[thirdMDIndex]/100;
    float z_init=mdsInGPU.anchorZ[thirdMDIndex]/100;
    float rt_init=mdsInGPU.anchorRt[thirdMDIndex]/100; //use the second MD as initial point

    if (moduleType3==1)  // 1: if MD3 is in 2s layer
    {
        x_init=mdsInGPU.anchorX[secondMDIndex]/100;
        y_init=mdsInGPU.anchorY[secondMDIndex]/100;
        z_init=mdsInGPU.anchorZ[secondMDIndex]/100;
        rt_init=mdsInGPU.anchorRt[secondMDIndex]/100;
    }

    //start from a circle of inner T3.
    // to determine the charge
    int charge=0;
    float slope3c=(y3-y_center)/(x3-x_center);
    float slope1c=(y1-y_center)/(x1-x_center);
    if((y3-y_center)>0 && (y1-y_center)>0) 
    {
        if (slope3c>slope1c) charge=-1; 
        else if (slope3c<slope1c) charge=1;
        if (slope1c>0 && slope3c<0) charge=-1;
        if (slope1c<0 && slope3c>0) charge=1;
    }
    else if((y3-y_center)<0 && (y1-y_center)<0) 
    {
        if (slope3c>slope1c) charge=-1; 
        else if (slope3c<slope1c) charge=1;
        if (slope1c<0 && slope3c>0) charge=1;
        if (slope1c>0 && slope3c<0) charge=-1;
    }
    else if ((y3-y_center)<0 && (y1-y_center)>0)
    {
        if ((x3-x_center)>0 && (x1-x_center)>0) charge = 1;
        else if ((x3-x_center)<0 && (x1-x_center)<0) charge = -1;
    }
    else if ((y3-y_center)>0 && (y1-y_center)<0)
    {
        if ((x3-x_center)>0 && (x1-x_center)>0) charge = -1;
        else if ((x3-x_center)<0 && (x1-x_center)<0) charge = 1;
    }

    float pseudo_phi = atan((y_init-y_center)/(x_init-x_center)); //actually represent pi/2-phi, wrt helix axis z
    float Pt=inner_pt, Px=Pt*abs(sin(pseudo_phi)), Py=Pt*abs(cos(pseudo_phi));

    if (x_init>x_center && y_init>y_center) //1st quad
    {
        if (charge==1) Py=-Py;
        if (charge==-1) Px=-Px;
    }
    if (x_init<x_center && y_init>y_center) //2nd quad
    {
        if (charge==-1) {Px=-Px; Py=-Py;}
    }
    if (x_init<x_center && y_init<y_center) //3rd quad
    {
        if (charge==1) Px=-Px;
        if (charge==-1) Py=-Py;
    }        
    if (x_init>x_center && y_init<y_center) //4th quad
    {
        if (charge==1) {Px=-Px; Py=-Py;}
    }

    //to get Pz, we use pt/pz=ds/dz, ds is the arclength between MD1 and MD3.
    float AO=sqrt((x1-x_center)*(x1-x_center)+(y1-y_center)*(y1-y_center));
    float BO=sqrt((x_init-x_center)*(x_init-x_center)+(y_init-y_center)*(y_init-y_center));
    float AB=sqrt((x1-x_init)*(x1-x_init)+(y1-y_init)*(y1-y_init)); 
    float dPhi = acos((AO*AO+BO*BO-AB*AB)/(2*AO*BO));
    float ds=innerRadius/100*dPhi;

//    float ds = sqrt((y_init-y1)*(y_init-y1)+(x_init-x1)*(x_init-x1)); //large ds->smallerPz->smaller residual
    float Pz=(z_init-z1)/ds*Pt;
    float p = sqrt(Px*Px+Py*Py+Pz*Pz);

    float B = 3.8112;
    float a = -0.299792*B*charge;

    float zsi, rtsi;
    int layeri, moduleTypei;
    float expectrt4=0,expectrt5=0,expectz4=0, expectz5=0;
    rzChiSquared=0;
    for(size_t i = 2; i < 6; i++)
    {
        if (i==2){
            zsi = z2;
            rtsi = rt2;
            layeri=layer2;
            moduleTypei=moduleType2;
        }
        else if (i==3) {
            zsi = z3;
            rtsi = rt3;
            layeri=layer3;
            moduleTypei=moduleType3;
        }
        else if (i==4){
            zsi = z4;
            rtsi = rt4;
            layeri=layer4;
            moduleTypei=moduleType4;
        }
        else if (i==5){
            zsi = z5;
            rtsi = rt5;
            layeri=layer5;
            moduleTypei=moduleType5;
        }

        if (moduleType3==0) { //0: ps
            if (i==3) continue;
        }
        else{
            if (i==2) continue;
        }

        // calculation is copied from PixelTriplet.cu SDL::computePT3RZChiSquared
        float diffr=0, diffz=0;

        float rou = a/p;
        // for barrel
        float s = (zsi-z_init)*p/Pz;
        float x = x_init + Px/a*sin(rou*s)-Py/a*(1-cos(rou*s));
        float y = y_init + Py/a*sin(rou*s)+Px/a*(1-cos(rou*s));
        diffr = (rtsi-sqrt(x*x+y*y))*100;
        if (i==4) expectrt4=sqrt(x*x+y*y);
        if (i==5) expectrt5=sqrt(x*x+y*y);

        // for endcap
        float paraA = rt_init*rt_init + 2*(Px*Px+Py*Py)/(a*a) + 2*(y_init*Px-x_init*Py)/a - rtsi*rtsi;
        float paraB = 2*(x_init*Px+y_init*Py)/a;
        float paraC = 2*(y_init*Px-x_init*Py)/a+2*(Px*Px+Py*Py)/(a*a);
        float A=paraB*paraB+paraC*paraC;
        float B=2*paraA*paraB;
        float C=paraA*paraA-paraC*paraC;
        float sol1 = (-B+sqrt(B*B-4*A*C))/(2*A);
        float sol2 = (-B-sqrt(B*B-4*A*C))/(2*A);
        float solz1 = asin(sol1)/rou*Pz/p+z_init;
        float solz2 = asin(sol2)/rou*Pz/p+z_init;
        float diffz1 = (solz1-zsi)*100;
        float diffz2 = (solz2-zsi)*100;
        diffz = (fabs(diffz1)<fabs(diffz2)) ? diffz1 : diffz2;
        if (i==4 && fabs(diffz1)<fabs(diffz2)) expectz4 = solz1;
        if (i==4 && fabs(diffz1)>fabs(diffz2)) expectz4 = solz2;
        if (i==5 && fabs(diffz1)<fabs(diffz2)) expectz5 = solz1;
        if (i==5 && fabs(diffz1)>fabs(diffz2)) expectz5 = solz2;

        residual = (layeri>6) ? diffr : diffz ;

        //PS Modules
        if(moduleTypei == 0)
        {
            error = 0.15f;
        }
        else //2S modules
        {
            error = 5.0f;
        }
        if (i==4) residual4=residual/error;
        if (i==5) residual5=residual/error;

        //check the tilted module, side: PosZ, NegZ, Center(for not tilted)
        float drdz;
        short side, subdets;
        if (i==2){
            drdz=abs(modulesInGPU.drdzs[lowerModuleIndex2]);
            side=modulesInGPU.sides[lowerModuleIndex2];
            subdets=modulesInGPU.subdets[lowerModuleIndex2];
        }
        if (i==3){
            drdz=abs(modulesInGPU.drdzs[lowerModuleIndex3]);
            side=modulesInGPU.sides[lowerModuleIndex3];
            subdets=modulesInGPU.subdets[lowerModuleIndex3];
        }
        if (i==2 || i==3){
            residual = (layeri <= 6 && ((side == SDL::Center) or (drdz < 1))) ? diffz : diffr;
//            residual_missing=residual;
            float projection_missing=1;
        if (drdz<1)
            projection_missing = ((subdets == SDL::Endcap) or (side == SDL::Center)) ? 1.f : 1/sqrt(1+drdz*drdz); // cos(atan(drdz)), if dr/dz<1
        if (drdz>1)
            projection_missing = ((subdets == SDL::Endcap) or (side == SDL::Center)) ? 1.f : drdz/sqrt(1+drdz*drdz);//sin(atan(drdz)), if dr/dz>1
            error=error*projection_missing;
            residual_missing=residual/error;
        }
        rzChiSquared += 12*(residual * residual)/(error * error);
    }
//    rzChiSquared = 12*(residual4 * residual4 + residual5 * residual5 + residual_missing * residual_missing);

//    if (isnan(rzChiSquared)) printf("rzChi2: %f, residual2: %f, inner_pt:%f, pseudo_phi: %f, charge: %i, Px:%f, Py:%f, x1:%f, x2:%f, x3:%f, x4:%f, x5:%f, y1:%f, y2:%f, y3:%f, y4:%f, y5:%f, z1:%f, z2:%f, z3:%f, z4:%f, z5:%f, x_center:%f, y_center:%f, slope1c:%f, slope3c:%f\n", rzChiSquared, residual_missing, inner_pt, pseudo_phi, charge, Px, Py, x1, x2, x3, x4, x5, y1, y2, y3, y4, y5, z1, z2, z3, z4, z5, x_center, y_center, slope1c, slope3c);

//    if(layer1 == 1 and layer2 == 2 and layer3 == 3 and layer4 == 12 and layer5 == 13){
//        printf("rt1:%f, rt2:%f, rt3:%f, rt4:%f, rt5:%f\n", rt1, rt2, rt3, rt4, rt5);
//        printf("x1:%f, x2:%f, x3:%f, x4:%f, x5:%f\n", x1, x2, x3, x4, x5);
//        printf("y1:%f, y2:%f, y3:%f, y4:%f, y5:%f\n", y1, y2, y3, y4, y5);
//        printf("z1:%f, z2:%f, z3:%f, z4:%f, z5:%f\n", z1, z2, z3, z4, z5);
//        printf("rt4_ex:%f, rt5_ex:%f\n", expectrt4, expectrt5);
//        printf("z4_ex:%f, z5_ex:%f\n", expectz4, expectz5);
//        printf("residual_missing:%f\n", residual_missing);
//        printf("Pt:%f, Px:%f, Py:%f, Pz:%f, charge: %i, residual_missing: %f, residual4: %f, residual5:%f, moduleType3:%i\n", Pt, Px, Py, Pz, charge, residual_missing, residual4, residual5, moduleType3);
//        printf("rzChi2: %f, residual2: %f, inner_pt:%f, pseudo_phi: %f, charge: %i, Px:%f, Py:%f, x1:%f, x2:%f, x3:%f, x4:%f, x5:%f, y1:%f, y2:%f, y3:%f, y4:%f, y5:%f, z1:%f, z2:%f, z3:%f, z4:%f, z5:%f, x_center:%f, y_center:%f, slope1c:%f, slope3c:%f\n", rzChiSquared, residual_missing, inner_pt, pseudo_phi, charge, Px, Py, x1, x2, x3, x4, x5, y1, y2, y3, y4, y5, z1, z2, z3, z4, z5, x_center, y_center, slope1c, slope3c);
//        printf("residual_missing:%f\n", residual_missing);
//    }

    // when building T5, apply 99% chi2 cuts as default, and add to pT5 collection. But when adding T5 to TC collections, appy 95% cut to reduce the fake rate
    TightCutFlag = false;
    //categories!
    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 4 and layer5 == 5) //11
        {
            if (rzChiSquared < 15.595f) TightCutFlag = 1;
            return rzChiSquared < 28.902f; 
        }
        else if(layer4 == 4 and layer5 == 12) //12
        {
            if (rzChiSquared < 14.614f) TightCutFlag = 1;
            return rzChiSquared < 23.037f;
        }
        else if(layer4 == 7 and layer5 == 8) //8
        {   
            if (rzChiSquared < 27.824f) TightCutFlag = 1;
            return rzChiSquared < 44.247f;
        }
        else if(layer4 == 7 and layer5 == 13) //9
        {
            if (rzChiSquared < 18.085f) TightCutFlag = 1;
            return rzChiSquared < 33.023f;
        }
        else if(layer4 == 12 and layer5 == 13) //10
        {
            if (rzChiSquared < 13.267f) TightCutFlag = 1;
            return rzChiSquared < 21.186f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 8 and layer5 == 9) //5
        {
            if (rzChiSquared < 60.195f) TightCutFlag = 1;
            return rzChiSquared < 117.118f;
        }
        if(layer4 == 8 and layer5 == 14) //6
        {
            if (rzChiSquared < 19.490f) TightCutFlag = 1;
            return rzChiSquared < 55.322f;
        }
        else if(layer4 == 13 and layer5 == 14) //7
        {
            if (rzChiSquared < 10.157f) TightCutFlag = 1;
            return rzChiSquared < 14.217f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8 and layer4 == 9) 
    {
        if (layer5 == 10) //3
        {
            if (rzChiSquared < 63.697f) TightCutFlag = 1;
            return rzChiSquared < 109.584f;
        }
        if (layer5 == 15) //4
        {
            if (rzChiSquared < 18.346f) TightCutFlag = 1;
            return rzChiSquared < 34.941f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 5 and layer5 == 6) //18
        {
            if (rzChiSquared < 6.053f) TightCutFlag = 1;
            return rzChiSquared < 8.629f;
        }
        else if(layer4 == 5 and layer5 == 12) //19
        {
            if (rzChiSquared < 5.693f) TightCutFlag = 1;
            return rzChiSquared < 7.929f;
        }

        else if(layer4 == 12 and layer5 == 13) //20
        {
            if (rzChiSquared < 5.44f) TightCutFlag = 1;
            return rzChiSquared < 7.627f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7) 
    {
        if(layer4 == 8 and layer5 == 14) //16
        {
            if (rzChiSquared < 23.730f) TightCutFlag = 1;
            return rzChiSquared < 23.748f;
        }
        if(layer4 == 13 and layer5 == 14) //17
        {
            if (rzChiSquared < 10.55f) TightCutFlag = 1;
            return rzChiSquared < 17.817f;
        }
    }

    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 15) //14
        {
            if (rzChiSquared < 24.558f) TightCutFlag = 1;
            return rzChiSquared < 40.918f;
        }
        else if(layer4 == 14 and layer5 == 15) //15
        {
            if (rzChiSquared < 8.752f) TightCutFlag = 1;
            return rzChiSquared < 13.678f;
        }
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 15 and layer5 == 16) //2
    {
        if (rzChiSquared < 7.994f) TightCutFlag = 1;
        return rzChiSquared < 11.622f;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 11) //0
    {
        if (rzChiSquared < 56.313f) TightCutFlag = 1;
        return rzChiSquared < 93.893f;
    }

    else if(layer1 == 7 and layer2 == 8 and layer3 == 9 and layer4 == 10 and layer5 == 16) //1
    {
        if (rzChiSquared < 22.041f) TightCutFlag = 1;
        return rzChiSquared < 37.087f;
    }
    return true;
}

__device__ bool SDL::checkIntervalOverlap(const float& firstMin, const float& firstMax, const float& secondMin, const float& secondMax)
{
    return ((firstMin <= secondMin) & (secondMin < firstMax)) |  ((secondMin < firstMin) & (firstMin < secondMax));
}

/*bounds for high Pt taken from : http://uaf-10.t2.ucsd.edu/~bsathian/SDL/T5_efficiency/efficiencies/new_efficiencies/efficiencies_20210513_T5_recovering_high_Pt_efficiencies/highE_radius_matching/highE_bounds.txt */

__device__ bool SDL::matchRadiiBBBBB(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound =  0.1512f;
    float bridgeInvRadiusErrorBound = 0.1781f;
    float outerInvRadiusErrorBound = 0.1840f;

    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 0.4449f;
        bridgeInvRadiusErrorBound = 0.4033f;
        outerInvRadiusErrorBound = 0.8016f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax);
}

__device__ bool SDL::matchRadiiBBBBE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{

    float innerInvRadiusErrorBound =  0.1781f;
    float bridgeInvRadiusErrorBound = 0.2167f;
    float outerInvRadiusErrorBound = 1.1116f;

    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 0.4750f;
        bridgeInvRadiusErrorBound = 0.3903f;
        outerInvRadiusErrorBound = 15.2120f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, bridgeInvRadiusMin, bridgeInvRadiusMax);
}

__device__ bool SDL::matchRadiiBBBEE12378(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound = 0.178f;
    float bridgeInvRadiusErrorBound = 0.507f;
    float outerInvRadiusErrorBound = 7.655f;

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, fminf(bridgeInvRadiusMin, 1.0f/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0f/bridgeRadiusMin2S));
}

__device__ bool SDL::matchRadiiBBBEE23478(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound = 0.2097f;
    float bridgeInvRadiusErrorBound = 0.8557f;
    float outerInvRadiusErrorBound = 24.0450f;

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, fminf(bridgeInvRadiusMin, 1.0f/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0f/bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBBBEE34578(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound = 0.066f;
    float bridgeInvRadiusErrorBound = 0.617f;
    float outerInvRadiusErrorBound = 2.688f;

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, fminf(bridgeInvRadiusMin, 1.0f/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0f/bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBBBEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{

    float innerInvRadiusErrorBound =  0.1840f;
    float bridgeInvRadiusErrorBound = 0.5971f;
    float outerInvRadiusErrorBound = 11.7102f;

    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf)) //as good as no selections
    {
        innerInvRadiusErrorBound = 1.0412f;
        outerInvRadiusErrorBound = 32.2737f;
        bridgeInvRadiusErrorBound = 10.9688f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, fminf(bridgeInvRadiusMin, 1.0f/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0f/bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBBEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound =  0.6376f;
    float bridgeInvRadiusErrorBound = 2.1381f;
    float outerInvRadiusErrorBound = 20.4179f;

    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf)) //as good as no selections!
    {
        innerInvRadiusErrorBound = 12.9173f;
        outerInvRadiusErrorBound = 25.6702f;
        bridgeInvRadiusErrorBound = 5.1700f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(innerInvRadiusMin, innerInvRadiusMax, fminf(bridgeInvRadiusMin, 1.0f/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0f/bridgeRadiusMin2S));

}

__device__ bool SDL::matchRadiiBEEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{

    float innerInvRadiusErrorBound =  1.9382f;
    float bridgeInvRadiusErrorBound = 3.7280f;
    float outerInvRadiusErrorBound = 5.7030f;


    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 23.2713f;
        outerInvRadiusErrorBound = 24.0450f;
        bridgeInvRadiusErrorBound = 21.7980f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(fminf(innerInvRadiusMin, 1.0/innerRadiusMax2S), fmaxf(innerInvRadiusMax, 1.0/innerRadiusMin2S), fminf(bridgeInvRadiusMin, 1.0/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0/bridgeRadiusMin2S));
}

__device__ bool SDL::matchRadiiEEEEE(const float& innerRadius, const float& bridgeRadius, const float& outerRadius, const float& innerRadiusMin2S, const float& innerRadiusMax2S, const float& bridgeRadiusMin2S, const float& bridgeRadiusMax2S, const float& outerRadiusMin2S, const float& outerRadiusMax2S, float& innerInvRadiusMin, float& innerInvRadiusMax, float& bridgeInvRadiusMin, float& bridgeInvRadiusMax, float& outerInvRadiusMin, float& outerInvRadiusMax)
{
    float innerInvRadiusErrorBound =  1.9382f;
    float bridgeInvRadiusErrorBound = 2.2091f;
    float outerInvRadiusErrorBound = 7.4084f;

    if(innerRadius > 2.0f/(2.f * k2Rinv1GeVf))
    {
        innerInvRadiusErrorBound = 22.5226f;
        bridgeInvRadiusErrorBound = 21.0966f;
        outerInvRadiusErrorBound = 19.1252f;
    }

    innerInvRadiusMax = (1.f + innerInvRadiusErrorBound) / innerRadius;
    innerInvRadiusMin = fmaxf(0.f, (1.f - innerInvRadiusErrorBound) / innerRadius);

    bridgeInvRadiusMax = (1.f + bridgeInvRadiusErrorBound) / bridgeRadius;
    bridgeInvRadiusMin = fmaxf(0.f, (1.f - bridgeInvRadiusErrorBound) / bridgeRadius);

    outerInvRadiusMax = (1.f + outerInvRadiusErrorBound) / outerRadius;
    outerInvRadiusMin = fmaxf(0.f, (1.f - outerInvRadiusErrorBound) / outerRadius);

    return checkIntervalOverlap(fminf(innerInvRadiusMin, 1.0/innerRadiusMax2S), fmaxf(innerInvRadiusMax, 1.0/innerRadiusMin2S), fminf(bridgeInvRadiusMin, 1.0/bridgeRadiusMax2S), fmaxf(bridgeInvRadiusMax, 1.0/bridgeRadiusMin2S));
}

__device__ void SDL::computeErrorInRadius(float* x1Vec, float* y1Vec, float* x2Vec, float* y2Vec, float* x3Vec, float* y3Vec, float& minimumRadius, float& maximumRadius)
{
    //brute force
    float candidateRadius;
    float g, f;
    minimumRadius = 123456789.f;
    maximumRadius = 0.f;
    for(size_t i = 0; i < 3; i++)
    {
        float x1 = x1Vec[i];
	float y1 = y1Vec[i];
        for(size_t j = 0; j < 3; j++)
        {
	    float x2 = x2Vec[j];
	    float y2 = y2Vec[j];
            for(size_t k = 0; k < 3; k++)
            {
	       float x3 = x3Vec[k];
               float y3 = y3Vec[k];
               candidateRadius = computeRadiusFromThreeAnchorHits(x1, y1, x2, y2, x3, y3, g, f);
               maximumRadius = fmaxf(candidateRadius, maximumRadius);
               minimumRadius = fminf(candidateRadius, minimumRadius);
            }
        }
    }
}
__device__ float SDL::computeRadiusFromThreeAnchorHits(float x1, float y1, float x2, float y2, float x3, float y3, float& g, float& f)
{
    float radius = 0.f;

    //writing manual code for computing radius, which obviously sucks
    //TODO:Use fancy inbuilt libraries like cuBLAS or cuSOLVE for this!
    //(g,f) -> center
    //first anchor hit - (x1,y1), second anchor hit - (x2,y2), third anchor hit - (x3, y3)

    /*
    if((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0)
    {
        return -1; //WTF man, three collinear points!
    }
    */

    float denomInv = 1.0f/((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3));

    float xy1sqr = x1 * x1 + y1 * y1;

    float xy2sqr = x2 * x2 + y2 * y2;

    float xy3sqr = x3 * x3 + y3 * y3;

    g = 0.5f * ((y3 - y2) * xy1sqr + (y1 - y3) * xy2sqr + (y2 - y1) * xy3sqr) * denomInv;

    f = 0.5f * ((x2 - x3) * xy1sqr + (x3 - x1) * xy2sqr + (x1 - x2) * xy3sqr) * denomInv;

    float c = ((x2 * y3 - x3 * y2) * xy1sqr + (x3 * y1 - x1 * y3) * xy2sqr + (x1 * y2 - x2 * y1) * xy3sqr) * denomInv;

    if(((y1 - y3) * (x2 - x3) - (x1 - x3) * (y2 - y3) == 0) || (g * g + f * f - c < 0))
    {
        printf("three collinear points or FATAL! r^2 < 0!\n");
	radius = -1.f;
    }
    else
      radius = sqrtf(g * g  + f * f - c);

    return radius;
}

__device__ bool SDL::T5HasCommonMiniDoublet(struct SDL::triplets& tripletsInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerTripletIndex, unsigned int outerTripletIndex)
{
    unsigned int innerOuterSegmentIndex = tripletsInGPU.segmentIndices[2 * innerTripletIndex + 1];
    unsigned int outerInnerSegmentIndex = tripletsInGPU.segmentIndices[2 * outerTripletIndex];
    unsigned int innerOuterOuterMiniDoubletIndex = segmentsInGPU.mdIndices[2 * innerOuterSegmentIndex + 1]; //inner triplet outer segment outer MD index
    unsigned int outerInnerInnerMiniDoubletIndex = segmentsInGPU.mdIndices[2 * outerInnerSegmentIndex]; //outer triplet inner segmnet inner MD index


    return (innerOuterOuterMiniDoubletIndex == outerInnerInnerMiniDoubletIndex);
}

__device__ void SDL::computeSigmasForRegression(SDL::modules& modulesInGPU, const uint16_t* lowerModuleIndices, float* delta1, float* delta2, float* slopes, bool* isFlat, int nPoints, bool anchorHits) 
{
   /*bool anchorHits required to deal with a weird edge case wherein 
     the hits ultimately used in the regression are anchor hits, but the
     lower modules need not all be Pixel Modules (in case of PS). Similarly,
     when we compute the chi squared for the non-anchor hits, the "partner module"
     need not always be a PS strip module, but all non-anchor hits sit on strip 
     modules.
    */
    ModuleType moduleType;
    short moduleSubdet, moduleSide;
    float inv1 = 0.01f/0.009f;
    float inv2 = 0.15f/0.009f;
    float inv3 = 2.4f/0.009f;
    for(size_t i=0; i<nPoints; i++)
    {
        moduleType = modulesInGPU.moduleType[lowerModuleIndices[i]];
        moduleSubdet = modulesInGPU.subdets[lowerModuleIndices[i]];
        moduleSide = modulesInGPU.sides[lowerModuleIndices[i]];
        float& drdz = modulesInGPU.drdzs[lowerModuleIndices[i]];
        slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]]; 
        //category 1 - barrel PS flat
        if(moduleSubdet == Barrel and moduleType == PS and moduleSide == Center)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            delta2[i] = inv1;//1.1111f;//0.01;
            slopes[i] = -999.f;
            isFlat[i] = true;
        }

        //category 2 - barrel 2S
        else if(moduleSubdet == Barrel and moduleType == TwoS)
        {
            delta1[i] = 1.f;//0.009;
            delta2[i] = 1.f;//0.009;
            slopes[i] = -999.f;
            isFlat[i] = true;
        }

        //category 3 - barrel PS tilted
        else if(moduleSubdet == Barrel and moduleType == PS and moduleSide != Center)
        {

            //delta1[i] = 0.01;
            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            if(anchorHits)
            {
                delta2[i] = (inv2 * drdz/sqrtf(1 + drdz * drdz));
            }
            else
            {
                delta2[i] = (inv3 * drdz/sqrtf(1 + drdz * drdz));
            }
        }
        //category 4 - endcap PS
        else if(moduleSubdet == Endcap and moduleType == PS)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            /*despite the type of the module layer of the lower module index,
            all anchor hits are on the pixel side and all non-anchor hits are
            on the strip side!*/
            if(anchorHits)
            {
                delta2[i] = inv2;//16.6666f;//0.15f;
            }
            else
            {
                delta2[i] = inv3;//266.666f;//2.4f;
            }
        }

        //category 5 - endcap 2S
        else if(moduleSubdet == Endcap and moduleType == TwoS)
        {
            delta1[i] = 1.f;//0.009;
            delta2[i] = 500.f*inv1;//555.5555f;//5.f;
            isFlat[i] = false;
        }
        else
        {
            printf("ERROR!!!!! I SHOULDN'T BE HERE!!!! subdet = %d, type = %d, side = %d\n", moduleSubdet, moduleType, moduleSide);
        }
    }
}

__device__ float SDL::computeRadiusUsingRegression(int nPoints, float* xs, float* ys, float* delta1, float* delta2, float* slopes, bool* isFlat, float& g, float& f, float* sigmas, float& chiSquared)
{
    float radius = 0.f;

    //some extra variables
    //the two variables will be caled x1 and x2, and y (which is x^2 + y^2)

    float sigmaX1Squared = 0.f;
    float sigmaX2Squared = 0.f;
    float sigmaX1X2 = 0.f; 
    float sigmaX1y = 0.f; 
    float sigmaX2y = 0.f;
    float sigmaY = 0.f;
    float sigmaX1 = 0.f;
    float sigmaX2 = 0.f;
    float sigmaOne = 0.f;

    float xPrime, yPrime, absArctanSlope, angleM;
    for(size_t i = 0; i < nPoints; i++)
    {
        //computing sigmas is a very tricky affair
        //if the module is tilted or endcap, we need to use the slopes properly!

        absArctanSlope = ((slopes[i] != 123456789) ? fabs(atanf(slopes[i])) : 0.5f*float(M_PI)); // Since C++ can't represent infinity, SDL_INF = 123456789 was used to represent infinity in the data table

        if(xs[i] > 0 and ys[i] > 0)
        {
            angleM = 0.5f*float(M_PI) - absArctanSlope;
        }
        else if(xs[i] < 0 and ys[i] > 0)
        {
            angleM = absArctanSlope + 0.5f*float(M_PI);
        }
        else if(xs[i] < 0 and ys[i] < 0)
        {
            angleM = -(absArctanSlope + 0.5f*float(M_PI));
        }
        else if(xs[i] > 0 and ys[i] < 0)
        {
            angleM = -(0.5f*float(M_PI) - absArctanSlope);
        }

        if(not isFlat[i])
        {
            xPrime = xs[i] * cosf(angleM) + ys[i] * sinf(angleM);
            yPrime = ys[i] * cosf(angleM) - xs[i] * sinf(angleM);
        }
        else
        {
            xPrime = xs[i];
            yPrime = ys[i];
        }
        sigmas[i] = 2 * sqrtf((xPrime * delta1[i]) * (xPrime * delta1[i]) + (yPrime * delta2[i]) * (yPrime * delta2[i]));

        sigmaX1Squared += (xs[i] * xs[i])/(sigmas[i] * sigmas[i]);
        sigmaX2Squared += (ys[i] * ys[i])/(sigmas[i] * sigmas[i]);
        sigmaX1X2 += (xs[i] * ys[i])/(sigmas[i] * sigmas[i]);
        sigmaX1y += (xs[i] * (xs[i] * xs[i] + ys[i] * ys[i]))/(sigmas[i] * sigmas[i]);
        sigmaX2y += (ys[i] * (xs[i] * xs[i] + ys[i] * ys[i]))/(sigmas[i] * sigmas[i]);
        sigmaY += (xs[i] * xs[i] + ys[i] * ys[i])/(sigmas[i] * sigmas[i]);
        sigmaX1 += xs[i]/(sigmas[i] * sigmas[i]);
        sigmaX2 += ys[i]/(sigmas[i] * sigmas[i]);
        sigmaOne += 1.0f/(sigmas[i] * sigmas[i]);
    }
    float denominator = (sigmaX1X2 - sigmaX1 * sigmaX2) * (sigmaX1X2 - sigmaX1 * sigmaX2) - (sigmaX1Squared - sigmaX1 * sigmaX1) * (sigmaX2Squared - sigmaX2 * sigmaX2);

    float twoG = ((sigmaX2y - sigmaX2 * sigmaY) * (sigmaX1X2 - sigmaX1 * sigmaX2) - (sigmaX1y - sigmaX1 * sigmaY) * (sigmaX2Squared - sigmaX2 * sigmaX2)) / denominator;
    float twoF = ((sigmaX1y - sigmaX1 * sigmaY) * (sigmaX1X2 - sigmaX1 * sigmaX2) - (sigmaX2y - sigmaX2 * sigmaY) * (sigmaX1Squared - sigmaX1 * sigmaX1)) / denominator;

    float c = -(sigmaY - twoG * sigmaX1 - twoF * sigmaX2)/sigmaOne;
    g = 0.5f*twoG;
    f = 0.5f*twoF;
    if(g * g + f * f - c < 0)
    {
        printf("FATAL! r^2 < 0!\n");
        return -1;
    }
    
    radius = sqrtf(g * g  + f * f - c);
    //compute chi squared
    chiSquared = 0.f;
    for(size_t i = 0; i < nPoints; i++)
    {
       chiSquared +=  (xs[i] * xs[i] + ys[i] * ys[i] - twoG * xs[i] - twoF * ys[i] + c) * (xs[i] * xs[i] + ys[i] * ys[i] - twoG * xs[i] - twoF * ys[i] + c) / (sigmas[i] * sigmas[i]);
    }
    return radius;
}

__device__ float SDL::computeChiSquared(int nPoints, float* xs, float* ys, float* delta1, float* delta2, float* slopes, bool* isFlat, float g, float f, float radius)
{
    // given values of (g, f, radius) and a set of points (and its uncertainties)
    //compute chi squared
    float c = g*g + f*f - radius*radius;
    float chiSquared = 0.f;
    float absArctanSlope, angleM, xPrime, yPrime, sigma;
    for(size_t i = 0; i < nPoints; i++)
    {
        absArctanSlope = ((slopes[i] != 123456789) ? fabs(atanf(slopes[i])) : 0.5f*float(M_PI)); // Since C++ can't represent infinity, SDL_INF = 123456789 was used to represent infinity in the data table
        if(xs[i] > 0 and ys[i] > 0)
        {
            angleM = 0.5f*float(M_PI) - absArctanSlope;
        }
        else if(xs[i] < 0 and ys[i] > 0)
        {
            angleM = absArctanSlope + 0.5f*float(M_PI);
        }
        else if(xs[i] < 0 and ys[i] < 0)
        {
            angleM = -(absArctanSlope + 0.5f*float(M_PI));
        }
        else if(xs[i] > 0 and ys[i] < 0)
        {
            angleM = -(0.5f*float(M_PI) - absArctanSlope);
        }

        if(not isFlat[i])
        {
            xPrime = xs[i] * cosf(angleM) + ys[i] * sinf(angleM);
            yPrime = ys[i] * cosf(angleM) - xs[i] * sinf(angleM);
        }
        else
        {
            xPrime = xs[i];
            yPrime = ys[i];
        }
        sigma = 2 * sqrtf((xPrime * delta1[i]) * (xPrime * delta1[i]) + (yPrime * delta2[i]) * (yPrime * delta2[i]));
        chiSquared +=  (xs[i] * xs[i] + ys[i] * ys[i] - 2 * g * xs[i] - 2 * f * ys[i] + c) * (xs[i] * xs[i] + ys[i] * ys[i] - 2 * g * xs[i] - 2 * f * ys[i] + c) / (sigma * sigma);
    }
    return chiSquared; 
}

__global__ void SDL::createQuintupletsInGPUv2(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, struct SDL::quintuplets& quintupletsInGPU, struct SDL::objectRanges& rangesInGPU, uint16_t nEligibleT5Modules)
{
    int gidy = blockIdx.y * blockDim.y + threadIdx.y;
    int npy = gridDim.y * blockDim.y;
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    int npx = gridDim.x * blockDim.x;
    int gidz = blockIdx.z * blockDim.z + threadIdx.z;
    int npz = gridDim.z * blockDim.z;

    for (int iter=gidz; iter < nEligibleT5Modules; iter+=npz){
      uint16_t lowerModule1 = rangesInGPU.indicesOfEligibleT5Modules[iter];


      unsigned int nInnerTriplets = tripletsInGPU.nTriplets[lowerModule1];
      for( unsigned int innerTripletArrayIndex =gidy; innerTripletArrayIndex < nInnerTriplets; innerTripletArrayIndex+=npy){

      unsigned int innerTripletIndex = rangesInGPU.tripletModuleIndices[lowerModule1] + innerTripletArrayIndex;
      uint16_t lowerModule2 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 1];
      uint16_t lowerModule3 = tripletsInGPU.lowerModuleIndices[3 * innerTripletIndex + 2];
      unsigned int nOuterTriplets = tripletsInGPU.nTriplets[lowerModule3];
        for (int outerTripletArrayIndex=gidx; outerTripletArrayIndex < nOuterTriplets; outerTripletArrayIndex+=npx)
        {
            unsigned int outerTripletIndex = rangesInGPU.tripletModuleIndices[lowerModule3] + outerTripletArrayIndex;
            uint16_t lowerModule4 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 1];
            uint16_t lowerModule5 = tripletsInGPU.lowerModuleIndices[3 * outerTripletIndex + 2];

            float innerRadius, outerRadius, bridgeRadius, regressionG, regressionF, regressionRadius, rzChiSquared, chiSquared, nonAnchorChiSquared; //required for making distributions
            bool TightCutFlag;
            bool success = runQuintupletDefaultAlgo(modulesInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerTripletIndex, outerTripletIndex, innerRadius, outerRadius,  bridgeRadius, regressionG, regressionF, regressionRadius, rzChiSquared, chiSquared, nonAnchorChiSquared, TightCutFlag);

            if(success)
            {
                short layer2_adjustment;
                int layer = modulesInGPU.layers[lowerModule1];
                if(layer == 1)
                {
                    layer2_adjustment = 1;
                } //get upper segment to be in second layer
                else if(layer == 2)
                {
                    layer2_adjustment = 0;
                } // get lower segment to be in second layer
                else
                {
                    return;
                } // ignore anything else TODO: move this to start, before object is made (faster)
                unsigned int totOccupancyQuintuplets = atomicAdd(&quintupletsInGPU.totOccupancyQuintuplets[lowerModule1], 1);
                if(totOccupancyQuintuplets >= (rangesInGPU.quintupletModuleIndices[lowerModule1 + 1] - rangesInGPU.quintupletModuleIndices[lowerModule1]))
                {
#ifdef Warnings
                    printf("Quintuplet excess alert! Module index = %d\n", lowerModule1);
#endif
                }
                else
                {
                    unsigned int quintupletModuleIndex = atomicAdd(&quintupletsInGPU.nQuintuplets[lowerModule1], 1);
                    //this if statement should never get executed!
                    if(rangesInGPU.quintupletModuleIndices[lowerModule1] == -1)
                    {
                        printf("Quintuplets : no memory for module at module index = %d\n", lowerModule1);
                    }
                    else
                    {
                        unsigned int quintupletIndex = rangesInGPU.quintupletModuleIndices[lowerModule1] +  quintupletModuleIndex;
                        float phi = mdsInGPU.anchorPhi[segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+layer2_adjustment]]];
                        float eta = mdsInGPU.anchorEta[segmentsInGPU.mdIndices[2*tripletsInGPU.segmentIndices[2*innerTripletIndex+layer2_adjustment]]];
                        float pt = (innerRadius+outerRadius)*3.8f*1.602f/(2*100*5.39f);
                        float scores = chiSquared + nonAnchorChiSquared;
                        addQuintupletToMemory(tripletsInGPU, quintupletsInGPU, innerTripletIndex, outerTripletIndex, lowerModule1, lowerModule2, lowerModule3, lowerModule4, lowerModule5, innerRadius, bridgeRadius, outerRadius, regressionG, regressionF, regressionRadius, rzChiSquared, chiSquared, nonAnchorChiSquared, pt,eta,phi,scores,layer,quintupletIndex, TightCutFlag);

                        tripletsInGPU.partOfT5[quintupletsInGPU.tripletIndices[2 * quintupletIndex]] = true;
                        tripletsInGPU.partOfT5[quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1]] = true;
                    }
                }
            }
        }
      }
    }
}

__device__ bool SDL::runQuintupletDefaultAlgoBBBB(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, uint16_t& innerInnerLowerModuleIndex, uint16_t& innerOuterLowerModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int& thirdMDIndex,
        unsigned int& fourthMDIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& dPhi, float& betaIn, float&
        betaOut, float& pt_beta, float& zLo, float& zHi, float& zLoPointed, float& zHiPointed, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut)
{
    bool pass = true;

    bool isPS_InLo = (modulesInGPU.moduleType[innerInnerLowerModuleIndex] == SDL::PS);
    bool isPS_OutLo = (modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS);

    float rt_InLo = mdsInGPU.anchorRt[firstMDIndex];
    float rt_InOut = mdsInGPU.anchorRt[secondMDIndex];
    float rt_OutLo = mdsInGPU.anchorRt[thirdMDIndex];

    float z_InLo = mdsInGPU.anchorZ[firstMDIndex];
    float z_InOut = mdsInGPU.anchorZ[secondMDIndex];
    float z_OutLo = mdsInGPU.anchorZ[thirdMDIndex];

    float alpha1GeV_OutLo = asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax));

    float rtRatio_OutLoInLo = rt_OutLo / rt_InLo; // Outer segment beginning rt divided by inner segment beginning rt;
    float dzDrtScale = tanf(alpha1GeV_OutLo) / alpha1GeV_OutLo; // The track can bend in r-z plane slightly
    float zpitch_InLo = (isPS_InLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);
    float zpitch_OutLo = (isPS_OutLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);

    zHi = z_InLo + (z_InLo + SDL::deltaZLum) * (rtRatio_OutLoInLo - 1.f) * (z_InLo < 0.f ? 1.f : dzDrtScale) + (zpitch_InLo + zpitch_OutLo);
    zLo = z_InLo + (z_InLo - SDL::deltaZLum) * (rtRatio_OutLoInLo - 1.f) * (z_InLo > 0.f ? 1.f : dzDrtScale) - (zpitch_InLo + zpitch_OutLo);


    //Cut 1 - z compatibility
    zOut = z_OutLo;
    rtOut = rt_OutLo;
    pass = pass and ((z_OutLo >= zLo) & (z_OutLo <= zHi));
    if(not pass) return pass;

    float drt_OutLo_InLo = (rt_OutLo - rt_InLo);
    float r3_InLo = sqrtf(z_InLo * z_InLo + rt_InLo * rt_InLo);
    float drt_InSeg = rt_InOut - rt_InLo;
    float dz_InSeg = z_InOut - z_InLo;
    float dr3_InSeg = sqrtf(rt_InOut * rt_InOut + z_InOut * z_InOut) - sqrtf(rt_InLo * rt_InLo + z_InLo * z_InLo);

    float coshEta = dr3_InSeg/drt_InSeg;
    float dzErr = (zpitch_InLo + zpitch_OutLo) * (zpitch_InLo + zpitch_OutLo) * 2.f;

    float sdlThetaMulsF = 0.015f * sqrtf(0.1f + 0.2f * (rt_OutLo - rt_InLo) / 50.f) * sqrtf(r3_InLo / rt_InLo);
    float sdlMuls = sdlThetaMulsF * 3.f / SDL::ptCut * 4.f; // will need a better guess than x4?
    dzErr += sdlMuls * sdlMuls * drt_OutLo_InLo * drt_OutLo_InLo / 3.f * coshEta * coshEta; //sloppy
    dzErr = sqrtf(dzErr);

    // Constructing upper and lower bound
    const float dzMean = dz_InSeg / drt_InSeg * drt_OutLo_InLo;
    const float zWindow = dzErr / drt_InSeg * drt_OutLo_InLo + (zpitch_InLo + zpitch_OutLo); //FIXME for SDL::ptCut lower than ~0.8 need to add curv path correction
    zLoPointed = z_InLo + dzMean * (z_InLo > 0.f ? 1.f : dzDrtScale) - zWindow;
    zHiPointed = z_InLo + dzMean * (z_InLo < 0.f ? 1.f : dzDrtScale) + zWindow;

    // Cut #2: Pointed Z (Inner segment two MD points to outer segment inner MD)
    pass =  pass and ((z_OutLo >= zLoPointed) & (z_OutLo <= zHiPointed));
    if(not pass) return pass;

    float sdlPVoff = 0.1f/rt_OutLo;
    sdlCut = alpha1GeV_OutLo + sqrtf(sdlMuls * sdlMuls + sdlPVoff * sdlPVoff);

    deltaPhiPos = SDL::deltaPhi(mdsInGPU.anchorX[secondMDIndex], mdsInGPU.anchorY[secondMDIndex], mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex]);
    // Cut #3: FIXME:deltaPhiPos can be tighter
    pass = pass and (fabsf(deltaPhiPos) <= sdlCut);
    if(not pass) return pass;

    float midPointX = 0.5f*(mdsInGPU.anchorX[firstMDIndex] + mdsInGPU.anchorX[thirdMDIndex]);
    float midPointY = 0.5f* (mdsInGPU.anchorY[firstMDIndex] + mdsInGPU.anchorY[thirdMDIndex]);
    float midPointZ = 0.5f*(mdsInGPU.anchorZ[firstMDIndex] + mdsInGPU.anchorZ[thirdMDIndex]);
    float diffX = mdsInGPU.anchorX[thirdMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float diffY = mdsInGPU.anchorY[thirdMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float diffZ = mdsInGPU.anchorZ[thirdMDIndex] - mdsInGPU.anchorZ[firstMDIndex];

    dPhi = SDL::deltaPhi(midPointX, midPointY, diffX, diffY);

    // Cut #4: deltaPhiChange
    pass = pass and (fabsf(dPhi) <= sdlCut);
    //lots of array accesses below. Cut here!
    if(not pass) return pass;

    // First obtaining the raw betaIn and betaOut values without any correction and just purely based on the mini-doublet hit positions

    float alpha_InLo  = __H2F(segmentsInGPU.dPhiChanges[innerSegmentIndex]);
    float alpha_OutLo = __H2F(segmentsInGPU.dPhiChanges[outerSegmentIndex]);

    bool isEC_lastLayer = modulesInGPU.subdets[outerOuterLowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS;

    float alpha_OutUp,alpha_OutUp_highEdge,alpha_OutUp_lowEdge;

    alpha_OutUp = SDL::deltaPhi(mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex], mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]);

    alpha_OutUp_highEdge = alpha_OutUp;
    alpha_OutUp_lowEdge = alpha_OutUp;

    float tl_axis_x = mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float tl_axis_y = mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float tl_axis_z = mdsInGPU.anchorZ[fourthMDIndex] - mdsInGPU.anchorZ[firstMDIndex];
    float tl_axis_highEdge_x = tl_axis_x;
    float tl_axis_highEdge_y = tl_axis_y;
    float tl_axis_lowEdge_x = tl_axis_x;
    float tl_axis_lowEdge_y = tl_axis_y;

    betaIn = alpha_InLo - SDL::deltaPhi(mdsInGPU.anchorX[firstMDIndex], mdsInGPU.anchorY[firstMDIndex], tl_axis_x, tl_axis_y);

    float betaInRHmin = betaIn;
    float betaInRHmax = betaIn;
    betaOut = -alpha_OutUp + SDL::deltaPhi(mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex], tl_axis_x, tl_axis_y);

    float betaOutRHmin = betaOut;
    float betaOutRHmax = betaOut;

    if(isEC_lastLayer)
    {
        alpha_OutUp_highEdge = SDL::deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], mdsInGPU.anchorHighEdgeX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]);
        alpha_OutUp_lowEdge = SDL::deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], mdsInGPU.anchorLowEdgeX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]);

        tl_axis_highEdge_x = mdsInGPU.anchorHighEdgeX[fourthMDIndex] - mdsInGPU.anchorX[firstMDIndex];
        tl_axis_highEdge_y = mdsInGPU.anchorHighEdgeY[fourthMDIndex] - mdsInGPU.anchorY[firstMDIndex];
        tl_axis_lowEdge_x = mdsInGPU.anchorLowEdgeX[fourthMDIndex] - mdsInGPU.anchorX[firstMDIndex];
        tl_axis_lowEdge_y = mdsInGPU.anchorLowEdgeY[fourthMDIndex] - mdsInGPU.anchorY[firstMDIndex];


        betaOutRHmin = -alpha_OutUp_highEdge + SDL::deltaPhi(mdsInGPU.anchorHighEdgeX[fourthMDIndex], mdsInGPU.anchorHighEdgeY[fourthMDIndex], tl_axis_highEdge_x, tl_axis_highEdge_y);
        betaOutRHmax = -alpha_OutUp_lowEdge + SDL::deltaPhi(mdsInGPU.anchorLowEdgeX[fourthMDIndex], mdsInGPU.anchorLowEdgeY[fourthMDIndex], tl_axis_lowEdge_x, tl_axis_lowEdge_y);
    }

    //beta computation
    float drt_tl_axis = sqrtf(tl_axis_x * tl_axis_x + tl_axis_y * tl_axis_y);
    float drt_tl_lowEdge = sqrtf(tl_axis_lowEdge_x * tl_axis_lowEdge_x + tl_axis_lowEdge_y * tl_axis_lowEdge_y);
    float drt_tl_highEdge = sqrtf(tl_axis_highEdge_x * tl_axis_highEdge_x + tl_axis_highEdge_y * tl_axis_highEdge_y);

    float corrF = 1.f;
    //innerOuterAnchor - innerInnerAnchor
    const float rt_InSeg = sqrtf((mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) * (mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) + (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]) * (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]));
    betaInCut = asinf(fminf((-rt_InSeg * corrF + drt_tl_axis) * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) + (0.02f / drt_InSeg);

    //Cut #5: first beta cut
    pass = pass and (fabsf(betaInRHmin) < betaInCut);
    if(not pass) return pass;

    float betaAv = 0.5f * (betaIn + betaOut);
    pt_beta = drt_tl_axis * SDL::k2Rinv1GeVf/sinf(betaAv);
    int lIn = 5;
    int lOut = isEC_lastLayer ? 11 : 5;
    float sdOut_dr = sqrtf((mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) * (mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) + (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]) * (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]));
    float sdOut_d = mdsInGPU.anchorRt[fourthMDIndex] - mdsInGPU.anchorRt[thirdMDIndex];

    const float diffDr = fabsf(rt_InSeg - sdOut_dr) / fabsf(rt_InSeg + sdOut_dr);

    SDL::runDeltaBetaIterationsT5(betaIn, betaOut, betaAv, pt_beta, rt_InSeg, sdOut_dr, drt_tl_axis, lIn);

    const float betaInMMSF = (fabsf(betaInRHmin + betaInRHmax) > 0) ? (2.f * betaIn / fabsf(betaInRHmin + betaInRHmax)) : 0.f; //mean value of min,max is the old betaIn
    const float betaOutMMSF = (fabsf(betaOutRHmin + betaOutRHmax) > 0) ? (2.f * betaOut / fabsf(betaOutRHmin + betaOutRHmax)) : 0.f;
    betaInRHmin *= betaInMMSF;
    betaInRHmax *= betaInMMSF;
    betaOutRHmin *= betaOutMMSF;
    betaOutRHmax *= betaOutMMSF;

    const float dBetaMuls = sdlThetaMulsF * 4.f / fminf(fabsf(pt_beta), SDL::pt_betaMax); //need to confimm the range-out value of 7 GeV


    const float alphaInAbsReg = fmaxf(fabsf(alpha_InLo), asinf(fminf(rt_InLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float alphaOutAbsReg = fmaxf(fabs(alpha_OutLo), asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float dBetaInLum = lIn < 11 ? 0.0f : fabsf(alphaInAbsReg*SDL::deltaZLum / z_InLo);
    const float dBetaOutLum = lOut < 11 ? 0.0f : fabsf(alphaOutAbsReg*SDL::deltaZLum / z_OutLo);
    const float dBetaLum2 = (dBetaInLum + dBetaOutLum) * (dBetaInLum + dBetaOutLum);
    const float sinDPhi = sinf(dPhi);

    const float dBetaRIn2 = 0; // TODO-RH
    // const float dBetaROut2 = 0; // TODO-RH
    float dBetaROut = 0;
    if(isEC_lastLayer)
    {
        dBetaROut = (sqrtf(mdsInGPU.anchorHighEdgeX[fourthMDIndex] * mdsInGPU.anchorHighEdgeX[fourthMDIndex] + mdsInGPU.anchorHighEdgeY[fourthMDIndex] * mdsInGPU.anchorHighEdgeY[fourthMDIndex]) - sqrtf(mdsInGPU.anchorLowEdgeX[fourthMDIndex] * mdsInGPU.anchorLowEdgeX[fourthMDIndex] + mdsInGPU.anchorLowEdgeY[fourthMDIndex] * mdsInGPU.anchorLowEdgeY[fourthMDIndex])) * sinDPhi / drt_tl_axis;
    }

    const float dBetaROut2 =  dBetaROut * dBetaROut;

    betaOutCut = asinf(fminf(drt_tl_axis*SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) //FIXME: need faster version
        + (0.02f / sdOut_d) + sqrtf(dBetaLum2 + dBetaMuls*dBetaMuls);

    //Cut #6: The real beta cut
    pass =  pass and ((fabsf(betaOut) < betaOutCut));
    if(not pass) return pass;

    float pt_betaIn = drt_tl_axis * SDL::k2Rinv1GeVf/sinf(betaIn);
    float pt_betaOut = drt_tl_axis * SDL::k2Rinv1GeVf / sinf(betaOut);
    float dBetaRes = 0.02f/fminf(sdOut_d,drt_InSeg);
    float dBetaCut2 = (dBetaRes*dBetaRes * 2.0f + dBetaMuls * dBetaMuls + dBetaLum2 + dBetaRIn2 + dBetaROut2
            + 0.25f * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)) * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)));

    float dBeta = betaIn - betaOut;
    deltaBetaCut = sqrtf(dBetaCut2);
    pass = pass and (dBeta * dBeta <= dBetaCut2);

    return pass;
}

__device__ bool SDL::runQuintupletDefaultAlgoBBEE(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, uint16_t& innerInnerLowerModuleIndex, uint16_t& innerOuterLowerModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int& thirdMDIndex,
        unsigned int& fourthMDIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& dPhi, float& betaIn, float&
        betaOut, float& pt_beta, float& zLo, float& rtLo, float& rtHi, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ)
{
    bool pass = true;
    bool isPS_InLo = (modulesInGPU.moduleType[innerInnerLowerModuleIndex] == SDL::PS);
    bool isPS_OutLo = (modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS);

    float rt_InLo = mdsInGPU.anchorRt[firstMDIndex];
    float rt_InOut = mdsInGPU.anchorRt[secondMDIndex];
    float rt_OutLo = mdsInGPU.anchorRt[thirdMDIndex];

    float z_InLo = mdsInGPU.anchorZ[firstMDIndex];
    float z_InOut = mdsInGPU.anchorZ[secondMDIndex];
    float z_OutLo = mdsInGPU.anchorZ[thirdMDIndex];

    float alpha1GeV_OutLo = asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax));

    float rtRatio_OutLoInLo = rt_OutLo / rt_InLo; // Outer segment beginning rt divided by inner segment beginning rt;
    float dzDrtScale = tanf(alpha1GeV_OutLo) / alpha1GeV_OutLo; // The track can bend in r-z plane slightly
    float zpitch_InLo = (isPS_InLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);
    float zpitch_OutLo = (isPS_OutLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);
    float zGeom = zpitch_InLo + zpitch_OutLo;

    zLo = z_InLo + (z_InLo - SDL::deltaZLum) * (rtRatio_OutLoInLo - 1.f) * (z_InLo > 0.f ? 1.f : dzDrtScale) - zGeom;

    // Cut #0: Preliminary (Only here in endcap case)
    pass = pass and (z_InLo * z_OutLo > 0);
    if(not pass) return pass;

    float dLum = copysignf(SDL::deltaZLum, z_InLo);
    bool isOutSgInnerMDPS = modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS;
    float rtGeom1 = isOutSgInnerMDPS ? SDL::pixelPSZpitch : SDL::strip2SZpitch;
    float zGeom1 = copysignf(zGeom,z_InLo);
    rtLo = rt_InLo * (1.f + (z_OutLo - z_InLo - zGeom1) / (z_InLo + zGeom1 + dLum) / dzDrtScale) - rtGeom1; //slope correction only on the lower end
    zOut = z_OutLo;
    rtOut = rt_OutLo;

    //Cut #1: rt condition
    pass =  pass and (rtOut >= rtLo);
    if(not pass) return pass;

    float zInForHi = z_InLo - zGeom1 - dLum;
    if(zInForHi * z_InLo < 0)
    {
        zInForHi = copysignf(0.1f,z_InLo);
    }
    rtHi = rt_InLo * (1.f + (z_OutLo - z_InLo + zGeom1) / zInForHi) + rtGeom1;

    //Cut #2: rt condition
    pass =  pass and ((rt_OutLo >= rtLo) & (rt_OutLo <= rtHi));
    if(not pass) return pass;

    float rIn = sqrtf(z_InLo * z_InLo + rt_InLo * rt_InLo);
    const float drtSDIn = rt_InOut - rt_InLo;
    const float dzSDIn = z_InOut - z_InLo;
    const float dr3SDIn = sqrtf(rt_InOut * rt_InOut + z_InOut * z_InOut) - sqrtf(rt_InLo * rt_InLo + z_InLo * z_InLo);

    const float coshEta = dr3SDIn / drtSDIn; //direction estimate
    const float dzOutInAbs = fabsf(z_OutLo - z_InLo);
    const float multDzDr = dzOutInAbs * coshEta / (coshEta * coshEta - 1.f);
    const float zGeom1_another = SDL::pixelPSZpitch; //What's this?
    kZ = (z_OutLo - z_InLo) / dzSDIn;
    float drtErr = zGeom1_another * zGeom1_another * drtSDIn * drtSDIn / dzSDIn / dzSDIn * (1.f - 2.f * kZ + 2.f * kZ * kZ); //Notes:122316
    const float sdlThetaMulsF = 0.015f * sqrtf(0.1f + 0.2f * (rt_OutLo - rt_InLo) / 50.f) * sqrtf(rIn / rt_InLo);
    const float sdlMuls = sdlThetaMulsF * 3.f / SDL::ptCut * 4.f; //will need a better guess than x4?
    drtErr += sdlMuls * sdlMuls * multDzDr * multDzDr / 3.f * coshEta * coshEta; //sloppy: relative muls is 1/3 of total muls
    drtErr = sqrtf(drtErr);
    const float drtMean = drtSDIn * dzOutInAbs / fabsf(dzSDIn); //
    const float rtWindow = drtErr + rtGeom1;
    const float rtLo_another = rt_InLo + drtMean / dzDrtScale - rtWindow;
    const float rtHi_another = rt_InLo + drtMean + rtWindow;

    //Cut #3: rt-z pointed
    pass =  pass and ((kZ >= 0) & (rtOut >= rtLo) & (rtOut <= rtHi));
    if(not pass) return pass;

    const float sdlPVoff = 0.1f / rt_OutLo;
    sdlCut = alpha1GeV_OutLo + sqrtf(sdlMuls * sdlMuls + sdlPVoff*sdlPVoff);


    deltaPhiPos = SDL::deltaPhi(mdsInGPU.anchorX[secondMDIndex], mdsInGPU.anchorY[secondMDIndex], mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex]);


    //Cut #4: deltaPhiPos can be tighter
    pass =  pass and (fabsf(deltaPhiPos) <= sdlCut);
    if(not pass) return pass;

    float midPointX = 0.5f*(mdsInGPU.anchorX[firstMDIndex] + mdsInGPU.anchorX[thirdMDIndex]);
    float midPointY = 0.5f* (mdsInGPU.anchorY[firstMDIndex] + mdsInGPU.anchorY[thirdMDIndex]);
    float midPointZ = 0.5f*(mdsInGPU.anchorZ[firstMDIndex] + mdsInGPU.anchorZ[thirdMDIndex]);
    float diffX = mdsInGPU.anchorX[thirdMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float diffY = mdsInGPU.anchorY[thirdMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float diffZ = mdsInGPU.anchorZ[thirdMDIndex] - mdsInGPU.anchorZ[firstMDIndex];

    dPhi = SDL::deltaPhi(midPointX, midPointY, diffX, diffY);
    // Cut #5: deltaPhiChange
    pass =  pass and (fabsf(dPhi) <= sdlCut);
    if(not pass) return pass;

    float sdIn_alpha     = __H2F(segmentsInGPU.dPhiChanges[innerSegmentIndex]);
    float sdIn_alpha_min = __H2F(segmentsInGPU.dPhiChangeMins[innerSegmentIndex]);
    float sdIn_alpha_max = __H2F(segmentsInGPU.dPhiChangeMaxs[innerSegmentIndex]);
    float sdOut_alpha = sdIn_alpha; //weird

    float sdOut_alphaOut = SDL::deltaPhi(mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex], mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]);

    float sdOut_alphaOut_min = SDL::phi_mpi_pi(__H2F(segmentsInGPU.dPhiChangeMins[outerSegmentIndex]) - __H2F(segmentsInGPU.dPhiMins[outerSegmentIndex]));
    float sdOut_alphaOut_max = SDL::phi_mpi_pi(__H2F(segmentsInGPU.dPhiChangeMaxs[outerSegmentIndex]) - __H2F(segmentsInGPU.dPhiMaxs[outerSegmentIndex]));

    float tl_axis_x = mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float tl_axis_y = mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float tl_axis_z = mdsInGPU.anchorZ[fourthMDIndex] - mdsInGPU.anchorZ[firstMDIndex];

    betaIn = sdIn_alpha - SDL::deltaPhi(mdsInGPU.anchorX[firstMDIndex], mdsInGPU.anchorY[firstMDIndex], tl_axis_x, tl_axis_y);

    float betaInRHmin = betaIn;
    float betaInRHmax = betaIn;
    betaOut = -sdOut_alphaOut + SDL::deltaPhi(mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex], tl_axis_x, tl_axis_y);

    float betaOutRHmin = betaOut;
    float betaOutRHmax = betaOut;

    bool isEC_secondLayer = (modulesInGPU.subdets[innerOuterLowerModuleIndex] == SDL::Endcap) and (modulesInGPU.moduleType[innerOuterLowerModuleIndex] == SDL::TwoS);

    if(isEC_secondLayer)
    {
        betaInRHmin = betaIn - sdIn_alpha_min + sdIn_alpha;
        betaInRHmax = betaIn - sdIn_alpha_max + sdIn_alpha;
    }

    betaOutRHmin = betaOut - sdOut_alphaOut_min + sdOut_alphaOut;
    betaOutRHmax = betaOut - sdOut_alphaOut_max + sdOut_alphaOut;

    float swapTemp;
    if(fabsf(betaOutRHmin) > fabsf(betaOutRHmax))
    {
        swapTemp = betaOutRHmin;
        betaOutRHmin = betaOutRHmax;
        betaOutRHmax = swapTemp;
    }

    if(fabsf(betaInRHmin) > fabsf(betaInRHmax))
    {
        swapTemp = betaInRHmin;
        betaInRHmin = betaInRHmax;
        betaInRHmax = swapTemp;
    }

    float sdIn_dr = sqrtf((mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) * (mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) + (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]) * (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]));
    float sdIn_d = rt_InOut - rt_InLo;

    float dr = sqrtf(tl_axis_x * tl_axis_x + tl_axis_y * tl_axis_y);
    const float corrF = 1.f;
    betaInCut = asinf(fminf((-sdIn_dr * corrF + dr) * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) + (0.02f / sdIn_d);

    //Cut #6: first beta cut
    pass =  pass and (fabsf(betaInRHmin) < betaInCut);
    if(not pass) return pass;

    float betaAv = 0.5f * (betaIn + betaOut);
    pt_beta = dr * SDL::k2Rinv1GeVf / sinf(betaAv);

    float lIn = 5;
    float lOut = 11;

    float sdOut_dr = sqrtf((mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) * (mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) + (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]) * (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]));
    float sdOut_d = mdsInGPU.anchorRt[fourthMDIndex] - mdsInGPU.anchorRt[thirdMDIndex];

    SDL::runDeltaBetaIterationsT5(betaIn, betaOut, betaAv, pt_beta, sdIn_dr, sdOut_dr, dr, lIn);

     const float betaInMMSF = (fabsf(betaInRHmin + betaInRHmax) > 0) ? (2.f * betaIn / fabsf(betaInRHmin + betaInRHmax)) : 0.; //mean value of min,max is the old betaIn
    const float betaOutMMSF = (fabsf(betaOutRHmin + betaOutRHmax) > 0) ? (2.f * betaOut / fabsf(betaOutRHmin + betaOutRHmax)) : 0.;
    betaInRHmin *= betaInMMSF;
    betaInRHmax *= betaInMMSF;
    betaOutRHmin *= betaOutMMSF;
    betaOutRHmax *= betaOutMMSF;

    const float dBetaMuls = sdlThetaMulsF * 4.f / fminf(fabsf(pt_beta), SDL::pt_betaMax); //need to confirm the range-out value of 7 GeV

    const float alphaInAbsReg = fmaxf(fabsf(sdIn_alpha), asinf(fminf(rt_InLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float alphaOutAbsReg = fmaxf(fabsf(sdOut_alpha), asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float dBetaInLum = lIn < 11 ? 0.0f : fabsf(alphaInAbsReg*SDL::deltaZLum / z_InLo);
    const float dBetaOutLum = lOut < 11 ? 0.0f : fabsf(alphaOutAbsReg*SDL::deltaZLum / z_OutLo);
    const float dBetaLum2 = (dBetaInLum + dBetaOutLum) * (dBetaInLum + dBetaOutLum);
    const float sinDPhi = sinf(dPhi);

    const float dBetaRIn2 = 0; // TODO-RH
    // const float dBetaROut2 = 0; // TODO-RH
    float dBetaROut = 0;
    if(modulesInGPU.moduleType[outerOuterLowerModuleIndex] == SDL::TwoS)
    {
        dBetaROut = (sqrtf(mdsInGPU.anchorHighEdgeX[fourthMDIndex] * mdsInGPU.anchorHighEdgeX[fourthMDIndex] + mdsInGPU.anchorHighEdgeY[fourthMDIndex] * mdsInGPU.anchorHighEdgeY[fourthMDIndex]) - sqrtf(mdsInGPU.anchorLowEdgeX[fourthMDIndex] * mdsInGPU.anchorLowEdgeX[fourthMDIndex] + mdsInGPU.anchorLowEdgeY[fourthMDIndex] * mdsInGPU.anchorLowEdgeY[fourthMDIndex])) * sinDPhi / dr;
    }

    const float dBetaROut2 = dBetaROut * dBetaROut;
    betaOutCut = asinf(fminf(dr*SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) //FIXME: need faster version
        + (0.02f / sdOut_d) + sqrtf(dBetaLum2 + dBetaMuls*dBetaMuls);

    //Cut #6: The real beta cut
    pass =  pass and (fabsf(betaOut) < betaOutCut);
    if(not pass) return pass;

    float pt_betaIn = dr * SDL::k2Rinv1GeVf/sinf(betaIn);
    float pt_betaOut = dr * SDL::k2Rinv1GeVf / sinf(betaOut);
    float dBetaRes = 0.02f/fminf(sdOut_d,sdIn_d);
    float dBetaCut2 = (dBetaRes*dBetaRes * 2.0f + dBetaMuls * dBetaMuls + dBetaLum2 + dBetaRIn2 + dBetaROut2
            + 0.25f * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)) * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)));
    float dBeta = betaIn - betaOut;
    deltaBetaCut = sqrtf(dBetaCut2);
    //Cut #7: Cut on dBet
    pass =  pass and (dBeta * dBeta <= dBetaCut2);

    return pass;
}

__device__ bool SDL::runQuintupletDefaultAlgoEEEE(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, uint16_t& innerInnerLowerModuleIndex, uint16_t& innerOuterLowerModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int& thirdMDIndex,
        unsigned int& fourthMDIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& dPhi, float& betaIn, float&
        betaOut, float& pt_beta, float& zLo, float& rtLo, float& rtHi, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ)
{
    bool pass = true;

    bool isPS_InLo = (modulesInGPU.moduleType[innerInnerLowerModuleIndex] == SDL::PS);
    bool isPS_OutLo = (modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS);

    float rt_InLo = mdsInGPU.anchorRt[firstMDIndex];
    float rt_InOut = mdsInGPU.anchorRt[secondMDIndex];
    float rt_OutLo = mdsInGPU.anchorRt[thirdMDIndex];

    float z_InLo = mdsInGPU.anchorZ[firstMDIndex];
    float z_InOut = mdsInGPU.anchorZ[secondMDIndex];
    float z_OutLo = mdsInGPU.anchorZ[thirdMDIndex];

    float alpha1GeV_OutLo = asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax));

    float rtRatio_OutLoInLo = rt_OutLo / rt_InLo; // Outer segment beginning rt divided by inner segment beginning rt;
    float dzDrtScale = tanf(alpha1GeV_OutLo) / alpha1GeV_OutLo; // The track can bend in r-z plane slightly
    float zpitch_InLo = (isPS_InLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);
    float zpitch_OutLo = (isPS_OutLo ? SDL::pixelPSZpitch : SDL::strip2SZpitch);
    float zGeom = zpitch_InLo + zpitch_OutLo;

    zLo = z_InLo + (z_InLo - SDL::deltaZLum) * (rtRatio_OutLoInLo - 1.f) * (z_InLo > 0.f ? 1.f : dzDrtScale) - zGeom; //slope-correction only on outer end

    // Cut #0: Preliminary (Only here in endcap case)
    pass =  pass and ((z_InLo * z_OutLo) > 0);
    if(not pass) return pass;

    float dLum = copysignf(SDL::deltaZLum, z_InLo);
    bool isOutSgInnerMDPS = modulesInGPU.moduleType[outerInnerLowerModuleIndex] == SDL::PS;
    bool isInSgInnerMDPS = modulesInGPU.moduleType[innerInnerLowerModuleIndex] == SDL::PS;

    float rtGeom = (isInSgInnerMDPS and isOutSgInnerMDPS) ? 2.f * SDL::pixelPSZpitch : (isInSgInnerMDPS or isOutSgInnerMDPS) ? SDL::pixelPSZpitch + SDL::strip2SZpitch : 2.f * SDL::strip2SZpitch;

    float zGeom1 = copysignf(zGeom,z_InLo);
    float dz = z_OutLo - z_InLo;
    rtLo = rt_InLo * (1.f + dz / (z_InLo + dLum) / dzDrtScale) - rtGeom; //slope correction only on the lower end

    zOut = z_OutLo;
    rtOut = rt_OutLo;

    //Cut #1: rt condition

    rtHi = rt_InLo * (1.f + dz / (z_InLo - dLum)) + rtGeom;

    pass =  pass and ((rtOut >= rtLo) & (rtOut <= rtHi));
    if(not pass) return pass;

    bool isInSgOuterMDPS = modulesInGPU.moduleType[innerOuterLowerModuleIndex] == SDL::PS;

    float drOutIn = rtOut - rt_InLo;
    const float drtSDIn = rt_InOut - rt_InLo;
    const float dzSDIn = z_InOut - z_InLo;
    const float dr3SDIn = sqrtf(rt_InOut * rt_InOut + z_InOut * z_InOut) - sqrtf(rt_InLo * rt_InLo + z_InLo * z_InLo);
    float coshEta = dr3SDIn / drtSDIn; //direction estimate
    float dzOutInAbs =  fabsf(z_OutLo - z_InLo);
    float multDzDr = dzOutInAbs * coshEta / (coshEta * coshEta - 1.f);

    kZ = (z_OutLo - z_InLo) / dzSDIn;
    float sdlThetaMulsF = 0.015f * sqrtf(0.1f + 0.2f * (rt_OutLo - rt_InLo) / 50.f);

    float sdlMuls = sdlThetaMulsF * 3.f / SDL::ptCut * 4.f; //will need a better guess than x4?

    float drtErr = sqrtf(SDL::pixelPSZpitch * SDL::pixelPSZpitch * 2.f / (dzSDIn * dzSDIn) * (dzOutInAbs * dzOutInAbs) + sdlMuls * sdlMuls * multDzDr * multDzDr / 3.f * coshEta * coshEta);

    float drtMean = drtSDIn * dzOutInAbs/fabsf(dzSDIn);
    float rtWindow = drtErr + rtGeom;
    float rtLo_point = rt_InLo + drtMean / dzDrtScale - rtWindow;
    float rtHi_point = rt_InLo + drtMean + rtWindow;

    // Cut #3: rt-z pointed
    // https://github.com/slava77/cms-tkph2-ntuple/blob/superDoubletLinked-91X-noMock/doubletAnalysis.C#L3765

    if (isInSgInnerMDPS and isInSgOuterMDPS) // If both PS then we can point
    {
        pass =  pass and (kZ >= 0 and rtOut >= rtLo_point and rtOut <= rtHi_point);
        if(not pass) return pass;
    }

    float sdlPVoff = 0.1f/rtOut;
    sdlCut = alpha1GeV_OutLo + sqrtf(sdlMuls * sdlMuls + sdlPVoff * sdlPVoff);

    deltaPhiPos = SDL::deltaPhi(mdsInGPU.anchorX[secondMDIndex], mdsInGPU.anchorY[secondMDIndex], mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex]);

    pass =  pass and (fabsf(deltaPhiPos) <= sdlCut);
    if(not pass) return pass;

    float midPointX = 0.5f*(mdsInGPU.anchorX[firstMDIndex] + mdsInGPU.anchorX[thirdMDIndex]);
    float midPointY = 0.5f* (mdsInGPU.anchorY[firstMDIndex] + mdsInGPU.anchorY[thirdMDIndex]);
    float midPointZ = 0.5f*(mdsInGPU.anchorZ[firstMDIndex] + mdsInGPU.anchorZ[thirdMDIndex]);
    float diffX = mdsInGPU.anchorX[thirdMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float diffY = mdsInGPU.anchorY[thirdMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float diffZ = mdsInGPU.anchorZ[thirdMDIndex] - mdsInGPU.anchorZ[firstMDIndex];

    dPhi = SDL::deltaPhi(midPointX, midPointY, diffX, diffY);

    // Cut #5: deltaPhiChange
    pass =  pass and ((fabsf(dPhi) <= sdlCut));
    if(not pass) return pass;

    float sdIn_alpha = __H2F(segmentsInGPU.dPhiChanges[innerSegmentIndex]);
    float sdOut_alpha = sdIn_alpha; //weird
    float sdOut_dPhiPos = SDL::deltaPhi(mdsInGPU.anchorX[thirdMDIndex], mdsInGPU.anchorY[thirdMDIndex], mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex]);

    float sdOut_dPhiChange = __H2F(segmentsInGPU.dPhiChanges[outerSegmentIndex]);
    float sdOut_dPhiChange_min = __H2F(segmentsInGPU.dPhiChangeMins[outerSegmentIndex]);
    float sdOut_dPhiChange_max = __H2F(segmentsInGPU.dPhiChangeMaxs[outerSegmentIndex]);

    float sdOut_alphaOutRHmin = SDL::phi_mpi_pi(sdOut_dPhiChange_min - sdOut_dPhiPos);
    float sdOut_alphaOutRHmax = SDL::phi_mpi_pi(sdOut_dPhiChange_max - sdOut_dPhiPos);
    float sdOut_alphaOut = SDL::phi_mpi_pi(sdOut_dPhiChange - sdOut_dPhiPos);

    float tl_axis_x = mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[firstMDIndex];
    float tl_axis_y = mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[firstMDIndex];
    float tl_axis_z = mdsInGPU.anchorZ[fourthMDIndex] - mdsInGPU.anchorZ[firstMDIndex];

    betaIn = sdIn_alpha - SDL::deltaPhi(mdsInGPU.anchorX[firstMDIndex], mdsInGPU.anchorY[firstMDIndex], tl_axis_x, tl_axis_y);

    float sdIn_alphaRHmin = __H2F(segmentsInGPU.dPhiChangeMins[innerSegmentIndex]);
    float sdIn_alphaRHmax = __H2F(segmentsInGPU.dPhiChangeMaxs[innerSegmentIndex]);
    float betaInRHmin = betaIn + sdIn_alphaRHmin - sdIn_alpha;
    float betaInRHmax = betaIn + sdIn_alphaRHmax - sdIn_alpha;

    betaOut = -sdOut_alphaOut + SDL::deltaPhi(mdsInGPU.anchorX[fourthMDIndex], mdsInGPU.anchorY[fourthMDIndex], tl_axis_x, tl_axis_y);

    float betaOutRHmin = betaOut - sdOut_alphaOutRHmin + sdOut_alphaOut;
    float betaOutRHmax = betaOut - sdOut_alphaOutRHmax + sdOut_alphaOut;

    float swapTemp;
    if(fabsf(betaOutRHmin) > fabsf(betaOutRHmax))
    {
        swapTemp = betaOutRHmin;
        betaOutRHmin = betaOutRHmax;
        betaOutRHmax = swapTemp;
    }

    if(fabsf(betaInRHmin) > fabsf(betaInRHmax))
    {
        swapTemp = betaInRHmin;
        betaInRHmin = betaInRHmax;
        betaInRHmax = swapTemp;
    }
    float sdIn_dr = sqrtf((mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) * (mdsInGPU.anchorX[secondMDIndex] - mdsInGPU.anchorX[firstMDIndex]) + (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]) * (mdsInGPU.anchorY[secondMDIndex] - mdsInGPU.anchorY[firstMDIndex]));
    float sdIn_d = rt_InOut - rt_InLo;

    float dr = sqrtf(tl_axis_x * tl_axis_x + tl_axis_y * tl_axis_y);
    const float corrF = 1.f;
    betaInCut = asinf(fminf((-sdIn_dr * corrF + dr) * SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) + (0.02f / sdIn_d);

    //Cut #6: first beta cut
    pass =  pass and (fabsf(betaInRHmin) < betaInCut);
    if(not pass) return pass;

    float betaAv = 0.5f * (betaIn + betaOut);
    pt_beta = dr * SDL::k2Rinv1GeVf / sinf(betaAv);


    int lIn= 11; //endcap
    int lOut = 13; //endcap

    float sdOut_dr = sqrtf((mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) * (mdsInGPU.anchorX[fourthMDIndex] - mdsInGPU.anchorX[thirdMDIndex]) + (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]) * (mdsInGPU.anchorY[fourthMDIndex] - mdsInGPU.anchorY[thirdMDIndex]));
    float sdOut_d = mdsInGPU.anchorRt[fourthMDIndex] - mdsInGPU.anchorRt[thirdMDIndex];

    float diffDr = fabsf(sdIn_dr - sdOut_dr)/fabs(sdIn_dr + sdOut_dr);

    SDL::runDeltaBetaIterationsT5(betaIn, betaOut, betaAv, pt_beta, sdIn_dr, sdOut_dr, dr, lIn);

     const float betaInMMSF = (fabsf(betaInRHmin + betaInRHmax) > 0) ? (2.f * betaIn / fabsf(betaInRHmin + betaInRHmax)) : 0.; //mean value of min,max is the old betaIn
    const float betaOutMMSF = (fabsf(betaOutRHmin + betaOutRHmax) > 0) ? (2.f * betaOut / fabsf(betaOutRHmin + betaOutRHmax)) : 0.;
    betaInRHmin *= betaInMMSF;
    betaInRHmax *= betaInMMSF;
    betaOutRHmin *= betaOutMMSF;
    betaOutRHmax *= betaOutMMSF;

    const float dBetaMuls = sdlThetaMulsF * 4.f / fminf(fabsf(pt_beta), SDL::pt_betaMax); //need to confirm the range-out value of 7 GeV

    const float alphaInAbsReg = fmaxf(fabsf(sdIn_alpha), asinf(fminf(rt_InLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float alphaOutAbsReg = fmaxf(fabsf(sdOut_alpha), asinf(fminf(rt_OutLo * SDL::k2Rinv1GeVf / 3.0f, SDL::sinAlphaMax)));
    const float dBetaInLum = lIn < 11 ? 0.0f : fabsf(alphaInAbsReg*SDL::deltaZLum / z_InLo);
    const float dBetaOutLum = lOut < 11 ? 0.0f : fabsf(alphaOutAbsReg*SDL::deltaZLum / z_OutLo);
    const float dBetaLum2 = (dBetaInLum + dBetaOutLum) * (dBetaInLum + dBetaOutLum);
    const float sinDPhi = sinf(dPhi);

    const float dBetaRIn2 = 0; // TODO-RH
    // const float dBetaROut2 = 0; // TODO-RH
    float dBetaROut2 = 0;//TODO-RH
    betaOutCut = asinf(fminf(dr*SDL::k2Rinv1GeVf / SDL::ptCut, SDL::sinAlphaMax)) //FIXME: need faster version
        + (0.02f / sdOut_d) + sqrtf(dBetaLum2 + dBetaMuls*dBetaMuls);

    //Cut #6: The real beta cut
    pass =  pass and (fabsf(betaOut) < betaOutCut);
    if(not pass) return pass;

    float pt_betaIn = dr * SDL::k2Rinv1GeVf/sinf(betaIn);
    float pt_betaOut = dr * SDL::k2Rinv1GeVf / sinf(betaOut);
    float dBetaRes = 0.02f/fminf(sdOut_d,sdIn_d);
    float dBetaCut2 = (dBetaRes*dBetaRes * 2.0f + dBetaMuls * dBetaMuls + dBetaLum2 + dBetaRIn2 + dBetaROut2
            + 0.25f * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)) * (fabsf(betaInRHmin - betaInRHmax) + fabsf(betaOutRHmin - betaOutRHmax)));
    float dBeta = betaIn - betaOut;
    //Cut #7: Cut on dBeta
    deltaBetaCut = sqrtf(dBetaCut2);

    pass =  pass and (dBeta * dBeta <= dBetaCut2);

    return pass;
}
__device__ bool SDL::runQuintupletDefaultAlgo(struct SDL::modules& modulesInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, uint16_t& innerInnerLowerModuleIndex, uint16_t& innerOuterLowerModuleIndex, uint16_t& outerInnerLowerModuleIndex, uint16_t& outerOuterLowerModuleIndex, unsigned int& innerSegmentIndex, unsigned int& outerSegmentIndex, unsigned int& firstMDIndex, unsigned int& secondMDIndex, unsigned int& thirdMDIndex, unsigned int& fourthMDIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& deltaPhi, float& betaIn, float&
        betaOut, float& pt_beta, float& zLo, float& zHi, float& rtLo, float& rtHi, float& zLoPointed, float& zHiPointed, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ)
{

    bool pass = false;

    zLo = -999;
    zHi = -999;
    rtLo = -999;
    rtHi = -999;
    zLoPointed = -999;
    zHiPointed = -999;
    kZ = -999;
    betaInCut = -999;

    short innerInnerLowerModuleSubdet = modulesInGPU.subdets[innerInnerLowerModuleIndex];
    short innerOuterLowerModuleSubdet = modulesInGPU.subdets[innerOuterLowerModuleIndex];
    short outerInnerLowerModuleSubdet = modulesInGPU.subdets[outerInnerLowerModuleIndex];
    short outerOuterLowerModuleSubdet = modulesInGPU.subdets[outerOuterLowerModuleIndex];

    if(innerInnerLowerModuleSubdet == SDL::Barrel
            and innerOuterLowerModuleSubdet == SDL::Barrel
            and outerInnerLowerModuleSubdet == SDL::Barrel
            and outerOuterLowerModuleSubdet == SDL::Barrel)
    {
        return runQuintupletDefaultAlgoBBBB(modulesInGPU,mdsInGPU,segmentsInGPU,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,innerSegmentIndex,outerSegmentIndex,firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, zHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut);
    }

    else if(innerInnerLowerModuleSubdet == SDL::Barrel
            and innerOuterLowerModuleSubdet == SDL::Barrel
            and outerInnerLowerModuleSubdet == SDL::Endcap
            and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
       return runQuintupletDefaultAlgoBBEE(modulesInGPU,mdsInGPU,segmentsInGPU,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,innerSegmentIndex,outerSegmentIndex,firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, rtLo, rtHi, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    }


    else if(innerInnerLowerModuleSubdet == SDL::Barrel
            and innerOuterLowerModuleSubdet == SDL::Barrel
            and outerInnerLowerModuleSubdet == SDL::Barrel
            and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
        return runQuintupletDefaultAlgoBBBB(modulesInGPU,mdsInGPU,segmentsInGPU,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,innerSegmentIndex,outerSegmentIndex,firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta,zLo, zHi, zLoPointed, zHiPointed, sdlCut, betaInCut, betaOutCut, deltaBetaCut);

    }

    else if(innerInnerLowerModuleSubdet == SDL::Barrel
            and innerOuterLowerModuleSubdet == SDL::Endcap
            and outerInnerLowerModuleSubdet == SDL::Endcap
            and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
        return runQuintupletDefaultAlgoBBEE(modulesInGPU,mdsInGPU,segmentsInGPU,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,innerSegmentIndex,outerSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, rtLo, rtHi, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);

    }

    else if(innerInnerLowerModuleSubdet == SDL::Endcap
            and innerOuterLowerModuleSubdet == SDL::Endcap
            and outerInnerLowerModuleSubdet == SDL::Endcap
            and outerOuterLowerModuleSubdet == SDL::Endcap)
    {
        return runQuintupletDefaultAlgoEEEE(modulesInGPU,mdsInGPU,segmentsInGPU,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,innerSegmentIndex,outerSegmentIndex, firstMDIndex, secondMDIndex, thirdMDIndex, fourthMDIndex, zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,pt_beta, zLo, rtLo, rtHi, sdlCut, betaInCut, betaOutCut, deltaBetaCut, kZ);
    }

    return pass;
}
__device__ void SDL::runDeltaBetaIterationsT5(float& betaIn, float& betaOut, float& betaAv, float & pt_beta, float sdIn_dr, float sdOut_dr, float dr, float lIn)
{
    if (lIn == 0)
    {
        betaOut += copysign(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaOut);
        return;
    }

    if (betaIn * betaOut > 0.f and (fabsf(pt_beta) < 4.f * SDL::pt_betaMax or (lIn >= 11 and fabsf(pt_beta) < 8.f * SDL::pt_betaMax)))   //and the pt_beta is well-defined; less strict for endcap-endcap
    {

        const float betaInUpd  = betaIn + copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        const float betaOutUpd = betaOut + copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaOut); //FIXME: need a faster version
        betaAv = 0.5f * (betaInUpd + betaOutUpd);

        //1st update
        //pt_beta = dr * k2Rinv1GeVf / sinf(betaAv); //get a better pt estimate
        const float pt_beta_inv = 1.f/fabsf(dr * k2Rinv1GeVf / sinf(betaAv)); //get a better pt estimate

        betaIn  += copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf *pt_beta_inv, SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaOut += copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf *pt_beta_inv, SDL::sinAlphaMax)), betaOut); //FIXME: need a faster version
        //update the av and pt
        betaAv = 0.5f * (betaIn + betaOut);
        //2nd update
        pt_beta = dr * SDL::k2Rinv1GeVf / sinf(betaAv); //get a better pt estimate
    }
    else if (lIn < 11 && fabsf(betaOut) < 0.2f * fabsf(betaIn) && fabsf(pt_beta) < 12.f * SDL::pt_betaMax)   //use betaIn sign as ref
    {

        const float pt_betaIn = dr * k2Rinv1GeVf / sinf(betaIn);

        const float betaInUpd  = betaIn + copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_betaIn), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        const float betaOutUpd = betaOut + copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_betaIn), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaAv = (fabsf(betaOut) > 0.2f * fabsf(betaIn)) ? (0.5f * (betaInUpd + betaOutUpd)) : betaInUpd;

        //1st update
        pt_beta = dr * SDL::k2Rinv1GeVf / sin(betaAv); //get a better pt estimate
        betaIn  += copysignf(asinf(fminf(sdIn_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        betaOut += copysignf(asinf(fminf(sdOut_dr * SDL::k2Rinv1GeVf / fabsf(pt_beta), SDL::sinAlphaMax)), betaIn); //FIXME: need a faster version
        //update the av and pt
        betaAv = 0.5f * (betaIn + betaOut);
        //2nd update
        pt_beta = dr * SDL::k2Rinv1GeVf / sin(betaAv); //get a better pt estimate

    }
}
