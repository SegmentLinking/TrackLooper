#include "TrackCandidate.cuh"

void SDL::trackCandidates::resetMemory(unsigned int maxTrackCandidates,hipStream_t stream)
{
    hipMemsetAsync(trackCandidateType,0, maxTrackCandidates * sizeof(short),stream);
    hipMemsetAsync(directObjectIndices, 0, maxTrackCandidates * sizeof(unsigned int),stream);
    hipMemsetAsync(objectIndices, 0,2 * maxTrackCandidates * sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidates, 0,sizeof(int),stream);
    hipMemsetAsync(nTrackCandidatespT3, 0,sizeof(int),stream);
    hipMemsetAsync(nTrackCandidatesT5, 0,sizeof(int),stream);
    hipMemsetAsync(nTrackCandidatespT5,0, sizeof(int),stream);
    hipMemsetAsync(nTrackCandidatespLS, 0,sizeof(int),stream);

    hipMemsetAsync(logicalLayers, 0, 7 * maxTrackCandidates * sizeof(uint8_t), stream);
    hipMemsetAsync(lowerModuleIndices, 0, 7 * maxTrackCandidates * sizeof(uint16_t), stream);
    hipMemsetAsync(hitIndices, 0, 14 * maxTrackCandidates * sizeof(unsigned int), stream);
    hipMemsetAsync(centerX, 0, maxTrackCandidates * sizeof(FPX), stream);
    hipMemsetAsync(centerY, 0, maxTrackCandidates * sizeof(FPX), stream);
    hipMemsetAsync(radius , 0, maxTrackCandidates * sizeof(FPX), stream);
}

void SDL::createTrackCandidatesInExplicitMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates,hipStream_t stream)
{
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    trackCandidatesInGPU.trackCandidateType = (short*)cms::cuda::allocate_device(dev,maxTrackCandidates * sizeof(short),stream);
    trackCandidatesInGPU.directObjectIndices = (unsigned int*)cms::cuda::allocate_device(dev,maxTrackCandidates * sizeof(unsigned int),stream);
    trackCandidatesInGPU.objectIndices = (unsigned int*)cms::cuda::allocate_device(dev,maxTrackCandidates * 2*sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidates= (int*)cms::cuda::allocate_device(dev, sizeof(int),stream);
    trackCandidatesInGPU.nTrackCandidatespT3= (int*)cms::cuda::allocate_device(dev, sizeof(int),stream);
    trackCandidatesInGPU.nTrackCandidatesT5= (int*)cms::cuda::allocate_device(dev, sizeof(int),stream);
    trackCandidatesInGPU.nTrackCandidatespT5= (int*)cms::cuda::allocate_device(dev, sizeof(int),stream);
    trackCandidatesInGPU.nTrackCandidatespLS= (int*)cms::cuda::allocate_device(dev, sizeof(int),stream);

    trackCandidatesInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_device(dev, 7 * maxTrackCandidates * sizeof(uint8_t), stream);
    trackCandidatesInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev, 7 * maxTrackCandidates * sizeof(uint16_t), stream);
    trackCandidatesInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, 14 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.centerX = (FPX*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(FPX), stream);
    trackCandidatesInGPU.centerY = (FPX*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(FPX), stream);
    trackCandidatesInGPU.radius  = (FPX*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(FPX), stream);

#else
    hipMalloc(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMalloc(&trackCandidatesInGPU.directObjectIndices, maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidates, sizeof(int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespT3, sizeof(int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatesT5, sizeof(int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespT5, sizeof(int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespLS, sizeof(int));

    hipMalloc(&trackCandidatesInGPU.logicalLayers, 7 * maxTrackCandidates * sizeof(uint8_t));
    hipMalloc(&trackCandidatesInGPU.lowerModuleIndices, 7 * maxTrackCandidates * sizeof(uint16_t));
    hipMalloc(&trackCandidatesInGPU.hitIndices, 14 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.centerX, maxTrackCandidates * sizeof(FPX));
    hipMalloc(&trackCandidatesInGPU.centerY, maxTrackCandidates * sizeof(FPX));
    hipMalloc(&trackCandidatesInGPU.radius , maxTrackCandidates * sizeof(FPX));
#endif
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidates,0, sizeof(int), stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatesT5,0, sizeof(int), stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT3,0, sizeof(int), stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT5,0, sizeof(int), stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespLS,0, sizeof(int), stream);
    hipMemsetAsync(trackCandidatesInGPU.logicalLayers, 0, 7 * maxTrackCandidates * sizeof(uint8_t), stream);
    hipMemsetAsync(trackCandidatesInGPU.lowerModuleIndices, 0, 7 * maxTrackCandidates * sizeof(uint16_t), stream);
    hipMemsetAsync(trackCandidatesInGPU.hitIndices, 0, 14 * maxTrackCandidates * sizeof(unsigned int), stream);
    hipStreamSynchronize(stream);
}

SDL::trackCandidates::trackCandidates()
{
    trackCandidateType = nullptr;
    directObjectIndices = nullptr;
    objectIndices = nullptr;
    nTrackCandidates = nullptr;
    nTrackCandidatesT5 = nullptr;
    nTrackCandidatespT3 = nullptr;
    nTrackCandidatespT5 = nullptr;
    nTrackCandidatespLS = nullptr;

    logicalLayers = nullptr;
    hitIndices = nullptr;
    lowerModuleIndices = nullptr;
    centerX = nullptr;
    centerY = nullptr;
    radius = nullptr;
}

SDL::trackCandidates::~trackCandidates()
{
}

void SDL::trackCandidates::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    //FIXME
    //hipFree(trackCandidateType);
    cms::cuda::free_device(dev,directObjectIndices);
    cms::cuda::free_device(dev,objectIndices);
    cms::cuda::free_device(dev,trackCandidateType);
    cms::cuda::free_device(dev,nTrackCandidates);
    cms::cuda::free_device(dev,nTrackCandidatespT3);
    cms::cuda::free_device(dev,nTrackCandidatesT5);
    cms::cuda::free_device(dev,nTrackCandidatespT5);
    cms::cuda::free_device(dev,nTrackCandidatespLS);

    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, centerX);
    cms::cuda::free_device(dev, centerY);
    cms::cuda::free_device(dev, radius);
}

void SDL::trackCandidates::freeMemory(hipStream_t stream)
{
    hipFree(trackCandidateType);
    hipFree(directObjectIndices);
    hipFree(objectIndices);
    hipFree(nTrackCandidates);
    hipFree(nTrackCandidatespT3);
    hipFree(nTrackCandidatesT5);
    hipFree(nTrackCandidatespT5);
    hipFree(nTrackCandidatespLS);

    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(lowerModuleIndices);
    hipFree(centerX);
    hipFree(centerY);
    hipFree(radius);
    
    hipStreamSynchronize(stream);
}
