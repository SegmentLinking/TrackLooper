#include "hip/hip_runtime.h"
#include "TrackCandidate.cuh"

#include "allocate.h"


void SDL::trackCandidates::resetMemory(unsigned int maxTrackCandidates,hipStream_t stream)
{
    hipMemsetAsync(trackCandidateType,0, maxTrackCandidates * sizeof(short),stream);
    hipMemsetAsync(directObjectIndices, 0, maxTrackCandidates * sizeof(unsigned int),stream);
    hipMemsetAsync(objectIndices, 0,2 * maxTrackCandidates * sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidates, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidatespT3, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidatesT5, 0,sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidatespT5,0, sizeof(unsigned int),stream);
    hipMemsetAsync(nTrackCandidatespLS, 0,sizeof(unsigned int),stream);

    hipMemsetAsync(logicalLayers, 0, 7 * maxTrackCandidates * sizeof(uint8_t), stream);
    hipMemsetAsync(lowerModuleIndices, 0, 7 * maxTrackCandidates * sizeof(uint16_t), stream);
    hipMemsetAsync(hitIndices, 0, 14 * maxTrackCandidates * sizeof(unsigned int), stream);
    hipMemsetAsync(pixelSeedIndex, 0, maxTrackCandidates * sizeof(int), stream);
    hipMemsetAsync(centerX, 0, maxTrackCandidates * sizeof(FPX), stream);
    hipMemsetAsync(centerY, 0, maxTrackCandidates * sizeof(FPX), stream);
    hipMemsetAsync(radius , 0, maxTrackCandidates * sizeof(FPX), stream);
}

void SDL::createTrackCandidatesInExplicitMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int maxTrackCandidates,hipStream_t stream)
{
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    trackCandidatesInGPU.trackCandidateType = (short*)cms::cuda::allocate_device(dev,maxTrackCandidates * sizeof(short),stream);
    trackCandidatesInGPU.directObjectIndices = (unsigned int*)cms::cuda::allocate_device(dev,maxTrackCandidates * sizeof(unsigned int),stream);
    trackCandidatesInGPU.objectIndices = (unsigned int*)cms::cuda::allocate_device(dev,maxTrackCandidates * 2*sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidates= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT3= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatesT5= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespT5= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    trackCandidatesInGPU.nTrackCandidatespLS= (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);

    trackCandidatesInGPU.logicalLayers = (uint8_t*)cms::cuda::allocate_device(dev, 7 * maxTrackCandidates * sizeof(uint8_t), stream);
    trackCandidatesInGPU.lowerModuleIndices = (uint16_t*)cms::cuda::allocate_device(dev, 7 * maxTrackCandidates * sizeof(uint16_t), stream);
    trackCandidatesInGPU.hitIndices = (unsigned int*)cms::cuda::allocate_device(dev, 14 * maxTrackCandidates * sizeof(unsigned int), stream);
    trackCandidatesInGPU.pixelSeedIndex = (int*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(int), stream);
    trackCandidatesInGPU.centerX = (FPX*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(FPX), stream);
    trackCandidatesInGPU.centerY = (FPX*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(FPX), stream);
    trackCandidatesInGPU.radius  = (FPX*)cms::cuda::allocate_device(dev, maxTrackCandidates * sizeof(FPX), stream);

#else
    hipMalloc(&trackCandidatesInGPU.trackCandidateType, maxTrackCandidates * sizeof(short));
    hipMalloc(&trackCandidatesInGPU.directObjectIndices, maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.objectIndices, 2 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidates, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespT3, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatesT5, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespT5, sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.nTrackCandidatespLS, sizeof(unsigned int));

    hipMalloc(&trackCandidatesInGPU.logicalLayers, 7 * maxTrackCandidates * sizeof(uint8_t));
    hipMalloc(&trackCandidatesInGPU.lowerModuleIndices, 7 * maxTrackCandidates * sizeof(uint16_t));
    hipMalloc(&trackCandidatesInGPU.hitIndices, 14 * maxTrackCandidates * sizeof(unsigned int));
    hipMalloc(&trackCandidatesInGPU.pixelSeedIndex, maxTrackCandidates * sizeof(int));
    hipMalloc(&trackCandidatesInGPU.centerX, maxTrackCandidates * sizeof(FPX));
    hipMalloc(&trackCandidatesInGPU.centerY, maxTrackCandidates * sizeof(FPX));
    hipMalloc(&trackCandidatesInGPU.radius , maxTrackCandidates * sizeof(FPX));
#endif
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidates,0, sizeof(unsigned int), stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatesT5,0, sizeof(unsigned int), stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT3,0, sizeof(unsigned int), stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespT5,0, sizeof(unsigned int), stream);
    hipMemsetAsync(trackCandidatesInGPU.nTrackCandidatespLS,0, sizeof(unsigned int), stream);
    hipMemsetAsync(trackCandidatesInGPU.logicalLayers, 0, 7 * maxTrackCandidates * sizeof(uint8_t), stream);
    hipMemsetAsync(trackCandidatesInGPU.lowerModuleIndices, 0, 7 * maxTrackCandidates * sizeof(uint16_t), stream);
    hipMemsetAsync(trackCandidatesInGPU.hitIndices, 0, 14 * maxTrackCandidates * sizeof(unsigned int), stream);
    hipMemsetAsync(trackCandidatesInGPU.pixelSeedIndex, 0, maxTrackCandidates * sizeof(int), stream);
    hipStreamSynchronize(stream);
}

ALPAKA_FN_ACC void SDL::addpLSTrackCandidateToMemory(struct trackCandidates& trackCandidatesInGPU, unsigned int trackletIndex, unsigned int trackCandidateIndex, uint4 hitIndices, int pixelSeedIndex)
{
    trackCandidatesInGPU.trackCandidateType[trackCandidateIndex] = 8;
    trackCandidatesInGPU.directObjectIndices[trackCandidateIndex] = trackletIndex;
    trackCandidatesInGPU.pixelSeedIndex[trackCandidateIndex] = pixelSeedIndex;

    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex] = trackletIndex;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex + 1] = trackletIndex;

    trackCandidatesInGPU.hitIndices[14 * trackCandidateIndex + 0] = hitIndices.x; // Order explanation in https://github.com/SegmentLinking/TrackLooper/issues/267
    trackCandidatesInGPU.hitIndices[14 * trackCandidateIndex + 1] = hitIndices.z;
    trackCandidatesInGPU.hitIndices[14 * trackCandidateIndex + 2] = hitIndices.y;
    trackCandidatesInGPU.hitIndices[14 * trackCandidateIndex + 3] = hitIndices.w;
}

ALPAKA_FN_ACC void SDL::addTrackCandidateToMemory(struct trackCandidates& trackCandidatesInGPU, short trackCandidateType, unsigned int innerTrackletIndex, unsigned int outerTrackletIndex, uint8_t* logicalLayerIndices, uint16_t* lowerModuleIndices, unsigned int* hitIndices, int pixelSeedIndex, float centerX, float centerY, float radius, unsigned int trackCandidateIndex, unsigned int directObjectIndex)
{
    trackCandidatesInGPU.trackCandidateType[trackCandidateIndex] = trackCandidateType;
    trackCandidatesInGPU.directObjectIndices[trackCandidateIndex] = directObjectIndex;
    trackCandidatesInGPU.pixelSeedIndex[trackCandidateIndex] = pixelSeedIndex;

    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex] = innerTrackletIndex;
    trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex + 1] = outerTrackletIndex;
    
    size_t limits = trackCandidateType == 7 ? 7 : 5;

    //send the starting pointer to the logicalLayer and hitIndices
    for(size_t i = 0; i < limits; i++)
    {
        trackCandidatesInGPU.logicalLayers[7 * trackCandidateIndex + i] = logicalLayerIndices[i];
        trackCandidatesInGPU.lowerModuleIndices[7 * trackCandidateIndex + i] = lowerModuleIndices[i];
    }
    for(size_t i = 0; i < 2 * limits; i++)
    {
        trackCandidatesInGPU.hitIndices[14 * trackCandidateIndex + i] = hitIndices[i];
    }
    trackCandidatesInGPU.centerX[trackCandidateIndex] = __F2H(centerX);
    trackCandidatesInGPU.centerY[trackCandidateIndex] = __F2H(centerY);
    trackCandidatesInGPU.radius[trackCandidateIndex]  = __F2H(radius);
}

SDL::trackCandidates::trackCandidates()
{
    trackCandidateType = nullptr;
    directObjectIndices = nullptr;
    objectIndices = nullptr;
    nTrackCandidates = nullptr;
    nTrackCandidatesT5 = nullptr;
    nTrackCandidatespT3 = nullptr;
    nTrackCandidatespT5 = nullptr;
    nTrackCandidatespLS = nullptr;

    logicalLayers = nullptr;
    hitIndices = nullptr;
    pixelSeedIndex = nullptr;
    lowerModuleIndices = nullptr;
    centerX = nullptr;
    centerY = nullptr;
    radius = nullptr;
}

SDL::trackCandidates::~trackCandidates()
{
}

void SDL::trackCandidates::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    //FIXME
    //hipFree(trackCandidateType);
    cms::cuda::free_device(dev,directObjectIndices);
    cms::cuda::free_device(dev,objectIndices);
    cms::cuda::free_device(dev,trackCandidateType);
    cms::cuda::free_device(dev,nTrackCandidates);
    cms::cuda::free_device(dev,nTrackCandidatespT3);
    cms::cuda::free_device(dev,nTrackCandidatesT5);
    cms::cuda::free_device(dev,nTrackCandidatespT5);
    cms::cuda::free_device(dev,nTrackCandidatespLS);

    cms::cuda::free_device(dev, logicalLayers);
    cms::cuda::free_device(dev, hitIndices);
    cms::cuda::free_device(dev, pixelSeedIndex);
    cms::cuda::free_device(dev, lowerModuleIndices);
    cms::cuda::free_device(dev, centerX);
    cms::cuda::free_device(dev, centerY);
    cms::cuda::free_device(dev, radius);
}
void SDL::trackCandidates::freeMemory(hipStream_t stream)
{
    hipFree(trackCandidateType);
    hipFree(directObjectIndices);
    hipFree(objectIndices);
    hipFree(nTrackCandidates);
    hipFree(nTrackCandidatespT3);
    hipFree(nTrackCandidatesT5);
    hipFree(nTrackCandidatespT5);
    hipFree(nTrackCandidatespLS);

    hipFree(logicalLayers);
    hipFree(hitIndices);
    hipFree(pixelSeedIndex);
    hipFree(lowerModuleIndices);
    hipFree(centerX);
    hipFree(centerY);
    hipFree(radius);
    
    hipStreamSynchronize(stream);
}

__global__ void SDL::addpT5asTrackCandidateInGPU(uint16_t nLowerModules, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, struct SDL::segments& segmentsInGPU, struct SDL::objectRanges& rangesInGPU)
{
    unsigned int nPixelQuintuplets = *pixelQuintupletsInGPU.nPixelQuintuplets;
    unsigned int pLS_offset = rangesInGPU.segmentModuleIndices[nLowerModules];

    for(int pixelQuintupletIndex = blockIdx.x * blockDim.x + threadIdx.x; pixelQuintupletIndex < nPixelQuintuplets; pixelQuintupletIndex += blockDim.x*gridDim.x)
    {
        if(pixelQuintupletsInGPU.isDup[pixelQuintupletIndex])
        {
            continue;
        }
        unsigned int trackCandidateIdx = atomicAdd(trackCandidatesInGPU.nTrackCandidates,1);
        atomicAdd(trackCandidatesInGPU.nTrackCandidatespT5,1);


        float radius = 0.5f*(__H2F(pixelQuintupletsInGPU.pixelRadius[pixelQuintupletIndex]) + __H2F(pixelQuintupletsInGPU.quintupletRadius[pixelQuintupletIndex]));
        unsigned int pT5PixelIndex =  pixelQuintupletsInGPU.pixelIndices[pixelQuintupletIndex];
        addTrackCandidateToMemory(trackCandidatesInGPU, 7/*track candidate type pT5=7*/, pT5PixelIndex, pixelQuintupletsInGPU.T5Indices[pixelQuintupletIndex], &pixelQuintupletsInGPU.logicalLayers[7 * pixelQuintupletIndex], &pixelQuintupletsInGPU.lowerModuleIndices[7 * pixelQuintupletIndex], &pixelQuintupletsInGPU.hitIndices[14 * pixelQuintupletIndex], segmentsInGPU.seedIdx[pT5PixelIndex - pLS_offset], __H2F(pixelQuintupletsInGPU.centerX[pixelQuintupletIndex]),
                            __H2F(pixelQuintupletsInGPU.centerY[pixelQuintupletIndex]),radius , trackCandidateIdx, pixelQuintupletIndex);
    }
}

__global__ void SDL::crossCleanpT3(struct SDL::modules& modulesInGPU, struct SDL::objectRanges& rangesInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU)
{
    unsigned int nPixelTriplets = *pixelTripletsInGPU.nPixelTriplets;
    for(int pixelTripletIndex = blockIdx.x * blockDim.x + threadIdx.x; pixelTripletIndex < nPixelTriplets; pixelTripletIndex += blockDim.x*gridDim.x)
    {
        if(pixelTripletsInGPU.isDup[pixelTripletIndex]) continue;
        //cross cleaning step
        float eta1 = __H2F(pixelTripletsInGPU.eta_pix[pixelTripletIndex]);
        float phi1 = __H2F(pixelTripletsInGPU.phi_pix[pixelTripletIndex]);

        int pixelModuleIndex = *modulesInGPU.nLowerModules;
        unsigned int prefix = rangesInGPU.segmentModuleIndices[pixelModuleIndex];

        unsigned int nPixelQuintuplets = *pixelQuintupletsInGPU.nPixelQuintuplets;
        for(int pixelQuintupletIndex = blockIdx.y * blockDim.y + threadIdx.y; pixelQuintupletIndex < nPixelQuintuplets; pixelQuintupletIndex += blockDim.y*gridDim.y)
        {
            unsigned int pLS_jx = pixelQuintupletsInGPU.pixelIndices[pixelQuintupletIndex];
            float eta2 = segmentsInGPU.eta[pLS_jx - prefix];
            float phi2 = segmentsInGPU.phi[pLS_jx - prefix];
            float dEta = abs(eta1-eta2);
            float dPhi = abs(phi1-phi2);
            if(dPhi > float(M_PI)){dPhi = dPhi - 2*float(M_PI);}
            float dR2 = dEta*dEta + dPhi*dPhi;
            if(dR2 < 1e-5f) pixelTripletsInGPU.isDup[pixelTripletIndex] = true;
        }
    }
}

__global__ void SDL::crossCleanT5(struct SDL::modules& modulesInGPU, struct SDL::quintuplets& quintupletsInGPU, struct SDL::pixelQuintuplets& pixelQuintupletsInGPU,struct SDL::pixelTriplets& pixelTripletsInGPU, struct SDL::objectRanges& rangesInGPU)
{
    int stepx = blockDim.x*gridDim.x;
    int stepy = blockDim.y*gridDim.y;
    int stepz = blockDim.z*gridDim.z;
    for(int innerInnerInnerLowerModuleArrayIndex = blockIdx.z * blockDim.z + threadIdx.z; innerInnerInnerLowerModuleArrayIndex < *(modulesInGPU.nLowerModules); innerInnerInnerLowerModuleArrayIndex+=stepz)
    {
        if(rangesInGPU.quintupletModuleIndices[innerInnerInnerLowerModuleArrayIndex] == -1) continue;
        unsigned int nQuints = quintupletsInGPU.nQuintuplets[innerInnerInnerLowerModuleArrayIndex];
        for(int innerObjectArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;innerObjectArrayIndex < nQuints;innerObjectArrayIndex+=stepy)
        {
            int quintupletIndex = rangesInGPU.quintupletModuleIndices[innerInnerInnerLowerModuleArrayIndex] + innerObjectArrayIndex;

            //don't add duplicate T5s or T5s that are accounted in pT5s
            if(quintupletsInGPU.isDup[quintupletIndex] or quintupletsInGPU.partOfPT5[quintupletIndex])
            {
                continue;//return;
            }
#ifdef Crossclean_T5
            int loop_bound = *pixelQuintupletsInGPU.nPixelQuintuplets + *pixelTripletsInGPU.nPixelTriplets; 
            //cross cleaning step
            float eta1 = __H2F(quintupletsInGPU.eta[quintupletIndex]);
            float phi1 = __H2F(quintupletsInGPU.phi[quintupletIndex]);

            for (unsigned int jx=blockIdx.x * blockDim.x + threadIdx.x; jx<loop_bound; jx+=stepx)
            {
                float eta2, phi2;
                if(jx < *pixelQuintupletsInGPU.nPixelQuintuplets)
                {
                    eta2 = __H2F(pixelQuintupletsInGPU.eta[jx]);
                    phi2 = __H2F(pixelQuintupletsInGPU.phi[jx]);
                }
                else
                {
                    eta2 = __H2F(pixelTripletsInGPU.eta[jx]);
                    phi2 = __H2F(pixelTripletsInGPU.phi[jx]);
                }

                float dEta = abs(eta1-eta2);
                float dPhi = abs(phi1-phi2);
                if(dPhi > float(M_PI)){dPhi = dPhi - 2*float(M_PI);}
                float dR2 = dEta*dEta + dPhi*dPhi;
                if(dR2 < 1e-3f) {quintupletsInGPU.isDup[quintupletIndex] = true;}//return;

            }
#endif
        }
    }
}

//Using Matt's block for the outer loop and thread for inner loop trick here!
//This will eliminate the need for another kernel just for adding the pLS, because we can __syncthreads()
__global__ void SDL::crossCleanpLS(struct SDL::modules& modulesInGPU, struct SDL::objectRanges& rangesInGPU, struct SDL::pixelTriplets& pixelTripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU,struct SDL::segments& segmentsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::hits& hitsInGPU, struct SDL::quintuplets& quintupletsInGPU)
{
    int pixelModuleIndex = *modulesInGPU.nLowerModules;
    unsigned int nPixels = segmentsInGPU.nSegments[pixelModuleIndex];
    for(int pixelArrayIndex = blockIdx.x * blockDim.x + threadIdx.x; pixelArrayIndex < nPixels; pixelArrayIndex+=blockDim.x*gridDim.x)
    {
        if((!segmentsInGPU.isQuad[pixelArrayIndex]) || (segmentsInGPU.isDup[pixelArrayIndex])) {continue;}

        float eta1 = segmentsInGPU.eta[pixelArrayIndex];
        float phi1 = segmentsInGPU.phi[pixelArrayIndex];
        unsigned int prefix = rangesInGPU.segmentModuleIndices[pixelModuleIndex];

        unsigned int nTrackCandidates = *(trackCandidatesInGPU.nTrackCandidates);
        for(int trackCandidateIndex = blockIdx.y * blockDim.y + threadIdx.y; trackCandidateIndex < nTrackCandidates; trackCandidateIndex+=blockDim.y*gridDim.y)
        {
            short type = trackCandidatesInGPU.trackCandidateType[trackCandidateIndex];
            unsigned int innerTrackletIdx = trackCandidatesInGPU.objectIndices[2 * trackCandidateIndex];
            if(type == 4) //T5
            {
                unsigned int quintupletIndex = innerTrackletIdx;//trackCandidatesInGPU.objectIndices[2*jx];//T5 index
                float eta2 = __H2F(quintupletsInGPU.eta[quintupletIndex]);
                float phi2 = __H2F(quintupletsInGPU.phi[quintupletIndex]);
                float dEta = abs(eta1-eta2);
                float dPhi = abs(phi1-phi2);
                if(dPhi > float(M_PI)){dPhi = dPhi - 2*float(M_PI);}
                float dR2 = dEta*dEta + dPhi*dPhi;
                if(dR2 < 1e-3f) {segmentsInGPU.isDup[pixelArrayIndex] = true;}
            }
            if(type == 5)  //pT3
            {
                int pLSIndex = pixelTripletsInGPU.pixelSegmentIndices[innerTrackletIdx];
                int npMatched = checkPixelHits(prefix+pixelArrayIndex,pLSIndex,mdsInGPU,segmentsInGPU,hitsInGPU);
                if(npMatched >0) {segmentsInGPU.isDup[pixelArrayIndex] = true;}

                int pT3Index = innerTrackletIdx;
                float eta2 = __H2F(pixelTripletsInGPU.eta_pix[pT3Index]);
                float phi2 = __H2F(pixelTripletsInGPU.phi_pix[pT3Index]);
                float dEta = abs(eta1-eta2);
                float dPhi = abs(phi1-phi2);
                if(dPhi > float(M_PI)){dPhi = dPhi - 2*float(M_PI);}
                float dR2 = dEta*dEta + dPhi*dPhi;
                if(dR2 < 0.000001f) {segmentsInGPU.isDup[pixelArrayIndex] = true;}
            }
            if(type == 7) //pT5
            {
                unsigned int pLSIndex = innerTrackletIdx;
                int npMatched = checkPixelHits(prefix+pixelArrayIndex,pLSIndex,mdsInGPU,segmentsInGPU,hitsInGPU);
                if(npMatched >0) {segmentsInGPU.isDup[pixelArrayIndex] = true;}

                float eta2 = segmentsInGPU.eta[pLSIndex - prefix];
                float phi2 = segmentsInGPU.phi[pLSIndex - prefix];
                float dEta = abs(eta1-eta2);
                float dPhi = abs(phi1-phi2);
                if(dPhi > float(M_PI)){dPhi = dPhi - 2*float(M_PI);}
                float dR2 = dEta*dEta + dPhi*dPhi;
                if(dR2 < 0.000001f) {segmentsInGPU.isDup[pixelArrayIndex] = true;}
            }
        }
    }
}

__global__ void SDL::addpT3asTrackCandidatesInGPU(uint16_t nLowerModules, struct SDL::pixelTriplets& pixelTripletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, struct SDL::segments& segmentsInGPU, struct SDL::objectRanges& rangesInGPU)
{
    unsigned int nPixelTriplets = *pixelTripletsInGPU.nPixelTriplets;
    unsigned int pLS_offset = rangesInGPU.segmentModuleIndices[nLowerModules];

    for(int pixelTripletIndex = blockIdx.x * blockDim.x + threadIdx.x; pixelTripletIndex < nPixelTriplets; pixelTripletIndex += blockDim.x*gridDim.x)
    {
        if((pixelTripletsInGPU.isDup[pixelTripletIndex])) continue;//return;
        unsigned int trackCandidateIdx = atomicAdd(trackCandidatesInGPU.nTrackCandidates,1);
        atomicAdd(trackCandidatesInGPU.nTrackCandidatespT3,1);
    
        float radius = 0.5f * (__H2F(pixelTripletsInGPU.pixelRadius[pixelTripletIndex]) + __H2F(pixelTripletsInGPU.tripletRadius[pixelTripletIndex]));
        unsigned int pT3PixelIndex =  pixelTripletsInGPU.pixelSegmentIndices[pixelTripletIndex];
        addTrackCandidateToMemory(trackCandidatesInGPU, 5/*track candidate type pT3=5*/, pixelTripletIndex, pixelTripletIndex, &pixelTripletsInGPU.logicalLayers[5 * pixelTripletIndex], &pixelTripletsInGPU.lowerModuleIndices[5 * pixelTripletIndex], &pixelTripletsInGPU.hitIndices[10 * pixelTripletIndex], segmentsInGPU.seedIdx[pT3PixelIndex - pLS_offset], __H2F(pixelTripletsInGPU.centerX[pixelTripletIndex]), __H2F(pixelTripletsInGPU.centerY[pixelTripletIndex]),radius,trackCandidateIdx, pixelTripletIndex);
    }    

}

__global__ void SDL::addT5asTrackCandidateInGPU(uint16_t nLowerModules, struct SDL::quintuplets& quintupletsInGPU, struct SDL::trackCandidates& trackCandidatesInGPU, struct SDL::objectRanges& rangesInGPU)
{
    for(uint16_t idx = blockIdx.y * blockDim.y + threadIdx.y; idx < nLowerModules; idx+= gridDim.y * blockDim.y)
    {
        if(rangesInGPU.quintupletModuleIndices[idx] == -1) continue;
        unsigned int nQuints = quintupletsInGPU.nQuintuplets[idx];
        for(unsigned int jdx = blockIdx.x * blockDim.x + threadIdx.x; jdx < nQuints; jdx += blockDim.x * gridDim.x)
        {
            int quintupletIndex = rangesInGPU.quintupletModuleIndices[idx] + jdx;

            if (quintupletsInGPU.isDup[quintupletIndex] or quintupletsInGPU.partOfPT5[quintupletIndex]) continue;

            if (!(quintupletsInGPU.TightCutFlag[quintupletIndex])) continue;

            unsigned int trackCandidateIdx = atomicAdd(trackCandidatesInGPU.nTrackCandidates,1);
            atomicAdd(trackCandidatesInGPU.nTrackCandidatesT5,1);
    
            addTrackCandidateToMemory(trackCandidatesInGPU, 4/*track candidate type T5=4*/, quintupletIndex, quintupletIndex, &quintupletsInGPU.logicalLayers[5 * quintupletIndex], &quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex], &quintupletsInGPU.hitIndices[10 * quintupletIndex], -1/*no pixel seed index for T5s*/, quintupletsInGPU.regressionG[quintupletIndex], quintupletsInGPU.regressionF[quintupletIndex], quintupletsInGPU.regressionRadius[quintupletIndex], trackCandidateIdx, quintupletIndex);
        } 
    }
}

__global__ void SDL::addpLSasTrackCandidateInGPU(uint16_t nLowerModules, struct SDL::trackCandidates& trackCandidatesInGPU, struct SDL::segments& segmentsInGPU)
{
    //int pixelArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x*gridDim.x;
    unsigned int nPixels = segmentsInGPU.nSegments[nLowerModules];
    for(int pixelArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;pixelArrayIndex < nPixels;  pixelArrayIndex +=step)
    {
        if((!segmentsInGPU.isQuad[pixelArrayIndex]) || (segmentsInGPU.isDup[pixelArrayIndex]))
        {
            continue;//return;
        }

        unsigned int trackCandidateIdx = atomicAdd(trackCandidatesInGPU.nTrackCandidates,1);
        atomicAdd(trackCandidatesInGPU.nTrackCandidatespLS,1);
        addpLSTrackCandidateToMemory(trackCandidatesInGPU, pixelArrayIndex, trackCandidateIdx, segmentsInGPU.pLSHitsIdxs[pixelArrayIndex], segmentsInGPU.seedIdx[pixelArrayIndex]);
    }
}


ALPAKA_FN_ACC int SDL::checkPixelHits(unsigned int ix, unsigned int jx,struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::hits& hitsInGPU)
{
    int phits1[4] = {-1,-1,-1,-1};
    int phits2[4] = {-1,-1,-1,-1};
    phits1[0] = hitsInGPU.idxs[mdsInGPU.anchorHitIndices[segmentsInGPU.mdIndices[2*ix]]];
    phits1[1] = hitsInGPU.idxs[mdsInGPU.anchorHitIndices[segmentsInGPU.mdIndices[2*ix+1]]];
    phits1[2] = hitsInGPU.idxs[mdsInGPU.outerHitIndices[segmentsInGPU.mdIndices[2*ix]]];
    phits1[3] = hitsInGPU.idxs[mdsInGPU.outerHitIndices[segmentsInGPU.mdIndices[2*ix+1]]];

    phits2[0] = hitsInGPU.idxs[mdsInGPU.anchorHitIndices[segmentsInGPU.mdIndices[2*jx]]];
    phits2[1] = hitsInGPU.idxs[mdsInGPU.anchorHitIndices[segmentsInGPU.mdIndices[2*jx+1]]];
    phits2[2] = hitsInGPU.idxs[mdsInGPU.outerHitIndices[segmentsInGPU.mdIndices[2*jx]]];
    phits2[3] = hitsInGPU.idxs[mdsInGPU.outerHitIndices[segmentsInGPU.mdIndices[2*jx+1]]];

    int npMatched = 0;

    for (int i =0; i<4;i++)
    {
        bool pmatched = false;
        if(phits1[i] == -1){continue;}
        for (int j =0; j<4; j++)
        {
            if(phits2[j] == -1){continue;}
            if(phits1[i] == phits2[j]){pmatched = true; break;}
        }
        if(pmatched){npMatched++;}
    }
    return npMatched;
}
