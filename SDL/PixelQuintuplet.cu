#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CONST_VAR __device__
#endif
# include "PixelQuintuplet.cuh"
#include "allocate.h"

SDL::pixelQuintuplets::pixelQuintuplets()
{
    pixelIndices = nullptr;
    T5Indices = nullptr;
    nPixelQuintuplets = nullptr;
    isDup = nullptr;
    score = nullptr;
}

SDL::pixelQuintuplets::~pixelQuintuplets()
{
}

void SDL::pixelQuintuplets::freeMemoryCache()
{
#ifdef Explicit_PT5
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,pixelIndices);
    cms::cuda::free_device(dev,T5Indices);
    cms::cuda::free_device(dev,nPixelQuintuplets);
    cms::cuda::free_device(dev,isDup);
    cms::cuda::free_device(dev,score);
    cms::cuda::free_device(dev,eta);
    cms::cuda::free_device(dev,phi);
#else
    cms::cuda::free_managed(pixelIndices);
    cms::cuda::free_managed(T5Indices);
    cms::cuda::free_managed(nPixelQuintuplets);
    cms::cuda::free_managed(isDup);
    cms::cuda::free_managed(score);
    cms::cuda::free_managed(eta);
    cms::cuda::free_managed(phi);
#endif
}
void SDL::pixelQuintuplets::freeMemory(hipStream_t stream)
{
    //hipFreeAsync(pixelIndices,stream);
    //hipFreeAsync(T5Indices,stream);
    //hipFreeAsync(nPixelQuintuplets,stream);
    //hipFreeAsync(isDup,stream);
    //hipFreeAsync(score,stream);
    //hipFreeAsync(eta,stream);
    //hipFreeAsync(phi,stream);
    hipFree(pixelIndices);
    hipFree(T5Indices);
    hipFree(nPixelQuintuplets);
    hipFree(isDup);
    hipFree(score);
    hipFree(eta);
    hipFree(phi);
#ifdef CUT_VALUE_DEBUG
    hipFree(rzChiSquared);
    hipFree(rPhiChiSquared);
    hipFree(rPhiChiSquaredInwards);
#endif
hipStreamSynchronize(stream);
}

void SDL::pixelQuintuplets::resetMemory(unsigned int maxPixelQuintuplets,hipStream_t stream)
{
    hipMemsetAsync(pixelIndices,0, maxPixelQuintuplets * sizeof(unsigned int),stream);
    hipMemsetAsync(T5Indices,0, maxPixelQuintuplets * sizeof(unsigned int),stream);
    hipMemsetAsync(nPixelQuintuplets,0, sizeof(unsigned int),stream);
    hipMemsetAsync(isDup,0, maxPixelQuintuplets * sizeof(bool),stream);
    hipMemsetAsync(score,0, maxPixelQuintuplets * sizeof(FPX),stream);
    hipMemsetAsync(eta, 0,maxPixelQuintuplets * sizeof(FPX),stream);
    hipMemsetAsync(phi, 0,maxPixelQuintuplets * sizeof(FPX),stream);
}
void SDL::createPixelQuintupletsInUnifiedMemory(struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int maxPixelQuintuplets,hipStream_t stream)
{
#ifdef CACHE_ALLOC
 //   hipStream_t stream=0;
    pixelQuintupletsInGPU.pixelIndices        = (unsigned int*)cms::cuda::allocate_managed(maxPixelQuintuplets * sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.T5Indices           = (unsigned int*)cms::cuda::allocate_managed(maxPixelQuintuplets * sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.nPixelQuintuplets   = (unsigned int*)cms::cuda::allocate_managed(sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.isDup               = (bool*)cms::cuda::allocate_managed(maxPixelQuintuplets * sizeof(bool),stream);
    pixelQuintupletsInGPU.score               = (FPX*)cms::cuda::allocate_managed(maxPixelQuintuplets * sizeof(FPX),stream);
    pixelQuintupletsInGPU.eta                 = (FPX*)cms::cuda::allocate_managed(maxPixelQuintuplets * sizeof(FPX),stream);
    pixelQuintupletsInGPU.phi                 = (FPX*)cms::cuda::allocate_managed(maxPixelQuintuplets * sizeof(FPX),stream);
#else
    hipMallocManaged(&pixelQuintupletsInGPU.pixelIndices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.T5Indices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.nPixelQuintuplets, sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.isDup, maxPixelQuintuplets * sizeof(bool));
    hipMallocManaged(&pixelQuintupletsInGPU.score, maxPixelQuintuplets * sizeof(FPX));
    hipMallocManaged(&pixelQuintupletsInGPU.eta, maxPixelQuintuplets * sizeof(FPX));
    hipMallocManaged(&pixelQuintupletsInGPU.phi, maxPixelQuintuplets * sizeof(FPX));
#ifdef CUT_VALUE_DEBUG
    hipMallocManaged(&pixelQuintupletsInGPU.rzChiSquared, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.rPhiChiSquared, maxPixelQuintuplets * sizeof(unsigned int));
    hipMallocManaged(&pixelQuintupletsInGPU.rPhiChiSquaredInwards, maxPixelQuintuplets * sizeof(unsigned int));
#endif
#endif

    hipMemsetAsync(pixelQuintupletsInGPU.nPixelQuintuplets, 0, sizeof(unsigned int),stream);
  hipStreamSynchronize(stream);
}

void SDL::createPixelQuintupletsInExplicitMemory(struct SDL::pixelQuintuplets& pixelQuintupletsInGPU, unsigned int maxPixelQuintuplets,hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    pixelQuintupletsInGPU.pixelIndices        = (unsigned int*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.T5Indices           = (unsigned int*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.nPixelQuintuplets   = (unsigned int*)cms::cuda::allocate_device(dev,sizeof(unsigned int),stream);
    pixelQuintupletsInGPU.isDup               = (bool*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(bool),stream);
    pixelQuintupletsInGPU.score               = (FPX*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(FPX),stream);
    pixelQuintupletsInGPU.eta                 = (FPX*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(FPX),stream);
    pixelQuintupletsInGPU.phi                 = (FPX*)cms::cuda::allocate_device(dev,maxPixelQuintuplets * sizeof(FPX),stream);
#else
    //hipMallocAsync(&pixelQuintupletsInGPU.pixelIndices, maxPixelQuintuplets * sizeof(unsigned int),stream);
    //hipMallocAsync(&pixelQuintupletsInGPU.T5Indices, maxPixelQuintuplets * sizeof(unsigned int),stream);
    //hipMallocAsync(&pixelQuintupletsInGPU.nPixelQuintuplets, sizeof(unsigned int),stream);
    //hipMallocAsync(&pixelQuintupletsInGPU.isDup, maxPixelQuintuplets * sizeof(bool),stream);
    //hipMallocAsync(&pixelQuintupletsInGPU.score, maxPixelQuintuplets * sizeof(float),stream);
    //hipMallocAsync(&pixelQuintupletsInGPU.eta, maxPixelQuintuplets * sizeof(float),stream);
    //hipMallocAsync(&pixelQuintupletsInGPU.phi, maxPixelQuintuplets * sizeof(float),stream);

    hipMalloc(&pixelQuintupletsInGPU.pixelIndices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.T5Indices, maxPixelQuintuplets * sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.nPixelQuintuplets, sizeof(unsigned int));
    hipMalloc(&pixelQuintupletsInGPU.isDup, maxPixelQuintuplets * sizeof(bool));
    hipMalloc(&pixelQuintupletsInGPU.score, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.eta, maxPixelQuintuplets * sizeof(FPX));
    hipMalloc(&pixelQuintupletsInGPU.phi, maxPixelQuintuplets * sizeof(FPX));

#endif
    hipMemsetAsync(pixelQuintupletsInGPU.nPixelQuintuplets, 0, sizeof(unsigned int),stream);
  hipStreamSynchronize(stream);
}

__device__ void SDL::rmPixelQuintupletToMemory(struct pixelQuintuplets& pixelQuintupletsInGPU, unsigned int pixelQuintupletIndex)
{

    pixelQuintupletsInGPU.isDup[pixelQuintupletIndex] = 1;
}
#ifdef CUT_VALUE_DEBUG
__device__ void SDL::addPixelQuintupletToMemory(struct pixelQuintuplets& pixelQuintupletsInGPU, unsigned int pixelIndex, unsigned int T5Index, unsigned int pixelQuintupletIndex, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards, float score)
#else
__device__ void SDL::addPixelQuintupletToMemory(struct pixelQuintuplets& pixelQuintupletsInGPU, unsigned int pixelIndex, unsigned int T5Index, unsigned int pixelQuintupletIndex, float score,float eta, float phi)
#endif
{
    pixelQuintupletsInGPU.pixelIndices[pixelQuintupletIndex] = pixelIndex;
    pixelQuintupletsInGPU.T5Indices[pixelQuintupletIndex] = T5Index;
    pixelQuintupletsInGPU.isDup[pixelQuintupletIndex] = 0;
    pixelQuintupletsInGPU.score[pixelQuintupletIndex] = __F2H(score);
    pixelQuintupletsInGPU.eta[pixelQuintupletIndex] = __F2H(eta);
    pixelQuintupletsInGPU.phi[pixelQuintupletIndex] = __F2H(phi);
    
#ifdef CUT_VALUE_DEBUG
    pixelQuintupletsInGPU.rzChiSquared[pixelQuintupletIndex] = rzChiSquared;
    pixelQuintupletsInGPU.rPhiChiSquared[pixelQuintupletIndex] = rPhiChiSquared;
    pixelQuintupletsInGPU.rPhiChiSquaredInwards[pixelQuintupletIndex] = rPhiChiSquaredInwards;
#endif
}

__device__ bool SDL::runPixelQuintupletDefaultAlgo(struct modules& modulesInGPU, struct hits& hitsInGPU, struct miniDoublets& mdsInGPU, struct segments& segmentsInGPU, struct triplets& tripletsInGPU, struct quintuplets& quintupletsInGPU, unsigned int& pixelSegmentIndex, unsigned int& quintupletIndex, float& rzChiSquared, float& rPhiChiSquared, float& rPhiChiSquaredInwards)
{
    bool pass = true;
    
    unsigned int pixelModuleIndex = segmentsInGPU.innerLowerModuleIndices[pixelSegmentIndex];

    unsigned int pixelSegmentArrayIndex = pixelSegmentIndex - (600 * pixelModuleIndex);

    unsigned int T5InnerT3Index = quintupletsInGPU.tripletIndices[2 * quintupletIndex];
    unsigned int T5OuterT3Index = quintupletsInGPU.tripletIndices[2 * quintupletIndex + 1];

    unsigned int firstSegmentIndex = tripletsInGPU.segmentIndices[2 * T5InnerT3Index];
    unsigned int secondSegmentIndex = tripletsInGPU.segmentIndices[2 * T5InnerT3Index + 1];
    unsigned int thirdSegmentIndex = tripletsInGPU.segmentIndices[2 * T5OuterT3Index];
    unsigned int fourthSegmentIndex = tripletsInGPU.segmentIndices[2 * T5OuterT3Index + 1];

    unsigned int pixelInnerMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex];
    unsigned int pixelOuterMDIndex = segmentsInGPU.mdIndices[2 * pixelSegmentIndex + 1];

    unsigned int pixelAnchorHitIndex1 = mdsInGPU.hitIndices[2 * pixelInnerMDIndex];
    unsigned int pixelNonAnchorHitIndex1 = mdsInGPU.hitIndices[2 * pixelInnerMDIndex + 1];
    unsigned int pixelAnchorHitIndex2 = mdsInGPU.hitIndices[2 * pixelOuterMDIndex];
    unsigned int pixelNonAnchorHitIndex2 = mdsInGPU.hitIndices[2 * pixelOuterMDIndex + 1];

    unsigned int anchorHitIndex1 = segmentsInGPU.innerMiniDoubletAnchorHitIndices[firstSegmentIndex];
    unsigned int anchorHitIndex2 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[firstSegmentIndex]; //same as second segment inner MD anchorhit index
    unsigned int anchorHitIndex3 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[secondSegmentIndex]; //same as third segment inner MD anchor hit index

    unsigned int anchorHitIndex4 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[thirdSegmentIndex]; //same as fourth segment inner MD anchor hit index
    unsigned int anchorHitIndex5 = segmentsInGPU.outerMiniDoubletAnchorHitIndices[fourthSegmentIndex];

    unsigned int lowerModuleIndex1 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex];
    unsigned int lowerModuleIndex2 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 1];
    unsigned int lowerModuleIndex3 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 2];
    unsigned int lowerModuleIndex4 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 3];
    unsigned int lowerModuleIndex5 = quintupletsInGPU.lowerModuleIndices[5 * quintupletIndex + 4];

    unsigned int lowerModuleIndices[] = {lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5};
    unsigned int anchorHits[] = {anchorHitIndex1, anchorHitIndex2, anchorHitIndex3, anchorHitIndex4, anchorHitIndex5};
    unsigned int pixelHits[] = {pixelAnchorHitIndex1, pixelAnchorHitIndex2};
    
    float pixelRadius, pixelRadiusError, tripletRadius, rPhiChiSquaredTemp, rzChiSquaredTemp, rPhiChiSquaredInwardsTemp;

    pass = pass & runPixelTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, pixelSegmentIndex, T5InnerT3Index, pixelRadius, pixelRadiusError, tripletRadius, rzChiSquaredTemp, rPhiChiSquaredTemp, rPhiChiSquaredInwardsTemp, false);

    rzChiSquared = computePT5RZChiSquared(modulesInGPU, hitsInGPU, pixelAnchorHitIndex1, pixelAnchorHitIndex2, anchorHits, lowerModuleIndices);

    rPhiChiSquared = computePT5RPhiChiSquared(modulesInGPU, hitsInGPU, segmentsInGPU, pixelSegmentArrayIndex, anchorHits, lowerModuleIndices);

    rPhiChiSquaredInwards = computePT5RPhiChiSquaredInwards(modulesInGPU, hitsInGPU, quintupletsInGPU, quintupletIndex, pixelHits);

    if(segmentsInGPU.circleRadius[pixelSegmentArrayIndex] < 5.0f/(2.f * k2Rinv1GeVf))
    {
        pass = pass & passPT5RZChiSquaredCuts(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rzChiSquared);

        pass = pass & passPT5RPhiChiSquaredCuts(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rPhiChiSquared);
    }
    
    //if(__H2F(quintupletsInGPU.regressionRadius[quintupletIndex]) < 5.0f/(2.f * k2Rinv1GeVf))
    //if(/*__H2F(*/quintupletsInGPU.regressionRadius[quintupletIndex] < 5.0f/(2.f * k2Rinv1GeVf))
    if(__H2F_T5(quintupletsInGPU.regressionRadius[quintupletIndex]) < 5.0f/(2.f * k2Rinv1GeVf))
    {
        pass = pass & passPT5RPhiChiSquaredInwardsCuts(modulesInGPU, lowerModuleIndex1, lowerModuleIndex2, lowerModuleIndex3, lowerModuleIndex4, lowerModuleIndex5, rPhiChiSquaredInwards);
    }

    //other cuts will be filled here!
    return pass;
}


__device__ bool SDL::passPT5RPhiChiSquaredCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, unsigned int lowerModuleIndex4, unsigned int lowerModuleIndex5, float rPhiChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 48.921f;
        }
        else if(layer4 == 4 and layer5 == 12)
        {
            return rPhiChiSquared < 97.948f;
        }
        else if(layer4 == 4 and layer5 == 5)
        {
            return rPhiChiSquared < 129.3f;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return rPhiChiSquared < 56.21f;
        }
        else if(layer4 == 7 and layer5 == 8)
        {
            return rPhiChiSquared < 74.198f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rPhiChiSquared < 21.265f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 37.058f;
        }
        else if(layer4 == 8 and layer5 == 9)
        {
            return rPhiChiSquared < 42.578f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 32.253f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 37.058f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 97.947f;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return rPhiChiSquared < 129.3f;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return rPhiChiSquared < 170.68f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {   
            return rPhiChiSquared < 48.92f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 74.2f;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 14 and layer5 == 15)
        {
            return rPhiChiSquared < 42.58f;
        }
        else if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 37.06f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 48.92f;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rPhiChiSquared < 85.25f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return rPhiChiSquared < 42.58f;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return rPhiChiSquared < 37.06f;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return rPhiChiSquared < 37.06f;
        }
    }
    return true;
}



__device__ bool SDL::passPT5RPhiChiSquaredInwardsCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, unsigned int lowerModuleIndex4, unsigned int lowerModuleIndex5, float rPhiChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 451.141f;
        }
        else if(layer4 == 4 and layer5 == 12)
        {
            return rPhiChiSquared < 786.173f;
        }
        else if(layer4 == 4 and layer5 == 5)
        {
            return rPhiChiSquared < 595.545f;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return rPhiChiSquared < 581.339f;
        }
        else if(layer4 == 7 and layer5 == 8)
        {
            return rPhiChiSquared < 112.537f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rPhiChiSquared < 225.322f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 1192.402f;
        }
        else if(layer4 == 8 and layer5 == 9)
        {
            return rPhiChiSquared < 786.173f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 1037.817f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 1808.536f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rPhiChiSquared < 684.253f;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return rPhiChiSquared < 684.253f;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return rPhiChiSquared < 684.253f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {   
            return rPhiChiSquared < 451.141f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rPhiChiSquared < 518.34f;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 14 and layer5 == 15)
        {
            return rPhiChiSquared < 2077.92f;
        }
        else if(layer4 == 9 and layer5 == 10)
        {
            return rPhiChiSquared < 74.20f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rPhiChiSquared < 1808.536f;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rPhiChiSquared < 786.173f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return rPhiChiSquared < 1574.076f;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return rPhiChiSquared < 5492.11f;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return rPhiChiSquared < 2743.037f;
        }
    }
    return true;
}

__device__ float SDL::computePT5RPhiChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, struct segments& segmentsInGPU, unsigned int pixelSegmentArrayIndex, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    /*
       Compute circle parameters from 3 pixel hits, and then use them to compute the chi squared for the outer hits
    */

    float g = segmentsInGPU.circleCenterX[pixelSegmentArrayIndex];
    float f = segmentsInGPU.circleCenterY[pixelSegmentArrayIndex];
    float radius = segmentsInGPU.circleRadius[pixelSegmentArrayIndex];
    float delta1[5], delta2[5], slopes[5];
    bool isFlat[5];
    float xs[5];
    float ys[5];
    float chiSquared = 0;
    for(size_t i = 0; i < 5; i++)
    {
        xs[i] = hitsInGPU.xs[anchorHits[i]];
        ys[i] = hitsInGPU.ys[anchorHits[i]];
    }

    computeSigmasForRegression_pT5(modulesInGPU, lowerModuleIndices, delta1, delta2, slopes, isFlat);
    chiSquared = computeChiSquared(5, xs, ys, delta1, delta2, slopes, isFlat, g, f, radius);

    return chiSquared;
}

__device__ bool SDL::passPT5RZChiSquaredCuts(struct modules& modulesInGPU, unsigned int lowerModuleIndex1, unsigned int lowerModuleIndex2, unsigned int lowerModuleIndex3, unsigned int lowerModuleIndex4, unsigned int lowerModuleIndex5, float& rzChiSquared)
{
    const int layer1 = modulesInGPU.layers[lowerModuleIndex1] + 6 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex1] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex1] == SDL::TwoS);
    const int layer2 = modulesInGPU.layers[lowerModuleIndex2] + 6 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex2] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex2] == SDL::TwoS);
    const int layer3 = modulesInGPU.layers[lowerModuleIndex3] + 6 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex3] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex3] == SDL::TwoS);
    const int layer4 = modulesInGPU.layers[lowerModuleIndex4] + 6 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex4] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex4] == SDL::TwoS);
    const int layer5 = modulesInGPU.layers[lowerModuleIndex5] + 6 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex5] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex5] == SDL::TwoS);

    
    if(layer1 == 1 and layer2 == 2 and layer3 == 3)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rzChiSquared < 451.141f;
        }
        else if(layer4 == 4 and layer5 == 12)
        {
            return rzChiSquared < 392.654f;
        }
        else if(layer4 == 4 and layer5 == 5)
        {
            return rzChiSquared < 225.322f;
        }
        else if(layer4 == 7 and layer5 == 13)
        {
            return rzChiSquared < 595.546f;
        }
        else if(layer4 == 7 and layer5 == 8)
        {
            return rzChiSquared < 196.111f;
        }
    }
    else if(layer1 == 1 and layer2 == 2 and layer3 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rzChiSquared < 297.446f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {   
            return rzChiSquared < 451.141f;
        }
        else if(layer4 == 8 and layer5 == 9)
        {
            return rzChiSquared < 518.339f;
        }
    }
    else if(layer1 == 1 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 9 and layer5 == 10)
        {
            return rzChiSquared < 341.75f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rzChiSquared < 341.75f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer3 == 4)
    {
        if(layer4 == 12 and layer5 == 13)
        {
            return rzChiSquared < 392.655f;
        }
        else if(layer4 == 5 and layer5 == 12)
        {
            return rzChiSquared < 341.75f;
        }
        else if(layer4 == 5 and layer5 == 6)
        {
            return rzChiSquared < 112.537f;
        }
    }
    else if(layer1 == 2 and layer2 == 3 and layer4 == 7)
    {
        if(layer4 == 13 and layer5 == 14)
        {
            return rzChiSquared < 595.545f;
        }
        else if(layer4 == 8 and layer5 == 14)
        {
            return rzChiSquared < 74.198f;
        }
    }
    else if(layer1 == 2 and layer2 == 7 and layer3 == 8)
    {
        if(layer4 == 14 and layer5 == 15)
        {
            return rzChiSquared < 518.339f;
        }
        else if(layer4 == 9 and layer5 == 10)
        {
            return rzChiSquared < 8.046f;
        }
        else if(layer4 == 9 and layer5 == 15)
        {
            return rzChiSquared < 451.141f;
        }
    }
    else if(layer1 == 3 and layer2 == 7 and layer3 == 8 and layer4 == 14 and layer5 == 15)
    {
        return rzChiSquared < 56.207f;
    }
    else if(layer1 == 7 and layer2 == 8 and layer3 == 9)
    {
        if(layer4 == 10 and layer5 == 11)
        {
            return rzChiSquared < 64.578f;
        }
        else if(layer4 == 10 and layer5 == 16)
        {
            return rzChiSquared < 85.250f;
        }
        else if(layer4 == 15 and layer5 == 16)
        {
            return rzChiSquared < 85.250f;
        }
    }
    return true;
}

__device__ float SDL::computePT5RPhiChiSquaredInwards(struct modules& modulesInGPU, struct hits& hitsInGPU, struct quintuplets& quintupletsInGPU, unsigned int quintupletIndex, unsigned int* pixelHits)
{
    /*Using the computed regression center and radius, compute the chi squared for the pixels*/
    float g = __H2F_T5(quintupletsInGPU.regressionG[quintupletIndex]);
    float f = __H2F_T5(quintupletsInGPU.regressionF[quintupletIndex]);
    float r = __H2F_T5(quintupletsInGPU.regressionRadius[quintupletIndex]);
    //float g = /*__H2F(*/quintupletsInGPU.regressionG[quintupletIndex];
    //float f = /*__H2F(*/quintupletsInGPU.regressionF[quintupletIndex];
    //float r = /*__H2F(*/quintupletsInGPU.regressionRadius[quintupletIndex];
    //float g = __H2F(quintupletsInGPU.regressionG[quintupletIndex]);
    //float f = __H2F(quintupletsInGPU.regressionF[quintupletIndex]);
    //float r = __H2F(quintupletsInGPU.regressionRadius[quintupletIndex]);
    float x, y;
    float chiSquared = 0;   
    for(size_t i = 0; i < 2; i++)
    {
        x = hitsInGPU.xs[pixelHits[i]];
        y = hitsInGPU.ys[pixelHits[i]];
        float residual = (x - g) * (x -g) + (y - f) * (y - f) - r * r;
        chiSquared += residual * residual;
    }
    //chiSquared /= 2;
    chiSquared *= 0.5f;
    return chiSquared;
}

__device__ float SDL::computePT5RZChiSquared(struct modules& modulesInGPU, struct hits& hitsInGPU, unsigned int& pixelAnchorHitIndex1, unsigned int& pixelAnchorHitIndex2, unsigned int* anchorHits, unsigned int* lowerModuleIndices)
{
    //use the two anchor hits of the pixel segment to compute the slope
    //then compute the pseudo chi squared of the five outer hits

    float& rtPix1 = hitsInGPU.rts[pixelAnchorHitIndex1];
    float& rtPix2 = hitsInGPU.rts[pixelAnchorHitIndex2];
    float& zPix1 = hitsInGPU.zs[pixelAnchorHitIndex1];
    float& zPix2 = hitsInGPU.zs[pixelAnchorHitIndex2];
    float slope = (zPix2 - zPix1)/(rtPix2 - rtPix1);
    float rtAnchor, zAnchor;
    float residual = 0;
    float error = 0;
    //hardcoded array indices!!!
    float RMSE = 0;
    float drdz;
    for(size_t i = 0; i < 5; i++)
    {
        unsigned int& anchorHitIndex = anchorHits[i];
        unsigned int& lowerModuleIndex = lowerModuleIndices[i];
        rtAnchor = hitsInGPU.rts[anchorHitIndex];
        zAnchor = hitsInGPU.zs[anchorHitIndex];

        const int moduleType = modulesInGPU.moduleType[lowerModuleIndex];
        const int moduleSide = modulesInGPU.sides[lowerModuleIndex];
        const int moduleLayerType = modulesInGPU.moduleLayerType[lowerModuleIndex];
        const int layer = modulesInGPU.layers[lowerModuleIndex] + 6 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap) + 5 * (modulesInGPU.subdets[lowerModuleIndex] == SDL::Endcap and modulesInGPU.moduleType[lowerModuleIndex] == SDL::TwoS);
        
        residual = (layer <= 6) ?  (zAnchor - zPix1) - slope * (rtAnchor - rtPix1) : (rtAnchor - rtPix1) - (zAnchor - zPix1)/slope;
        
        //PS Modules
        if(moduleType == 0)
        {
            error = 0.15f;
        }
        else //2S modules
        {
            error = 5.0f;
        }

        //special dispensation to tilted PS modules!
        if(moduleType == 0 and layer <= 6 and moduleSide != Center)
        {
            if(moduleLayerType == Strip)
            {
                drdz = modulesInGPU.drdzs[lowerModuleIndex];
            }
            else
            {
                drdz = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(lowerModuleIndex)];
            }

            //error *= 1.f/sqrtf(1.f + drdz * drdz);
            error /= sqrtf(1.f + drdz * drdz);
        }
        RMSE += (residual * residual)/(error * error);
    }

    RMSE = sqrtf(0.2f * RMSE);
    return RMSE;
}
__device__ void SDL::computeSigmasForRegression_pT5(SDL::modules& modulesInGPU, const unsigned int* lowerModuleIndices, float* delta1, float* delta2, float* slopes, bool* isFlat, int nPoints, bool anchorHits)
{
   /*bool anchorHits required to deal with a weird edge case wherein
     the hits ultimately used in the regression are anchor hits, but the
     lower modules need not all be Pixel Modules (in case of PS). Similarly,
     when we compute the chi squared for the non-anchor hits, the "partner module"
     need not always be a PS strip module, but all non-anchor hits sit on strip
     modules.
    */
    ModuleType moduleType;
    short moduleSubdet, moduleSide;
    ModuleLayerType moduleLayerType;
    float drdz;
    float inv1 = 0.01f/0.009f;
    float inv2 = 0.15f/0.009f;
    float inv3 = 2.4f/0.009f;
    for(size_t i=0; i<nPoints; i++)
    {
        moduleType = modulesInGPU.moduleType[lowerModuleIndices[i]];
        moduleSubdet = modulesInGPU.subdets[lowerModuleIndices[i]];
        moduleSide = modulesInGPU.sides[lowerModuleIndices[i]];
        moduleLayerType = modulesInGPU.moduleLayerType[lowerModuleIndices[i]];
        //category 1 - barrel PS flat
        if(moduleSubdet == Barrel and moduleType == PS and moduleSide == Center)
        {
            //delta1[i] = 0.01;
            //delta2[i] = 0.01;
            delta1[i] = inv1;//1.1111f;//0.01;
            delta2[i] = inv1;//1.1111f;//0.01;
            slopes[i] = -999.f;
            isFlat[i] = true;
        }

        //category 2 - barrel 2S
        else if(moduleSubdet == Barrel and moduleType == TwoS)
        {
            //delta1[i] = 0.009;
            //delta2[i] = 0.009;
            delta1[i] = 1.f;//0.009;
            delta2[i] = 1.f;//0.009;
            slopes[i] = -999.f;
            isFlat[i] = true;
        }

        //category 3 - barrel PS tilted
        else if(moduleSubdet == Barrel and moduleType == PS and moduleSide != Center)
        {

            //get drdz
            if(moduleLayerType == Strip)
            {
                drdz = modulesInGPU.drdzs[lowerModuleIndices[i]];
                slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
            }
            else
            {
                drdz = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndex(lowerModuleIndices[i])];
                slopes[i] = modulesInGPU.slopes[modulesInGPU.partnerModuleIndex(lowerModuleIndices[i])];
            }

            //delta1[i] = 0.01;
            delta1[i] = inv1;//1.1111f;//0.01;
            isFlat[i] = false;

            if(anchorHits)
            {
                //delta2[i] = (0.15f * drdz/sqrtf(1 + drdz * drdz));
                delta2[i] = (inv2 * drdz/sqrtf(1 + drdz * drdz));
                //delta2[i] = (inv2 * drdz*rsqrt(1 + drdz * drdz));
            }
            else
            {
                //delta2[i] = (2.4f * drdz/sqrtf(1 + drdz * drdz));
                delta2[i] = (inv3 * drdz/sqrtf(1 + drdz * drdz));
                //delta2[i] = (inv3 * drdz*rsqrt(1 + drdz * drdz));
            }
        }
        //category 4 - endcap PS
        else if(moduleSubdet == Endcap and moduleType == PS)
        {
            delta1[i] = inv1;//1.1111f;//0.01;
            //delta1[i] = 0.01;
            if(moduleLayerType == Strip)
            {
                slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
            }
            else
            {
                slopes[i] = modulesInGPU.slopes[modulesInGPU.partnerModuleIndex(lowerModuleIndices[i])];

            }
            isFlat[i] = false;

            /*despite the type of the module layer of the lower module index,
            all anchor hits are on the pixel side and all non-anchor hits are
            on the strip side!*/
            if(anchorHits)
            {
                delta2[i] = inv2;//16.6666f;//0.15f;
                //delta2[i] = 0.15f;
            }
            else
            {
                //delta2[i] = 2.4f;
                delta2[i] = inv3;//266.666f;//2.4f;
            }
        }

        //category 5 - endcap 2S
        else if(moduleSubdet == Endcap and moduleType == TwoS)
        {
            //delta1[i] = 0.009;
            //delta2[i] = 5.f;
            delta1[i] = 1.f;//0.009;
            delta2[i] = 500.f*inv1;//555.5555f;//5.f;
            slopes[i] = modulesInGPU.slopes[lowerModuleIndices[i]];
            isFlat[i] = false;
        }
        else
        {
            printf("ERROR!!!!! I SHOULDN'T BE HERE!!!! subdet = %d, type = %d, side = %d\n", moduleSubdet, moduleType, moduleSide);
        }
    }
    //divide everyone by the smallest possible values of delta1 and delta2
//    for(size_t i = 0; i < 5; i++)
//    {
//        delta1[i] /= 0.009;
//        delta2[i] /= 0.009;
//    }
}
