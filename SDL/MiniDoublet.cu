#include "hip/hip_runtime.h"
#include "MiniDoublet.cuh"

void SDL::miniDoublets::resetMemory(unsigned int nMemoryLocationsx, unsigned int nLowerModules,hipStream_t stream)
{
    hipMemsetAsync(anchorHitIndices,0, nMemoryLocationsx * 3 * sizeof(unsigned int),stream);
    hipMemsetAsync(dphichanges,0, nMemoryLocationsx * 9 * sizeof(float),stream);
    hipMemsetAsync(nMDs,0, (nLowerModules + 1) * sizeof(int),stream);
    hipMemsetAsync(totOccupancyMDs,0, (nLowerModules + 1) * sizeof(unsigned int),stream);
}

__global__ void SDL::createMDArrayRangesGPU(struct modules& modulesInGPU, struct objectRanges& rangesInGPU)
{
    short module_subdets;
    short module_layers;
    short module_rings;
    float module_eta;

    __shared__ unsigned int nTotalMDs; //start!   
    nTotalMDs = 0; //start!   
    __syncthreads();
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int np = gridDim.x * blockDim.x;
    for(uint16_t i = gid; i < *modulesInGPU.nLowerModules; i+= np)
    {
        module_subdets = modulesInGPU.subdets[i];
        module_layers = modulesInGPU.layers[i];
        module_rings = modulesInGPU.rings[i];
        module_eta = abs(modulesInGPU.eta[i]);
        unsigned int occupancy;
        unsigned int category_number, eta_number;
        if (module_layers<=3 && module_subdets==5) category_number = 0;
        else if (module_layers>=4 && module_subdets==5) category_number = 1;
        else if (module_layers<=2 && module_subdets==4 && module_rings>=11) category_number = 2;
        else if (module_layers>=3 && module_subdets==4 && module_rings>=8) category_number = 2;
        else if (module_layers<=2 && module_subdets==4 && module_rings<=10) category_number = 3;
        else if (module_layers>=3 && module_subdets==4 && module_rings<=7) category_number = 3;

        if (module_eta<0.75) eta_number=0;
        else if (module_eta>0.75 && module_eta<1.5) eta_number=1;
        else if (module_eta>1.5  && module_eta<2.25) eta_number=2;
        else if (module_eta>2.25 && module_eta<3) eta_number=3;

        if (category_number == 0 && eta_number == 0) occupancy = 49;
        else if (category_number == 0 && eta_number == 1) occupancy = 42;
        else if (category_number == 0 && eta_number == 2) occupancy = 37;
        else if (category_number == 0 && eta_number == 3) occupancy = 41;
        else if (category_number == 1) occupancy = 100;
        else if (category_number == 2 && eta_number == 1) occupancy = 16;
        else if (category_number == 2 && eta_number == 2) occupancy = 19;
        else if (category_number == 3 && eta_number == 1) occupancy = 14;
        else if (category_number == 3 && eta_number == 2) occupancy = 20;
        else if (category_number == 3 && eta_number == 3) occupancy = 25;

        unsigned int nTotMDs= atomicAdd(&nTotalMDs,occupancy);
        rangesInGPU.miniDoubletModuleIndices[i] = nTotMDs; 
        rangesInGPU.miniDoubletModuleOccupancy[i] = occupancy;
    }
    __syncthreads();
    if(threadIdx.x==0){
      rangesInGPU.miniDoubletModuleIndices[*modulesInGPU.nLowerModules] = nTotalMDs;
      //*nTotalMDsx=nTotalMDs;
      *rangesInGPU.device_nTotalMDs=nTotalMDs;
    }

}

//FIXME:Add memory locations for the pixel MDs here!
void SDL::createMDsInExplicitMemory(struct miniDoublets& mdsInGPU, unsigned int nMemoryLocations, uint16_t nLowerModules, unsigned int maxPixelMDs,hipStream_t stream)
{
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    mdsInGPU.anchorHitIndices = (unsigned int*)cms::cuda::allocate_device(dev,nMemoryLocations * 2 * sizeof(unsigned int), stream);
    mdsInGPU.moduleIndices = (uint16_t*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(uint16_t), stream);
    mdsInGPU.dphichanges = (float*)cms::cuda::allocate_device(dev,nMemoryLocations*9*sizeof(float),stream);
    mdsInGPU.nMDs = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(int),stream);
    mdsInGPU.totOccupancyMDs = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(int),stream);
    mdsInGPU.anchorX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 6 * sizeof(float), stream);
    mdsInGPU.anchorHighEdgeX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 4 * sizeof(float), stream);
    mdsInGPU.outerX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 6 * sizeof(float), stream);
    mdsInGPU.outerHighEdgeX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 4 * sizeof(float), stream);
    mdsInGPU.nMemoryLocations = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);
#else
    hipMalloc(&mdsInGPU.anchorHitIndices, nMemoryLocations * 2 * sizeof(unsigned int));
    hipMalloc(&mdsInGPU.moduleIndices, nMemoryLocations * sizeof(uint16_t));
    hipMalloc(&mdsInGPU.dphichanges, nMemoryLocations *9* sizeof(float));
    hipMalloc(&mdsInGPU.nMDs, (nLowerModules + 1) * sizeof(int)); 
    hipMalloc(&mdsInGPU.totOccupancyMDs, (nLowerModules + 1) * sizeof(int)); 
    hipMalloc(&mdsInGPU.anchorX, nMemoryLocations * 6 * sizeof(float));
    hipMalloc(&mdsInGPU.anchorHighEdgeX, nMemoryLocations * 4 * sizeof(float));
    hipMalloc(&mdsInGPU.outerX, nMemoryLocations * 6 * sizeof(float));
    hipMalloc(&mdsInGPU.outerHighEdgeX, nMemoryLocations * 4 * sizeof(float));
    hipMalloc(&mdsInGPU.nMemoryLocations, sizeof(unsigned int));
#endif
    hipMemsetAsync(mdsInGPU.nMDs,0, (nLowerModules + 1) *sizeof(int),stream);
    hipMemsetAsync(mdsInGPU.totOccupancyMDs,0, (nLowerModules + 1) *sizeof(int),stream);
    hipStreamSynchronize(stream);

    mdsInGPU.outerHitIndices = mdsInGPU.anchorHitIndices + nMemoryLocations;
    mdsInGPU.dzs  = mdsInGPU.dphichanges + nMemoryLocations;
    mdsInGPU.dphis  = mdsInGPU.dphichanges + 2*nMemoryLocations;
    mdsInGPU.shiftedXs  = mdsInGPU.dphichanges + 3*nMemoryLocations;
    mdsInGPU.shiftedYs  = mdsInGPU.dphichanges + 4*nMemoryLocations;
    mdsInGPU.shiftedZs  = mdsInGPU.dphichanges + 5*nMemoryLocations;
    mdsInGPU.noShiftedDzs  = mdsInGPU.dphichanges + 6*nMemoryLocations;
    mdsInGPU.noShiftedDphis  = mdsInGPU.dphichanges + 7*nMemoryLocations;
    mdsInGPU.noShiftedDphiChanges  = mdsInGPU.dphichanges + 8*nMemoryLocations;

    mdsInGPU.anchorY = mdsInGPU.anchorX + nMemoryLocations;
    mdsInGPU.anchorZ = mdsInGPU.anchorX + 2 * nMemoryLocations;
    mdsInGPU.anchorRt = mdsInGPU.anchorX + 3 * nMemoryLocations;
    mdsInGPU.anchorPhi = mdsInGPU.anchorX + 4 * nMemoryLocations;
    mdsInGPU.anchorEta = mdsInGPU.anchorX + 5 * nMemoryLocations;

    mdsInGPU.anchorHighEdgeY = mdsInGPU.anchorHighEdgeX + nMemoryLocations;
    mdsInGPU.anchorLowEdgeX = mdsInGPU.anchorHighEdgeX + 2 * nMemoryLocations;
    mdsInGPU.anchorLowEdgeY = mdsInGPU.anchorHighEdgeX + 3 * nMemoryLocations;

    mdsInGPU.outerY = mdsInGPU.outerX + nMemoryLocations;
    mdsInGPU.outerZ = mdsInGPU.outerX + 2 * nMemoryLocations;
    mdsInGPU.outerRt = mdsInGPU.outerX + 3 * nMemoryLocations;
    mdsInGPU.outerPhi = mdsInGPU.outerX + 4 * nMemoryLocations;
    mdsInGPU.outerEta = mdsInGPU.outerX + 5 * nMemoryLocations;

    mdsInGPU.outerHighEdgeY = mdsInGPU.outerHighEdgeX + nMemoryLocations;
    mdsInGPU.outerLowEdgeX = mdsInGPU.outerHighEdgeX + 2 * nMemoryLocations;
    mdsInGPU.outerLowEdgeY = mdsInGPU.outerHighEdgeX + 3 * nMemoryLocations;
}

SDL::miniDoublets::miniDoublets()
{
    anchorHitIndices = nullptr;
    outerHitIndices = nullptr;
    moduleIndices = nullptr;
    nMDs = nullptr;
    totOccupancyMDs = nullptr;
    dphichanges = nullptr;

    dzs = nullptr;
    dphis = nullptr;

    shiftedXs = nullptr;
    shiftedYs = nullptr;
    shiftedZs = nullptr;
    noShiftedDzs = nullptr;
    noShiftedDphis = nullptr;
    noShiftedDphiChanges = nullptr;
    
    anchorX = nullptr;
    anchorY = nullptr;
    anchorZ = nullptr;
    anchorRt = nullptr;
    anchorPhi = nullptr;
    anchorEta = nullptr;
    anchorHighEdgeX = nullptr;
    anchorHighEdgeY = nullptr;
    anchorLowEdgeX = nullptr;
    anchorLowEdgeY = nullptr;
    outerX = nullptr;
    outerY = nullptr;
    outerZ = nullptr;
    outerRt = nullptr;
    outerPhi = nullptr;
    outerEta = nullptr;
    outerHighEdgeX = nullptr;
    outerHighEdgeY = nullptr;
    outerLowEdgeX = nullptr;
    outerLowEdgeY = nullptr;
}

SDL::miniDoublets::~miniDoublets()
{
}

void SDL::miniDoublets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,anchorHitIndices);
    cms::cuda::free_device(dev, moduleIndices);
    cms::cuda::free_device(dev,dphichanges);
    cms::cuda::free_device(dev,nMDs);
    cms::cuda::free_device(dev,totOccupancyMDs);
    cms::cuda::free_device(dev, anchorX);
    cms::cuda::free_device(dev, anchorHighEdgeX);
    cms::cuda::free_device(dev, outerX);
    cms::cuda::free_device(dev, outerHighEdgeX);
    cms::cuda::free_device(dev, nMemoryLocations);
}

void SDL::miniDoublets::freeMemory(hipStream_t stream)
{
    hipFree(anchorHitIndices);
    hipFree(moduleIndices);
    hipFree(nMDs);
    hipFree(totOccupancyMDs);
    hipFree(dphichanges);
    hipFree(anchorX);
    hipFree(anchorHighEdgeX);
    hipFree(outerX);
    hipFree(outerHighEdgeX);
    hipFree(nMemoryLocations);
}

void SDL::printMD(struct miniDoublets& mdsInGPU, struct hits& hitsInGPU, SDL::modules& modulesInGPU, unsigned int mdIndex)
{
    std::cout<<std::endl;
    std::cout << "dz " << mdsInGPU.dzs[mdIndex] << std::endl;
    std::cout << "dphi " << mdsInGPU.dphis[mdIndex] << std::endl;
    std::cout << "dphinoshift " << mdsInGPU.noShiftedDphis[mdIndex] << std::endl;
    std::cout << "dphichange " << mdsInGPU.dphichanges[mdIndex] << std::endl;
    std::cout << "dphichangenoshift " << mdsInGPU.noShiftedDphiChanges[mdIndex] << std::endl;
    std::cout << std::endl;
    std::cout << "Anchor Hit " << std::endl;
    std::cout << "------------------------------" << std::endl;
    unsigned int lowerHitIndex = mdsInGPU.anchorHitIndices[mdIndex];
    unsigned int upperHitIndex = mdsInGPU.outerHitIndices[mdIndex];
    {
        IndentingOStreambuf indent(std::cout);
        printHit(hitsInGPU, modulesInGPU, lowerHitIndex);
    }
    std::cout << "Non-anchor Hit " << std::endl;
    std::cout << "------------------------------" << std::endl;
    {
        IndentingOStreambuf indent(std::cout);
        printHit(hitsInGPU, modulesInGPU, upperHitIndex);
    }
}

__global__ void SDL::addMiniDoubletRangesToEventExplicit(struct modules& modulesInGPU, struct miniDoublets& mdsInGPU, struct objectRanges& rangesInGPU,struct hits& hitsInGPU)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int np = gridDim.x * blockDim.x;
    for(uint16_t i = gid; i < *modulesInGPU.nLowerModules; i+= np)
    {
        if(mdsInGPU.nMDs[i] == 0 or hitsInGPU.hitRanges[i * 2] == -1)
        {
            rangesInGPU.mdRanges[i * 2] = -1;
            rangesInGPU.mdRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU.mdRanges[i * 2] = rangesInGPU.miniDoubletModuleIndices[i] ;
            rangesInGPU.mdRanges[i * 2 + 1] = rangesInGPU.miniDoubletModuleIndices[i] + mdsInGPU.nMDs[i] - 1;
        }
    }
}
