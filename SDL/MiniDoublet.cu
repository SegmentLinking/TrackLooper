#include "MiniDoublet.cuh"

void SDL::miniDoublets::resetMemory(unsigned int nMemoryLocationsx, unsigned int nLowerModules,hipStream_t stream)
{
    hipMemsetAsync(anchorHitIndices,0, nMemoryLocationsx * 3 * sizeof(unsigned int),stream);
    hipMemsetAsync(dphichanges,0, nMemoryLocationsx * 9 * sizeof(float),stream);
    hipMemsetAsync(nMDs,0, (nLowerModules + 1) * sizeof(int),stream);
    hipMemsetAsync(totOccupancyMDs,0, (nLowerModules + 1) * sizeof(unsigned int),stream);
}

//FIXME:Add memory locations for the pixel MDs here!
void SDL::createMDsInExplicitMemory(struct miniDoublets& mdsInGPU, unsigned int nMemoryLocations, uint16_t nLowerModules, unsigned int maxPixelMDs,hipStream_t stream)
{
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    mdsInGPU.anchorHitIndices = (unsigned int*)cms::cuda::allocate_device(dev,nMemoryLocations * 2 * sizeof(unsigned int), stream);
    mdsInGPU.moduleIndices = (uint16_t*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(uint16_t), stream);
    mdsInGPU.dphichanges = (float*)cms::cuda::allocate_device(dev,nMemoryLocations*9*sizeof(float),stream);
    mdsInGPU.nMDs = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(int),stream);
    mdsInGPU.totOccupancyMDs = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(int),stream);
    mdsInGPU.anchorX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 6 * sizeof(float), stream);
    mdsInGPU.anchorHighEdgeX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 4 * sizeof(float), stream);
    mdsInGPU.outerX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 6 * sizeof(float), stream);
    mdsInGPU.outerHighEdgeX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 4 * sizeof(float), stream);
    mdsInGPU.nMemoryLocations = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);
#else
    hipMalloc(&mdsInGPU.anchorHitIndices, nMemoryLocations * 2 * sizeof(unsigned int));
    hipMalloc(&mdsInGPU.moduleIndices, nMemoryLocations * sizeof(uint16_t));
    hipMalloc(&mdsInGPU.dphichanges, nMemoryLocations *9* sizeof(float));
    hipMalloc(&mdsInGPU.nMDs, (nLowerModules + 1) * sizeof(int)); 
    hipMalloc(&mdsInGPU.totOccupancyMDs, (nLowerModules + 1) * sizeof(int)); 
    hipMalloc(&mdsInGPU.anchorX, nMemoryLocations * 6 * sizeof(float));
    hipMalloc(&mdsInGPU.anchorHighEdgeX, nMemoryLocations * 4 * sizeof(float));
    hipMalloc(&mdsInGPU.outerX, nMemoryLocations * 6 * sizeof(float));
    hipMalloc(&mdsInGPU.outerHighEdgeX, nMemoryLocations * 4 * sizeof(float));
    hipMalloc(&mdsInGPU.nMemoryLocations, sizeof(unsigned int));
#endif
    hipMemsetAsync(mdsInGPU.nMDs,0, (nLowerModules + 1) *sizeof(int),stream);
    hipMemsetAsync(mdsInGPU.totOccupancyMDs,0, (nLowerModules + 1) *sizeof(int),stream);
    hipStreamSynchronize(stream);

    mdsInGPU.outerHitIndices = mdsInGPU.anchorHitIndices + nMemoryLocations;
    mdsInGPU.dzs  = mdsInGPU.dphichanges + nMemoryLocations;
    mdsInGPU.dphis  = mdsInGPU.dphichanges + 2*nMemoryLocations;
    mdsInGPU.shiftedXs  = mdsInGPU.dphichanges + 3*nMemoryLocations;
    mdsInGPU.shiftedYs  = mdsInGPU.dphichanges + 4*nMemoryLocations;
    mdsInGPU.shiftedZs  = mdsInGPU.dphichanges + 5*nMemoryLocations;
    mdsInGPU.noShiftedDzs  = mdsInGPU.dphichanges + 6*nMemoryLocations;
    mdsInGPU.noShiftedDphis  = mdsInGPU.dphichanges + 7*nMemoryLocations;
    mdsInGPU.noShiftedDphiChanges  = mdsInGPU.dphichanges + 8*nMemoryLocations;

    mdsInGPU.anchorY = mdsInGPU.anchorX + nMemoryLocations;
    mdsInGPU.anchorZ = mdsInGPU.anchorX + 2 * nMemoryLocations;
    mdsInGPU.anchorRt = mdsInGPU.anchorX + 3 * nMemoryLocations;
    mdsInGPU.anchorPhi = mdsInGPU.anchorX + 4 * nMemoryLocations;
    mdsInGPU.anchorEta = mdsInGPU.anchorX + 5 * nMemoryLocations;

    mdsInGPU.anchorHighEdgeY = mdsInGPU.anchorHighEdgeX + nMemoryLocations;
    mdsInGPU.anchorLowEdgeX = mdsInGPU.anchorHighEdgeX + 2 * nMemoryLocations;
    mdsInGPU.anchorLowEdgeY = mdsInGPU.anchorHighEdgeX + 3 * nMemoryLocations;

    mdsInGPU.outerY = mdsInGPU.outerX + nMemoryLocations;
    mdsInGPU.outerZ = mdsInGPU.outerX + 2 * nMemoryLocations;
    mdsInGPU.outerRt = mdsInGPU.outerX + 3 * nMemoryLocations;
    mdsInGPU.outerPhi = mdsInGPU.outerX + 4 * nMemoryLocations;
    mdsInGPU.outerEta = mdsInGPU.outerX + 5 * nMemoryLocations;

    mdsInGPU.outerHighEdgeY = mdsInGPU.outerHighEdgeX + nMemoryLocations;
    mdsInGPU.outerLowEdgeX = mdsInGPU.outerHighEdgeX + 2 * nMemoryLocations;
    mdsInGPU.outerLowEdgeY = mdsInGPU.outerHighEdgeX + 3 * nMemoryLocations;
}

SDL::miniDoublets::miniDoublets()
{
    anchorHitIndices = nullptr;
    outerHitIndices = nullptr;
    moduleIndices = nullptr;
    nMDs = nullptr;
    totOccupancyMDs = nullptr;
    dphichanges = nullptr;

    dzs = nullptr;
    dphis = nullptr;

    shiftedXs = nullptr;
    shiftedYs = nullptr;
    shiftedZs = nullptr;
    noShiftedDzs = nullptr;
    noShiftedDphis = nullptr;
    noShiftedDphiChanges = nullptr;
    
    anchorX = nullptr;
    anchorY = nullptr;
    anchorZ = nullptr;
    anchorRt = nullptr;
    anchorPhi = nullptr;
    anchorEta = nullptr;
    anchorHighEdgeX = nullptr;
    anchorHighEdgeY = nullptr;
    anchorLowEdgeX = nullptr;
    anchorLowEdgeY = nullptr;
    outerX = nullptr;
    outerY = nullptr;
    outerZ = nullptr;
    outerRt = nullptr;
    outerPhi = nullptr;
    outerEta = nullptr;
    outerHighEdgeX = nullptr;
    outerHighEdgeY = nullptr;
    outerLowEdgeX = nullptr;
    outerLowEdgeY = nullptr;
}

SDL::miniDoublets::~miniDoublets()
{
}

void SDL::miniDoublets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,anchorHitIndices);
    cms::cuda::free_device(dev, moduleIndices);
    cms::cuda::free_device(dev,dphichanges);
    cms::cuda::free_device(dev,nMDs);
    cms::cuda::free_device(dev,totOccupancyMDs);
    cms::cuda::free_device(dev, anchorX);
    cms::cuda::free_device(dev, anchorHighEdgeX);
    cms::cuda::free_device(dev, outerX);
    cms::cuda::free_device(dev, outerHighEdgeX);
    cms::cuda::free_device(dev, nMemoryLocations);
}

void SDL::miniDoublets::freeMemory(hipStream_t stream)
{
    hipFree(anchorHitIndices);
    hipFree(moduleIndices);
    hipFree(nMDs);
    hipFree(totOccupancyMDs);
    hipFree(dphichanges);
    hipFree(anchorX);
    hipFree(anchorHighEdgeX);
    hipFree(outerX);
    hipFree(outerHighEdgeX);
    hipFree(nMemoryLocations);
}

void SDL::printMD(struct miniDoublets& mdsInGPU, struct hits& hitsInGPU, SDL::modules& modulesInGPU, unsigned int mdIndex)
{
    std::cout<<std::endl;
    std::cout << "dz " << mdsInGPU.dzs[mdIndex] << std::endl;
    std::cout << "dphi " << mdsInGPU.dphis[mdIndex] << std::endl;
    std::cout << "dphinoshift " << mdsInGPU.noShiftedDphis[mdIndex] << std::endl;
    std::cout << "dphichange " << mdsInGPU.dphichanges[mdIndex] << std::endl;
    std::cout << "dphichangenoshift " << mdsInGPU.noShiftedDphiChanges[mdIndex] << std::endl;
    std::cout << std::endl;
    std::cout << "Anchor Hit " << std::endl;
    std::cout << "------------------------------" << std::endl;
    unsigned int lowerHitIndex = mdsInGPU.anchorHitIndices[mdIndex];
    unsigned int upperHitIndex = mdsInGPU.outerHitIndices[mdIndex];
    {
        IndentingOStreambuf indent(std::cout);
        printHit(hitsInGPU, modulesInGPU, lowerHitIndex);
    }
    std::cout << "Non-anchor Hit " << std::endl;
    std::cout << "------------------------------" << std::endl;
    {
        IndentingOStreambuf indent(std::cout);
        printHit(hitsInGPU, modulesInGPU, upperHitIndex);
    }
}
