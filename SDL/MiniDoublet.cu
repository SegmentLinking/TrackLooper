#include "MiniDoublet.cuh"

//FIXME:Add memory locations for the pixel MDs here!
void SDL::createMDsInExplicitMemory(struct miniDoublets& mdsInGPU, unsigned int nMemoryLocations, uint16_t nLowerModules, unsigned int maxPixelMDs,hipStream_t stream)
{
#ifdef CACHE_ALLOC
    int dev;
    hipGetDevice(&dev);
    mdsInGPU.anchorHitIndices = (unsigned int*)cms::cuda::allocate_device(dev,nMemoryLocations * 2 * sizeof(unsigned int), stream);
    mdsInGPU.moduleIndices = (uint16_t*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(uint16_t), stream);
    mdsInGPU.dphichanges = (float*)cms::cuda::allocate_device(dev,nMemoryLocations*9*sizeof(float),stream);
    mdsInGPU.nMDs = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(int),stream);
    mdsInGPU.totOccupancyMDs = (int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(int),stream);
    mdsInGPU.anchorX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 6 * sizeof(float), stream);
    mdsInGPU.anchorHighEdgeX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 4 * sizeof(float), stream);
    mdsInGPU.outerX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 6 * sizeof(float), stream);
    mdsInGPU.outerHighEdgeX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 4 * sizeof(float), stream);
    mdsInGPU.nMemoryLocations = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);
#else
    hipMalloc(&mdsInGPU.anchorHitIndices, nMemoryLocations * 2 * sizeof(unsigned int));
    hipMalloc(&mdsInGPU.moduleIndices, nMemoryLocations * sizeof(uint16_t));
    hipMalloc(&mdsInGPU.dphichanges, nMemoryLocations *9* sizeof(float));
    hipMalloc(&mdsInGPU.nMDs, (nLowerModules + 1) * sizeof(int)); 
    hipMalloc(&mdsInGPU.totOccupancyMDs, (nLowerModules + 1) * sizeof(int)); 
    hipMalloc(&mdsInGPU.anchorX, nMemoryLocations * 6 * sizeof(float));
    hipMalloc(&mdsInGPU.anchorHighEdgeX, nMemoryLocations * 4 * sizeof(float));
    hipMalloc(&mdsInGPU.outerX, nMemoryLocations * 6 * sizeof(float));
    hipMalloc(&mdsInGPU.outerHighEdgeX, nMemoryLocations * 4 * sizeof(float));
    hipMalloc(&mdsInGPU.nMemoryLocations, sizeof(unsigned int));
#endif
    hipMemsetAsync(mdsInGPU.nMDs,0, (nLowerModules + 1) *sizeof(int),stream);
    hipMemsetAsync(mdsInGPU.totOccupancyMDs,0, (nLowerModules + 1) *sizeof(int),stream);
    hipStreamSynchronize(stream);

    mdsInGPU.outerHitIndices = mdsInGPU.anchorHitIndices + nMemoryLocations;
    mdsInGPU.dzs  = mdsInGPU.dphichanges + nMemoryLocations;
    mdsInGPU.dphis  = mdsInGPU.dphichanges + 2*nMemoryLocations;
    mdsInGPU.shiftedXs  = mdsInGPU.dphichanges + 3*nMemoryLocations;
    mdsInGPU.shiftedYs  = mdsInGPU.dphichanges + 4*nMemoryLocations;
    mdsInGPU.shiftedZs  = mdsInGPU.dphichanges + 5*nMemoryLocations;
    mdsInGPU.noShiftedDzs  = mdsInGPU.dphichanges + 6*nMemoryLocations;
    mdsInGPU.noShiftedDphis  = mdsInGPU.dphichanges + 7*nMemoryLocations;
    mdsInGPU.noShiftedDphiChanges  = mdsInGPU.dphichanges + 8*nMemoryLocations;

    mdsInGPU.anchorY = mdsInGPU.anchorX + nMemoryLocations;
    mdsInGPU.anchorZ = mdsInGPU.anchorX + 2 * nMemoryLocations;
    mdsInGPU.anchorRt = mdsInGPU.anchorX + 3 * nMemoryLocations;
    mdsInGPU.anchorPhi = mdsInGPU.anchorX + 4 * nMemoryLocations;
    mdsInGPU.anchorEta = mdsInGPU.anchorX + 5 * nMemoryLocations;

    mdsInGPU.anchorHighEdgeY = mdsInGPU.anchorHighEdgeX + nMemoryLocations;
    mdsInGPU.anchorLowEdgeX = mdsInGPU.anchorHighEdgeX + 2 * nMemoryLocations;
    mdsInGPU.anchorLowEdgeY = mdsInGPU.anchorHighEdgeX + 3 * nMemoryLocations;

    mdsInGPU.outerY = mdsInGPU.outerX + nMemoryLocations;
    mdsInGPU.outerZ = mdsInGPU.outerX + 2 * nMemoryLocations;
    mdsInGPU.outerRt = mdsInGPU.outerX + 3 * nMemoryLocations;
    mdsInGPU.outerPhi = mdsInGPU.outerX + 4 * nMemoryLocations;
    mdsInGPU.outerEta = mdsInGPU.outerX + 5 * nMemoryLocations;

    mdsInGPU.outerHighEdgeY = mdsInGPU.outerHighEdgeX + nMemoryLocations;
    mdsInGPU.outerLowEdgeX = mdsInGPU.outerHighEdgeX + 2 * nMemoryLocations;
    mdsInGPU.outerLowEdgeY = mdsInGPU.outerHighEdgeX + 3 * nMemoryLocations;
}

SDL::miniDoublets::miniDoublets()
{
    anchorHitIndices = nullptr;
    outerHitIndices = nullptr;
    moduleIndices = nullptr;
    nMDs = nullptr;
    totOccupancyMDs = nullptr;
    dphichanges = nullptr;

    dzs = nullptr;
    dphis = nullptr;

    shiftedXs = nullptr;
    shiftedYs = nullptr;
    shiftedZs = nullptr;
    noShiftedDzs = nullptr;
    noShiftedDphis = nullptr;
    noShiftedDphiChanges = nullptr;
    
    anchorX = nullptr;
    anchorY = nullptr;
    anchorZ = nullptr;
    anchorRt = nullptr;
    anchorPhi = nullptr;
    anchorEta = nullptr;
    anchorHighEdgeX = nullptr;
    anchorHighEdgeY = nullptr;
    anchorLowEdgeX = nullptr;
    anchorLowEdgeY = nullptr;
    outerX = nullptr;
    outerY = nullptr;
    outerZ = nullptr;
    outerRt = nullptr;
    outerPhi = nullptr;
    outerEta = nullptr;
    outerHighEdgeX = nullptr;
    outerHighEdgeY = nullptr;
    outerLowEdgeX = nullptr;
    outerLowEdgeY = nullptr;
}

SDL::miniDoublets::~miniDoublets()
{
}

void SDL::miniDoublets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,anchorHitIndices);
    cms::cuda::free_device(dev, moduleIndices);
    cms::cuda::free_device(dev,dphichanges);
    cms::cuda::free_device(dev,nMDs);
    cms::cuda::free_device(dev,totOccupancyMDs);
    cms::cuda::free_device(dev, anchorX);
    cms::cuda::free_device(dev, anchorHighEdgeX);
    cms::cuda::free_device(dev, outerX);
    cms::cuda::free_device(dev, outerHighEdgeX);
    cms::cuda::free_device(dev, nMemoryLocations);
}

void SDL::miniDoublets::freeMemory(hipStream_t stream)
{
    hipFree(anchorHitIndices);
    hipFree(moduleIndices);
    hipFree(nMDs);
    hipFree(totOccupancyMDs);
    hipFree(dphichanges);
    hipFree(anchorX);
    hipFree(anchorHighEdgeX);
    hipFree(outerX);
    hipFree(outerHighEdgeX);
    hipFree(nMemoryLocations);
}
