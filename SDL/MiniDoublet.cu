#include "hip/hip_runtime.h"
# include "MiniDoublet.cuh"
#define SDL_INF 123456789

#include "allocate.h"

void SDL::miniDoublets::resetMemory(unsigned int nMemoryLocationsx, unsigned int nLowerModules,hipStream_t stream)

{
    hipMemsetAsync(anchorHitIndices,0, nMemoryLocationsx * 3 * sizeof(unsigned int),stream);
    hipMemsetAsync(dphichanges,0, nMemoryLocationsx * 9 * sizeof(float),stream);
    hipMemsetAsync(nMDs,0, (nLowerModules + 1) * sizeof(unsigned int),stream);
    hipMemsetAsync(totOccupancyMDs,0, (nLowerModules + 1) * sizeof(unsigned int),stream);
}


__global__ void SDL::createMDArrayRangesGPU(struct modules& modulesInGPU, struct objectRanges& rangesInGPU)//, unsigned int* nTotalMDsx)
{
    short module_subdets;
    short module_layers;
    short module_rings;
    float module_eta;

    __shared__ unsigned int nTotalMDs; //start!   
    nTotalMDs = 0; //start!   
    __syncthreads();
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int np = gridDim.x * blockDim.x;
    for(uint16_t i = gid; i < *modulesInGPU.nLowerModules; i+= np)
    {
        module_subdets = modulesInGPU.subdets[i];
        module_layers = modulesInGPU.layers[i];
        module_rings = modulesInGPU.rings[i];
        module_eta = abs(modulesInGPU.eta[i]);
        unsigned int occupancy;
        unsigned int category_number, eta_number;
        if (module_layers<=3 && module_subdets==5) category_number = 0;
        else if (module_layers>=4 && module_subdets==5) category_number = 1;
        else if (module_layers<=2 && module_subdets==4 && module_rings>=11) category_number = 2;
        else if (module_layers>=3 && module_subdets==4 && module_rings>=8) category_number = 2;
        else if (module_layers<=2 && module_subdets==4 && module_rings<=10) category_number = 3;
        else if (module_layers>=3 && module_subdets==4 && module_rings<=7) category_number = 3;

        if (module_eta<0.75) eta_number=0;
        else if (module_eta>0.75 && module_eta<1.5) eta_number=1;
        else if (module_eta>1.5  && module_eta<2.25) eta_number=2;
        else if (module_eta>2.25 && module_eta<3) eta_number=3;

        if (category_number == 0 && eta_number == 0) occupancy = 49;
        else if (category_number == 0 && eta_number == 1) occupancy = 42;
        else if (category_number == 0 && eta_number == 2) occupancy = 37;
        else if (category_number == 0 && eta_number == 3) occupancy = 41;
        else if (category_number == 1) occupancy = 100;
        else if (category_number == 2 && eta_number == 1) occupancy = 16;
        else if (category_number == 2 && eta_number == 2) occupancy = 19;
        else if (category_number == 3 && eta_number == 1) occupancy = 14;
        else if (category_number == 3 && eta_number == 2) occupancy = 20;
        else if (category_number == 3 && eta_number == 3) occupancy = 25;

        unsigned int nTotMDs= atomicAdd(&nTotalMDs,occupancy);
        rangesInGPU.miniDoubletModuleIndices[i] = nTotMDs; 
        rangesInGPU.miniDoubletModuleOccupancy[i] = occupancy;
    }
    __syncthreads();
    if(threadIdx.x==0){
      rangesInGPU.miniDoubletModuleIndices[*modulesInGPU.nLowerModules] = nTotalMDs;
      //*nTotalMDsx=nTotalMDs;
      *rangesInGPU.device_nTotalMDs=nTotalMDs;
    }

}

//FIXME:Add memory locations for the pixel MDs here!
void SDL::createMDsInExplicitMemory(struct miniDoublets& mdsInGPU, unsigned int nMemoryLocations, uint16_t nLowerModules, unsigned int maxPixelMDs,hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream=0;
    int dev;
    hipGetDevice(&dev);
    mdsInGPU.anchorHitIndices = (unsigned int*)cms::cuda::allocate_device(dev,nMemoryLocations * 2 * sizeof(unsigned int), stream);
    mdsInGPU.moduleIndices = (uint16_t*)cms::cuda::allocate_device(dev, nMemoryLocations * sizeof(uint16_t), stream);
    mdsInGPU.dphichanges = (float*)cms::cuda::allocate_device(dev,nMemoryLocations*9*sizeof(float),stream);
    mdsInGPU.nMDs = (unsigned int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(unsigned int),stream);
    mdsInGPU.totOccupancyMDs = (unsigned int*)cms::cuda::allocate_device(dev, (nLowerModules + 1) *sizeof(unsigned int),stream);
    mdsInGPU.anchorX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 6 * sizeof(float), stream);
    mdsInGPU.anchorHighEdgeX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 4 * sizeof(float), stream);
    mdsInGPU.outerX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 6 * sizeof(float), stream);
    mdsInGPU.outerHighEdgeX = (float*)cms::cuda::allocate_device(dev, nMemoryLocations * 4 * sizeof(float), stream);

    mdsInGPU.nMemoryLocations = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int), stream);

#else
    hipMalloc(&mdsInGPU.anchorHitIndices, nMemoryLocations * 2 * sizeof(unsigned int));
    hipMalloc(&mdsInGPU.moduleIndices, nMemoryLocations * sizeof(uint16_t));
    hipMalloc(&mdsInGPU.dphichanges, nMemoryLocations *9* sizeof(float));
    hipMalloc(&mdsInGPU.nMDs, (nLowerModules + 1) * sizeof(unsigned int)); 
    hipMalloc(&mdsInGPU.totOccupancyMDs, (nLowerModules + 1) * sizeof(unsigned int)); 
    hipMalloc(&mdsInGPU.anchorX, nMemoryLocations * 6 * sizeof(float));
    hipMalloc(&mdsInGPU.anchorHighEdgeX, nMemoryLocations * 4 * sizeof(float));
    hipMalloc(&mdsInGPU.outerX, nMemoryLocations * 6 * sizeof(float));
    hipMalloc(&mdsInGPU.outerHighEdgeX, nMemoryLocations * 4 * sizeof(float));

    hipMalloc(&mdsInGPU.nMemoryLocations, sizeof(unsigned int));

#endif
    hipMemsetAsync(mdsInGPU.nMDs,0, (nLowerModules + 1) *sizeof(unsigned int),stream);
    hipMemsetAsync(mdsInGPU.totOccupancyMDs,0, (nLowerModules + 1) *sizeof(unsigned int),stream);
    hipStreamSynchronize(stream);

    mdsInGPU.outerHitIndices = mdsInGPU.anchorHitIndices + nMemoryLocations;
    mdsInGPU.dzs  = mdsInGPU.dphichanges + nMemoryLocations;
    mdsInGPU.dphis  = mdsInGPU.dphichanges + 2*nMemoryLocations;
    mdsInGPU.shiftedXs  = mdsInGPU.dphichanges + 3*nMemoryLocations;
    mdsInGPU.shiftedYs  = mdsInGPU.dphichanges + 4*nMemoryLocations;
    mdsInGPU.shiftedZs  = mdsInGPU.dphichanges + 5*nMemoryLocations;
    mdsInGPU.noShiftedDzs  = mdsInGPU.dphichanges + 6*nMemoryLocations;
    mdsInGPU.noShiftedDphis  = mdsInGPU.dphichanges + 7*nMemoryLocations;
    mdsInGPU.noShiftedDphiChanges  = mdsInGPU.dphichanges + 8*nMemoryLocations;

    mdsInGPU.anchorY = mdsInGPU.anchorX + nMemoryLocations;
    mdsInGPU.anchorZ = mdsInGPU.anchorX + 2 * nMemoryLocations;
    mdsInGPU.anchorRt = mdsInGPU.anchorX + 3 * nMemoryLocations;
    mdsInGPU.anchorPhi = mdsInGPU.anchorX + 4 * nMemoryLocations;
    mdsInGPU.anchorEta = mdsInGPU.anchorX + 5 * nMemoryLocations;

    mdsInGPU.anchorHighEdgeY = mdsInGPU.anchorHighEdgeX + nMemoryLocations;
    mdsInGPU.anchorLowEdgeX = mdsInGPU.anchorHighEdgeX + 2 * nMemoryLocations;
    mdsInGPU.anchorLowEdgeY = mdsInGPU.anchorHighEdgeX + 3 * nMemoryLocations;

    mdsInGPU.outerY = mdsInGPU.outerX + nMemoryLocations;
    mdsInGPU.outerZ = mdsInGPU.outerX + 2 * nMemoryLocations;
    mdsInGPU.outerRt = mdsInGPU.outerX + 3 * nMemoryLocations;
    mdsInGPU.outerPhi = mdsInGPU.outerX + 4 * nMemoryLocations;
    mdsInGPU.outerEta = mdsInGPU.outerX + 5 * nMemoryLocations;

    mdsInGPU.outerHighEdgeY = mdsInGPU.outerHighEdgeX + nMemoryLocations;
    mdsInGPU.outerLowEdgeX = mdsInGPU.outerHighEdgeX + 2 * nMemoryLocations;
    mdsInGPU.outerLowEdgeY = mdsInGPU.outerHighEdgeX + 3 * nMemoryLocations;
}

//#ifdef CUT_VALUE_DEBUG
//ALPAKA_FN_ACC void SDL::addMDToMemory(struct miniDoublets& mdsInGPU, struct hits& hitsInGPU, struct modules& modulesInGPU, unsigned int lowerHitIdx, unsigned int upperHitIdx, uint16_t& lowerModuleIdx, float dz, float drt, float dPhi, float dPhiChange, float shiftedX, float shiftedY, float shiftedZ, float noShiftedDz, float noShiftedDphi, float noShiftedDPhiChange, float dzCut, float drtCut, float miniCut, unsigned int idx)
//#else
ALPAKA_FN_ACC void SDL::addMDToMemory(struct miniDoublets& mdsInGPU, struct hits& hitsInGPU, struct modules& modulesInGPU, unsigned int lowerHitIdx, unsigned int upperHitIdx, uint16_t& lowerModuleIdx, float dz, float dPhi, float dPhiChange, float shiftedX, float shiftedY, float shiftedZ, float noShiftedDz, float noShiftedDphi, float noShiftedDPhiChange, unsigned int idx)
//#endif
{
    //the index into which this MD needs to be written will be computed in the kernel
    //nMDs variable will be incremented in the kernel, no need to worry about that here
    
    mdsInGPU.moduleIndices[idx] = lowerModuleIdx;
    unsigned int anchorHitIndex, outerHitIndex;
    if(modulesInGPU.moduleType[lowerModuleIdx] == PS and modulesInGPU.moduleLayerType[lowerModuleIdx] == Strip)
    {
        mdsInGPU.anchorHitIndices[idx] = upperHitIdx;
        mdsInGPU.outerHitIndices[idx] = lowerHitIdx;

        anchorHitIndex = upperHitIdx;
        outerHitIndex = lowerHitIdx;
    }
    else
    {
        mdsInGPU.anchorHitIndices[idx] = lowerHitIdx;
        mdsInGPU.outerHitIndices[idx] = upperHitIdx;

        anchorHitIndex = lowerHitIdx;
        outerHitIndex = upperHitIdx;
    }

    mdsInGPU.dphichanges[idx] = dPhiChange;

    mdsInGPU.dphis[idx] = dPhi;
    mdsInGPU.dzs[idx] = dz;
    mdsInGPU.shiftedXs[idx] = shiftedX;
    mdsInGPU.shiftedYs[idx] = shiftedY;
    mdsInGPU.shiftedZs[idx] = shiftedZ;

    mdsInGPU.noShiftedDzs[idx] = noShiftedDz;
    mdsInGPU.noShiftedDphis[idx] = noShiftedDphi;
    mdsInGPU.noShiftedDphiChanges[idx] = noShiftedDPhiChange;
//#ifdef CUT_VALUE_DEBUG
//    mdsInGPU.dzCuts[idx] = dzCut;
//    mdsInGPU.drtCuts[idx] = drtCut;
//    mdsInGPU.miniCuts[idx] = miniCut;
//#endif

    mdsInGPU.anchorX[idx] = hitsInGPU.xs[anchorHitIndex];
    mdsInGPU.anchorY[idx] = hitsInGPU.ys[anchorHitIndex];
    mdsInGPU.anchorZ[idx] = hitsInGPU.zs[anchorHitIndex];
    mdsInGPU.anchorRt[idx] = hitsInGPU.rts[anchorHitIndex];
    mdsInGPU.anchorPhi[idx] = hitsInGPU.phis[anchorHitIndex];
    mdsInGPU.anchorEta[idx] = hitsInGPU.etas[anchorHitIndex];
    mdsInGPU.anchorHighEdgeX[idx] = hitsInGPU.highEdgeXs[anchorHitIndex];
    mdsInGPU.anchorHighEdgeY[idx] = hitsInGPU.highEdgeYs[anchorHitIndex];
    mdsInGPU.anchorLowEdgeX[idx] = hitsInGPU.lowEdgeXs[anchorHitIndex];
    mdsInGPU.anchorLowEdgeY[idx] = hitsInGPU.lowEdgeYs[anchorHitIndex];

    mdsInGPU.outerX[idx] = hitsInGPU.xs[outerHitIndex];
    mdsInGPU.outerY[idx] = hitsInGPU.ys[outerHitIndex];
    mdsInGPU.outerZ[idx] = hitsInGPU.zs[outerHitIndex];
    mdsInGPU.outerRt[idx] = hitsInGPU.rts[outerHitIndex];
    mdsInGPU.outerPhi[idx] = hitsInGPU.phis[outerHitIndex];
    mdsInGPU.outerEta[idx] = hitsInGPU.etas[outerHitIndex];
    mdsInGPU.outerHighEdgeX[idx] = hitsInGPU.highEdgeXs[outerHitIndex];
    mdsInGPU.outerHighEdgeY[idx] = hitsInGPU.highEdgeYs[outerHitIndex];
    mdsInGPU.outerLowEdgeX[idx] = hitsInGPU.lowEdgeXs[outerHitIndex];
    mdsInGPU.outerLowEdgeY[idx] = hitsInGPU.lowEdgeYs[outerHitIndex];
}

ALPAKA_FN_ACC bool SDL::runMiniDoubletDefaultAlgoBarrel(struct modules& modulesInGPU, /*struct hits& hitsInGPU,*/ uint16_t& lowerModuleIndex, uint16_t& upperModuleIndex, unsigned int lowerHitIndex, unsigned int upperHitIndex, float& dz, float& dPhi, float& dPhiChange, float& shiftedX, float& shiftedY, float& shiftedZ, float& noshiftedDz, float& noShiftedDphi, float& noShiftedDphiChange, float xLower,float yLower, float zLower, float rtLower,float xUpper,float yUpper,float zUpper,float rtUpper)
{

    bool pass = true; 
    dz = zLower - zUpper;     
    const float dzCut = modulesInGPU.moduleType[lowerModuleIndex] == PS ? 2.f : 10.f;
    //const float sign = ((dz > 0) - (dz < 0)) * ((hitsInGPU.zs[lowerHitIndex] > 0) - (hitsInGPU.zs[lowerHitIndex] < 0));
    const float sign = ((dz > 0) - (dz < 0)) * ((zLower > 0) - (zLower < 0));
    const float invertedcrossercut = (fabsf(dz) > 2) * sign;

    pass = pass  and ((fabsf(dz) < dzCut) & (invertedcrossercut <= 0));
    if(not pass) return pass;

    float miniCut = 0;

    miniCut = modulesInGPU.moduleLayerType[lowerModuleIndex] == Pixel ?  dPhiThreshold(rtLower, modulesInGPU, lowerModuleIndex) : dPhiThreshold(rtUpper, modulesInGPU, lowerModuleIndex); 

    // Cut #2: dphi difference
    // Ref to original code: https://github.com/slava77/cms-tkph2-ntuple/blob/184d2325147e6930030d3d1f780136bc2dd29ce6/doubletAnalysis.C#L3085
    float xn = 0.f, yn = 0.f;// , zn = 0;
    float shiftedRt;
    if (modulesInGPU.sides[lowerModuleIndex] != Center) // If barrel and not center it is tilted
    {
        // Shift the hits and calculate new xn, yn position
        float shiftedCoords[3];
        shiftStripHits(modulesInGPU, lowerModuleIndex, upperModuleIndex, lowerHitIndex, upperHitIndex, shiftedCoords,xLower,yLower,zLower,rtLower,xUpper,yUpper,zUpper,rtUpper);
        xn = shiftedCoords[0];
        yn = shiftedCoords[1];

        // Lower or the upper hit needs to be modified depending on which one was actually shifted
        if (modulesInGPU.moduleLayerType[lowerModuleIndex] == Pixel)
        {
            shiftedX = xn;
            shiftedY = yn;
            shiftedZ = zUpper;
            shiftedRt = sqrt(xn * xn + yn * yn);

            dPhi = deltaPhi(xLower,yLower,shiftedX, shiftedY); //function from Hit.cu
            noShiftedDphi = deltaPhi(xLower, yLower, xUpper, yUpper);
        }
        else
        {
            shiftedX = xn;
            shiftedY = yn;
            shiftedZ = zLower;
            shiftedRt = sqrt(xn * xn + yn * yn);
            dPhi = deltaPhi(shiftedX, shiftedY, xUpper, yUpper);
            noShiftedDphi = deltaPhi(xLower,yLower,xUpper,yUpper);

        }
    }
    else
    {
        dPhi = deltaPhi(xLower, yLower, xUpper, yUpper);
        noShiftedDphi = dPhi;
    }

    pass = pass & (fabsf(dPhi) < miniCut);
    if(not pass) return pass;

    // Cut #3: The dphi change going from lower Hit to upper Hit
    // Ref to original code: https://github.com/slava77/cms-tkph2-ntuple/blob/184d2325147e6930030d3d1f780136bc2dd29ce6/doubletAnalysis.C#L3076
    if (modulesInGPU.sides[lowerModuleIndex]!= Center)
    {
        // When it is tilted, use the new shifted positions
// TODO: This is somewhat of an mystery.... somewhat confused why this is the case
        if (modulesInGPU.moduleLayerType[lowerModuleIndex] != Pixel)
        {
            // dPhi Change should be calculated so that the upper hit has higher rt.
            // In principle, this kind of check rt_lower < rt_upper should not be necessary because the hit shifting should have taken care of this.
            // (i.e. the strip hit is shifted to be aligned in the line of sight from interaction point to pixel hit of PS module guaranteeing rt ordering)
            // But I still placed this check for safety. (TODO: After cheking explicitly if not needed remove later?)
            // setDeltaPhiChange(lowerHit.rt() < upperHitMod.rt() ? lowerHit.deltaPhiChange(upperHitMod) : upperHitMod.deltaPhiChange(lowerHit));


            dPhiChange = (rtLower < shiftedRt) ? deltaPhiChange(xLower, yLower, shiftedX, shiftedY) : deltaPhiChange(shiftedX, shiftedY, xLower, yLower); 
            noShiftedDphiChange = rtLower < rtUpper ? deltaPhiChange(xLower,yLower, xUpper, yUpper) : deltaPhiChange(xUpper, yUpper, xLower, yLower);
        }
        else
        {
            // dPhi Change should be calculated so that the upper hit has higher rt.
            // In principle, this kind of check rt_lower < rt_upper should not be necessary because the hit shifting should have taken care of this.
            // (i.e. the strip hit is shifted to be aligned in the line of sight from interaction point to pixel hit of PS module guaranteeing rt ordering)
            // But I still placed this check for safety. (TODO: After cheking explicitly if not needed remove later?)

            dPhiChange = (shiftedRt < rtUpper) ? deltaPhiChange(shiftedX, shiftedY, xUpper, yUpper) : deltaPhiChange(xUpper, yUpper, shiftedX, shiftedY);
            noShiftedDphiChange = rtLower < rtUpper ? deltaPhiChange(xLower,yLower, xUpper, yUpper) : deltaPhiChange(xUpper, yUpper, xLower, yLower);
        }
    }
    else
    {
        // When it is flat lying module, whichever is the lowerSide will always have rt lower
        dPhiChange = deltaPhiChange(xLower, yLower, xUpper, yUpper);
        noShiftedDphiChange = dPhiChange;
    }

    pass = pass & (fabsf(dPhiChange) < miniCut);

    return pass;
}

ALPAKA_FN_ACC bool SDL::runMiniDoubletDefaultAlgoEndcap(struct modules& modulesInGPU, uint16_t& lowerModuleIndex, uint16_t& upperModuleIndex, unsigned int lowerHitIndex, unsigned int upperHitIndex, float& drt, float& dPhi, float& dPhiChange, float& shiftedX, float& shiftedY, float& shiftedZ, float& noshiftedDz, float& noShiftedDphi, float& noShiftedDphichange,float xLower, float yLower, float zLower, float rtLower,float xUpper,float yUpper,float zUpper,float rtUpper)
{

    bool pass = true; 

    // There are series of cuts that applies to mini-doublet in a "endcap" region

    // Cut #1 : dz cut. The dz difference can't be larger than 1cm. (max separation is 4mm for modules in the endcap)
    // Ref to original code: https://github.com/slava77/cms-tkph2-ntuple/blob/184d2325147e6930030d3d1f780136bc2dd29ce6/doubletAnalysis.C#L3093
    // For PS module in case when it is tilted a different dz (after the strip hit shift) is calculated later.

    float dz = zLower - zUpper; // Not const since later it might change depending on the type of module

    const float dzCut = 1.f;

    pass = pass & (fabsf(dz) < dzCut);
    if(not pass) return pass;
    // Cut #2 : drt cut. The dz difference can't be larger than 1cm. (max separation is 4mm for modules in the endcap)
    // Ref to original code: https://github.com/slava77/cms-tkph2-ntuple/blob/184d2325147e6930030d3d1f780136bc2dd29ce6/doubletAnalysis.C#L3100
    const float drtCut = modulesInGPU.moduleType[lowerModuleIndex] == PS ? 2.f : 10.f;
    drt = rtLower - rtUpper;
    pass = pass & (fabs(drt) < drtCut);
    if(not pass) return pass;
    // The new scheme shifts strip hits to be "aligned" along the line of sight from interaction point to the pixel hit (if it is PS modules)
    float xn = 0, yn = 0, zn = 0;

    float shiftedCoords[3];
    shiftStripHits(modulesInGPU, /*hitsInGPU,*/ lowerModuleIndex, upperModuleIndex, lowerHitIndex, upperHitIndex, shiftedCoords,xLower,yLower,zLower,rtLower,xUpper,yUpper,zUpper,rtUpper);

    xn = shiftedCoords[0];
    yn = shiftedCoords[1];
    zn = shiftedCoords[2];

    if (modulesInGPU.moduleType[lowerModuleIndex] == PS)
    {
        // Appropriate lower or upper hit is modified after checking which one was actually shifted
        if (modulesInGPU.moduleLayerType[lowerModuleIndex] == Pixel)
        {
            // SDL::Hit upperHitMod(upperHit);
            // upperHitMod.setXYZ(xn, yn, upperHit.z());
            // setDeltaPhi(lowerHit.deltaPhi(upperHitMod));
            shiftedX = xn;
            shiftedY = yn;
            shiftedZ = zUpper;
            dPhi = deltaPhi(xLower, yLower, shiftedX, shiftedY);
            noShiftedDphi = deltaPhi(xLower, yLower, xUpper, yUpper);
        }
        else
        {
            // SDL::Hit lowerHitMod(lowerHit);
            // lowerHitMod.setXYZ(xn, yn, lowerHit.z());
            // setDeltaPhi(lowerHitMod.deltaPhi(upperHit));
            shiftedX = xn;
            shiftedY = yn;
            shiftedZ = zLower;
            dPhi = deltaPhi(shiftedX, shiftedY, xUpper, yUpper);
            noShiftedDphi = deltaPhi(xLower, yLower, xUpper, yUpper);
        }
    }
    else
    {
        shiftedX = xn;
        shiftedY = yn;
        shiftedZ = zUpper;
        dPhi = deltaPhi(xLower, yLower, xn, yn);
        noShiftedDphi = deltaPhi(xLower, yLower, xUpper, yUpper);
    }

    // dz needs to change if it is a PS module where the strip hits are shifted in order to properly account for the case when a tilted module falls under "endcap logic"
    // if it was an endcap it will have zero effect
    if (modulesInGPU.moduleType[lowerModuleIndex] == PS)
    {
        dz = modulesInGPU.moduleLayerType[lowerModuleIndex] == Pixel ? zLower - zn : zUpper - zn; 
    }

    float miniCut = 0;
    miniCut = modulesInGPU.moduleLayerType[lowerModuleIndex] == Pixel ?  dPhiThreshold(rtLower, modulesInGPU, lowerModuleIndex,dPhi, dz) :  dPhiThreshold(rtUpper, modulesInGPU, lowerModuleIndex, dPhi, dz);

    pass = pass & (fabsf(dPhi) < miniCut);
    if(not pass) return pass;

    // Cut #4: Another cut on the dphi after some modification
    // Ref to original code: https://github.com/slava77/cms-tkph2-ntuple/blob/184d2325147e6930030d3d1f780136bc2dd29ce6/doubletAnalysis.C#L3119-L3124

    
    float dzFrac = fabsf(dz) / fabsf(zLower);
    dPhiChange = dPhi / dzFrac * (1.f + dzFrac);
    noShiftedDphichange = noShiftedDphi / dzFrac * (1.f + dzFrac);
    pass = pass & (fabsf(dPhiChange) < miniCut);

    return pass;
}

ALPAKA_FN_ACC bool SDL::runMiniDoubletDefaultAlgo(struct modules& modulesInGPU, /*struct hits& hitsInGPU,*/ uint16_t& lowerModuleIndex, uint16_t& upperModuleIndex, unsigned int lowerHitIndex, unsigned int upperHitIndex, float& dz, float& dPhi, float& dPhiChange, float& shiftedX, float& shiftedY, float& shiftedZ, float& noShiftedDz, float& noShiftedDphi, float& noShiftedDphiChange, float xLower, float yLower, float zLower, float rtLower,float xUpper,float yUpper,float zUpper,float rtUpper)
{
   //bool pass;
   if(modulesInGPU.subdets[lowerModuleIndex] == Barrel)
   {
        return runMiniDoubletDefaultAlgoBarrel(modulesInGPU, /*hitsInGPU,*/ lowerModuleIndex, upperModuleIndex, lowerHitIndex, upperHitIndex, dz, dPhi, dPhiChange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange,xLower,yLower,zLower,rtLower, xUpper,yUpper,zUpper,rtUpper);
   } 
   else
   {
       return runMiniDoubletDefaultAlgoEndcap(modulesInGPU, /*hitsInGPU,*/ lowerModuleIndex, upperModuleIndex, lowerHitIndex, upperHitIndex, dz, dPhi, dPhiChange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange,xLower,yLower,zLower,rtLower, xUpper,yUpper,zUpper,rtUpper);

   }
   //return pass;
}

ALPAKA_FN_ACC inline float SDL::dPhiThreshold(/*struct hits& hitsInGPU,*/float rt, struct modules& modulesInGPU, /*unsigned int hitIndex,*/ uint16_t& moduleIndex, float dPhi, float dz)
{
    // =================================================================
    // Various constants
    // =================================================================
    //mean of the horizontal layer position in y; treat this as R below

    // =================================================================
    // Computing some components that make up the cut threshold
    // =================================================================

    unsigned int iL = modulesInGPU.layers[moduleIndex] - 1;
    const float miniSlope = asinf(min(rt * k2Rinv1GeVf / ptCut, sinAlphaMax));
    const float rLayNominal = ((modulesInGPU.subdets[moduleIndex]== Barrel) ? miniRminMeanBarrel[iL] : miniRminMeanEndcap[iL]);
    const float miniPVoff = 0.1f / rLayNominal;
    const float miniMuls = ((modulesInGPU.subdets[moduleIndex] == Barrel) ? miniMulsPtScaleBarrel[iL] * 3.f / ptCut : miniMulsPtScaleEndcap[iL] * 3.f / ptCut);
    const bool isTilted = modulesInGPU.subdets[moduleIndex] == Barrel and modulesInGPU.sides[moduleIndex] != Center;
    //the lower module is sent in irrespective of its layer type. We need to fetch the drdz properly

    float drdz;
    if(isTilted)
    {
        if(modulesInGPU.moduleType[moduleIndex] == PS and modulesInGPU.moduleLayerType[moduleIndex] == Strip)
        {
            drdz = modulesInGPU.drdzs[moduleIndex];
        }
        else
        {
            drdz = modulesInGPU.drdzs[modulesInGPU.partnerModuleIndices[moduleIndex]];
        }
    }
    else
    {
        drdz = 0;
    }
    const float miniTilt = ((isTilted) ? 0.5f * pixelPSZpitch * drdz / sqrt(1.f + drdz * drdz) / moduleGapSize(modulesInGPU,moduleIndex) : 0);

    // Compute luminous region requirement for endcap
    const float miniLum = fabsf(dPhi * deltaZLum/dz); // Balaji's new error
    // const float miniLum = abs(deltaZLum / lowerHit.z()); // Old error


    // =================================================================
    // Return the threshold value
    // =================================================================
    // Following condition is met if the module is central and flatly lying
    if (modulesInGPU.subdets[moduleIndex] == Barrel and modulesInGPU.sides[moduleIndex] == Center)
    {
        return miniSlope + sqrt(miniMuls * miniMuls + miniPVoff * miniPVoff);
    }
    // Following condition is met if the module is central and tilted
    else if (modulesInGPU.subdets[moduleIndex] == Barrel and modulesInGPU.sides[moduleIndex] != Center) //all types of tilted modules
    {
        return miniSlope + sqrt(miniMuls * miniMuls + miniPVoff * miniPVoff + miniTilt * miniTilt * miniSlope * miniSlope);
    }
    // If not barrel, it is Endcap
    else
    {
        return miniSlope + sqrt(miniMuls * miniMuls + miniPVoff * miniPVoff + miniLum * miniLum);
    }
}

ALPAKA_FN_ACC inline float SDL::isTighterTiltedModules(struct modules& modulesInGPU, uint16_t& moduleIndex)
{
    // The "tighter" tilted modules are the subset of tilted modules that have smaller spacing
    // This is the same as what was previously considered as"isNormalTiltedModules"
    // See Figure 9.1 of https://cds.cern.ch/record/2272264/files/CMS-TDR-014.pdf
    short subdet = modulesInGPU.subdets[moduleIndex];
    short layer = modulesInGPU.layers[moduleIndex];
    short side = modulesInGPU.sides[moduleIndex];
    short rod = modulesInGPU.rods[moduleIndex];

    if (
           (subdet == Barrel and side != Center and layer== 3)
           or (subdet == Barrel and side == NegZ and layer == 2 and rod > 5)
           or (subdet == Barrel and side == PosZ and layer == 2 and rod < 8)
           or (subdet == Barrel and side == NegZ and layer == 1 and rod > 9)
           or (subdet == Barrel and side == PosZ and layer == 1 and rod < 4)
       )
        return true;
    else
        return false;

}

ALPAKA_FN_ACC inline float SDL::moduleGapSize(struct modules& modulesInGPU, uint16_t& moduleIndex)
{
    float miniDeltaTilted[3] = {0.26f, 0.26f, 0.26f};
    float miniDeltaFlat[6] ={0.26f, 0.16f, 0.16f, 0.18f, 0.18f, 0.18f};
    float miniDeltaLooseTilted[3] = {0.4f,0.4f,0.4f};
    float miniDeltaEndcap[5][15];

    for (size_t i = 0; i < 5; i++)
    {
        for (size_t j = 0; j < 15; j++)
        {
            if (i == 0 || i == 1)
            {
                if (j < 10)
                {
                    miniDeltaEndcap[i][j] = 0.4f;
                }
                else
                {
                    miniDeltaEndcap[i][j] = 0.18f;
                }
            }
            else if (i == 2 || i == 3)
            {
                if (j < 8)
                {
                    miniDeltaEndcap[i][j] = 0.4f;
                }
                else
                {
                    miniDeltaEndcap[i][j]  = 0.18f;
                }
            }
            else
            {
                if (j < 9)
                {
                    miniDeltaEndcap[i][j] = 0.4f;
                }
                else
                {
                    miniDeltaEndcap[i][j] = 0.18f;
                }
            }
        }
    }


    unsigned int iL = modulesInGPU.layers[moduleIndex]-1;
    unsigned int iR = modulesInGPU.rings[moduleIndex] - 1;
    short subdet = modulesInGPU.subdets[moduleIndex];
    short side = modulesInGPU.sides[moduleIndex];

    float moduleSeparation = 0;

    if (subdet == Barrel and side == Center)
    {
        moduleSeparation = miniDeltaFlat[iL];
    }
    else if (isTighterTiltedModules(modulesInGPU, moduleIndex))
    {
        moduleSeparation = miniDeltaTilted[iL];
    }
    else if (subdet == Endcap)
    {
        moduleSeparation = miniDeltaEndcap[iL][iR];
    }
    else //Loose tilted modules
    {
        moduleSeparation = miniDeltaLooseTilted[iL];
    }

    return moduleSeparation;
}

ALPAKA_FN_ACC void SDL::shiftStripHits(struct modules& modulesInGPU, /*struct hits& hitsInGPU,*/ uint16_t& lowerModuleIndex, uint16_t& upperModuleIndex, unsigned int lowerHitIndex, unsigned int upperHitIndex, float* shiftedCoords, float xLower, float yLower, float zLower, float rtLower,float xUpper,float yUpper,float zUpper,float rtUpper)
{

    // This is the strip shift scheme that is explained in http://uaf-10.t2.ucsd.edu/~phchang/talks/PhilipChang20190607_SDL_Update.pdf (see backup slides)
    // The main feature of this shifting is that the strip hits are shifted to be "aligned" in the line of sight from interaction point to the the pixel hit.
    // (since pixel hit is well defined in 3-d)
    // The strip hit is shifted along the strip detector to be placed in a guessed position where we think they would have actually crossed
    // The size of the radial direction shift due to module separation gap is computed in "radial" size, while the shift is done along the actual strip orientation
    // This means that there may be very very subtle edge effects coming from whether the strip hit is center of the module or the at the edge of the module
    // But this should be relatively minor effect

    // dependent variables for this if statement
    // lowerModule
    // lowerHit
    // upperHit
    // SDL::endcapGeometry
    // SDL::tiltedGeometry

    // Some variables relevant to the function
    float xp; // pixel x (pixel hit x)
    float yp; // pixel y (pixel hit y)
    float zp; // pixel y (pixel hit y)
    float rtp; // pixel y (pixel hit y)
    float xa; // "anchor" x (the anchor position on the strip module plane from pixel hit)
    float ya; // "anchor" y (the anchor position on the strip module plane from pixel hit)
    float xo; // old x (before the strip hit is moved up or down)
    float yo; // old y (before the strip hit is moved up or down)
    float xn; // new x (after the strip hit is moved up or down)
    float yn; // new y (after the strip hit is moved up or down)
    float abszn; // new z in absolute value
    float zn; // new z with the sign (+/-) accounted
    float angleA; // in r-z plane the theta of the pixel hit in polar coordinate is the angleA
    float angleB; // this is the angle of tilted module in r-z plane ("drdz"), for endcap this is 90 degrees
    bool isEndcap; // If endcap, drdz = infinity
    //unsigned int pixelHitIndex; // Pointer to the pixel hit
    //unsigned int stripHitIndex; // Pointer to the strip hit
    float moduleSeparation;
    float drprime; // The radial shift size in x-y plane projection
    float drprime_x; // x-component of drprime
    float drprime_y; // y-component of drprime
    float& slope = modulesInGPU.slopes[lowerModuleIndex]; // The slope of the possible strip hits for a given module in x-y plane
    float absArctanSlope;
    float angleM; // the angle M is the angle of rotation of the module in x-y plane if the possible strip hits are along the x-axis, then angleM = 0, and if the possible strip hits are along y-axis angleM = 90 degrees
    float absdzprime; // The distance between the two points after shifting
    float& drdz_ = modulesInGPU.drdzs[lowerModuleIndex];
    // Assign hit pointers based on their hit type
    if (modulesInGPU.moduleType[lowerModuleIndex] == PS)
    {
// TODO: This is somewhat of an mystery.... somewhat confused why this is the case
        if (modulesInGPU.subdets[lowerModuleIndex] == Barrel ? modulesInGPU.moduleLayerType[lowerModuleIndex] != Pixel : modulesInGPU.moduleLayerType[lowerModuleIndex] == Pixel)
        {
            //old to delete
       //     pixelHitIndex = lowerHitIndex;
       //     stripHitIndex = upperHitIndex;
    
            //new to replace
            xo =xUpper;
            yo =yUpper;
            xp =xLower;
            yp =yLower;
            zp =zLower;
            rtp =rtLower;
            xp =xLower;
            yp =yLower;
            zp =zLower;
            rtp =rtLower;
        }
        else
        {
     //       pixelHitIndex = upperHitIndex;
     //       stripHitIndex = lowerHitIndex;
            //new to replace
            xo = xLower;
            yo = yLower;
            xp = xUpper;
            yp = yUpper;
            zp = zUpper;
            rtp=rtUpper;
            xp = xUpper;
            yp = yUpper;
            zp = zUpper;
            rtp=rtUpper;
        }
    }
    else // if (lowerModule.moduleType() == SDL::Module::TwoS) // If it is a TwoS module (if this is called likely an endcap module) then anchor the inner hit and shift the outer hit
    {
        //pixelHitIndex = lowerHitIndex; // Even though in this case the "pixelHitPtr" is really just a strip hit, we pretend it is the anchoring pixel hit
        //stripHitIndex = upperHitIndex;
            xo =xUpper;
            yo =yUpper;
            xp =xLower;
            yp =yLower;
            zp =zLower;
            rtp =rtLower;
            xp =xLower;
            yp =yLower;
            zp =zLower;
            rtp =rtLower;
    }

    // If it is endcap some of the math gets simplified (and also computers don't like infinities)
    isEndcap = modulesInGPU.subdets[lowerModuleIndex]== Endcap;

    // NOTE: TODO: Keep in mind that the sin(atan) function can be simplifed to something like x / sqrt(1 + x^2) and similar for cos
    // I am not sure how slow sin, atan, cos, functions are in c++. If x / sqrt(1 + x^2) are faster change this later to reduce arithmetic computation time

    // The pixel hit is used to compute the angleA which is the theta in polar coordinate
    // angleA = atanf(pixelHitPtr->rt() / pixelHitPtr->z() + (pixelHitPtr->z() < 0 ? M_PI : 0)); // Shift by pi if the z is negative so that the value of the angleA stays between 0 to pi and not -pi/2 to pi/2

    angleA = fabsf(atanf(rtp / zp));
    angleB = ((isEndcap) ? float(M_PI) / 2.f : atanf(drdz_)); // The tilt module on the postive z-axis has negative drdz slope in r-z plane and vice versa


    moduleSeparation = moduleGapSize(modulesInGPU, lowerModuleIndex);

    // Sign flips if the pixel is later layer
    if (modulesInGPU.moduleType[lowerModuleIndex] == PS and modulesInGPU.moduleLayerType[lowerModuleIndex] != Pixel)
    {
        moduleSeparation *= -1;
    }

    drprime = (moduleSeparation / sinf(angleA + angleB)) * sinf(angleA);
    
    // Compute arctan of the slope and take care of the slope = infinity case
    absArctanSlope = ((slope != SDL_INF) ? fabs(atanf(slope)) : float(M_PI) / 2.f); // Since C++ can't represent infinity, SDL_INF = 123456789 was used to represent infinity in the data table


    // Depending on which quadrant the pixel hit lies, we define the angleM by shifting them slightly differently
    if (xp > 0 and yp > 0)
    {
        angleM = absArctanSlope;
    }
    else if (xp > 0 and yp < 0)
    {
        angleM = float(M_PI) - absArctanSlope;
    }
    else if (xp < 0 and yp < 0)
    {
        angleM = float(M_PI) + absArctanSlope;
    }
    else // if (xp < 0 and yp > 0)
    {
        angleM = 2.f * float(M_PI) - absArctanSlope;
    }

    // Then since the angleM sign is taken care of properly
    drprime_x = drprime * sinf(angleM);
    drprime_y = drprime * cosf(angleM);

    // The new anchor position is
    xa = xp + drprime_x;
    ya = yp + drprime_y;

    // The original strip hit position
    //xo = hitsInGPU.xs[stripHitIndex];
    //yo = hitsInGPU.ys[stripHitIndex];

    // Compute the new strip hit position (if the slope vaule is in special condition take care of the exceptions)
    if (slope == SDL_INF) // Special value designated for tilted module when the slope is exactly infinity (module lying along y-axis)
    {
        xn = xa; // New x point is simply where the anchor is
        yn = yo; // No shift in y
    }
    else if (slope == 0)
    {
        xn = xo; // New x point is simply where the anchor is
        yn = ya; // No shift in y
    }
    else
    {
        xn = (slope * xa + (1.f / slope) * xo - ya + yo) / (slope + (1.f / slope)); // new xn
        yn = (xn - xa) * slope + ya; // new yn
    }

    // Computing new Z position
    absdzprime = fabsf(moduleSeparation / sinf(angleA + angleB) * cosf(angleA)); // module separation sign is for shifting in radial direction for z-axis direction take care of the sign later

    // Depending on which one as closer to the interactin point compute the new z wrt to the pixel properly
    if (modulesInGPU.moduleLayerType[lowerModuleIndex] == Pixel)
    {
        abszn = fabsf(zp) + absdzprime;
    }
    else
    {
        abszn = fabsf(zp) - absdzprime;
    }

    zn = abszn * ((zp > 0) ? 1 : -1); // Apply the sign of the zn


    shiftedCoords[0] = xn;
    shiftedCoords[1] = yn;
    shiftedCoords[2] = zn;
}

SDL::miniDoublets::miniDoublets()
{
    anchorHitIndices = nullptr;
    outerHitIndices = nullptr;
    moduleIndices = nullptr;
    nMDs = nullptr;
    totOccupancyMDs = nullptr;
    dphichanges = nullptr;

    dzs = nullptr;
    dphis = nullptr;

    shiftedXs = nullptr;
    shiftedYs = nullptr;
    shiftedZs = nullptr;
    noShiftedDzs = nullptr;
    noShiftedDphis = nullptr;
    noShiftedDphiChanges = nullptr;
    
    anchorX = nullptr;
    anchorY = nullptr;
    anchorZ = nullptr;
    anchorRt = nullptr;
    anchorPhi = nullptr;
    anchorEta = nullptr;
    anchorHighEdgeX = nullptr;
    anchorHighEdgeY = nullptr;
    anchorLowEdgeX = nullptr;
    anchorLowEdgeY = nullptr;
    outerX = nullptr;
    outerY = nullptr;
    outerZ = nullptr;
    outerRt = nullptr;
    outerPhi = nullptr;
    outerEta = nullptr;
    outerHighEdgeX = nullptr;
    outerHighEdgeY = nullptr;
    outerLowEdgeX = nullptr;
    outerLowEdgeY = nullptr;
}

SDL::miniDoublets::~miniDoublets()
{
}

void SDL::miniDoublets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,anchorHitIndices);
    cms::cuda::free_device(dev, moduleIndices);
    cms::cuda::free_device(dev,dphichanges);
    cms::cuda::free_device(dev,nMDs);
    cms::cuda::free_device(dev,totOccupancyMDs);
    cms::cuda::free_device(dev, anchorX);
    cms::cuda::free_device(dev, anchorHighEdgeX);
    cms::cuda::free_device(dev, outerX);
    cms::cuda::free_device(dev, outerHighEdgeX);
    cms::cuda::free_device(dev, nMemoryLocations);
}


void SDL::miniDoublets::freeMemory(hipStream_t stream)
{
    hipFree(anchorHitIndices);
    hipFree(moduleIndices);
    hipFree(nMDs);
    hipFree(totOccupancyMDs);
    hipFree(dphichanges);
    hipFree(anchorX);
    hipFree(anchorHighEdgeX);
    hipFree(outerX);
    hipFree(outerHighEdgeX);
    hipFree(nMemoryLocations);
}

void SDL::printMD(struct miniDoublets& mdsInGPU, struct hits& hitsInGPU, SDL::modules& modulesInGPU, unsigned int mdIndex)
{
    std::cout<<std::endl;
    std::cout << "dz " << mdsInGPU.dzs[mdIndex] << std::endl;
    std::cout << "dphi " << mdsInGPU.dphis[mdIndex] << std::endl;
    std::cout << "dphinoshift " << mdsInGPU.noShiftedDphis[mdIndex] << std::endl;
    std::cout << "dphichange " << mdsInGPU.dphichanges[mdIndex] << std::endl;
    std::cout << "dphichangenoshift " << mdsInGPU.noShiftedDphiChanges[mdIndex] << std::endl;
    std::cout << std::endl;
    std::cout << "Anchor Hit " << std::endl;
    std::cout << "------------------------------" << std::endl;
    unsigned int lowerHitIndex = mdsInGPU.anchorHitIndices[mdIndex];
    unsigned int upperHitIndex = mdsInGPU.outerHitIndices[mdIndex];
    {
        IndentingOStreambuf indent(std::cout);
        printHit(hitsInGPU, modulesInGPU, lowerHitIndex);
    }
    std::cout << "Non-anchor Hit " << std::endl;
    std::cout << "------------------------------" << std::endl;
    {
        IndentingOStreambuf indent(std::cout);
        printHit(hitsInGPU, modulesInGPU, upperHitIndex);
    }
}

__global__ void SDL::createMiniDoubletsInGPUv2(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::objectRanges& rangesInGPU)
{
    int blockxSize = blockDim.x*gridDim.x;
    int blockySize = blockDim.y*gridDim.y;
    //int blockzSize = blockDim.z*gridDim.z;
    for(uint16_t lowerModuleIndex = blockIdx.y * blockDim.y + threadIdx.y; lowerModuleIndex< (*modulesInGPU.nLowerModules); lowerModuleIndex += blockySize)
    {
        uint16_t upperModuleIndex = modulesInGPU.partnerModuleIndices[lowerModuleIndex];
        int nLowerHits = hitsInGPU.hitRangesnLower[lowerModuleIndex];
        int nUpperHits = hitsInGPU.hitRangesnUpper[lowerModuleIndex];
        if(hitsInGPU.hitRangesLower[lowerModuleIndex] == -1) continue;
        const int maxHits = max(nUpperHits,nLowerHits);
        unsigned int upHitArrayIndex = hitsInGPU.hitRangesUpper[lowerModuleIndex];
        unsigned int loHitArrayIndex = hitsInGPU.hitRangesLower[lowerModuleIndex];
        int limit = nUpperHits*nLowerHits;
        for(int hitIndex = blockIdx.x * blockDim.x + threadIdx.x; hitIndex< limit; hitIndex += blockxSize)
        {
            int lowerHitIndex =  hitIndex / nUpperHits;
            int upperHitIndex =  hitIndex % nUpperHits;
            if(upperHitIndex >= nUpperHits) continue;
            if(lowerHitIndex >= nLowerHits) continue;
            unsigned int lowerHitArrayIndex = loHitArrayIndex + lowerHitIndex;
            float xLower = hitsInGPU.xs[lowerHitArrayIndex];
            float yLower = hitsInGPU.ys[lowerHitArrayIndex];
            float zLower = hitsInGPU.zs[lowerHitArrayIndex];
            float rtLower = hitsInGPU.rts[lowerHitArrayIndex];
            unsigned int upperHitArrayIndex = upHitArrayIndex+upperHitIndex;
            float xUpper = hitsInGPU.xs[upperHitArrayIndex];
            float yUpper = hitsInGPU.ys[upperHitArrayIndex];
            float zUpper = hitsInGPU.zs[upperHitArrayIndex];
            float rtUpper = hitsInGPU.rts[upperHitArrayIndex];

            float dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;

            //float dzCut, drtCut;//, miniCut;
            bool success = runMiniDoubletDefaultAlgo(modulesInGPU, lowerModuleIndex, upperModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, xLower,yLower,zLower,rtLower,xUpper,yUpper,zUpper,rtUpper);
if(success)
            {
                unsigned int totOccupancyMDs = atomicAdd(&mdsInGPU.totOccupancyMDs[lowerModuleIndex],1);
                if(totOccupancyMDs >= (rangesInGPU.miniDoubletModuleOccupancy[lowerModuleIndex]))
                {
#ifdef Warnings
                    printf("Mini-doublet excess alert! Module index =  %d\n",lowerModuleIndex);
#endif
                }
                else
                {
                    unsigned int mdModuleIndex = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);
                    unsigned int mdIndex = rangesInGPU.miniDoubletModuleIndices[lowerModuleIndex] + mdModuleIndex;

//#ifdef CUT_VALUE_DEBUG
//                    addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz,drt, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, dzCut, drtCut, miniCut, mdIndex);
//#else
                    addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIndex);
//#endif
                }

            }
        }
    }
}
__global__ void SDL::addMiniDoubletRangesToEventExplicit(struct modules& modulesInGPU, struct miniDoublets& mdsInGPU, struct objectRanges& rangesInGPU,struct hits& hitsInGPU)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int np = gridDim.x * blockDim.x;
    for(uint16_t i = gid; i < *modulesInGPU.nLowerModules; i+= np)
    {
        if(mdsInGPU.nMDs[i] == 0 or hitsInGPU.hitRanges[i * 2] == -1)
        {
            rangesInGPU.mdRanges[i * 2] = -1;
            rangesInGPU.mdRanges[i * 2 + 1] = -1;
        }
        else
        {
            rangesInGPU.mdRanges[i * 2] = rangesInGPU.miniDoubletModuleIndices[i] ;
            rangesInGPU.mdRanges[i * 2 + 1] = rangesInGPU.miniDoubletModuleIndices[i] + mdsInGPU.nMDs[i] - 1;
        }
    }
}
