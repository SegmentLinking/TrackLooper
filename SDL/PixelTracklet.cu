#include "hip/hip_runtime.h"
# include "PixelTracklet.cuh"

//#ifdef CACHE_ALLOC
#include "allocate.h"
//#endif

void SDL::createPixelTrackletsInExplicitMemory(struct pixelTracklets& pixelTrackletsInGPU, unsigned int maxPixelTracklets,hipStream_t stream)
{
#ifdef CACHE_ALLOC
//    hipStream_t stream = 0;
    int dev;
    hipGetDevice(&dev);

    pixelTrackletsInGPU.segmentIndices = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelTracklets * sizeof(unsigned int) * 2,stream);
    pixelTrackletsInGPU.lowerModuleIndices = (unsigned int*)cms::cuda::allocate_device(dev, maxPixelTracklets * sizeof(unsigned int) * 2,stream);//split up to avoid runtime error of exceeding max byte allocation at a time
    pixelTrackletsInGPU.nPixelTracklets = (unsigned int*)cms::cuda::allocate_device(dev, sizeof(unsigned int),stream);
    pixelTrackletsInGPU.zOut = (float*)cms::cuda::allocate_device(dev, maxPixelTracklets * sizeof(float) * 4,stream);
    pixelTrackletsInGPU.betaIn = (float*)cms::cuda::allocate_device(dev, maxPixelTracklets * sizeof(float) * 3,stream);

#else
    hipMalloc(&pixelTrackletsInGPU.segmentIndices, 2 * maxPixelTracklets * sizeof(unsigned int));
    hipMalloc(&pixelTrackletsInGPU.lowerModuleIndices, 2 * maxPixelTracklets * sizeof(unsigned int));
    hipMalloc(&pixelTrackletsInGPU.nPixelTracklets, sizeof(unsigned int));
    hipMalloc(&pixelTrackletsInGPU.zOut, maxPixelTracklets *4* sizeof(float));
    hipMalloc(&pixelTrackletsInGPU.betaIn, maxPixelTracklets *3* sizeof(float));
#endif
    pixelTrackletsInGPU.rtOut = pixelTrackletsInGPU.zOut + maxPixelTracklets;
    pixelTrackletsInGPU.deltaPhiPos = pixelTrackletsInGPU.zOut + maxPixelTracklets * 2;
    pixelTrackletsInGPU.deltaPhi = pixelTrackletsInGPU.zOut + maxPixelTracklets * 3;
    pixelTrackletsInGPU.betaOut = pixelTrackletsInGPU.betaIn + maxPixelTracklets;
    pixelTrackletsInGPU.pt_beta = pixelTrackletsInGPU.betaIn + maxPixelTracklets * 2;

    hipMemsetAsync(pixelTrackletsInGPU.nPixelTracklets, 0, sizeof(unsigned int),stream);
}

#ifdef CUT_VALUE_DEBUG
ALPAKA_FN_ACC void SDL::addPixelTrackletToMemory(struct pixelTracklets& pixelTrackletsInGPU, unsigned int innerSegmentIndex, unsigned int outerSegmentIndex, unsigned int innerInnerLowerModuleIndex, unsigned int innerOuterLowerModuleIndex, unsigned int outerInnerLowerModuleIndex, unsigned int outerOuterLowerModuleIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& deltaPhi, float& betaIn, float& betaOut, float pt_beta, float& zLo, float& zHi, float& rtLo, float& rtHi, float& zLoPointed, float&
        zHiPointed, float& sdlCut, float& betaInCut, float& betaOutCut, float& deltaBetaCut, float& kZ, unsigned int pixelTrackletIndex)
#else
ALPAKA_FN_ACC void SDL::addPixelTrackletToMemory(struct pixelTracklets& pixelTrackletsInGPU, unsigned int innerSegmentIndex, unsigned int outerSegmentIndex, unsigned int innerInnerLowerModuleIndex, unsigned int innerOuterLowerModuleIndex, unsigned int outerInnerLowerModuleIndex, unsigned int outerOuterLowerModuleIndex, float& zOut, float& rtOut, float& deltaPhiPos, float& deltaPhi, float& betaIn, float& betaOut, float pt_beta, unsigned int pixelTrackletIndex)
#endif
{
    pixelTrackletsInGPU.segmentIndices[2 * pixelTrackletIndex] = innerSegmentIndex;
    pixelTrackletsInGPU.segmentIndices[2 * pixelTrackletIndex + 1] = outerSegmentIndex;
    pixelTrackletsInGPU.lowerModuleIndices[2 * pixelTrackletIndex] = outerInnerLowerModuleIndex;
    pixelTrackletsInGPU.lowerModuleIndices[2 * pixelTrackletIndex + 1] = outerOuterLowerModuleIndex;

    pixelTrackletsInGPU.zOut[pixelTrackletIndex] = zOut;
    pixelTrackletsInGPU.rtOut[pixelTrackletIndex] = rtOut;
    pixelTrackletsInGPU.deltaPhiPos[pixelTrackletIndex] = deltaPhiPos;
    pixelTrackletsInGPU.deltaPhi[pixelTrackletIndex] = deltaPhi;

    pixelTrackletsInGPU.betaIn[pixelTrackletIndex] = betaIn;
    pixelTrackletsInGPU.betaOut[pixelTrackletIndex] = betaOut;
    pixelTrackletsInGPU.pt_beta[pixelTrackletIndex] = pt_beta;

#ifdef CUT_VALUE_DEBUG
    pixelTrackletsInGPU.zLo[pixelTrackletIndex] = zLo;
    pixelTrackletsInGPU.zHi[pixelTrackletIndex] = zHi;
    pixelTrackletsInGPU.rtLo[pixelTrackletIndex] = rtLo;
    pixelTrackletsInGPU.rtHi[pixelTrackletIndex] = rtHi;
    pixelTrackletsInGPU.zLoPointed[pixelTrackletIndex] = zLoPointed;
    pixelTrackletsInGPU.zHiPointed[pixelTrackletIndex] = zHiPointed;
    pixelTrackletsInGPU.sdlCut[pixelTrackletIndex] = sdlCut;
    pixelTrackletsInGPU.betaInCut[pixelTrackletIndex] = betaInCut;
    pixelTrackletsInGPU.betaOutCut[pixelTrackletIndex] = betaOutCut;
    pixelTrackletsInGPU.deltaBetaCut[pixelTrackletIndex] = deltaBetaCut;
    pixelTrackletsInGPU.kZ[pixelTrackletIndex] = kZ;
#endif

}

void SDL::pixelTracklets::freeMemoryCache()
{
    int dev;
    hipGetDevice(&dev);
    cms::cuda::free_device(dev,segmentIndices);
    cms::cuda::free_device(dev,lowerModuleIndices);
    cms::cuda::free_device(dev,zOut);
    cms::cuda::free_device(dev,betaIn);
    cms::cuda::free_device(dev,nPixelTracklets);
}

void SDL::pixelTracklets::freeMemory()
{
    hipFree(segmentIndices);
    hipFree(lowerModuleIndices);
    hipFree(nPixelTracklets);
    hipFree(zOut);
    hipFree(betaIn);
#ifdef CUT_VALUE_DEBUG
    hipFree(zLo);
    hipFree(zHi);
    hipFree(rtLo);
    hipFree(rtHi);
    hipFree(zLoPointed);
    hipFree(zHiPointed);
    hipFree(sdlCut);
    hipFree(betaInCut);
    hipFree(betaOutCut);
    hipFree(deltaBetaCut);
    hipFree(kZ);
#endif
}

SDL::pixelTracklets::pixelTracklets()
{
    segmentIndices = nullptr;
    lowerModuleIndices = nullptr;
    nPixelTracklets = nullptr;
    zOut = nullptr;
    rtOut = nullptr;

    deltaPhiPos = nullptr;
    deltaPhi = nullptr;
    betaIn = nullptr;
    betaOut = nullptr;
    pt_beta = nullptr;
#ifdef CUT_VALUE_DEBUG
    zLo = nullptr;
    zHi = nullptr;
    rtLo = nullptr;
    rtHi = nullptr;
    zLoPointed = nullptr;
    zHiPointed = nullptr;
    sdlCut = nullptr;
    betaInCut = nullptr;
    betaOutCut = nullptr;
    deltaBetaCut = nullptr;
    kZ = nullptr;
#endif

}

SDL::pixelTracklets::~pixelTracklets()
{

}

